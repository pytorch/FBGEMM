#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include "codegen/embedding_forward_template_helpers.cuh"

using namespace at;
using namespace fbgemm_gpu;

{% if not dense %}
constexpr int32_t kCacheLocationMissing = -1;
{% endif %}
enum {
  DEVICE = 0,
  MANAGED = 1,
  MANAGED_CACHING = 2,
};

constexpr size_t kForwardMaxThreads = 512;

// TODO: optimization to use multiple warps per row.
template <typename emb_t, typename cache_t, size_t kMaxVecsPerThread>
__global__
__launch_bounds__(kForwardMaxThreads) void {{ "dense" if dense else "split" }}_embedding_codegen_grad_indice_weights_kernel(
    // [\sum_t E_t x D_t]
    const PackedTensorAccessor32<acc_type<cache_t, true>, 2, RestrictPtrTraits>
        grad_output,
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> dev_weights,
    {% if not dense %}
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> uvm_weights,
    PackedTensorAccessor64<cache_t, 2, RestrictPtrTraits> lxu_cache_weights,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> weights_offsets,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> D_offsets,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        indices, // [N = \sum_{b,t} L_{b,t} total indices, i.e. flattened
                 // [B][T][L]
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        offsets, // [B x T + 1]
    {% if not dense %}
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        lxu_cache_locations,
    {% endif %}
    PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        feature_requires_grad, // [T],
    PackedTensorAccessor32<acc_type<cache_t, true>, 1, RestrictPtrTraits>
        grad_indice_weights
    ) {
    int32_t B = grad_output.size(0);
    int32_t T = D_offsets.size(0) - 1;
    int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
    int32_t t = b_t / B;
    int32_t b = b_t % B;

    if (b_t >= B * T) {
        return;
    }
    int64_t weights_offset = weights_offsets[t];
    int32_t D_start = D_offsets[t];
    int32_t D_end = D_offsets[t + 1];
    int32_t D = D_end - D_start;
    int64_t indices_start = offsets[t * B + b];
    int64_t indices_end = offsets[t * B + b + 1];
    int32_t L = indices_end - indices_start;
    if (feature_requires_grad.size(0) > 0 && !feature_requires_grad[t]) {
        // If the table does not require gradient computation, we set the gradient to zero.
        for (int32_t l_start = 0; l_start < L; l_start += kWarpSize) {
            int32_t l = l_start + threadIdx.x;
            if (l < L) {
                grad_indice_weights[indices_start + l] = 0.0;
            }
        }
        return;
    }

    const emb_t* __restrict__ weights;
    {% if not dense %}
    const auto placement = weights_placements[t];
    if (placement == DEVICE) {
        weights = &dev_weights[weights_offset];
    } else {
        weights = &uvm_weights[weights_offset];
    }
    {% else %}
    weights = &dev_weights[weights_offset];
    {% endif %}


    Vec4T<acc_type<cache_t, true>> grad_out[kMaxVecsPerThread];
    #pragma unroll kMaxVecsPerThread
    for (int32_t i = 0;
        i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
        ++i) {
        int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
        Vec4T<acc_type<cache_t, true>> go((&grad_output[b][0]) + D_start + d);
        grad_out[i] = go;
    }

    for (int32_t l_start = 0; l_start < L; l_start += kWarpSize) {
        int32_t l = l_start + threadIdx.x;
        int64_t idx = l < L ? indices[indices_start + l] : 0;
        {% if not dense %}
        int32_t cache_idx = (placement == MANAGED_CACHING && l < L) ? lxu_cache_locations[indices_start + l] : 0;
        {% endif %}
        for (auto j = 0; j < kWarpSize && l_start + j < L; ++j) {
            int64_t idx_j = __shfl_sync(0xFFFFFFFF, idx, j);
            {% if not dense %}
            int32_t cache_idx_j = __shfl_sync(0xFFFFFFFF, cache_idx, j);
            {% endif %}
            acc_type<cache_t, true> grad_indice_weight = 0.0;

        #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                ++i) {
                int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
                {% if not dense %}
                if (placement == MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                    Vec4T<cache_t> weight(&lxu_cache_weights[cache_idx_j][d]);
                    grad_indice_weight += weight.acc.x * grad_out[i].acc.x +
                        weight.acc.y * grad_out[i].acc.y +
                        weight.acc.z * grad_out[i].acc.z + weight.acc.w * grad_out[i].acc.w;
                } else {
                    int32_t D_emb = D;
                    if (std::is_same<emb_t, uint8_t>::value) {
                        D_emb += kINT8QparamsBytes;
                    }
                    auto weight_row = WeightRow<emb_t, cache_t, acc_type<cache_t, true>>(
                        const_cast<emb_t*>(&weights[idx_j * D_emb]),
                        nullptr,
                        D,
                        nullptr);
                    float2 qparams;
                    if (std::is_same<emb_t, uint8_t>::value) {
                        qparams = weight_row.load_qparams();
                    }
                    Vec4T<acc_type<cache_t, true>> weight =
                    weight_row.load(d, qparams);
                    grad_indice_weight += weight.acc.x * grad_out[i].acc.x +
                        weight.acc.y * grad_out[i].acc.y +
                        weight.acc.z * grad_out[i].acc.z + weight.acc.w * grad_out[i].acc.w;
                }
                {% else %}
                int32_t D_emb = D;
                if (std::is_same<emb_t, uint8_t>::value) {
                    D_emb += kINT8QparamsBytes;
                }
                auto weight_row = WeightRow<emb_t, cache_t, acc_type<cache_t, true>>(
                    const_cast<emb_t*>(&weights[idx_j * D_emb]),
                    nullptr,
                    D,
                    nullptr);
                float2 qparams;
                if (std::is_same<emb_t, uint8_t>::value) {
                    qparams = weight_row.load_qparams();
                }
                Vec4T<acc_type<cache_t, true>> weight =
                weight_row.load(d, qparams);
                grad_indice_weight += weight.acc.x * grad_out[i].acc.x +
                    weight.acc.y * grad_out[i].acc.y +
                    weight.acc.z * grad_out[i].acc.z + weight.acc.w * grad_out[i].acc.w;
                {% endif %}
            }
            grad_indice_weight =
                warpReduceAllSum<acc_type<cache_t, true>>(grad_indice_weight);
            if (threadIdx.x == 0) {
                grad_indice_weights[indices_start + l_start + j] = grad_indice_weight;
            }
        }
    }
}

Tensor {{ "dense" if dense else "split" }}_embedding_codegen_grad_indice_weights_cuda(
    Tensor grad_output,
    Tensor dev_weights,
    {% if not dense %}
    Tensor uvm_weights,
    Tensor lxu_cache_weights,
    Tensor weights_placements,
    {% endif %}
    Tensor weights_offsets,
    Tensor D_offsets,
    int64_t max_D,
    Tensor indices,
    Tensor offsets,
    {% if not dense %}
    Tensor lxu_cache_locations,
    {% endif %}
    Tensor feature_requires_grad) {
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());
    const auto T = D_offsets.size(0) - 1;
    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    const auto B = (offsets.size(0) - 1) / T;
    TORCH_CHECK(B >= 0);
    TORCH_CHECK(max_D <= {{ max_embedding_dim }});
    auto grad_indice_weights = empty_like(indices, indices.options().dtype(grad_output.dtype()));
    if (B == 0) {
      return grad_indice_weights;
    }
    feature_requires_grad = feature_requires_grad.defined() ? feature_requires_grad : empty({0}, indices.options().dtype(kInt));
    {% if not dense %}
    DISPATCH_EMB_CACHE_TYPES(
    {% else %}
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    {% endif %}
        dev_weights.type(),
        {% if not dense %}
        lxu_cache_weights.type(),
        {% endif %}
        "split_embedding_codegen_grad_indice_weights_kernel",
        ([&] {
            {% for kMaxVecsPerThread in range(1, max_embedding_dim // 128 + 1) %}
            if (max_D <= {{ 128 * kMaxVecsPerThread }}) {
            {{ "dense" if dense else "split" }}_embedding_codegen_grad_indice_weights_kernel<
                {% if not dense %}
                emb_t,
                cache_t,
                {% else %}
                scalar_t,
                scalar_t,
                {% endif %}
                {{ kMaxVecsPerThread }}><<<
                div_round_up((B * T), kForwardMaxThreads / kWarpSize),
                dim3(kWarpSize, kForwardMaxThreads / kWarpSize),
                0,
                at::cuda::getCurrentCUDAStream()>>>(
                {% if not dense %}
                grad_output.packed_accessor32<
                    acc_type<cache_t, true>,
                    2,
                    RestrictPtrTraits>(),
                dev_weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                {% else %}
                grad_output.packed_accessor32<
                    acc_type<scalar_t, true>,
                    2,
                    RestrictPtrTraits>(),
                dev_weights.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),
                {% endif %}
                {% if not dense %}
                uvm_weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                lxu_cache_weights.packed_accessor64<cache_t, 2, RestrictPtrTraits>(),
                weights_placements.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                {% endif %}
                weights_offsets.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                D_offsets.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                offsets.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                {% if not dense %}
                lxu_cache_locations.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                {% endif %}
                feature_requires_grad.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                {% if not dense %}
                grad_indice_weights.packed_accessor32<acc_type<cache_t, true>, 1, RestrictPtrTraits>()
                {% else %}
                grad_indice_weights.packed_accessor32<acc_type<scalar_t, true>, 1, RestrictPtrTraits>()
                {% endif %}
            );
            return;
            }
            {% endfor %}
        }));

  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return grad_indice_weights;
}
