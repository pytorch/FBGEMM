#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// clang-format off
{% set wdesc =  "weighted" if weighted else "unweighted" %}
#include "codegen/embedding_forward_template_helpers.cuh"

using namespace fbgemm_gpu;
using Tensor = at::Tensor;

namespace nbit {

/*
  Looping over the weight types is requires to generate all the C++ template
  declarations (not definitions) that will be invoked by the function
  `Tensor int_nbit_split_embedding*_codegen_forward_*_cuda(...)` later in the
  same generated source file.
*/
{% for emb_weight_type in ["FP32", "FP16", "FP8", "INT8", "INT4", "INT2"] %}
template<typename index_t, typename output_t, size_t OutputRowsPerThread, size_t WarpsPerBlock, size_t InputRowsInFlight, size_t MinNum128BRows, size_t MaxNum128BRows, bool DeviceOnly>
__launch_bounds__(WarpsPerBlock * kWarpSize)
__global__ void {{ type_map[emb_weight_type].enum_name }}_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel_small_L(
  const at::PackedTensorAccessor64<uint8_t, 1, at::RestrictPtrTraits> dev_weights,
  const at::PackedTensorAccessor64<uint8_t, 1, at::RestrictPtrTraits> uvm_weights,
  const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
  const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
  const at::PackedTensorAccessor32<uint8_t, 1, at::RestrictPtrTraits> weights_tys,
  {% if not nobag %}
  const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
  {% else %}
  const int64_t D,
  {% endif %}
  FixedDivisor fd_B, // FixedDivisor(div_round_up(B, OutputRowsPerThread))
  const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
  const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
  {% if not nobag %}
  const int64_t pooling_mode,
  {% endif %}
  const int64_t row_alignment,
  {% if weighted %}
  at::PackedTensorAccessor32<float, 1, at::RestrictPtrTraits>
      indice_weights,
  {% endif %}
  {% if type_map[emb_weight_type].enum_name == "FP8" %}
  const int exponent_bits,
  const int exponent_bias,
  {% endif %}
  at::PackedTensorAccessor32<output_t, 2, at::RestrictPtrTraits>
      output, // [B][total_D],
  const at::PackedTensorAccessor64<uint8_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
  const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> lxu_cache_locations
  );
{% endfor %} // for emb_weight_type in ["FP32", "FP16", "FP8", "INT8", "INT4", "INT2"]

}

Tensor int_nbit_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_cuda(
    Tensor dev_weights,
    Tensor uvm_weights,
    Tensor weights_placements,
    Tensor weights_offsets,
    Tensor weights_tys,
    {% if not nobag %}
    Tensor D_offsets,
    const int64_t total_D,
    {% else %}
    const int64_t D,
    {% endif %}
    const int64_t max_int2_D,
    const int64_t max_int4_D,
    const int64_t max_int8_D,
    const int64_t max_float16_D,
    const int64_t max_float32_D,
    Tensor indices,
    Tensor offsets,
    {% if not nobag %}
    const int64_t pooling_mode,
    {% endif %}
    const int64_t row_alignment,
    {% if weighted %}
    Tensor indice_weights,
    {% endif %}
    const int64_t output_dtype,
    Tensor lxu_cache_weights,
    Tensor lxu_cache_locations,
    const int64_t max_float8_D,
    const int64_t fp8_exponent_bits,
    const int64_t fp8_exponent_bias
) {
    TENSOR_ON_CUDA_GPU(dev_weights);
    TENSORS_ON_SAME_DEVICE(uvm_weights, dev_weights);
    TENSORS_ON_SAME_DEVICE(weights_placements, dev_weights);
    TENSORS_ON_SAME_DEVICE(weights_offsets, dev_weights);
    TENSORS_ON_SAME_DEVICE(weights_tys, dev_weights);
    {% if not nobag %}
    TENSORS_ON_SAME_DEVICE(D_offsets, dev_weights);
    {% endif %}
    TENSORS_ON_SAME_DEVICE(indices, dev_weights);
    TENSORS_ON_SAME_DEVICE(offsets, dev_weights);
    {% if weighted %}
    TENSORS_EMPTY_OR_ON_SAME_DEVICE(indice_weights, dev_weights);
    {% endif %}
    TENSORS_EMPTY_OR_ON_SAME_DEVICE(lxu_cache_weights, dev_weights);
    TENSORS_EMPTY_OR_ON_SAME_DEVICE(lxu_cache_locations, dev_weights);

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());

    // kernels assume indices are contiguous.
    indices = indices.contiguous();

    {% if not nobag %}
    const int32_t T = D_offsets.numel() - 1;
    {% else %}
    const int32_t total_L = indices.numel();
    const int32_t T = weights_offsets.numel();
    {% endif %}
    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    const int32_t B = (offsets.size(0) - 1) / T;
    TORCH_CHECK(B >= 0);

    {% if not nobag %}
    TORCH_CHECK(total_D > 0);
    {% else %}
    TORCH_CHECK(D > 0);
    {% endif %}

    Tensor output;
    const int kINT8QparamsBytes = 8;
    SparseType o_dtype = static_cast<SparseType>(output_dtype);
    TORCH_CHECK(o_dtype == SparseType::FP32 || o_dtype == SparseType::FP16 || o_dtype == SparseType::BF16 || o_dtype == SparseType::INT8);
    {% if not nobag %}
    int64_t total_adjusted_D = total_D;
    if (o_dtype == SparseType::INT8) {
        total_adjusted_D += T * kINT8QparamsBytes;
    }
    if (indices.numel() == 0) {
      output = at::zeros({B, total_adjusted_D}, dev_weights.options().dtype(getScalarType(o_dtype)));
    }
    else {
      output = at::empty({B, total_adjusted_D}, dev_weights.options().dtype(getScalarType(o_dtype)));
    }
    {% else %}
    int64_t adjusted_D = D;
    if (o_dtype == SparseType::INT8) {
        adjusted_D += T * kINT8QparamsBytes;
    }
    if (total_L == 0) {
      output = at::zeros({total_L, adjusted_D}, dev_weights.options().dtype(getScalarType(o_dtype)));
    }
    else {
      output = at::empty({total_L, adjusted_D}, dev_weights.options().dtype(getScalarType(o_dtype)));
    }

    {% endif %}

    if (B == 0 || indices.numel() == 0) {
      return output;
    }

    using index_t = int32_t;

    constexpr int32_t kWarpsPerBlock = 4;

    const auto device_only = lxu_cache_weights.numel() == 0 && uvm_weights.numel() == 0;
    #define Y(...) \
      if (device_only) { \
        X(true, __VA_ARGS__) \
      } else { \
        X(false, __VA_ARGS__) \
      };

    // launch 2-bit kernel
    #define X(DeviceOnly, OutputRowsPerThread, InputRowsInFlight, MinNum128BRows, MaxNum128BRows) \
    nbit::INT2_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel_small_L<index_t, output_t, OutputRowsPerThread, kWarpsPerBlock, InputRowsInFlight, MinNum128BRows, MaxNum128BRows, DeviceOnly><<< \
        nbit::div_round_up(T * nbit::div_round_up(B, OutputRowsPerThread), kWarpsPerBlock), \
        dim3(kWarpSize, kWarpsPerBlock), \
        0, \
        at::cuda::getCurrentCUDAStream()>>>( \
        dev_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        uvm_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(), \
        weights_tys.packed_accessor32<uint8_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        {% else %} \
        D, \
        {% endif %} \
        FixedDivisor(div_round_up(B, OutputRowsPerThread)), \
        indices.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        offsets.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        pooling_mode, \
        {% endif %} \
        row_alignment, \
        {% if weighted %} indice_weights.packed_accessor32<float, 1, at::RestrictPtrTraits>(), {% endif %} \
        output.packed_accessor32<output_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_weights.packed_accessor64<uint8_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>() \
    ); \
    C10_CUDA_KERNEL_LAUNCH_CHECK(); \

    DISPATCH_OUTPUT_TYPES(output.scalar_type(), "int2_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_kernel", ([&] {
      if (max_int2_D > 0) {
        auto max_int2_128b_rows = nbit::div_round_up(nbit::padded_row_size_in_bytes(max_int2_D, SparseType::INT2, row_alignment), 128);
        TORCH_CHECK(max_int2_128b_rows <= 4);
        if (max_int2_128b_rows > 0) {
          Y(2, 16, 0, 1);
        }
        if (max_int2_128b_rows > 1) {
          Y(2, 8, 1, 2);
        }
        if (max_int2_128b_rows > 2) {
          Y(2, 8, 2, 4);
        }
      }
    }));
    #undef X


    // launch 4-bit kernel
    #define X(DeviceOnly, OutputRowsPerThread, InputRowsInFlight, MinNum128BRows, MaxNum128BRows) \
    nbit::INT4_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel_small_L<index_t, output_t, OutputRowsPerThread, kWarpsPerBlock, InputRowsInFlight, MinNum128BRows, MaxNum128BRows, DeviceOnly><<< \
        nbit::div_round_up(T * nbit::div_round_up(B, OutputRowsPerThread), kWarpsPerBlock), \
        dim3(kWarpSize, kWarpsPerBlock), \
        0, \
        at::cuda::getCurrentCUDAStream()>>>( \
        dev_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        uvm_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(), \
        weights_tys.packed_accessor32<uint8_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        {% else %} \
        D, \
        {% endif %} \
        FixedDivisor(div_round_up(B, OutputRowsPerThread)), \
        indices.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        offsets.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        pooling_mode, \
        {% endif %} \
        row_alignment, \
        {% if weighted %} indice_weights.packed_accessor32<float, 1, at::RestrictPtrTraits>(), {% endif %} \
        output.packed_accessor32<output_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_weights.packed_accessor64<uint8_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>() \
    ); \
    C10_CUDA_KERNEL_LAUNCH_CHECK(); \

    DISPATCH_OUTPUT_TYPES(output.scalar_type(), "int4_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_kernel", ([&] {
      if (max_int4_D > 0) {
        auto max_int4_128b_rows = nbit::div_round_up(nbit::padded_row_size_in_bytes(max_int4_D, SparseType::INT4, row_alignment), 128);
        TORCH_CHECK(max_int4_128b_rows <= 8);
        if (max_int4_128b_rows > 0) {
          Y(4, 8, 0, 1);
        }
        if (max_int4_128b_rows > 1) {
          Y(2, 8, 1, 2);
        }
        if (max_int4_128b_rows > 2) {
          Y(1, 4, 2, 4);
        }
        if (max_int4_128b_rows > 4) {
          Y(1, 4, 4, 8);
        }
      }
    }));
    #undef X

    // launch 8-bit int kernel
    #define X(DeviceOnly, OutputRowsPerThread, InputRowsInFlight, MinNum128BRows, MaxNum128BRows) \
    nbit::INT8_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel_small_L<index_t, output_t, OutputRowsPerThread, kWarpsPerBlock, InputRowsInFlight, MinNum128BRows, MaxNum128BRows, DeviceOnly><<< \
        nbit::div_round_up(T * nbit::div_round_up(B, OutputRowsPerThread), kWarpsPerBlock), \
        dim3(kWarpSize, kWarpsPerBlock), \
        0, \
        at::cuda::getCurrentCUDAStream()>>>( \
        dev_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        uvm_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(), \
        weights_tys.packed_accessor32<uint8_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        {% else %} \
        D, \
        {% endif %} \
        FixedDivisor(div_round_up(B, OutputRowsPerThread)), \
        indices.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        offsets.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        pooling_mode, \
        {% endif %} \
        row_alignment, \
        {% if weighted %} indice_weights.packed_accessor32<float, 1, at::RestrictPtrTraits>(), {% endif %} \
        output.packed_accessor32<output_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_weights.packed_accessor64<uint8_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>() \
    ); \
    C10_CUDA_KERNEL_LAUNCH_CHECK(); \

    DISPATCH_OUTPUT_TYPES(output.scalar_type(), "int8_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_kernel", ([&] {
      if (max_int8_D > 0) {
        auto max_int8_128b_rows = nbit::div_round_up(nbit::padded_row_size_in_bytes(max_int8_D, SparseType::INT8, row_alignment), 128);
        TORCH_CHECK(max_int8_128b_rows <= 16);
        if (max_int8_128b_rows > 0) {
          Y(2, 8, 0, 1);
        }
        if (max_int8_128b_rows > 1) {
          Y(2, 4, 1, 2);
        }
        if (max_int8_128b_rows > 2) {
          Y(2, 4, 2, 4);
        }
        if (max_int8_128b_rows > 4) {
          Y(2, 4, 4, 8);
        }
        if (max_int8_128b_rows > 8) {
          Y(2, 2, 8, 16);
        }
      }
    }));
    #undef X

    // launch 8-bit float kernel
    #define X(DeviceOnly, OutputRowsPerThread, InputRowsInFlight, MinNum128BRows, MaxNum128BRows) \
    nbit::FP8_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel_small_L<index_t, output_t, OutputRowsPerThread, kWarpsPerBlock, InputRowsInFlight, MinNum128BRows, MaxNum128BRows, DeviceOnly><<< \
        nbit::div_round_up(T * nbit::div_round_up(B, OutputRowsPerThread), kWarpsPerBlock), \
        dim3(kWarpSize, kWarpsPerBlock), \
        0, \
        at::cuda::getCurrentCUDAStream()>>>( \
        dev_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        uvm_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(), \
        weights_tys.packed_accessor32<uint8_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        {% else %} \
        D, \
        {% endif %} \
        FixedDivisor(div_round_up(B, OutputRowsPerThread)), \
        indices.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        offsets.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        pooling_mode, \
        {% endif %} \
        row_alignment, \
        {% if weighted %} indice_weights.packed_accessor32<float, 1, at::RestrictPtrTraits>(), {% endif %} \
        fp8_exponent_bits, \
        fp8_exponent_bias, \
        output.packed_accessor32<output_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_weights.packed_accessor64<uint8_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>() \
    ); \
    C10_CUDA_KERNEL_LAUNCH_CHECK(); \

    DISPATCH_OUTPUT_TYPES(output.scalar_type(), "fp8_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_kernel", ([&] {
      if (max_float8_D > 0) {
        auto max_fp8_128b_rows = nbit::div_round_up(nbit::padded_row_size_in_bytes(max_float8_D, SparseType::FP8, row_alignment), 128);
        TORCH_CHECK(max_fp8_128b_rows <= 16);
        if (max_fp8_128b_rows > 0) {
          Y(2, 8, 0, 1);
        }
        if (max_fp8_128b_rows > 1) {
          Y(2, 4, 1, 2);
        }
        if (max_fp8_128b_rows > 2) {
          Y(2, 4, 2, 4);
        }
        if (max_fp8_128b_rows > 4) {
          Y(2, 4, 4, 8);
        }
        if (max_fp8_128b_rows > 8) {
          Y(2, 2, 4, 8);
        }
      }
    }));
    #undef X

    // launch 16-bit kernel
    #define X(DeviceOnly, OutputRowsPerThread, InputRowsInFlight, MinNum128BRows, MaxNum128BRows) \
    nbit::FP16_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel_small_L<index_t, output_t, OutputRowsPerThread, kWarpsPerBlock, InputRowsInFlight, MinNum128BRows, MaxNum128BRows, DeviceOnly><<< \
        nbit::div_round_up(T * nbit::div_round_up(B, OutputRowsPerThread), kWarpsPerBlock), \
        dim3(kWarpSize, kWarpsPerBlock), \
        0, \
        at::cuda::getCurrentCUDAStream()>>>( \
        dev_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        uvm_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(), \
        weights_tys.packed_accessor32<uint8_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        {% else %} \
        D, \
        {% endif %} \
        FixedDivisor(div_round_up(B, OutputRowsPerThread)), \
        indices.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        offsets.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        pooling_mode, \
        {% endif %} \
        row_alignment, \
        {% if weighted %} indice_weights.packed_accessor32<float, 1, at::RestrictPtrTraits>(), {% endif %} \
        output.packed_accessor32<output_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_weights.packed_accessor64<uint8_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>() \
    ); \
    C10_CUDA_KERNEL_LAUNCH_CHECK(); \

    DISPATCH_OUTPUT_TYPES(output.scalar_type(), "fp16_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_kernel", ([&] {
      if (max_float16_D > 0) {
        auto max_fp16_128b_rows = nbit::div_round_up(nbit::padded_row_size_in_bytes(max_float16_D, SparseType::FP16, row_alignment), 128);
        TORCH_CHECK(max_fp16_128b_rows <= 32);
        if (max_fp16_128b_rows > 0) {
          Y(2, 8, 0, 2);
        }
        if (max_fp16_128b_rows > 2) {
          Y(2, 8, 2, 4);
        }
        if (max_fp16_128b_rows > 4) {
          Y(2, 4, 4, 8);
        }
        if (max_fp16_128b_rows > 8) {
          Y(2, 2, 8, 16);
        }
        if (max_fp16_128b_rows > 16) {
          Y(2, 1, 16, 32);
        }
      }
    }));
    #undef X

    // launch 32-bit kernel
    #define X(DeviceOnly, OutputRowsPerThread, InputRowsInFlight, MinNum128BRows, MaxNum128BRows) \
    nbit::FP32_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel_small_L<index_t, output_t, OutputRowsPerThread, kWarpsPerBlock, InputRowsInFlight, MinNum128BRows, MaxNum128BRows, DeviceOnly><<< \
        nbit::div_round_up(T * nbit::div_round_up(B, OutputRowsPerThread), kWarpsPerBlock), \
        dim3(kWarpSize, kWarpsPerBlock), \
        0, \
        at::cuda::getCurrentCUDAStream()>>>( \
        dev_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        uvm_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(), \
        weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(), \
        weights_tys.packed_accessor32<uint8_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(), \
        {% else %} \
        D, \
        {% endif %} \
        FixedDivisor(div_round_up(B, OutputRowsPerThread)), \
        indices.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        offsets.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(), \
        {% if not nobag %} \
        pooling_mode, \
        {% endif %} \
        row_alignment, \
        {% if weighted %} indice_weights.packed_accessor32<float, 1, at::RestrictPtrTraits>(), {% endif %} \
        output.packed_accessor32<output_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_weights.packed_accessor64<uint8_t, 2, at::RestrictPtrTraits>(), \
        lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>() \
    ); \
    C10_CUDA_KERNEL_LAUNCH_CHECK(); \

    DISPATCH_OUTPUT_TYPES(output.scalar_type(), "fp32_split_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_kernel", ([&] {
      if (max_float32_D > 0) {
        auto max_fp32_128b_rows = nbit::div_round_up(nbit::padded_row_size_in_bytes(max_float32_D, SparseType::FP32, row_alignment), 128);
        TORCH_CHECK(max_fp32_128b_rows <= 64);
        if (max_fp32_128b_rows > 0) {
          Y(2, 4, 0, 4);
        }
        if (max_fp32_128b_rows > 4) {
          Y(2, 2, 4, 16);
        }
        if (max_fp32_128b_rows > 16) {
          Y(1, 1, 16, 32);
        }
        if (max_fp32_128b_rows > 32) {
          Y(1, 1, 32, 64);
        }
      }
    }));
    #undef X

    return output;
}

// clang-format on
