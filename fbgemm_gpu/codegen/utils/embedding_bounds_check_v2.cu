#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "fbgemm_gpu/utils/embedding_bounds_check_common.cuh"

template <typename index_t, bool vbe, BoundsCheckMode bounds_check_mode>
__global__ __launch_bounds__(kMaxThreads) void bounds_check_indices_kernel_v2(
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        rows_per_table,
    pta::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    pta::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    const int32_t* const B_offsets, // Use a raw pointer to avoid creating a
                                    // dummy PackedTensorAccessor
    pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> warning,
    FixedDivisor fd,
    const int32_t* const b_t_map,
    const int32_t info_B_num_bits,
    const int32_t info_B_mask,
    TORCH_DSA_KERNEL_ARGS) {
  int32_t T = rows_per_table.size(0);
  int32_t total_B = offsets.size(0) - 1;
  int32_t B = vbe ? 0 : (total_B / T);

  const index_t num_indices = indices.size(0);
  const auto b_t_start = blockIdx.x * blockDim.y + threadIdx.y;
  index_t invalid_i = -1, invalid_idx = -1;
  int32_t invalid_b_t = -1;
  int64_t warning_inc = 0;

  // Check the last element
  if (b_t_start == 0 && threadIdx.x == 0) {
    if (bounds_check_mode == BoundsCheckMode::FATAL) {
      CUDA_KERNEL_ASSERT2(num_indices == offsets[total_B]);
    } else if (bounds_check_mode == BoundsCheckMode::WARNING) {
      if (num_indices != offsets[total_B]) {
        if (gpuAtomicIncrement(&warning[0]) == 0) {
          printf(
              "EmbeddingBoundsCheck (VBE %s): the last element in offsets is incorrect for "
              "total batch size %s: %d, total table num T: %d, "
              " last element in offsets: %lld, indices size: %lld. "
              " Setting the last element in offsets to be indices size.\n",
              vbe ? "true" : "false",
              vbe ? "total_B" : "B",
              vbe ? total_B : B,
              T,
              static_cast<int64_t>(offsets[total_B]),
              static_cast<int64_t>(num_indices));
        }
        offsets[total_B] = num_indices;
      }
    } else if (bounds_check_mode == BoundsCheckMode::IGNORE) {
      if (num_indices != offsets[total_B]) {
        offsets[total_B] = num_indices;
      }
    }
  }

  for (auto b_t = blockIdx.x * blockDim.y + threadIdx.y; b_t < total_B;
       b_t += blockDim.y * gridDim.x) {
    // Compute b and t
    int32_t b;
    int32_t t;
    if (vbe) {
      const auto info = *reinterpret_cast<const uint32_t*>(&b_t_map[b_t]);
      *reinterpret_cast<uint32_t*>(&t) = info >> info_B_num_bits;
      *reinterpret_cast<uint32_t*>(&b) = info & info_B_mask;
    } else {
      fd.DivMod(b_t, &t, &b);
    }

    const auto num_rows = rows_per_table[t];
    auto indices_start = offsets[b_t];
    auto indices_end = offsets[b_t + 1];

    if (bounds_check_mode == BoundsCheckMode::FATAL) {
      CUDA_KERNEL_ASSERT2(indices_start >= 0);
      CUDA_KERNEL_ASSERT2(indices_start <= indices_end);
      CUDA_KERNEL_ASSERT2(indices_end <= num_indices);
    } else if (bounds_check_mode == BoundsCheckMode::WARNING) {
      if (indices_start < 0 || indices_start > indices_end ||
          indices_end > num_indices) {
        if (threadIdx.x == 0 && gpuAtomicIncrement(&warning[0]) == 0) {
          printf(
              "EmbeddingBoundsCheck (VBE %s): (at least one) Out of bounds access for "
              "batch: %d, table: %d, indices_start: %lld, indices_end: %lld,"
              " num_indices: %lld. Setting indices_start and indices_end within "
              "the range.\n",
              vbe ? "true" : "false",
              b,
              t,
              static_cast<int64_t>(indices_start),
              static_cast<int64_t>(indices_end),
              static_cast<int64_t>(num_indices));
        }
        adjust_offset_kernel(
            indices_start,
            indices_end,
            num_indices,
            &offsets[b_t],
            &offsets[b_t + 1]);
      }
    } else if (bounds_check_mode == BoundsCheckMode::IGNORE) {
      adjust_offset_kernel(
          indices_start,
          indices_end,
          num_indices,
          &offsets[b_t],
          &offsets[b_t + 1]);
    }

    const auto L = indices_end - indices_start;
    for (index_t i = static_cast<index_t>(threadIdx.x); i < L;
         i += static_cast<index_t>(fbgemm_gpu::kWarpSize)) {
      const auto idx = indices[indices_start + i];
      if (idx == -1) {
        // -1 indicates pruned rows.
        continue;
      }
      if (bounds_check_mode == BoundsCheckMode::FATAL) {
        CUDA_KERNEL_ASSERT2(
            idx >= 0 && "Failed idx >= 0 in bounds_check_indices");
        CUDA_KERNEL_ASSERT2(
            idx < num_rows && "Failed idx < num_rows in bounds_check_indices");
      } else if (bounds_check_mode == BoundsCheckMode::WARNING) {
        if (idx < 0 || idx >= num_rows) {
          invalid_i = i;
          invalid_idx = idx;
          invalid_b_t = b_t;
          indices[indices_start + i] = 0;
          warning_inc += 1;
        }
      } else if (bounds_check_mode == BoundsCheckMode::IGNORE) {
        if (idx < 0 || idx >= num_rows) {
          indices[indices_start + i] = 0;
        }
      }
    }
  } // for b_t

  if (warning_inc > 0) {
    gpuAtomicAdd(&warning[0], warning_inc);
  }
  if (bounds_check_mode == BoundsCheckMode::WARNING && invalid_i != -1 &&
      static_cast<int64_t>(atomicAdd(
          reinterpret_cast<unsigned long long int*>(&warning[0]), 0)) == 0) {
    int32_t b;
    int32_t t;

    fd.DivMod(invalid_b_t, &t, &b);

    int32_t B = vbe ? (B_offsets[t + 1] - B_offsets[t]) : (total_B / T);

    printf(
        "EmbeddingBoundsCheck (VBE %s): (at least one) Out of bounds access for "
        "batch: %d, table: %d, bag element: %lld, idx: %lld, num_rows: %lld, "
        "indices_start: %lld, indices_end: %lld, T: %d, B: %d, b_t: %d. "
        "Setting idx to zero.\n",
        vbe ? "true" : "false",
        b,
        t,
        static_cast<int64_t>(invalid_i),
        static_cast<int64_t>(invalid_idx),
        rows_per_table[t],
        static_cast<int64_t>(offsets[invalid_b_t]),
        static_cast<int64_t>(offsets[invalid_b_t + 1]),
        T,
        B,
        invalid_b_t);
  }
}

void _bounds_check_indices_cuda_v2(
    Tensor& rows_per_table,
    Tensor& indices,
    Tensor& offsets,
    BoundsCheckMode bounds_check_mode,
    Tensor& warning,
    const std::optional<Tensor>& weights,
    const std::optional<Tensor>& B_offsets,
    int64_t /*max_B*/,
    const std::optional<Tensor>& b_t_map,
    int32_t info_B_num_bits,
    uint32_t info_B_mask,
    int64_t /*T*/,
    int64_t B,
    int64_t total_B,
    bool vbe,
    bool prefetch_pipeline) {
  if (vbe) {
    TORCH_CHECK(b_t_map.has_value());
    TENSOR_NDIM_EQUALS(b_t_map.value(), 1);
  }

  CUDA_DEVICE_GUARD(rows_per_table);

  if (bounds_check_mode == BoundsCheckMode::WARNING) {
    warning.zero_();
  }

  constexpr size_t kNumThreads = 1024;
  auto grid_dim =
      min(div_round_up(total_B, kNumThreads / fbgemm_gpu::kWarpSize),
          get_max_thread_blocks_());
  if (prefetch_pipeline) {
    // Limit the grid size to PREFETCH_KERNEL_MAX_BLOCKS if running this kernel
    // on the prefetch stream
    constexpr int PREFETCH_KERNEL_MAX_BLOCKS = 8;
    grid_dim = min(grid_dim, PREFETCH_KERNEL_MAX_BLOCKS);
  }

#define INVOKE_BOUNDS_CHECK_INDICES(MODE)                                      \
  if (bounds_check_mode == MODE) {                                             \
    AT_DISPATCH_INDEX_TYPES(                                                   \
        indices.scalar_type(), "bounds_check_indices_cuda", [&] {              \
          [[maybe_unused]] const auto func_name =                              \
              "bounds_check_indices_cuda_v2";                                  \
          const auto bounds_check_kernel =                                     \
              (vbe ? bounds_check_indices_kernel_v2<index_t, true, MODE>       \
                   : bounds_check_indices_kernel_v2<index_t, false, MODE>);    \
          FBGEMM_LAUNCH_DSA_KERNEL(                                            \
              bounds_check_kernel,                                             \
              grid_dim,                                                        \
              dim3(                                                            \
                  fbgemm_gpu::kWarpSize, kNumThreads / fbgemm_gpu::kWarpSize), \
              0,                                                               \
              at::cuda::getCurrentCUDAStream(),                                \
              PTA_B(rows_per_table, int64_t, 1, 32),                           \
              PTA_B(indices, index_t, 1, 32),                                  \
              PTA_B(offsets, index_t, 1, 32),                                  \
              vbe ? B_offsets.value().data_ptr<int32_t>() : nullptr,           \
              PTA_B(warning, int64_t, 1, 32),                                  \
              FixedDivisor(B),                                                 \
              vbe ? b_t_map.value().data_ptr<int32_t>() : nullptr,             \
              info_B_num_bits,                                                 \
              info_B_mask);                                                    \
        });                                                                    \
  }

  INVOKE_BOUNDS_CHECK_INDICES(BoundsCheckMode::FATAL)
  INVOKE_BOUNDS_CHECK_INDICES(BoundsCheckMode::WARNING)
  INVOKE_BOUNDS_CHECK_INDICES(BoundsCheckMode::IGNORE)

#undef INVOKE_BOUNDS_CHECK_INDICES
}
