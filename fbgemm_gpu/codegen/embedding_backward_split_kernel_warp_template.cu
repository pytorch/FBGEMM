#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

// clang-format off
{%- set wdesc = "weighted" if weighted else "unweighted" %}
{%- set ndesc = "_nobag" if nobag else "" %}
{%- set vdesc = "_vbe" if vbe else "" %}

#include "fbgemm_gpu/embedding_backward_template_helpers.cuh"
#include "fbgemm_gpu/fbgemm_tensor_accessor.h"
#include "fbgemm_gpu/split_embeddings_utils.cuh"
{%- if optimizer != "none" and not dense %}
#include "gen_embedding_optimizer_{{ optimizer }}_split_device_kernel.cuh"
{%- endif %}
#include "gen_embedding_backward_{{ kdesc }}_split_device_kernel.cuh"
#include "gen_embedding_backward_common_split_device_kernel.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

////////////////////////////////////////////////////////////////////////////////
// Kernel Template Definition
////////////////////////////////////////////////////////////////////////////////

template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    int32_t kFixedMaxVecsPerThread,
    int32_t kThreadGroupSize,
    bool kUseVecBlocking>
__global__ __launch_bounds__(kBackwardMaxThreads) void
{%- if is_index_select %}
batch_index_select_dim0_codegen_backward_kernel_warp_per_row(
{%- else %}
split_embedding{{ ndesc }}_backward_codegen_{{ optimizer }}_{{ wdesc }}{{ vdesc }}_kernel_warp_per_row_1(
{%- endif %}
    const pta::PackedTensorAccessor64<grad_t, {{ "1" if is_index_select else "2" }}, at::RestrictPtrTraits> grad_output,
    {%- if optimizer != "none" %}
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    pta::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag or is_index_select %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int64_t D,
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> hash_size_cumsum,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_linear_indices_run,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_cumulative_run_lengths,
    {%- if not nobag %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- else %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- endif %}
    {%- if not dense %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_lxu_cache_locations,
    const bool use_uniq_cache_locations,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> table_unique_indices_offsets,
    {%- endif %}
    {%- if weighted %}
    const pta::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {%- endif %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_num_runs,
    int32_t max_segment_length_per_warp,
    {%- if not dense and optimizer != "none" %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {%- else %}
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> grad_dev_weights,
    {%- endif %} // if not dense and optimizer != "none"
    {%- if not nobag and vbe %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> B_offsets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> row_output_offsets,
    {%- endif %}
    {%- if not nobag %}
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    {%- endif %}
    const int32_t max_D,
    const int32_t max_vecs_per_thread,
    {%- if is_index_select %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> grad_offsets,
    const bool permute_output_dim_0_1
    {%- else %}
    {{ args.split_kernel_args | replace_pta_namespace() | join(",\n    ") }}
    {%- endif %}
) {
    {%- if not nobag %}
    int32_t T = D_offsets.size(0) - 1;
    {%- else %}
    int32_t T = weights_offsets.size(0);
    {%- endif %}
    const int32_t start_run_id = blockIdx.x * blockDim.y + threadIdx.y;

#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
    const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
    const unsigned int shfl_sync_mask = 0xffffffffu;
#endif
    constexpr int VEC_WIDTH = 4;
    constexpr auto kIsInt8 = std::is_same<emb_t, uint8_t>::value;

    struct SharedMemory<Vec4TAcc<cache_t>> smem;
    const int32_t grad_sum_stride = max_D / VEC_WIDTH;
    auto* smem_grad_sum = (kUseVecBlocking || kIsInt8)
      ? smem.getPointer() + threadIdx.y * grad_sum_stride
      : nullptr;

    for (uint32_t run_id = start_run_id;
         run_id < sorted_linear_indices_run.size(0) && run_id < sorted_linear_indices_num_runs[0];
             run_id += gridDim.x * blockDim.y) {

        const int64_t linear_index = sorted_linear_indices_run[run_id];
        const int32_t segment_start =
            sorted_linear_indices_cumulative_run_lengths[run_id];
        const int32_t segment_end =
            sorted_linear_indices_cumulative_run_lengths[run_id + 1];
        const int32_t SL = segment_end - segment_start;


        if (SL >= max_segment_length_per_warp) {
            continue;
        }

        // now, each segment corresponds to exactly one table `t` and row in
        // that table (`idx`). Thus, we can hoist out some of the book-keeping.
        {%- if not nobag %}
        const auto info_0 = reinterpret_cast<const uint32_t*>(&sorted_infos[0])[segment_start];
        const auto t_0 = info_0 >> info_B_num_bits;
        {%- else %}
        const auto info_0 = sorted_infos[segment_start];
        int32_t t_0 = info_0 % T;
        {%- endif %}

        int64_t hash_size = hash_size_cumsum[t_0];
        {%- if not nobag or is_index_select %}
        const auto D_start_t0 = D_offsets[t_0];
        // D can be hoisted here because D is the same if features share the
        // same table, but D_start is different
        const int32_t D = D_offsets[t_0 + 1] - D_start_t0;
        {%- if is_index_select %}
        // grad_offset can be hoisted here for batch_index_select because it
        // does not allow multiple features to share a single embedding table
        const auto grad_offset = permute_output_dim_0_1 ? D_start_t0 : grad_offsets[t_0];
        const auto grad_stride = permute_output_dim_0_1 ? D_offsets[T] : D;
        {%- endif %}
        {%- endif %}
        int64_t idx = linear_index - hash_size;

        const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
        const int32_t sl_start = 0;
        const int32_t sl_end = SL;
        Vec4TAcc<cache_t> grad_sum[kFixedMaxVecsPerThread];
        constexpr int32_t kGroupVecWidth = kThreadGroupSize * VEC_WIDTH;
        const int32_t num_vecs = (D + kGroupVecWidth - 1) / kGroupVecWidth;

        compute_grad_sum_{{ kdesc }}<
          grad_t,
          cache_t,
          kFixedMaxVecsPerThread,
          kThreadGroupSize,
          VEC_WIDTH,
          kUseVecBlocking>(
            grad_sum,
            smem_grad_sum,
            grad_output,
            {%- if not nobag or is_index_select %}
            D_offsets,
            {%- endif %}
            D,
            T,
            sorted_infos,
            {%- if weighted %}
            sorted_indice_weights,
            {%- endif %}
            {%- if not nobag and vbe %}
            B_offsets,
            row_output_offsets,
            {%- endif %}
            {%- if is_index_select %}
            grad_offset,
            grad_stride,
            {%- endif %}
            {%- if not nobag %}
            info_B_num_bits,
            info_B_mask,
            {%- endif %}
            segment_start,
            sl_start,
            sl_end,
            shfl_sync_mask,
            num_vecs
        );

        // Copy value to max_vecs to make max_vecs_per_thread known at compile time
        // when kUseVecBlocking == false
        const int32_t max_vecs =
            kUseVecBlocking ? max_vecs_per_thread : kFixedMaxVecsPerThread;

        {%- if not dense and optimizer != "none" %}
        split_{{ optimizer }}_table_update_kernel<
          emb_t,
          cache_t,
          kFixedMaxVecsPerThread,
          kThreadGroupSize,
          VEC_WIDTH,
          kUseVecBlocking>(
              dev_weights,
              uvm_weights,
              lxu_cache_weights,
              weights_placements,
              weights_offsets,
              sorted_lxu_cache_locations,
              grad_sum,
              smem_grad_sum,
              smem_grad_sum, // shared_weight_update_row (reuse smem_grad_sum)
              stochastic_rounding,
              stochastic_rounding_philox_args,
              run_id,
              use_uniq_cache_locations
                  ? (run_id - table_unique_indices_offsets[t_0])
                  : segment_start,
              D,
              t_0,
              idx,
              shfl_sync_mask,
              max_vecs,
              {{ args.split_function_arg_names | join(", ") }}
        );
        {%- else %}
        // Write deduplicated gradient to grad_dev_weights gradient is sparse
        // for split_embedding and dense for dense_embedding
        {%- if dense %}
        const int64_t weights_offset = weights_offsets[t_0];
        {%- else %}
        // Compute offset of sparse gradient
        const int64_t weights_offset = run_id * max_D;
        idx = 0;
        {%- endif %}
        store_grad_sum<
            emb_t,
            cache_t,
            kFixedMaxVecsPerThread,
            kThreadGroupSize,
            VEC_WIDTH,
            kUseVecBlocking>(
              grad_dev_weights,
              grad_sum,
              kUseVecBlocking ? smem_grad_sum : nullptr,
              D,
              weights_offset,
              idx,
              max_vecs
        );
        {%- endif %} // if not dense and optimizer != "none"
    }
}


////////////////////////////////////////////////////////////////////////////////
// Explicit Template Instantiations
////////////////////////////////////////////////////////////////////////////////

/*
    Explicitly instantiate the kernel function template.  The instantiations are
    based on the types enumerated by DISPATCH_EMB_GRAD_CACHE_TYPES macro used in
    embedding_backward_split_template.cu
*/

{%- macro template_instantiation(
      emb_type,
      grad_type,
      cache_type,
      kFixedMaxVecsPerThread,
      kThreadGroupSize,
      kUseVecBlocking
    )
%}
template __global__ __launch_bounds__(kBackwardMaxThreads) void
{%- if is_index_select %}
batch_index_select_dim0_codegen_backward_kernel_warp_per_row
{%- else %}
split_embedding{{ ndesc }}_backward_codegen_{{ optimizer }}_{{ wdesc }}{{ vdesc }}_kernel_warp_per_row_1
{%- endif %}
< {{ emb_type }},
  {{ grad_type }},
  {{ cache_type }},
  {{ kFixedMaxVecsPerThread }},
  {{ kThreadGroupSize }},
  {{ kUseVecBlocking }}
> (
    const pta::PackedTensorAccessor64<{{ grad_type }}, {{ "1" if is_index_select else "2" }}, at::RestrictPtrTraits> grad_output,
    {%- if optimizer != "none" %}
    pta::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    pta::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> uvm_weights,
    pta::PackedTensorAccessor64<{{ cache_type }}, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag or is_index_select %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int64_t D,
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> hash_size_cumsum,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_linear_indices_run,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_cumulative_run_lengths,
    {%- if not nobag %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- else %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- endif %}
    {%- if not dense %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_lxu_cache_locations,
    const bool use_uniq_cache_locations,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> table_unique_indices_offsets,
    {%- endif %}
    {%- if weighted %}
    const pta::PackedTensorAccessor32<at::acc_type<{{ cache_type }}, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {%- endif %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_num_runs,
    int32_t max_segment_length_per_warp,
    {%- if not dense and optimizer != "none" %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {%- else %}
    pta::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> grad_dev_weights,
    {%- endif %} // if not dense and optimizer != "none"
    {%- if not nobag and vbe %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> B_offsets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> row_output_offsets,
    {%- endif %}
    {%- if not nobag %}
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    {%- endif %}
    const int32_t max_D,
    const int32_t max_vecs_per_thread,
    {%- if is_index_select %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> grad_offsets,
    const bool permute_output_dim_0_1
    {%- else %}
    {{ args.split_kernel_args_no_defaults | replace_pta_namespace() | join(",\n    ") | replace("cache_t", cache_type) }}
    {%- endif %}
);
{%- endmacro %}

{%- macro bulk_template_instantiations(kFixedMaxVecsPerThread, kThreadGroupSize, kUseVecBlocking) %}
    {%- for grad_type in ['float', 'at::Half', 'at::BFloat16'] %}
    {%- for emb_type in ['float', 'at::Half'] %}
    {%- for cache_type in ['float', 'at::Half'] %}
        {{ template_instantiation(
            emb_type,
            grad_type,
            cache_type,
            kFixedMaxVecsPerThread,
            kThreadGroupSize,
            kUseVecBlocking
          )
        }}
    {%- endfor %}
    {%- endfor %}
    {%- endfor %}
{%- endmacro %}


{%- if is_experimental_optimizer %}

{{
  bulk_template_instantiations(
    fixed_max_vecs_per_thread["backward"],
    'kWarpSize',
    'true'
  )
}}

{%- else %}

{%- macro instantiate_templates(use_subwarp_shuffle) %}
{%- for (kFixedMaxVecsPerThread, kThreadGroupSize, kUseVecBlocking)
    in get_max_vecs_template_configs(
        items_per_warp,
        fixed_max_vecs_per_thread["backward"],
        use_subwarp_shuffle,
        use_vec_blocking=True,
    )
%}
    {{
      bulk_template_instantiations(
        kFixedMaxVecsPerThread,
        kThreadGroupSize,
        kUseVecBlocking,
      )
    }}
{%- endfor %}
{%- endmacro %}


////////////////////////////////////////////////////////////////////////////////
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
////////////////////////////////////////////////////////////////////////////////

{#- /*
    Explicitly instantiate kernels for the FBGEMM_USE_SUBWARP_SHUFFLE case

    Please see get_max_vecs_template_configs in
    codegen/embedding_common_code_generator.py for more details
*/ #}

{{ instantiate_templates(use_subwarp_shuffle=True) }}

////////////////////////////////////////////////////////////////////////////////
#else
////////////////////////////////////////////////////////////////////////////////

{#- /*
    Explicitly instantiate kernels for the non-FBGEMM_USE_SUBWARP_SHUFFLE case

    Please see get_max_vecs_template_configs in
    codegen/embedding_common_code_generator.py for more details
*/ #}

{{ instantiate_templates(use_subwarp_shuffle=False) }}

////////////////////////////////////////////////////////////////////////////////
#endif
////////////////////////////////////////////////////////////////////////////////

{%- endif %}
        // clang-format on
