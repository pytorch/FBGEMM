#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

// clang-format off
{%- set wdesc = "weighted" if weighted else "unweighted" %}
{%- set ndesc = "_nobag" if nobag else "" %}
{%- set vdesc = "_vbe" if vbe else "" %}

#include "fbgemm_gpu/embedding_backward_template_helpers.cuh"
#include "fbgemm_gpu/fbgemm_tensor_accessor.h"
#include "fbgemm_gpu/split_embeddings_utils.cuh"
{%- if optimizer != "none" and not dense %}
#include "gen_embedding_optimizer_{{ optimizer }}_split_device_kernel.cuh"
{%- endif %}
#include "gen_embedding_backward_{{ kdesc }}_split_device_kernel.cuh"
#include "gen_embedding_backward_common_split_device_kernel.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

////////////////////////////////////////////////////////////////////////////////
// Kernel Template Definition
////////////////////////////////////////////////////////////////////////////////

{%- macro sync_grad_sums(kBlockDim) %}
    {%- set kWarpId = kBlockDim // 2 %}
    {%- set d_vec = "(vec * kThreadGroupSize + lane_id)" %}
    if (blockDim.y >= {{ kBlockDim }}) {
      if (warp_id < {{ kWarpId }}) {
        for (int32_t vec = 0; vec < max_vecs && {{ d_vec }} * VEC_WIDTH < D; ++vec) {
          const int32_t d_vec = {{ d_vec }};
          smem_grad_sum[d_vec] = vec4_acc(
              smem_grad_sum[d_vec],
              smem_grad_sum[d_vec +
                  {{ kWarpId }} * max_vecs * kThreadGroupSize]);
        }
      }
      __syncthreads();
  }
{%- endmacro %}

template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    int32_t kFixedMaxVecsPerThread,
    int32_t kThreadGroupSize,
    bool kUseVecBlocking>
__global__ __launch_bounds__(kMaxThreads) void
{%- if is_index_select %}
batch_index_select_dim0_codegen_backward_kernel_cta_per_row(
{%- else %}
split_embedding{{ ndesc }}_backward_codegen_{{ optimizer }}_{{ wdesc }}{{ vdesc }}_kernel_cta_per_row_1(
{%- endif %}
    const pta::PackedTensorAccessor64<grad_t, {{ "1" if is_index_select else "2" }}, at::RestrictPtrTraits> grad_output,
    {%- if optimizer != "none" %}
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    pta::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    {%- endif %} // if optimizer != "none"
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag or is_index_select %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int64_t D,
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> hash_size_cumsum,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_linear_indices_run,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_cumulative_run_lengths,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_ids,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> num_long_run_ids,
    {%- if not nobag %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- else %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- endif %}
    {%- if not dense %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_lxu_cache_locations,
    const bool use_uniq_cache_locations,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> table_unique_indices_offsets,
    {%- endif %}
    {%- if weighted %}
    const pta::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {%- endif %}
    {%- if not dense and optimizer != "none" %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {%- else %}
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> grad_dev_weights,
    {%- if optimizer == "none" %}
    const int32_t max_D,
    {%- endif %}
    {%- endif %} // if not dense and optimizer != "none"
    {%- if not nobag and vbe %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> B_offsets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> row_output_offsets,
    {%- endif %}
    {%- if not nobag %}
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    {%- endif %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_id_to_really_long_run_ids,
    pta::PackedTensorAccessor32<at::acc_type<cache_t, true>, 2, at::RestrictPtrTraits> temp_grad_accum,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> grad_accum_counter,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms,
    const int32_t max_vecs_per_thread,
    {%- if is_index_select %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> grad_offsets,
    const bool permute_output_dim_0_1
    {%- else %}
    {{ args.split_kernel_args | replace_pta_namespace() | join(",\n    ") }}
    {%- endif %}
) {
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
  const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
  const unsigned int shfl_sync_mask = 0xffffffffu;
#endif
  constexpr int VEC_WIDTH = 4;
  constexpr auto kIsInt8 = std::is_same<emb_t, uint8_t>::value;
  int32_t T = weights_offsets.size(0);
  const int32_t num_long_runs = num_long_run_ids[0];
  const int32_t warp_id = threadIdx.y;
  const int32_t lane_id = threadIdx.x;

  // Copy value to max_vecs to make max_vecs_per_thread known at compile time
  // when kUseVecBlocking == false
  const int32_t max_vecs =
      kUseVecBlocking ? max_vecs_per_thread : kFixedMaxVecsPerThread;
  struct SharedMemory<Vec4TAcc<cache_t>> smem;
  auto* smem_grad_sum =
      smem.getPointer() + warp_id * max_vecs * kThreadGroupSize;

  for (int32_t long_run_id = blockIdx.x; long_run_id < num_long_runs; long_run_id += gridDim.x) {
        // The first thread block in the really long run has run_id in long_run_ids
        // and the rest have the negative of its offset (see find_long_segments kernel).
        int32_t cta_rank_on_current_run = 0;
        int32_t current_run_id = long_run_ids[long_run_id];
        if (current_run_id < 0) {
            cta_rank_on_current_run = -long_run_ids[long_run_id];
            current_run_id = long_run_ids[long_run_id - cta_rank_on_current_run];
        }
        const int32_t run_length =
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1] -
            sorted_linear_indices_cumulative_run_lengths[current_run_id];
        // This computation must agree with how we compute num_ctas_for_run in
        // find_long_segments kernel!
        const int32_t num_ctas_on_current_run =
            use_deterministic_algorithms ? 1 : div_round_up(run_length, max_segment_length_per_cta);


        const int64_t linear_index = sorted_linear_indices_run[current_run_id];
        const int32_t segment_start =
            sorted_linear_indices_cumulative_run_lengths[current_run_id] +
            cta_rank_on_current_run * max_segment_length_per_cta;
        const int32_t segment_end = std::min(
            use_deterministic_algorithms ? INT_MAX : segment_start + max_segment_length_per_cta,
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1]);
        const int32_t SL = segment_end - segment_start;

        // Note that with shared embedding tables we can have multiple tables
        // (i.e. different values of `t` sharing the same segment).
        {%- if not nobag %}
        const auto info_0 = reinterpret_cast<const uint32_t*>(&sorted_infos[0])[segment_start];
        const auto t_0 = info_0 >> info_B_num_bits;
        {%- else %}
        const auto info_0 = sorted_infos[segment_start];
        int32_t t_0 = info_0 % T;
        {%- endif %}

        int64_t hash_size = hash_size_cumsum[t_0];
        {%- if not nobag or is_index_select %}
        const int32_t D_start_t0 = D_offsets[t_0];
        // D can be hoisted here because D is the same if features share the
        // same table, but D_start is different
        const int32_t D = D_offsets[t_0 + 1] - D_start_t0;
        {%- if is_index_select %}
        // grad_offset can be hoisted here for batch_index_select because it
        // does not allow multiple features to share a single embedding table
        const auto grad_offset = permute_output_dim_0_1 ? D_start_t0 : grad_offsets[t_0];
        const auto grad_stride = permute_output_dim_0_1 ? D_offsets[T] : D;
        {%- endif %}
        {%- endif %}
        int64_t idx = linear_index - hash_size;

        const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
        const int32_t sl_start = SL_per_warp * warp_id;
        const int32_t sl_end = min(SL_per_warp * (warp_id + 1), SL);

        // Accumulate gradients (compute grad_sum)
        Vec4TAcc<cache_t> grad_sum[kFixedMaxVecsPerThread];
        constexpr int32_t kGroupVecWidth = kThreadGroupSize * VEC_WIDTH;
        const int32_t num_vecs = (D + kGroupVecWidth - 1) / kGroupVecWidth;

        compute_grad_sum_{{ kdesc }}<
          grad_t,
          cache_t,
          kFixedMaxVecsPerThread,
          kThreadGroupSize,
          VEC_WIDTH,
          kUseVecBlocking>(
            grad_sum,
            smem_grad_sum,
            grad_output,
            {%- if not nobag or is_index_select %}
            D_offsets,
            {%- endif %}
            D,
            T,
            sorted_infos,
            {%- if weighted %}
            sorted_indice_weights,
            {%- endif %}
            {%- if not nobag and vbe %}
            B_offsets,
            row_output_offsets,
            {%- endif %}
            {%- if is_index_select %}
            grad_offset,
            grad_stride,
            {%- endif %}
            {%- if not nobag %}
            info_B_num_bits,
            info_B_mask,
            {%- endif %}
            segment_start,
            sl_start,
            sl_end,
            shfl_sync_mask,
            num_vecs
        );
        // Do shared memory reduction only if we used multiple warps.
        if (SL > SL_per_warp) {
            __syncthreads();

            {{ sync_grad_sums(32) }}
            {{ sync_grad_sums(16) }}
            {{ sync_grad_sums(8) }}
            {{ sync_grad_sums(4) }}

            if (warp_id == 0) {
                {{
                   generate_optimized_grad_sum_loop_access(
                       """
                        {grad_vec} = vec4_acc(
                            smem_grad_sum[d_vec],
                            smem_grad_sum[d_vec + max_vecs * kThreadGroupSize]
                        );
                       """
                   )
                }}
            }
        }

        if (warp_id != 0) {
            continue;
        }

        if (num_ctas_on_current_run > 1) {
            int really_long_run_id = long_run_id_to_really_long_run_ids[long_run_id];
            Vec4TAcc<cache_t> *temp_grad_accum_ptr =
                reinterpret_cast<Vec4TAcc<cache_t>*>(&temp_grad_accum[really_long_run_id][0]);
            {{
                generate_optimized_grad_sum_loop_access(
                    """
                    gpuAtomicAdd(&temp_grad_accum_ptr[d_vec].acc.x, {grad_vec}.acc.x);
                    gpuAtomicAdd(&temp_grad_accum_ptr[d_vec].acc.y, {grad_vec}.acc.y);
                    gpuAtomicAdd(&temp_grad_accum_ptr[d_vec].acc.z, {grad_vec}.acc.z);
                    gpuAtomicAdd(&temp_grad_accum_ptr[d_vec].acc.w, {grad_vec}.acc.w);
                    """
                )
            }}

            int counter;
            if (threadIdx.x == 0) {
                __threadfence();
                counter = gpuAtomicAdd(&grad_accum_counter[really_long_run_id], -1);
            }
            counter = SHFL_SYNC(counter, 0);
            // Only the thread block accumulated the gradient last does the weight update.
            if (counter > 1) {
                continue;
            }
            CUDA_KERNEL_ASSERT(counter == 1 && "Invalid grad_accum_counter. Race condition?");
            {{
                generate_optimized_grad_sum_loop_access(
                    """
                    {grad_vec} = temp_grad_accum_ptr[d_vec];
                    """
                )
            }}
        }

        {%- if not dense and optimizer != "none" %}
        split_{{ optimizer }}_table_update_kernel<
          emb_t,
          cache_t,
          kFixedMaxVecsPerThread,
          kThreadGroupSize,
          VEC_WIDTH,
          kUseVecBlocking>(
              dev_weights,
              uvm_weights,
              lxu_cache_weights,
              weights_placements,
              weights_offsets,
              sorted_lxu_cache_locations,
              grad_sum,
              kUseVecBlocking ? smem_grad_sum : nullptr,
              kIsInt8 ? smem_grad_sum : nullptr,
              stochastic_rounding,
              stochastic_rounding_philox_args,
              current_run_id,
              use_uniq_cache_locations
                  ? (current_run_id - table_unique_indices_offsets[t_0])
                  : segment_start,
              D,
              t_0,
              idx,
              shfl_sync_mask,
              max_vecs,
              {{ args.split_function_arg_names | join(", ") }}
        );
        {%- else %}
        // Write deduplicated gradient to grad_dev_weights gradient is sparse
        // for split_embedding and dense for dense_embedding
        {%- if dense %}
        const int64_t weights_offset = weights_offsets[t_0];
        {%- else %}
        // Compute offset of sparse gradient
        const int64_t weights_offset = current_run_id * max_D;
        idx = 0;
        {%- endif %}
        store_grad_sum<
          emb_t,
          cache_t,
          kFixedMaxVecsPerThread,
          kThreadGroupSize,
          VEC_WIDTH,
          kUseVecBlocking>(
              grad_dev_weights,
              grad_sum,
              kUseVecBlocking ? smem_grad_sum : nullptr,
              D,
              weights_offset,
              idx,
              max_vecs
        );
        {%- endif %}
    } // for each run
}

////////////////////////////////////////////////////////////////////////////////
// Explicit Template Instantiations
////////////////////////////////////////////////////////////////////////////////

/*
    Explicitly instantiate the kernel function template.  The instantiations are
    based on the types enumerated by DISPATCH_EMB_GRAD_CACHE_TYPES macro used in
    embedding_backward_split_template.cu
*/

{%- macro template_instantiation(
      emb_type,
      grad_type,
      cache_type,
      kFixedMaxVecsPerThread,
      kThreadGroupSize,
      kUseVecBlocking
    )
%}
template __global__ __launch_bounds__(kMaxThreads) void
{%- if is_index_select %}
batch_index_select_dim0_codegen_backward_kernel_cta_per_row
{%- else %}
split_embedding{{ ndesc }}_backward_codegen_{{ optimizer }}_{{ wdesc }}{{ vdesc }}_kernel_cta_per_row_1
{%- endif %}
< {{ emb_type }},
  {{ grad_type }},
  {{ cache_type }},
  {{ kFixedMaxVecsPerThread }},
  {{ kThreadGroupSize }},
  {{ kUseVecBlocking }}
> (
    const pta::PackedTensorAccessor64<{{ grad_type }}, {{ "1" if is_index_select else "2" }}, at::RestrictPtrTraits> grad_output,
    {%- if optimizer != "none" %}
    pta::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    pta::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> uvm_weights,
    pta::PackedTensorAccessor64<{{ cache_type }}, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {%- endif %}
    {%- endif %} // if optimizer != "none"
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag or is_index_select %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int64_t D,
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> hash_size_cumsum,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_linear_indices_run,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_cumulative_run_lengths,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_ids,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> num_long_run_ids,
    {%- if not nobag %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- else %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- endif %}
    {%- if not dense %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_lxu_cache_locations,
    const bool use_uniq_cache_locations,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> table_unique_indices_offsets,
    {%- endif %}
    {%- if weighted %}
    const pta::PackedTensorAccessor32<at::acc_type<{{ cache_type }}, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {%- endif %}
    {%- if not dense and optimizer != "none" %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {%- else %}
    pta::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> grad_dev_weights,
    {%- if optimizer == "none" %}
    const int32_t max_D,
    {%- endif %}
    {%- endif %} // if not dense and optimizer != "none"
    {%- if not nobag and vbe %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> B_offsets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> row_output_offsets,
    {%- endif %}
    {%- if not nobag %}
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    {%- endif %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_id_to_really_long_run_ids,
    pta::PackedTensorAccessor32<at::acc_type<{{ cache_type }}, true>, 2, at::RestrictPtrTraits> temp_grad_accum,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> grad_accum_counter,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms,
    const int32_t max_vecs_per_thread,
    {%- if is_index_select %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> grad_offsets,
    const bool permute_output_dim_0_1
    {%- else %}
    {{ args.split_kernel_args_no_defaults | replace_pta_namespace() | join(",\n    ") | replace("cache_t", cache_type) }}
    {%- endif %}
);
{%- endmacro %}

{%- macro bulk_template_instantiations(kFixedMaxVecsPerThread, kThreadGroupSize, kUseVecBlocking) %}
    {%- for grad_type in ['float', 'at::Half', 'at::BFloat16'] %}
    {%- for emb_type in ['float', 'at::Half'] %}
    {%- for cache_type in ['float', 'at::Half'] %}
        {{ template_instantiation(
            emb_type,
            grad_type,
            cache_type,
            kFixedMaxVecsPerThread,
            kThreadGroupSize,
            kUseVecBlocking)
         }}
    {%- endfor %}
    {%- endfor %}
    {%- endfor %}
{%- endmacro %}


{%- if is_experimental_optimizer %}

{{
  bulk_template_instantiations(
    fixed_max_vecs_per_thread["backward"],
    'kWarpSize',
    'true'
  )
}}

{%- else %}

{%- macro instantiate_templates(use_subwarp_shuffle) %}
{%- for (kFixedMaxVecsPerThread, kThreadGroupSize, kUseVecBlocking)
    in get_max_vecs_template_configs(
        items_per_warp,
        fixed_max_vecs_per_thread["backward"],
        use_subwarp_shuffle,
        use_vec_blocking=True,
    )
%}
    {{
      bulk_template_instantiations(
        kFixedMaxVecsPerThread,
        kThreadGroupSize,
        kUseVecBlocking,
      )
    }}
{%- endfor %}
{%- endmacro %}

////////////////////////////////////////////////////////////////////////////////
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
////////////////////////////////////////////////////////////////////////////////

{#- /*
    Explicitly instantiate kernels for the FBGEMM_USE_SUBWARP_SHUFFLE case

    Please see get_max_vecs_template_configs in
    codegen/embedding_common_code_generator.py for more details
*/ #}

{{ instantiate_templates(use_subwarp_shuffle=True) }}

////////////////////////////////////////////////////////////////////////////////
#else
////////////////////////////////////////////////////////////////////////////////

{#- /*
    Explicitly instantiate kernels for the non-FBGEMM_USE_SUBWARP_SHUFFLE case

    Please see get_max_vecs_template_configs in
    codegen/embedding_common_code_generator.py for more details
*/ #}

{{ instantiate_templates(use_subwarp_shuffle=False) }}

////////////////////////////////////////////////////////////////////////////////
#endif
////////////////////////////////////////////////////////////////////////////////

{%- endif %}
  // clang-format on
