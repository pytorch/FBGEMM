#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// clang-format off
{%- set wdesc = "weighted" if weighted else "unweighted" %}
#include "fbgemm_gpu/embedding_backward_template_helpers.cuh"
#include "fbgemm_gpu/split_embeddings_utils.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    size_t kMaxVecsPerThread,
    int32_t kThreadGroupSize >
__global__ __launch_bounds__(kMaxThreads) void
split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1(
    const at::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits> grad_output,
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {%- endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int32_t B,
    int64_t D,
    {%- endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        hash_size_cumsum,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_run,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_cumulative_run_lengths,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        long_run_ids,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        num_long_run_ids,
    {%- if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- else %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- endif %}
    {%- if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_lxu_cache_locations,
    {%- endif %}
    {%- if weighted %}
    const at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {%- endif %}
    {%- if not dense %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {%- else %}
    at::PackedTensorAccessor64<cache_t, 1, at::RestrictPtrTraits> grad_dev_weights,
    {%- endif %}
    {%- if not nobag %}
    FixedDivisor fd,
    {%- endif %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_id_to_really_long_run_ids,
    at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 2, at::RestrictPtrTraits> temp_grad_accum,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> grad_accum_counter,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms,
    {{ args.split_kernel_args | join(",\n    ") }}) {
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
  const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
  const unsigned int shfl_sync_mask = 0xffffffffu;
#endif
  constexpr int VEC_WIDTH = 4;
  int32_t T = weights_offsets.size(0);
  {%- if not nobag %}
  const int32_t B = grad_output.size(0);
  {%- endif %}
  const int32_t num_long_runs = num_long_run_ids[0];
  for (int32_t long_run_id = blockIdx.x; long_run_id < num_long_runs; long_run_id += gridDim.x) {
        // The first thread block in the really long run has run_id in long_run_ids
        // and the rest have the negative of its offset (see find_long_segments kernel).
        int32_t cta_rank_on_current_run = 0;
        int32_t current_run_id = long_run_ids[long_run_id];
        if (current_run_id < 0) {
            cta_rank_on_current_run = -long_run_ids[long_run_id];
            current_run_id = long_run_ids[long_run_id - cta_rank_on_current_run];
        }
        const int32_t run_length =
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1] -
            sorted_linear_indices_cumulative_run_lengths[current_run_id];
        // This computation must agree with how we compute num_ctas_for_run in
        // find_long_segments kernel!
        const int32_t num_ctas_on_current_run =
            use_deterministic_algorithms ? 1 : div_round_up(run_length, max_segment_length_per_cta);


        const int64_t linear_index = sorted_linear_indices_run[current_run_id];
        const int32_t segment_start =
            sorted_linear_indices_cumulative_run_lengths[current_run_id] +
            cta_rank_on_current_run * max_segment_length_per_cta;
        const int32_t segment_end = std::min(
            use_deterministic_algorithms ? INT_MAX : segment_start + max_segment_length_per_cta,
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1]);
        const int32_t SL = segment_end - segment_start;
        const int32_t warp_id = threadIdx.y;
        const int32_t lane_id = threadIdx.x;

        // Note that with shared embedding tables we can have multiple tables
        // (i.e. different values of `t` sharing the same segment).
        //
        const auto info_0 = sorted_infos[segment_start];

        {%- if not nobag %}
        int32_t t_0 = fd.Div(info_0); //info_0 / B;
        {%- else %}
        int32_t t_0 = info_0 % T;
        {%- endif %}

        int64_t hash_size = hash_size_cumsum[t_0];
        {%- if not nobag %}
        int32_t D = D_offsets[t_0 + 1] - D_offsets[t_0];
        {%- endif %}
        int64_t idx = linear_index - hash_size;

        const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
        const int32_t sl_start = SL_per_warp * warp_id;
        const int32_t sl_end = min(SL_per_warp * (warp_id + 1), SL);
        Vec4T<at::acc_type<cache_t, true>> grad_sum[kMaxVecsPerThread];
        for (int32_t sl = sl_start; sl < sl_end; sl += kThreadGroupSize) {
            int32_t sl_j = sl + threadIdx.x;
            {%- if not nobag %}
            int32_t b_t = sl_j < sl_end ? sorted_infos[segment_start + sl_j] : 0;
            int32_t b; //= b_t % B;
            int32_t t; //= b_t / B;
            fd.DivMod(b_t, &t, &b);
            int32_t D_start = sl_j < sl_end ? D_offsets[t] : 0;
            {%- else %}
            int64_t l_t = sl_j < sl_end ? sorted_infos[segment_start + sl_j] : 0;
            int32_t l = l_t / T;
            {%- endif %}
            {%- if weighted %}
            at::acc_type<cache_t, true> idx_weight = sl_j < sl_end ? sorted_indice_weights[segment_start + sl_j] : 0.0;
            {%- endif %}
            for (int32_t j = 0; j < kThreadGroupSize && sl + j < sl_end; ++j) {
                {%- if not nobag %}
                int32_t b_j = SHFL_SYNC(b, j);
                int32_t D_start_j = SHFL_SYNC(D_start, j);
                {%- else %}
                int32_t l_j = SHFL_SYNC(l, j);
                {%- endif %}

                {%- if weighted %}
                at::acc_type<cache_t, true> idx_weight_j = SHFL_SYNC(idx_weight, j);
                {%- endif %}

        #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    {%- if not nobag %}
                    Vec4T<at::acc_type<grad_t, true>> grad_out_vec(
                        &grad_output[b_j][0] + D_start_j + d);
                    {%- else %}
                    Vec4T<at::acc_type<grad_t, true>> grad_out_vec(&grad_output[l_j][d]);
                    {%- endif %}
                    {%- if weighted %}
                    grad_sum[i].fma_(grad_out_vec, idx_weight_j);
                    {%- else %}
                    grad_sum[i].add_(grad_out_vec);
                    {%- endif %}
                }
            }
        }
        // do shared memory reduction only if we used multiple warps.
        if (SL > SL_per_warp) {
            struct SharedMemory<Vec4T<at::acc_type<cache_t, true>>> smem;
            Vec4T<at::acc_type<cache_t, true>>* shared_grad_sums = smem.getPointer();

    #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
            shared_grad_sums
                [lane_id + i * kThreadGroupSize +
                warp_id * kMaxVecsPerThread * kThreadGroupSize] = grad_sum[i];
            }
            __syncthreads();
            if (blockDim.y >= 32) {
            if (warp_id < 16) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 16) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 16) {
            if (warp_id < 8) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 8) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 8) {
            if (warp_id < 4) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 4) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 4) {
            if (warp_id < 2) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 2) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (warp_id == 0) {
    #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                grad_sum[i] = vec4_acc(
                    shared_grad_sums
                        [lane_id + i * kThreadGroupSize +
                        warp_id * kMaxVecsPerThread * kThreadGroupSize],
                    shared_grad_sums
                        [lane_id + i * kThreadGroupSize +
                        (warp_id + 1) * kMaxVecsPerThread * kThreadGroupSize]);
            }
            }
        }

        if (warp_id != 0) {
            continue;
        }

        if (num_ctas_on_current_run > 1) {
            int really_long_run_id = long_run_id_to_really_long_run_ids[long_run_id];
            Vec4T<at::acc_type<cache_t, true>> *temp_grad_accum_ptr =
                reinterpret_cast<Vec4T<at::acc_type<cache_t, true>>*>(&temp_grad_accum[really_long_run_id][0]);
#pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.x, grad_sum[i].acc.x);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.y, grad_sum[i].acc.y);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.z, grad_sum[i].acc.z);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.w, grad_sum[i].acc.w);
            }
            int counter;
            if (threadIdx.x == 0) {
                __threadfence();
                counter = gpuAtomicAdd(&grad_accum_counter[really_long_run_id], -1);
            }
            counter = SHFL_SYNC(counter, 0);
            // Only the thread block accumulated the gradient last does the weight update.
            if (counter > 1) {
                continue;
            }
            CUDA_KERNEL_ASSERT(counter == 1 && "Invalid grad_accum_counter. Race condition?");
#pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                grad_sum[i] = temp_grad_accum_ptr[lane_id + i * kThreadGroupSize];
            }
        }

        int64_t weights_offset = weights_offsets[t_0];
        {%- if not dense %}
        emb_t* __restrict__ weights{nullptr};
        cache_t* __restrict__ cache_weights{nullptr};
        int32_t D_emb = D;
        if (std::is_same<emb_t, uint8_t>::value) {
            D_emb += kINT8QparamsBytes;
        }
        const auto weights_placement = static_cast<PlacementType>(weights_placements[t_0]);
        if (weights_placement == PlacementType::DEVICE) {
            weights = &dev_weights[weights_offset + idx * D_emb];
        } else {
            weights = &uvm_weights[weights_offset + idx * D_emb];
        }
        if (weights_placement == PlacementType::MANAGED_CACHING) {
            int32_t cache_idx = sorted_lxu_cache_locations[segment_start];
            if (cache_idx != kCacheLocationMissing) {
                cache_weights = &lxu_cache_weights[cache_idx][0];
            }
        }
        {%- for tensor in args.split_tensors %}
        at::acc_type<cache_t, true>* __restrict__ {{ tensor }};
        const auto {{ tensor }}_placement = static_cast<PlacementType>({{ tensor }}_placements[t_0]);
        int64_t {{ tensor }}_offset = {{ tensor }}_offsets[t_0];
        if ({{ tensor }}_placement == PlacementType::DEVICE) {
            {{ tensor }} = &{{ tensor }}_dev[{{ tensor }}_offset];
        } else {
            {{ tensor }} = &{{ tensor }}_uvm[{{ tensor }}_offset];
        }
        {%- endfor %}


        struct SharedMemory<Vec4T<at::acc_type<cache_t, true>>> weight_update_buffer;
        Vec4T<at::acc_type<cache_t, true>>* shared_weight_update_row = weight_update_buffer.getPointer();

        auto weight_row_template = WeightRow<emb_t, cache_t, at::acc_type<cache_t, true>>(weights, cache_weights, D, nullptr);
        if (!std::is_same<emb_t, float>::value && stochastic_rounding) {
            StochasticRoundingRNGState state;
            // different for every *run* and every *thread*.
            auto stochastic_rounding_seeds =
                at::cuda::philox::unpack(stochastic_rounding_philox_args);
            stochastic_rounding_init(
                std::get<0>(stochastic_rounding_seeds) ^
                    std::get<1>(stochastic_rounding_seeds),
                threadIdx.x + current_run_id * blockDim.x,
                &state);
            weight_row_template.set_stoc_state(&state);
        }

        float2 qparams_template;
        if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
            qparams_template = weight_row_template.load_qparams();
        }

        {{ split_precomputation }}

        float2 qparams_new;
#pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            Vec4T<at::acc_type<cache_t, true>> weight_new = weight_row_template.load(d, qparams_template);
            auto& grad = grad_sum[i];
            {{ split_weight_update }}
            if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
                shared_weight_update_row[lane_id + i * kThreadGroupSize] = weight_new;
            } else {
                weight_row_template.store(weight_new, d, qparams_new); // qparams_new not used if embedding is not int8
            }
        }
        if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
            // calculate qparams from updated weight row
            qparams_new = thrust_find_qparams<at::acc_type<cache_t, true>>(shared_weight_update_row, D);
            weight_row_template.store_qparams(qparams_new);

#pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                weight_row_template.store(shared_weight_update_row[lane_id + i * kThreadGroupSize], d, qparams_new);
            }
        }
        {%- else %}
#pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            auto& grad = grad_sum[i];
            grad.store(&grad_dev_weights[weights_offset + idx * D + d]);
        }
        {%- endif %}
    } // for each run
}

/*
    Explicitly instantiate the kernel function template.  The instantiations are
    based on the types enumerated by DISPATCH_EMB_GRAD_CACHE_TYPES macro used in
    embedding_backward_split_template.cu
*/

{%- for grad_type in ['float', 'at::Half'] %}
{%- for emb_type in ['uint8_t', 'float', 'at::Half'] %}
{%- for cache_type in ['float', 'at::Half'] %}

////////////////////////////////////////////////////////////////////////////////
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
////////////////////////////////////////////////////////////////////////////////

{#- /*
    Compute the Cartesian product of (kMaxVecsPerThread, kThreadGroupSize)
    in the FBGEMM_USE_SUBWARP_SHUFFLE case

    constexpr int kMaxVecsPerThread = std::max({{ kMaxElemPerThread }} / 4, 1);
    constexpr int kThreadGroupSize = kWarpSize / std::max(4 / {{ kMaxElemPerThread }}, 1);

    This is needed to compute the unique tuples to use for explicit instantiation,
    so that we can avoid duplicate template instantiations.
*/ #}
{%- set tuples = [] %}
{%- for kMaxElemPerThread in range(1, max_embedding_dim // (items_per_warp // 4) + 1) %}
{%- if kMaxElemPerThread in [1, 2] or kMaxElemPerThread % 4 == 0 %}
    {%- set t0 = [ (kMaxElemPerThread // 4), 1 ] | max %}
    {%- set t1 = [ 4 // kMaxElemPerThread, 1] | max %}
    {%- set temp = tuples.append((t0, "(kWarpSize / " ~ t1 ~ ")")) %}
{%- endif %}
{%- endfor %}

{#- /* Enumerate over the unique tuples */ #}
{%- for (kMaxVecsPerThread, kThreadGroupSize) in tuples | unique %}

template __global__ __launch_bounds__(kMaxThreads)
void split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1
< {{ emb_type }},
  {{ grad_type }},
  {{ cache_type }},
  {{ kMaxVecsPerThread }},
  {{ kThreadGroupSize }}
> (
    const at::PackedTensorAccessor64<{{ grad_type  }}, 2, at::RestrictPtrTraits> grad_output,
    at::PackedTensorAccessor64<{{ emb_type  }}, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    at::PackedTensorAccessor64<{{ emb_type  }}, 1, at::RestrictPtrTraits> uvm_weights,
    at::PackedTensorAccessor64<{{ cache_type  }}, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int32_t B,
    int64_t D,
    {%- endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> hash_size_cumsum,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_linear_indices_run,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_cumulative_run_lengths,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_ids,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> num_long_run_ids,
    {%- if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- else %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- endif %}
    {%- if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_lxu_cache_locations,
    {%- endif %}
    {%- if weighted %}
    const at::PackedTensorAccessor32<at::acc_type<{{ cache_type  }}, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {%- endif %}
    {%- if not dense %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {%- else %}
    at::PackedTensorAccessor64<{{ cache_type  }}, 1, at::RestrictPtrTraits> grad_dev_weights,
    {%- endif %}
    {%- if not nobag %}
    FixedDivisor fd,
    {%- endif %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_id_to_really_long_run_ids,
    at::PackedTensorAccessor32<at::acc_type<{{ cache_type  }}, true>, 2, at::RestrictPtrTraits> temp_grad_accum,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> grad_accum_counter,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms,
    {{ args.split_kernel_args_no_defaults | join(",\n    ") | replace("cache_t", cache_type) }});

{%- endfor %}

////////////////////////////////////////////////////////////////////////////////
#else
////////////////////////////////////////////////////////////////////////////////

{#- /*
    Compute the Cartesian product of (kMaxVecsPerThread, kThreadGroupSize)
    in the non-FBGEMM_USE_SUBWARP_SHUFFLE case

    constexpr int kMaxVecsPerThread = std::max({{ kMaxElemPerThread }} / 4, 1);
    constexpr int kThreadGroupSize = kWarpSize;
*/ #}
{%- set tuples = [] %}
{%- for kMaxElemPerThread in range(1, max_embedding_dim // (items_per_warp // 4) + 1) %}
{%- if kMaxElemPerThread in [1, 2] or kMaxElemPerThread % 4 == 0 %}
    {%- set t0 = [ (kMaxElemPerThread // 4), 1 ] | max %}
    {%- set t1 = [ 4 // kMaxElemPerThread, 1] | max %}
    {%- set temp = tuples.append((t0, "kWarpSize")) %}
{%- endif %}
{%- endfor %}

{#- /* Enumerate over the unique tuples */ #}
{%- for (kMaxVecsPerThread, kThreadGroupSize) in tuples | unique %}

template __global__ __launch_bounds__(kMaxThreads)
void split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1
< {{ emb_type }},
  {{ grad_type }},
  {{ cache_type }},
  {{ kMaxVecsPerThread }},
  {{ kThreadGroupSize }}
> (
    const at::PackedTensorAccessor64<{{ grad_type  }}, 2, at::RestrictPtrTraits> grad_output,
    at::PackedTensorAccessor64<{{ emb_type  }}, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    at::PackedTensorAccessor64<{{ emb_type  }}, 1, at::RestrictPtrTraits> uvm_weights,
    at::PackedTensorAccessor64<{{ cache_type  }}, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int32_t B,
    int64_t D,
    {%- endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> hash_size_cumsum,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_linear_indices_run,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_cumulative_run_lengths,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_ids,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> num_long_run_ids,
    {%- if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- else %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {%- endif %}
    {%- if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_lxu_cache_locations,
    {%- endif %}
    {%- if weighted %}
    const at::PackedTensorAccessor32<at::acc_type<{{ cache_type  }}, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {%- endif %}
    {%- if not dense %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {%- else %}
    at::PackedTensorAccessor64<{{ cache_type  }}, 1, at::RestrictPtrTraits> grad_dev_weights,
    {%- endif %}
    {%- if not nobag %}
    FixedDivisor fd,
    {%- endif %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_id_to_really_long_run_ids,
    at::PackedTensorAccessor32<at::acc_type<{{ cache_type  }}, true>, 2, at::RestrictPtrTraits> temp_grad_accum,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> grad_accum_counter,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms,
    {{ args.split_kernel_args_no_defaults | join(",\n    ") | replace("cache_t", cache_type) }});

{%- endfor %}

////////////////////////////////////////////////////////////////////////////////
#endif
////////////////////////////////////////////////////////////////////////////////

{%- endfor %}
{%- endfor %}
{%- endfor %}

        // clang-format on
