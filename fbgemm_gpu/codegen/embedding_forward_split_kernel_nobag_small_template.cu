#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

{#
// @lint-ignore LINTIGNORE
// @lint-ignore-every CLANGFORMAT
// clang-format off
// Note: clang-format off doesn't work with this templaterized code,
// so we need to keep lint-ignore-every.
// See https://fburl.com/dw9ljh4h
#}

{%- set wdesc =  "weighted" if weighted else "unweighted" %}
#include "codegen/embedding_forward_template_helpers.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

template <
    typename emb_t,
    typename cache_t,
    typename output_t,
    typename index_t,
    size_t kThreadGroupSize
    >
__launch_bounds__(kForwardMaxThreads) __global__ void
{{ "dense" if dense else "split" }}_embedding_nobag_codegen_forward_unweighted_small_kernel(
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    int64_t D,
    FixedDivisor fd_B,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    {%- if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> lxu_cache_locations,
    {%- endif %}
    at::PackedTensorAccessor64<output_t, 2, at::RestrictPtrTraits> output // [B][total_D],
    ) {
    int32_t T = weights_offsets.size(0);
    int32_t B = (offsets.size(0) - 1) / T;
    int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
    if (b_t >= B * T) {
        return;
    }
    int32_t t;
    int32_t b;
    fd_B.DivMod(b_t, &t, &b);
    int64_t weights_offset = weights_offsets[t];
    index_t indices_start = offsets[t * B + b];
    index_t indices_end = offsets[t * B + b + 1];
    int32_t L = indices_end - indices_start;
    const emb_t* __restrict__ weights;
    {%- if not dense %}
    const auto placement = static_cast<PlacementType>(weights_placements[t]);
    if (placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset];
    } else {
        weights = &uvm_weights[weights_offset];
    }
    {%- else %}
    weights = &dev_weights[weights_offset];
    {%- endif %}

    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }

    const int32_t group_start = threadIdx.x / kThreadGroupSize * kThreadGroupSize;
    const int32_t group_end = group_start + kThreadGroupSize;
    const int32_t d = threadIdx.x % kThreadGroupSize * 4;

    for (int32_t l_start = 0; l_start < L; l_start += kWarpSize) {
        int32_t l = l_start + threadIdx.x;
        int64_t idx = l < L ? indices[indices_start + l] : 0;
        {%- if not dense %}
        int32_t cache_idx = (placement == PlacementType::MANAGED_CACHING && l < L) ? lxu_cache_locations[indices_start + l] : 0;
        {%- endif %}
        for (auto j = group_start; j < group_end && l_start + j < L; ++j) {
            int64_t idx_j = shfl_sync(idx, j);
            int64_t output_j = indices_start + l_start + j;
            {%- if not dense %}
            int32_t cache_idx_j = shfl_sync(cache_idx, j);
            {%- endif %}

            {%- if not dense %}

            // assume cache is fp16/fp32 which doesn't require qparams
            float2 qparams_cache = make_float2(0.0f, 0.0f);

            {%- endif %}
            auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
                const_cast<emb_t*>(&weights[idx_j * D_emb]),
                nullptr,
                D,
                nullptr);
            float2 qparams_emb;
            if (std::is_same<emb_t, uint8_t>::value) {
                qparams_emb = weight_row_emb.load_qparams();
            }

            if (d < D) {
                {%- if not dense %}
                if (placement == PlacementType::MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                    auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
                        const_cast<emb_t*>(&weights[idx_j * D_emb]),
                        const_cast<cache_t*>(&lxu_cache_weights[cache_idx_j][0]),
                        D,
                        nullptr);
                    Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                    weight.store(&output[output_j][d]);
                } else {
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    weight.store(&output[output_j][d]);
                }
                {%- else %}
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    weight.store(&output[output_j][d]);
                {%- endif %}
            }
        }
    }
}

/*
    Explicitly instantiate the kernel function template.  The instantiations are
    based on the types enumerated by DISPATCH_EMB_GRAD_CACHE_TYPES macro used in
    embedding_forward_split_template.cu
*/

{%- for output_type in ['uint8_t', 'at::Half', 'float'] %}
{%- for emb_type in ['uint8_t', 'float', 'at::Half'] %}
{%- for cache_type in ['float', 'at::Half'] %}
{%- for kEmbeddingSize in [4, 8, 16, 32] %}
{%- set index_type = 'int64_t' %}

template __launch_bounds__(kForwardMaxThreads) __global__
void {{ "dense" if dense else "split" }}_embedding_nobag_codegen_forward_unweighted_small_kernel
<
  {{ emb_type }},
  {{ cache_type }},
  {{ output_type }},
  {{ index_type }},
  {{ kEmbeddingSize // 4 }}
> (
    const at::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    const at::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor64<{{ cache_type }}, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    int64_t D,
    FixedDivisor fd_B,
    const at::PackedTensorAccessor32<{{ index_type }}, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<{{ index_type }}, 1, at::RestrictPtrTraits> offsets,
    {%- if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> lxu_cache_locations,
    {%- endif %}
    at::PackedTensorAccessor64<{{ output_type }}, 2, at::RestrictPtrTraits> output);

{%- endfor %}
{%- endfor %}
{%- endfor %}
{%- endfor %}
