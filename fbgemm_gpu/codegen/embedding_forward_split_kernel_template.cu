#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

{#-
// @lint-ignore LINTIGNORE
// @lint-ignore-every CLANGFORMAT
// clang-format off
// Note: clang-format off doesn't work with this templaterized code,
// so we need to keep lint-ignore-every.
// See https://fburl.com/dw9ljh4h
#}

{%- set wdesc =  "weighted" if weighted else "unweighted" %}
{%- set vbe_desc = "_vbe" if vbe else "" %}
#include "codegen/embedding_forward_template_helpers.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

template <
    typename emb_t,
    typename cache_t,
    typename output_t,
    {%- if not dense %}
    bool use_lxu_cache,
    {%- endif %}
    typename index_t,
    {%- if not nobag %}
    size_t kMaxVecsPerThread,
    {%- endif %}
    size_t kThreadGroupSize >
__launch_bounds__(kForwardMaxThreads) __global__
void {{ "dense" if dense else "split" }}_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}{{ vbe_desc }}_kernel(
    const pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    const pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    const pta::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int64_t D,
    {%- endif %}
    {%- if vbe %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> output_offsets,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> b_t_map,
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    {%- else %}
    FixedDivisor fd_B,
    {%- endif %}
    const pta::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const pta::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    {%- if not nobag %}
    int64_t pooling_mode,
    {%- endif %}
    {%- if weighted %}
    pta::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits> indice_weights,
    {%- endif %}
    {%- if not dense %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> lxu_cache_locations,
    {%- endif %}
    pta::PackedTensorAccessor64<output_t, 2, at::RestrictPtrTraits> output // [B][total_D]
    ) {

// shfl_sync_mask is implicitly used by SHFL_SYNC
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
    const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
    const unsigned int shfl_sync_mask = 0xffffffffu;
#endif

    // Elements are processed 4 at a time through fbgemm_gpu::Vec4 (CUDA float4, 16 bytes)
    constexpr int VEC_WIDTH = 4;

    // Determine the linearized warp ID, and exit early if needed
    int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
    if (b_t >= offsets.size(0) - 1) {
        return;
    }

    // Determine the Table and Training Example IDs
    int32_t t;  // Table ID
    int32_t b;  // Training Example ID
    {%- if vbe %}
    const auto info = reinterpret_cast<const uint32_t*>(&b_t_map[b_t])[0];
    reinterpret_cast<uint32_t*>(&t)[0] = info >> info_B_num_bits;
    reinterpret_cast<uint32_t*>(&b)[0] = info & info_B_mask;
    {%- else %}
    fd_B.DivMod(b_t, &t, &b);
    {%- endif %}

    // From the Table ID, fetch its weight tensor offset, locate that position
    // in the input weights tensor, and set the weights table pointer
    const emb_t* __restrict__ weights;
    int64_t weights_offset = weights_offsets[t];
    {%- if not dense %}
    const auto placement = static_cast<PlacementType>(weights_placements[t]);
    if (placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset];
    } else {
        weights = &uvm_weights[weights_offset];
    }
    {%- else %}
    weights = &dev_weights[weights_offset];
    {%- endif %}

    // Get total number of tables
    int32_t T = weights_offsets.size(0);

    // Determine the number of indices (pooling factor) to look up within the bag
    index_t indices_start = offsets[b_t];
    index_t indices_end = offsets[b_t + 1];
    int32_t L = indices_end - indices_start;

    // Get the offsets of the embedding dimensions of the tables and determine D
    {%- if not nobag %}
    int32_t D_start = D_offsets[t];
    int32_t D_end = D_offsets[t + 1];
    int32_t D = D_end - D_start;
    {%- endif %}

    // D is computed in the bag case or provided as function arg in the nobag case
    // (nobag only supports the case where the embedding dimensions are the same for all tables)
    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }

    {%- if not nobag %}
    // Determine if we're doing mean pooling
    const bool mean_pooling = static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN;

    // Compute 1/L - this is used to compute the mean later on
    const float inv_L = (mean_pooling && L != 0) ? static_cast<float>(1.0) / L: static_cast<float>(1.0);

    // Set up the accumulator buffer
    Vec4T<cache_t> accumulators[kMaxVecsPerThread];
    {%- endif %}

    // Iterate over each kThreadGroupSize-sized subset of L indices in the bag
    for (int32_t l_start = 0; l_start < L; l_start += kThreadGroupSize) {
        // Determine the L index that this thread will load data from in cooperative load
        int32_t l = l_start + threadIdx.x;
        // Cooperatively load the indices
        int64_t idx = l < L ? indices[indices_start + l] : 0;

        {%- if not dense %}
        // Cooperatively load the cache's indices
        int32_t cache_idx = (use_lxu_cache && placement == PlacementType::MANAGED_CACHING && l < L) ? lxu_cache_locations[indices_start + l] : 0;
        {%- endif %}

        {%- if weighted %}
        // Cooperatively load the positional weight indices
        at::acc_type<cache_t, true> idx_weight = l < L ? indice_weights[indices_start + l] : 0;
        {%- endif %}

        // Iterate over kThreadGroupSize indices
        for (auto j = 0; j < kThreadGroupSize && l_start + j < L; ++j) {
            // Load index from thread j in the group
            int64_t idx_j = SHFL_SYNC(idx, j);

            {%- if nobag %}
            int64_t output_j = indices_start + l_start + j;
            {%- endif %}

            {%- if not dense %}
            // Load cache's index from thread j in the group
            int32_t cache_idx_j = use_lxu_cache ? SHFL_SYNC(cache_idx, j) : 0;
            {%- endif %}

            {%- if weighted %}
            // Load positional weight index from thread j in the group
            at::acc_type<cache_t, true> idx_weight_j = SHFL_SYNC(idx_weight, j);
            {%- endif %}

            {%- if not dense %}
            // use_lxu_cache is a compile time condition
            if (use_lxu_cache && placement == PlacementType::MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                // Load the embedding table row from cache to the buffer
                auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
                    const_cast<emb_t*>(&weights[idx_j * D_emb]),
                    const_cast<cache_t*>(&lxu_cache_weights[cache_idx_j][0]),
                    D,
                    nullptr);

                // Assume cache is fp16/fp32 which doesn't require qparams
                float2 qparams_cache = make_float2(0.0f, 0.0f);

                {%- if not nobag %}
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    // Load Vec4 from cache
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);

                    {%- if weighted %}
                    // Accumulate the weight * positional weight
                    accumulators[i].fma_(weight, idx_weight_j);
                    {%- else %}
                    // Accumulate the weight
                    accumulators[i].add_(weight);
                    {%- endif %}
                }
                {%- else %}
                for (int32_t i = 0; i < D; i += kThreadGroupSize * VEC_WIDTH) {
                    int32_t d = i + threadIdx.x * VEC_WIDTH;
                    if (d < D) {
                        // Since there is no pooling, simply copy the weight to output
                        Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                        weight.store(&output[output_j][d]);
                    }
                }
                {%- endif %}

            } else { // else row is not in cache
            {%- endif %}
                // Load the embedding table row from memory to the buffer
                auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
                    const_cast<emb_t*>(&weights[idx_j * D_emb]),
                    nullptr,
                    D,
                    nullptr);

                // Load the two quantization params (scale and bias) from the end of the embedding table row (2 floats)
                [[maybe_unused]] float2 qparams_emb;
                if (std::is_same<emb_t, uint8_t>::value) {
                    qparams_emb = weight_row_emb.load_qparams();
                }

                {%- if not nobag %}
                // Iterate over the row of elements in the weights table, in 4-element strides between adjacent threads
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    // Figure out the position in the embedding table row to load
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;

                    // Fused load-and-dequantize from the buffer
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);

                    {%- if weighted %}
                    // Accumulate the weight * positional weight
                    accumulators[i].fma_(weight, idx_weight_j);
                    {%- else %}
                    // Accumulate the weight
                    accumulators[i].add_(weight);
                    {%- endif %}
                }
                {%- else %}
                for (int32_t i = 0; i < D; i += kThreadGroupSize * VEC_WIDTH) {
                    int32_t d = i + threadIdx.x * VEC_WIDTH;
                    if (d < D) {
                        // Since there is no pooling, simply copy the weight to output
                        Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                        weight.store(&output[output_j][d]);
                    }
                }
                {%- endif %}
            {%- if not dense %}
            } // else row is not in cache
            {%- endif %}
        }
    }

    {%- if not nobag %}
    // If weight type is FP32/16
    if constexpr (!std::is_same_v<output_t, uint8_t>) {
        {%- if vbe %}
        output_t* output_ = &output[0][output_offsets[b_t]];
        {%- else %}
        output_t* output_ = &output[b][D_start];
        {%- endif %}

        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
             i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
             ++i) {
            // Compute the mean (for mean pooling) and store directly to memory as is
            accumulators[i].mul_(inv_L);
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            accumulators[i].store(output_ + d);
        }

    } else {
        // Else weight type is INT8
        float thread_local_min = std::numeric_limits<float>::max();
        float thread_local_max = std::numeric_limits<float>::lowest();
        float2 qparams;

        // Accumulate the min and max values
        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            // Simultaneously multiply by 1/L to compute the mean
            accumulators[i].mul_(inv_L);
            thread_local_max = max(thread_local_max, vec4_max(accumulators[i]));
            thread_local_min = min(thread_local_max, vec4_min(accumulators[i]));
        }

        // Construct the quantization parameters from the min and max values
        qparams = warp_find_qparams(thread_local_min, thread_local_max);
        int output_D_start = D_start + t * 8;
        int output_D_end = output_D_start + D;

        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            // Fused quantize-and-store to memory
            nearest_rounding_vector<output_t, cache_t>(&output[b][output_D_start + d], accumulators[i], qparams);
        }

        // Write out the qparams to the front of the embedding table row
        if (threadIdx.x == 0) {
            store_qparams_to_row(&output[b][output_D_end], qparams);
        }

    }
    {%- endif %}
}


////////////////////////////////////////////////////////////////////////////////
// Explicit Template Instantiations
////////////////////////////////////////////////////////////////////////////////

/*
    Explicitly instantiate the kernel function template.  The instantiations are
    based on the types enumerated by DISPATCH_EMB_CACHE_TYPES macro used in
    embedding_forward_split_template.cu
*/

{%- macro template_instantiation(emb_type, cache_type, output_type, use_cache, kMaxVecsPerThread, kThreadGroupSize) %}
template __launch_bounds__(kForwardMaxThreads) __global__
void {{ "dense" if dense else "split" }}_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}{{ vbe_desc }}_kernel
<
    {{ emb_type }},
    {{ cache_type }},
    {{ output_type }},
    {%- if not dense %}
    {{ use_cache }},
    {%- endif %}
    int64_t,
    {%- if not nobag %}
    {{- kMaxVecsPerThread }},
    {%- endif %}
    {{ kThreadGroupSize }}
> (
    const pta::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> dev_weights,
    {%- if not dense %}
    const pta::PackedTensorAccessor64<{{ emb_type }}, 1, at::RestrictPtrTraits> uvm_weights,
    const pta::PackedTensorAccessor64<{{ cache_type }}, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {%- if not nobag %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {%- else %}
    int64_t D,
    {%- endif %}
    {%- if vbe %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> output_offsets,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> b_t_map,
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    {%- else %}
    FixedDivisor fd_B,
    {%- endif %}
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> indices,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> offsets,
    {%- if not nobag %}
    int64_t pooling_mode,
    {%- endif %}
    {%- if weighted %}
    pta::PackedTensorAccessor32<at::acc_type<{{ cache_type }}, true>, 1, at::RestrictPtrTraits> indice_weights,
    {%- endif %}
    {%- if not dense %}
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> lxu_cache_locations,
    {%- endif %}
    pta::PackedTensorAccessor64<{{ output_type }}, 2, at::RestrictPtrTraits> output);
{%- endmacro %}

{%- macro bulk_template_instantiations(use_cache, kMaxVecsPerThread, kThreadGroupSize) %}
    {%- for emb_type in ['uint8_t', 'float', 'at::Half'] %}
    {%- for cache_type in ['float', 'at::Half'] %}
    {%- for output_type in ['uint8_t', 'at::Half', 'float'] %}
        {{ template_instantiation(emb_type, cache_type, output_type, use_cache, kMaxVecsPerThread, kThreadGroupSize) }}
    {%- endfor %}
    {%- endfor %}
    {%- endfor %}
{%- endmacro %}


////////////////////////////////////////////////////////////////////////////////
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
////////////////////////////////////////////////////////////////////////////////

{#- /*
    Compute the Cartesian product of (use_cache, kMaxVecsPerThread, kThreadGroupSize)
    in the FBGEMM_USE_SUBWARP_SHUFFLE case

    constexpr int kMaxVecsPerThread = std::max({{ kMaxElemPerThread }} / 4, 1);
    constexpr int kThreadGroupSize = kWarpSize / std::max(4 / {{ kMaxElemPerThread }}, 1);

    This is needed to compute the unique tuples to use for explicit instantiation,
    so that we can avoid duplicate template instantiations.
*/ #}
{%- set tuples = [] %}
{%- for use_cache in ['true', 'false'] %}
{%- for kMaxElemPerThread in range(1, max_embedding_dim // (items_per_warp // 4) + 1) %}
{%- if kMaxElemPerThread in [1, 2] or kMaxElemPerThread % 4 == 0 %}
    {%- set t0 = use_cache if not dense else "NULL" %}
    {%- set t1 = [ (kMaxElemPerThread // 4), 1 ] | max if not nobag else "NULL" %}
    {%- set t2 = [ 4 // kMaxElemPerThread, 1] | max %}
    {%- set temp = tuples.append((t0, t1, "(kWarpSize / " ~ t2 ~ ")")) %}
{%- endif %}
{%- endfor %}
{%- endfor %}

{#- /*
    Enumerate over the unique tuples (NULL means the field is not materialized
    for the template context, e.g. where nobag = true):

    (true,·1,·(kWarpSize·/·4))
    (true,·1,·(kWarpSize·/·2))
    (true,·1,·(kWarpSize·/·1))
    (true,·2,·(kWarpSize·/·1))
    (true,·3,·(kWarpSize·/·1))
    (true,·4,·(kWarpSize·/·1))
    (true,·5,·(kWarpSize·/·1))
    (true,·6,·(kWarpSize·/·1))
    (true,·7,·(kWarpSize·/·1))
    (true,·8,·(kWarpSize·/·1))
    (false,·1,·(kWarpSize·/·4))
    (false,·1,·(kWarpSize·/·2))
    (false,·1,·(kWarpSize·/·1))
    (false,·2,·(kWarpSize·/·1))
    (false,·3,·(kWarpSize·/·1))
    (false,·4,·(kWarpSize·/·1))
    (false,·5,·(kWarpSize·/·1))
    (false,·6,·(kWarpSize·/·1))
    (false,·7,·(kWarpSize·/·1))
    (false,·8,·(kWarpSize·/·1))

    (NULL,·1,·(kWarpSize·/·4))
    (NULL,·1,·(kWarpSize·/·2))
    (NULL,·1,·(kWarpSize·/·1))
    (NULL,·2,·(kWarpSize·/·1))
    (NULL,·3,·(kWarpSize·/·1))
    (NULL,·4,·(kWarpSize·/·1))
    (NULL,·5,·(kWarpSize·/·1))
    (NULL,·6,·(kWarpSize·/·1))
    (NULL,·7,·(kWarpSize·/·1))
    (NULL,·8,·(kWarpSize·/·1))

    (true,·NULL,·(kWarpSize·/·4))
    (true,·NULL,·(kWarpSize·/·2))
    (true,·NULL,·(kWarpSize·/·1))
    (false,·NULL,·(kWarpSize·/·4))
    (false,·NULL,·(kWarpSize·/·2))
    (false,·NULL,·(kWarpSize·/·1))

    (NULL,·NULL,·(kWarpSize·/·4))
    (NULL,·NULL,·(kWarpSize·/·2))
    (NULL,·NULL,·(kWarpSize·/·1))
*/ #}
{%- for (use_cache, kMaxVecsPerThread, kThreadGroupSize) in tuples | unique %}
    {{ bulk_template_instantiations(use_cache, kMaxVecsPerThread, kThreadGroupSize) }}
{%- endfor %}

////////////////////////////////////////////////////////////////////////////////
#else
////////////////////////////////////////////////////////////////////////////////

{#- /*
    Compute the Cartesian product of (use_cache, kMaxVecsPerThread, kThreadGroupSize)
    in the non-FBGEMM_USE_SUBWARP_SHUFFLE case

    constexpr int kMaxVecsPerThread = std::max({{ kMaxElemPerThread }} / 4, 1);
    constexpr int kThreadGroupSize = kWarpSize;
*/ #}
{%- set tuples = [] %}
{%- for use_cache in ['true', 'false'] %}
{%- for kMaxElemPerThread in range(1, max_embedding_dim // (items_per_warp // 4) + 1) %}
{%- if kMaxElemPerThread in [1, 2] or kMaxElemPerThread % 4 == 0 %}
    {%- set t0 = use_cache if not dense else "NULL" %}
    {%- set t1 = [ (kMaxElemPerThread // 4), 1 ] | max if not nobag else "NULL" %}
    {%- set temp = tuples.append((t0, t1, "kWarpSize")) %}
{%- endif %}
{%- endfor %}
{%- endfor %}

{#- /*
    Enumerate over the unique tuples (NULL means the field is not materialized
    for the template context, e.g. where nobag = true):

    (true,·1,·kWarpSize)
    (true,·2,·kWarpSize)
    (true,·3,·kWarpSize)
    (true,·4,·kWarpSize)
    (true,·5,·kWarpSize)
    (true,·6,·kWarpSize)
    (true,·7,·kWarpSize)
    (true,·8,·kWarpSize)
    (false,·1,·kWarpSize)
    (false,·2,·kWarpSize)
    (false,·3,·kWarpSize)
    (false,·4,·kWarpSize)
    (false,·5,·kWarpSize)
    (false,·6,·kWarpSize)
    (false,·7,·kWarpSize)
    (false,·8,·kWarpSize)

    (NULL,·1,·kWarpSize)
    (NULL,·2,·kWarpSize)
    (NULL,·3,·kWarpSize)
    (NULL,·4,·kWarpSize)
    (NULL,·5,·kWarpSize)
    (NULL,·6,·kWarpSize)
    (NULL,·7,·kWarpSize)
    (NULL,·8,·kWarpSize)

    (true,·NULL,·kWarpSize)
    (false,·NULL,·kWarpSize)

    (NULL,·NULL,·kWarpSize)
*/ #}
{%- for (use_cache, kMaxVecsPerThread, kThreadGroupSize) in tuples | unique %}
    {{ bulk_template_instantiations(use_cache, kMaxVecsPerThread, kThreadGroupSize) }}
{%- endfor %}

////////////////////////////////////////////////////////////////////////////////
#endif
////////////////////////////////////////////////////////////////////////////////
