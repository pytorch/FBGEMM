#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

{#
// @lint-ignore LINTIGNORE
// @lint-ignore-every CLANGFORMAT
// clang-format off
// Note: clang-format off doesn't work with this templaterized code,
// so we need to keep lint-ignore-every.
// See https://fburl.com/dw9ljh4h
#}

{% set wdesc =  "weighted" if weighted else "unweighted" %}
#include "codegen/embedding_forward_template_helpers.cuh"

#define SHFL_SYNC(val, srcLane) shfl_sync(val, srcLane, kThreadGroupSize, shfl_sync_mask)

{% if not dense %}
constexpr int32_t kCacheLocationMissing = -1;
{% endif %}

constexpr size_t kForwardMaxThreads = 512;

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

{% if not weighted %}
template <
    typename emb_t,
    typename cache_t,
    typename output_t,
    typename index_t,
    size_t kThreadGroupSize
    >
__launch_bounds__(kForwardMaxThreads)
__global__ void {{ "dense" if dense else "split" }}_embedding_nobag_codegen_forward_unweighted_small_kernel(
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    int64_t D,
    FixedDivisor fd_B,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        lxu_cache_locations,
    {% endif %}
    at::PackedTensorAccessor64<output_t, 2, at::RestrictPtrTraits>
        output // [B][total_D],
    ) {
    int32_t T = weights_offsets.size(0);
    int32_t B = (offsets.size(0) - 1) / T;
    int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
    if (b_t >= B * T) {
        return;
    }
    int32_t t;
    int32_t b;
    fd_B.DivMod(b_t, &t, &b);
    int64_t weights_offset = weights_offsets[t];
    index_t indices_start = offsets[t * B + b];
    index_t indices_end = offsets[t * B + b + 1];
    int32_t L = indices_end - indices_start;
    const emb_t* __restrict__ weights;
    {% if not dense %}
    const auto placement = static_cast<PlacementType>(weights_placements[t]);
    if (placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset];
    } else {
        weights = &uvm_weights[weights_offset];
    }
    {% else %}
    weights = &dev_weights[weights_offset];
    {% endif %}

    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }

    const int32_t group_start = threadIdx.x / kThreadGroupSize * kThreadGroupSize;
    const int32_t group_end = group_start + kThreadGroupSize;
    const int32_t d = threadIdx.x % kThreadGroupSize * 4;

    for (int32_t l_start = 0; l_start < L; l_start += kWarpSize) {
        int32_t l = l_start + threadIdx.x;
        int64_t idx = l < L ? indices[indices_start + l] : 0;
        {% if not dense %}
        int32_t cache_idx = (placement == PlacementType::MANAGED_CACHING && l < L) ? lxu_cache_locations[indices_start + l] : 0;
        {% endif %}
        for (auto j = group_start; j < group_end && l_start + j < L; ++j) {
            int64_t idx_j = shfl_sync(idx, j);
            int64_t output_j = indices_start + l_start + j;
            {% if not dense %}
            int32_t cache_idx_j = shfl_sync(cache_idx, j);
            {% endif %}

            {% if not dense %}
            auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
                const_cast<emb_t*>(&weights[idx_j * D_emb]),
                const_cast<cache_t*>(&lxu_cache_weights[cache_idx_j][0]),
                D,
                nullptr);
            // assume cache is fp16/fp32 which doesn't require qparams
            float2 qparams_cache = make_float2(0.0f, 0.0f);

            {% endif %}
            auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
                const_cast<emb_t*>(&weights[idx_j * D_emb]),
                nullptr,
                D,
                nullptr);
            float2 qparams_emb;
            if (std::is_same<emb_t, uint8_t>::value) {
                qparams_emb = weight_row_emb.load_qparams();
            }

            if (d < D) {
                {% if not dense %}
                if (placement == PlacementType::MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                    Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                    weight.store(&output[output_j][d]);
                } else {
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    weight.store(&output[output_j][d]);
                }
                {% else %}
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    weight.store(&output[output_j][d]);
                {% endif %}
            }
        }
    }
}
{% endif %}

{% for nobag in [True, False] %}
{% if not nobag or not weighted %}
template <
    typename emb_t,
    typename cache_t,
    typename output_t,
    {% if not dense %}
    bool use_lxu_cache,
    {% endif %}
    typename index_t,
    {% if not nobag %}
    size_t kMaxVecsPerThread,
    {% endif %}
    size_t kThreadGroupSize = kWarpSize
    >
__launch_bounds__(kForwardMaxThreads)
__global__ void {{ "dense" if dense else "split" }}_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel(
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {% else %}
    int64_t D,
    {% endif %}
    FixedDivisor fd_B,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    {% if not nobag %}
    int64_t pooling_mode,
    {% endif %}
    {% if weighted %}
    at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits>
        indice_weights,
    {% endif %}
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        lxu_cache_locations,
    {% endif %}
    at::PackedTensorAccessor64<output_t, 2, at::RestrictPtrTraits>
        output // [B][total_D],
    ) {
    int32_t T = weights_offsets.size(0);
    {% if not nobag %}
    const bool mean_pooling = static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN;
    int32_t B = output.size(0);
    {% else %}
    int32_t B = (offsets.size(0) - 1) / T;
    {% endif %}
    int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
    if (b_t >= B * T) {
        return;
    }
    int32_t t;
    int32_t b;
    fd_B.DivMod(b_t, &t, &b);
    int64_t weights_offset = weights_offsets[t];
    {% if not nobag %}
    int32_t D_start = D_offsets[t];
    int32_t D_end = D_offsets[t + 1];
    int32_t D = D_end - D_start;
    {% endif %}
    index_t indices_start = offsets[t * B + b];
    index_t indices_end = offsets[t * B + b + 1];
    int32_t L = indices_end - indices_start;
    const emb_t* __restrict__ weights;
    {% if not dense %}
    const auto placement = static_cast<PlacementType>(weights_placements[t]);
    if (placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset];
    } else {
        weights = &uvm_weights[weights_offset];
    }
    {% else %}
    weights = &dev_weights[weights_offset];
    {% endif %}

    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }

    constexpr int VEC_WIDTH = 4;
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
    const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
    const unsigned int shfl_sync_mask = 0xffffffffu;
#endif

    {% if not nobag %}
    const float inv_L = (mean_pooling && L != 0) ? static_cast<float>(1.0) / L: static_cast<float>(1.0);
    Vec4T<cache_t> accumulators[kMaxVecsPerThread];
    {% endif %}
    for (int32_t l_start = 0; l_start < L; l_start += kThreadGroupSize) {
        int32_t l = l_start + threadIdx.x;
        int64_t idx = l < L ? indices[indices_start + l] : 0;
        {% if not dense %}
        int32_t cache_idx = (use_lxu_cache && placement == PlacementType::MANAGED_CACHING && l < L) ? lxu_cache_locations[indices_start + l] : 0;
        {% endif %}
        {% if weighted %}
        at::acc_type<cache_t, true> idx_weight = l < L ? indice_weights[indices_start + l] : 0;
        {% endif %}
        for (auto j = 0; j < kThreadGroupSize && l_start + j < L; ++j) {
            int64_t idx_j = SHFL_SYNC(idx, j);
            {% if nobag %}
            int64_t output_j = indices_start + l_start + j;
            {% endif %}
            {% if not dense %}
            int32_t cache_idx_j = use_lxu_cache ? SHFL_SYNC(cache_idx, j) : 0;
            {% endif %}

            {% if weighted %}
            at::acc_type<cache_t, true> idx_weight_j = SHFL_SYNC(idx_weight, j);
            {% endif %}

            {% if not dense %}
            // use_lxu_cache is a compile time condition
            if (use_lxu_cache && placement == PlacementType::MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
                    const_cast<emb_t*>(&weights[idx_j * D_emb]),
                    const_cast<cache_t*>(&lxu_cache_weights[cache_idx_j][0]),
                    D,
                    nullptr);
                // assume cache is fp16/fp32 which doesn't require qparams
                float2 qparams_cache = make_float2(0.0f, 0.0f);

                {% if not nobag %}
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                    {% if weighted %}
                    accumulators[i].fma_(weight, idx_weight_j);
                    {% else %}
                    accumulators[i].add_(weight);
                    {% endif %}
                }
                {% else %}
                for (int32_t i = 0; i < D; i += kThreadGroupSize * VEC_WIDTH) {
                    int32_t d = i + threadIdx.x * VEC_WIDTH;
                    if (d < D) {
                        Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                        weight.store(&output[output_j][d]);
                    }
                }
                {% endif %}
            }
            else { // else row is not in cache
            {% endif %}
                auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
                    const_cast<emb_t*>(&weights[idx_j * D_emb]),
                    nullptr,
                    D,
                    nullptr);
                float2 qparams_emb;
                if (std::is_same<emb_t, uint8_t>::value) {
                    qparams_emb = weight_row_emb.load_qparams();
                }
                {% if not nobag %}
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    {% if weighted %}
                    accumulators[i].fma_(weight, idx_weight_j);
                    {% else %}
                    accumulators[i].add_(weight);
                    {% endif %}
                }
                {% else %}
                for (int32_t i = 0; i < D; i += kThreadGroupSize * VEC_WIDTH) {
                    int32_t d = i + threadIdx.x * VEC_WIDTH;
                    if (d < D) {
                        Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                        weight.store(&output[output_j][d]);
                    }
                }
                {% endif %}
            {% if not dense %}
            } // else row is not in cache
            {% endif %}
        }
    }

    {% if not nobag %}
    if (!std::is_same<output_t, uint8_t>::value) {
        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
        i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
        ++i) {
            accumulators[i].mul_(inv_L);
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            accumulators[i].store(&output[b][D_start + d]);
        }
    } else {
        // apply per feature row-wise int8
        float thread_local_min = std::numeric_limits<float>::max();
        float thread_local_max = std::numeric_limits<float>::lowest();
        float2 qparams;

        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            accumulators[i].mul_(inv_L);
            thread_local_max = max(thread_local_max, vec4_max(accumulators[i]));
            thread_local_min = min(thread_local_max, vec4_min(accumulators[i]));
        }

        qparams = warp_find_qparams(thread_local_min, thread_local_max);
        int output_D_start = D_start + t * 8;
        int output_D_end = output_D_start + D;

        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            nearest_rounding_vector<output_t, cache_t>(&output[b][output_D_start + d], accumulators[i], qparams);
        }
        if (threadIdx.x == 0) {
            store_qparams_to_row(&output[b][output_D_end], qparams);
        }

    }
    {% endif %}
}

Tensor {{ "dense" if dense else "split" }}_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_cuda(
    Tensor dev_weights,
    {% if not dense %}
    Tensor uvm_weights,
    Tensor lxu_cache_weights,
    Tensor weights_placements,
    {% endif %}
    Tensor weights_offsets,
    {% if not nobag %}
    Tensor D_offsets,
    int64_t total_D,
    int64_t max_D,
    {% else %}
    int64_t D,
    {% endif %}
    Tensor indices,
    Tensor offsets,
    {% if not nobag %}
    int64_t pooling_mode,
    {% endif %}
    {% if weighted %}
    Tensor indice_weights,
    {% endif %}
    {% if not dense %}
    Tensor lxu_cache_locations,
    {% endif %}
    int64_t output_dtype,
    int64_t unused
) {
    TENSOR_ON_CUDA_GPU(dev_weights);
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(uvm_weights);
    TENSOR_ON_CUDA_GPU(lxu_cache_weights);
    TENSOR_ON_CUDA_GPU(weights_placements);
    {% endif %}
    TENSOR_ON_CUDA_GPU(weights_offsets);
    {% if not nobag %}
    TENSOR_ON_CUDA_GPU(D_offsets);
    {% endif %}
    TENSOR_ON_CUDA_GPU(indices);
    TENSOR_ON_CUDA_GPU(offsets);
    {% if weighted %}
    TENSOR_ON_CUDA_GPU(indice_weights);
    {% endif %}
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(lxu_cache_locations);
    {% endif %}

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());

    {% if not nobag %}
    int32_t T = D_offsets.numel() - 1;
    {% else %}
    int32_t total_L = indices.numel();
    int32_t T = weights_offsets.numel();
    {% endif %}
    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    int32_t B = (offsets.size(0) - 1) / T;
    TORCH_CHECK(B >= 0);
    {% if not nobag %}
    TORCH_CHECK(total_D > 0);
    TORCH_CHECK(total_D % 4 == 0);
    TORCH_CHECK(max_D <= {{ max_embedding_dim }});
    {% else %}
    TORCH_CHECK(D > 0);
    TORCH_CHECK(D % 4 == 0);
    {% endif %}

    Tensor output;
    {% if nobag %}
    SparseType o_dtype = static_cast<SparseType>(output_dtype);
    TORCH_CHECK(o_dtype == SparseType::FP32 || o_dtype == SparseType::FP16 ||
                o_dtype == SparseType::BF16 || o_dtype == SparseType::INT8);
    int64_t adjusted_D = D;
    if (o_dtype == SparseType::INT8) {
        adjusted_D += T * kINT8QparamsBytes;
    }
    output = at::empty({total_L, adjusted_D}, dev_weights.options().dtype(getScalarType(o_dtype)));
    {% else %}
    SparseType o_dtype = static_cast<SparseType>(output_dtype);
    TORCH_CHECK(o_dtype == SparseType::FP32 || o_dtype == SparseType::FP16 ||
                o_dtype == SparseType::BF16 || o_dtype == SparseType::INT8);
    int64_t total_adjusted_D = total_D;
    if (o_dtype == SparseType::INT8) {
        total_adjusted_D += T * kINT8QparamsBytes;
    }
    output = at::empty({B, total_adjusted_D}, dev_weights.options().dtype(getScalarType(o_dtype)));

    {% endif %}

    if (B == 0) {
        return output;
    }

    DISPATCH_EMB_CACHE_OUTPUT_TYPES(
        dev_weights.scalar_type(),
        {% if not dense %}
        lxu_cache_weights.scalar_type(),
        {% else %}
        dev_weights.scalar_type(),
        {% endif %}
        output.scalar_type(),
        "batched_embedding{{ "_nobag" if nobag else "" }}_forward_kernel_2", [&] {
        {% if not dense %}
        // Check if LXU cache is used
        bool use_lxu_cache = lxu_cache_weights.numel() > 0;
        {% endif %}
        {% if not nobag %}
        {% for use_cache in ["false", "true"] %}
        // The dense case does not have cache so we have to generate code for
        // only one case (value of use_cache does not matter)
        {% if (not dense) or (use_cache == "true") %}
        {% if not dense %}
        if (use_lxu_cache == {{ use_cache }}) {
        {% endif %}
            // kMaxElemPerThread is # of elements handled by thread if we use a full warp for a row
            // We consider kMaxElemPerThread 1 and 2, and then a multiple of 4.
            {% for kMaxElemPerThread in range(1, max_embedding_dim // (items_per_warp // 4) + 1) %}
            {% if kMaxElemPerThread in [1, 2] or kMaxElemPerThread % 4 == 0 %}
            if (max_D <= {{ items_per_warp // 4 * kMaxElemPerThread }}) {
                // hipcc can't use max in constexpr
                constexpr int kMaxVecsPerThread = {{ kMaxElemPerThread }} / 4 >= 1 ? {{ kMaxElemPerThread }} / 4 : 1;
                // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
                constexpr int kThreadGroupSize = kWarpSize / std::max(4 / {{ kMaxElemPerThread }}, 1);
#else
                constexpr int kThreadGroupSize = kWarpSize;
#endif
                {% if not dense %}
                split_embedding_codegen_forward_{{ wdesc }}_kernel<emb_t, cache_t, output_t, {{ use_cache }}, int64_t, kMaxVecsPerThread, kThreadGroupSize><<<
                {% else %}
                dense_embedding_codegen_forward_{{ wdesc }}_kernel<emb_t, cache_t, output_t, int64_t, kMaxVecsPerThread, kThreadGroupSize><<<
                {% endif %}
                    div_round_up((B * T), kForwardMaxThreads / kThreadGroupSize),
                    dim3(kThreadGroupSize, kForwardMaxThreads / kThreadGroupSize),
                    0,
                    at::cuda::getCurrentCUDAStream()>>>(
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    {% if not dense %}
                    uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                    weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    FixedDivisor(B),
                    indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    pooling_mode,
                    {% if weighted %}
                    indice_weights.packed_accessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not dense %}
                    lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    output.packed_accessor64<
                        output_t,
                        2,
                        at::RestrictPtrTraits>()
                    );

                return;
            }
            {% endif %}
            {% endfor %}
        {% if not dense %}
        } // if (use_lxu_cache == {{ use_cache }})
        {% endif %}
        {% endif %} // if (not dense) or (use_cache == "true")
        {% endfor %} // for use_cache in ["false", "true"]
        {% else %}
        {% for kEmbeddingSize in [4, 8, 16, 32] %}
        if (D <= {{ kEmbeddingSize }}) {
        {% if not dense %}
        split_embedding_nobag_codegen_forward_unweighted_small_kernel<emb_t, cache_t, output_t, int64_t, {{ kEmbeddingSize // 4 }}><<<
        {% else %}
        dense_embedding_nobag_codegen_forward_unweighted_small_kernel<emb_t, cache_t, output_t, int64_t, {{ kEmbeddingSize // 4 }}><<<
        {% endif %}
            div_round_up((B * T), kForwardMaxThreads / kWarpSize),
            dim3(kWarpSize, kForwardMaxThreads / kWarpSize),
            0,
            at::cuda::getCurrentCUDAStream()>>>(
            dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
            {% if not dense %}
            uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
            lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
            weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            {% endif %}
            weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            D,
            FixedDivisor(B),
            indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            {% if not dense %}
            lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            {% endif %}
            output.packed_accessor64<
                output_t,
                2,
                at::RestrictPtrTraits>()
            );
            return;
        }
        {% endfor %}
        {% for use_cache in ["false", "true"] %}
        // The dense case does not have cache so we have to generate code for
        // only one case (value of use_cache does not matter)
        {% if (not dense) or (use_cache == "true") %}
        {% if not dense %}
        if (use_lxu_cache == {{ use_cache }}) {
            split_embedding_nobag_codegen_forward_unweighted_kernel<emb_t, cache_t, output_t, {{ use_cache }}, int64_t><<<
        {% else %}
            dense_embedding_nobag_codegen_forward_unweighted_kernel<emb_t, cache_t, output_t, int64_t><<<
        {% endif %}
                div_round_up((B * T), kForwardMaxThreads / kWarpSize),
                dim3(kWarpSize, kForwardMaxThreads / kWarpSize),
                0,
                at::cuda::getCurrentCUDAStream()>>>(
                dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                {% if not dense %}
                uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                {% endif %}
                weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                D,
                FixedDivisor(B),
                indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                {% if not dense %}
                lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                {% endif %}
                output.packed_accessor64<
                    output_t,
                    2,
                    at::RestrictPtrTraits>()
                );
                return;
        {% if not dense %}
        } // if (use_lxu_cache == {{ use_cache }})
        {% endif %}
        {% endif %} // if (not dense) or (use_cache == "true")
        {% endfor %} // for use_cache in ["false", "true"]
        {% endif %}
        });

  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return output;
}
{% endif %}
{% endfor %}
    // clang-format on
