#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

{#
// @lint-ignore LINTIGNORE
// @lint-ignore-every CLANGFORMAT
// clang-format off
// Note: clang-format off doesn't work with this templaterized code,
// so we need to keep lint-ignore-every.
// See https://fburl.com/dw9ljh4h
#}

{% set wdesc =  "weighted" if weighted else "unweighted" %}
#include "codegen/embedding_forward_template_helpers.cuh"

{% if not dense %}
constexpr int32_t kCacheLocationMissing = -1;
{% endif %}

constexpr size_t kForwardMaxThreads = 512;

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

{% for nobag in [True, False] %}
{% if not nobag or not weighted %}
template <
    typename emb_t,
    typename cache_t,
    {% if not dense %}
    typename output_t,
    {% endif %}
    typename index_t
    {% if not nobag %}
    ,size_t kMaxVecsPerThread
    {% endif %}
    >
__launch_bounds__(kForwardMaxThreads)
__global__ void {{ "dense" if dense else "split" }}_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel(
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {% else %}
    int64_t D,
    {% endif %}
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    {% if not nobag %}
    int64_t pooling_mode,
    {% endif %}
    {% if weighted %}
    at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits>
        indice_weights,
    {% endif %}
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        lxu_cache_locations,
    at::PackedTensorAccessor32<output_t, 2, at::RestrictPtrTraits>
        output // [B][total_D],
    {% else %}
    at::PackedTensorAccessor32<at::acc_type<cache_t,true>, 2, at::RestrictPtrTraits>
        output // [B][total_D],
    {% endif %}
    ) {
    {% if not nobag %}
    int32_t B = output.size(0);
    int32_t T = D_offsets.size(0) - 1;
    {% else %}
    int32_t T = weights_offsets.size(0);
    int32_t B = (offsets.size(0) - 1) / T;
    {% endif %}
    int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
    int32_t t = b_t / B;
    int32_t b = b_t % B;

    if (b_t >= B * T) {
        return;
    }
    int64_t weights_offset = weights_offsets[t];
    {% if not nobag %}
    int32_t D_start = D_offsets[t];
    int32_t D_end = D_offsets[t + 1];
    int32_t D = D_end - D_start;
    {% endif %}
    index_t indices_start = offsets[t * B + b];
    index_t indices_end = offsets[t * B + b + 1];
    int32_t L = indices_end - indices_start;
    const emb_t* __restrict__ weights;
    {% if not dense %}
    const auto placement = static_cast<PlacementType>(weights_placements[t]);
    if (placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset];
    } else {
        weights = &uvm_weights[weights_offset];
    }
    {% else %}
    weights = &dev_weights[weights_offset];
    {% endif %}

    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }

    {% if not nobag %}
    Vec4T<cache_t> accumulators[kMaxVecsPerThread];
    {% endif %}
    for (int32_t l_start = 0; l_start < L; l_start += kWarpSize) {
        int32_t l = l_start + threadIdx.x;
        int64_t idx = l < L ? indices[indices_start + l] : 0;
        {% if not dense %}
        int32_t cache_idx = (placement == PlacementType::MANAGED_CACHING && l < L) ? lxu_cache_locations[indices_start + l] : 0;
        {% endif %}
        {% if weighted %}
        at::acc_type<cache_t, true> idx_weight = l < L ? indice_weights[indices_start + l] : 0;
        {% endif %}
        for (auto j = 0; j < kWarpSize && l_start + j < L; ++j) {
            int64_t idx_j = shfl_sync(idx, j);
            {% if nobag %}
            int64_t output_j = indices_start + l_start + j;
            {% endif %}
            {% if not dense %}
            int32_t cache_idx_j = shfl_sync(cache_idx, j);
            {% endif %}

            {% if weighted %}
            at::acc_type<cache_t, true> idx_weight_j = shfl_sync(idx_weight, j);
            {% endif %}

            {% if not dense %}
            auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
                const_cast<emb_t*>(&weights[idx_j * D_emb]),
                const_cast<cache_t*>(&lxu_cache_weights[cache_idx_j][0]),
                D,
                nullptr);
            float2 qparams_cache; // assume cache is fp16/fp32 which doesn't require qparams

            {% endif %}
            auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
                const_cast<emb_t*>(&weights[idx_j * D_emb]),
                nullptr,
                D,
                nullptr);
            float2 qparams_emb;
            if (std::is_same<emb_t, uint8_t>::value) {
                qparams_emb = weight_row_emb.load_qparams();
            }

            {% if not nobag %}
            #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                ++i) {
                int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
                {% if not dense %}
                if (placement == PlacementType::MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                    Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                    {% if weighted %}
                    accumulators[i].fma_(weight, idx_weight_j);
                    {% else %}
                    accumulators[i].add_(weight);
                    {% endif %}
                } else {
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    {% if weighted %}
                    accumulators[i].fma_(weight, idx_weight_j);
                    {% else %}
                    accumulators[i].add_(weight);
                    {% endif %}
                }
                {% else %}
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    {% if weighted %}
                    accumulators[i].fma_(weight, idx_weight_j);
                    {% else %}
                    accumulators[i].add_(weight);
                    {% endif %}
                {% endif %}
            }
            {% else %}
            for (int32_t i = 0; i < D; i+=4 * kWarpSize) {
                int32_t d = i + threadIdx.x * 4;
                if (d < D) {
                    {% if not dense %}
                    if (placement == PlacementType::MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                        Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                        weight.store(&output[output_j][d]);
                    } else {
                        Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                        weight.store(&output[output_j][d]);
                    }
                    {% else %}
                        Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                        weight.store(&output[output_j][d]);
                    {% endif %}
                }
            }
            {% endif %}
        }
    }

    {% if not nobag %}
    {% if not dense %}
    if (!std::is_same<output_t, uint8_t>::value) {
        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
        i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
        ++i) {
            if (static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN && L != 0) {
                accumulators[i].mul_(1.0 / L);
            }
            int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
            accumulators[i].store(&output[b][D_start + d]);
        }
    } else {
        // apply per feature row-wise int8
        float thread_local_min = std::numeric_limits<float>::max();
        float thread_local_max = std::numeric_limits<float>::lowest();
        float2 qparams;

        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
            ++i) {
            if (static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN && L != 0) {
                accumulators[i].mul_(1.0 / L);
            }
            thread_local_max = max(thread_local_max, vec4_max(accumulators[i]));
            thread_local_min = min(thread_local_max, vec4_min(accumulators[i]));
        }

        qparams = warp_find_qparams(thread_local_min, thread_local_max);
        int output_D_start = D_start + t * 8;
        int output_D_end = output_D_start + D;

        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
            ++i) {
            int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
            nearest_rounding_vector<output_t, cache_t>(&output[b][output_D_start + d], accumulators[i], qparams);
        }
        if (threadIdx.x == 0) {
            store_qparams_to_row(&output[b][output_D_end], qparams);
        }

    }
    {% else %}
    // no pooled embedding quantization fusion for dense embeddings
    #pragma unroll kMaxVecsPerThread
    for (int32_t i = 0;
        i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
        ++i) {
        int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
        if (static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN && L != 0) {
            accumulators[i].mul_(1.0 / L);
        }
        accumulators[i].store(&output[b][D_start + d]);
    }
    {% endif %}
    {% endif %}
}

Tensor {{ "dense" if dense else "split" }}_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_cuda(
    Tensor dev_weights,
    {% if not dense %}
    Tensor uvm_weights,
    Tensor lxu_cache_weights,
    Tensor weights_placements,
    {% endif %}
    Tensor weights_offsets,
    {% if not nobag %}
    Tensor D_offsets,
    int64_t total_D,
    int64_t max_D,
    {% else %}
    int64_t D,
    {% endif %}
    Tensor indices,
    Tensor offsets,
    {% if not nobag %}
    int64_t pooling_mode,
    {% endif %}
    {% if weighted %}
    Tensor indice_weights,
    {% endif %}
    {% if not dense %}
    Tensor lxu_cache_locations,
    {% endif %}
    {% if not dense and not nobag %}
    int64_t output_dtype,
    {% endif %}
    int64_t unused
) {
    TENSOR_ON_CUDA_GPU(dev_weights);
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(uvm_weights);
    TENSOR_ON_CUDA_GPU(lxu_cache_weights);
    TENSOR_ON_CUDA_GPU(weights_placements);
    {% endif %}
    TENSOR_ON_CUDA_GPU(weights_offsets);
    {% if not nobag %}
    TENSOR_ON_CUDA_GPU(D_offsets);
    {% endif %}
    TENSOR_ON_CUDA_GPU(indices);
    TENSOR_ON_CUDA_GPU(offsets);
    {% if weighted %}
    TENSOR_ON_CUDA_GPU(indice_weights);
    {% endif %}
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(lxu_cache_locations);
    {% endif %}

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());

    {% if not nobag %}
    int32_t T = D_offsets.numel() - 1;
    {% else %}
    int32_t total_L = indices.numel();
    int32_t T = weights_offsets.numel();
    {% endif %}
    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    int32_t B = (offsets.size(0) - 1) / T;
    TORCH_CHECK(B >= 0);
    {% if not nobag %}
    TORCH_CHECK(total_D > 0);
    TORCH_CHECK(total_D % 4 == 0);
    TORCH_CHECK(max_D <= {{ max_embedding_dim }});
    {% else %}
    TORCH_CHECK(D > 0);
    TORCH_CHECK(D % 4 == 0);
    {% endif %}

    {% if nobag %}
    Tensor output = at::empty({total_L, D}, dev_weights.options().dtype(at::kFloat));
    {% else %}
    Tensor output;
    {% if dense %}
    if (dev_weights.type().scalarType() == at::kHalf || dev_weights.type().scalarType() == at::kByte) {
        output = at::empty({B, total_D}, dev_weights.options().dtype(at::kFloat));
    } else {
        output = at::empty({B, total_D}, dev_weights.options());
    }
    {% else %}
    SparseType o_dtype = static_cast<SparseType>(output_dtype);
    TORCH_CHECK(o_dtype == SparseType::FP32 || o_dtype == SparseType::FP16 ||
                o_dtype == SparseType::BF16 || o_dtype == SparseType::INT8);
    if (o_dtype == SparseType::FP32) {
        output = at::empty({B, total_D}, dev_weights.options().dtype(at::kFloat));
    } else if (o_dtype == SparseType::FP16) {
        output = at::empty({B, total_D}, dev_weights.options().dtype(at::kHalf));
    } else if (o_dtype == SparseType::BF16) {
        output = at::empty({B, total_D}, dev_weights.options().dtype(at::kBFloat16));
    } else if (o_dtype == SparseType::INT8) {
        output = at::empty({B, int64_t(total_D + T * kINT8QparamsBytes)}, dev_weights.options().dtype(at::kByte));
    }
    {% endif %}
    {% endif %}

    if (B == 0) {
        return output;
    }

    {% if not dense %}
    DISPATCH_EMB_CACHE_OUTPUT_TYPES(
    {% else %}
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    {% endif %}
        dev_weights.type(),
        {% if not dense %}
        lxu_cache_weights.type(),
        output.type(),
        {% endif %}
        "batched_embedding{{ "_nobag" if nobag else "" }}_forward_kernel_2", [&] {
        {% if not nobag %}
        {% for kMaxVecsPerThread in range(1, max_embedding_dim // 128 + 1) %}
        if (max_D <= {{ 128 * kMaxVecsPerThread }}) {
            {% if not dense %}
            split_embedding_codegen_forward_{{ wdesc }}_kernel<emb_t, cache_t, output_t, int64_t, {{ kMaxVecsPerThread }}><<<
            {% else %}
            dense_embedding_codegen_forward_{{ wdesc }}_kernel<scalar_t, scalar_t, int64_t, {{ kMaxVecsPerThread }}><<<
            {% endif %}
                div_round_up((B * T), kForwardMaxThreads / kWarpSize),
                dim3(kWarpSize, kForwardMaxThreads / kWarpSize),
                0,
                at::cuda::getCurrentCUDAStream()>>>(
                dev_weights.packed_accessor64<{{ "scalar_t" if dense else "emb_t" }}, 1, at::RestrictPtrTraits>(),
                {% if not dense %}
                uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                {% endif %}
                weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                pooling_mode,
                {% if weighted %}
                indice_weights.packed_accessor32<at::acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>, 1, at::RestrictPtrTraits>(),
                {% endif %}
                {% if not dense %}
                lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                output.packed_accessor32<
                    output_t,
                    2,
                    at::RestrictPtrTraits>()
                );
                {% else %}
                output.packed_accessor32<
                    at::acc_type<scalar_t, true>,
                    2,
                    at::RestrictPtrTraits>()
                );
                {% endif %}

            return;
        }
        {% endfor %}
        {% else %}
        {% if not dense %}
        split_embedding_nobag_codegen_forward_unweighted_kernel<emb_t, cache_t, output_t, int64_t><<<
        {% else %}
        dense_embedding_nobag_codegen_forward_unweighted_kernel<scalar_t, scalar_t, int64_t><<<
        {% endif %}
            div_round_up((B * T), kForwardMaxThreads / kWarpSize),
            dim3(kWarpSize, kForwardMaxThreads / kWarpSize),
            0,
            at::cuda::getCurrentCUDAStream()>>>(
            dev_weights.packed_accessor64<{{ "scalar_t" if dense else "emb_t" }}, 1, at::RestrictPtrTraits>(),
            {% if not dense %}
            uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
            lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
            weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            {% endif %}
            weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            D,
            indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            {% if not dense %}
            lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            output.packed_accessor32<
                output_t,
                2,
                at::RestrictPtrTraits>()
            );
            {% else %}
            output.packed_accessor32<
                at::acc_type<scalar_t, true>,
                2,
                at::RestrictPtrTraits>()
            );
            {% endif %}

            return;
        {% endif %}
        });

  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return output;
}
{% endif %}
{% endfor %}
    // clang-format on
