#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

{#
// @lint-ignore LINTIGNORE
// @lint-ignore-every CLANGFORMAT
// clang-format off
// Note: clang-format off doesn't work with this templaterized code,
// so we need to keep lint-ignore-every.
// See https://fburl.com/dw9ljh4h
#}

{% set wdesc =  "weighted" if weighted else "unweighted" %}
#include "codegen/embedding_forward_template_helpers.cuh"
#include <unistd.h>
#include <limits.h>

#define SHFL_SYNC(val, srcLane) shfl_sync(val, srcLane, kThreadGroupSize, shfl_sync_mask)

#ifdef __HIP_PLATFORM_HCC__
#include "hip_kernel/split_tbe_fwd.hip.hpp"
#endif

{% if not dense %}
constexpr int32_t kCacheLocationMissing = -1;
{% endif %}

constexpr size_t kForwardMaxThreads = 512;

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

{% if not weighted %}
template <
    typename emb_t,
    typename cache_t,
    {% if not dense %}
    typename output_t,
    {% endif %}
    typename index_t,
    size_t kThreadGroupSize
    >
__launch_bounds__(kForwardMaxThreads)
__global__ void {{ "dense" if dense else "split" }}_embedding_nobag_codegen_forward_unweighted_small_kernel(
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    int64_t D,
    FixedDivisor fd_B,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        lxu_cache_locations,
    at::PackedTensorAccessor32<output_t, 2, at::RestrictPtrTraits>
        output // [B][total_D],
    {% else %}
    at::PackedTensorAccessor32<at::acc_type<cache_t,true>, 2, at::RestrictPtrTraits>
        output // [B][total_D],
    {% endif %}
    ) {
    int32_t T = weights_offsets.size(0);
    int32_t B = (offsets.size(0) - 1) / T;
    int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
    if (b_t >= B * T) {
        return;
    }
    int32_t t;
    int32_t b;
    fd_B.DivMod(b_t, &t, &b);
    int64_t weights_offset = weights_offsets[t];
    index_t indices_start = offsets[t * B + b];
    index_t indices_end = offsets[t * B + b + 1];
    int32_t L = indices_end - indices_start;
    const emb_t* __restrict__ weights;
    {% if not dense %}
    const auto placement = static_cast<PlacementType>(weights_placements[t]);
    if (placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset];
    } else {
        weights = &uvm_weights[weights_offset];
    }
    {% else %}
    weights = &dev_weights[weights_offset];
    {% endif %}

    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }

    const int32_t group_start = threadIdx.x / kThreadGroupSize * kThreadGroupSize;
    const int32_t group_end = group_start + kThreadGroupSize;
    const int32_t d = threadIdx.x % kThreadGroupSize * 4;

    for (int32_t l_start = 0; l_start < L; l_start += kWarpSize) {
        int32_t l = l_start + threadIdx.x;
        int64_t idx = l < L ? indices[indices_start + l] : 0;
        {% if not dense %}
        int32_t cache_idx = (placement == PlacementType::MANAGED_CACHING && l < L) ? lxu_cache_locations[indices_start + l] : 0;
        {% endif %}
        for (auto j = group_start; j < group_end && l_start + j < L; ++j) {
            int64_t idx_j = shfl_sync(idx, j);
            int64_t output_j = indices_start + l_start + j;
            {% if not dense %}
            int32_t cache_idx_j = shfl_sync(cache_idx, j);
            {% endif %}

            {% if not dense %}
            auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
                const_cast<emb_t*>(&weights[idx_j * D_emb]),
                const_cast<cache_t*>(&lxu_cache_weights[cache_idx_j][0]),
                D,
                nullptr);
            // assume cache is fp16/fp32 which doesn't require qparams
            float2 qparams_cache = make_float2(0.0f, 0.0f);

            {% endif %}
            auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
                const_cast<emb_t*>(&weights[idx_j * D_emb]),
                nullptr,
                D,
                nullptr);
            float2 qparams_emb;
            if (std::is_same<emb_t, uint8_t>::value) {
                qparams_emb = weight_row_emb.load_qparams();
            }

            if (d < D) {
                {% if not dense %}
                if (placement == PlacementType::MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                    Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                    weight.store(&output[output_j][d]);
                } else {
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    weight.store(&output[output_j][d]);
                }
                {% else %}
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    weight.store(&output[output_j][d]);
                {% endif %}
            }
        }
    }
}
{% endif %}

{% for nobag in [True, False] %}
{% if not nobag or not weighted %}
template <
    typename emb_t,
    typename cache_t,
    {% if not dense %}
    typename output_t,
    bool use_lxu_cache,
    {% endif %}
    typename index_t,
    {% if not nobag %}
    size_t kMaxVecsPerThread,
    {% endif %}
    size_t kThreadGroupSize = kWarpSize
    >
__launch_bounds__(kForwardMaxThreads)
__global__ void {{ "dense" if dense else "split" }}_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_kernel(
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    const at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {% else %}
    int64_t D,
    {% endif %}
    FixedDivisor fd_B,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    {% if not nobag %}
    int64_t pooling_mode,
    {% endif %}
    {% if weighted %}
    at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits>
        indice_weights,
    {% endif %}
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        lxu_cache_locations,
    at::PackedTensorAccessor32<output_t, 2, at::RestrictPtrTraits>
        output // [B][total_D],
    {% else %}
    at::PackedTensorAccessor32<at::acc_type<cache_t,true>, 2, at::RestrictPtrTraits>
        output // [B][total_D],
    {% endif %}
    ) {
    int32_t T = weights_offsets.size(0);
    {% if not nobag %}
    const bool mean_pooling = static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN;
    int32_t B = output.size(0);
    {% else %}
    int32_t B = (offsets.size(0) - 1) / T;
    {% endif %}
    int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
    if (b_t >= B * T) {
        return;
    }
    int32_t t;
    int32_t b;
    fd_B.DivMod(b_t, &t, &b);
    int64_t weights_offset = weights_offsets[t];
    {% if not nobag %}
    int32_t D_start = D_offsets[t];
    int32_t D_end = D_offsets[t + 1];
    int32_t D = D_end - D_start;
    {% endif %}
    index_t indices_start = offsets[t * B + b];
    index_t indices_end = offsets[t * B + b + 1];
    int32_t L = indices_end - indices_start;
    const emb_t* __restrict__ weights;
    {% if not dense %}
    const auto placement = static_cast<PlacementType>(weights_placements[t]);
    if (placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset];
    } else {
        weights = &uvm_weights[weights_offset];
    }
    {% else %}
    weights = &dev_weights[weights_offset];
    {% endif %}

    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }

    constexpr int VEC_WIDTH = 4;
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
    const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
    const unsigned int shfl_sync_mask = 0xffffffffu;
#endif

    {% if not nobag %}
    const float inv_L = (mean_pooling && L != 0) ? static_cast<float>(1.0) / L: static_cast<float>(1.0);
    Vec4T<cache_t> accumulators[kMaxVecsPerThread];
    {% endif %}
    for (int32_t l_start = 0; l_start < L; l_start += kThreadGroupSize) {
        int32_t l = l_start + threadIdx.x;
        int64_t idx = l < L ? indices[indices_start + l] : 0;
        {% if not dense %}
        int32_t cache_idx = (use_lxu_cache && placement == PlacementType::MANAGED_CACHING && l < L) ? lxu_cache_locations[indices_start + l] : 0;
        {% endif %}
        {% if weighted %}
        at::acc_type<cache_t, true> idx_weight = l < L ? indice_weights[indices_start + l] : 0;
        {% endif %}
        for (auto j = 0; j < kThreadGroupSize && l_start + j < L; ++j) {
            int64_t idx_j = SHFL_SYNC(idx, j);
            {% if nobag %}
            int64_t output_j = indices_start + l_start + j;
            {% endif %}
            {% if not dense %}
            int32_t cache_idx_j = use_lxu_cache ? SHFL_SYNC(cache_idx, j) : 0;
            {% endif %}

            {% if weighted %}
            at::acc_type<cache_t, true> idx_weight_j = SHFL_SYNC(idx_weight, j);
            {% endif %}

            {% if not dense %}
            // use_lxu_cache is a compile time condition
            if (use_lxu_cache && placement == PlacementType::MANAGED_CACHING && cache_idx_j != kCacheLocationMissing) {
                auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
                    const_cast<emb_t*>(&weights[idx_j * D_emb]),
                    const_cast<cache_t*>(&lxu_cache_weights[cache_idx_j][0]),
                    D,
                    nullptr);
                // assume cache is fp16/fp32 which doesn't require qparams
                float2 qparams_cache = make_float2(0.0f, 0.0f);

                {% if not nobag %}
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                    {% if weighted %}
                    accumulators[i].fma_(weight, idx_weight_j);
                    {% else %}
                    accumulators[i].add_(weight);
                    {% endif %}
                }
                {% else %}
                for (int32_t i = 0; i < D; i += kThreadGroupSize * VEC_WIDTH) {
                    int32_t d = i + threadIdx.x * VEC_WIDTH;
                    if (d < D) {
                        Vec4T<cache_t> weight = weight_row_cache.load(d, qparams_cache);
                        weight.store(&output[output_j][d]);
                    }
                }
                {% endif %}
            }
            else { // else row is not in cache
            {% endif %}
                auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
                    const_cast<emb_t*>(&weights[idx_j * D_emb]),
                    nullptr,
                    D,
                    nullptr);
                float2 qparams_emb;
                if (std::is_same<emb_t, uint8_t>::value) {
                    qparams_emb = weight_row_emb.load_qparams();
                }
                {% if not nobag %}
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                    {% if weighted %}
                    accumulators[i].fma_(weight, idx_weight_j);
                    {% else %}
                    accumulators[i].add_(weight);
                    {% endif %}
                }
                {% else %}
                for (int32_t i = 0; i < D; i += kThreadGroupSize * VEC_WIDTH) {
                    int32_t d = i + threadIdx.x * VEC_WIDTH;
                    if (d < D) {
                        Vec4T<cache_t> weight = weight_row_emb.load(d, qparams_emb);
                        weight.store(&output[output_j][d]);
                    }
                }
                {% endif %}
            {% if not dense %}
            } // else row is not in cache
            {% endif %}
        }
    }

    {% if not nobag %}
    {% if not dense %}
    if (!std::is_same<output_t, uint8_t>::value) {
        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
        i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
        ++i) {
            accumulators[i].mul_(inv_L);
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            accumulators[i].store(&output[b][D_start + d]);
        }
    } else {
        // apply per feature row-wise int8
        float thread_local_min = std::numeric_limits<float>::max();
        float thread_local_max = std::numeric_limits<float>::lowest();
        float2 qparams;

        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            accumulators[i].mul_(inv_L);
            thread_local_max = max(thread_local_max, vec4_max(accumulators[i]));
            thread_local_min = min(thread_local_max, vec4_min(accumulators[i]));
        }

        qparams = warp_find_qparams(thread_local_min, thread_local_max);
        int output_D_start = D_start + t * 8;
        int output_D_end = output_D_start + D;

        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            nearest_rounding_vector<output_t, cache_t>(&output[b][output_D_start + d], accumulators[i], qparams);
        }
        if (threadIdx.x == 0) {
            store_qparams_to_row(&output[b][output_D_end], qparams);
        }

    }
    {% else %}
    // no pooled embedding quantization fusion for dense embeddings
    #pragma unroll kMaxVecsPerThread
    for (int32_t i = 0;
        i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
        ++i) {
        int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
        if (static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN && L != 0) {
            accumulators[i].mul_(1.0 / L);
        }
        accumulators[i].store(&output[b][D_start + d]);
    }
    {% endif %}
    {% endif %}
}

Tensor {{ "dense" if dense else "split" }}_embedding{{ "_nobag" if nobag else "" }}_codegen_forward_{{ wdesc }}_cuda(
    Tensor dev_weights,
    {% if not dense %}
    Tensor uvm_weights,
    Tensor lxu_cache_weights,
    Tensor weights_placements,
    {% endif %}
    Tensor weights_offsets,
    {% if not nobag %}
    Tensor D_offsets,
    int64_t total_D,
    int64_t max_D,
    {% else %}
    int64_t D,
    {% endif %}
    Tensor indices,
    Tensor offsets,
    {% if not nobag %}
    int64_t pooling_mode,
    {% endif %}
    {% if weighted %}
    Tensor indice_weights,
    {% endif %}
    {% if not dense %}
    Tensor lxu_cache_locations,
    {% endif %}
    {% if not dense %}
    int64_t output_dtype,
    {% endif %}
    int64_t unused
) {
    TENSOR_ON_CUDA_GPU(dev_weights);
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(uvm_weights);
    TENSOR_ON_CUDA_GPU(lxu_cache_weights);
    TENSOR_ON_CUDA_GPU(weights_placements);
    {% endif %}
    TENSOR_ON_CUDA_GPU(weights_offsets);
    {% if not nobag %}
    TENSOR_ON_CUDA_GPU(D_offsets);
    {% endif %}
    TENSOR_ON_CUDA_GPU(indices);
    TENSOR_ON_CUDA_GPU(offsets);
    {% if weighted %}
    TENSOR_ON_CUDA_GPU(indice_weights);
    {% endif %}
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(lxu_cache_locations);
    {% endif %}

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());

    {% if not nobag %}
    int32_t T = D_offsets.numel() - 1;
    {% else %}
    int32_t total_L = indices.numel();
    int32_t T = weights_offsets.numel();
    {% endif %}
    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    int32_t B = (offsets.size(0) - 1) / T;
    TORCH_CHECK(B >= 0);
    {% if not nobag %}
    TORCH_CHECK(total_D > 0);
    TORCH_CHECK(total_D % 4 == 0);
    TORCH_CHECK(max_D <= {{ max_embedding_dim }});
    {% else %}
    TORCH_CHECK(D > 0);
    TORCH_CHECK(D % 4 == 0);
    {% endif %}

    Tensor output;
    {% if nobag %}
    {% if dense %}
        output = at::empty({total_L, D}, dev_weights.options().dtype(at::kFloat));
    {% else %}
    SparseType o_dtype = static_cast<SparseType>(output_dtype);
    TORCH_CHECK(o_dtype == SparseType::FP32 || o_dtype == SparseType::FP16 ||
                o_dtype == SparseType::BF16 || o_dtype == SparseType::INT8);
    int64_t adjusted_D = D;
    if (o_dtype == SparseType::INT8) {
        adjusted_D += T * kINT8QparamsBytes;
    }
    output = at::empty({total_L, adjusted_D}, dev_weights.options().dtype(getScalarType(o_dtype)));
    {% endif %}
    {% else %}
    {% if dense %}
    if (dev_weights.scalar_type() == at::kHalf || dev_weights.scalar_type() == at::kByte) {
        output = at::empty({B, total_D}, dev_weights.options().dtype(at::kFloat));
    } else {
        output = at::empty({B, total_D}, dev_weights.options());
    }
    {% else %}
    SparseType o_dtype = static_cast<SparseType>(output_dtype);
    TORCH_CHECK(o_dtype == SparseType::FP32 || o_dtype == SparseType::FP16 ||
                o_dtype == SparseType::BF16 || o_dtype == SparseType::INT8);
    int64_t total_adjusted_D = total_D;
    if (o_dtype == SparseType::INT8) {
        total_adjusted_D += T * kINT8QparamsBytes;
    }
    output = at::empty({B, total_adjusted_D}, dev_weights.options().dtype(getScalarType(o_dtype)));

    {% endif %}
    {% endif %}

    if (B == 0) {
        return output;
    }

#ifdef __HIP_PLATFORM_HCC__  // HIP Optimal Kernel
    /*
     * current limitations
     1. sparse, and bag
     2. embedding dims in [64, 128, 192, 256]
     3. yet to support mixed embedding dims (loosely guarded below)
     4. yet to support non-uniform table locations (all be on devs)
     5. yet to support duplicate tables from some cases in exact optim (fbgemm_gpu/split_embedding_configs.py)
     */
    {% if not nobag %}
    {% if not dense %}

    // weight param cnt
    int64_t wcnts = dev_weights.numel();
    // mixed hypothesis
    bool mixed_ls = (total_D != (max_D * T));
    // execution guards
    bool guard_ex = (wcnts > 0 && !mixed_ls);

    // all Ts on device
    std::vector<int32_t> wplas(weights_placements.data_ptr<int32_t>(), weights_placements.data_ptr<int32_t>() + weights_placements.numel());
    bool all_devs = std::accumulate(wplas.begin(), wplas.end(), 0) == 0;
    // no duplicate in weight offsets (which is the case exact optim used sometimes)
    std::vector<int64_t> woffs(weights_offsets.data_ptr<int64_t>(), weights_offsets.data_ptr<int64_t>() + weights_offsets.numel());
    std::vector<int64_t>::iterator it = std::unique(woffs.begin(), woffs.end());
    // not support duplicated weights table yet
    bool no_dupt = (it == woffs.end());

    if (guard_ex)  guard_ex = all_devs && no_dupt;

    if (guard_ex && (dev_weights.scalar_type() == at::ScalarType::Half || dev_weights.scalar_type() == at::ScalarType::Float)) {
        constexpr uint32_t workgroup_size = 256;
        constexpr uint32_t wave_size = 64;

        uint32_t bags_per_workgroup = workgroup_size / wave_size;
        uint32_t grids[3] = {(B + bags_per_workgroup - 1) / bags_per_workgroup, (uint32_t)T, 1};
        uint32_t blocks[3] = {workgroup_size, 1, 1};
        int64_t E = wcnts / T / max_D;

	std::string prec = dev_weights.scalar_type() == at::ScalarType::Half  ? "fp16" : "fp32";

        {
            struct {
                void            *output;
                void         *emb_table;
                const int64_t  *indices;
                const int64_t  *offsets;
                int64_t    pooling_mode;
                {% if weighted %}
                float   *indice_weights;
                {% endif %}
                uint32_t        emb_dim;
                uint32_t          batch;
                uint32_t       num_rows;
                uint32_t     num_tables;
            } args;
            size_t arg_size = sizeof(args);
            args.output = output.packed_accessor32<float, 2, at::RestrictPtrTraits>().data();
            if (dev_weights.scalar_type() == at::ScalarType::Half)
                args.emb_table = dev_weights.packed_accessor64<at::Half, 1, at::RestrictPtrTraits>().data();
            else
                args.emb_table = dev_weights.packed_accessor64<float, 1, at::RestrictPtrTraits>().data();
            args.indices = indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>().data();
            args.offsets = offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>().data();
            args.pooling_mode = pooling_mode;
            {% if weighted %}
            args.indice_weights = indice_weights.packed_accessor32<float, 1, at::RestrictPtrTraits>().data();
            {% endif %}
            args.emb_dim = (uint32_t) max_D;
            args.batch = (uint32_t) B;
            args.num_rows = E;
            args.num_tables = (uint32_t) T;

            {% for kDimSize in [64, 128, 192, 256, 384, 512, 640, 768, 896, 1024] %}
            if (max_D <= {{ kDimSize }}) {
                if (prec == "fp16") {
                    hipLaunchKernelGGL(split_tbe_fwd_{{ wdesc }}_hip_kernel_fp16_e{{ kDimSize }},
                        dim3(grids[0], grids[1], grids[2]),
                        dim3(blocks[0], blocks[1], blocks[2]),
                        0, 0,
                        (float *)args.output, (const half *)args.emb_table, args.indices, args.offsets, args.pooling_mode,
                        {% if weighted %}
                        args.indice_weights,
                        {% endif %}
                        args.emb_dim, args.batch, args.num_rows, args.num_tables);
                } else {    // only 2 emb_t: fp16, fp32 for now
                    hipLaunchKernelGGL(split_tbe_fwd_{{ wdesc }}_hip_kernel_fp32_e{{ kDimSize }},
                        dim3(grids[0], grids[1], grids[2]),
                        dim3(blocks[0], blocks[1], blocks[2]),
                        0, 0,
                        (float *)args.output, (const float *)args.emb_table, args.indices, args.offsets, args.pooling_mode,
                        {% if weighted %}
                        args.indice_weights,
                        {% endif %}
                        args.emb_dim, args.batch, args.num_rows, args.num_tables);
                }
		return output;
            }
            {% endfor %}
        }
    }
    {% endif %}  // not dense
    {% endif %}  // not nobag
#endif  // HIP Optimal Kernel

    {% if not dense %}
    DISPATCH_EMB_CACHE_OUTPUT_TYPES(
    {% else %}
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    {% endif %}
        dev_weights.scalar_type(),
        {% if not dense %}
        lxu_cache_weights.scalar_type(),
        output.scalar_type(),
        {% endif %}
        "batched_embedding{{ "_nobag" if nobag else "" }}_forward_kernel_2", [&] {
        {% if not dense %}
        // Check if LXU cache is used
        bool use_lxu_cache = lxu_cache_weights.numel() > 0;
        {% endif %}
        {% if not nobag %}
        {% for use_cache in ["false", "true"] %}
        // The dense case does not have cache so we have to generate code for
        // only one case (value of use_cache does not matter)
        {% if (not dense) or (use_cache == "true") %}
        {% if not dense %}
        if (use_lxu_cache == {{ use_cache }}) {
        {% endif %}
            // kMaxElemPerThread is # of elements handled by thread if we use a full warp for a row
            // We consider kMaxElemPerThread 1 and 2, and then a multiple of 4.
            {% for kMaxElemPerThread in range(1, max_embedding_dim // (items_per_warp // 4) + 1) %}
            {% if kMaxElemPerThread in [1, 2] or kMaxElemPerThread % 4 == 0 %}
            if (max_D <= {{ items_per_warp // 4 * kMaxElemPerThread }}) {
                // hipcc can't use max in constexpr
                constexpr int kMaxVecsPerThread = {{ kMaxElemPerThread }} / 4 >= 1 ? {{ kMaxElemPerThread }} / 4 : 1;
                // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
                constexpr int kThreadGroupSize = kWarpSize / std::max(4 / {{ kMaxElemPerThread }}, 1);
#else
                constexpr int kThreadGroupSize = kWarpSize;
#endif
                {% if not dense %}
                split_embedding_codegen_forward_{{ wdesc }}_kernel<emb_t, cache_t, output_t, {{ use_cache }}, int64_t, kMaxVecsPerThread, kThreadGroupSize><<<
                {% else %}
                dense_embedding_codegen_forward_{{ wdesc }}_kernel<scalar_t, scalar_t, int64_t, kMaxVecsPerThread, kThreadGroupSize><<<
                {% endif %}
                    div_round_up((B * T), kForwardMaxThreads / kThreadGroupSize),
                    dim3(kThreadGroupSize, kForwardMaxThreads / kThreadGroupSize),
                    0,
                    at::cuda::getCurrentCUDAStream()>>>(
                    dev_weights.packed_accessor64<{{ "scalar_t" if dense else "emb_t" }}, 1, at::RestrictPtrTraits>(),
                    {% if not dense %}
                    uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                    weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    FixedDivisor(B),
                    indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    pooling_mode,
                    {% if weighted %}
                    indice_weights.packed_accessor32<at::acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not dense %}
                    lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    output.packed_accessor32<
                        output_t,
                        2,
                        at::RestrictPtrTraits>()
                    );
                    {% else %}
                    output.packed_accessor32<
                        at::acc_type<scalar_t, true>,
                        2,
                        at::RestrictPtrTraits>()
                    );
                    {% endif %}

                return;
            }
            {% endif %}
            {% endfor %}
        {% if not dense %}
        } // if (use_lxu_cache == {{ use_cache }})
        {% endif %}
        {% endif %} // if (not dense) or (use_cache == "true")
        {% endfor %} // for use_cache in ["false", "true"]
        {% else %}
        {% for kEmbeddingSize in [4, 8, 16, 32] %}
        if (D <= {{ kEmbeddingSize }}) {
        {% if not dense %}
        split_embedding_nobag_codegen_forward_unweighted_small_kernel<emb_t, cache_t, output_t, int64_t, {{ kEmbeddingSize // 4 }}><<<
        {% else %}
        dense_embedding_nobag_codegen_forward_unweighted_small_kernel<scalar_t, scalar_t, int64_t, {{ kEmbeddingSize // 4 }}><<<
        {% endif %}
            div_round_up((B * T), kForwardMaxThreads / kWarpSize),
            dim3(kWarpSize, kForwardMaxThreads / kWarpSize),
            0,
            at::cuda::getCurrentCUDAStream()>>>(
            dev_weights.packed_accessor64<{{ "scalar_t" if dense else "emb_t" }}, 1, at::RestrictPtrTraits>(),
            {% if not dense %}
            uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
            lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
            weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            {% endif %}
            weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            D,
            FixedDivisor(B),
            indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            {% if not dense %}
            lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            output.packed_accessor32<
                output_t,
                2,
                at::RestrictPtrTraits>()
            );
            {% else %}
            output.packed_accessor32<
                at::acc_type<scalar_t, true>,
                2,
                at::RestrictPtrTraits>()
            );
            {% endif %}

            return;
        }
        {% endfor %}
        {% for use_cache in ["false", "true"] %}
        // The dense case does not have cache so we have to generate code for
        // only one case (value of use_cache does not matter)
        {% if (not dense) or (use_cache == "true") %}
        {% if not dense %}
        if (use_lxu_cache == {{ use_cache }}) {
            split_embedding_nobag_codegen_forward_unweighted_kernel<emb_t, cache_t, output_t, {{ use_cache }}, int64_t><<<
        {% else %}
            dense_embedding_nobag_codegen_forward_unweighted_kernel<scalar_t, scalar_t, int64_t><<<
        {% endif %}
                div_round_up((B * T), kForwardMaxThreads / kWarpSize),
                dim3(kWarpSize, kForwardMaxThreads / kWarpSize),
                0,
                at::cuda::getCurrentCUDAStream()>>>(
                dev_weights.packed_accessor64<{{ "scalar_t" if dense else "emb_t" }}, 1, at::RestrictPtrTraits>(),
                {% if not dense %}
                uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                {% endif %}
                weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                D,
                FixedDivisor(B),
                indices.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                {% if not dense %}
                lxu_cache_locations.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                output.packed_accessor32<
                    output_t,
                    2,
                    at::RestrictPtrTraits>()
                );
                {% else %}
                output.packed_accessor32<
                    at::acc_type<scalar_t, true>,
                    2,
                    at::RestrictPtrTraits>()
                );
                {% endif %}

                return;
        {% if not dense %}
        } // if (use_lxu_cache == {{ use_cache }})
        {% endif %}
        {% endif %} // if (not dense) or (use_cache == "true")
        {% endfor %} // for use_cache in ["false", "true"]
        {% endif %}
        });

  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return output;
}
{% endif %}
{% endfor %}
    // clang-format on
