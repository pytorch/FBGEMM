#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// clang-format off
{% set wdesc = "weighted" if weighted else "unweighted" %}
#include "fbgemm_gpu/embedding_backward_template_helpers.cuh"
#include "fbgemm_gpu/split_embeddings_utils.cuh"

#define SHFL_SYNC(val, srcLane) shfl_sync(val, srcLane, kThreadGroupSize, shfl_sync_mask)

{% if not dense %}
constexpr int32_t kCacheLocationMissing = -1;
{% endif %}

constexpr size_t kBackwardMaxThreads = 512;

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

namespace {

// Based on the empirical study, max grid size that is 64x larger than the
// number of SMs gives good performance across the board
constexpr int MAX_THREAD_BLOCKS_FACTOR = 64;

int get_max_thread_blocks_() {
  return MAX_THREAD_BLOCKS_FACTOR * at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
}

} // namespace

__global__ __launch_bounds__(kMaxThreads) void
split_embedding_backward_codegen_{{ optimizer }}_{{ wdesc }}_find_long_segments(
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_num_runs,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_run_lengths,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        long_run_ids,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        num_long_run_ids,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        long_run_id_to_really_long_run_ids,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        num_really_long_run_ids,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        grad_accum_counter,
    const int32_t max_segment_length_per_warp,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms) {
  const int32_t num_runs = sorted_linear_indices_num_runs[0];
  for (auto run_id = blockIdx.x * blockDim.x + threadIdx.x; run_id < num_runs; run_id += blockDim.x * gridDim.x) {
    if (sorted_linear_indices_run_lengths[run_id] >= max_segment_length_per_warp) {
        // A segment with length > max_segment_length_per_cta is handled by more than 1 thread block.
        const int num_ctas_for_run =
            use_deterministic_algorithms ? 1 : div_round_up(sorted_linear_indices_run_lengths[run_id], max_segment_length_per_cta);
        const auto long_run_idx = gpuAtomicAdd(&num_long_run_ids[0], num_ctas_for_run);
        // The first thread block in the really long run gets run_id in long_run_ids
        // and the rest get the negative of its offset.
        long_run_ids[long_run_idx] = run_id;
        for (int i = 1; i < num_ctas_for_run; ++i) {
            long_run_ids[long_run_idx + i] = -i;
        }
        if (num_ctas_for_run > 1) {
            const auto really_long_run_idx = gpuAtomicAdd(&num_really_long_run_ids[0], 1);
            grad_accum_counter[really_long_run_idx] = num_ctas_for_run;
            for (int i = 0; i < num_ctas_for_run; ++i) {
                long_run_id_to_really_long_run_ids[long_run_idx + i] = really_long_run_idx;
            }
        }
    }
  }
}

template <typename grad_t>
__global__ __launch_bounds__(kMaxThreads) void grad_mean_kernel(
    const at::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits>
        grad_output,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> offsets,
    at::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits>
        grad_output_mean) {
  int32_t B = grad_output.size(0);
  int32_t T = D_offsets.size(0) - 1;
  int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
  int32_t b = b_t % B;
  int32_t t = b_t / B;

  if (b_t >= B * T) {
    return;
  }
  int32_t D_start = D_offsets[t];
  int32_t D_end = D_offsets[t + 1];
  int32_t D = D_end - D_start;
  int64_t indices_start = offsets[t * B + b];
  int64_t indices_end = offsets[t * B + b + 1];
  int32_t L = indices_end - indices_start;

  if (L != 0) {
    for (int32_t d = threadIdx.x; d * 4 < D; d += blockDim.x) {
      Vec4T<grad_t> grad_out_vec(&grad_output[b][D_start + d * 4]);
      grad_out_vec.mul_(1.0 / L);
      grad_out_vec.store(&grad_output_mean[b][D_start + d * 4]);
    }
  } else {
    for (int32_t d = threadIdx.x; d * 4 < D; d += blockDim.x) {
      Vec4T<grad_t> grad_out_vec(&grad_output[b][D_start + d * 4]);
      grad_out_vec.store(&grad_output_mean[b][D_start + d * 4]);
    }
  }
}

{% for nobag in [True, False] %}
{% if not nobag or not weighted %}
template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    size_t kMaxVecsPerThread,
    int32_t kThreadGroupSize = kWarpSize>
__global__ __launch_bounds__(kMaxThreads) void
split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1(
    const at::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits> grad_output,
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {% else %}
    int32_t B,
    int64_t D,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        hash_size_cumsum,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_run,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_cumulative_run_lengths,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        long_run_ids,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        num_long_run_ids,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {% else %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {% endif %}
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_lxu_cache_locations,
    {% endif %}
    {% if weighted %}
    const at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {% endif %}
    {% if not dense %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {% else %}
    at::PackedTensorAccessor64<cache_t, 1, at::RestrictPtrTraits> grad_dev_weights,
    {% endif %}
    {% if not nobag %}
    FixedDivisor fd,
    {% endif %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_id_to_really_long_run_ids,
    at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 2, at::RestrictPtrTraits> temp_grad_accum,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> grad_accum_counter,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms,
    {{ args.split_kernel_args | join(", ") }}) {
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
  const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
  const unsigned int shfl_sync_mask = 0xffffffffu;
#endif
  constexpr int VEC_WIDTH = 4;
  int32_t T = weights_offsets.size(0);
  {% if not nobag %}
  const int32_t B = grad_output.size(0);
  {% endif %}
  const int32_t num_long_runs = num_long_run_ids[0];
  for (int32_t long_run_id = blockIdx.x; long_run_id < num_long_runs; long_run_id += gridDim.x) {
        // The first thread block in the really long run has run_id in long_run_ids
        // and the rest have the negative of its offset (see find_long_segments kernel).
        int32_t cta_rank_on_current_run = 0;
        int32_t current_run_id = long_run_ids[long_run_id];
        if (current_run_id < 0) {
            cta_rank_on_current_run = -long_run_ids[long_run_id];
            current_run_id = long_run_ids[long_run_id - cta_rank_on_current_run];
        }
        const int32_t run_length =
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1] -
            sorted_linear_indices_cumulative_run_lengths[current_run_id];
        // This computation must agree with how we compute num_ctas_for_run in
        // find_long_segments kernel!
        const int32_t num_ctas_on_current_run =
            use_deterministic_algorithms ? 1 : div_round_up(run_length, max_segment_length_per_cta);


        const int64_t linear_index = sorted_linear_indices_run[current_run_id];
        const int32_t segment_start =
            sorted_linear_indices_cumulative_run_lengths[current_run_id] +
            cta_rank_on_current_run * max_segment_length_per_cta;
        const int32_t segment_end = std::min(
            use_deterministic_algorithms ? INT_MAX : segment_start + max_segment_length_per_cta,
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1]);
        const int32_t SL = segment_end - segment_start;
        const int32_t warp_id = threadIdx.y;
        const int32_t lane_id = threadIdx.x;

        // Note that with shared embedding tables we can have multiple tables
        // (i.e. different values of `t` sharing the same segment).
        //
        const auto info_0 = sorted_infos[segment_start];

        {% if not nobag %}
        int32_t t_0 = fd.Div(info_0); //info_0 / B;
        {% else %}
        int32_t t_0 = info_0 % T;
        {% endif %}

        int64_t hash_size = hash_size_cumsum[t_0];
        {% if not nobag %}
        int32_t D = D_offsets[t_0 + 1] - D_offsets[t_0];
        {% endif %}
        int64_t idx = linear_index - hash_size;

        const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
        const int32_t sl_start = SL_per_warp * warp_id;
        const int32_t sl_end = min(SL_per_warp * (warp_id + 1), SL);
        Vec4T<at::acc_type<cache_t, true>> grad_sum[kMaxVecsPerThread];
        for (int32_t sl = sl_start; sl < sl_end; sl += kThreadGroupSize) {
            int32_t sl_j = sl + threadIdx.x;
            {% if not nobag %}
            int32_t b_t = sl_j < sl_end ? sorted_infos[segment_start + sl_j] : 0;
            int32_t b; //= b_t % B;
            int32_t t; //= b_t / B;
            fd.DivMod(b_t, &t, &b);
            int32_t D_start = sl_j < sl_end ? D_offsets[t] : 0;
            {% else %}
            int64_t l_t = sl_j < sl_end ? sorted_infos[segment_start + sl_j] : 0;
            int32_t l = l_t / T;
            {% endif %}
            {% if weighted %}
            at::acc_type<cache_t, true> idx_weight = sl_j < sl_end ? sorted_indice_weights[segment_start + sl_j] : 0.0;
            {% endif %}
            for (int32_t j = 0; j < kThreadGroupSize && sl + j < sl_end; ++j) {
                {% if not nobag %}
                int32_t b_j = SHFL_SYNC(b, j);
                int32_t D_start_j = SHFL_SYNC(D_start, j);
                {% else %}
                int32_t l_j = SHFL_SYNC(l, j);
                {% endif %}

                {% if weighted %}
                at::acc_type<cache_t, true> idx_weight_j = SHFL_SYNC(idx_weight, j);
                {% endif %}

        #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    {% if not nobag %}
                    Vec4T<at::acc_type<grad_t, true>> grad_out_vec(
                        &grad_output[b_j][0] + D_start_j + d);
                    {% else %}
                    Vec4T<at::acc_type<grad_t, true>> grad_out_vec(&grad_output[l_j][d]);
                    {% endif %}
                    {% if weighted %}
                    grad_sum[i].fma_(grad_out_vec, idx_weight_j);
                    {% else %}
                    grad_sum[i].add_(grad_out_vec);
                    {% endif %}
                }
            }
        }
        // do shared memory reduction only if we used multiple warps.
        if (SL > SL_per_warp) {
            struct SharedMemory<Vec4T<at::acc_type<cache_t, true>>> smem;
            Vec4T<at::acc_type<cache_t, true>>* shared_grad_sums = smem.getPointer();

    #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
            shared_grad_sums
                [lane_id + i * kThreadGroupSize +
                warp_id * kMaxVecsPerThread * kThreadGroupSize] = grad_sum[i];
            }
            __syncthreads();
            if (blockDim.y >= 32) {
            if (warp_id < 16) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 16) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 16) {
            if (warp_id < 8) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 8) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 8) {
            if (warp_id < 4) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 4) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 4) {
            if (warp_id < 2) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 2) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (warp_id == 0) {
    #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                grad_sum[i] = vec4_acc(
                    shared_grad_sums
                        [lane_id + i * kThreadGroupSize +
                        warp_id * kMaxVecsPerThread * kThreadGroupSize],
                    shared_grad_sums
                        [lane_id + i * kThreadGroupSize +
                        (warp_id + 1) * kMaxVecsPerThread * kThreadGroupSize]);
            }
            }
        }

        if (warp_id != 0) {
            continue;
        }

        if (num_ctas_on_current_run > 1) {
            int really_long_run_id = long_run_id_to_really_long_run_ids[long_run_id];
            Vec4T<at::acc_type<cache_t, true>> *temp_grad_accum_ptr =
                reinterpret_cast<Vec4T<at::acc_type<cache_t, true>>*>(&temp_grad_accum[really_long_run_id][0]);
#pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.x, grad_sum[i].acc.x);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.y, grad_sum[i].acc.y);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.z, grad_sum[i].acc.z);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.w, grad_sum[i].acc.w);
            }
            int counter;
            if (threadIdx.x == 0) {
                __threadfence();
                counter = gpuAtomicAdd(&grad_accum_counter[really_long_run_id], -1);
            }
            counter = SHFL_SYNC(counter, 0);
            // Only the thread block accumulated the gradient last does the weight update.
            if (counter > 1) {
                continue;
            }
            CUDA_KERNEL_ASSERT(counter == 1 && "Invalid grad_accum_counter. Race condition?");
#pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                grad_sum[i] = temp_grad_accum_ptr[lane_id + i * kThreadGroupSize];
            }
        }

        int64_t weights_offset = weights_offsets[t_0];
        {% if not dense %}
        emb_t* __restrict__ weights{nullptr};
        cache_t* __restrict__ cache_weights{nullptr};
        int32_t D_emb = D;
        if (std::is_same<emb_t, uint8_t>::value) {
            D_emb += kINT8QparamsBytes;
        }
        const auto weights_placement = static_cast<PlacementType>(weights_placements[t_0]);
        if (weights_placement == PlacementType::DEVICE) {
            weights = &dev_weights[weights_offset + idx * D_emb];
        } else {
            weights = &uvm_weights[weights_offset + idx * D_emb];
        }
        if (weights_placement == PlacementType::MANAGED_CACHING) {
            int32_t cache_idx = sorted_lxu_cache_locations[segment_start];
            if (cache_idx != kCacheLocationMissing) {
                cache_weights = &lxu_cache_weights[cache_idx][0];
            }
        }
        {% for tensor in args.split_tensors %}
        at::acc_type<cache_t, true>* __restrict__ {{ tensor }};
        const auto {{ tensor }}_placement = static_cast<PlacementType>({{ tensor }}_placements[t_0]);
        int64_t {{ tensor }}_offset = {{ tensor }}_offsets[t_0];
        if ({{ tensor }}_placement == PlacementType::DEVICE) {
            {{ tensor }} = &{{ tensor }}_dev[{{ tensor }}_offset];
        } else {
            {{ tensor }} = &{{ tensor }}_uvm[{{ tensor }}_offset];
        }
        {% endfor %}


        struct SharedMemory<Vec4T<at::acc_type<cache_t, true>>> weight_update_buffer;
        Vec4T<at::acc_type<cache_t, true>>* shared_weight_update_row = weight_update_buffer.getPointer();

        auto weight_row_template = WeightRow<emb_t, cache_t, at::acc_type<cache_t, true>>(weights, cache_weights, D, nullptr);
        if (!std::is_same<emb_t, float>::value && stochastic_rounding) {
            StochasticRoundingRNGState state;
            // different for every *run* and every *thread*.
            auto stochastic_rounding_seeds =
                at::cuda::philox::unpack(stochastic_rounding_philox_args);
            stochastic_rounding_init(
                std::get<0>(stochastic_rounding_seeds) ^
                    std::get<1>(stochastic_rounding_seeds),
                threadIdx.x + current_run_id * blockDim.x,
                &state);
            weight_row_template.set_stoc_state(&state);
        }

        float2 qparams_template;
        if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
            qparams_template = weight_row_template.load_qparams();
        }

        {{ split_precomputation }}

        float2 qparams_new;
#pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            Vec4T<at::acc_type<cache_t, true>> weight_new = weight_row_template.load(d, qparams_template);
            auto& grad = grad_sum[i];
            {{ split_weight_update }}
            if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
                shared_weight_update_row[lane_id + i * kThreadGroupSize] = weight_new;
            } else {
                weight_row_template.store(weight_new, d, qparams_new); // qparams_new not used if embedding is not int8
            }
        }
        if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
            // calculate qparams from updated weight row
            qparams_new = thrust_find_qparams<at::acc_type<cache_t, true>>(shared_weight_update_row, D);
            weight_row_template.store_qparams(qparams_new);

#pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                weight_row_template.store(shared_weight_update_row[lane_id + i * kThreadGroupSize], d, qparams_new);
            }
        }
        {% else %}
#pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            auto& grad = grad_sum[i];
            grad.store(&grad_dev_weights[weights_offset + idx * D + d]);
        }
        {% endif %}
    } // for each run
}


template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    size_t kMaxVecsPerThread,
    int32_t kThreadGroupSize = kWarpSize>
__global__
__launch_bounds__(kBackwardMaxThreads)
void
split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_warp_per_row_1(
    const at::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits>
        grad_output,
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {% else %}
    int32_t B,
    int64_t D,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        hash_size_cumsum,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_run,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_cumulative_run_lengths,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {% else %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {% endif %}
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_lxu_cache_locations,
    {% endif %}
    {% if weighted %}
    const at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {% endif %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_num_runs,
    int32_t max_segment_length_per_warp,
    {% if not dense %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {% else %}
    at::PackedTensorAccessor64<cache_t, 1, at::RestrictPtrTraits> grad_dev_weights,
    {% endif %}
    {% if not nobag %}
    FixedDivisor fd,
    {% endif %}
    {{ args.split_kernel_args | join(", ") }}) {

    {% if not nobag %}
    int32_t T = D_offsets.size(0) - 1;
    const int32_t B = grad_output.size(0);
    {% else %}
    int32_t T = weights_offsets.size(0);
    {% endif %}
    const int32_t start_run_id = blockIdx.x * blockDim.y + threadIdx.y;

#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
    const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
    const unsigned int shfl_sync_mask = 0xffffffffu;
#endif
    constexpr int VEC_WIDTH = 4;

    for (uint32_t run_id = start_run_id;
         run_id < sorted_linear_indices_run.size(0) && run_id < sorted_linear_indices_num_runs[0];
         run_id += gridDim.x * blockDim.y) {

    const int64_t linear_index = sorted_linear_indices_run[run_id];
    const int32_t segment_start =
        sorted_linear_indices_cumulative_run_lengths[run_id];
    const int32_t segment_end =
        sorted_linear_indices_cumulative_run_lengths[run_id + 1];
    const int32_t SL = segment_end - segment_start;

    if (SL >= max_segment_length_per_warp) {
        continue;
    }

    // now, each segment corresponds to exactly one table `t` and row in
    // that table (`idx`). Thus, we can hoist out some of the book-keeping.
    const auto info_0 = sorted_infos[segment_start];

    {% if not nobag %}
    int32_t t_0 = fd.Div(info_0); // info_0 / B;
    {% else %}
    int32_t t_0 = info_0 % T;
    {% endif %}

    int64_t hash_size = hash_size_cumsum[t_0];
    {% if not nobag %}
    int32_t D = D_offsets[t_0 + 1] - D_offsets[t_0];
    {% endif %}
    int64_t idx = linear_index - hash_size;

    const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
    const int32_t sl_start = 0;
    const int32_t sl_end = SL;
    Vec4T<at::acc_type<cache_t, true>> grad_sum[kMaxVecsPerThread];
    for (int32_t sl = sl_start; sl < sl_end; sl += kThreadGroupSize) {
        int32_t sl_j = sl + threadIdx.x;
        {% if not nobag %}
        int32_t b_t = sl_j < sl_end ? sorted_infos[segment_start + sl_j] : 0;
        int32_t b; //= b_t % B;
        int32_t t; //= b_t / B;
        fd.DivMod(b_t, &t, &b);
        int32_t D_start = D_offsets[t];
        {% else %}
        int64_t l_t = sl_j < sl_end ? sorted_infos[segment_start + sl_j] : 0;
        int32_t l = l_t / T;
        {% endif %}
        {% if weighted %}
        at::acc_type<cache_t, true> idx_weight = sl_j < sl_end ? sorted_indice_weights[segment_start + sl_j] : 0.0;
        {% endif %}

        for (int32_t j = 0; j < kThreadGroupSize && sl + j < sl_end; ++j) {
            {% if not nobag %}
            int32_t b_j = SHFL_SYNC(b, j);
            int32_t D_start_j = SHFL_SYNC(D_start, j);
            {% else %}
            int32_t l_j = SHFL_SYNC(l, j);
            {% endif %}
            {% if weighted %}
            at::acc_type<cache_t, true> idx_weight_j = SHFL_SYNC(idx_weight, j);
            {% endif %}

            #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                {% if not nobag %}
                Vec4T<at::acc_type<grad_t, true>> grad_out_vec(
                    &grad_output[b_j][0] + D_start_j + d);
                {% else %}
                Vec4T<at::acc_type<grad_t, true>> grad_out_vec(&grad_output[l_j][d]);
                {% endif %}
                {% if weighted %}
                grad_sum[i].fma_(grad_out_vec, idx_weight_j);
                {% else %}
                grad_sum[i].add_(grad_out_vec);
                {% endif %}
            }
        }
    }
    int64_t weights_offset = weights_offsets[t_0];
    {% if not dense %}
    emb_t* __restrict__ weights{nullptr};
    cache_t* __restrict__ cache_weights{nullptr};
    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }
    const auto weights_placement = static_cast<PlacementType>(weights_placements[t_0]);
    if (weights_placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset + idx * D_emb];
    } else {
        weights = &uvm_weights[weights_offset + idx * D_emb];
    }
    if (weights_placement == PlacementType::MANAGED_CACHING) {
        int32_t cache_idx = sorted_lxu_cache_locations[segment_start];
        if (cache_idx != kCacheLocationMissing) {
            cache_weights = &lxu_cache_weights[cache_idx][0];
        }
    }
    {% for tensor in args.split_tensors %}
    at::acc_type<cache_t, true>* __restrict__ {{ tensor }};
    const auto {{ tensor }}_placement = static_cast<PlacementType>({{ tensor }}_placements[t_0]);
    int64_t {{ tensor }}_offset = {{ tensor }}_offsets[t_0];
    if ({{ tensor }}_placement == PlacementType::DEVICE) {
        {{ tensor }} = &{{ tensor }}_dev[{{ tensor }}_offset];
    } else {
        {{ tensor }} = &{{ tensor }}_uvm[{{ tensor }}_offset];
    }
    {% endfor %}

    struct SharedMemory<Vec4T<at::acc_type<cache_t, true>>> weight_update_buffer;
    Vec4T<at::acc_type<cache_t, true>>* shared_weight_update_row = weight_update_buffer.getPointer();
    auto weight_row_template = WeightRow<emb_t, cache_t, at::acc_type<cache_t, true>>(weights, cache_weights, D, nullptr);
    if (!std::is_same<emb_t, float>::value && stochastic_rounding) {
        StochasticRoundingRNGState state;
        // different for every *run* and every *thread*.
        auto stochastic_rounding_seeds =
            at::cuda::philox::unpack(stochastic_rounding_philox_args);
        stochastic_rounding_init(
            std::get<0>(stochastic_rounding_seeds) ^
                std::get<1>(stochastic_rounding_seeds),
            threadIdx.x + run_id * blockDim.x,
            &state);
        weight_row_template.set_stoc_state(&state);
    }
    float2 qparams_template;
    if (std::is_same<emb_t, uint8_t>::value && !cache_weights){
        qparams_template = weight_row_template.load_qparams();
    }

    {{ split_precomputation }}

    float2 qparams_new;
    #pragma unroll kMaxVecsPerThread
    for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
        int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
        Vec4T<at::acc_type<cache_t, true>> weight_new = weight_row_template.load(d, qparams_template);
        auto& grad = grad_sum[i];
        {{ split_weight_update }}
        if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
            shared_weight_update_row[threadIdx.x + (i + threadIdx.y * kMaxVecsPerThread) * kThreadGroupSize] = weight_new;
        } else {
            weight_row_template.store(weight_new, d, qparams_new); // qparams_new not used if type is not int8
        }
    }

    if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
        // calculate new qparams after row update
        qparams_new = thrust_find_qparams<at::acc_type<cache_t, true>>(&shared_weight_update_row[threadIdx.y * kMaxVecsPerThread * kThreadGroupSize], D);
        weight_row_template.store_qparams(qparams_new);

        // fetch cached updated row from shared mem and quantize on-the-fly when saving to lowp embedding
        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
            int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            weight_row_template.store(shared_weight_update_row[threadIdx.x + (i + threadIdx.y * kMaxVecsPerThread) * kThreadGroupSize], d, qparams_new);
        }
    }
    {% else %}
#pragma unroll kMaxVecsPerThread
    for (int32_t i = 0;
        i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
        ++i) {
        int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
        auto& grad = grad_sum[i];
        grad.store(&grad_dev_weights[weights_offset + idx * D + d]);
    }
    {% endif %}

    }
}

{{ "void" if not dense else "Tensor" }} split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_exact_cuda(
    Tensor grad_output,
    Tensor dev_weights,
    {% if not dense %}
    Tensor uvm_weights,
    Tensor lxu_cache_weights,
    Tensor weights_placements,
    {% endif %}
    Tensor weights_offsets,
    {% if not nobag %}
    Tensor D_offsets,
    int64_t max_D,
    {% else %}
    int64_t D,
    {% endif %}
    Tensor hash_size_cumsum,
    int64_t total_hash_size_bits,
    Tensor indices,
    Tensor offsets,
    {% if not nobag %}
    int64_t pooling_mode,
    {% endif %}
    {% if weighted %}
    Tensor indice_weights,
    {% endif %}
    {% if not dense %}
    Tensor lxu_cache_locations,
    {% endif %}
    int64_t unused_,
    int64_t max_segment_length_per_warp,
    {% if not dense %}
    bool stochastic_rounding,
    {% endif %}
    {{ args.split_function_args | join(", ") }}) {

    TENSOR_ON_CUDA_GPU(grad_output);
    TENSOR_ON_CUDA_GPU(dev_weights);
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(uvm_weights);
    TENSOR_ON_CUDA_GPU(lxu_cache_weights);
    TENSOR_ON_CUDA_GPU(weights_placements);
    {% endif %}
    TENSOR_ON_CUDA_GPU(weights_offsets);
    {% if not nobag %}
    TENSOR_ON_CUDA_GPU(D_offsets);
    {% endif %}
    TENSOR_ON_CUDA_GPU(hash_size_cumsum);
    TENSOR_ON_CUDA_GPU(indices);
    TENSOR_ON_CUDA_GPU(offsets);
    {% if weighted %}
    TENSOR_ON_CUDA_GPU(indice_weights);
    {% endif %}
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(lxu_cache_locations);
    {% endif %}

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());

    {% if dense %}
    auto grad_dev_weights = zeros_like(dev_weights);
    {% endif %}

    // short-circuit if there are zero indices.
    if (indices.numel() == 0) {
        return {{ "grad_dev_weights" if dense else "" }};
    }

    {% if not nobag %}
    int32_t T = D_offsets.numel() - 1;
    {% else %}
    int32_t T = weights_offsets.numel();
    {% endif %}

    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    const auto B = (offsets.size(0) - 1) / T;
    TORCH_CHECK(B > 0);
    auto BT_block_size = kMaxThreads / kWarpSize;
    TORCH_CHECK(BT_block_size * kWarpSize <= kMaxThreads);
    {% if nobag %}
    auto max_D = D;
    {% endif %}
    TORCH_CHECK(max_D <= {{ max_embedding_dim }});

    // V100: 96 KB; A100: 160 KB.
    int max_shared_bytes = 0;
#ifndef __HIP_PLATFORM_HCC__
    hipDeviceGetAttribute(&max_shared_bytes, hipDeviceAttributeSharedMemPerBlockOptin, dev_weights.get_device());
#else
    // MI100 has 64 KB local memory (shared memory) per workgroup
    max_shared_bytes = 64 << 10;
#endif
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    int shared_kb = max_shared_bytes >> 10;
    // V100: 64 KB; A100: 96 KB.
#ifndef __HIP_PLATFORM_HCC__
    // Use 2/3 of the available GPU shared mem; leave rooms for L1$.
    int used_shared_kb = round_down(shared_kb * 2 / 3, 16);
    TORCH_CHECK(used_shared_kb > 0);
#else
    // MI100 has independent shared mem and L1
    int used_shared_kb = shared_kb;
#endif
    int used_shared_bytes = used_shared_kb << 10;

    Tensor linear_indices, linear_indices_sorted;
    Tensor infos_sorted;
    Tensor sorted_linear_indices_run, sorted_linear_indices_run_lengths,
        sorted_linear_indices_num_runs,
        sorted_linear_indices_cumulative_run_lengths;
    std::tie(
        linear_indices,
        linear_indices_sorted,
        infos_sorted,
        sorted_linear_indices_run,
        sorted_linear_indices_run_lengths,
        sorted_linear_indices_num_runs,
        sorted_linear_indices_cumulative_run_lengths) =
        transpose_embedding_input(
            hash_size_cumsum,
            total_hash_size_bits,
            indices,
            offsets,
            {{"true" if nobag else "false"}});

    {% if not dense %}
    auto lxu_cache_locations_sorted = at::empty_like(lxu_cache_locations);
    if (lxu_cache_locations.size(0) > 0) {
        size_t temp_storage_bytes = 0;
        AT_CUDA_CHECK(radix_sort_pairs(
            nullptr,
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            lxu_cache_locations.data_ptr<int32_t>(),
            lxu_cache_locations_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
        auto temp_storage = at::empty(
            {static_cast<int64_t>(temp_storage_bytes)},
            indices.options().dtype(at::kByte));
        AT_CUDA_CHECK(radix_sort_pairs(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            lxu_cache_locations.data_ptr<int32_t>(),
            lxu_cache_locations_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
    }
    {% endif %}

    DISPATCH_EMB_GRAD_CACHE_TYPES(
        dev_weights.scalar_type(),
        grad_output.scalar_type(),
        {% if not dense %}
        lxu_cache_weights.scalar_type(),
        {% else %}
        dev_weights.scalar_type(),
        {% endif %}
            "split_embedding_backward_{{ optimizer }}_exact_kernel",
        [&] {
            {% if weighted %}
            auto indice_weights_sorted = at::empty_like(indice_weights);
            {
            size_t temp_storage_bytes = 0;
            AT_CUDA_CHECK(radix_sort_pairs(
                nullptr,
                temp_storage_bytes,
                linear_indices.data_ptr<int64_t>(),
                linear_indices_sorted.data_ptr<int64_t>(),
                indice_weights.data_ptr<at::acc_type<cache_t, true>>(),
                indice_weights_sorted.data_ptr<at::acc_type<cache_t, true>>(),
                linear_indices.numel(),
                0,
                total_hash_size_bits,
                at::cuda::getCurrentCUDAStream(),
                false));
            auto temp_storage = at::empty(
                {static_cast<int64_t>(temp_storage_bytes)},
                indices.options().dtype(at::kByte));
            AT_CUDA_CHECK(radix_sort_pairs(
                temp_storage.data_ptr(),
                temp_storage_bytes,
                linear_indices.data_ptr<int64_t>(),
                linear_indices_sorted.data_ptr<int64_t>(),
                indice_weights.data_ptr<at::acc_type<cache_t, true>>(),
                indice_weights_sorted.data_ptr<at::acc_type<cache_t, true>>(),
                linear_indices.numel(),
                0,
                total_hash_size_bits,
                at::cuda::getCurrentCUDAStream(),
                false));
            }
            {% endif %}

            // early memory release
            linear_indices.reset();
            linear_indices_sorted.reset();

            auto grad_output_accessor = grad_output.packed_accessor64<grad_t, 2, at::RestrictPtrTraits>();
            {% if not nobag %}
            Tensor grad_output_mean;
            if (static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN) {
              grad_output_mean = at::empty_like(grad_output);
              grad_mean_kernel<grad_t>
                  <<<div_round_up((B * T), kMaxThreads / kWarpSize),
                     dim3(kWarpSize, kMaxThreads / kWarpSize),
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      grad_output_accessor,
                      D_offsets
                          .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                      offsets
                          .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                      grad_output_mean.packed_accessor64<
                          grad_t, 2, at::RestrictPtrTraits>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              grad_output_accessor = grad_output_mean.packed_accessor64<
                  grad_t, 2, at::RestrictPtrTraits>();
            }
            {% endif %}

            {% if not dense %}
            at::PhiloxCudaState rng_engine_inputs;
            if (stochastic_rounding && !std::is_same<emb_t, float>::value) {
                auto gen = at::cuda::detail::getDefaultCUDAGenerator();
                std::lock_guard<std::mutex> lock(gen.mutex());
                rng_engine_inputs =
                    at::check_generator<at::CUDAGeneratorImpl>(gen)
                        ->philox_cuda_state(4);
            }
            {% endif %}
            // kMaxElemPerThread is # of elements handled by thread if we use a full warp for a row
            // We consider kMaxElemPerThread 1 and 2, and then a multiple of 4.
            {% for kMaxElemPerThread in range(1, max_embedding_dim // (items_per_warp // 4) + 1) %}
            {% if kMaxElemPerThread in [1, 2] or kMaxElemPerThread % 4 == 0 %}
            if (max_D <= {{ items_per_warp // 4 * kMaxElemPerThread }}) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = {{ kMaxElemPerThread }} / 4 >= 1 ? {{ kMaxElemPerThread }} / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / {{ kMaxElemPerThread }}, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty_like(sorted_linear_indices_run_lengths);
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty_like(sorted_linear_indices_run_lengths);
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

            split_embedding_backward_codegen_{{ optimizer }}_{{ wdesc }}_find_long_segments<<<
                div_round_up(indices.numel(), kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                sorted_linear_indices_num_runs.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                sorted_linear_indices_run_lengths.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                num_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                long_run_id_to_really_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                num_really_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                grad_accum_counter.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(long_run_ids.numel(), kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB.
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    {% if not dense %}
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                    weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    {% if not nobag %}
                    D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    B,
                    D,
                    {% endif %}
                    hash_size_cumsum.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    sorted_linear_indices_run
                        .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    sorted_linear_indices_cumulative_run_lengths
                        .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    num_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% if not nobag %}
                    infos_sorted.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    infos_sorted.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not dense %}
                    lxu_cache_locations_sorted.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if weighted %}
                    indice_weights_sorted.packed_accessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not dense %}
                    stochastic_rounding,
                    rng_engine_inputs,
                    {% else %}
                    grad_dev_weights.packed_accessor64<cache_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not nobag %}
                    FixedDivisor(B),
                    {% endif %}
                    long_run_id_to_really_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    temp_grad_accum.packed_accessor32<at::acc_type<cache_t, true>, 2, at::RestrictPtrTraits>(),
                    grad_accum_counter.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    {{ args.split_kernel_arg_constructors | join(", ") }});
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(sorted_linear_indices_run.numel(), kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB.
#endif
            }

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    {% if not dense %}
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                    weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    {% if not nobag %}
                    D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    B,
                    D,
                    {% endif %}
                    hash_size_cumsum.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    sorted_linear_indices_run
                        .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    sorted_linear_indices_cumulative_run_lengths
                        .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% if not nobag %}
                    infos_sorted.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    infos_sorted.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not dense %}
                    lxu_cache_locations_sorted.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if weighted %}
                    indice_weights_sorted.packed_accessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    sorted_linear_indices_num_runs
                        .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    max_segment_length_per_warp,
                    {% if not dense %}
                    stochastic_rounding,
                    rng_engine_inputs,
                    {% else %}
                    grad_dev_weights.packed_accessor64<cache_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not nobag %}
                    FixedDivisor(B),
                    {% endif %}
                    {{ args.split_kernel_arg_constructors | join(", ") }});
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
        {% endif %}
        {% endfor %}
        });

    return {{ "grad_dev_weights" if dense else "" }};
}
{% endif %}
{% endfor %}
// clang-format on
