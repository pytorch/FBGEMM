#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
{% set wdesc = "weighted" if weighted else "unweighted" %}
#include "codegen/embedding_backward_template_helpers.cuh"

{% if not dense %}
constexpr int32_t kCacheLocationMissing = -1;
{% endif %}
enum {
  DEVICE = 0,
  MANAGED = 1,
  MANAGED_CACHING = 2,
};

constexpr size_t kBackwardMaxThreads = 512;

using namespace at;
using namespace fbgemm_gpu;

template <
    typename emb_t,
    typename cache_t,
    size_t kMaxVecsPerThread>
__global__ void
__launch_bounds__(kMaxThreads)
split_embedding_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1(
    const PackedTensorAccessor32<acc_type<cache_t, true>, 2, RestrictPtrTraits>
        grad_output,
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> dev_weights,
    {% if not dense %}
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> uvm_weights,
    PackedTensorAccessor64<cache_t, 2, RestrictPtrTraits> lxu_cache_weights,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> weights_offsets,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> D_offsets,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        hash_size_cumsum,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        sorted_linear_indices_run,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_linear_indices_cumulative_run_lengths,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_linear_indices_run_lengths,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> sorted_infos,
    {% if not dense %}
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_lxu_cache_locations,
    {% endif %}
    {% if weighted %}
    const PackedTensorAccessor32<acc_type<cache_t, true>, 1, RestrictPtrTraits> sorted_indice_weights,
    {% endif %}
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_linear_indices_num_runs,
    int32_t max_segment_length_per_warp,
    {% if not dense %}
    bool stochastic_rounding,
    PhiloxCudaState stochastic_rounding_philox_args,
    {% else %}
    PackedTensorAccessor64<cache_t, 1, RestrictPtrTraits> grad_dev_weights,
    {% endif %}
    FixedDivisor fd,
    {{ args.split_kernel_args | join(", ") }}) {
  int32_t T = D_offsets.size(0) - 1;
  const int32_t B = grad_output.size(0);
  const int32_t num_runs = sorted_linear_indices_num_runs[0];

  for (int32_t run_id = blockIdx.x * kWarpSize; run_id < num_runs;
       run_id += kWarpSize * gridDim.x) {
        const int32_t candidate_run_id = run_id + threadIdx.x;
        int candidate_run_active = candidate_run_id < num_runs &&
            sorted_linear_indices_run_lengths[candidate_run_id] >=
                max_segment_length_per_warp;
        uint32_t candidate_mask = __ballot_sync(0xFFFFFFFF, candidate_run_active);
    while (candidate_mask != 0) {
        int32_t current_thread_id = __ffs(candidate_mask) - 1;
        candidate_mask ^= (static_cast<uint32_t>(1) << current_thread_id);
        int32_t current_run_id = run_id + current_thread_id;
        const int64_t linear_index = sorted_linear_indices_run[current_run_id];
        const int32_t segment_start =
            sorted_linear_indices_cumulative_run_lengths[current_run_id];
        const int32_t segment_end =
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1];
        const int32_t SL = segment_end - segment_start;
        // TODO: should never be hit!
        if (SL < max_segment_length_per_warp) {
            return;
        }
        const int32_t warp_id = threadIdx.y;
        const int32_t lane_id = threadIdx.x;

        // Note that with shared embedding tables we can have multiple tables
        // (i.e. different values of `t` sharing the same segment).
        //
        const auto info_0 = sorted_infos[segment_start];
        int32_t t_0 = fd.Div(info_0); //info_0 / B;
        int64_t hash_size = hash_size_cumsum[t_0];
        int32_t D = D_offsets[t_0 + 1] - D_offsets[t_0];
        int64_t idx = linear_index - hash_size;

        const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
        const int32_t sl_start = SL_per_warp * warp_id;
        const int32_t sl_end = min(SL_per_warp * (warp_id + 1), SL);
        Vec4T<acc_type<cache_t, true>> grad_sum[kMaxVecsPerThread];
        for (int32_t sl = sl_start; sl < sl_end; sl += kWarpSize) {
            int32_t sl_j = sl + threadIdx.x;
            int32_t b_t = sl_j < sl_end ? sorted_infos[segment_start + sl_j] : 0;
            int32_t b; //= b_t % B;
            int32_t t; //= b_t / B;
            fd.DivMod(b_t, &t, &b);
            int32_t D_start = sl_j < sl_end ? D_offsets[t] : 0;
            {% if weighted %}
            acc_type<cache_t, true> idx_weight = sl_j < sl_end ? sorted_indice_weights[segment_start + sl_j] : 0.0;
            {% endif %}

            for (int32_t j = 0; j < kWarpSize && sl + j < sl_end; ++j) {
                int32_t b_j = __shfl_sync(0xFFFFFFFF, b, j);
                int32_t D_start_j = __shfl_sync(0xFFFFFFFF, D_start, j);
                {% if weighted %}
                acc_type<cache_t, true> idx_weight_j = __shfl_sync(0xFFFFFFFF, idx_weight, j);
                {% endif %}

        #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                    ++i) {
                    int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
                    Vec4T<acc_type<cache_t, true>> grad_out_vec(
                        &grad_output[b_j][0] + D_start_j + d);
                    {% if weighted %}
                    grad_sum[i].fma_(grad_out_vec, idx_weight_j);
                    {% else %}
                    grad_sum[i].acc.x += grad_out_vec.acc.x;
                    grad_sum[i].acc.y += grad_out_vec.acc.y;
                    grad_sum[i].acc.z += grad_out_vec.acc.z;
                    grad_sum[i].acc.w += grad_out_vec.acc.w;
                    {% endif %}
                }
            }
        }
        // do shared memory reduction only if we used multiple blocks.
        if (SL > SL_per_warp) {
            struct SharedMemory<Vec4T<acc_type<cache_t, true>>> smem;
            Vec4T<acc_type<cache_t, true>>* shared_grad_sums = smem.getPointer();

    #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                ++i) {
            shared_grad_sums
                [lane_id + i * kWarpSize +
                warp_id * kMaxVecsPerThread * kWarpSize] = grad_sum[i];
            }
            __syncthreads();
            if (blockDim.y >= 32) {
            if (warp_id < 16) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    4 * kWarpSize * i + threadIdx.x * 4 < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kWarpSize +
                    warp_id * kMaxVecsPerThread * kWarpSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kWarpSize +
                                warp_id * kMaxVecsPerThread * kWarpSize],
                            shared_grad_sums
                                [lane_id + i * kWarpSize +
                                (warp_id + 16) * kMaxVecsPerThread * kWarpSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 16) {
            if (warp_id < 8) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    4 * kWarpSize * i + threadIdx.x * 4 < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kWarpSize +
                    warp_id * kMaxVecsPerThread * kWarpSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kWarpSize +
                                warp_id * kMaxVecsPerThread * kWarpSize],
                            shared_grad_sums
                                [lane_id + i * kWarpSize +
                                (warp_id + 8) * kMaxVecsPerThread * kWarpSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 8) {
            if (warp_id < 4) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    4 * kWarpSize * i + threadIdx.x * 4 < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kWarpSize +
                    warp_id * kMaxVecsPerThread * kWarpSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kWarpSize +
                                warp_id * kMaxVecsPerThread * kWarpSize],
                            shared_grad_sums
                                [lane_id + i * kWarpSize +
                                (warp_id + 4) * kMaxVecsPerThread * kWarpSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 4) {
            if (warp_id < 2) {
    #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    4 * kWarpSize * i + threadIdx.x * 4 < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kWarpSize +
                    warp_id * kMaxVecsPerThread * kWarpSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kWarpSize +
                                warp_id * kMaxVecsPerThread * kWarpSize],
                            shared_grad_sums
                                [lane_id + i * kWarpSize +
                                (warp_id + 2) * kMaxVecsPerThread * kWarpSize]);
                }
            }
            __syncthreads();
            }
            if (warp_id == 0) {
    #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                ++i) {
                grad_sum[i] = vec4_acc(
                    shared_grad_sums
                        [lane_id + i * kWarpSize +
                        warp_id * kMaxVecsPerThread * kWarpSize],
                    shared_grad_sums
                        [lane_id + i * kWarpSize +
                        (warp_id + 1) * kMaxVecsPerThread * kWarpSize]);
            }
            }
        }

        if (warp_id == 0) {
            int64_t weights_offset = weights_offsets[t_0];
            {% if not dense %}
            emb_t* __restrict__ weights{nullptr};
            cache_t* __restrict__ cache_weights{nullptr};
            int32_t D_emb = D;
            if (std::is_same<emb_t, uint8_t>::value) {
                D_emb += kINT8QparamsBytes;
            }
            const auto weights_placement = weights_placements[t_0];
            if (weights_placement == DEVICE) {
                weights = &dev_weights[weights_offset + idx * D_emb];
            } else {
                weights = &uvm_weights[weights_offset + idx * D_emb];
            }
            if (weights_placement == MANAGED_CACHING) {
                int32_t cache_idx = sorted_lxu_cache_locations[segment_start];
                if (cache_idx != kCacheLocationMissing) {
                    cache_weights = &lxu_cache_weights[cache_idx][0];
                }
            }
            {% for tensor in args.split_tensors %}
            acc_type<cache_t, true>* __restrict__ {{ tensor }};
            const auto {{ tensor }}_placement = {{ tensor }}_placements[t_0];
            int64_t {{ tensor }}_offset = {{ tensor }}_offsets[t_0];
            if ({{ tensor }}_placement == DEVICE) {
                {{ tensor }} = &{{ tensor }}_dev[{{ tensor }}_offset];
            } else {
                {{ tensor }} = &{{ tensor }}_uvm[{{ tensor }}_offset];
            }
            {% endfor %}

            {{ split_precomputation }}

            struct SharedMemory<Vec4T<acc_type<cache_t, true>>> weight_update_buffer;
            Vec4T<acc_type<cache_t, true>>* shared_weight_update_row = weight_update_buffer.getPointer();

            auto weight_row_template = WeightRow<emb_t, cache_t, acc_type<cache_t, true>>(weights, cache_weights, D, nullptr);
            if (!std::is_same<emb_t, float>::value && stochastic_rounding) {
                StochasticRoundingRNGState state;
                // different for every *run* and every *thread*.
                auto stochastic_rounding_seeds =
                    at::cuda::philox::unpack(stochastic_rounding_philox_args);
                stochastic_rounding_init(
                    std::get<0>(stochastic_rounding_seeds) ^
                        std::get<1>(stochastic_rounding_seeds),
                    threadIdx.x + current_run_id * blockDim.x,
                    &state);
                weight_row_template.set_stoc_state(&state);
            }

            float2 qparams_template;
            if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
                qparams_template = weight_row_template.load_qparams();
            }
            float2 qparams_new;
            #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                    i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                    ++i) {
                int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
                Vec4T<acc_type<cache_t, true>> weight_new = weight_row_template.load(d, qparams_template);
                auto& grad = grad_sum[i];
                {{ split_weight_update }}
                if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
                    shared_weight_update_row[lane_id + i * kWarpSize] = weight_new;
                } else {
                    weight_row_template.store(weight_new, d, qparams_new); // qparams_new not used if embedding is not int8
                }
            }
            if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
                // calculate qparams from updated weight row
                qparams_new = thrust_find_qparams<acc_type<cache_t, true>>(shared_weight_update_row, D);
                weight_row_template.store_qparams(qparams_new);

                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                        i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                        ++i) {
                    int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
                    weight_row_template.store(shared_weight_update_row[lane_id + i * kWarpSize], d, qparams_new);
                }
            }
            {% else %}
        #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                ++i) {
                int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
                auto& grad = grad_sum[i];
                grad.store(&grad_dev_weights[weights_offset + idx * D + d]);
            }
            {% endif %}
        }
    }
  }
}


template <
    typename emb_t,
    typename cache_t,
    size_t kMaxVecsPerThread>
__global__
__launch_bounds__(kBackwardMaxThreads)
void
split_embedding_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_warp_per_row_1(
    const PackedTensorAccessor32<acc_type<cache_t,true>, 2, RestrictPtrTraits>
        grad_output,
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> dev_weights,
    {% if not dense %}
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> uvm_weights,
    PackedTensorAccessor64<cache_t, 2, RestrictPtrTraits> lxu_cache_weights,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> weights_offsets,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> D_offsets,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        hash_size_cumsum,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        sorted_linear_indices_run,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_linear_indices_cumulative_run_lengths,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_linear_indices_run_lengths,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> sorted_infos,
    {% if not dense %}
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_lxu_cache_locations,
    {% endif %}
    {% if weighted %}
    const PackedTensorAccessor32<acc_type<cache_t, true>, 1, RestrictPtrTraits> sorted_indice_weights,
    {% endif %}
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_linear_indices_num_runs,
    int32_t max_segment_length_per_warp,
    {% if not dense %}
    bool stochastic_rounding,
    PhiloxCudaState stochastic_rounding_philox_args,
    {% else %}
    PackedTensorAccessor64<cache_t, 1, RestrictPtrTraits> grad_dev_weights,
    {% endif %}
    FixedDivisor fd,
    {{ args.split_kernel_args | join(", ") }}) {

    const int32_t T = D_offsets.size(0) - 1;
    const int32_t B = grad_output.size(0);
    const int32_t run_id = blockIdx.x * blockDim.y + threadIdx.y;

    if (run_id >= sorted_linear_indices_run.size(0)) {
        return;
    }
    if (run_id >= sorted_linear_indices_num_runs[0]) {
        return;
    }
    const int64_t linear_index = sorted_linear_indices_run[run_id];
    const int32_t segment_start =
        sorted_linear_indices_cumulative_run_lengths[run_id];
    const int32_t segment_end =
        sorted_linear_indices_cumulative_run_lengths[run_id + 1];
    const int32_t SL = segment_end - segment_start;

    if (SL >= max_segment_length_per_warp) {
        return;
    }

    // now, each segment corresponds to exactly one table `t` and row in
    // that table (`idx`). Thus, we can hoist out some of the book-keeping.
    const auto info_0 = sorted_infos[segment_start];
    int32_t t_0 = fd.Div(info_0); // info_0 / B;

    int64_t hash_size = hash_size_cumsum[t_0];
    int32_t D = D_offsets[t_0 + 1] - D_offsets[t_0];
    int64_t idx = linear_index - hash_size;

    const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
    const int32_t sl_start = 0;
    const int32_t sl_end = SL;
    Vec4T<acc_type<cache_t, true>> grad_sum[kMaxVecsPerThread];
    for (int32_t sl = sl_start; sl < sl_end; sl += kWarpSize) {
        int32_t sl_j = sl + threadIdx.x;
        int32_t b_t = sl_j < sl_end ? sorted_infos[segment_start + sl_j] : 0;
        int32_t b; //= b_t % B;
        int32_t t; //= b_t / B;
        fd.DivMod(b_t, &t, &b);
        int32_t D_start = D_offsets[t];
        {% if weighted %}
        acc_type<cache_t, true> idx_weight = sl_j < sl_end ? sorted_indice_weights[segment_start + sl_j] : 0.0;
        {% endif %}

        for (int32_t j = 0; j < kWarpSize && sl + j < sl_end; ++j) {
            int32_t b_j = __shfl_sync(0xFFFFFFFF, b, j);
            int32_t D_start_j = __shfl_sync(0xFFFFFFFF, D_start, j);
            {% if weighted %}
            acc_type<cache_t, true> idx_weight_j = __shfl_sync(0xFFFFFFFF, idx_weight, j);
            {% endif %}

    #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
            ++i) {
            int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
            Vec4T<acc_type<cache_t, true>> grad_out_vec(
                &grad_output[b_j][0] + D_start_j + d);
                {% if weighted %}
                grad_sum[i].fma_(grad_out_vec, idx_weight_j);
                {% else %}
                grad_sum[i].acc.x += grad_out_vec.acc.x;
                grad_sum[i].acc.y += grad_out_vec.acc.y;
                grad_sum[i].acc.z += grad_out_vec.acc.z;
                grad_sum[i].acc.w += grad_out_vec.acc.w;
                {% endif %}
            }
        }
    }
    int64_t weights_offset = weights_offsets[t_0];
    {% if not dense %}
    emb_t* __restrict__ weights{nullptr};
    cache_t* __restrict__ cache_weights{nullptr};
    int32_t D_emb = D;
    if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
    }
    const auto weights_placement = weights_placements[t_0];
    if (weights_placement == DEVICE) {
        weights = &dev_weights[weights_offset + idx * D_emb];
    } else {
        weights = &uvm_weights[weights_offset + idx * D_emb];
    }
    if (weights_placement == MANAGED_CACHING) {
        int32_t cache_idx = sorted_lxu_cache_locations[segment_start];
        if (cache_idx != kCacheLocationMissing) {
            cache_weights = &lxu_cache_weights[cache_idx][0];
        }
    }
    {% for tensor in args.split_tensors %}
    acc_type<cache_t, true>* __restrict__ {{ tensor }};
    const auto {{ tensor }}_placement = {{ tensor }}_placements[t_0];
    int64_t {{ tensor }}_offset = {{ tensor }}_offsets[t_0];
    if ({{ tensor }}_placement == DEVICE) {
        {{ tensor }} = &{{ tensor }}_dev[{{ tensor }}_offset];
    } else {
        {{ tensor }} = &{{ tensor }}_uvm[{{ tensor }}_offset];
    }
    {% endfor %}

    {{ split_precomputation }}
    struct SharedMemory<Vec4T<acc_type<cache_t, true>>> weight_update_buffer;
    Vec4T<acc_type<cache_t, true>>* shared_weight_update_row = weight_update_buffer.getPointer();
    auto weight_row_template = WeightRow<emb_t, cache_t, acc_type<cache_t, true>>(weights, cache_weights, D, nullptr);
    if (!std::is_same<emb_t, float>::value && stochastic_rounding) {
        StochasticRoundingRNGState state;
        // different for every *run* and every *thread*.
        auto stochastic_rounding_seeds =
            at::cuda::philox::unpack(stochastic_rounding_philox_args);
        stochastic_rounding_init(
            std::get<0>(stochastic_rounding_seeds) ^
                std::get<1>(stochastic_rounding_seeds),
            threadIdx.x + run_id * blockDim.x,
            &state);
        weight_row_template.set_stoc_state(&state);
    }
    float2 qparams_template;
    if (std::is_same<emb_t, uint8_t>::value && !cache_weights){
        qparams_template = weight_row_template.load_qparams();
    }
    float2 qparams_new;
    #pragma unroll kMaxVecsPerThread
    for (int32_t i = 0;
            i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
            ++i) {
        int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
        Vec4T<acc_type<cache_t, true>> weight_new = weight_row_template.load(d, qparams_template);
        auto& grad = grad_sum[i];
        {{ split_weight_update }}
        if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
            shared_weight_update_row[threadIdx.x + i * kWarpSize + threadIdx.y * kMaxVecsPerThread * kWarpSize] = weight_new;
        } else {
            weight_row_template.store(weight_new, d, qparams_new); // qparams_new not used if type is not int8
        }
    }

    if (std::is_same<emb_t, uint8_t>::value && !cache_weights) {
        // calculate new qparams after row update
        qparams_new = thrust_find_qparams<acc_type<cache_t, true>>(&shared_weight_update_row[threadIdx.y * kMaxVecsPerThread * kWarpSize], D);
        weight_row_template.store_qparams(qparams_new);

        // fetch cached updated row from shared mem and quantize on-the-fly when saving to lowp embedding
        #pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
                i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
                ++i) {
            int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
            weight_row_template.store(shared_weight_update_row[threadIdx.x  + i * kWarpSize + threadIdx.y * kMaxVecsPerThread * kWarpSize], d, qparams_new);
        }
    }
    {% else %}
#pragma unroll kMaxVecsPerThread
    for (int32_t i = 0;
        i < kMaxVecsPerThread && 4 * kWarpSize * i + threadIdx.x * 4 < D;
        ++i) {
        int32_t d = 4 * kWarpSize * i + threadIdx.x * 4;
        auto& grad = grad_sum[i];
        grad.store(&grad_dev_weights[weights_offset + idx * D + d]);
    }
    {% endif %}
}

template <typename cache_t, typename emb_t>
__global__ void __launch_bounds__(kMaxThreads) grad_mean_kernel(
    const PackedTensorAccessor32<acc_type<cache_t, true>, 2, RestrictPtrTraits>
        grad_output,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> D_offsets,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> offsets,
    PackedTensorAccessor32<acc_type<cache_t, true>, 2, RestrictPtrTraits>
        grad_output_mean) {
  int32_t B = grad_output.size(0);
  int32_t T = D_offsets.size(0) - 1;
  int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
  int32_t b = b_t % B;
  int32_t t = b_t / B;

  if (b_t >= B * T) {
    return;
  }
  int32_t D_start = D_offsets[t];
  int32_t D_end = D_offsets[t + 1];
  int32_t D = D_end - D_start;
  int64_t indices_start = offsets[t * B + b];
  int64_t indices_end = offsets[t * B + b + 1];
  int32_t L = indices_end - indices_start;

  if (L != 0) {
    for (int32_t d = threadIdx.x; d * 4 < D; d += blockDim.x) {
      Vec4T<acc_type<cache_t, true>> grad_out_vec(&grad_output[b][D_start + d * 4]);
      grad_out_vec.acc.x /= L;
      grad_out_vec.acc.y /= L;
      grad_out_vec.acc.z /= L;
      grad_out_vec.acc.w /= L;
      grad_out_vec.store(&grad_output_mean[b][D_start + d * 4]);
    }
  } else {
    for (int32_t d = threadIdx.x; d * 4 < D; d += blockDim.x) {
      Vec4T<acc_type<cache_t, true>> grad_out_vec(&grad_output[b][D_start + d * 4]);
      grad_out_vec.store(&grad_output_mean[b][D_start + d * 4]);
    }
  }
}

{{ "void" if not dense else "Tensor" }} split_embedding_backward_codegen_{{ optimizer }}_{{ wdesc }}_exact_cuda(
    Tensor grad_output,
    Tensor dev_weights,
    {% if not dense %}
    Tensor uvm_weights,
    Tensor lxu_cache_weights,
    Tensor weights_placements,
    {% endif %}
    Tensor weights_offsets,
    Tensor D_offsets,
    int64_t max_D,
    Tensor hash_size_cumsum,
    int64_t total_hash_size_bits,
    Tensor indices,
    Tensor offsets,
    int64_t pooling_mode,
    {% if weighted %}
    Tensor indice_weights,
    {% endif %}
    {% if not dense %}
    Tensor lxu_cache_locations,
    {% endif %}
    int64_t unused_,
    int64_t max_segment_length_per_warp,
    {% if not dense %}
    bool stochastic_rounding,
    {% endif %}
    {{ args.split_function_args | join(", ") }}) {
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());

    {% if dense %}
    auto grad_dev_weights = zeros_like(dev_weights);
    {% endif %}

    // short-circuit if there are zero indices.
    if (indices.numel() == 0) {
        return {{ "grad_dev_weights" if dense else "" }};
    }

    int32_t T = D_offsets.numel() - 1;
    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    const auto B = (offsets.size(0) - 1) / T;
    TORCH_CHECK(B > 0);
    auto BT_block_size = kMaxThreads / kWarpSize;
    TORCH_CHECK(BT_block_size * kWarpSize <= kMaxThreads);
    TORCH_CHECK(max_D <= {{ max_embedding_dim }});

    auto infos = at::empty_like(indices, indices.options().dtype(kInt));
    auto infos_sorted = at::empty_like(infos);
    auto linear_indices = at::empty_like(indices);
    auto linear_indices_sorted = at::empty_like(indices);
    linearize_index_kernel<<<
        div_round_up(B * T, kMaxThreads),
        kMaxThreads,
        0,
        at::cuda::getCurrentCUDAStream()>>>(
        hash_size_cumsum.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
        indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
        offsets.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
        infos.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
        linear_indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    {
        size_t temp_storage_bytes = 0;
        AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
            nullptr,
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            infos.data_ptr<int32_t>(),
            infos_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
        auto temp_storage = at::empty(
            {static_cast<int64_t>(temp_storage_bytes)},
            indices.options().dtype(kByte));
        AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            infos.data_ptr<int32_t>(),
            infos_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
    }
    {% if not dense %}
    auto lxu_cache_locations_sorted = at::empty_like(lxu_cache_locations);
    if (lxu_cache_locations.size(0) > 0) {
        size_t temp_storage_bytes = 0;
        AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
            nullptr,
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            lxu_cache_locations.data_ptr<int32_t>(),
            lxu_cache_locations_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
        auto temp_storage = at::empty(
            {static_cast<int64_t>(temp_storage_bytes)},
            indices.options().dtype(kByte));
        AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            lxu_cache_locations.data_ptr<int32_t>(),
            lxu_cache_locations_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
    }
    {% endif %}
    auto sorted_linear_indices_run = at::empty_like(indices);
    auto sorted_linear_indices_run_lengths =
        at::zeros_like(indices, indices.options().dtype(kInt));
    auto sorted_linear_indices_num_runs =
        at::zeros({1}, indices.options().dtype(kInt));

    {
        size_t temp_storage_bytes = 0;
        AT_CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
            nullptr,
            temp_storage_bytes,
            linear_indices_sorted.data_ptr<int64_t>(),
            sorted_linear_indices_run.data_ptr<int64_t>(),
            sorted_linear_indices_run_lengths.data_ptr<int32_t>(),
            sorted_linear_indices_num_runs.data_ptr<int32_t>(),
            linear_indices_sorted.numel(),
            at::cuda::getCurrentCUDAStream()));
        // Allocate temporary storage
        auto temp_storage = at::empty(
            {static_cast<int64_t>(temp_storage_bytes)},
            indices.options().dtype(kByte));
        // Run encoding
        AT_CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            linear_indices_sorted.data_ptr<int64_t>(),
            sorted_linear_indices_run.data_ptr<int64_t>(),
            sorted_linear_indices_run_lengths.data_ptr<int32_t>(),
            sorted_linear_indices_num_runs.data_ptr<int32_t>(),
            linear_indices_sorted.numel(),
            at::cuda::getCurrentCUDAStream()));
    }

    auto sorted_linear_indices_cumulative_run_lengths =
        asynchronous_complete_cumsum(sorted_linear_indices_run_lengths);

    {% if not dense %}
    DISPATCH_EMB_CACHE_TYPES(
    {% else %}
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    {% endif %}
        dev_weights.type(),
        {% if not dense %}
        lxu_cache_weights.type(),
        {% endif %}
        "split_embedding_backward_{{ optimizer }}_exact_kernel",
        ([&] {

            {% if weighted %}
            auto indice_weights_sorted = at::empty_like(indice_weights);
            {
            size_t temp_storage_bytes = 0;
            AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
                nullptr,
                temp_storage_bytes,
                linear_indices.data_ptr<int64_t>(),
                linear_indices_sorted.data_ptr<int64_t>(),
                {% if not dense %}
                indice_weights.data_ptr<acc_type<cache_t, true>>(),
                indice_weights_sorted.data_ptr<acc_type<cache_t, true>>(),
                {% else %}
                indice_weights.data_ptr<acc_type<scalar_t, true>>(),
                indice_weights_sorted.data_ptr<acc_type<scalar_t, true>>(),
                {% endif %}
                linear_indices.numel(),
                0,
                total_hash_size_bits,
                at::cuda::getCurrentCUDAStream(),
                false));
            auto temp_storage = at::empty(
                {static_cast<int64_t>(temp_storage_bytes)},
                indices.options().dtype(kByte));
            AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
                temp_storage.data_ptr(),
                temp_storage_bytes,
                linear_indices.data_ptr<int64_t>(),
                linear_indices_sorted.data_ptr<int64_t>(),
                {% if not dense %}
                indice_weights.data_ptr<acc_type<cache_t, true>>(),
                indice_weights_sorted.data_ptr<acc_type<cache_t, true>>(),
                {% else %}
                indice_weights.data_ptr<acc_type<scalar_t, true>>(),
                indice_weights_sorted.data_ptr<acc_type<scalar_t, true>>(),
                {% endif %}
                linear_indices.numel(),
                0,
                total_hash_size_bits,
                at::cuda::getCurrentCUDAStream(),
                false));
            }
            {% endif %}

            auto grad_output_accessor = grad_output.packed_accessor32<
                acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>,
                2,
                RestrictPtrTraits>();
            Tensor grad_output_mean;
            if (pooling_mode == MEAN) {
              grad_output_mean = at::empty_like(grad_output);
              grad_mean_kernel<{{ "scalar_t, scalar_t" if dense else "cache_t, emb_t" }}>
                  <<<div_round_up((B * T), kMaxThreads / kWarpSize),
                     dim3(kWarpSize, kMaxThreads / kWarpSize),
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      grad_output_accessor,
                      D_offsets
                          .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                      offsets
                          .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                      grad_output_mean.packed_accessor32<
                          acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>,
                          2,
                          RestrictPtrTraits>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              grad_output_accessor = grad_output_mean.packed_accessor32<
                  acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>,
                  2,
                  RestrictPtrTraits>();
            }

            {% if not dense %}
            PhiloxCudaState rng_engine_inputs;
            if (stochastic_rounding && !std::is_same<emb_t, float>::value) {
                auto gen = at::cuda::detail::getDefaultCUDAGenerator();
                std::lock_guard<std::mutex> lock(gen.mutex());
                rng_engine_inputs =
                    at::check_generator<at::CUDAGeneratorImpl>(gen)
                        ->philox_cuda_state(4);
            }
            {% endif %}
            {% for kMaxVecsPerThread in range(1, max_embedding_dim // 128 + 1) %}
            if (max_D <= {{ 128 * kMaxVecsPerThread }}) {
            // Stay under 64K of shared memory (96K in total), BT_block_size must be a power of two.
            // B
            while(BT_block_size * sizeof(acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>) * 4 * kWarpSize * {{ kMaxVecsPerThread }} >= 64 * 1024) {
                BT_block_size /= 2;
            }
            if (std::is_same<{{ "scalar_t" if dense else "emb_t" }}, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }
            split_embedding_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1<
                {% if not dense %}
                emb_t,
                cache_t,
                {% else %}
                scalar_t,
                scalar_t,
                {% endif %}
                {{ kMaxVecsPerThread }}>
                <<<div_round_up(linear_indices.numel(), 32 * kWarpSize),
                    dim3(kWarpSize, BT_block_size),
                    BT_block_size * sizeof(acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>) * 4 * kWarpSize *
                        {{ kMaxVecsPerThread }},
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    {% if not dense %}
                    dev_weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                    uvm_weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                    lxu_cache_weights.packed_accessor64<cache_t, 2, RestrictPtrTraits>(),
                    weights_placements.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    {% else %}
                    dev_weights.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),
                    {% endif %}
                    weights_offsets.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                    D_offsets.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    hash_size_cumsum.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                    sorted_linear_indices_run
                        .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                    sorted_linear_indices_cumulative_run_lengths
                        .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    sorted_linear_indices_run_lengths
                        .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    infos_sorted.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    {% if not dense %}
                    lxu_cache_locations_sorted.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    {% endif %}
                    {% if weighted %}
                    indice_weights_sorted.packed_accessor32<acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>, 1, RestrictPtrTraits>(),
                    {% endif %}
                    sorted_linear_indices_num_runs
                        .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    max_segment_length_per_warp,
                    {% if not dense %}
                    stochastic_rounding,
                    rng_engine_inputs,
                    {% else %}
                    grad_dev_weights.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),
                    {% endif %}
                    FixedDivisor(B),
                    {{ args.split_kernel_arg_constructors | join(", ") }});
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_warp_per_row_1<
                {% if not dense %}
                emb_t,
                cache_t,
                {% else %}
                scalar_t,
                scalar_t,
                {% endif %}
                {{ kMaxVecsPerThread }}>
                <<<div_round_up(linear_indices.numel(), kBackwardMaxThreads / kWarpSize),
                    dim3(kWarpSize, kBackwardMaxThreads / kWarpSize),
                    BT_block_size * sizeof(
                    acc_type<
                    {% if not dense %}
                    cache_t
                    {% else %}
                    scalar_t
                    {% endif %},
                    true>) * 4 * kWarpSize *
                        {{ kMaxVecsPerThread }},
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    {% if not dense %}
                    dev_weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                    uvm_weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                    lxu_cache_weights.packed_accessor64<cache_t, 2, RestrictPtrTraits>(),
                    weights_placements.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    {% else %}
                    dev_weights.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),
                    {% endif %}
                    weights_offsets.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                    D_offsets.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    hash_size_cumsum.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                    sorted_linear_indices_run
                        .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                    sorted_linear_indices_cumulative_run_lengths
                        .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    sorted_linear_indices_run_lengths
                        .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    infos_sorted.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    {% if not dense %}
                    lxu_cache_locations_sorted.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    {% endif %}
                    {% if weighted %}
                    indice_weights_sorted.packed_accessor32<acc_type<{{ "scalar_t" if dense else "cache_t" }}, true>, 1, RestrictPtrTraits>(),
                    {% endif %}
                    sorted_linear_indices_num_runs
                        .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                    max_segment_length_per_warp,
                    {% if not dense %}
                    stochastic_rounding,
                    rng_engine_inputs,
                    {% else %}
                    grad_dev_weights.packed_accessor64<scalar_t, 1, RestrictPtrTraits>(),
                    {% endif %}
                    FixedDivisor(B),
                    {{ args.split_kernel_arg_constructors | join(", ") }});
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
        {% endfor %}
        }));

    return {{ "grad_dev_weights" if dense else "" }};
}
