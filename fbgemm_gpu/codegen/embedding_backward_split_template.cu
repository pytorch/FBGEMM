#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
// clang-format off
{% set wdesc = "weighted" if weighted else "unweighted" %}
#include "fbgemm_gpu/embedding_backward_template_helpers.cuh"
#include "fbgemm_gpu/split_embeddings_utils.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    size_t kMaxVecsPerThread,
    int32_t kThreadGroupSize>
__global__ __launch_bounds__(kMaxThreads) void
split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1(
    const at::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits> grad_output,
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {% else %}
    int32_t B,
    int64_t D,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        hash_size_cumsum,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_run,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_cumulative_run_lengths,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        long_run_ids,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        num_long_run_ids,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {% else %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {% endif %}
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_lxu_cache_locations,
    {% endif %}
    {% if weighted %}
    const at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {% endif %}
    {% if not dense %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {% else %}
    at::PackedTensorAccessor64<cache_t, 1, at::RestrictPtrTraits> grad_dev_weights,
    {% endif %}
    {% if not nobag %}
    FixedDivisor fd,
    {% endif %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_id_to_really_long_run_ids,
    at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 2, at::RestrictPtrTraits> temp_grad_accum,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> grad_accum_counter,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms,
    {{ args.split_kernel_args | join(",\n    ") }});


template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    size_t kMaxVecsPerThread,
    int32_t kThreadGroupSize = kWarpSize>
__global__
__launch_bounds__(kBackwardMaxThreads)
void
split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_warp_per_row_1(
    const at::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits>
        grad_output,
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    {% if not dense %}
    at::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    at::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    {% else %}
    int32_t B,
    int64_t D,
    {% endif %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        hash_size_cumsum,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_run,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_cumulative_run_lengths,
    {% if not nobag %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    {% else %}
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_infos,
    {% endif %}
    {% if not dense %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_lxu_cache_locations,
    {% endif %}
    {% if weighted %}
    const at::PackedTensorAccessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits> sorted_indice_weights,
    {% endif %}
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_num_runs,
    int32_t max_segment_length_per_warp,
    {% if not dense %}
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    {% else %}
    at::PackedTensorAccessor64<cache_t, 1, at::RestrictPtrTraits> grad_dev_weights,
    {% endif %}
    {% if not nobag %}
    FixedDivisor fd,
    {% endif %}
    {{ args.split_kernel_args | join(", ") }});


{{ "void" if not dense else "Tensor" }} split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_exact_cuda(
    Tensor grad_output,
    Tensor dev_weights,
    {% if not dense %}
    Tensor uvm_weights,
    Tensor lxu_cache_weights,
    Tensor weights_placements,
    {% endif %}
    Tensor weights_offsets,
    {% if not nobag %}
    Tensor D_offsets,
    int64_t max_D,
    {% else %}
    int64_t D,
    {% endif %}
    Tensor hash_size_cumsum,
    int64_t total_hash_size_bits,
    Tensor indices,
    Tensor offsets,
    {% if not nobag %}
    int64_t pooling_mode,
    {% endif %}
    {% if weighted %}
    Tensor indice_weights,
    {% endif %}
    {% if not dense %}
    Tensor lxu_cache_locations,
    {% endif %}
    int64_t unused_,
    int64_t max_segment_length_per_warp,
    {% if not dense %}
    bool stochastic_rounding,
    {% endif %}
    {{ args.split_function_args | join(", ") }}) {

    TENSOR_ON_CUDA_GPU(grad_output);
    TENSOR_ON_CUDA_GPU(dev_weights);
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(uvm_weights);
    TENSOR_ON_CUDA_GPU(lxu_cache_weights);
    TENSOR_ON_CUDA_GPU(weights_placements);
    {% endif %}
    TENSOR_ON_CUDA_GPU(weights_offsets);
    {% if not nobag %}
    TENSOR_ON_CUDA_GPU(D_offsets);
    {% endif %}
    TENSOR_ON_CUDA_GPU(hash_size_cumsum);
    TENSOR_ON_CUDA_GPU(indices);
    TENSOR_ON_CUDA_GPU(offsets);
    {% if weighted %}
    TENSOR_ON_CUDA_GPU(indice_weights);
    {% endif %}
    {% if not dense %}
    TENSOR_ON_CUDA_GPU(lxu_cache_locations);
    {% endif %}

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());

    {% if dense %}
    auto grad_dev_weights = zeros_like(dev_weights);
    {% endif %}

    // short-circuit if there are zero indices.
    if (indices.numel() == 0) {
        return {{ "grad_dev_weights" if dense else "" }};
    }

    {% if not nobag %}
    int32_t T = D_offsets.numel() - 1;
    {% else %}
    int32_t T = weights_offsets.numel();
    {% endif %}

    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    const auto B = (offsets.size(0) - 1) / T;
    TORCH_CHECK(B > 0);
    auto BT_block_size = kMaxThreads / kWarpSize;
    TORCH_CHECK(BT_block_size * kWarpSize <= kMaxThreads);
    {% if nobag %}
    auto max_D = D;
    {% endif %}
    TORCH_CHECK(max_D <= {{ max_embedding_dim }});

    // V100: 96 KB; A100: 160 KB.
    int max_shared_bytes = 0;
#ifndef __HIP_PLATFORM_HCC__
    hipDeviceGetAttribute(&max_shared_bytes, hipDeviceAttributeSharedMemPerBlockOptin, dev_weights.get_device());
#else
    // MI100 has 64 KB local memory (shared memory) per workgroup
    max_shared_bytes = 64 << 10;
#endif
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    int shared_kb = max_shared_bytes >> 10;
    // V100: 64 KB; A100: 96 KB.
#ifndef __HIP_PLATFORM_HCC__
    // Use 2/3 of the available GPU shared mem; leave rooms for L1$.
    int used_shared_kb = round_down(shared_kb * 2 / 3, 16);
    TORCH_CHECK(used_shared_kb > 0);
#else
    // MI100 has independent shared mem and L1
    int used_shared_kb = shared_kb;
#endif
    int used_shared_bytes = used_shared_kb << 10;

    Tensor linear_indices, linear_indices_sorted;
    Tensor infos_sorted;
    Tensor sorted_linear_indices_run, sorted_linear_indices_run_lengths,
        sorted_linear_indices_num_runs,
        sorted_linear_indices_cumulative_run_lengths;
    std::tie(
        linear_indices,
        linear_indices_sorted,
        infos_sorted,
        sorted_linear_indices_run,
        sorted_linear_indices_run_lengths,
        sorted_linear_indices_num_runs,
        sorted_linear_indices_cumulative_run_lengths) =
        transpose_embedding_input(
            hash_size_cumsum,
            total_hash_size_bits,
            indices,
            offsets,
            {{"true" if nobag else "false"}});

    {% if not dense %}
    auto lxu_cache_locations_sorted = at::empty_like(lxu_cache_locations);
    if (lxu_cache_locations.size(0) > 0) {
        size_t temp_storage_bytes = 0;
        AT_CUDA_CHECK(radix_sort_pairs(
            nullptr,
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            lxu_cache_locations.data_ptr<int32_t>(),
            lxu_cache_locations_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
        auto temp_storage = at::empty(
            {static_cast<int64_t>(temp_storage_bytes)},
            indices.options().dtype(at::kByte));
        AT_CUDA_CHECK(radix_sort_pairs(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            lxu_cache_locations.data_ptr<int32_t>(),
            lxu_cache_locations_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
    }
    {% endif %}

    DISPATCH_EMB_GRAD_CACHE_TYPES(
        dev_weights.scalar_type(),
        grad_output.scalar_type(),
        {% if not dense %}
        lxu_cache_weights.scalar_type(),
        {% else %}
        dev_weights.scalar_type(),
        {% endif %}
            "split_embedding_backward_{{ optimizer }}_exact_kernel",
        [&] {
            {% if weighted %}
            auto indice_weights_sorted = at::empty_like(indice_weights);
            {
            size_t temp_storage_bytes = 0;
            AT_CUDA_CHECK(radix_sort_pairs(
                nullptr,
                temp_storage_bytes,
                linear_indices.data_ptr<int64_t>(),
                linear_indices_sorted.data_ptr<int64_t>(),
                indice_weights.data_ptr<at::acc_type<cache_t, true>>(),
                indice_weights_sorted.data_ptr<at::acc_type<cache_t, true>>(),
                linear_indices.numel(),
                0,
                total_hash_size_bits,
                at::cuda::getCurrentCUDAStream(),
                false));
            auto temp_storage = at::empty(
                {static_cast<int64_t>(temp_storage_bytes)},
                indices.options().dtype(at::kByte));
            AT_CUDA_CHECK(radix_sort_pairs(
                temp_storage.data_ptr(),
                temp_storage_bytes,
                linear_indices.data_ptr<int64_t>(),
                linear_indices_sorted.data_ptr<int64_t>(),
                indice_weights.data_ptr<at::acc_type<cache_t, true>>(),
                indice_weights_sorted.data_ptr<at::acc_type<cache_t, true>>(),
                linear_indices.numel(),
                0,
                total_hash_size_bits,
                at::cuda::getCurrentCUDAStream(),
                false));
            }
            {% endif %}

            // early memory release
            linear_indices.reset();
            linear_indices_sorted.reset();

            auto grad_output_accessor = grad_output.packed_accessor64<grad_t, 2, at::RestrictPtrTraits>();
            {% if not nobag %}
            Tensor grad_output_mean;
            if (static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN) {
              grad_output_mean = at::empty_like(grad_output);
              grad_mean_kernel<grad_t>
                  <<<div_round_up((B * T), kMaxThreads / kWarpSize),
                     dim3(kWarpSize, kMaxThreads / kWarpSize),
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      grad_output_accessor,
                      D_offsets
                          .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                      offsets
                          .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                      grad_output_mean.packed_accessor64<
                          grad_t, 2, at::RestrictPtrTraits>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              grad_output_accessor = grad_output_mean.packed_accessor64<
                  grad_t, 2, at::RestrictPtrTraits>();
            }
            {% endif %}

            {% if not dense %}
            at::PhiloxCudaState rng_engine_inputs;
            if (stochastic_rounding && !std::is_same<emb_t, float>::value) {
                auto gen = at::cuda::detail::getDefaultCUDAGenerator();
                std::lock_guard<std::mutex> lock(gen.mutex());
                rng_engine_inputs =
                    at::check_generator<at::CUDAGeneratorImpl>(gen)
                        ->philox_cuda_state(4);
            }
            {% endif %}
            // kMaxElemPerThread is # of elements handled by thread if we use a full warp for a row
            // We consider kMaxElemPerThread 1 and 2, and then a multiple of 4.
            {% for kMaxElemPerThread in range(1, max_embedding_dim // (items_per_warp // 4) + 1) %}
            {% if kMaxElemPerThread in [1, 2] or kMaxElemPerThread % 4 == 0 %}
            if (max_D <= {{ items_per_warp // 4 * kMaxElemPerThread }}) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = {{ kMaxElemPerThread }} / 4 >= 1 ? {{ kMaxElemPerThread }} / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / {{ kMaxElemPerThread }}, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty_like(sorted_linear_indices_run_lengths);
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty_like(sorted_linear_indices_run_lengths);
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(indices.numel(), kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                sorted_linear_indices_num_runs.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                sorted_linear_indices_run_lengths.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                num_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                long_run_id_to_really_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                num_really_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                grad_accum_counter.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(long_run_ids.numel(), kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB.
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    {% if not dense %}
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                    weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    {% if not nobag %}
                    D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    B,
                    D,
                    {% endif %}
                    hash_size_cumsum.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    sorted_linear_indices_run
                        .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    sorted_linear_indices_cumulative_run_lengths
                        .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    num_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% if not nobag %}
                    infos_sorted.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    infos_sorted.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not dense %}
                    lxu_cache_locations_sorted.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if weighted %}
                    indice_weights_sorted.packed_accessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not dense %}
                    stochastic_rounding,
                    rng_engine_inputs,
                    {% else %}
                    grad_dev_weights.packed_accessor64<cache_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not nobag %}
                    FixedDivisor(B),
                    {% endif %}
                    long_run_id_to_really_long_run_ids.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    temp_grad_accum.packed_accessor32<at::acc_type<cache_t, true>, 2, at::RestrictPtrTraits>(),
                    grad_accum_counter.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    {{ args.split_kernel_arg_constructors | join(", ") }});
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(sorted_linear_indices_run.numel(), kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB.
#endif
            }

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding{{ "_nobag" if nobag else "" }}_backward_codegen_{{ optimizer }}_{{ wdesc }}_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    {% if not dense %}
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    uvm_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    lxu_cache_weights.packed_accessor64<cache_t, 2, at::RestrictPtrTraits>(),
                    weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    dev_weights.packed_accessor64<emb_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    {% if not nobag %}
                    D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    B,
                    D,
                    {% endif %}
                    hash_size_cumsum.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    sorted_linear_indices_run
                        .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    sorted_linear_indices_cumulative_run_lengths
                        .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% if not nobag %}
                    infos_sorted.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% else %}
                    infos_sorted.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not dense %}
                    lxu_cache_locations_sorted.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if weighted %}
                    indice_weights_sorted.packed_accessor32<at::acc_type<cache_t, true>, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    sorted_linear_indices_num_runs
                        .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                    max_segment_length_per_warp,
                    {% if not dense %}
                    stochastic_rounding,
                    rng_engine_inputs,
                    {% else %}
                    grad_dev_weights.packed_accessor64<cache_t, 1, at::RestrictPtrTraits>(),
                    {% endif %}
                    {% if not nobag %}
                    FixedDivisor(B),
                    {% endif %}
                    {{ args.split_kernel_arg_constructors | join(", ") }});
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
        {% endif %}
        {% endfor %}
        });

    return {{ "grad_dev_weights" if dense else "" }};
}

// clang-format on
