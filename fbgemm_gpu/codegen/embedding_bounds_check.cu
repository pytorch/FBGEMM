#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include "fbgemm_gpu/embedding_backward_template_helpers.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

template <typename index_t>
__global__ __launch_bounds__(kMaxThreads) void bounds_check_indices_kernel(
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        rows_per_table,
    at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    int64_t bounds_check_mode_,
    at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> warning,
    FixedDivisor fd) {
  int32_t T = rows_per_table.size(0);
  int32_t B = (offsets.size(0) - 1) / T;

  int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
  int32_t b; // = b_t % B;
  int32_t t; // = b_t / B;
  fd.DivMod(b_t, &t, &b);
  if (t >= T) {
    return;
  }
  auto bounds_check_mode = static_cast<BoundsCheckMode>(bounds_check_mode_);

  auto num_rows = rows_per_table[t];
  auto indices_start = offsets[t * B + b];
  auto indices_end = offsets[t * B + b + 1];
  index_t num_indices = indices.size(0);

  if (bounds_check_mode == BoundsCheckMode::FATAL) {
    CUDA_KERNEL_ASSERT(indices_start >= 0);
    CUDA_KERNEL_ASSERT(indices_start <= indices_end);
    CUDA_KERNEL_ASSERT(indices_end <= num_indices);
  } else if (bounds_check_mode == BoundsCheckMode::WARNING) {
    if (indices_start < 0 || indices_start > indices_end ||
        indices_end > num_indices) {
      if (gpuAtomicIncrement(&warning[0]) == 0) {
        printf(
            "EmbeddingBoundsCheck: (at least one) Out of bounds access for "
            "batch: %lld, table: %lld, indices_start: %lld, indices_end: %lld,"
            " num_indices: %lld. Setting indices_start and indices_end within "
            "the range.\n",
            int64_t(b),
            int64_t(t),
            int64_t(indices_start),
            int64_t(indices_end),
            int64_t(num_indices));
      }
      indices_start = std::max(
          static_cast<index_t>(0), std::min(indices_start, num_indices));
      indices_end = std::max(indices_start, std::min(indices_end, num_indices));
      offsets[t * B + b] = indices_start;
      offsets[t * B + b + 1] = indices_end;
    }
  } else if (bounds_check_mode == BoundsCheckMode::IGNORE) {
    indices_start =
        std::max(static_cast<index_t>(0), std::min(indices_start, num_indices));
    indices_end = std::max(indices_start, std::min(indices_end, num_indices));
    offsets[t * B + b] = indices_start;
    offsets[t * B + b + 1] = indices_end;
  }

  auto L = indices_end - indices_start;
  for (index_t i = (index_t)threadIdx.x; i < L;
       i += (index_t)fbgemm_gpu::kWarpSize) {
    auto idx = indices[indices_start + i];
    if (idx == -1) {
      // -1 indicates pruned rows.
      continue;
    }
    if (bounds_check_mode == BoundsCheckMode::FATAL) {
      CUDA_KERNEL_ASSERT(idx >= 0 && "Failed idx >= 0 in bounds_check_indices");
      CUDA_KERNEL_ASSERT(
          idx < num_rows && "Failed idx < num_rows in bounds_check_indices");
    } else if (bounds_check_mode == BoundsCheckMode::WARNING) {
      if (idx < 0 || idx >= num_rows) {
        if (gpuAtomicIncrement(&warning[0]) == 0) {
          printf(
              "EmbeddingBoundsCheck: (at least one) Out of bounds access for batch: %lld, table: %lld, bag element: %lld, idx: %lld, num_rows: %lld, indices_start: %lld, T: %d, B: %d, b_t: %d. Setting idx to zero.\n",
              int64_t(b),
              int64_t(t),
              int64_t(i),
              int64_t(idx),
              num_rows,
              int64_t(indices_start),
              T,
              B,
              b_t);
        }
        indices[indices_start + i] = 0;
      }
    } else if (bounds_check_mode == BoundsCheckMode::IGNORE) {
      if (idx < 0 || idx >= num_rows) {
        indices[indices_start + i] = 0;
      }
    }
  }
}

void bounds_check_indices_cuda(
    Tensor rows_per_table,
    Tensor indices,
    Tensor offsets,
    int64_t bounds_check_mode_,
    Tensor warning) {
  TENSOR_ON_CUDA_GPU(rows_per_table);
  TENSOR_ON_CUDA_GPU(indices);
  TENSOR_ON_CUDA_GPU(offsets);
  TENSOR_ON_CUDA_GPU(warning);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(rows_per_table.get_device());

  int32_t T = rows_per_table.size(0);
  int32_t B = (offsets.size(0) - 1) / T;
  if (B == 0 || T == 0) {
    return;
  }
  auto bounds_check_mode = static_cast<BoundsCheckMode>(bounds_check_mode_);
  if (bounds_check_mode == BoundsCheckMode::WARNING) {
    warning.zero_();
  }
  constexpr size_t kNumThreads = 256;

  AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "bounds_check_indices", [&] {
    bounds_check_indices_kernel<index_t>
        <<<div_round_up(B * T, kNumThreads / fbgemm_gpu::kWarpSize),
           dim3(fbgemm_gpu::kWarpSize, kNumThreads / fbgemm_gpu::kWarpSize),
           0,
           at::cuda::getCurrentCUDAStream()>>>(
            rows_per_table
                .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            indices.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(),
            offsets.packed_accessor32<index_t, 1, at::RestrictPtrTraits>(),
            bounds_check_mode_,
            warning.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            FixedDivisor(B));
  });
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}
