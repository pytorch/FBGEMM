#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

// Define __TEMPLATE_SOURCE_FILE__, which is similar to __FILE__, but is used in
// the FBGEMM codebase to denote the template source file in auto-generated
// code.
#define __TEMPLATE_SOURCE_FILE__ "FOO/BAR/BAZ-123.cpp"

// Enable tensor value checking before and after executing kernels
#define FBGEMM_GPU_TENSORCHECK

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDADeviceAssertion.h>
#include <hip/hip_runtime.h>
#include <gmock/gmock.h>
#include <gtest/gtest.h>
#include <torch/types.h> // @manual=//caffe2:torch-cpp-cpu

#include "fbgemm_gpu/utils/host_device_buffer_pair.cuh"
#include "fbgemm_gpu/utils/kernel_launcher.cuh"
#include "fbgemm_gpu/utils/tensor_accessor_builder.h"

namespace fbgemm_gpu::utils {

#define U32(x) static_cast<uint32_t>(x)

using ::testing::HasSubstr;
using source_location = fbgemm_gpu::utils::source_location;

// Define the constant value that is immune to preprocesor #defined and #undefs
// and use this for template_source_file test.  Value should match the value in
// the __TEMPLATE_SOURCE_FILE__ macro.
constexpr auto TemplateSourceFileReference = "FOO/BAR/BAZ-123.cpp";

////////////////////////////////////////////////////////////////////////////////
// Test Kernels
////////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void array_sum_kernel(T* C, const T* A, const T* B, size_t size) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    C[idx] = A[idx] + B[idx];
  }
}

template <typename T>
__global__ void array_sum_dsa_kernel(
    T* C,
    const T* A,
    const T* B,
    size_t size,
    TORCH_DSA_KERNEL_ARGS) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    C[idx] = A[idx] + B[idx];
  }
}

template <typename T>
__global__ void tensor_sum_kernel(
    pta::PackedTensorAccessor64<T, 1, at::RestrictPtrTraits> C,
    const pta::PackedTensorAccessor64<T, 1, at::RestrictPtrTraits> A,
    const pta::PackedTensorAccessor64<T, 1, at::RestrictPtrTraits> B,
    TORCH_DSA_KERNEL_ARGS) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < C.size(0)) {
    C[idx] = A[idx] + B[idx];
  }
}

__device__ unsigned int xor128_rand_int(uint32_t seed) {
  auto x = seed ^ (blockIdx.x * blockDim.x + threadIdx.x);
  x ^= x << 13;
  x ^= x >> 17;
  x ^= x << 5;
  return x;
}

template <typename T>
__global__ void tensor_sum_kernel_bad_output(
    pta::PackedTensorAccessor64<T, 1, at::RestrictPtrTraits> C,
    const pta::PackedTensorAccessor64<T, 1, at::RestrictPtrTraits> A,
    const pta::PackedTensorAccessor64<T, 1, at::RestrictPtrTraits> B,
    TORCH_DSA_KERNEL_ARGS) {
  const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  auto seed = xor128_rand_int(42);

  if (idx < C.size(0)) {
    if (seed = xor128_rand_int(seed); seed % 100 != 0) {
      // 99% chance of normal value
      C[idx] = A[idx] + B[idx];

    } else {
      seed = xor128_rand_int(seed);

      if (seed % 3 == 0) {
        C[idx] = std::numeric_limits<T>::quiet_NaN();

      } else if (seed % 3 == 1) {
        C[idx] = std::numeric_limits<T>::infinity();

      } else {
        C[idx] = std::numeric_limits<T>::infinity();
      }
    }
  }
}

__global__ void always_fail_assertion_kernel(
    const int a,
    TORCH_DSA_KERNEL_ARGS) {
  CUDA_KERNEL_ASSERT2((a != a) && "This assertion should always fail");
}

auto sample_tensors(const long size) {
  auto A = torch::full(
      {size},
      2,
      torch::dtype(torch::kFloat32)
          .device(torch::kCUDA, at::cuda::current_device()));

  auto B = torch::full(
      {size},
      3,
      torch::dtype(torch::kFloat32)
          .device(torch::kCUDA, at::cuda::current_device()));

  auto C = torch::full(
      {size},
      -1,
      torch::dtype(torch::kFloat32)
          .device(torch::kCUDA, at::cuda::current_device()));

  return std::make_tuple(A, B, C);
}

////////////////////////////////////////////////////////////////////////////////
// Kernel Launcher Tests
////////////////////////////////////////////////////////////////////////////////

TEST(KernelLauncherTest, template_source_file) {
  {
    constexpr auto context = SOURCE_CONTEXT_CURRENT("kernel");
    const auto launcher = KernelLauncher<true>(context);

    ASSERT_THAT(launcher.context.description(), HasSubstr(__FILE__));
    ASSERT_THAT(
        launcher.context.description(), HasSubstr(TemplateSourceFileReference));
  }

  {
    constexpr auto context =
        SourceContext(source_location::current(), "kernel", "", "");
    const auto launcher = KernelLauncher<true>(context);

    ASSERT_THAT(
        launcher.context.description(),
        Not(HasSubstr(TemplateSourceFileReference)));
  }
}

TEST(KernelLauncherTest, array_kernel_launch) {
  constexpr auto size = 1024;
  auto A = HostDeviceBufferPair<float>(size, 2);
  auto B = HostDeviceBufferPair<float>(size, 3);
  auto C = HostDeviceBufferPair<float>(size, -1);

  EXPECT_NO_THROW({
    FBGEMM_LAUNCH_KERNEL(
        array_sum_kernel<float>,
        8,
        1024,
        0,
        at::cuda::getCurrentCUDAStream(),
        C.device,
        A.device,
        B.device,
        size);

    C.syncToHost();

    for (const auto x : C.host) {
      EXPECT_EQ(x, 5.0f);
    }
  });
}

TEST(KernelLauncherTest, array_kernel_launch_dsa) {
  constexpr auto size = 1024;
  auto A = HostDeviceBufferPair<float>(size, 2);
  auto B = HostDeviceBufferPair<float>(size, 3);
  auto C = HostDeviceBufferPair<float>(size, -1);

  EXPECT_NO_THROW({
    FBGEMM_LAUNCH_DSA_KERNEL(
        array_sum_dsa_kernel<float>,
        8,
        1024,
        0,
        at::cuda::getCurrentCUDAStream(),
        C.device,
        A.device,
        B.device,
        size);

    C.syncToHost();

    for (const auto x : C.host) {
      EXPECT_EQ(x, 5.0f);
    }
  });
}

TEST(KernelLauncherTest, tensor_kernel_launch) {
  const auto size = 1024;
  // Not using structured bindings bc it fails on ROCm with:
  // `capturing a structured binding is not yet supported in OpenMP`
  at::Tensor A, B, C;
  std::tie(A, B, C) = sample_tensors(size);

  // Test normal kernel launch succeeds
  EXPECT_NO_THROW({
    FBGEMM_LAUNCH_DSA_KERNEL(
        tensor_sum_kernel<float>,
        8,
        1024,
        0,
        at::cuda::getCurrentCUDAStream(),
        PTA_B(C, float, 1, 64),
        PTA_B(A, float, 1, 64),
        PTA_B(B, float, 1, 64));
  });

  EXPECT_EQ(
      C.equal(torch::full(
          {size},
          5,
          torch::dtype(torch::kFloat32)
              .device(torch::kCUDA, at::cuda::current_device()))),
      true);
}

TEST(KernelLauncherTest, kernel_launch_checks) {
  const auto size = 1024;
  // Not using structured bindings bc it fails on ROCm with:
  // `capturing a structured binding is not yet supported in OpenMP`
  at::Tensor A, B, C;
  std::tie(A, B, C) = sample_tensors(size);

  const auto device = at::cuda::getCurrentCUDAStream().device_index();
  const auto properties = *at::cuda::getDeviceProperties(device);
  const auto grid_max = properties.maxGridSize;
  const auto block_max = properties.maxThreadsDim;

  // Test grid size bounds checking
  EXPECT_THROW(
      {
        FBGEMM_LAUNCH_DSA_KERNEL(
            tensor_sum_kernel<float>,
            // grid dims are too large
            grid_max[0] + 1,
            1024,
            0,
            at::cuda::getCurrentCUDAStream(),
            PTA_B(C, float, 1, 64),
            PTA_B(A, float, 1, 64),
            PTA_B(B, float, 1, 64));
      },
      std::exception);

  // Test block size bounds checking
  EXPECT_THROW(
      {
        FBGEMM_LAUNCH_DSA_KERNEL(
            tensor_sum_kernel<float>,
            8,
            // block dims are too large
            block_max[0] + 1,
            0,
            at::cuda::getCurrentCUDAStream(),
            PTA_B(C, float, 1, 64),
            PTA_B(A, float, 1, 64),
            PTA_B(B, float, 1, 64));
      },
      std::exception);

#if defined(__HIP_PLATFORM_AMD__) || \
    (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700))
  // Test max thread count
  EXPECT_THROW(
      {
        FBGEMM_LAUNCH_DSA_KERNEL(
            tensor_sum_kernel<float>,
            // Both grid and block dims conform, but the total number of
            // threads exceeds the max
            {U32(grid_max[0]), U32(grid_max[1]), U32(grid_max[2])},
            {U32(block_max[0]), U32(block_max[1]), U32(block_max[2])},
            0,
            at::cuda::getCurrentCUDAStream(),
            PTA_B(C, float, 1, 64),
            PTA_B(A, float, 1, 64),
            PTA_B(B, float, 1, 64));
      },
      std::exception);
#endif

  // Test shared memory size bounds checking
  EXPECT_THROW(
      {
        FBGEMM_LAUNCH_DSA_KERNEL(
            tensor_sum_kernel<float>,
            8,
            1024,
    // Requested shared memory size is too large
#ifdef __HIP_PLATFORM_AMD__
            properties.sharedMemPerBlock + 1,
#else
            properties.sharedMemPerBlockOptin + 1,
#endif
            at::cuda::getCurrentCUDAStream(),
            PTA_B(C, float, 1, 64),
            PTA_B(A, float, 1, 64),
            PTA_B(B, float, 1, 64));
      },
      std::exception);
}

TEST(KernelLauncherTest, tensor_value_checks) {
  const auto size = 1024;
  // Not using structured bindings bc it fails on ROCm with:
  // `capturing a structured binding is not yet supported in OpenMP`
  at::Tensor A, B, C;
  std::tie(A, B, C) = sample_tensors(size);

  {
    // Test for bad INPUT tensors
    const float values[] = {
        std::numeric_limits<float>::quiet_NaN(),
        std::numeric_limits<float>::infinity(),
        -std::numeric_limits<float>::infinity(),
    };

    for (const auto value : values) {
      // Set a bad value
      auto i = rand() % size;
      A[i] = value;

      EXPECT_THROW(
          {
            FBGEMM_LAUNCH_DSA_KERNEL(
                tensor_sum_kernel<float>,
                8,
                1024,
                0,
                at::cuda::getCurrentCUDAStream(),
                PTA_B(C, float, 1, 64),
                PTA_B(A, float, 1, 64),
                PTA_B(B, float, 1, 64));
          },
          std::exception);

      // Unset the bad value
      A[i] = 1;
    }

    for (const auto value : values) {
      // Set a bad value
      auto i = rand() % size;
      B[i] = value;

      EXPECT_THROW(
          {
            FBGEMM_LAUNCH_DSA_KERNEL(
                tensor_sum_kernel<float>,
                8,
                1024,
                0,
                at::cuda::getCurrentCUDAStream(),
                PTA_B(C, float, 1, 64),
                PTA_B(A, float, 1, 64),
                PTA_B(B, float, 1, 64));
          },
          std::exception);

      // Unset the bad value
      B[i] = 1;
    }
  }

  {
    // Test for bad OUTPUT tensors
    EXPECT_THROW(
        {
          FBGEMM_LAUNCH_DSA_KERNEL(
              tensor_sum_kernel_bad_output<float>,
              8,
              1024,
              0,
              at::cuda::getCurrentCUDAStream(),
              PTA_B(C, float, 1, 64),
              PTA_B(A, float, 1, 64),
              PTA_B(B, float, 1, 64));
        },
        std::exception);
  }
}

// NOTE: This test currently fails in fbcode CI for HIP with the following
// error (but runs without issues on both NVIDIA and AMD machines):
//
// void fbgemm_gpu::utils::always_fail_assertion_kernel(const int,
// c10::hip::DeviceAssertionsData *const, uint32_t): Device-side assertion `(a
// != a) && "This assertion should always fail"' failed. :0:rocdevice.cpp :2984:
// 1311044151769 us: [pid:1082329 tid:0x7fc06c9ff640] Callback: Queue
// 0x7fc06b500000 aborting with error : HSA_STATUS_ERROR_EXCEPTION: An HSAIL
// operation resulted in a hardware exception. code: 0x1016
//
// Disabled for now until we can figure out why this is happening.
#ifndef __HIPCC__

TEST(KernelLauncherTest, throws_dsa_exception) {
  FBGEMM_LAUNCH_DSA_KERNEL(
      always_fail_assertion_kernel,
      1,
      1,
      0,
      at::cuda::getCurrentCUDAStream(),
      42);

  EXPECT_NO_THROW({
    try {
      c10::cuda::device_synchronize();
      throw std::runtime_error("Test didn't fail, but should have.");

    } catch (const c10::Error& err) {
      const auto err_str = std::string(err.what());

      ASSERT_THAT(
          err_str,
          HasSubstr(
              "CUDA device-side assertion failures were found on GPU #0!"));

      ASSERT_THAT(
          err_str,
          HasSubstr("File containing kernel launch = [" __TEMPLATE_SOURCE_FILE__
                    "] " __FILE__));

      ASSERT_THAT(
          err_str,
          HasSubstr(
              "Name of kernel launched that led to failure = always_fail_assertion_kernel"));
    }
  });
}

#endif // __HIPCC__

} // namespace fbgemm_gpu::utils
