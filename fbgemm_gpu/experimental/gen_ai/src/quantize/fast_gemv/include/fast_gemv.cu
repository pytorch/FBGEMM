#include "hip/hip_runtime.h"
// @lint-ignore-every LICENSELINT

/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

// MIT License

// Copyright (c) 2023 Siping Wang

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

/*
 * The source code contained in this file is pulled from original
 * github repo: https://github.com/wangsiping97/FastGEMV.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cutlass/float8.h>
#include <cutlass/numeric_conversion.h>
#include <driver_functions.h>

#include "fast_gemv.cuh"
#include "utility.cuh"

using SizeType32 = std::size_t;

///////////////////////////// NORMAL //////////////////////////////
// thread_per_block = blockDim.x
// blockDim.y <= SHARED_MEM_MAX_ROWS
__global__ void gemv_bf16(
    __hip_bfloat16* mat,
    __hip_bfloat16* vec,
    __hip_bfloat16* res,
    const unsigned int k,
    const unsigned int m,
    const unsigned int n,
    unsigned int num_per_thread) {
  float sum[MAX_M_SIZE] = {0.0f};
  const auto tid = threadIdx.x;
  const auto row = blockIdx.y * blockDim.y + threadIdx.y;
  const auto start_idx = threadIdx.x;
  float4* mat4 = reinterpret_cast<float4*>(mat);
  float4* vec4 = reinterpret_cast<float4*>(vec);

#pragma unroll
  for (int iter = 0; iter < num_per_thread >> 3; iter++) {
    auto j = start_idx + iter * blockDim.x;
    if (j < k >> 3) {
      const auto mat_val = mat4[row * (k >> 3) + j];
      const bfloat16_2* mat_h1 = (bfloat16_2*)&mat_val.x;
      const bfloat16_2* mat_h2 = (bfloat16_2*)&mat_val.y;
      const bfloat16_2* mat_h3 = (bfloat16_2*)&mat_val.z;
      const bfloat16_2* mat_h4 = (bfloat16_2*)&mat_val.w;
#pragma unroll
      for (int col = 0; col < m; col++) {
        const auto vec_val = vec4[col * (k >> 3) + j];
        const bfloat16_2* vec_h1 = (bfloat16_2*)&vec_val.x;
        const bfloat16_2* vec_h2 = (bfloat16_2*)&vec_val.y;
        const bfloat16_2* vec_h3 = (bfloat16_2*)&vec_val.z;
        const bfloat16_2* vec_h4 = (bfloat16_2*)&vec_val.w;
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h1->x),
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(mat_h1->x),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h1->y),
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(mat_h1->y),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h2->x),
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(mat_h2->x),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h2->y),
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(mat_h2->y),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h3->x),
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(mat_h3->x),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h3->y),
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(mat_h3->y),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h4->x),
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(mat_h4->x),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h4->y),
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(mat_h4->y),
            sum[col]);
      }
    }
  }
#pragma unroll
  for (int col = 0; col < m; col++) {
    sum[col] = warpReduceSum(sum[col], blockDim.x);
  }

  if (blockDim.x <= WARP_SIZE) {
    if (tid == 0) {
      for (int col = 0; col < m; col++) {
        res[row + col * n] = __float2bfloat16(sum[col]);
      }
    }
    return;
  }

  // Shared mem for partial sums (one per warp in the block)
  static __shared__ float warpLevelSums[SHARED_MEM_MAX_ROWS][WARP_SIZE];
  const auto laneId = threadIdx.x % WARP_SIZE;
  const auto warpId = threadIdx.x / WARP_SIZE;
#pragma unroll
  for (int col = 0; col < m; col++) {
    if (laneId == 0)
      warpLevelSums[threadIdx.y][warpId] = sum[col];
    __syncthreads();
    // read from shared memory only if that warp existed
    sum[col] = (threadIdx.x < blockDim.x / WARP_SIZE)
        ? warpLevelSums[threadIdx.y][laneId]
        : 0.0;
    // Final reduce using first warp
    if (warpId == 0)
      sum[col] = warpReduceSum(sum[col], blockDim.x / WARP_SIZE);
    if (tid == 0) {
      res[row + col * n] = __float2bfloat16(sum[col]);
    }
  }
}

///////////////////////////// QUANTIZED-FLOAT8-MIXED
/////////////////////////////////

__global__ void gemv_quantized_bf16_fp8(
    cutlass::float_e4m3_t* mat,
    __hip_bfloat16* vec,
    __hip_bfloat16* res,
    const unsigned int k,
    const unsigned int m,
    const unsigned int n,
    float const* scale,
    unsigned int num_per_thread) {
  float sum[MAX_M_SIZE] = {0.0f};
  // each thread load num_per_thread elements from global
  const auto tid = threadIdx.x;
  const auto row = blockIdx.y * blockDim.y + threadIdx.y;
  const auto start_idx = threadIdx.x;
  half4* mat4 = reinterpret_cast<half4*>(mat);
  float4* vec4 = reinterpret_cast<float4*>(vec);

#pragma unroll
  for (int iter = 0; iter < num_per_thread >> 3; iter++) {
    auto j = start_idx + iter * blockDim.x;
    if (j < k >> 3) {
      const auto mat_val = mat4[row * (k >> 3) + j];
      const fp8_2* mat_h1 = (fp8_2*)&mat_val.x;
      const fp8_2* mat_h2 = (fp8_2*)&mat_val.y;
      const fp8_2* mat_h3 = (fp8_2*)&mat_val.z;
      const fp8_2* mat_h4 = (fp8_2*)&mat_val.w;
#pragma unroll
      for (int col = 0; col < m; col++) {
        const auto vec_val = vec4[col * (k >> 3) + j];
        const bfloat16_2* vec_h1 = (bfloat16_2*)&vec_val.x;
        const bfloat16_2* vec_h2 = (bfloat16_2*)&vec_val.y;
        const bfloat16_2* vec_h3 = (bfloat16_2*)&vec_val.z;
        const bfloat16_2* vec_h4 = (bfloat16_2*)&vec_val.w;
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h1->x),
            cutlass::NumericConverter<float, cutlass::float_e4m3_t>::convert(
                mat_h1->x),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h1->y),
            cutlass::NumericConverter<float, cutlass::float_e4m3_t>::convert(
                mat_h1->y),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h2->x),
            cutlass::NumericConverter<float, cutlass::float_e4m3_t>::convert(
                mat_h2->x),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h2->y),
            cutlass::NumericConverter<float, cutlass::float_e4m3_t>::convert(
                mat_h2->y),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h3->x),
            cutlass::NumericConverter<float, cutlass::float_e4m3_t>::convert(
                mat_h3->x),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h3->y),
            cutlass::NumericConverter<float, cutlass::float_e4m3_t>::convert(
                mat_h3->y),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h4->x),
            cutlass::NumericConverter<float, cutlass::float_e4m3_t>::convert(
                mat_h4->x),
            sum[col]);
        sum[col] = fma(
            cutlass::NumericConverter<float, __hip_bfloat16>::convert(vec_h4->y),
            cutlass::NumericConverter<float, cutlass::float_e4m3_t>::convert(
                mat_h4->y),
            sum[col]);
      }
    }
  }
#pragma unroll
  for (int col = 0; col < m; col++) {
    sum[col] *= (*scale);
    sum[col] = warpReduceSum(sum[col], blockDim.x);
  }

  if (blockDim.x <= WARP_SIZE) {
    if (tid == 0) {
#pragma unroll
      for (int col = 0; col < m; col++) {
        res[row + col * n] = __float2bfloat16(sum[col]);
      }
    }
    return;
  }

  // Shared mem for partial sums (one per warp in the block)
  static __shared__ float warpLevelSums[SHARED_MEM_MAX_ROWS][WARP_SIZE];
  const auto laneId = threadIdx.x % WARP_SIZE;
  const auto warpId = threadIdx.x / WARP_SIZE;
#pragma unroll
  for (int col = 0; col < m; col++) {
    if (laneId == 0)
      warpLevelSums[threadIdx.y][warpId] = sum[col];
    __syncthreads();
    // read from shared memory only if that warp existed
    sum[col] = (threadIdx.x < blockDim.x / WARP_SIZE)
        ? warpLevelSums[threadIdx.y][laneId]
        : 0.0;
    // Final reduce using first warp
    if (warpId == 0)
      sum[col] = warpReduceSum(sum[col], blockDim.x / WARP_SIZE);
    if (tid == 0) {
      res[row + col * n] = __float2bfloat16(sum[col]);
    }
  }
}

///////////////////////////// QUANTIZED-INT4 //////////////////////////////

// based on previous experiments, num_per_thread can >= 16
__global__ void gemv_quantized_int4(
    uint4_2* mat,
    half* vec,
    half* res,
    unsigned int n,
    half scale,
    half zero_point,
    unsigned int num_per_thread) {
  float sum = 0;
  // each thread load num_per_thread elements from global
  auto tid = threadIdx.x;
  auto row = blockIdx.y * blockDim.y + threadIdx.y;
  auto start_idx = threadIdx.x;
  uint4_2_4* mat4 = reinterpret_cast<uint4_2_4*>(mat);
  float4* vec4 = reinterpret_cast<float4*>(vec);

  float zero_point_f = static_cast<float>(zero_point);
  float scale_f = static_cast<float>(scale);

#pragma unroll
  for (int iter = 0; iter < num_per_thread >> 4; iter++) {
    unsigned int j = 2 * (start_idx + iter * blockDim.x);
    if (j < n >> 3) {
      float4 vec_val_1 = vec4[j]; // 8 half
      float4 vec_val_2 = vec4[j + 1];
      const bfloat16_2* vec_h1 = (bfloat16_2*)&vec_val_1.x;
      const bfloat16_2* vec_h2 = (bfloat16_2*)&vec_val_1.y;
      const bfloat16_2* vec_h3 = (bfloat16_2*)&vec_val_1.z;
      const bfloat16_2* vec_h4 = (bfloat16_2*)&vec_val_1.w;
      const bfloat16_2* vec_h5 = (bfloat16_2*)&vec_val_2.x;
      const bfloat16_2* vec_h6 = (bfloat16_2*)&vec_val_2.y;
      const bfloat16_2* vec_h7 = (bfloat16_2*)&vec_val_2.z;
      const bfloat16_2* vec_h8 = (bfloat16_2*)&vec_val_2.w;

      uint4_2_4 mat_val_1 = mat4[row * (n >> 3) + j];
      uint4_2_4 mat_val_2 = mat4[row * (n >> 3) + j + 1];
      const uint4_2* mat_h1 = (uint4_2*)&mat_val_1.x;
      const uint4_2* mat_h2 = (uint4_2*)&mat_val_1.y;
      const uint4_2* mat_h3 = (uint4_2*)&mat_val_1.z;
      const uint4_2* mat_h4 = (uint4_2*)&mat_val_1.w;
      const uint4_2* mat_h5 = (uint4_2*)&mat_val_2.x;
      const uint4_2* mat_h6 = (uint4_2*)&mat_val_2.y;
      const uint4_2* mat_h7 = (uint4_2*)&mat_val_2.z;
      const uint4_2* mat_h8 = (uint4_2*)&mat_val_2.w;

      sum += static_cast<float>(vec_h1->x) *
          (static_cast<float>(mat_h1->getX()) - zero_point_f);
      sum += static_cast<float>(vec_h1->y) *
          (static_cast<float>(mat_h1->getY()) - zero_point_f);
      sum += static_cast<float>(vec_h2->x) *
          (static_cast<float>(mat_h2->getX()) - zero_point_f);
      sum += static_cast<float>(vec_h2->y) *
          (static_cast<float>(mat_h2->getY()) - zero_point_f);
      sum += static_cast<float>(vec_h3->x) *
          (static_cast<float>(mat_h3->getX()) - zero_point_f);
      sum += static_cast<float>(vec_h3->y) *
          (static_cast<float>(mat_h3->getY()) - zero_point_f);
      sum += static_cast<float>(vec_h4->x) *
          (static_cast<float>(mat_h4->getX()) - zero_point_f);
      sum += static_cast<float>(vec_h4->y) *
          (static_cast<float>(mat_h4->getY()) - zero_point_f);
      sum += static_cast<float>(vec_h5->x) *
          (static_cast<float>(mat_h5->getX()) - zero_point_f);
      sum += static_cast<float>(vec_h5->y) *
          (static_cast<float>(mat_h5->getY()) - zero_point_f);
      sum += static_cast<float>(vec_h6->x) *
          (static_cast<float>(mat_h6->getX()) - zero_point_f);
      sum += static_cast<float>(vec_h6->y) *
          (static_cast<float>(mat_h6->getY()) - zero_point_f);
      sum += static_cast<float>(vec_h7->x) *
          (static_cast<float>(mat_h7->getX()) - zero_point_f);
      sum += static_cast<float>(vec_h7->y) *
          (static_cast<float>(mat_h7->getY()) - zero_point_f);
      sum += static_cast<float>(vec_h8->x) *
          (static_cast<float>(mat_h8->getX()) - zero_point_f);
      sum += static_cast<float>(vec_h8->y) *
          (static_cast<float>(mat_h8->getY()) - zero_point_f);
    }
  }

  sum *= scale_f;

  sum = warpReduceSum(sum, blockDim.x);

  if (blockDim.x <= WARP_SIZE) {
    if (tid == 0) {
      res[row] = __float2half(sum);
    }
    return;
  }

  // Shared mem for partial sums (one per warp in the block)
  static __shared__ float warpLevelSums[SHARED_MEM_MAX_ROWS][WARP_SIZE];
  const auto laneId = threadIdx.x % WARP_SIZE;
  const auto warpId = threadIdx.x / WARP_SIZE;
  if (laneId == 0)
    warpLevelSums[threadIdx.y][warpId] = sum;
  __syncthreads();
  // read from shared memory only if that warp existed
  sum = (threadIdx.x < blockDim.x / WARP_SIZE)
      ? warpLevelSums[threadIdx.y][laneId]
      : 0.0;
  // Final reduce using first warp
  if (warpId == 0)
    sum = warpReduceSum(sum, blockDim.x / WARP_SIZE);
  if (tid == 0) {
    res[row] = __float2half(sum);
  }
}
