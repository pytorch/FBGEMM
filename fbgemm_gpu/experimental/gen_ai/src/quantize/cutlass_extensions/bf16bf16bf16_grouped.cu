#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <cutlass/util/device_memory.h>
#include <cutlass/util/packed_stride.hpp>

// clang-format off
// The fixed ordering of the headers is required for CUTLASS 3.2+
#include <cute/tensor.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>     // @manual
#include <cutlass/gemm/device/gemm_universal_adapter.h>       // @manual
#include <cutlass/epilogue/collective/collective_builder.hpp> // @manual
// clang-format on

#include "cutlass_extensions/include/kernel_mode.h"

namespace fbgemm_gpu {

#if CUDART_VERSION >= 12000

namespace GroupedGemmBF16Args {
using ProblemShape =
    cutlass::gemm::GroupProblemShape<cute::Shape<int, int, int>>;
using ElementInputA = cutlass::bfloat16_t;
using ElementInputB = cutlass::bfloat16_t;
using ElementOutput = cutlass::bfloat16_t;
using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::ColumnMajor;
using LayoutOutput = cutlass::layout::RowMajor;
using ElementAccumulator = float;
using ArchTag = cutlass::arch::Sm90;
using OperatorClass = cutlass::arch::OpClassTensorOp;
using StageCountType = cutlass::gemm::collective::StageCountAuto;
// Template structure to encapsulate configurations
template <
    int TB_M,
    int TB_N,
    int TB_K,
    int TBS_M,
    int TBS_N,
    int TBS_K,
    bool PONG>
struct GroupedGemmConfigs {
  using TileShape =
      cute::Shape<cute::Int<TB_M>, cute::Int<TB_N>, cute::Int<TB_K>>;
  using ClusterShape =
      cute::Shape<cute::Int<TBS_M>, cute::Int<TBS_N>, cute::Int<TBS_K>>;
  using CooperativeSchedule =
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedCooperative;
  using PongSchedule = cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpong;
  using CooperativeEpilogueSchedule =
      cutlass::epilogue::PtrArrayTmaWarpSpecializedCooperative;
  using PongEpilogueSchedule =
      cutlass::epilogue::PtrArrayTmaWarpSpecializedPingpong;
  using KernelSchedule =
      cute::conditional_t<PONG, PongSchedule, CooperativeSchedule>;
  using EpilogueSchedule = cute::
      conditional_t<PONG, PongEpilogueSchedule, CooperativeEpilogueSchedule>;
  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          TileShape,
          ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAccumulator,
          ElementAccumulator,
          ElementOutput,
          LayoutOutput*,
          128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementOutput,
          LayoutOutput*,
          128 / cutlass::sizeof_bits<ElementOutput>::value,
          EpilogueSchedule,
          cutlass::epilogue::fusion::LinearCombination<
              ElementOutput,
              ElementAccumulator>>::CollectiveOp;
  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          ElementInputA,
          LayoutInputA*,
          128 / cutlass::sizeof_bits<ElementInputA>::value,
          ElementInputB,
          LayoutInputB*,
          128 / cutlass::sizeof_bits<ElementInputB>::value,
          ElementAccumulator,
          TileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          KernelSchedule>::CollectiveOp;
  using GemmKernel = cutlass::gemm::kernel::
      GemmUniversal<ProblemShape, CollectiveMainloop, CollectiveEpilogue>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  using StrideInputA = typename Gemm::GemmKernel::InternalStrideA;
  using StrideInputB = typename Gemm::GemmKernel::InternalStrideB;
  using StrideOutput = typename Gemm::GemmKernel::InternalStrideD;
};
} // namespace GroupedGemmBF16Args

__global__ void set_dynamic_kernel_args_kernel(
    GroupedGemmBF16Args::ElementInputA* x_ptr,
    GroupedGemmBF16Args::ElementInputB* w_ptr,
    int64_t* input_args_ptr,
    int64_t* output_args_ptr,
    GroupedGemmBF16Args::ElementOutput* output_data,
    int x_ptr_offset,
    int w_ptr_offset,
    int problem_shape_buf_offset,
    int stride_buf_offset,
    int stride_size,
    int problem_count,
    int problem_shape_size,
    int64_t* zero_start_index_M,
    int M,
    int N,
    int K) {
  int group_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (group_index < problem_count) {
    int64_t* x_ptr_ = input_args_ptr + x_ptr_offset;
    int64_t* w_ptr_ = input_args_ptr + w_ptr_offset;
    uint8_t* problem_shape_buf =
        reinterpret_cast<uint8_t*>(input_args_ptr + problem_shape_buf_offset);
    uint8_t* stride_buf =
        reinterpret_cast<uint8_t*>(input_args_ptr + stride_buf_offset);

    GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape*
        problem_shape_ptr = reinterpret_cast<
            GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape*>(
            problem_shape_buf);
    // Pass dummy configs to get Stride structure
    GroupedGemmBF16Args::GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
        StrideInputA* stride_input_A_ptr = reinterpret_cast<
            GroupedGemmBF16Args::
                GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
                    StrideInputA*>(stride_buf);
    GroupedGemmBF16Args::GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
        StrideInputB* stride_input_B_ptr = reinterpret_cast<
            GroupedGemmBF16Args::
                GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
                    StrideInputB*>(stride_buf + stride_size);
    GroupedGemmBF16Args::GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
        StrideOutput* stride_output_ptr = reinterpret_cast<
            GroupedGemmBF16Args::
                GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
                    StrideOutput*>(stride_buf + (stride_size * 2));

    output_args_ptr[group_index] =
        reinterpret_cast<int64_t>(output_data + (group_index * M * N));

    // Write kernel arguments directly to memory.
    x_ptr_[group_index] =
        reinterpret_cast<int64_t>(x_ptr + (group_index * M * K));
    w_ptr_[group_index] =
        reinterpret_cast<int64_t>(w_ptr + (group_index * N * K));
    problem_shape_ptr[group_index] =
        GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape(
            zero_start_index_M[group_index], N, K);
    stride_input_A_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmBF16Args::
            GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::StrideInputA{},
        {zero_start_index_M[group_index], K, 1});
    stride_input_B_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmBF16Args::
            GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::StrideInputB{},
        {N, K, 1});
    stride_output_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmBF16Args::
            GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::StrideOutput{},
        {zero_start_index_M[group_index], N, 1});
  }
}

__global__ void set_static_kernel_args_kernel(
    GroupedGemmBF16Args::ElementInputA* x_ptr,
    GroupedGemmBF16Args::ElementInputB* w_ptr,
    int64_t* input_args_ptr,
    int64_t* output_args_ptr,
    GroupedGemmBF16Args::ElementOutput* output_data,
    int x_ptr_offset,
    int w_ptr_offset,
    int problem_shape_buf_offset,
    int stride_buf_offset,
    int stride_size,
    int problem_count,
    int problem_shape_size,
    int group_index,
    int M,
    int N,
    int K) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // We only set one group's information per kernel launch.
  if (idx == 0) {
    int64_t* x_ptr_ = input_args_ptr + x_ptr_offset;
    int64_t* w_ptr_ = input_args_ptr + w_ptr_offset;
    uint8_t* problem_shape_buf =
        reinterpret_cast<uint8_t*>(input_args_ptr + problem_shape_buf_offset);
    uint8_t* stride_buf =
        reinterpret_cast<uint8_t*>(input_args_ptr + stride_buf_offset);

    GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape*
        problem_shape_ptr = reinterpret_cast<
            GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape*>(
            problem_shape_buf);
    // Pass dummy configs to get Stride structure
    GroupedGemmBF16Args::GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
        StrideInputA* stride_input_A_ptr = reinterpret_cast<
            GroupedGemmBF16Args::
                GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
                    StrideInputA*>(stride_buf);
    GroupedGemmBF16Args::GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
        StrideInputB* stride_input_B_ptr = reinterpret_cast<
            GroupedGemmBF16Args::
                GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
                    StrideInputB*>(stride_buf + stride_size);
    GroupedGemmBF16Args::GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
        StrideOutput* stride_output_ptr = reinterpret_cast<
            GroupedGemmBF16Args::
                GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::
                    StrideOutput*>(stride_buf + (stride_size * 2));

    output_args_ptr[group_index] = reinterpret_cast<int64_t>(output_data);

    // Write kernel arguments directly to memory.
    x_ptr_[group_index] = reinterpret_cast<int64_t>(x_ptr);
    w_ptr_[group_index] = reinterpret_cast<int64_t>(w_ptr);
    problem_shape_ptr[group_index] =
        GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape(M, N, K);
    stride_input_A_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmBF16Args::
            GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::StrideInputA{},
        {M, K, 1});
    stride_input_B_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmBF16Args::
            GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::StrideInputB{},
        {N, K, 1});
    stride_output_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmBF16Args::
            GroupedGemmConfigs<128, 128, 128, 2, 1, 1, true>::StrideOutput{},
        {M, N, 1});
  }
}

template <
    int TB_M,
    int TB_N,
    int TB_K,
    int TBS_M,
    int TBS_N,
    int TBS_K,
    bool PONG>
std::vector<at::Tensor> bf16bf16bf16_grouped_impl(
    at::TensorList X, // BF16
    at::TensorList W, // BF16
    std::vector<at::Tensor> output_tensor,
    std::optional<at::Tensor> zero_start_index_M) {
  int problem_count = X.size();
  TORCH_CHECK(W.size() == problem_count);
  TORCH_CHECK(
      !zero_start_index_M.has_value() ||
      zero_start_index_M->size(0) == problem_count);
  if (problem_count == 0) {
    return {};
  }
  using GroupedGemmConfigs = GroupedGemmBF16Args::
      GroupedGemmConfigs<TB_M, TB_N, TB_K, TBS_M, TBS_N, TBS_K, PONG>;

  at::Tensor output_args =
      at::empty({problem_count}, X[0].options().dtype(at::kLong));

  const int64_t problem_shape_size = problem_count *
      ((int64_t)sizeof(
          GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape));
  const int64_t stride_size = problem_count *
      ((int64_t)sizeof(typename GroupedGemmConfigs::StrideInputA));

  at::Tensor input_args = at::empty(
      {problem_count * 3 + problem_shape_size + stride_size * 3},
      X[0].options().dtype(at::kLong));

  int x_ptr_offset = 0;
  int w_ptr_offset = problem_count * sizeof(int64_t);
  int problem_shape_buf_offset = problem_count * 2 * sizeof(int64_t);
  int stride_buf_offset =
      problem_count * 2 * sizeof(int64_t) + problem_shape_size;

  TORCH_CHECK(
      !zero_start_index_M.has_value() ||
          zero_start_index_M->dtype() == at::kLong,
      "zero_start_index_M must be int64.");

  // Set arguments
  // Here we support two methods for initializing arguments. If
  // zero_start_index_M is provided, we assume that M is dynamic and N and K are
  // fixed. This is useful for cuda graphs. If it is not provided we run in
  // eager mode.
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  if (zero_start_index_M.has_value()) {
    // When zero_start_M is provided, we run in dynamic shape mode.
    int M = X[0].size(0);
    int N = W[0].size(0);
    int K = X[0].size(1);
    // Make sure that inputs are allocated in sequential memory as required by
    // this mode.
    for (int i = 1; i < problem_count; i++) {
      // Check that all inputs are allocated directly following preceding input.
      TORCH_CHECK(
          X[i].data_ptr() ==
              (reinterpret_cast<GroupedGemmBF16Args::ElementInputA*>(
                   X[i - 1].data_ptr()) +
               (M * K)),
          "Inputs must be sequential in memory to support dynamic M, but X is not.");
      TORCH_CHECK(
          W[i].data_ptr() ==
              (reinterpret_cast<GroupedGemmBF16Args::ElementInputB*>(
                   W[i - 1].data_ptr()) +
               (N * K)),
          "Inputs must be sequential in memory to support dynamic M, but W is not.");
      TORCH_CHECK(
          output_tensor[i].data_ptr() ==
              (reinterpret_cast<GroupedGemmBF16Args::ElementOutput*>(
                   output_tensor[i - 1].data_ptr()) +
               (M * N)),
          "Inputs must be sequential in memory to support dynamic M, but output is not.");
    }
    int blockSize = std::min(1024, problem_count);
    int numBlocks = (problem_count + blockSize - 1) / blockSize;
    set_dynamic_kernel_args_kernel<<<numBlocks, blockSize, 0, stream>>>(
        reinterpret_cast<GroupedGemmBF16Args::ElementInputA*>(X[0].data_ptr()),
        reinterpret_cast<GroupedGemmBF16Args::ElementInputB*>(W[0].data_ptr()),
        input_args.data_ptr<int64_t>(),
        output_args.data_ptr<int64_t>(),
        reinterpret_cast<GroupedGemmBF16Args::ElementOutput*>(
            output_tensor[0].data_ptr()),
        x_ptr_offset,
        w_ptr_offset,
        problem_shape_buf_offset,
        stride_buf_offset,
        stride_size,
        problem_count,
        problem_shape_size,
        reinterpret_cast<int64_t*>(zero_start_index_M.value().data_ptr()),
        M,
        N,
        K);
  } else {
    // Otherwise run in static mode, which can support arbitrary N and K.
    int blockSize = 256;
    int numBlocks = 1;
    // Iterate over groups and launch one kernel to set each up.
    for (int i = 0; i < problem_count; i++) {
      int M = X[i].size(0);
      int N = W[i].size(0);
      int K = X[i].size(1);
      set_static_kernel_args_kernel<<<numBlocks, blockSize, 0, stream>>>(
          reinterpret_cast<GroupedGemmBF16Args::ElementInputA*>(
              X[i].data_ptr()),
          reinterpret_cast<GroupedGemmBF16Args::ElementInputB*>(
              W[i].data_ptr()),
          input_args.data_ptr<int64_t>(),
          output_args.data_ptr<int64_t>(),
          reinterpret_cast<GroupedGemmBF16Args::ElementOutput*>(
              output_tensor[i].data_ptr()),
          x_ptr_offset,
          w_ptr_offset,
          problem_shape_buf_offset,
          stride_buf_offset,
          stride_size,
          problem_count,
          problem_shape_size,
          i,
          M,
          N,
          K);
    }
  }

  // Get appropriate pointers to each component of the kernel args.
  int64_t* output_ptr = output_args.data_ptr<int64_t>();
  int64_t* x_ptr = input_args.data_ptr<int64_t>() + x_ptr_offset;
  int64_t* w_ptr = input_args.data_ptr<int64_t>() + w_ptr_offset;
  uint8_t* problem_shape_buf = reinterpret_cast<uint8_t*>(
      input_args.data_ptr<int64_t>() + problem_shape_buf_offset);
  uint8_t* stride_buf = reinterpret_cast<uint8_t*>(
      input_args.data_ptr<int64_t>() + stride_buf_offset);

  GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape* problem_shape_ptr =
      reinterpret_cast<
          GroupedGemmBF16Args::ProblemShape::UnderlyingProblemShape*>(
          problem_shape_buf);
  typename GroupedGemmConfigs::StrideInputA* stride_input_A_ptr =
      reinterpret_cast<typename GroupedGemmConfigs::StrideInputA*>(stride_buf);
  typename GroupedGemmConfigs::StrideInputB* stride_input_B_ptr =
      reinterpret_cast<typename GroupedGemmConfigs::StrideInputB*>(
          stride_buf + stride_size);
  typename GroupedGemmConfigs::StrideOutput* stride_output_ptr =
      reinterpret_cast<typename GroupedGemmConfigs::StrideOutput*>(
          stride_buf + (stride_size * 2));

  typename GroupedGemmConfigs::Gemm::Arguments arguments;
  decltype(arguments.epilogue.thread) fusion_args;
  fusion_args.alpha = 1.0;
  fusion_args.dAlpha = {cute::_0{}, cute::_0{}, 0};

  arguments = typename GroupedGemmConfigs::Gemm::Arguments{
      cutlass::gemm::GemmUniversalMode::kGrouped,
      {problem_count, problem_shape_ptr, nullptr},
      {reinterpret_cast<const GroupedGemmBF16Args::ElementInputA**>(x_ptr),
       stride_input_A_ptr,
       reinterpret_cast<const GroupedGemmBF16Args::ElementInputB**>(w_ptr),
       stride_input_B_ptr},
      {fusion_args,
       reinterpret_cast<const GroupedGemmBF16Args::ElementOutput**>(output_ptr),
       stride_output_ptr,
       reinterpret_cast<GroupedGemmBF16Args::ElementOutput**>(output_ptr),
       stride_output_ptr}};

  typename GroupedGemmConfigs::Gemm gemm;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size =
      GroupedGemmConfigs::Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  at::Tensor workspace =
      at::empty(workspace_size, X[0].options().dtype(at::kByte));

  // Check the problem size is supported or not
  cutlass::Status status = gemm.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm.initialize(
      arguments, reinterpret_cast<uint8_t*>(workspace.data_ptr()));
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm(at::cuda::getCurrentCUDAStream());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error(
        std::string("cutlass cannot run") +
        cutlass::cutlassGetStatusString(status));
  }

  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return output_tensor;
}

std::vector<at::Tensor> dispatch_bf16_grouped_kernel(
    at::TensorList x_group, // BF16
    at::TensorList w_group, // BF16
    std::vector<at::Tensor> output_tensor,
    std::optional<at::Tensor> zero_start_index_M) {
  KernelMode kernel = get_grouped_kernel_mode(x_group, w_group);
  if (kernel == KernelMode::Small) {
    return bf16bf16bf16_grouped_impl<64, 128, 128, 2, 1, 1, true>(
        x_group, w_group, output_tensor, zero_start_index_M);
  } else if (kernel == KernelMode::Large) {
    return bf16bf16bf16_grouped_impl<128, 128, 128, 2, 1, 1, true>(
        x_group, w_group, output_tensor, zero_start_index_M);
  } else {
    return bf16bf16bf16_grouped_impl<128, 128, 128, 1, 2, 1, true>(
        x_group, w_group, output_tensor, zero_start_index_M);
  }
}

std::vector<at::Tensor> bf16bf16bf16_grouped(
    at::TensorList x_group, // BF16
    at::TensorList w_group, // BF16
    std::optional<std::vector<at::Tensor>> output = std::nullopt) {
  TORCH_CHECK(!output.has_value(), "Preallocated output not yet supported.");
  // Initialize output tensor.
  int problem_count = x_group.size();
  std::vector<at::Tensor> output_tensor;
  for (int i = 0; i < problem_count; i++) {
    int M = x_group[i].size(0);
    int N = w_group[i].size(0);
    output_tensor.push_back(
        at::empty({M, N}, x_group[i].options().dtype(at::kBFloat16)));
  }
  return dispatch_bf16_grouped_kernel(
      x_group, w_group, output_tensor, std::nullopt);
}

at::Tensor bf16bf16bf16_grouped_dynamic(
    at::TensorList x_group, // BF16
    at::TensorList w_group, // BF16
    std::optional<at::Tensor> zero_start_index_M = std::nullopt) {
  std::vector<at::Tensor> output_groups;
  at::Tensor output_full;
  int problem_count = x_group.size();
  int N = w_group[0].size(0);
  int K = x_group[0].size(1);
  if (zero_start_index_M.has_value()) {
    int M = x_group[0].size(0);
    // Fill output with zeros to simplify integration. This prevents nans from
    // showing up in the tensor.
    output_full = at::zeros(
        {problem_count, M, N}, x_group[0].options().dtype(at::kBFloat16));
    // Split the output into groups.
    output_groups = at::unbind(output_full, 0);
  } else {
    // If not provided, we try to allocate a single blob that can store each
    // group.
    int total_M = 0;
    std::vector<int> group_sizes = {};
    for (int i = 0; i < problem_count; i++) {
      TORCH_CHECK(
          x_group[i].size(1) == K && w_group[i].size(0) == N,
          "Dynamic grouped gemm requires fixed N and K.");
      int group_M = x_group[i].size(0);
      total_M += group_M;
      group_sizes.push_back(group_M);
    }
    // Allocate a contiguous array for all groups.
    output_full =
        at::empty({total_M, N}, x_group[0].options().dtype(at::kBFloat16));
    // Split the full array into appropriate groups.
    // We do this with narrow to make sure there are no extra copies.
    int offset = 0;
    for (int size : group_sizes) {
      output_groups.push_back(output_full.narrow(0, offset, size));
      offset += size;
    }
  }
  // Run kernel to populate output tensor.
  dispatch_bf16_grouped_kernel(
      x_group, w_group, output_groups, zero_start_index_M);
  // Return coalesced view of output.
  return output_full;
}

#else

std::vector<at::Tensor> bf16bf16bf16_grouped(
    at::TensorList /* x_group */, // BF16
    at::TensorList /* w_group */, // BF16
    std::optional<std::vector<at::Tensor>> /* output */) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

at::Tensor bf16bf16bf16_grouped_dynamic(
    at::TensorList /* x_group */, // BF16
    at::TensorList /* w_group */, // BF16
    std::optional<at::Tensor> /* zero_start_index_M */) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

#endif

} // namespace fbgemm_gpu
