/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "fbgemm_gpu/quantize/tuning_cache.hpp"
#include "fbgemm_gpu/quantize/utils.h"

#if defined(CUDA_VERSION) && (CUDA_VERSION >= 12080)
#include "mx8mx8bf16_grouped/mx8mx8bf16_grouped_manifest.cuh"
#endif

namespace fbgemm_gpu {

#if defined(CUDA_VERSION) && (CUDA_VERSION >= 12080)

Kernel_mx8mx8bf16_grouped get_kernel_via_tuning(
    int M,
    int N,
    int K,
    int G,
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor output,
    at::Tensor offsets) {
  static TuningCache cache("mx8mx8bf16_grouped");

  M = nextPowerOf2(M);
  N = nextPowerOf2(N);
  K = nextPowerOf2(K);
  const std::string shape_key =
      std::to_string(M) + "_" + std::to_string(N) + "_" + std::to_string(K);

  const auto& kernels = get_mx8mx8bf16_grouped_kernels();
  auto kernel = cache.findBestKernelMaybeAutotune(
      shape_key, kernels, XQ, WQ, x_scale, w_scale, output, G, offsets);

  return kernel;
}

Kernel_mx8mx8bf16_grouped
get_kernel_via_heuristics(int M, int N, int K, int G) {
  if (M <= 128) {
    if (N <= 512) {
      return mx8mx8bf16_grouped_256_64_256_2_1_1;
    } else if (N <= 1024) {
      if (K <= 4096) {
        return mx8mx8bf16_grouped_256_64_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_128_64_256_1_1_1;
      }
    } else {
      return mx8mx8bf16_grouped_256_128_256_2_1_1;
    }
  } else if (M <= 512) {
    if (N <= 512) {
      return mx8mx8bf16_grouped_256_128_256_2_1_1;
    } else if (N <= 4096) {
      if (K <= 1024) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    } else if (N <= 8192) {
      return mx8mx8bf16_grouped_256_128_256_2_1_1;
    } else {
      if (K <= 512) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else if (K <= 4096) {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      } else if (K <= 8192) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    }
  } else if (M <= 1024) {
    if (N <= 2048) {
      if (K <= 1024) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    } else if (N <= 4096) {
      return mx8mx8bf16_grouped_256_128_256_2_1_1;
    } else if (N <= 8192) {
      if (K <= 512) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    } else {
      return mx8mx8bf16_grouped_256_128_256_2_1_1;
    }
  } else if (M <= 2048) {
    if (N <= 1024) {
      if (K <= 1024) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    } else if (N <= 2048) {
      return mx8mx8bf16_grouped_256_128_256_2_1_1;
    } else {
      if (K <= 512) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    }
  } else if (M <= 4096) {
    if (N <= 512) {
      if (K <= 512) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    } else if (N <= 1024) {
      return mx8mx8bf16_grouped_256_128_256_2_1_1;
    } else {
      if (K <= 512) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    }
  } else if (M <= 8192) {
    if (K <= 512) {
      return mx8mx8bf16_grouped_256_256_256_2_1_1;
    } else {
      return mx8mx8bf16_grouped_256_128_256_2_1_1;
    }
  } else {
    if (N <= 8192) {
      if (K <= 512) {
        return mx8mx8bf16_grouped_256_256_256_2_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    } else {
      if (K <= 512) {
        return mx8mx8bf16_grouped_128_64_256_1_1_1;
      } else {
        return mx8mx8bf16_grouped_256_128_256_2_1_1;
      }
    }
  }
}

at::Tensor dispatch_mx8_grouped_kernel(
    int M,
    int N,
    int K,
    int G,
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor output,
    at::Tensor offsets) {
  // Select kernel to run via heuristics.
  auto kernel = [&]() {
    if (std::getenv("FBGEMM_AUTOTUNE_ENABLE")) {
      return get_kernel_via_tuning(
          M, N, K, G, XQ, WQ, x_scale, w_scale, output, offsets);
    } else {
      return get_kernel_via_heuristics(M, N, K, G);
    }
  }();
  // Invoke kernel
  return kernel(XQ, WQ, x_scale, w_scale, output, G, offsets);
}

at::Tensor mx8mx8bf16_grouped_mm(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor offsets,
    std::optional<at::Tensor> output) {
  TORCH_CHECK(offsets.dtype() == at::kInt, "offsets must be int32.");
  TORCH_CHECK(offsets.dim() == 1, "offsets must be 1D tensor.");
  TORCH_CHECK(XQ.is_contiguous(), "XQ must be row major.");
  TORCH_CHECK(WQ.transpose(-2, -1).is_contiguous(), "WQ must be column major.");
  TORCH_CHECK(x_scale.is_contiguous(), "x_scale must be contiguous.");
  TORCH_CHECK(w_scale.is_contiguous(), "w_scale must be contiguous.");

  int64_t G = offsets.size(0);
  int64_t M = XQ.size(0);
  int64_t N = WQ.size(-1);
  int64_t K = WQ.size(-2);

  at::Tensor output_actual;

  // 2d-3d case.
  if (XQ.dim() == 2 && WQ.dim() == 3) {
    // Alias for clarity that groups are along M dimension for 2d-3d case.
    int64_t total_M = M;

    // Allocate output tensor if necessary.
    output_actual = output.has_value()
        ? output.value()
        : at::empty({total_M, N}, XQ.options().dtype(at::kBFloat16));

    TORCH_CHECK(
        XQ.size(-1) == K && WQ.size(0) == G,
        "for 2d-3d grouped GEMM, XQ shape must be (total_M, K) and WQ shape must be (G, K, N).");

    TORCH_CHECK(
        output_actual.dim() == 2 && output_actual.size(0) == total_M &&
            output_actual.size(1) == N,
        "for 2d-3d grouped GEMM, output shape must be (total_M, N).");

    // Normalized jagged dim for heuristics
    M /= G;
    // 2d-2d case.
  } else if (XQ.dim() == 2 && WQ.dim() == 2) {
    // Alias for clarity that groups are along K dimension for 2d-2d case.
    int64_t total_K = K;

    // Allocate output tensor if necessary.
    output_actual = output.has_value()
        ? output.value()
        : at::empty({G, M, N}, XQ.options().dtype(at::kBFloat16));

    TORCH_CHECK(
        XQ.dim() == 2 && WQ.dim() == 2 && WQ.size(-2) == total_K,
        "for 2d-2d grouped GEMM, XQ shape must be (M, total_K) and WQ shape must be (total_K, N).");

    TORCH_CHECK(
        output_actual.dim() == 3 && output_actual.size(0) == G &&
            output_actual.size(1) == M && output_actual.size(2) == N,
        "for 2d-2d grouped GEMM, output shape must be (G, M, N).");
    // Normalized jagged dim for heuristics
    K /= G;
  } else {
    TORCH_CHECK(false, "Invalid input shapes. Must be one of 2D-2D, 2D-3D.");
  }

  // Early exit for empty inputs.
  if (M == 0) {
    return output_actual;
  }

  // Return continuous view of output.
  return dispatch_mx8_grouped_kernel(
      M, N, K, G, XQ, WQ, x_scale, w_scale, output_actual, offsets);
}

#else

at::Tensor mx8mx8bf16_grouped_mm(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor offsets,
    std::optional<at::Tensor> output) {
  throw std::runtime_error(
      "CUDA version is older than 12.8"); // requires CUDA>=12.8
}

#endif

} // namespace fbgemm_gpu
