/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "cutlass_extensions/include/kernel_mode.h"
#include "cutlass_extensions/include/threadblock.h"

namespace fbgemm_gpu {

#if CUDART_VERSION >= 12000

template <
    int TB_M,
    int TB_N,
    int TB_K,
    int TBS_M,
    int TBS_N,
    int TBS_K,
    bool PONG,
    typename INPUT_DTYPE,
    typename WEIGHT_SCALE_DTYPE>
at::Tensor f8i4bf16_rowwise_impl(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // INT4
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor w_zp) {
  int M = XQ.size(0);
  int N = WQ.size(0);
  int K = XQ.size(1);

  int num_groups = w_scale.size(0);

  TORCH_CHECK(XQ.is_cuda() && XQ.is_contiguous());
  TORCH_CHECK(WQ.is_cuda() && WQ.is_contiguous());
  TORCH_CHECK(x_scale.is_cuda() && x_scale.is_contiguous());
  TORCH_CHECK(w_scale.is_cuda() && w_scale.is_contiguous());
  TORCH_CHECK(w_zp.is_cuda() && w_zp.is_contiguous());
  TORCH_CHECK(K >= num_groups && K % num_groups == 0);

  int group_size = K / num_groups;

  auto Y = at::empty({M, N}, XQ.options().dtype(at::kBFloat16));

  using ElementInputA = INPUT_DTYPE;
  using LayoutInputA = cutlass::layout::ColumnMajor;
  constexpr int AlignmentInputA =
      128 /
      cutlass::sizeof_bits<
          ElementInputA>::value; // Memory access granularity/alignment of A
                                 // matrix in units of elements (up to 16 bytes)

  using ElementInputB = cutlass::int4b_t;
  using LayoutInputB = cutlass::layout::RowMajor;
  constexpr int AlignmentInputB =
      128 /
      cutlass::sizeof_bits<
          ElementInputB>::value; // Memory access granularity/alignment of B
                                 // matrix in units of elements (up to 16 bytes)

  using ElementScale = WEIGHT_SCALE_DTYPE;
  using ElementZeroPoint = WEIGHT_SCALE_DTYPE;
  using ElementComputeEpilogue = float;
  using ElementAccumulator = float;

  using ElementOutput = cutlass::bfloat16_t;
  using LayoutOutput = cutlass::layout::ColumnMajor;
  constexpr int AlignmentOutput =
      128 /
      cutlass::sizeof_bits<
          ElementOutput>::value; // Memory access granularity/alignment of C
                                 // matrix in units of elements (up to 16 bytes)

  using ArchTag = cutlass::arch::Sm90; // Tag indicating the minimum SM that
                                       // supports the intended feature
  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using TileShape = cute::Shape<
      cute::Int<TB_M>,
      cute::Int<TB_N>,
      cute::Int<TB_K>>; // Threadblock-level
                        // tile size
  using ClusterShape = cute::Shape<
      cute::Int<TBS_M>,
      cute::Int<TBS_N>,
      cute::Int<TBS_K>>; // Shape of the
                         // threadblocks in a
                         // cluster
  using DefaultSchedule = cutlass::gemm::KernelTmaWarpSpecializedMixedInput;
  using PongSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedPingpongMixedInput;
  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecialized;
  using EpilogueTileType = cutlass::epilogue::collective::EpilogueTileAuto;
  using MainLoopSchedule =
      cute::conditional_t<PONG, PongSchedule, DefaultSchedule>;

  // Implement rowwise scaling epilogue for x
  using XScale = cutlass::epilogue::fusion::Sm90RowBroadcast<
      PONG ? 2 : 1,
      TileShape,
      ElementComputeEpilogue,
      cute::Stride<cute::Int<0>, cute::Int<1>, cute::Int<0>>>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using Compute0 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies,
      ElementOutput, // First stage output type.
      ElementComputeEpilogue, // First stage input types.
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EpilogueEVT =
      cutlass::epilogue::fusion::Sm90EVT<Compute0, XScale, Accum>;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          TileShape,
          ClusterShape,
          EpilogueTileType,
          ElementAccumulator,
          ElementAccumulator,
          ElementOutput,
          LayoutOutput,
          AlignmentOutput,
          ElementOutput,
          LayoutOutput,
          AlignmentOutput,
          EpilogueSchedule,
          EpilogueEVT>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          cute::tuple<ElementInputB, ElementScale, ElementZeroPoint>,
          LayoutInputB,
          AlignmentInputB,
          ElementInputA,
          LayoutInputA,
          AlignmentInputA,
          ElementAccumulator,
          TileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          MainLoopSchedule>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cute::Shape<int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideInputA = typename Gemm::GemmKernel::StrideA;
  using StrideInputB = typename Gemm::GemmKernel::StrideB;
  using StrideOutput = typename Gemm::GemmKernel::StrideC;
  using StrideS = typename CollectiveMainloop::StrideScale;

  StrideInputA stride_a = cutlass::make_cute_packed_stride(
      StrideInputA{}, cute::make_shape(M, K, cute::Int<1>{}));
  StrideInputB stride_b = cutlass::make_cute_packed_stride(
      StrideInputB{}, cute::make_shape(N, K, cute::Int<1>{}));
  StrideOutput stride_output = cutlass::make_cute_packed_stride(
      StrideOutput{}, cute::make_shape(N, M, cute::Int<1>{}));
  StrideS stride_S = cutlass::make_cute_packed_stride(
      StrideS{}, cute::make_shape(N, num_groups, cute::Int<1>{}));

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {N, M, K},
      {reinterpret_cast<ElementInputB*>(WQ.data_ptr()),
       stride_b,
       reinterpret_cast<ElementInputA*>(XQ.data_ptr()),
       stride_a,
       reinterpret_cast<ElementScale*>(w_scale.data_ptr()),
       stride_S,
       group_size,
       reinterpret_cast<ElementZeroPoint*>(w_zp.data_ptr())},
      {{},
       (ElementOutput*)Y.data_ptr<at::BFloat16>(),
       stride_output,
       (ElementOutput*)Y.data_ptr<at::BFloat16>(),
       stride_output}};

  arguments.epilogue.thread = {
      {reinterpret_cast<ElementComputeEpilogue*>(
          x_scale.data_ptr())}, // x_scale
      {}, // Accumulator
      {}, // Multiplies
  };

  Gemm gemm;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm(at::cuda::getCurrentCUDAStream());

  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error(
        std::string("cutlass cannot run") +
        cutlass::cutlassGetStatusString(status));
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return Y;
}

template <typename InputDType, typename WEIGHT_SCALE_DTYPE>
at::Tensor dispatch_f8i4bf16_rowwise_kernel(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor w_zp) {
  KernelMode kernel = get_kernel_mode(XQ, WQ);
  if (kernel == KernelMode::Small) {
    return f8i4bf16_rowwise_impl<
        64,
        128,
        128,
        2,
        1,
        1,
        true,
        InputDType,
        WEIGHT_SCALE_DTYPE>(XQ, WQ, x_scale, w_scale, w_zp);
  } else if (kernel == KernelMode::Large) {
    return f8i4bf16_rowwise_impl<
        128,
        128,
        128,
        2,
        1,
        1,
        true,
        InputDType,
        WEIGHT_SCALE_DTYPE>(XQ, WQ, x_scale, w_scale, w_zp);
  } else {
    return f8i4bf16_rowwise_impl<
        128,
        128,
        128,
        2,
        1,
        1,
        false,
        InputDType,
        WEIGHT_SCALE_DTYPE>(XQ, WQ, x_scale, w_scale, w_zp);
  }
}

at::Tensor f8i4bf16_rowwise(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // INT4
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor w_zp) {
  // Check datatypes.
  TORCH_CHECK(
      x_scale.dtype() == at::kFloat, "Input scale tensor must be float32.");
  TORCH_CHECK(
      (w_scale.dtype() == at::kFloat && w_zp.dtype() == at::kFloat) ||
          (w_scale.dtype() == at::kHalf && w_zp.dtype() == at::kHalf) ||
          (w_scale.dtype() == at::kBFloat16 && w_zp.dtype() == at::kBFloat16),
      "Weight scale and zero point tensors must be float32, bfloat16, or float16, and dtype of weight scale and zero point tensors must be the same .");

  // Templatize based on input and weight scale/zero point dtype.
  bool use_e5m2 = XQ.dtype() == at::kFloat8_e5m2;

  if (w_scale.dtype() == at::kFloat) {
    if (use_e5m2) {
      return dispatch_f8i4bf16_rowwise_kernel<cutlass::float_e5m2_t, float>(
          XQ, WQ, x_scale, w_scale, w_zp);
    } else {
      return dispatch_f8i4bf16_rowwise_kernel<cutlass::float_e4m3_t, float>(
          XQ, WQ, x_scale, w_scale, w_zp);
    }
  } else if (w_scale.dtype() == at::kHalf) {
    if (use_e5m2) {
      return dispatch_f8i4bf16_rowwise_kernel<
          cutlass::float_e5m2_t,
          cutlass::half_t>(XQ, WQ, x_scale, w_scale, w_zp);
    } else {
      return dispatch_f8i4bf16_rowwise_kernel<
          cutlass::float_e4m3_t,
          cutlass::half_t>(XQ, WQ, x_scale, w_scale, w_zp);
    }
  } else if (w_scale.dtype() == at::kBFloat16) {
    if (use_e5m2) {
      return dispatch_f8i4bf16_rowwise_kernel<
          cutlass::float_e5m2_t,
          cutlass::bfloat16_t>(XQ, WQ, x_scale, w_scale, w_zp);
    } else {
      return dispatch_f8i4bf16_rowwise_kernel<
          cutlass::float_e4m3_t,
          cutlass::bfloat16_t>(XQ, WQ, x_scale, w_scale, w_zp);
    }
  } else {
    throw std::runtime_error(
        "Weight scale and zero point data type not supported in f8i4bf16_rowwise");
  }
}

#else

at::Tensor f8i4bf16_rowwise(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // INT4
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor w_zp) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

#endif

} // namespace fbgemm_gpu
