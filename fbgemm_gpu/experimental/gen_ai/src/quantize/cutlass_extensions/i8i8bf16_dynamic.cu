/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/DeviceGuard.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <ATen/cuda/Atomic.cuh>
#if !(                                                  \
    defined(USE_ROCM) ||                                \
    ((defined(CUDA_VERSION) && CUDA_VERSION < 11000) || \
     (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))))
#include <hipblaslt.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <cuda/atomic>
#elif (defined(USE_ROCM))
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hipblaslt/hipblaslt.h>
#endif
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAGuard.h>
#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/half.h>
#include <cutlass/numeric_types.h>
#include <cutlass/trace.h>
#include <cutlass/util/host_tensor.h>
#include "cublas_utils.h"

#if CUDART_VERSION >= 12000
#include <hip/hip_fp8.h>
#endif

// clang-format off
// The fixed ordering of the headers is required for CUTLASS 3.2+
#include <cute/tensor.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>     // @manual
#include <cutlass/gemm/device/gemm_universal_adapter.h>       // @manual
#include <cutlass/epilogue/collective/collective_builder.hpp> // @manual
// clang-format on

#include <cute/atom/mma_atom.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>

#include "cutlass_extensions/include/kernel_mode.h"
#include "cutlass_extensions/include/threadblock.h"
#include "fp8_blockwise_cutlass_helpers.h"

namespace fbgemm_gpu {

template <int TB_M, int TB_N, int TB_K, int W_M, int W_N, int W_K>
at::Tensor i8i8bf16_dynamic_impl(
    at::Tensor XQ, // INT8
    at::Tensor WQ, // INT8
    at::Tensor scale,
    int64_t split_k) {
  auto M = XQ.size(0);
  auto N = WQ.size(0);
  auto K = XQ.size(1);

  TORCH_CHECK(XQ.is_cuda() && XQ.is_contiguous());
  TORCH_CHECK(WQ.is_cuda() && WQ.is_contiguous());

  auto Y = at::empty({M, N}, XQ.options().dtype(at::kBFloat16));

  using ElementOutput = cutlass::bfloat16_t;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = float;
  using ElementInputA = int8_t; // <- data type of elements in input matrix A
  using ElementInputB = int8_t; // <- data type of elements in input matrix B

  // The code section below describes matrix layout of input and output
  // matrices. Column Major for Matrix A, Row Major for Matrix B and Row Major
  // for Matrix C
  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  using Gemm = cutlass::gemm::device::Gemm<
      int8_t,
      cutlass::layout::RowMajor,
      int8_t,
      cutlass::layout::ColumnMajor,
      ElementOutput,
      cutlass::layout::RowMajor,
      ElementAccumulator,
      cutlass::arch::OpClassTensorOp,
      cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<TB_M, TB_N, TB_K>, // ThreadBlockShape
      cutlass::gemm::GemmShape<W_M, W_N, W_K>, // WarpShape
      cutlass::gemm::GemmShape<16, 8, 32>, // InstructionShape
      cutlass::epilogue::thread::LinearCombinationOnDevice<
          ElementOutput,
          128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator,
          ElementComputeEpilogue>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
      3,
      16,
      16,
      true>;

  auto input_size = cutlass::MatrixCoord(M, K);
  auto weight_size = cutlass::MatrixCoord(K, N);
  auto output_size = cutlass::MatrixCoord(M, N);

  // constexpr int kSparse = Gemm::kSparse;
  // How many elements of A are covered per ElementE
  // constexpr int kElementsPerElementE = Gemm::kElementsPerElementE;
  // The size of individual meta data
  // constexpr int kMetaSizeInBits = Gemm::kMetaSizeInBits;
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  cutlass::TensorRef<ElementInputA, LayoutInputA> input_ref(
      XQ.data_ptr<ElementInputA>(), LayoutInputA::packed(input_size));
  cutlass::TensorRef<ElementInputB, LayoutInputB> weight_ref(
      WQ.data_ptr<ElementInputB>(), LayoutInputB::packed(weight_size));
  cutlass::TensorRef<ElementOutput, LayoutOutput> out_ref(
      (ElementOutput*)Y.data_ptr<at::BFloat16>(),
      LayoutOutput::packed(output_size));

  typename Gemm::Arguments arguments{
      problem_size,
      input_ref,
      weight_ref,
      out_ref,
      out_ref,
      {scale.data_ptr<float>()},
      int(split_k)};
  Gemm gemm_op;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  auto workspace =
      at::empty({int64_t(workspace_size)}, Y.options().dtype(at::kChar));

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(
      arguments, workspace.data_ptr(), at::cuda::getCurrentCUDAStream());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm_op(at::cuda::getCurrentCUDAStream());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error(
        std::string("cutlass cannot run") +
        cutlass::cutlassGetStatusString(status));
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return Y;
}

at::Tensor i8i8bf16_dynamic(
    at::Tensor XQ, // INT8
    at::Tensor WQ, // INT8
    at::Tensor scale,
    int64_t split_k) {
  auto M = XQ.size(0);
  auto N = WQ.size(0);
  auto K = XQ.size(1);
  if (M <= 128 && N >= K) {
    return i8i8bf16_dynamic_impl<64, 128, 64, 32, 64, 64>(
        XQ, WQ, scale, split_k);
  } else if (M <= 128 && N < K) {
    return i8i8bf16_dynamic_impl<64, 64, 128, 32, 32, 128>(
        XQ, WQ, scale, split_k);
  } else {
    return i8i8bf16_dynamic_impl<256, 128, 64, 64, 64, 64>(
        XQ, WQ, scale, split_k);
  }
}

} // namespace fbgemm_gpu
