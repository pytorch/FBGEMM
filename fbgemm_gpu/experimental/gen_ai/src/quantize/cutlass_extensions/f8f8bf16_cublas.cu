/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAGuard.h>

#include "cublas_utils.h"

namespace fbgemm_gpu {

#if CUDART_VERSION >= 12000

at::Tensor f8f8bf16_cublas(
    at::Tensor A, // FP8
    at::Tensor B, // FP8
    std::optional<at::Tensor> Ainvs = c10::nullopt,
    std::optional<at::Tensor> Binvs = c10::nullopt,
    bool use_fast_accum = true,
    std::optional<at::Tensor> output = c10::nullopt) {
  auto m = A.size(0);
  auto n = B.size(0);
  auto k = A.size(1);
  size_t workspaceSize = CUBLAS_WORKSPACE_SIZE;
  const int8_t fastAccuMode = use_fast_accum ? 1 : 0;

  TORCH_CHECK(A.is_cuda() && A.is_contiguous());
  TORCH_CHECK(B.is_cuda() && B.is_contiguous());

  hipblasLtHandle_t ltHandle;
  checkCublasStatus(hipblasLtCreate(&ltHandle));
  auto& allocator = *::c10::cuda::CUDACachingAllocator::get();
  auto workspace = allocator.allocate(workspaceSize);
  if (output.has_value()) {
    auto output_tensor = output.value();
    TORCH_CHECK(output_tensor.is_cuda());
    TORCH_CHECK(output_tensor.is_contiguous());
    TORCH_CHECK(
        output_tensor.numel() == m * n,
        "output_tensor.numel=",
        output_tensor.numel(),
        ", m=",
        m,
        ", n=",
        n);
    TORCH_CHECK(output_tensor.options().dtype() == at::kBFloat16);
  }

  const hipDataType A_type = HIP_R_8F_E4M3;
  const hipDataType B_type = HIP_R_8F_E4M3;
  const hipDataType D_type = HIP_R_16BF;

  float one = 1.0;
  float zero = 0.0;

  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  hipblasLtMatmulDesc_t operationDesc = nullptr;
  hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Ddesc = nullptr;
  hipblasLtMatmulPreference_t preference = nullptr;
  int returnedResults = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  hipblasComputeType_t gemm_compute_type = HIPBLAS_COMPUTE_32F;
  // Create matrix descriptors. Not setting any extra attributes.

  auto lda = k;
  auto ldb = k;
  auto ldd = n;
  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, A_type, k, m, lda));
  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, B_type, k, n, ldb));
  checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, D_type, n, m, ldd));

  checkCublasStatus(
      hipblasLtMatmulDescCreate(&operationDesc, gemm_compute_type, HIP_R_32F));
  checkCublasStatus(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
  checkCublasStatus(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

  checkCublasStatus(hipblasLtMatmulDescSetAttribute(
      operationDesc,
      CUBLASLT_MATMUL_DESC_FAST_ACCUM,
      &fastAccuMode,
      sizeof(fastAccuMode)));

  if (Ainvs.has_value()) {
    const float* Ainvs_pt = Ainvs.value().data_ptr<float>();
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(
        operationDesc,
        HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER,
        &Ainvs_pt,
        sizeof(Ainvs_pt)));
  }

  if (Binvs.has_value()) {
    const float* Binvs_pt = Binvs.value().data_ptr<float>();
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(
        operationDesc,
        HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER,
        &Binvs_pt,
        sizeof(Binvs_pt)));
  }

  checkCublasStatus(hipblasLtMatmulDescSetAttribute(
      operationDesc,
      HIPBLASLT_MATMUL_DESC_EPILOGUE,
      &epilogue,
      sizeof(epilogue)));

  checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));

  checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(
      preference,
      HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
      &workspaceSize,
      sizeof(workspaceSize)));

  checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(
      ltHandle,
      operationDesc,
      Bdesc,
      Adesc,
      Ddesc,
      Ddesc,
      preference,
      1,
      &heuristicResult,
      &returnedResults));

  if (returnedResults == 0)
    throw std::runtime_error("Unable to find any suitable algorithms");

  // D = alpha * (A * B) + beta * C
  // Warmup
  auto Y = output.value_or(at::empty({m, n}, A.options().dtype(at::kBFloat16)));
  checkCublasStatus(hipblasLtMatmul(
      ltHandle,
      operationDesc,
      static_cast<const void*>(&one), /* alpha */
      B.data_ptr(), /* B */
      Bdesc,
      A.data_ptr(), /* A */
      Adesc,
      static_cast<const void*>(&zero), /* beta */
      nullptr, /* C */
      Ddesc,
      Y.data_ptr(), /* D */
      Ddesc,
      &heuristicResult.algo, /* algo */
      workspace.mutable_get(), /* workspace */
      workspaceSize,
      at::cuda::getCurrentCUDAStream())); /* stream */
  return Y;
}

#else

at::Tensor f8f8bf16_cublas(
    at::Tensor A, // FP8
    at::Tensor B, // FP8
    std::optional<at::Tensor> Ainvs = c10::nullopt,
    std::optional<at::Tensor> Binvs = c10::nullopt,
    bool use_fast_accum = true,
    std::optional<at::Tensor> output = c10::nullopt) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

#endif

} // namespace fbgemm_gpu
