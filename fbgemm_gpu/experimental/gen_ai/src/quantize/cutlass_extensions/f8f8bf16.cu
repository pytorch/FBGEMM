/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/packed_stride.hpp>

// clang-format off
// The fixed ordering of the headers is required for CUTLASS 3.2+
#include <cute/tensor.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>     // @manual
#include <cutlass/gemm/device/gemm_universal_adapter.h>       // @manual
#include <cutlass/epilogue/collective/collective_builder.hpp> // @manual
// clang-format on

namespace fbgemm_gpu {

#if CUDART_VERSION >= 12000

// Cutlass tensorwise kernel
template <
    int TB_M,
    int TB_N,
    int TB_K,
    int TBS_M,
    int TBS_N,
    int TBS_K,
    bool FAST_ACCUM>
at::Tensor f8f8bf16_impl(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor scale) {
  // XQ: M x K
  // WQ: N x K
  // output: M x N
  int M = size_to_dim_(XQ.dim() - 1, XQ.sizes());
  int N = WQ.size(0);
  int K = WQ.size(1);
  // 1. If the input tensor is {M, K}, the output tensor is {M, N}.
  // 2. If the input tensor is {b, M, K}, the output tensor is {b, M, N}.
  auto out_sizes = XQ.sizes().vec();
  out_sizes.back() = N;

  // Handle case where inputs are empty.
  if (M == 0 || N == 0 || K == 0) {
    return at::zeros(out_sizes, XQ.options().dtype(at::kBFloat16));
  }

  TORCH_CHECK(XQ.is_cuda() && XQ.is_contiguous());
  TORCH_CHECK(WQ.is_cuda() && WQ.is_contiguous());

  auto Y = at::empty(out_sizes, XQ.options().dtype(at::kBFloat16));

  using ElementInputA = cutlass::float_e4m3_t;
  using LayoutInputA = cutlass::layout::RowMajor;
  constexpr int AlignmentInputA =
      128 /
      cutlass::sizeof_bits<
          ElementInputA>::value; // Memory access granularity/alignment of A
                                 // matrix in units of elements (up to 16 bytes)

  using ElementInputB = cutlass::float_e4m3_t;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  constexpr int AlignmentInputB =
      128 /
      cutlass::sizeof_bits<
          ElementInputB>::value; // Memory access granularity/alignment of B
                                 // matrix in units of elements (up to 16 bytes)

  using ElementOutput = cutlass::bfloat16_t;
  using LayoutOutput = cutlass::layout::ColumnMajor;
  constexpr int AlignmentOutput =
      128 /
      cutlass::sizeof_bits<
          ElementOutput>::value; // Memory access granularity/alignment of C
                                 // matrix in units of elements (up to 16 bytes)

  using ElementAccumulator = float;
  using ElementComputeEpilogue = float;
  using ArchTag = cutlass::arch::Sm90; // Tag indicating the minimum SM that
                                       // supports the intended feature
  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using TileShape = cute::Shape<
      cute::Int<TB_M>,
      cute::Int<TB_N>,
      cute::Int<TB_K>>; // Threadblock-level
                        // tile size
  using ClusterShape = cute::Shape<
      cute::Int<TBS_M>,
      cute::Int<TBS_N>,
      cute::Int<TBS_K>>; // Shape of the
                         // threadblocks in a
                         // cluster
  using StageCountType =
      cutlass::gemm::collective::StageCountAuto; // Stage count maximized
                                                 // based on the tile size
  using KernelSchedule = cutlass::gemm::collective::
      KernelScheduleAuto; // Kernel to launch based on the default setting in
                          // the Collective Builder

  using MainLoopSchedule = cute::conditional_t<
      FAST_ACCUM,
      cutlass::gemm::KernelTmaWarpSpecializedFP8FastAccum,
      cutlass::gemm::KernelTmaWarpSpecialized>;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          ElementInputA,
          LayoutInputA,
          AlignmentInputA,
          ElementInputB,
          LayoutInputB,
          AlignmentInputB,
          ElementAccumulator,
          TileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAuto,
          MainLoopSchedule>::CollectiveOp;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          TileShape,
          ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAccumulator,
          ElementComputeEpilogue,
          ElementOutput,
          LayoutOutput,
          AlignmentOutput,
          ElementOutput,
          LayoutOutput,
          AlignmentOutput,
          cutlass::epilogue::collective::EpilogueScheduleAuto>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cute::Shape<int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideInputA = typename Gemm::GemmKernel::StrideA;
  using StrideInputB = typename Gemm::GemmKernel::StrideB;
  using StrideOutput = typename Gemm::GemmKernel::StrideC;

  StrideInputA stride_a = cutlass::make_cute_packed_stride(
      StrideInputA{}, cute::make_shape(M, K, 1));
  StrideInputB stride_b = cutlass::make_cute_packed_stride(
      StrideInputB{}, cute::make_shape(N, K, 1));
  StrideOutput stride_output = cutlass::make_cute_packed_stride(
      StrideOutput{}, cute::make_shape(N, M, 1));

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {N, M, K},
      {reinterpret_cast<ElementInputB*>(WQ.data_ptr()),
       stride_b,
       reinterpret_cast<ElementInputA*>(XQ.data_ptr()),
       stride_a},
      {{scale.data_ptr<float>(), 0},
       (ElementOutput*)Y.data_ptr<at::BFloat16>(),
       stride_output,
       (ElementOutput*)Y.data_ptr<at::BFloat16>(),
       stride_output}};
  Gemm gemm;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm(at::cuda::getCurrentCUDAStream());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error(
        std::string("cutlass cannot run") +
        cutlass::cutlassGetStatusString(status));
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return Y;
}

at::Tensor f8f8bf16(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor scale,
    bool use_fast_accum) {
  auto M = XQ.size(0);
  // auto K = XQ.size(1);
  // auto N = WQ.size(0);
  if (use_fast_accum) {
    if (M <= 128) {
      return f8f8bf16_impl<64, 128, 128, 2, 1, 1, true>(XQ, WQ, scale);
    } else {
      return f8f8bf16_impl<128, 128, 128, 1, 2, 1, true>(XQ, WQ, scale);
    }
  } else {
    if (M <= 128) {
      return f8f8bf16_impl<64, 128, 128, 2, 1, 1, false>(XQ, WQ, scale);
    } else {
      return f8f8bf16_impl<128, 128, 128, 1, 2, 1, false>(XQ, WQ, scale);
    }
  }
}

#else

at::Tensor f8f8bf16(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor scale,
    bool use_fast_accum) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

#endif

} // namespace fbgemm_gpu
