/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <cutlass/util/device_memory.h>
#include <cutlass/util/packed_stride.hpp>

// clang-format off
// The fixed ordering of the headers is required for CUTLASS 3.2+
#include <cute/tensor.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>     // @manual
#include <cutlass/gemm/device/gemm_universal_adapter.h>       // @manual
#include <cutlass/epilogue/collective/collective_builder.hpp> // @manual
// clang-format on

#include "cutlass_extensions/include/kernel_mode.h"

namespace fbgemm_gpu {

#if CUDART_VERSION >= 12000

template <
    int TB_M,
    int TB_N,
    int TB_K,
    int TBS_M,
    int TBS_N,
    int TBS_K,
    bool PONG,
    bool FAST_ACCUM>
at::Tensor f8f8bf16_tensorwise_impl(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    double scale) {
  // XQ: M x K
  // WQ: N x K
  // output: M x N
  int M = size_to_dim_(XQ.dim() - 1, XQ.sizes());
  int N = WQ.size(0);
  int K = WQ.size(1);
  // 1. If the input tensor is {M, K}, the output tensor is {M, N}.
  // 2. If the input tensor is {b, M, K}, the output tensor is {b, M, N}.
  auto out_sizes = XQ.sizes().vec();
  out_sizes.back() = N;

  TORCH_CHECK(XQ.is_cuda() && XQ.is_contiguous());
  TORCH_CHECK(WQ.is_cuda() && WQ.is_contiguous());

  auto Y = at::empty(out_sizes, XQ.options().dtype(at::kBFloat16));

  using ElementInputA = cutlass::float_e4m3_t;
  using LayoutInputA = cutlass::layout::RowMajor;
  constexpr int AlignmentInputA = 16 /
      sizeof(ElementInputA); // Memory access granularity/alignment of A
                             // matrix in units of elements (up to 16 bytes)

  using ElementInputB = cutlass::float_e4m3_t;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  constexpr int AlignmentInputB = 16 /
      sizeof(ElementInputB); // Memory access granularity/alignment of B
                             // matrix in units of elements (up to 16 bytes)

  using ElementOutput = cutlass::bfloat16_t;
  using LayoutOutput = cutlass::layout::RowMajor;
  constexpr int AlignmentOutput = 16 /
      sizeof(ElementOutput); // Memory access granularity/alignment of C
                             // matrix in units of elements (up to 16 bytes)

  using ElementAccumulator = float;
  using ElementComputeEpilogue = float;
  using ArchTag = cutlass::arch::Sm90; // Tag indicating the minimum SM that
                                       // supports the intended feature
  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using TileShape = cute::Shape<
      cute::Int<TB_M>,
      cute::Int<TB_N>,
      cute::Int<TB_K>>; // Threadblock-level
                        // tile size
  using ClusterShape = cute::Shape<
      cute::Int<TBS_M>,
      cute::Int<TBS_N>,
      cute::Int<TBS_K>>; // Shape of the
                         // threadblocks in a
                         // cluster
  using StageCountType =
      cutlass::gemm::collective::StageCountAuto; // Stage count maximized
                                                 // based on the tile size
  using KernelSchedule = cutlass::gemm::collective::
      KernelScheduleAuto; // Kernel to launch based on the default setting in
                          // the Collective Builder

  using DefaultSchedule = cutlass::gemm::KernelTmaWarpSpecialized;
  using PongSchedule = cutlass::gemm::KernelTmaWarpSpecializedPingpong;
  using FastDefaultSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedFP8FastAccum;
  using FastPongSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
  using SlowAccum = cute::conditional_t<PONG, PongSchedule, DefaultSchedule>;
  using FastAccum =
      cute::conditional_t<PONG, FastPongSchedule, FastDefaultSchedule>;
  using MainLoopSchedule =
      cute::conditional_t<FAST_ACCUM, FastAccum, SlowAccum>;

  using Scale_ =
      cutlass::epilogue::fusion::Sm90ScalarBroadcast<ElementComputeEpilogue>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using Compute0 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies,
      ElementOutput, // First stage output type.
      ElementComputeEpilogue, // First stage input types.
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EpilogueEVT =
      cutlass::epilogue::fusion::Sm90EVT<Compute0, Scale_, Accum>;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          TileShape,
          ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAccumulator,
          ElementComputeEpilogue,
          ElementOutput,
          LayoutOutput,
          AlignmentOutput,
          ElementOutput,
          LayoutOutput,
          AlignmentOutput,
          cutlass::epilogue::TmaWarpSpecialized,
          EpilogueEVT>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          ElementInputA,
          LayoutInputA,
          AlignmentInputA,
          ElementInputB,
          LayoutInputB,
          AlignmentInputB,
          ElementAccumulator,
          TileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          MainLoopSchedule>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cute::Shape<int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideInputA = typename Gemm::GemmKernel::StrideA;
  using StrideInputB = typename Gemm::GemmKernel::StrideB;
  using StrideOutput = typename Gemm::GemmKernel::StrideC;

  StrideInputA stride_a = cutlass::make_cute_packed_stride(
      StrideInputA{}, cute::make_shape(M, K, cute::Int<1>{}));
  StrideInputB stride_b = cutlass::make_cute_packed_stride(
      StrideInputB{}, cute::make_shape(N, K, cute::Int<1>{}));
  StrideOutput stride_output = cutlass::make_cute_packed_stride(
      StrideOutput{}, cute::make_shape(M, N, cute::Int<1>{}));

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K},
      {reinterpret_cast<ElementInputA*>(XQ.data_ptr()),
       stride_a,
       reinterpret_cast<ElementInputB*>(WQ.data_ptr()),
       stride_b},
      {{},
       (ElementOutput*)Y.data_ptr<at::BFloat16>(),
       stride_output,
       (ElementOutput*)Y.data_ptr<at::BFloat16>(),
       stride_output}};

  arguments.epilogue.thread = {
      {float(scale)}, // scale
      {}, // Accumulator
      {}, // Multiplies
  };

  Gemm gemm;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm(at::cuda::getCurrentCUDAStream());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error(
        std::string("cutlass cannot run") +
        cutlass::cutlassGetStatusString(status));
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return Y;
}

at::Tensor f8f8bf16_tensorwise(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    double scale,
    bool use_fast_accum) {
  KernelMode kernel = get_kernel_mode(XQ, WQ);
  if (kernel == KernelMode::Small) {
    return f8f8bf16_tensorwise_impl<64, 128, 128, 2, 1, 1, true, true>(
        XQ, WQ, scale);
  } else if (kernel == KernelMode::Large) {
    return f8f8bf16_tensorwise_impl<128, 128, 128, 2, 1, 1, true, true>(
        XQ, WQ, scale);
  } else {
    return f8f8bf16_tensorwise_impl<128, 128, 128, 1, 2, 1, false, true>(
        XQ, WQ, scale);
  }
}

#else

at::Tensor f8f8bf16_tensorwise(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    double scale,
    bool use_fast_accum) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

#endif

} // namespace fbgemm_gpu
