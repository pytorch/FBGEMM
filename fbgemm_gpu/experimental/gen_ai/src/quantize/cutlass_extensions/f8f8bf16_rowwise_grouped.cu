#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <cutlass/util/device_memory.h>
#include <cutlass/util/packed_stride.hpp>

// clang-format off
// The fixed ordering of the headers is required for CUTLASS 3.2+
#include <cute/tensor.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>     // @manual
#include <cutlass/gemm/device/gemm_universal_adapter.h>       // @manual
#include <cutlass/epilogue/collective/collective_builder.hpp> // @manual
// clang-format on

#include "cutlass_extensions/include/kernel_mode.h"

namespace fbgemm_gpu {

#if CUDART_VERSION >= 12000

namespace GroupedGemmArgs {
using ProblemShape =
    cutlass::gemm::GroupProblemShape<cute::Shape<int, int, int>>;
using ElementInputA = cutlass::float_e4m3_t;
using ElementInputB = cutlass::float_e4m3_t;
using ElementOutput = cutlass::bfloat16_t;
using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::ColumnMajor;
using LayoutOutput = cutlass::layout::RowMajor;
using ElementAccumulator = float;
using ElementComputeEpilogue = float;
using ArchTag = cutlass::arch::Sm90;
using OperatorClass = cutlass::arch::OpClassTensorOp;
using StageCountType = cutlass::gemm::collective::StageCountAuto;
// Template structure to encapsulate configurations
template <
    int TB_M,
    int TB_N,
    int TB_K,
    int TBS_M,
    int TBS_N,
    int TBS_K,
    bool PONG>
struct GroupedGemmConfigs {
  using TileShape =
      cute::Shape<cute::Int<TB_M>, cute::Int<TB_N>, cute::Int<TB_K>>;
  using ClusterShape =
      cute::Shape<cute::Int<TBS_M>, cute::Int<TBS_N>, cute::Int<TBS_K>>;
  using CooperativeSchedule =
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedCooperativeFP8FastAccum;
  using PongSchedule =
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpongFP8FastAccum;
  using CooperativeEpilogueSchedule =
      cutlass::epilogue::PtrArrayTmaWarpSpecializedCooperative;
  using PongEpilogueSchedule =
      cutlass::epilogue::PtrArrayTmaWarpSpecializedPingpong;
  using KernelSchedule =
      cute::conditional_t<PONG, PongSchedule, CooperativeSchedule>;
  using EpilogueSchedule = cute::
      conditional_t<PONG, PongEpilogueSchedule, CooperativeEpilogueSchedule>;

  // Implement rowwise scaling epilogue.
  using XScale = cutlass::epilogue::fusion::Sm90ColBroadcast<
      0,
      TileShape,
      ElementComputeEpilogue*,
      ElementComputeEpilogue,
      cute::Stride<cute::Int<1>, cute::Int<0>, cute::Int<0>>>;

  using WScale = cutlass::epilogue::fusion::Sm90RowBroadcast<
      0,
      TileShape,
      ElementComputeEpilogue*,
      ElementComputeEpilogue,
      cute::Stride<cute::Int<0>, cute::Int<1>, cute::Int<0>>>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using Compute0 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies,
      ElementComputeEpilogue, // First stage output type.
      ElementComputeEpilogue, // First stage input types.
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 =
      cutlass::epilogue::fusion::Sm90EVT<Compute0, WScale, Accum>;

  using Compute1 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies,
      ElementOutput,
      ElementComputeEpilogue, // Second stage input types.
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute1 =
      cutlass::epilogue::fusion::Sm90EVT<Compute1, XScale, EVTCompute0>;

  using EpilogueEVT = EVTCompute1;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90,
          cutlass::arch::OpClassTensorOp,
          TileShape,
          ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAccumulator,
          ElementAccumulator,
          ElementOutput,
          LayoutOutput*,
          128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementOutput,
          LayoutOutput*,
          128 / cutlass::sizeof_bits<ElementOutput>::value,
          EpilogueSchedule,
          EpilogueEVT>::CollectiveOp;
  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          ElementInputA,
          LayoutInputA*,
          128 / cutlass::sizeof_bits<ElementInputA>::value,
          ElementInputB,
          LayoutInputB*,
          128 / cutlass::sizeof_bits<ElementInputB>::value,
          ElementAccumulator,
          TileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          KernelSchedule>::CollectiveOp;
  using GemmKernel = cutlass::gemm::kernel::
      GemmUniversal<ProblemShape, CollectiveMainloop, CollectiveEpilogue>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  using StrideInputA = typename Gemm::GemmKernel::InternalStrideA;
  using StrideInputB = typename Gemm::GemmKernel::InternalStrideB;
  using StrideOutput = typename Gemm::GemmKernel::InternalStrideD;
};
} // namespace GroupedGemmArgs

__global__ void set_kernel_args_kernel(
    int64_t xq_ptr,
    int64_t wq_ptr,
    int64_t x_scale_ptr,
    int64_t w_scale_ptr,
    int64_t* input_args_ptr,
    int64_t* output_args_ptr,
    at::BFloat16* output_data,
    int output_offset,
    int xq_ptr_offset,
    int wq_ptr_offset,
    int x_scale_ptr_offset,
    int w_scale_ptr_offset,
    int problem_shape_buf_offset,
    int stride_buf_offset,
    int stride_size,
    int group_count,
    int problem_shape_size,
    int group_index,
    int M,
    int N,
    int K) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // Each kernel annoyingly can only set the kernel args for one group.
  // This could only be avoided with complicated memory management.
  if (idx == 0) {
    int64_t* xq_ptr_ = input_args_ptr + xq_ptr_offset;
    int64_t* wq_ptr_ = input_args_ptr + wq_ptr_offset;
    int64_t* x_scale_ptr_ = input_args_ptr + x_scale_ptr_offset;
    int64_t* w_scale_ptr_ = input_args_ptr + w_scale_ptr_offset;
    uint8_t* problem_shape_buf =
        reinterpret_cast<uint8_t*>(input_args_ptr + problem_shape_buf_offset);
    uint8_t* stride_buf =
        reinterpret_cast<uint8_t*>(input_args_ptr + stride_buf_offset);

    GroupedGemmArgs::ProblemShape::UnderlyingProblemShape* problem_shape_ptr =
        reinterpret_cast<
            GroupedGemmArgs::ProblemShape::UnderlyingProblemShape*>(
            problem_shape_buf);
    // Pass dummy configs to get Stride structure
    GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
        StrideInputA* stride_input_A_ptr = reinterpret_cast<
            GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
                StrideInputA*>(stride_buf);
    GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
        StrideInputB* stride_input_B_ptr = reinterpret_cast<
            GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
                StrideInputB*>(stride_buf + stride_size);
    GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
        StrideOutput* stride_output_ptr = reinterpret_cast<
            GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
                StrideOutput*>(stride_buf + (stride_size * 2));

    output_args_ptr[group_index] =
        reinterpret_cast<int64_t>(output_data + output_offset);

    // Write kernel arguments directly to memory.
    xq_ptr_[group_index] = xq_ptr;
    wq_ptr_[group_index] = wq_ptr;
    x_scale_ptr_[group_index] = x_scale_ptr;
    w_scale_ptr_[group_index] = w_scale_ptr;
    problem_shape_ptr[group_index] =
        GroupedGemmArgs::ProblemShape::UnderlyingProblemShape(M, N, K);
    stride_input_A_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmArgs::
            GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::StrideInputA{},
        {M, K, 1});
    stride_input_B_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmArgs::
            GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::StrideInputB{},
        {N, K, 1});
    stride_output_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmArgs::
            GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::StrideOutput{},
        {M, N, 1});
  }
}

__global__ void set_dynamic_kernel_args_kernel(
    GroupedGemmArgs::ElementInputA* xq_ptr,
    GroupedGemmArgs::ElementInputB* wq_ptr,
    GroupedGemmArgs::ElementComputeEpilogue* x_scale_ptr,
    GroupedGemmArgs::ElementComputeEpilogue* w_scale_ptr,
    int64_t* input_args_ptr,
    int64_t* output_args_ptr,
    GroupedGemmArgs::ElementOutput* output_data,
    int xq_ptr_offset,
    int wq_ptr_offset,
    int x_scale_ptr_offset,
    int w_scale_ptr_offset,
    int problem_shape_buf_offset,
    int stride_buf_offset,
    int stride_size,
    int group_count,
    int problem_shape_size,
    std::optional<int64_t*> zero_start_index_M,
    std::optional<int64_t*> M_sizes,
    int M,
    int N,
    int K) {
  int group_index = blockIdx.x * blockDim.x + threadIdx.x;
  // If this thread corresponds to a valid group, write kernel args to device
  // memory.
  if (group_index < group_count) {
    int64_t* xq_ptr_ = input_args_ptr + xq_ptr_offset;
    int64_t* wq_ptr_ = input_args_ptr + wq_ptr_offset;
    int64_t* x_scale_ptr_ = input_args_ptr + x_scale_ptr_offset;
    int64_t* w_scale_ptr_ = input_args_ptr + w_scale_ptr_offset;
    uint8_t* problem_shape_buf =
        reinterpret_cast<uint8_t*>(input_args_ptr + problem_shape_buf_offset);
    uint8_t* stride_buf =
        reinterpret_cast<uint8_t*>(input_args_ptr + stride_buf_offset);

    GroupedGemmArgs::ProblemShape::UnderlyingProblemShape* problem_shape_ptr =
        reinterpret_cast<
            GroupedGemmArgs::ProblemShape::UnderlyingProblemShape*>(
            problem_shape_buf);
    // Pass dummy configs to get Stride structure
    GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
        StrideInputA* stride_input_A_ptr = reinterpret_cast<
            GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
                StrideInputA*>(stride_buf);
    GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
        StrideInputB* stride_input_B_ptr = reinterpret_cast<
            GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
                StrideInputB*>(stride_buf + stride_size);
    GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
        StrideOutput* stride_output_ptr = reinterpret_cast<
            GroupedGemmArgs::GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::
                StrideOutput*>(stride_buf + (stride_size * 2));

    int offset_M;
    int kernel_M;
    if (zero_start_index_M.has_value()) {
      // For inputs with padding, M is fixed and the number of rows
      // to operate on is available in zero_start_index_M.
      offset_M = group_index * M;
      kernel_M = zero_start_index_M.value()[group_index];
    } else {
      // M for this group is pulled directly from M_sizes.
      kernel_M = M_sizes.value()[group_index];
      // We compute the offset by getting the cumulative sum over
      // prior groups.
      offset_M = 0;
      for (int i = 0; i < group_index; i++) {
        offset_M += M_sizes.value()[i];
      }
    }

    output_args_ptr[group_index] =
        reinterpret_cast<int64_t>(output_data + (offset_M * N));

    // Write kernel arguments directly to memory.
    xq_ptr_[group_index] = reinterpret_cast<int64_t>(xq_ptr + (offset_M * K));
    wq_ptr_[group_index] =
        reinterpret_cast<int64_t>(wq_ptr + (group_index * N * K));
    x_scale_ptr_[group_index] =
        reinterpret_cast<int64_t>(x_scale_ptr + offset_M);
    w_scale_ptr_[group_index] =
        reinterpret_cast<int64_t>(w_scale_ptr + (group_index * N));
    problem_shape_ptr[group_index] =
        GroupedGemmArgs::ProblemShape::UnderlyingProblemShape(kernel_M, N, K);
    stride_input_A_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmArgs::
            GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::StrideInputA{},
        {kernel_M, K, 1});
    stride_input_B_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmArgs::
            GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::StrideInputB{},
        {N, K, 1});
    stride_output_ptr[group_index] = cutlass::make_cute_packed_stride(
        typename GroupedGemmArgs::
            GroupedGemmConfigs<128, 256, 128, 2, 1, 1, false>::StrideOutput{},
        {kernel_M, N, 1});
  }
}

template <
    typename InputType,
    int TB_M,
    int TB_N,
    int TB_K,
    int TBS_M,
    int TBS_N,
    int TBS_K,
    bool PONG>
at::Tensor f8f8bf16_rowwise_grouped_impl(
    InputType XQ, // FP8
    InputType WQ, // FP8
    InputType x_scale,
    InputType w_scale,
    at::Tensor output,
    std::optional<at::Tensor> zero_start_index_M,
    std::optional<at::Tensor> M_sizes) {
  int group_count;
  at::TensorOptions options;
  if constexpr (std::is_same_v<InputType, at::TensorList>) {
    group_count = XQ.size();
    options = XQ[0].options();
    TORCH_CHECK(WQ.size() == group_count);
  } else {
    TORCH_CHECK(
        zero_start_index_M.has_value() != M_sizes.has_value(),
        "One of zero_start_index_M or M_sizes must be provided.");
    group_count = WQ.size(0);
    options = XQ.options();
  }
  if (group_count == 0) {
    return at::Tensor();
  }
  using GroupedGemmConfigs = GroupedGemmArgs::
      GroupedGemmConfigs<TB_M, TB_N, TB_K, TBS_M, TBS_N, TBS_K, PONG>;

  int64_t total_output_size = 0;
  std::vector<int64_t> output_sizes;
  output_sizes.reserve(group_count);
  at::Tensor output_args = at::empty({group_count}, options.dtype(at::kLong));

  const int64_t problem_shape_size = group_count *
      ((int64_t)sizeof(GroupedGemmArgs::ProblemShape::UnderlyingProblemShape));
  const int64_t stride_size = group_count *
      ((int64_t)sizeof(typename GroupedGemmConfigs::StrideInputA));

  // TODO: Though pointer buffer with 1000 is suitable all of our usecases, we
  // should refactor pointer buffer with better general strategy to avoid this
  // number
  at::Tensor input_args = at::empty(
      {group_count * 4 + problem_shape_size + stride_size * 3 + 1000},
      options.dtype(at::kLong));

  int xq_ptr_offset = 0;
  int wq_ptr_offset = group_count * sizeof(int64_t);
  int x_scale_ptr_offset = group_count * 2 * sizeof(int64_t);
  int w_scale_ptr_offset = group_count * 3 * sizeof(int64_t);
  int problem_shape_buf_offset = group_count * 4 * sizeof(int64_t);
  int stride_buf_offset =
      group_count * 4 * sizeof(int64_t) + problem_shape_size;

  if constexpr (std::is_same_v<InputType, at::TensorList>) {
    for (int i = 0; i < group_count; ++i) {
      const int64_t output_size = XQ[i].size(0) * WQ[i].size(0);
      total_output_size += output_size;
      output_sizes.push_back(output_size);
    }
  } else {
    // When inputs are pregrouped, output has G * M * N elements.
    total_output_size = group_count * XQ.size(1) * WQ.size(1);
    for (int i = 0; i < group_count; ++i) {
      output_sizes.push_back(XQ.size(1) * WQ.size(1));
    }
  }

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  // Set kernel arguments for tensor list inputs.
  // The strategy here is to iterate over each group and set the corresponding
  // device memory separately. This is the best way to allow true dynamic
  // shapes.
  if constexpr (std::is_same_v<InputType, at::TensorList>) {
    int const blockSize = 256;
    int const numBlocks = 1;
    int64_t output_offset = 0;

    for (int i = 0; i < group_count; ++i) {
      int M, N, K;
      int64_t xq_ptr, wq_ptr, x_scale_ptr, w_scale_ptr;
      // Compute buffer pointers based on input type.
      M = XQ[i].size(0);
      N = WQ[i].size(0);
      K = XQ[i].size(1);
      TORCH_CHECK_EQ(WQ[i].size(1), K);
      // Calculate data pointer for this group.
      xq_ptr = reinterpret_cast<int64_t>(XQ[i].data_ptr());
      wq_ptr = reinterpret_cast<int64_t>(WQ[i].data_ptr());
      x_scale_ptr = reinterpret_cast<int64_t>(x_scale[i].data_ptr());
      w_scale_ptr = reinterpret_cast<int64_t>(w_scale[i].data_ptr());
      set_kernel_args_kernel<<<numBlocks, blockSize, 0, stream>>>(
          xq_ptr,
          wq_ptr,
          x_scale_ptr,
          w_scale_ptr,
          input_args.data_ptr<int64_t>(),
          output_args.data_ptr<int64_t>(),
          output.data_ptr<at::BFloat16>(),
          output_offset,
          xq_ptr_offset,
          wq_ptr_offset,
          x_scale_ptr_offset,
          w_scale_ptr_offset,
          problem_shape_buf_offset,
          stride_buf_offset,
          stride_size,
          group_count,
          problem_shape_size,
          i,
          M,
          N,
          K);
      output_offset += output_sizes[i];
    }
  } else {
    // For Tensor inputs, we can set all group arguments in a single kernel
    // launch.
    TORCH_CHECK(
        !zero_start_index_M.has_value() ||
            zero_start_index_M->dtype() == at::kLong,
        "zero_start_index_M must be int64.");

    TORCH_CHECK(
        !M_sizes.has_value() || M_sizes->dtype() == at::kLong,
        "M_sizes must be int64.");
    int const blockSize = std::min(1024, group_count);
    int const numBlocks = (group_count + blockSize - 1) / blockSize;
    // When m_offsets is used, XQ is shape [total_M, K]. When zero_start_index_M
    // is used, shape is [G, M, K].
    int M = XQ.size(XQ.dim() - 2);
    int N = WQ.size(1);
    int K = WQ.size(2);
    std::optional<int64_t*> zero_start_index_M_ptr = std::nullopt;
    std::optional<int64_t*> M_sizes_ptr = std::nullopt;
    if (zero_start_index_M.has_value()) {
      zero_start_index_M_ptr =
          reinterpret_cast<int64_t*>(zero_start_index_M.value().data_ptr());
    }
    if (M_sizes.has_value()) {
      M_sizes_ptr = reinterpret_cast<int64_t*>(M_sizes.value().data_ptr());
    }
    set_dynamic_kernel_args_kernel<<<numBlocks, blockSize, 0, stream>>>(
        reinterpret_cast<GroupedGemmArgs::ElementInputA*>(XQ.data_ptr()),
        reinterpret_cast<GroupedGemmArgs::ElementInputB*>(WQ.data_ptr()),
        reinterpret_cast<GroupedGemmArgs::ElementComputeEpilogue*>(
            x_scale.data_ptr()),
        reinterpret_cast<GroupedGemmArgs::ElementComputeEpilogue*>(
            w_scale.data_ptr()),
        input_args.data_ptr<int64_t>(),
        output_args.data_ptr<int64_t>(),
        reinterpret_cast<GroupedGemmArgs::ElementOutput*>(output.data_ptr()),
        xq_ptr_offset,
        wq_ptr_offset,
        x_scale_ptr_offset,
        w_scale_ptr_offset,
        problem_shape_buf_offset,
        stride_buf_offset,
        stride_size,
        group_count,
        problem_shape_size,
        zero_start_index_M_ptr,
        M_sizes_ptr,
        M,
        N,
        K);
  }

  int64_t* output_ptr = output_args.data_ptr<int64_t>();
  int64_t* xq_ptr = input_args.data_ptr<int64_t>() + xq_ptr_offset;
  int64_t* wq_ptr = input_args.data_ptr<int64_t>() + wq_ptr_offset;
  int64_t* x_scale_ptr = input_args.data_ptr<int64_t>() + x_scale_ptr_offset;
  int64_t* w_scale_ptr = input_args.data_ptr<int64_t>() + w_scale_ptr_offset;
  uint8_t* problem_shape_buf = reinterpret_cast<uint8_t*>(
      input_args.data_ptr<int64_t>() + problem_shape_buf_offset);
  uint8_t* stride_buf = reinterpret_cast<uint8_t*>(
      input_args.data_ptr<int64_t>() + stride_buf_offset);

  GroupedGemmArgs::ProblemShape::UnderlyingProblemShape* problem_shape_ptr =
      reinterpret_cast<GroupedGemmArgs::ProblemShape::UnderlyingProblemShape*>(
          problem_shape_buf);
  typename GroupedGemmConfigs::StrideInputA* stride_input_A_ptr =
      reinterpret_cast<typename GroupedGemmConfigs::StrideInputA*>(stride_buf);
  typename GroupedGemmConfigs::StrideInputB* stride_input_B_ptr =
      reinterpret_cast<typename GroupedGemmConfigs::StrideInputB*>(
          stride_buf + stride_size);
  typename GroupedGemmConfigs::StrideOutput* stride_output_ptr =
      reinterpret_cast<typename GroupedGemmConfigs::StrideOutput*>(
          stride_buf + (stride_size * 2));

  typename GroupedGemmConfigs::Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGrouped,
      {group_count, problem_shape_ptr, nullptr},
      {reinterpret_cast<const GroupedGemmArgs::ElementInputA**>(xq_ptr),
       stride_input_A_ptr,
       reinterpret_cast<const GroupedGemmArgs::ElementInputB**>(wq_ptr),
       stride_input_B_ptr},
      {{},
       reinterpret_cast<const GroupedGemmArgs::ElementOutput**>(output_ptr),
       stride_output_ptr,
       reinterpret_cast<GroupedGemmArgs::ElementOutput**>(output_ptr),
       stride_output_ptr}};

  arguments.epilogue.thread = {
      {reinterpret_cast<const GroupedGemmArgs::ElementComputeEpilogue**>(
          x_scale_ptr)}, // x_scale
      // compute_0
      {
          {reinterpret_cast<const GroupedGemmArgs::ElementComputeEpilogue**>(
              w_scale_ptr)}, // w_scale
          {}, // Accumulator
          {} // Multiplies
      },
      {}, // Multiplies
  };

  typename GroupedGemmConfigs::Gemm gemm;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size =
      GroupedGemmConfigs::Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  at::Tensor workspace =
      at::empty(workspace_size, XQ[0].options().dtype(at::kByte));

  // Check the problem size is supported or not
  cutlass::Status status = gemm.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm.initialize(
      arguments, reinterpret_cast<uint8_t*>(workspace.data_ptr()));
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm(at::cuda::getCurrentCUDAStream());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error(
        std::string("cutlass cannot run") +
        cutlass::cutlassGetStatusString(status));
  }

  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return output;
}

// FP8 Tensorwise grouped cutlass kernel dispatch.
template <typename InputType>
at::Tensor dispatch_fp8_grouped_kernel(
    InputType XQ, // FP8
    InputType WQ, // FP8
    InputType x_scale,
    InputType w_scale,
    at::Tensor output,
    std::optional<at::Tensor> zero_start_index_M = std::nullopt,
    std::optional<at::Tensor> M_sizes = std::nullopt) {
  KernelMode kernel = get_grouped_kernel_mode(XQ, WQ);
  if (kernel == KernelMode::Small) {
    return f8f8bf16_rowwise_grouped_impl<
        InputType,
        64,
        128,
        128,
        2,
        1,
        1,
        true>(XQ, WQ, x_scale, w_scale, output, zero_start_index_M, M_sizes);
  } else if (kernel == KernelMode::Large) {
    return f8f8bf16_rowwise_grouped_impl<
        InputType,
        128,
        256,
        128,
        2,
        1,
        1,
        false>(XQ, WQ, x_scale, w_scale, output, zero_start_index_M, M_sizes);
  } else {
    return f8f8bf16_rowwise_grouped_impl<
        InputType,
        128,
        256,
        64,
        2,
        1,
        1,
        false>(XQ, WQ, x_scale, w_scale, output, zero_start_index_M, M_sizes);
  }
}

std::vector<at::Tensor> f8f8bf16_rowwise_grouped(
    at::TensorList XQ, // FP8
    at::TensorList WQ, // FP8
    at::TensorList x_scale,
    at::TensorList w_scale,
    std::optional<std::vector<at::Tensor>> output = std::nullopt) {
  at::Tensor Y;
  int group_count = XQ.size();
  std::vector<int64_t> output_sizes;
  if (output.has_value()) {
    // Handle initialization check for output list.
    std::vector<at::Tensor> output_;
    output_ = output.value();
    TORCH_CHECK(
        output_.size() == group_count,
        "Output and input must have same number of groups.");
    // Check that output shapes are correct.
    for (int i = 0; i < group_count; i++) {
      int M = XQ[i].size(0);
      int N = WQ[i].size(0);
      int out_M = output_[i].size(0);
      int out_N = output_[i].size(1);
      TORCH_CHECK(
          M == out_M && N == out_N,
          "Output tensors do not have the expected shape.");
      TORCH_CHECK(
          output_[i].dtype() == at::kBFloat16,
          "Output dtype must be bfloat16.");
      output_sizes.push_back(out_M * out_N);
    }
    Y = at::stack(output.value(), 0);
    // Otherwise allocate a new output tensor.
  } else {
    int64_t total_output_size = 0;
    for (int i = 0; i < group_count; ++i) {
      const int64_t output_size = XQ[i].size(0) * WQ[i].size(0);
      total_output_size += output_size;
      output_sizes.push_back(output_size);
    }
    Y = at::empty(total_output_size, XQ[0].options().dtype(at::kBFloat16));
  }

  // Run kernel.
  at::Tensor g_out =
      dispatch_fp8_grouped_kernel<at::TensorList>(XQ, WQ, x_scale, w_scale, Y);

  // Return grouped view of output.
  std::vector<at::Tensor> output_group = g_out.split(output_sizes);
  for (int i = 0; i < group_count; ++i) {
    output_group[i] = output_group[i].view({XQ[i].size(0), WQ[i].size(0)});
  }
  return output_group;
}

at::Tensor f8f8bf16_rowwise_grouped_stacked(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor M_sizes,
    std::optional<at::Tensor> output = std::nullopt) {
  int total_M = XQ.size(0);
  int N = WQ.size(1);
  int group_count = M_sizes.size(0);
  TORCH_CHECK(
      M_sizes.device() == XQ.device(),
      "M_sizes must be on same device as inputs.");
  TORCH_CHECK(
      WQ.dim() == 3 && WQ.size(0) == group_count,
      "Weights should be shape [G, N, K].")
  at::Tensor Y = at::empty(total_M * N, XQ.options().dtype(at::kBFloat16));
  // Early exit for empty inputs.
  if (total_M == 0) {
    return Y.view({total_M, N});
  }
  // Return continuous view of output.
  at::Tensor out = dispatch_fp8_grouped_kernel<at::Tensor>(
      XQ, WQ, x_scale, w_scale, Y, std::nullopt, M_sizes);
  return out.view({total_M, N});
}

at::Tensor f8f8bf16_rowwise_grouped_dynamic(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor zero_start_index_M,
    bool zeroing_output_tensor = true) {
  TORCH_CHECK(
      zero_start_index_M.device() == XQ.device(),
      "zero_start_index_M must be on same device as inputs.");
  int group_count = XQ.size(0);
  int M = XQ.size(1);
  int N = WQ.size(1);
  int K = XQ.size(0);
  int total_output_size = group_count * M * N;
  at::Tensor Y;
  if (zeroing_output_tensor) {
    Y = at::zeros(total_output_size, XQ.options().dtype(at::kBFloat16));
  } else {
    Y = at::empty(total_output_size, XQ.options().dtype(at::kBFloat16));
  }

  // Return continuous view of output.
  at::Tensor output = dispatch_fp8_grouped_kernel<at::Tensor>(
      XQ, WQ, x_scale, w_scale, Y, zero_start_index_M);
  // View as proper shape.
  return output.view({-1, M, N});
}

#else

std::vector<at::Tensor> f8f8bf16_rowwise_grouped(
    at::TensorList XQ, // FP8
    at::TensorList WQ, // FP8
    at::TensorList x_scale,
    at::TensorList w_scale,
    std::optional<std::vector<at::Tensor>> output = std::nullopt) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

at::Tensor f8f8bf16_rowwise_grouped_stacked(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor M_sizes,
    std::optional<at::Tensor> output = std::nullopt) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

at::Tensor f8f8bf16_rowwise_grouped_dynamic(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor zero_start_index_M,
    bool zeroing_output_tensor = true) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

#endif

} // namespace fbgemm_gpu
