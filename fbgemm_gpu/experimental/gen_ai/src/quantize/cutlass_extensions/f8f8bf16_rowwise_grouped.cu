/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
// clang-format on

#include "f8f8bf16_rowwise_grouped/f8f8bf16_rowwise_grouped_manifest.cuh"
#include "f8f8bf16_rowwise_grouped_sm100/f8f8bf16_rowwise_grouped_manifest.cuh"
#include "fbgemm_gpu/quantize/tuning_cache.hpp"
#include "fbgemm_gpu/quantize/utils.h"

namespace fbgemm_gpu {

#if CUDART_VERSION >= 12000

namespace {
TuningCache& getTuningCache() {
  // This kernel has multiple APIs templated based on InputType, so we use this
  // to have a single cache instance across APIs.
  static TuningCache cache("f8f8bf16_rowwise_grouped");
  return cache;
}
} // namespace

template <typename InputType>
Kernel_f8f8bf16_rowwise_grouped<InputType>
get_kernel_via_heuristics(int total_M, int max_N, int max_K, int G) {
  static int arch = -1;
  // Avoid expensive hipGetDeviceProperties call.
  if (arch < 0) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (prop.major >= 10) {
      arch = 10;
      int runtimeVersion;
      C10_CUDA_CHECK(hipRuntimeGetVersion(&runtimeVersion));
      TORCH_CHECK(
          runtimeVersion >= 12080,
          "FP8 grouped GEMM on sm100a or above requires cuda >= 12.8");
    } else {
      arch = 9;
    }
  }

  // Use heuristics to pick the best kernel implementation.
  if (arch == 10) {
    // Llama4 shapes
    if ((max_N == 5120 && max_K == 1024) || (max_N == 2048 && max_K == 5120)) {
      if (total_M <= 256) {
        return f8f8bf16_rowwise_grouped_256_32_128_2_1_1_10_f;
      } else if (total_M <= 512) {
        return f8f8bf16_rowwise_grouped_256_64_128_2_1_1_10_f;
      } else if (total_M <= 1024) {
        return f8f8bf16_rowwise_grouped_256_128_128_2_1_1_10_f;
      } else {
        return f8f8bf16_rowwise_grouped_256_256_128_2_1_1_10_f;
      }
    }

    // Fallback to legacy heuristic.
    if (total_M <= 64 || (total_M <= 256 and max_N <= 1024)) {
      if (max_K <= 4096) {
        return f8f8bf16_rowwise_grouped_256_32_128_2_1_1_10_f;
      } else {
        return f8f8bf16_rowwise_grouped_128_32_128_2_1_1_10_f;
      }
    } else if (total_M <= 512) {
      if (max_N <= 1024) {
        return f8f8bf16_rowwise_grouped_128_64_128_2_1_1_10_f;
      } else if (max_N <= 8192) {
        if (max_K <= 2048) {
          return f8f8bf16_rowwise_grouped_256_32_128_2_1_1_10_f;
        } else if (max_K <= 4096) {
          return f8f8bf16_rowwise_grouped_128_32_128_2_1_1_10_f;
        } else {
          return f8f8bf16_rowwise_grouped_128_64_128_2_1_1_10_f;
        }
      }
    } else if (total_M <= 1024) {
      if (max_N <= 1024) {
        return f8f8bf16_rowwise_grouped_128_128_128_2_1_1_10_f;
      } else if (max_N <= 8192) {
        if (max_K <= 2048) {
          return f8f8bf16_rowwise_grouped_256_64_128_2_1_1_10_f;
        } else if (max_K <= 4096) {
          return f8f8bf16_rowwise_grouped_128_64_128_2_1_1_10_f;
        } else {
          return f8f8bf16_rowwise_grouped_128_128_128_2_1_1_10_f;
        }
      }
    } else if (total_M <= 2048) {
      if (max_N <= 1024) {
        return f8f8bf16_rowwise_grouped_256_256_128_2_1_1_10_f;
      } else if (max_N <= 8192) {
        if (max_K <= 2048) {
          return f8f8bf16_rowwise_grouped_256_128_128_2_1_1_10_f;
        } else if (max_K <= 4096) {
          return f8f8bf16_rowwise_grouped_128_128_128_2_1_1_10_f;
        }
      }
    }
    return f8f8bf16_rowwise_grouped_256_256_128_2_1_1_10_f;
  } else {
    // LLama4 16E
    if (max_N == 2048 && max_K == 5120 && G == 16) {
      if (total_M <= 256) {
        return f8f8bf16_rowwise_grouped_128_16_128_2_1_1_9_f;
      } else if (total_M <= 512) {
        return f8f8bf16_rowwise_grouped_128_32_128_2_1_1_9_f;
      } else if (total_M <= 1024) {
        return f8f8bf16_rowwise_grouped_128_64_128_2_1_1_9_f;
      } else if (total_M <= 2048) {
        return f8f8bf16_rowwise_grouped_128_128_128_2_1_1_9_f;
      } else {
        return f8f8bf16_rowwise_grouped_128_256_128_1_1_1_9_f;
      }
    }
    if (max_N == 5120 && max_K == 1024 && G == 16) {
      if (total_M <= 16) {
        return f8f8bf16_rowwise_grouped_128_16_128_2_1_1_9_f;
      } else if (total_M <= 256) {
        return f8f8bf16_rowwise_grouped_256_32_128_4_1_1_9_f;
      } else if (total_M <= 512) {
        return f8f8bf16_rowwise_grouped_256_32_128_1_1_1_9_f;
      } else if (total_M <= 1024) {
        return f8f8bf16_rowwise_grouped_256_64_128_1_1_1_9_f;
      } else if (total_M <= 1536) {
        return f8f8bf16_rowwise_grouped_256_128_128_4_1_1_9_f;
      } else if (total_M <= 2048) {
        return f8f8bf16_rowwise_grouped_256_64_128_1_1_1_9_f;
      } else if (total_M <= 4096) {
        return f8f8bf16_rowwise_grouped_128_256_128_2_1_1_9_f;
      } else {
        return f8f8bf16_rowwise_grouped_128_256_128_1_1_1_9_f;
      }
    }
    // LLama4 128E
    if (max_N == 5120 && max_K == 1024 && G == 128) {
      if (total_M <= 128) {
        return f8f8bf16_rowwise_grouped_256_16_128_2_1_1_9_f;
      } else if (total_M <= 256) {
        return f8f8bf16_rowwise_grouped_256_16_128_2_1_1_9_f;
      } else if (total_M <= 2048) {
        return f8f8bf16_rowwise_grouped_256_16_128_1_1_1_9_f;
      } else if (total_M <= 4096) {
        return f8f8bf16_rowwise_grouped_256_32_128_1_1_1_9_f;
      } else if (total_M <= 8192) {
        return f8f8bf16_rowwise_grouped_256_64_128_1_1_1_9_f;
      } else if (total_M <= 12288) {
        return f8f8bf16_rowwise_grouped_256_128_128_2_1_1_9_f;
      } else if (total_M <= 32768) {
        return f8f8bf16_rowwise_grouped_256_128_128_1_1_1_9_f;
      } else {
        return f8f8bf16_rowwise_grouped_128_256_128_2_1_1_9_f;
      }
    }
    if (max_N == 2048 && max_K == 5120 && G == 128) {
      if (total_M <= 128) {
        return f8f8bf16_rowwise_grouped_256_16_128_1_1_1_9_f;
      } else if (total_M <= 512) {
        return f8f8bf16_rowwise_grouped_256_16_128_4_1_1_9_f;
      } else if (total_M <= 1024) {
        return f8f8bf16_rowwise_grouped_256_32_128_4_1_1_9_f;
      } else if (total_M <= 2048) {
        return f8f8bf16_rowwise_grouped_256_16_128_2_1_1_9_f;
      } else if (total_M <= 4096) {
        return f8f8bf16_rowwise_grouped_256_32_128_4_1_1_9_f;
      } else if (total_M <= 8192) {
        return f8f8bf16_rowwise_grouped_256_64_128_4_1_1_9_f;
      } else if (total_M <= 16384) {
        return f8f8bf16_rowwise_grouped_128_128_128_1_1_1_9_f;
      } else {
        return f8f8bf16_rowwise_grouped_128_256_128_2_1_1_9_f;
      }
    }

    if (total_M <= 16) {
      return f8f8bf16_rowwise_grouped_128_16_128_1_1_1_9_f;
    } else if (total_M <= 32) {
      return f8f8bf16_rowwise_grouped_128_32_128_1_1_1_9_f;
    } else if (total_M <= 64) {
      return f8f8bf16_rowwise_grouped_128_64_128_1_1_1_9_f;
    } else if (total_M <= 128) {
      return f8f8bf16_rowwise_grouped_128_128_128_1_1_1_9_f;
    } else if (total_M <= 512) {
      return f8f8bf16_rowwise_grouped_256_128_128_2_1_1_9_f;
    } else {
      return f8f8bf16_rowwise_grouped_128_256_128_2_1_1_9_f;
    }
  }
}

template <typename InputType>
Kernel_f8f8bf16_rowwise_grouped<InputType> get_kernel_via_tuning(
    int total_M,
    int max_N,
    int max_K,
    int G,
    InputType XQ, // FP8
    InputType WQ, // FP8
    InputType x_scale,
    InputType w_scale,
    at::Tensor output,
    std::optional<at::Tensor> zero_start_index_M = std::nullopt,
    std::optional<at::Tensor> M_sizes = std::nullopt) {
  auto& cache = getTuningCache();

  // Reducing amount of auto tuning by rounding up total_M to next power of 2.
  total_M = nextPowerOf2(total_M);
  // Use (total_M, max_N, max_K, G) shape as the key.
  const std::string shape_key = std::to_string(total_M) + "_" +
      std::to_string(max_N) + "_" + std::to_string(max_K) + "_" +
      std::to_string(G);
  const auto& kernels = get_f8f8bf16_rowwise_grouped_kernels<InputType>();
  auto kernel = cache.findBestKernelMaybeAutotune(
      shape_key,
      kernels,
      XQ,
      WQ,
      x_scale,
      w_scale,
      output,
      zero_start_index_M,
      M_sizes);

  return kernel;
}

// FP8 rowwise grouped cutlass kernel dispatch.
template <typename InputType>
at::Tensor dispatch_fp8_grouped_kernel(
    int total_M,
    int max_N,
    int max_K,
    int G,
    InputType XQ, // FP8
    InputType WQ, // FP8
    InputType x_scale,
    InputType w_scale,
    at::Tensor output,
    std::optional<at::Tensor> zero_start_index_M = std::nullopt,
    std::optional<at::Tensor> M_sizes = std::nullopt) {
  // Select kernel to run via heuristics or tuning.
  auto kernel = [&]() {
    if (std::getenv("FBGEMM_AUTOTUNE_ENABLE")) {
      return get_kernel_via_tuning(
          total_M,
          max_N,
          max_K,
          G,
          XQ,
          WQ,
          x_scale,
          w_scale,
          output,
          zero_start_index_M,
          M_sizes);
    } else {
      return get_kernel_via_heuristics<InputType>(total_M, max_N, max_K, G);
    }
  }();
  // Invoke kernel
  return kernel(XQ, WQ, x_scale, w_scale, output, zero_start_index_M, M_sizes);
}

template <typename OutputType>
OutputType _f8f8bf16_rowwise_grouped(
    at::TensorList XQ, // FP8
    at::TensorList WQ, // FP8
    at::TensorList x_scale,
    at::TensorList w_scale) {
  at::Tensor Y;
  int64_t total_M = 0;
  int64_t max_N = 0;
  int64_t max_K = 0;
  int64_t G = XQ.size();

  // Allocate output tensor.
  std::vector<int64_t> output_sizes;
  int64_t total_output_size = 0;
  for (int i = 0; i < G; ++i) {
    int64_t M = XQ[i].size(0);
    int64_t N = WQ[i].size(0);
    int64_t K = WQ[i].size(1);
    total_M += M;
    if (N > max_N) {
      max_N = N;
    }
    if (K > max_K) {
      max_K = K;
    }
    const int64_t output_size = M * N;
    total_output_size += output_size;
    output_sizes.push_back(output_size);
  }
  Y = at::empty(total_output_size, XQ[0].options().dtype(at::kBFloat16));

  // Run kernel.
  at::Tensor g_out = dispatch_fp8_grouped_kernel<at::TensorList>(
      total_M, max_N, max_K, G, XQ, WQ, x_scale, w_scale, Y);

  // Return appropriate output type.
  if constexpr (std::is_same_v<OutputType, at::Tensor>) {
    int64_t N = WQ[0].size(0);
    return g_out.view({total_M, N});
  } else {
    // Return grouped view of output.
    std::vector<at::Tensor> output_group = g_out.split(output_sizes);
    for (int i = 0; i < G; ++i) {
      output_group[i] = output_group[i].view({XQ[i].size(0), WQ[i].size(0)});
    }
    return output_group;
  }
}

std::vector<at::Tensor> f8f8bf16_rowwise_grouped(
    at::TensorList XQ, // FP8
    at::TensorList WQ, // FP8
    at::TensorList x_scale,
    at::TensorList w_scale) {
  return _f8f8bf16_rowwise_grouped<std::vector<at::Tensor>>(
      XQ, WQ, x_scale, w_scale);
}

at::Tensor f8f8bf16_rowwise_grouped_cat(
    at::TensorList XQ, // FP8
    at::TensorList WQ, // FP8
    at::TensorList x_scale,
    at::TensorList w_scale) {
  return _f8f8bf16_rowwise_grouped<at::Tensor>(XQ, WQ, x_scale, w_scale);
}

at::Tensor f8f8bf16_rowwise_grouped_stacked(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor M_sizes) {
  int64_t total_M = XQ.size(0);
  int64_t N = WQ.size(1);
  int64_t K = WQ.size(2);
  int64_t G = M_sizes.size(0);
  TORCH_CHECK(
      M_sizes.device() == XQ.device(),
      "M_sizes must be on same device as inputs.");
  TORCH_CHECK(
      WQ.dim() == 3 && WQ.size(0) == G, "Weights should be shape [G, N, K].")
  at::Tensor Y = at::empty(total_M * N, XQ.options().dtype(at::kBFloat16));
  // Early exit for empty inputs.
  if (total_M == 0) {
    return Y.view({total_M, N});
  }
  // Return continuous view of output.
  at::Tensor out = dispatch_fp8_grouped_kernel<at::Tensor>(
      total_M, N, K, G, XQ, WQ, x_scale, w_scale, Y, std::nullopt, M_sizes);
  return out.view({total_M, N});
}

at::Tensor f8f8bf16_rowwise_grouped_dynamic(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor zero_start_index_M,
    bool zeroing_output_tensor = true) {
  TORCH_CHECK(
      zero_start_index_M.device() == XQ.device(),
      "zero_start_index_M must be on same device as inputs.");
  int64_t G = XQ.size(0);
  int64_t M = XQ.size(1);
  int64_t N = WQ.size(1);
  int64_t K = WQ.size(2);
  int64_t total_output_size = G * M * N;
  at::Tensor Y;
  if (zeroing_output_tensor) {
    Y = at::zeros(total_output_size, XQ.options().dtype(at::kBFloat16));
  } else {
    Y = at::empty(total_output_size, XQ.options().dtype(at::kBFloat16));
  }

  // Return continuous view of output.
  at::Tensor output = dispatch_fp8_grouped_kernel<at::Tensor>(
      G * M, N, K, G, XQ, WQ, x_scale, w_scale, Y, zero_start_index_M);
  // View as proper shape.
  return output.view({G, M, N});
}

#else

std::vector<at::Tensor> f8f8bf16_rowwise_grouped(
    at::TensorList XQ, // FP8
    at::TensorList WQ, // FP8
    at::TensorList x_scale,
    at::TensorList w_scale) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

at::Tensor f8f8bf16_rowwise_grouped_cat(
    at::TensorList XQ, // FP8
    at::TensorList WQ, // FP8
    at::TensorList x_scale,
    at::TensorList w_scale) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

at::Tensor f8f8bf16_rowwise_grouped_stacked(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor M_sizes) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

at::Tensor f8f8bf16_rowwise_grouped_dynamic(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor zero_start_index_M,
    bool zeroing_output_tensor = true) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

#endif

} // namespace fbgemm_gpu
