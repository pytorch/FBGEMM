#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <optional>

#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <c10/cuda/CUDAStream.h>
#ifdef USE_ROCM
#include <hip/hip_fp16.h>
#else
#include <hip/hip_bf16.h>
#endif
#include <torch/torch.h>

#define DISPATCH_CASE_FLOATING_TYPES(...)              \
  AT_DISPATCH_CASE(at::ScalarType::Float, __VA_ARGS__) \
  AT_DISPATCH_CASE(at::ScalarType::BFloat16, __VA_ARGS__)

namespace fbgemm_gpu {

namespace {

#ifdef USE_ROCM
constexpr int kNumThreadsPerWarp = 64;
#else
constexpr int kNumThreadsPerWarp = 32;
#endif

__inline__ constexpr int ceil_of_ratio(int a, int b) {
  return (a + b - 1) / b;
};

#ifdef USE_ROCM
__device__ __forceinline__ int atomic_add_relaxed(int* addr, int inc) {
  return __hip_atomic_fetch_add(
      addr, inc, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
};

__device__ __forceinline__ int atomic_add_release(int* addr, int inc) {
  return __hip_atomic_fetch_add(
      addr, inc, __ATOMIC_RELEASE, __HIP_MEMORY_SCOPE_AGENT);
};

__device__ __forceinline__ int load_aquire(int* addr) {
  return __hip_atomic_load(addr, __ATOMIC_ACQUIRE, __HIP_MEMORY_SCOPE_AGENT);
};
#else
__device__ __forceinline__ int atomic_add_relaxed(int* addr, int inc) {
  int val;
  asm volatile("atom.relaxed.gpu.global.add.s32 %0, [%1], %2;\n"
               : "=r"(val)
               : "l"(addr), "r"(inc));
  return val;
};

__device__ __forceinline__ int atomic_add_release(int* addr, int inc) {
  int val;
  asm volatile("atom.release.gpu.global.add.s32 %0, [%1], %2;\n"
               : "=r"(val)
               : "l"(addr), "r"(inc));
  return val;
};

__device__ __forceinline__ int load_aquire(int* addr) {
  int val;
  asm volatile("ld.global.acquire.gpu.b32 %0, [%1];\n" : "=r"(val) : "l"(addr));
  return val;
};
#endif

template <
    class DataType,
    class IndexType,
    int NumExperts,
    int NumTokensPerTile,
    int TopK>
struct SharedStorage {
  DataType routing_scores[NumTokensPerTile * NumExperts * TopK];
  IndexType expert_indices[NumTokensPerTile * NumExperts * TopK];
  IndexType token_count_cumsums[NumExperts];
};

template <class DataType, class IndexType>
struct Params {
  // 1. Inputs
  // 1.1. Routing scores.
  const DataType* routing_scores;
  const int stride_t;
  const int stride_e;
  // 1.2. Expert ranges.
  const int expert_index_start;
  const int expert_index_end;
  // 1.3. Token counts.
  const IndexType* valid_token_count;
  const int num_tokens;
  const int num_tokens_per_cta;

  // 2. Buffers
  IndexType* buffered_expert_indices;
  IndexType* buffered_token_indices;

  // 3. Outputs
  IndexType* token_count_per_expert;
  IndexType* shuffled_expert_indices;
  IndexType* shuffled_token_indices;
};

template <class DataType, class IndexType>
__device__ __forceinline__ void merge_top1(
    DataType* routing_scores,
    IndexType* expert_indices,
    int lhs_smem_index,
    int rhs_smem_index) {
  auto lhs_score = routing_scores[lhs_smem_index];
  auto rhs_score = routing_scores[rhs_smem_index];
  auto lhs_expert_index = expert_indices[lhs_smem_index];
  auto rhs_expert_index = expert_indices[rhs_smem_index];

  bool lhs_larger = lhs_score >= rhs_score;
  routing_scores[lhs_smem_index] = lhs_larger ? lhs_score : rhs_score;
  expert_indices[lhs_smem_index] =
      lhs_larger ? lhs_expert_index : rhs_expert_index;
}

template <class DataType, class IndexType>
__device__ __forceinline__ void merge_top2(
    DataType* routing_scores,
    IndexType* expert_indices,
    int lhs_smem_index,
    int rhs_smem_index,
    bool skip_duplicates) {
  auto lhs_score0 = routing_scores[lhs_smem_index];
  auto lhs_score1 = routing_scores[lhs_smem_index + 1];
  auto rhs_score0 = routing_scores[rhs_smem_index];
  auto rhs_score1 = routing_scores[rhs_smem_index + 1];
  auto lhs_expert_index0 = expert_indices[lhs_smem_index];
  auto lhs_expert_index1 = expert_indices[lhs_smem_index + 1];
  auto rhs_expert_index0 = expert_indices[rhs_smem_index];
  auto rhs_expert_index1 = expert_indices[rhs_smem_index + 1];

  if (lhs_score0 >= rhs_score0) {
    routing_scores[lhs_smem_index] = lhs_score0;
    expert_indices[lhs_smem_index] = lhs_expert_index0;

    if ((lhs_score1 >= rhs_score0) && !skip_duplicates) {
      routing_scores[lhs_smem_index + 1] = lhs_score1;
      expert_indices[lhs_smem_index + 1] = lhs_expert_index1;
    } else {
      routing_scores[lhs_smem_index + 1] = rhs_score0;
      expert_indices[lhs_smem_index + 1] = rhs_expert_index0;
    }
  } else {
    routing_scores[lhs_smem_index] = rhs_score0;
    expert_indices[lhs_smem_index] = rhs_expert_index0;

    if ((rhs_score1 > lhs_score0) && !skip_duplicates) {
      routing_scores[lhs_smem_index + 1] = rhs_score1;
      expert_indices[lhs_smem_index + 1] = rhs_expert_index1;
    } else {
      routing_scores[lhs_smem_index + 1] = lhs_score0;
      expert_indices[lhs_smem_index + 1] = lhs_expert_index0;
    }
  }
}

template <class DataType, class IndexType, int K>
__device__ __forceinline__ void merge_topk(
    DataType* routing_scores,
    IndexType* expert_indices,
    int lhs_smem_index,
    int rhs_smem_index,
    int num_valid_values) {
  /**
  @param num_valid_values At first log2(K) calls to this function, both inputs
  would only contain num_valid_values = 2 ** (step_number - 1) meaningful
  values, the rest are duplicates. So the function should take exactly
  num_valid_values elements from the first input and num_valid_values elements
  from the second input. After the first log2(K) calls, all elements in the
  inputs are meaningful, and num_valid_values doesn't have any effect (i.e.
  num_valid_values >= K).
  */
  // Temporary arrays to store the merged result
  DataType merged_scores[K];
  IndexType merged_indices[K];

  // Pointers to the left and right arrays
  DataType* lhs_scores = &routing_scores[lhs_smem_index];
  DataType* rhs_scores = &routing_scores[rhs_smem_index];
  IndexType* lhs_indices = &expert_indices[lhs_smem_index];
  IndexType* rhs_indices = &expert_indices[rhs_smem_index];

  // Merge the two sorted arrays (assuming both are sorted in descending order)
  int lhs_idx = 0;
  int rhs_idx = 0;
  int merged_idx = 0;

  int max_from_one_array = std::min(num_valid_values, K);

  // Get the top K elements from the two arrays
  while (merged_idx < K) {
    // If we've exhausted the left array, take from the right
    if (lhs_idx >= max_from_one_array) {
      merged_scores[merged_idx] = rhs_scores[rhs_idx];
      merged_indices[merged_idx] = rhs_indices[rhs_idx];
      rhs_idx++;
      merged_idx++;
      continue;
    }

    // If we've exhausted the right array, take from the left
    if (rhs_idx >= max_from_one_array) {
      merged_scores[merged_idx] = lhs_scores[lhs_idx];
      merged_indices[merged_idx] = lhs_indices[lhs_idx];
      lhs_idx++;
      merged_idx++;
      continue;
    }

    // Compare the current elements from both arrays and take the larger one
    if (lhs_scores[lhs_idx] >= rhs_scores[rhs_idx]) {
      merged_scores[merged_idx] = lhs_scores[lhs_idx];
      merged_indices[merged_idx] = lhs_indices[lhs_idx];
      lhs_idx++;
    } else {
      merged_scores[merged_idx] = rhs_scores[rhs_idx];
      merged_indices[merged_idx] = rhs_indices[rhs_idx];
      rhs_idx++;
    }
    merged_idx++;
  }

  // Write the merged result back to the left-hand side positions
  for (int i = 0; i < K; i++) {
    routing_scores[lhs_smem_index + i] = merged_scores[i];
    expert_indices[lhs_smem_index + i] = merged_indices[i];
  }
}

template <
    class DataType,
    class IndexType,
    int NumExperts,
    int NumTokensPerTile,
    int TopK>
__global__ void index_shuffling_kernel(Params<DataType, IndexType> params) {
  // scores: [num_tokens, num_experts]
  // counts: [num_experts]
  // expert_indices/shuffled_expert_indices: [num_tokens]
  // token_indices/shuffled_token_indices: [num_tokens]

  // When the number of experts is large, increase the total number of threads
  // to make the parallel reduction below work. Note that this logic is
  // duplicated inside index_shuffling_torch, but for const instead of
  // constexpr.
  constexpr int kNumWarps = (NumExperts > 128) ? 8 : 4;
  constexpr int kNumThreads = kNumThreadsPerWarp * kNumWarps;

  extern __shared__ char shared_memory[];
  auto& smem = *reinterpret_cast<
      SharedStorage<DataType, IndexType, NumExperts, NumTokensPerTile, TopK>*>(
      shared_memory);

  const auto tidx = threadIdx.x;
  const auto bidx = blockIdx.x;

  const int num_total_tokens = params.num_tokens;
  const int token_index_start = bidx * params.num_tokens_per_cta;
  if (token_index_start >= num_total_tokens) {
    return;
  }

  const int token_index_end =
      std::min(token_index_start + params.num_tokens_per_cta, num_total_tokens);
  const int num_valid_tokens =
      params.valid_token_count ? *params.valid_token_count : num_total_tokens;

  const int expert_index_start = params.expert_index_start;
  const int expert_index_end = params.expert_index_end;

  const int stride_t = params.stride_t;
  const int stride_e = params.stride_e;

  const DataType zero = static_cast<DataType>(-INFINITY);
  for (int token_index_offset = token_index_start;
       token_index_offset < token_index_end;
       token_index_offset += NumTokensPerTile) {
    // 1. Read scores
    // TODO(shikaili): vectorized. asynchronous.
#pragma unroll
    for (int i = tidx; i < NumTokensPerTile * NumExperts; i += kNumThreads) {
      const int token_index = token_index_offset + i / NumExperts;
      const int expert_index = i % NumExperts;
#pragma unroll
      for (int j = 0; j < TopK; j++) {
        smem.routing_scores[i * TopK + j] = (token_index < num_valid_tokens)
            ? params.routing_scores
                  [token_index * stride_t + expert_index * stride_e]
            : zero;
        smem.expert_indices[i * TopK + j] = expert_index;
      }
    }
    __syncthreads();

    // 2. Top-1 Reduction
    static_assert(NumExperts % 2 == 0, "");
    // When there are many experts, increase kNumParallelReductionThreads to
    // make sure it's above NumExperts/2, so the parallel reduction covers all
    // experts.
    constexpr int kNumParallelReductionThreads =
        (NumExperts > 128) ? 256 : (NumExperts / 2);

    static_assert(kNumThreads % kNumParallelReductionThreads == 0, "");
    constexpr int kNumParallelReductionGroups =
        kNumThreads / kNumParallelReductionThreads;

    static_assert(NumTokensPerTile % kNumParallelReductionGroups == 0, "");

    // 2D parallel reduction. No bank conflicts.
    for (int num_reduced_threads = 1;
         num_reduced_threads <= kNumParallelReductionThreads;
         num_reduced_threads <<= 1) {
#pragma unroll
      for (int local_token_offset = 0; local_token_offset < NumTokensPerTile;
           local_token_offset += kNumParallelReductionGroups) {
        if (!(tidx & (num_reduced_threads - 1))) {
          int local_token_index =
              local_token_offset + tidx / kNumParallelReductionThreads;
          int lhs_expert_offset = (tidx % kNumParallelReductionThreads) * 2;
          int lhs_smem_index =
              (local_token_index * NumExperts + lhs_expert_offset) * TopK;

          int rhs_smem_index = lhs_smem_index + num_reduced_threads * TopK;
          if (lhs_expert_offset + num_reduced_threads < NumExperts) {
            if (TopK == 1) {
              merge_top1(
                  smem.routing_scores,
                  smem.expert_indices,
                  lhs_smem_index,
                  rhs_smem_index);
            } else if (TopK == 2) {
              merge_top2(
                  smem.routing_scores,
                  smem.expert_indices,
                  lhs_smem_index,
                  rhs_smem_index,
                  num_reduced_threads == 1);
            } else {
              merge_topk<DataType, IndexType, 4>(
                  smem.routing_scores,
                  smem.expert_indices,
                  lhs_smem_index,
                  rhs_smem_index,
                  num_reduced_threads);
            }
          }
        }
        if (TopK > 1) {
          __syncthreads();
        }
      }
#ifdef USE_ROCM
      __syncthreads();
#else
      if constexpr (kNumParallelReductionThreads <= kNumThreadsPerWarp) {
        __syncwarp();
      } else {
        __syncthreads();
      }
#endif
    }
    if constexpr (kNumParallelReductionThreads > kNumThreadsPerWarp) {
      __syncthreads();
    }

    // 3. Counting
#pragma unroll
    for (int i = tidx; i < NumTokensPerTile; i += kNumThreads) {
      int local_token_index = i;
      int token_index = token_index_offset + i;
      if (token_index < num_valid_tokens) {
#pragma unroll
        for (int j = 0; j < TopK; j++) {
          auto expert_index =
              smem.expert_indices[local_token_index * NumExperts * TopK + j];
          params.buffered_expert_indices[token_index * TopK + j] = expert_index;
          if (expert_index >= expert_index_start &&
              expert_index < expert_index_end) {
            auto token_index_in_expert = atomic_add_relaxed(
                &params.token_count_per_expert[expert_index], 1);
            params.buffered_token_indices[token_index * TopK + j] =
                token_index_in_expert;
          }
        }
      }
    }
    __syncthreads();
  }

  if (tidx == 0) {
    int processed_tokens = 0;
    int* processed_tokens_addr = &params.token_count_per_expert[NumExperts];

    int inc = token_index_end - token_index_start;
    atomic_add_release(processed_tokens_addr, inc);

    do {
      processed_tokens = load_aquire(processed_tokens_addr);
    } while (processed_tokens != num_total_tokens);
  }
  __syncthreads();

  // 4. Scan
  for (int i = tidx; i < NumExperts; i += kNumThreads) {
    smem.token_count_cumsums[i] = params.token_count_per_expert[i];
  }
  __syncthreads();

  if (tidx == 0) {
    // TODO(shikaili): parallel.
#pragma unroll
    for (int i = 1; i < NumExperts; ++i) {
      smem.token_count_cumsums[i] += smem.token_count_cumsums[i - 1];
    }
  }
  __syncthreads();

  // 5. Store
  auto get_token_count_cumsum = [&smem](int index) {
    return index == 0 ? 0 : smem.token_count_cumsums[index - 1];
  };

  const int token_count_cumsum_start =
      get_token_count_cumsum(expert_index_start);
  const int token_count_cumsum_end = get_token_count_cumsum(expert_index_end);
  const int num_selected_tokens =
      token_count_cumsum_end - token_count_cumsum_start;

  for (int global_token_offset = bidx * params.num_tokens_per_cta;
       global_token_offset < (bidx + 1) * params.num_tokens_per_cta;
       global_token_offset += kNumThreads) {
    int token_index = global_token_offset + tidx;
    if (token_index < num_valid_tokens) {
#pragma unroll
      for (int j = 0; j < TopK; j++) {
        int expert_index =
            params.buffered_expert_indices[token_index * TopK + j];
        if (expert_index >= expert_index_start &&
            expert_index < expert_index_end) {
          int new_token_index_in_expert =
              params.buffered_token_indices[token_index * TopK + j];
          int new_token_index = get_token_count_cumsum(expert_index) -
              token_count_cumsum_start + new_token_index_in_expert;
          params.shuffled_expert_indices[new_token_index] =
              expert_index - expert_index_start;
          params.shuffled_token_indices[new_token_index] = token_index;
        }
      }
    }
  }

  if (tidx == 0 && bidx == 0) {
    params.token_count_per_expert[NumExperts + 1] = num_selected_tokens;
  }
}
} // namespace

std::tuple<at::Tensor, at::Tensor, at::Tensor> index_shuffling_torch(
    const at::Tensor& routing_scores,
    const std::optional<int64_t>& expert_index_start,
    const std::optional<int64_t>& expert_index_end,
    const std::optional<at::Tensor>& valid_token_count,
    const int64_t top_k = 1) {
  TORCH_CHECK(
      routing_scores.dtype() == torch::kBFloat16 ||
          routing_scores.dtype() == torch::kFloat,
      "routing_scores must be either BFloat16 or Float");

  using IndexType = int32_t;

  // Declare tensors outside the dispatch to ensure they're accessible for the
  // return statement
  at::Tensor token_count_per_expert;
  at::Tensor shuffled_expert_indices;
  at::Tensor shuffled_token_indices;

  AT_DISPATCH_SWITCH(
      routing_scores.scalar_type(),
      "index_shuffling_params",
      DISPATCH_CASE_FLOATING_TYPES([&] {
        using DataType = scalar_t;

        TORCH_CHECK(routing_scores.dim() == 2);
        const int num_tokens = routing_scores.size(0);
        const int num_experts = routing_scores.size(1);
        TORCH_CHECK(
            num_experts == 16 || num_experts == 32 || num_experts == 128 ||
            num_experts == 320);

        TORCH_CHECK(top_k == 1 || top_k == 2 || top_k == 4);

        auto allocate_index_tensor = [&](int size) {
          return at::empty(
              {size},
              at::TensorOptions().dtype(at::kInt).device(
                  routing_scores.device()));
        };
        token_count_per_expert = allocate_index_tensor(num_experts + 2);
        shuffled_expert_indices = allocate_index_tensor(num_tokens * top_k);
        shuffled_token_indices = allocate_index_tensor(num_tokens * top_k);
        at::Tensor buffered_expert_indices =
            allocate_index_tensor(num_tokens * top_k);
        at::Tensor buffered_token_indices =
            allocate_index_tensor(num_tokens * top_k);
#ifdef USE_ROCM
        // TODO(shikaili): hipMetsetAsync is more expensive than ATen set zero.
        token_count_per_expert.zero_();
#else
        hipMemsetAsync(
            token_count_per_expert.data_ptr(),
            0,
            token_count_per_expert.numel() *
                token_count_per_expert.dtype().itemsize(),
            at::cuda::getCurrentCUDAStream());
#endif

        // Avoid expensive `hipGetDeviceProperties` call.
        static int num_sms = -1;
        if (num_sms < 0) {
          hipDeviceProp_t deviceProp;
          hipGetDeviceProperties(&deviceProp, 0);
          num_sms = deviceProp.multiProcessorCount;
        }

#ifdef USE_ROCM
        constexpr int kNumTokensPerTileFewExperts = 32;
#else
        constexpr int kNumTokensPerTileFewExperts = 16;
#endif

        void* kernel;
        int smem_size;

// Reducing tile size as problem size increases to avoid
// hipErrorCooperativeLaunchTooLarge.
// TopK > 1 is not supported on AMD yet.
#ifndef USE_ROCM
#define DISPATCH(E, B, K, S)          \
  if (S <= 128) {                     \
    DISPATCH_K(E, B, K);              \
  } else if (storage_factor <= 256) { \
    DISPATCH_K(E, B / 2, K);          \
  } else if (storage_factor <= 512) { \
    DISPATCH_K(E, B / 4, K);          \
  } else {                            \
    DISPATCH_K(E, B / 8, K);          \
  }
#else
#define DISPATCH(E, B, K, S) \
  TORCH_CHECK(K == 1);       \
  DISPATCH_EB(E, 8, 1)
#endif

#define DISPATCH_K(E, B, K) \
  if (K == 1) {             \
    DISPATCH_EB(E, B, 1)    \
  } else if (K == 2) {      \
    DISPATCH_EB(E, B, 2)    \
  } else {                  \
    TORCH_CHECK(K == 4);    \
    DISPATCH_EB(E, B, 4)    \
  }
#define DISPATCH_EB(E, B, K)                                            \
  kernel = (void*)index_shuffling_kernel<DataType, IndexType, E, B, K>; \
  smem_size = sizeof(SharedStorage<DataType, IndexType, E, B, K>);

        int storage_factor = top_k * num_experts;

        if (num_experts == 16) {
          DISPATCH_K(16, kNumTokensPerTileFewExperts, top_k)
        } else if (num_experts == 32) {
          DISPATCH_K(32, kNumTokensPerTileFewExperts, top_k)
        } else if (num_experts == 128) {
          DISPATCH(128, kNumTokensPerTileFewExperts, top_k, storage_factor)
        } else {
          TORCH_CHECK(num_experts == 320);
          DISPATCH(320, kNumTokensPerTileFewExperts, top_k, storage_factor)
        }
    // This is to avoid build errors (divisibility asserts and local memory
    // overflow) on AMD.
#ifndef USE_ROCM
        const int num_tokens_per_tile = (storage_factor <= 128)
            ? kNumTokensPerTileFewExperts
            : ((storage_factor <= 256)
                   ? kNumTokensPerTileFewExperts / 2
                   : ((storage_factor <= 512)
                          ? kNumTokensPerTileFewExperts / 4
                          : kNumTokensPerTileFewExperts / 8));
#else
        const int num_tokens_per_tile = (num_experts <= 128)
            ? kNumTokensPerTileFewExperts
            : kNumTokensPerTileFewExperts / 4;
#endif

        const int num_tiles = ceil_of_ratio(num_tokens, num_tokens_per_tile);
        const int num_ctas = std::min(num_tiles, num_sms);
        const int num_tiles_per_cta = ceil_of_ratio(
            ceil_of_ratio(num_tokens, num_ctas), num_tokens_per_tile);
        const int num_tokens_per_cta = num_tiles_per_cta * num_tokens_per_tile;

        Params<DataType, IndexType> params = {
            // Inputs
            .routing_scores =
                reinterpret_cast<DataType*>(routing_scores.data_ptr()),
            .stride_t = static_cast<int>(routing_scores.stride(0)),
            .stride_e = static_cast<int>(routing_scores.stride(1)),
            .expert_index_start =
                expert_index_start.has_value() ? int(*expert_index_start) : 0,
            .expert_index_end = expert_index_end.has_value()
                ? int(*expert_index_end)
                : num_experts,
            .valid_token_count = reinterpret_cast<IndexType*>(
                valid_token_count.has_value() ? valid_token_count->data_ptr()
                                              : nullptr),
            .num_tokens = num_tokens,
            .num_tokens_per_cta = num_tokens_per_cta,
            // Buffer
            .buffered_expert_indices = reinterpret_cast<IndexType*>(
                buffered_expert_indices.data_ptr()),
            .buffered_token_indices =
                reinterpret_cast<IndexType*>(buffered_token_indices.data_ptr()),
            // Outputs
            .token_count_per_expert =
                reinterpret_cast<IndexType*>(token_count_per_expert.data_ptr()),
            .shuffled_expert_indices = reinterpret_cast<IndexType*>(
                shuffled_expert_indices.data_ptr()),
            .shuffled_token_indices = reinterpret_cast<IndexType*>(
                shuffled_token_indices.data_ptr())};
        const int num_warps = (num_experts > 128) ? 8 : 4;
        const int num_threads = kNumThreadsPerWarp * num_warps;
        dim3 grids(num_ctas);
        dim3 blocks(num_threads);
        void* args[] = {(void*)&params};
        auto stream = at::cuda::getCurrentCUDAStream();

#ifdef USE_ROCM
        // hipLaunchCooperativeKernel seems to cause incorrect memory order
        // across kernel launches.
        C10_CUDA_CHECK(hipLaunchKernel(
            (void*)kernel, grids, blocks, args, smem_size, stream));
#else
        if (smem_size >= 48 * 1024) {
          C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
              kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
        }
        C10_CUDA_CHECK(hipLaunchCooperativeKernel(
            (void*)kernel, grids, blocks, args, smem_size, stream));
#endif
      }));

  return std::make_tuple(
      token_count_per_expert, shuffled_expert_indices, shuffled_token_indices);
}

} // namespace fbgemm_gpu
