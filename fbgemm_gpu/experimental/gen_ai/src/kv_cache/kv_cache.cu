#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/DeviceGuard.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/Atomic.cuh>
#include <algorithm>
#include "c10/core/ScalarType.h"
#include "c10/util/BFloat16.h"
#include "kv_cache.h"

#ifndef USE_ROCM
#include <mma.h>
#endif
#include <hipcub/hipcub.hpp>

#include "fbgemm_gpu/utils/cuda_block_count.h"
#include "fbgemm_gpu/utils/vec_quant.cuh"

#include <torch/torch.h>

template <typename func_t>
void set_gpu_max_dynamic_shared_memory(
    func_t kernel,
    const int smem_bytes,
    const int device) {
  // V100: 96 KB; A100: 160 KB; H100: 228 KB.
  int max_shared_bytes = 0;
  C10_CUDA_CHECK(hipDeviceGetAttribute(
      &max_shared_bytes,
#ifndef __HIP_PLATFORM_AMD__
      hipDeviceAttributeSharedMemPerBlockOptin,
#else
      hipDeviceAttributeMaxSharedMemoryPerBlock,
#endif
      device));
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  TORCH_CHECK(
      smem_bytes <= max_shared_bytes,
      "Try to allocate ",
      smem_bytes / 1024,
      " KB of shared memory but only ",
      max_shared_bytes / 1024,
      " KB is available");

  C10_CUDA_CHECK(hipFuncSetAttribute(
      (void*)kernel, hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes));
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

namespace fbgemm_gpu {

template <int KVQuantNumGroups = 1>
__global__ void dequantize_int4_cache_kernel(
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_K, // [B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_V, // [B][MAX_T][N_KVH][D_H // G]
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> kv_seqlen,
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_K_dq, // [B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_V_dq // [B][MAX_T][N_KVH][D_H]
) {
  auto N_KVH = cache_K.size(2);
  auto D_H = cache_K_dq.size(3);

  auto b = blockIdx.x;
  // only need to dequantize this far.
  auto max_t = kv_seqlen[b];

  // one warp per T/H
  for (auto t_h = threadIdx.y + blockIdx.y * blockDim.y; t_h < max_t * N_KVH;
       t_h += blockDim.y * gridDim.y) {
    auto h = t_h % N_KVH;
    auto t = t_h / N_KVH;

    auto* row_k = &cache_K[b][t][h][0];
    auto* row_v = &cache_V[b][t][h][0];
    bfx8 kv_dq;
    if (KVQuantNumGroups == 1) {
      __half2 k_shift_scale;
      __half2 v_shift_scale;
      *reinterpret_cast<uint*>(&k_shift_scale) =
          *reinterpret_cast<uint*>(&row_k[0]);
      *reinterpret_cast<uint*>(&v_shift_scale) =
          *reinterpret_cast<uint*>(&row_v[0]);
      if (4 * threadIdx.x >= D_H) {
        continue;
      }
      uint32_t kq = *reinterpret_cast<uint16_t*>(&row_k[threadIdx.x * 2 + 4]);
      uint32_t vq = *reinterpret_cast<uint16_t*>(&row_v[threadIdx.x * 2 + 4]);

      uint32_t packed = kq | (vq << 16);
      kv_dq = dequantize_packed_int4(packed, k_shift_scale, v_shift_scale);

    } else {
      __half2 k_shift_scale;
      __half2 v_shift_scale;
      auto group_size = D_H / KVQuantNumGroups;
      auto group_idx = threadIdx.x * 4 / group_size;

      *reinterpret_cast<uint*>(&k_shift_scale) =
          *reinterpret_cast<uint*>(&row_k[4 * group_idx]);
      *reinterpret_cast<uint*>(&v_shift_scale) =
          *reinterpret_cast<uint*>(&row_v[4 * group_idx]);

      int32_t int4_qparam_offset = 4 * KVQuantNumGroups;

      if (4 * threadIdx.x >= D_H) {
        continue;
      }

      uint32_t kq = *reinterpret_cast<uint16_t*>(
          &row_k[threadIdx.x * 2 + int4_qparam_offset]);
      uint32_t vq = *reinterpret_cast<uint16_t*>(
          &row_v[threadIdx.x * 2 + int4_qparam_offset]);

      uint32_t packed = kq | (vq << 16);

      kv_dq = dequantize_packed_int4(packed, k_shift_scale, v_shift_scale);
    }
    // now, write our outputs
    auto* row_k_dq = &cache_K_dq[b][t][h][0];
    auto* row_v_dq = &cache_V_dq[b][t][h][0];

    *reinterpret_cast<uint2*>(&row_k_dq[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&kv_dq.vals[0]);
    *reinterpret_cast<uint2*>(&row_v_dq[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&kv_dq.vals[2]);
  }
}

#define CALL_DEQUANTIZE_INT4_CACHE_GROUPWISE_KERNEL(NUM_GROUPS, ...)          \
  dequantize_int4_cache_kernel<                                               \
      NUM_GROUPS><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(  \
      cache_K.packed_accessor64<uint8_t, 4, at::RestrictPtrTraits>(),         \
      cache_V.packed_accessor64<uint8_t, 4, at::RestrictPtrTraits>(),         \
      kv_seqlen.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),       \
      cache_K_dq.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(), \
      cache_V_dq.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>());

std::tuple<at::Tensor, at::Tensor> dequantize_int4_cache(
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor kv_seqlen,
    std::optional<int64_t> num_groups,
    std::optional<at::Tensor> qparam_k,
    std::optional<at::Tensor> qparam_v) {
  // allocate DQ outputs
  TORCH_CHECK(cache_K.is_cuda());
  TORCH_CHECK(cache_V.is_cuda());
  TORCH_CHECK(kv_seqlen.is_cuda());
  TORCH_CHECK(
      !qparam_k.has_value(),
      "CUDA doesn't support external qparams in dequantize_int4_cache");
  TORCH_CHECK(
      !qparam_v.has_value(),
      "CUDA doesn't support external qparams in dequantize_int4_cache");
  auto B = cache_K.size(0);
  auto MAX_T = cache_K.size(1);
  auto N_KVH = cache_K.size(2);
  auto D_HQ = cache_K.size(3);
  // D_HQ == D_H // 2 + 8 (int4 + 4xhalf qparams)
  auto num_groups_ = num_groups ? num_groups.value() : 1;
  auto int4_qparam_offset = 4 * num_groups_;
  auto D_H = (D_HQ - int4_qparam_offset) * 2;

  auto cache_K_dq =
      at::zeros({B, MAX_T, N_KVH, D_H}, cache_K.options().dtype(at::kBFloat16));
  auto cache_V_dq =
      at::zeros({B, MAX_T, N_KVH, D_H}, cache_K.options().dtype(at::kBFloat16));

  if (B == 0) {
    return {cache_K_dq, cache_V_dq};
  }

  constexpr int32_t kMaxBlocks = 256;
  dim3 blocks(B, std::max<int32_t>(1, kMaxBlocks / B));
  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);
  CALL_INT4_KERNEL_WITH_KV_GROUPWISE_QUANT_CHECK(
      CALL_DEQUANTIZE_INT4_CACHE_GROUPWISE_KERNEL, num_groups_)
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return {cache_K_dq, cache_V_dq};
}

template <typename T>
__device__ void get_dst_row(
    T** dst_row,
    at::PackedTensorAccessor64<T, 4, at::RestrictPtrTraits>&
        cache_KV, // [B][MAX_T][N_KVH][D_H +4 or D_H]
    int32_t b,
    int32_t h,
    int32_t cache_loc_t,
    int32_t page_size,
    int32_t* block_tables,
    int32_t block_tables_b_stride) {
  if (block_tables == nullptr) {
    *dst_row = &cache_KV[b][cache_loc_t][h][0];
  } else {
    int page_logical_idx = cache_loc_t / page_size;
    int page_offset = cache_loc_t % page_size;
    int page_physical_idx =
        block_tables[b * block_tables_b_stride + page_logical_idx];
    *dst_row = &cache_KV[0][page_physical_idx * page_size + page_offset][h][0];
  }
}

enum class PositionEmbeddingMode { ROPE = 0, XPOS = 1, NOPE = 2 };
enum class KVQuantRecipe { perTokenScaling = 0, perHeadScaling = 1 };
enum class QKV { Q, K, V };

template <typename T, KVQuantRecipe recipe = KVQuantRecipe::perTokenScaling>
DEVICE_INLINE void quantize_fp8_kv(
    fx4 dst,
    T* dst_row_q,
    __half2* qparam = nullptr,
    bool do_norm = false);

DEVICE_INLINE void per_row_norm(fx4& dst) {
  float sum = fx4_dot(dst, dst);
  // Warp reduce sum
  sum = warpReduceSum(sum);
  float rsqr = rsqrtf(sum / D_H);
  dst = fx4_scale(dst, rsqr);
}

DEVICE_INLINE void per_head_amax(fx4& dst, float* amax) {
  dst = fx4_abs(dst);
  auto thread_max = fmaxf(fmaxf(fmaxf(dst.x, dst.y), dst.z), dst.w);
  CUDA_KERNEL_ASSERT(thread_max >= 0)
  unsigned mask = ballot_sync(4 * threadIdx.x < D_H, 0xFFFFFFFF);
  float warp_max = warpReduceMax(thread_max, mask);
  // Use atomic operation to update the global maximum
  if (threadIdx.x == 0) {
    // CUDA_KERNEL_ASSERT(warp_max >= 0);
    atomicMax(reinterpret_cast<int*>(amax), __float_as_int(warp_max));
  }
}
DEVICE_INLINE void per_row_amax(fx4& dst, float* amax) {
  dst = fx4_abs(dst);
  auto thread_max = fmaxf(fmaxf(fmaxf(dst.x, dst.y), dst.z), dst.w);
  unsigned mask = ballot_sync(4 * threadIdx.x < D_H, 0xFFFFFFFF);
  float warp_max = warpReduceMax(thread_max, mask);
  // write amax
  if (threadIdx.x == 0) {
    CUDA_KERNEL_ASSERT(uintptr_t(amax) % 4 == 0);
    *amax = warp_max;
  }
}
__global__ void nope_qkv_varseq_prefill_kernel(
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XQ, // [B_T][N_H][D_H]
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XK, // [B_T][N_KVH][D_H]
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XV, // [B_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_K, // [B][MAX_T][N_KVH][D_H] or
                 // [1][MAX_PAGES * PAGE_SIZE][N_KVH][D_H] for paged attention
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_V, // [B][MAX_T][N_KVH][D_H] or
                 // [1][MAX_PAGES * PAGE_SIZE][N_KVH][D_H] for paged attention
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XQ_O, // [B_T][N_H][D]
    int32_t* varseq_batch, // in decoding case we have T == 1 and so just pass
                           // nullptr
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> varseq_seqpos,
    int32_t* block_tables, // [B][MAX_PAGES], maps logical pages to physical
                           // ones for paged attention
    int32_t page_size,
    int32_t block_tables_b_stride,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        varseq_cache_seqpos,
    int64_t* actual_batch_size =
        nullptr, // When running in CUDA graph mode, the actual batch size
                 // can be smaller than block_tables.size(0). In this case
                 // rows of block_tables beyond actual_batch_size are not
                 // initialized, and using them wil cause undefined
                 // behavior. To prevent this, when actual_batch_size is
                 // provided, the kernel exits if the current batch index is
                 // larger of equal to actual_batch_size,
    bool update_kv = true) {
  // Launch b_t_(sum(h)) warps.
  auto b_t_hh = blockIdx.x * blockDim.y + threadIdx.y;
  auto B_T = XQ.size(0);
  int N_KVH = 0;
  if (update_kv) {
    N_KVH = XK.size(1);
  }
  auto N_H = XQ.size(1);
  auto D_H = XQ.size(2);
  auto HH = 2 * N_KVH + N_H;

  auto hh = b_t_hh % HH;
  auto b_t = b_t_hh / HH;
  if (b_t >= B_T) {
    return;
  }
  auto seqpos_t = varseq_seqpos[b_t];
  if (seqpos_t == -1) {
    return;
  }
  auto cache_loc_t = varseq_cache_seqpos[b_t];
  auto b = varseq_batch ? varseq_batch[b_t] : b_t;

  if (actual_batch_size != nullptr && b_t >= *actual_batch_size) {
    return;
  }

  at::BFloat16* src_row;
  at::BFloat16* dst_row;
  auto h = 0;
  QKV qkv;
  if (hh < N_H) {
    h = hh;
    src_row = &XQ[b_t][h][0];
    dst_row = &XQ_O[b_t][h][0];
    qkv = QKV::Q;
  } else if (hh < N_H + N_KVH) {
    h = hh - N_H;
    src_row = &XK[b_t][h][0];
    qkv = QKV::K;
    get_dst_row(
        &dst_row,
        cache_K,
        b,
        h,
        cache_loc_t,
        page_size,
        block_tables,
        block_tables_b_stride);
  } else {
    h = hh - N_H - N_KVH;
    src_row = &XV[b_t][h][0];
    qkv = QKV::V;
    get_dst_row(
        &dst_row,
        cache_V,
        b,
        h,
        cache_loc_t,
        page_size,
        block_tables,
        block_tables_b_stride);
  }

  for (auto head_id = 4 * threadIdx.x; head_id < D_H;
       head_id += kThreadsPerWarp * 4) {
    // assert D_H % 4 == 0;
    // load 4 elements per thread in a warp.
    if (head_id >= D_H) {
      return;
    }
    bfx4 src;
    *reinterpret_cast<uint2*>(&src) =
        *reinterpret_cast<uint2*>(&src_row[head_id]);

    if (qkv == QKV::Q || update_kv) {
      *reinterpret_cast<uint2*>(&dst_row[head_id]) =
          *reinterpret_cast<uint2*>(&src);
    }
  }
}

template <PositionEmbeddingMode Mode>
__global__ void rope_xpos_qkv_varseq_prefill_kernel(
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XQ, // [B_T][N_H][D_H]
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XK, // [B_T][N_KVH][D_H]
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XV, // [B_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_K, // [B][MAX_T][N_KVH][D_H] or
                 // [1][MAX_PAGES * PAGE_SIZE][N_KVH][D_H] for paged attention
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_V, // [B][MAX_T][N_KVH][D_H] or
                 // [1][MAX_PAGES * PAGE_SIZE][N_KVH][D_H] for paged attention
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XQ_O, // [B_T][N_H][D]
    int32_t* varseq_batch, // in decoding case we have T == 1 and so just pass
                           // nullptr
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> varseq_seqpos,
    double theta,
    double gamma,
    double scale_base,
    double exponent_offset,
    int32_t* block_tables, // [B][MAX_PAGES], maps logical pages to physical
                           // ones for paged attention
    int32_t page_size,
    int32_t block_tables_b_stride,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        varseq_cache_seqpos,
    int64_t* actual_batch_size =
        nullptr, // When running in CUDA graph mode, the actual batch size
                 // can be smaller than block_tables.size(0). In this case
                 // rows of block_tables beyond actual_batch_size are not
                 // initialized, and using them wil cause undefined
                 // behavior. To prevent this, when actual_batch_size is
                 // provided, the kernel exits if the current batch index is
                 // larger of equal to actual_batch_size,
    bool rope_scaling = false,
    int64_t old_context_len = 8192,
    double scaling_factor = 16,
    double lo_freq_factor = 1,
    double hi_freq_factor = 32,
    bool write_k_back = false,
    bool update_kv = true) {
  // Launch b_t_(sum(h)) warps.
  auto b_t_hh = blockIdx.x * blockDim.y + threadIdx.y;
  auto B_T = XQ.size(0);
  int N_KVH = 0;
  if (update_kv) {
    N_KVH = XK.size(1);
  } else {
    assert(!write_k_back);
  }
  auto N_H = XQ.size(1);
  auto D_H = XQ.size(2);
  auto HH = 2 * N_KVH + N_H;

  auto hh = b_t_hh % HH;
  auto b_t = b_t_hh / HH;
  if (b_t >= B_T) {
    return;
  }
  auto seqpos_t = varseq_seqpos[b_t];
  if (seqpos_t == -1) {
    return;
  }
  auto cache_loc_t = varseq_cache_seqpos[b_t];
  auto b = varseq_batch ? varseq_batch[b_t] : b_t;

  if (actual_batch_size != nullptr && b_t >= *actual_batch_size) {
    return;
  }

  at::BFloat16* src_row;
  at::BFloat16* dst_row;
  auto h = 0;
  QKV qkv;
  if (hh < N_H) {
    h = hh;
    src_row = &XQ[b_t][h][0];
    dst_row = &XQ_O[b_t][h][0];
    qkv = QKV::Q;
  } else if (hh < N_H + N_KVH) {
    h = hh - N_H;
    src_row = &XK[b_t][h][0];

    get_dst_row(
        &dst_row,
        cache_K,
        b,
        h,
        cache_loc_t,
        page_size,
        block_tables,
        block_tables_b_stride);
    qkv = QKV::K;
  } else {
    h = hh - N_H - N_KVH;
    src_row = &XV[b_t][h][0];
    get_dst_row(
        &dst_row,
        cache_V,
        b,
        h,
        cache_loc_t,
        page_size,
        block_tables,
        block_tables_b_stride);
    qkv = QKV::V;
  }

  for (auto head_id = 4 * threadIdx.x; head_id < D_H;
       head_id += kThreadsPerWarp * 4) {
    // assert D_H % 4 == 0;
    // load 4 elements per thread in a warp.
    if (head_id >= D_H) {
      return;
    }

    bfx4 src;
    *reinterpret_cast<uint2*>(&src) =
        *reinterpret_cast<uint2*>(&src_row[head_id]);
    if (qkv == QKV::V) {
      if (update_kv) {
        *reinterpret_cast<uint2*>(&dst_row[head_id]) =
            *reinterpret_cast<uint2*>(&src);
      }
    } else { // qk requires rope
      if (update_kv || qkv == QKV::Q) {
        int32_t offset_0 = ((head_id) / 2 + 0);
        int32_t offset_1 = ((head_id) / 2 + 1);

        double powers_0 = offset_0 * 2;
        double powers_1 = offset_1 * 2;

        double freqs_0 = pow(theta, powers_0 / -static_cast<double>(D_H));
        double freqs_1 = pow(theta, powers_1 / -static_cast<double>(D_H));
        if (rope_scaling) {
          double lo_freq_wavelen = old_context_len / lo_freq_factor;
          double hi_freq_wavelen = old_context_len / hi_freq_factor;
          double wavelen_0 = 2 * M_PI / freqs_0;
          if (wavelen_0 >= hi_freq_wavelen && wavelen_0 > lo_freq_wavelen) {
            freqs_0 = freqs_0 / scaling_factor;
          } else if (wavelen_0 >= hi_freq_wavelen) {
            double smooth = (old_context_len / wavelen_0 - lo_freq_factor) /
                (hi_freq_factor - lo_freq_factor);
            freqs_0 =
                (1 - smooth) * freqs_0 / scaling_factor + smooth * freqs_0;
          }
          double wavelen_1 = 2 * M_PI / freqs_1;
          if (wavelen_1 >= hi_freq_wavelen && wavelen_1 > lo_freq_wavelen) {
            freqs_1 = freqs_1 / scaling_factor;
          } else if (wavelen_1 >= hi_freq_wavelen) {
            double smooth = (old_context_len / wavelen_1 - lo_freq_factor) /
                (hi_freq_factor - lo_freq_factor);
            freqs_1 =
                (1 - smooth) * freqs_1 / scaling_factor + smooth * freqs_1;
          }
        }
        freqs_0 = static_cast<double>(seqpos_t) * freqs_0;
        freqs_1 = static_cast<double>(seqpos_t) * freqs_1;

        double sin_0, sin_1, cos_0, cos_1;
        sincos(freqs_0, &sin_0, &cos_0);
        sincos(freqs_1, &sin_1, &cos_1);

        auto src_0 = bf1622float2(src.vals[0]);
        auto src_1 = bf1622float2(src.vals[1]);

        double dst_x, dst_y, dst_z, dst_w;

        dst_x = static_cast<double>(src_0.x) * cos_0 -
            static_cast<double>(src_0.y) * sin_0;
        dst_y = static_cast<double>(src_0.y) * cos_0 +
            static_cast<double>(src_0.x) * sin_0;

        dst_z = static_cast<double>(src_1.x) * cos_1 -
            static_cast<double>(src_1.y) * sin_1;
        dst_w = static_cast<double>(src_1.y) * cos_1 +
            static_cast<double>(src_1.x) * sin_1;

        if (Mode == PositionEmbeddingMode::XPOS) {
          double gamma_0 = (powers_0 + gamma * D_H) / (D_H + gamma * D_H);
          double gamma_1 = (powers_1 + gamma * D_H) / (D_H + gamma * D_H);
          double scale_base_ = (qkv == QKV::Q) ? scale_base : -scale_base;
          double factor_0 = pow(
              gamma_0,
              (static_cast<double>(seqpos_t) - exponent_offset) / scale_base_);
          double factor_1 = pow(
              gamma_1,
              (static_cast<double>(seqpos_t) - exponent_offset) / scale_base_);

          dst_x *= factor_0;
          dst_y *= factor_0;
          dst_z *= factor_1;
          dst_w *= factor_1;
        }

        fx4 dst;
        dst.x = __double2float_rn(dst_x);
        dst.y = __double2float_rn(dst_y);
        dst.z = __double2float_rn(dst_z);
        dst.w = __double2float_rn(dst_w);

        bfx4 dst_;
        dst_.vals[0] = __floats2bfloat162_rn(dst.x, dst.y);
        dst_.vals[1] = __floats2bfloat162_rn(dst.z, dst.w);
        if (update_kv || qkv == QKV::Q) {
          *reinterpret_cast<uint2*>(&dst_row[head_id]) =
              *reinterpret_cast<uint2*>(&dst_);
        }

        if (write_k_back && qkv == QKV::K) {
          // Also write back to the source row
          *reinterpret_cast<uint2*>(&src_row[head_id]) =
              *reinterpret_cast<uint2*>(&dst_);
        }
      }
    }
  }
}

template <PositionEmbeddingMode EmbMode>
DEVICE_INLINE fx4 rope_xpos(
    bfx4 src,
    int32_t seqpos_t,
    QKV head,
    double theta,
    double gamma,
    double scale_base,
    double exponent_offset,
    bool rope_scaling = false,
    int64_t old_context_len = 8192,
    double scaling_factor = 16,
    double lo_freq_factor = 1,
    double hi_freq_factor = 32) {
  fx4 dst; // read 4 bf16 from src and store in 4 float registers
  if (head == QKV::V || EmbMode == PositionEmbeddingMode::NOPE) {
    return bfx4_to_fx4(src);
  }
  int32_t offset_0 = ((4 * threadIdx.x) / 2 + 0);
  int32_t offset_1 = ((4 * threadIdx.x) / 2 + 1);

  double powers_0 = offset_0 * 2;
  double powers_1 = offset_1 * 2;

  double freqs_0 = pow(theta, powers_0 / -static_cast<double>(D_H));
  double freqs_1 = pow(theta, powers_1 / -static_cast<double>(D_H));

  if (rope_scaling) {
    // From https://github.com/fairinternal/llm_inference/pull/391
    // See https://arxiv.org/pdf/2309.16039 , https://fburl.com/eyhqrzhn
    double lo_freq_wavelen = old_context_len / lo_freq_factor;
    double hi_freq_wavelen = old_context_len / hi_freq_factor;
    double wavelen_0 = 2 * M_PI / freqs_0;
    if (wavelen_0 >= hi_freq_wavelen && wavelen_0 > lo_freq_wavelen) {
      freqs_0 = freqs_0 / scaling_factor;
    } else if (wavelen_0 >= hi_freq_wavelen) {
      double smooth = (old_context_len / wavelen_0 - lo_freq_factor) /
          (hi_freq_factor - lo_freq_factor);
      freqs_0 = (1 - smooth) * freqs_0 / scaling_factor + smooth * freqs_0;
    }
    double wavelen_1 = 2 * M_PI / freqs_1;
    if (wavelen_1 >= hi_freq_wavelen && wavelen_1 > lo_freq_wavelen) {
      freqs_1 = freqs_1 / scaling_factor;
    } else if (wavelen_1 >= hi_freq_wavelen) {
      double smooth = (old_context_len / wavelen_1 - lo_freq_factor) /
          (hi_freq_factor - lo_freq_factor);
      freqs_1 = (1 - smooth) * freqs_1 / scaling_factor + smooth * freqs_1;
    }
  }
  freqs_0 = static_cast<double>(seqpos_t) * freqs_0;
  freqs_1 = static_cast<double>(seqpos_t) * freqs_1;

  double sin_0, sin_1, cos_0, cos_1;
  sincos(freqs_0, &sin_0, &cos_0);
  sincos(freqs_1, &sin_1, &cos_1);

  auto src_0 = bf1622float2(src.vals[0]);
  auto src_1 = bf1622float2(src.vals[1]);

  double dst_x, dst_y, dst_z, dst_w;

  dst_x = static_cast<double>(src_0.x) * cos_0 -
      static_cast<double>(src_0.y) * sin_0;
  dst_y = static_cast<double>(src_0.y) * cos_0 +
      static_cast<double>(src_0.x) * sin_0;

  dst_z = static_cast<double>(src_1.x) * cos_1 -
      static_cast<double>(src_1.y) * sin_1;
  dst_w = static_cast<double>(src_1.y) * cos_1 +
      static_cast<double>(src_1.x) * sin_1;

  if (EmbMode == PositionEmbeddingMode::XPOS) {
    double gamma_0 = (powers_0 + gamma * D_H) / (D_H + gamma * D_H);
    double gamma_1 = (powers_1 + gamma * D_H) / (D_H + gamma * D_H);
    double scale_base_ = (head == QKV::Q) ? scale_base : -scale_base;
    double factor_0 =
        pow(gamma_0,
            (static_cast<double>(seqpos_t) - exponent_offset) / scale_base_);
    double factor_1 =
        pow(gamma_1,
            (static_cast<double>(seqpos_t) - exponent_offset) / scale_base_);
    dst_x *= factor_0;
    dst_y *= factor_0;
    dst_z *= factor_1;
    dst_w *= factor_1;
  }

  dst.x = __double2float_rn(dst_x);
  dst.y = __double2float_rn(dst_y);
  dst.z = __double2float_rn(dst_z);
  dst.w = __double2float_rn(dst_w);

  return dst;
}

template <int KVQuantNumGroups = 1>
DEVICE_INLINE void
quantize_int4_kv(fx4 dst, uint8_t* dst_row_q, bool do_norm = false) {
  if (do_norm) {
    per_row_norm(dst);
  }
  auto thread_min = fminf(fminf(fminf(dst.x, dst.y), dst.z), dst.w);
  auto thread_max = fmaxf(fmaxf(fmaxf(dst.x, dst.y), dst.z), dst.w);

  float warp_min, warp_max;

  int32_t int4_qparam_offset = 4;
  if (KVQuantNumGroups == 1) {
    unsigned mask = ballot_sync(4 * threadIdx.x < D_H, 0xFFFFFFFF);
    warp_min = -warpReduceMax(-thread_min, mask);
    warp_max = warpReduceMax(thread_max, mask);
  } else {
    auto group_size = D_H / KVQuantNumGroups;
    auto group_idx = threadIdx.x * 4 / group_size;
    int4_qparam_offset = 4 * KVQuantNumGroups;
    unsigned masks[KVQuantNumGroups];
    for (int i = 0; i < KVQuantNumGroups; ++i) {
      masks[i] = ballot_sync(group_idx == i, 0xFFFFFFFF);
    }
    warp_min = -warpReduceMax(-thread_min, masks[group_idx]);
    warp_max = warpReduceMax(thread_max, masks[group_idx]);
  }

  auto scale = (warp_max - warp_min) / 15.0f;
  auto inv_scale = 15.0 / (scale * 15.0 + 1.0e-8);
  auto shift = warp_min;

  auto x_0 = __float2int_rn((dst.x - shift) * inv_scale) & 0xF;
  auto x_1 = __float2int_rn((dst.y - shift) * inv_scale) & 0xF;
  auto x_2 = __float2int_rn((dst.z - shift) * inv_scale) & 0xF;
  auto x_3 = __float2int_rn((dst.w - shift) * inv_scale) & 0xF;

  uint16_t packed = 0;

  packed |= (x_0 << 0);
  packed |= (x_1 << 4);
  packed |= (x_2 << 8);
  packed |= (x_3 << 12);

  // each threadIdx.x writes 2 bytes with 4+4 byte offset for scale/shift

  CUDA_KERNEL_ASSERT(
      uintptr_t(&dst_row_q[2 * threadIdx.x + int4_qparam_offset]) % 2 == 0);

  *reinterpret_cast<uint16_t*>(
      &dst_row_q[2 * threadIdx.x + int4_qparam_offset]) = packed;
  if (threadIdx.x == 0) {
    CUDA_KERNEL_ASSERT(uintptr_t(&dst_row_q[0]) % 4 == 0);
    __half2 qparams = __floats2half2_rn(scale, shift);
    *reinterpret_cast<__half2*>(&dst_row_q[0]) = qparams;
  }
  if (KVQuantNumGroups > 1) {
    int32_t group_size = D_H / KVQuantNumGroups;
    if (threadIdx.x > 0 && threadIdx.x * 4 % group_size == 0) {
      auto group_idx = threadIdx.x * 4 / group_size;
      int32_t qparam_offset = 4 * group_idx;
      CUDA_KERNEL_ASSERT(uintptr_t(&dst_row_q[qparam_offset]) % 4 == 0);
      __half2 qparams = __floats2half2_rn(scale, shift);
      *reinterpret_cast<__half2*>(&dst_row_q[qparam_offset]) = qparams;
    }
  }
}

#define CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL(                  \
    NUM_GROUPS,                                                              \
    DTYPE,                                                                   \
    EMB_MODE,                                                                \
    VARSEQ_BATCH,                                                            \
    VARSEQ_SEQPOS,                                                           \
    THETA,                                                                   \
    GAMMA,                                                                   \
    SCALE_BASE,                                                              \
    EXPO_OFFSET,                                                             \
    block_tables,                                                            \
    page_size,                                                               \
    block_tables_b_stride,                                                   \
    varseq_cache_seqpos,                                                     \
    actual_batch_size,                                                       \
    rope_scaling,                                                            \
    old_context_len,                                                         \
    scaling_factor,                                                          \
    lo_freq_factor,                                                          \
    hi_freq_factor,                                                          \
    write_k_back,                                                            \
    k_norm)                                                                  \
  rope_xpos_qkv_varseq_prefill_kernel_quantized<EMB_MODE, DTYPE, NUM_GROUPS> \
      <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(            \
          XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),    \
          XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),    \
          XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),    \
          cache_K.packed_accessor64<uint8_t, 4, at::RestrictPtrTraits>(),    \
          cache_V.packed_accessor64<uint8_t, 4, at::RestrictPtrTraits>(),    \
          qparam_k_ptr,                                                      \
          qparam_v_ptr,                                                      \
          XQ_O.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),  \
          VARSEQ_BATCH,                                                      \
          VARSEQ_SEQPOS,                                                     \
          THETA,                                                             \
          GAMMA,                                                             \
          SCALE_BASE,                                                        \
          EXPO_OFFSET,                                                       \
          block_tables,                                                      \
          page_size,                                                         \
          block_tables_b_stride,                                             \
          varseq_cache_seqpos,                                               \
          actual_batch_size,                                                 \
          rope_scaling,                                                      \
          old_context_len,                                                   \
          scaling_factor,                                                    \
          lo_freq_factor,                                                    \
          hi_freq_factor,                                                    \
          write_k_back,                                                      \
          k_norm);

#if (defined(USE_ROCM) && ROCM_VERSION >= 60200) || \
    (defined(CUDA_VERSION) && CUDA_VERSION >= 12000)
class FP8_E4M3_MAX {
 public:
#ifndef USE_ROCM
  static constexpr float value = 448.0;
#else
  static constexpr float value = 240.0;
#endif
};
class FP8_E5M2_MAX {
 public:
  static constexpr float value = 57344.0;
};
#endif

template <
    PositionEmbeddingMode EmbMode,
    CacheLogicalDtype kCacheDtype,
    int KVQuantNumGroups = 1>
__global__ void rope_xpos_qkv_varseq_prefill_kernel_quantized(
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XQ, // [B_T][N_H][D_H]
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XK, // [B_T][N_KVH][D_H]
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XV, // [B_T][N_KVH][D_H]
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_K, // [B][MAX_T][N_KVH][D_H] or
                 // [1][MAX_PAGES * PAGE_SIZE][N_KVH][D_H] for paged attention
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_V, // [B][MAX_T][N_KVH][D_H] or
                 // [1][MAX_PAGES * PAGE_SIZE][N_KVH][D_H] for paged attention
    int32_t* qparam_k_ptr,
    int32_t* qparam_v_ptr,
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XQ_O, // [B_T][N_H][D]
    int32_t* varseq_batch, // in decoding case we have T == 1 and so just
                           // pass nullptr
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> varseq_seqpos,
    double theta,
    double gamma,
    double scale_base,
    double exponent_offset,
    int32_t* block_tables, // [B][MAX_PAGES], maps logical pages to physical
                           // ones for paged attention
    int32_t page_size,
    int32_t block_tables_b_stride,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        varseq_cache_seqpos,
    int64_t* actual_batch_size =
        nullptr, // When running in CUDA graph mode, the actual batch size
                 // can be smaller than block_tables.size(0). In this case
                 // rows of block_tables beyond actual_batch_size are not
                 // initialized, and using them wil cause undefined
                 // behavior. To prevent this, when actual_batch_size is
                 // provided, the kernel exits if the current batch index is
                 // larger of equal to actual_batch_size,
    bool rope_scaling = false,
    int64_t old_context_len = 8192,
    double scaling_factor = 16,
    double lo_freq_factor = 1,
    double hi_freq_factor = 32,
    bool write_k_back = false,
    bool k_norm = false) {
  // Launch b_t_(sum(h)) warps.
  auto b_t_hh = blockIdx.x * blockDim.y +
      threadIdx.y; // Block = [kThreadsPerWarp, kWarpsPerBlock]
  // Each warp handles a single head XQ or XK or XV of a single token..
  // That would be 1 x 128 distributed among 32 threads in the warp.
  // Each thread should handle 4 elements.
  auto B_T = XQ.size(0);
  auto N_KVH = XK.size(1);
  auto N_H = XQ.size(1);
  auto D_H = XQ.size(2);

  auto HH = 2 * N_KVH + N_H;

  auto hh = b_t_hh % HH;
  auto b_t = b_t_hh / HH;
  if (b_t >= B_T) {
    return;
  }
  auto seqpos_t = varseq_seqpos[b_t];
  if (seqpos_t == -1) {
    return;
  }
  auto cache_loc_t = varseq_cache_seqpos[b_t];
  auto b = varseq_batch ? varseq_batch[b_t] : b_t;

  if (actual_batch_size != nullptr && b_t >= *actual_batch_size) {
    return;
  }

  at::BFloat16* src_row = nullptr;
  at::BFloat16* dst_row = nullptr;
  uint8_t* dst_row_q = nullptr;
  auto h = 0;
  QKV qkv;
  if (hh < N_H) {
    h = hh;
    src_row = &XQ[b_t][h][0];
    dst_row = &XQ_O[b_t][h][0];
    qkv = QKV::Q;
  } else if (hh < N_H + N_KVH) {
    h = hh - N_H;
    src_row = &XK[b_t][h][0];
    get_dst_row(
        &dst_row_q,
        cache_K,
        b,
        h,
        cache_loc_t,
        page_size,
        block_tables,
        block_tables_b_stride);
    qkv = QKV::K;
  } else {
    h = hh - N_H - N_KVH;
    src_row = &XV[b_t][h][0];
    get_dst_row(
        &dst_row_q,
        cache_V,
        b,
        h,
        cache_loc_t,
        page_size,
        block_tables,
        block_tables_b_stride);
    qkv = QKV::V;
  }

  // load 4 elements per thread in a warp.

  // Each thread should handle D_H//32 = 4 elements.
  CUDA_KERNEL_ASSERT(D_H <= 4 * kThreadsPerWarp);
  if (4 * threadIdx.x >= D_H) {
    return;
  }
  bfx4 src;
  *reinterpret_cast<uint2*>(&src) =
      *reinterpret_cast<uint2*>(&src_row[4 * threadIdx.x]);

  fx4 dst = rope_xpos<EmbMode>(
      src,
      seqpos_t,
      qkv,
      theta,
      gamma,
      scale_base,
      exponent_offset,
      rope_scaling,
      old_context_len,
      scaling_factor,
      lo_freq_factor,
      hi_freq_factor);
  if (k_norm && qkv != QKV::V) {
    // fusing K/Q norm into the kernel
    per_row_norm(dst);
  }

  if (qkv == QKV::K && write_k_back) {
    // Also write back to the source row
    bfx4 dst_bf16 = fx4_to_bfx4(dst);
    *reinterpret_cast<uint2*>(&src_row[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&dst_bf16);
  }
  if (qkv == QKV::Q) {
    // write to dst_row
    bfx4 dst_bf16 = fx4_to_bfx4(dst);
    CUDA_KERNEL_ASSERT(uintptr_t(&dst_row[4 * threadIdx.x]) % 8 == 0);
    *reinterpret_cast<uint2*>(&dst_row[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&dst_bf16);
  } else {
    // quantize and write to dst_row
    auto D_H = XQ.size(2);
    auto D_H_q = cache_K.size(3);
    __half2* qparam_row = nullptr;
    if (kCacheDtype == CacheLogicalDtype::FP8) {
      if (qparam_k_ptr == nullptr) {
        CUDA_KERNEL_ASSERT(D_H_q - D_H == 4);
      } else {
        auto T = cache_K.size(1);
        size_t idx = 0;
        if (block_tables == nullptr) {
          idx = b * (T * N_KVH) + (size_t)cache_loc_t * N_KVH + h;
        } else {
          // This is duplicate computation with get_dst_row above.
          // TODO: Maybe clean up and merge later.
          int page_logical_idx = cache_loc_t / page_size;
          int page_offset = cache_loc_t % page_size;
          int page_physical_idx =
              block_tables[b * block_tables_b_stride + page_logical_idx];
          int physical_t = page_physical_idx * page_size + page_offset;
          idx = physical_t * N_KVH + h;
        }
        if (qkv == QKV::K) {
          qparam_row = reinterpret_cast<__half2*>(&qparam_k_ptr[idx]);
        } else {
          qparam_row = reinterpret_cast<__half2*>(&qparam_v_ptr[idx]);
        }
      }
      quantize_fp8_kv(dst, dst_row_q, qparam_row);
    } else if (kCacheDtype == CacheLogicalDtype::INT4) {
      CUDA_KERNEL_ASSERT(D_H_q - D_H / 2 == 4 * KVQuantNumGroups);
      quantize_int4_kv<KVQuantNumGroups>(dst, dst_row_q);
    }
  }
}

template <
    PositionEmbeddingMode EmbMode,
    CacheLogicalDtype kCacheDtype,
    int KVQuantNumGroups = 1>
__global__ void rope_xpos_qkv_varseq_prefill_kernel_fp8(
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XQ, // [B_T][N_H][D_H]
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XK, // [B_T][N_KVH][D_H]
    at::PackedTensorAccessor32<at::BFloat16, 3, at::RestrictPtrTraits>
        XV, // [B_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::Float8_e4m3fn, 4, at::RestrictPtrTraits>
        cache_K, // [B][MAX_T][N_KVH][D_H +4]
    at::PackedTensorAccessor64<at::Float8_e4m3fn, 4, at::RestrictPtrTraits>
        cache_V, // [B][MAX_T][N_KVH][D_H + 4]
    int32_t* qparam_k_ptr,
    int32_t* qparam_v_ptr,
    int32_t* varseq_batch, // in decoding case we have T == 1 and so just
                           // pass nullptr
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> varseq_seqpos,
    double theta,
    double gamma,
    double scale_base,
    double exponent_offset,
    int32_t* block_tables, // [B][MAX_PAGES], maps logical pages to physical
                           // ones for paged attention
    int32_t page_size,
    int32_t block_tables_b_stride,
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        varseq_cache_seqpos,
    int64_t* actual_batch_size =
        nullptr, // When running in CUDA graph mode, the actual batch size
                 // can be smaller than block_tables.size(0). In this case
                 // rows of block_tables beyond actual_batch_size are not
                 // initialized, and using them wil cause undefined
                 // behavior. To prevent this, when actual_batch_size is
                 // provided, the kernel exits if the current batch index is
                 // larger of equal to actual_batch_size,
    bool rope_scaling = false,
    int64_t old_context_len = 8192,
    double scaling_factor = 16,
    double lo_freq_factor = 1,
    double hi_freq_factor = 32,
    bool write_k_back = false,
    bool k_norm = false,
    float* amax = nullptr,
    bool* is_precalculated_qparam = nullptr) {
  // Launch b_t_(sum(h)) warps.
  auto b_t_hh = blockIdx.x * blockDim.y +
      threadIdx.y; // Block = [kThreadsPerWarp, kWarpsPerBlock]
  // Each warp handles a single head XQ or XK or XV of a single token..
  // That would be 1 x 128 distributed among 32 threads in the warp.
  // Each thread should handle 4 elements.
  auto B_T = XQ.size(0);
  auto N_KVH = XK.size(1);
  auto N_H = XQ.size(1);
  auto D_H = XQ.size(2);

  auto HH = 2 * N_KVH + N_H;

  auto hh = b_t_hh % HH;
  auto b_t = b_t_hh / HH;
  if (b_t >= B_T) {
    return;
  }
  auto seqpos_t = varseq_seqpos[b_t];
  if (seqpos_t == -1) {
    return;
  }
  auto cache_loc_t = varseq_cache_seqpos[b_t];
  auto b = varseq_batch ? varseq_batch[b_t] : b_t;

  if (actual_batch_size != nullptr && b_t >= *actual_batch_size) {
    return;
  }

  at::BFloat16* src_row = nullptr;
  // dst_row_q dtype is fp8 not uint8_t
  at::Float8_e4m3fn* dst_row_q = nullptr;
  auto h = 0;
  QKV qkv;
  if (hh < N_H) {
    h = hh;
    src_row = &XQ[b_t][h][0];
    qkv = QKV::Q;
  } else if (hh < N_H + N_KVH) {
    h = hh - N_H;
    src_row = &XK[b_t][h][0];
    get_dst_row(
        &dst_row_q,
        cache_K,
        b,
        h,
        cache_loc_t,
        page_size,
        block_tables,
        block_tables_b_stride);
    qkv = QKV::K;
  } else {
    h = hh - N_H - N_KVH;
    src_row = &XV[b_t][h][0];
    get_dst_row(
        &dst_row_q,
        cache_V,
        b,
        h,
        cache_loc_t,
        page_size,
        block_tables,
        block_tables_b_stride);
    qkv = QKV::V;
  }

  // load 4 elements per thread in a warp.

  // Each thread should handle D_H//32 = 4 elements.
  CUDA_KERNEL_ASSERT(D_H <= 4 * kThreadsPerWarp);
  if (4 * threadIdx.x >= D_H) {
    return;
  }
  bfx4 src;
  *reinterpret_cast<uint2*>(&src) =
      *reinterpret_cast<uint2*>(&src_row[4 * threadIdx.x]);

  fx4 dst = rope_xpos<EmbMode>(
      src,
      seqpos_t,
      qkv,
      theta,
      gamma,
      scale_base,
      exponent_offset,
      rope_scaling,
      old_context_len,
      scaling_factor,
      lo_freq_factor,
      hi_freq_factor);
  if (k_norm && qkv != QKV::V) {
    // fusing K/Q norm into the kernel
    per_row_norm(dst);
  }
  // Write back to the source row for Q and K
  if (qkv != QKV::V && write_k_back) {
    // Also write back to the source row
    bfx4 dst_bf16 = fx4_to_bfx4(dst);
    *reinterpret_cast<uint2*>(&src_row[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&dst_bf16);
  }
  // This kernel does not write to xq_o
  bool is_precalculated_qparam_b_t = is_precalculated_qparam
      ? is_precalculated_qparam[b_t]
      : true; // for decode it is true
  if (qkv != QKV::Q && is_precalculated_qparam_b_t) {
    // only write to cache if batch lane has a pre-calculated qparam
    // quantize and write to dst_row
    CUDA_KERNEL_ASSERT(qparam_k_ptr != nullptr)
    // TODO: Change qparam ptr to float*
    __half2* qparam_row = nullptr;

    size_t idx = b * N_KVH + h;
    if (qkv == QKV::K) {
      qparam_row = reinterpret_cast<__half2*>(&qparam_k_ptr[idx]);
    } else {
      qparam_row = reinterpret_cast<__half2*>(&qparam_v_ptr[idx]);
    }
    quantize_fp8_kv<at::Float8_e4m3fn, KVQuantRecipe::perHeadScaling>(
        dst, dst_row_q, qparam_row);
  } else {
    // qkv == Q or qparam is not precalculated
    CUDA_KERNEL_ASSERT(amax != nullptr);
    // per_row_amax(dst, &amax[b_t * HH + hh]);
    per_head_amax(dst, &amax[b * HH + hh]);
  }
}

at::Tensor nope_qkv_varseq_prefill(
    at::Tensor XQ,
    std::optional<at::Tensor> XK_,
    std::optional<at::Tensor> XV_,
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor varseq_batch,
    at::Tensor varseq_seqpos,
    std::optional<at::Tensor> block_tables,
    int64_t page_size,
    std::optional<at::Tensor> varseq_cache_seqpos,
    int64_t cache_logical_dtype_int,
    std::optional<int64_t> num_groups,
    std::optional<at::Tensor> qparam_k = std::nullopt,
    std::optional<at::Tensor> qparam_v = std::nullopt,
    bool k_norm = false,
    bool update_kv = true,
    std::optional<at::Tensor> amax_qkv = std::nullopt,
    std::optional<at::Tensor> kv_quant_scale_precomputed = std::nullopt) {
  auto B_T = XQ.size(0);
  auto N_H = XQ.size(1);

  auto N_KVH = 0;

  at::Tensor XK, XV;
  if (!update_kv) {
    assert(XK_.has_value() == false);
    XK = at::empty_like(XQ);
    // at::zeros({0, 0, 0}, at::BFloat16); // at::zeros(0);
    XV = at::empty_like(XQ);
    // at::zeros({0, 0, 0}, at::BFloat16);
  } else {
    XK = XK_.value();
    XV = XV_.value();
    N_KVH = XK.size(1);
  }

  TORCH_CHECK(XQ.size(2) % 4 == 0);
  TORCH_CHECK(XQ.size(2) <= 512);

  int32_t num_warps = B_T * (2 * N_KVH + N_H);
  TORCH_CHECK(num_warps > 0);

  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);
  dim3 blocks(cuda_calc_xblock_count(num_warps, kWarpsPerBlock));

  TORCH_CHECK(varseq_batch.is_contiguous());
  TORCH_CHECK(varseq_batch.numel() == B_T);
  auto XQ_O = at::empty_like(XQ);

  auto varseq_cache_seqpos_ = varseq_cache_seqpos.value_or(varseq_seqpos);

  int32_t* block_tables_ptr = nullptr;
  int32_t block_tables_b_stride = 0;
  if (block_tables.has_value()) {
    block_tables_ptr = static_cast<int32_t*>(block_tables.value().data_ptr());
    block_tables_b_stride = block_tables.value().stride(0);
  }
  CacheLogicalDtype cache_logical_dtype =
      static_cast<CacheLogicalDtype>(cache_logical_dtype_int);
  if (cache_K.dtype() == at::kBFloat16) {
    nope_qkv_varseq_prefill_kernel<<<
        blocks,
        threads,
        0,
        at::cuda::getCurrentCUDAStream()>>>(
        XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
        XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
        XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
        cache_K.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
        cache_V.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
        XQ_O.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
        varseq_batch.data_ptr<int32_t>(),
        varseq_seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
        block_tables_ptr,
        page_size,
        block_tables_b_stride,
        varseq_cache_seqpos_
            .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
        nullptr,
        update_kv);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    auto num_groups_ = num_groups ? num_groups.value() : 1;
    int32_t* qparam_k_ptr = nullptr;
    int32_t* qparam_v_ptr = nullptr;
    if (qparam_k.has_value()) {
      qparam_k_ptr = static_cast<int32_t*>(qparam_k.value().data_ptr());
      qparam_v_ptr = static_cast<int32_t*>(qparam_v.value().data_ptr());
    }
    auto varseq_batch_ = varseq_batch.data_ptr<int32_t>();
    auto varseq_seqpos_ =
        varseq_seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>();
    if (cache_logical_dtype == CacheLogicalDtype::FP8) {
#if (defined(USE_ROCM) && ROCM_VERSION >= 60200) || \
    (defined(CUDA_VERSION) && CUDA_VERSION >= 12000)
      CUDA_KERNEL_ASSERT(num_groups_ == 1);
      if (cache_K.dtype() == at::kFloat8_e4m3fn) {
        float* amax_ptr = nullptr;
        bool* is_precalculated_qparam = nullptr;
        if (amax_qkv.has_value()) {
          amax_ptr = static_cast<float*>(amax_qkv.value().data_ptr());
        }
        if (kv_quant_scale_precomputed.has_value()) {
          is_precalculated_qparam =
              static_cast<bool*>(kv_quant_scale_precomputed.value().data_ptr());
        }
        rope_xpos_qkv_varseq_prefill_kernel_fp8<
            PositionEmbeddingMode::NOPE,
            CacheLogicalDtype::FP8,
            1><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            cache_K.packed_accessor64<
                at::Float8_e4m3fn,
                4,
                at::RestrictPtrTraits>(),
            cache_V.packed_accessor64<
                at::Float8_e4m3fn,
                4,
                at::RestrictPtrTraits>(),
            qparam_k_ptr,
            qparam_v_ptr,
            varseq_batch_,
            varseq_seqpos_,
            0,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            varseq_cache_seqpos_
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            nullptr,
            false,
            0,
            0,
            0,
            0,
            true, // write_k_back and q too if we are doing norm.
            k_norm,
            amax_ptr,
            is_precalculated_qparam);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL(
            1,
            CacheLogicalDtype::FP8,
            PositionEmbeddingMode::NOPE,
            varseq_batch_,
            varseq_seqpos_,
            0,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            (varseq_cache_seqpos_
                 .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
            nullptr,
            false,
            0,
            0,
            0,
            0,
            false,
            k_norm);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
#else
      throw std::runtime_error("CUDA version is older than 12.0");
#endif
    } else {
      CALL_INT4_KERNEL_WITH_KV_GROUPWISE_QUANT_CHECK(
          CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL,
          num_groups_,
          CacheLogicalDtype::INT4,
          PositionEmbeddingMode::NOPE,
          varseq_batch_,
          varseq_seqpos_,
          0,
          0,
          0,
          0,
          block_tables_ptr,
          page_size,
          block_tables_b_stride,
          (varseq_cache_seqpos_
               .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
          nullptr,
          false,
          0,
          0,
          0,
          0,
          false,
          k_norm);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
  return XQ_O;
}

at::Tensor nope_qkv_decoding(
    at::Tensor XQ,
    std::optional<at::Tensor> XK_,
    std::optional<at::Tensor> XV_,
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor seqpos,
    std::optional<at::Tensor> block_tables,
    int64_t page_size,
    std::optional<at::Tensor> actual_batch_size,
    std::optional<at::Tensor> batch,
    std::optional<at::Tensor> cache_seqpos,
    int64_t cache_logical_dtype_int,
    std::optional<int64_t> num_groups,
    std::optional<at::Tensor> qparam_k = std::nullopt,
    std::optional<at::Tensor> qparam_v = std::nullopt,
    bool k_norm = false,
    bool update_kv = true,
    std::optional<at::Tensor> amax_qkv = std::nullopt) {
  auto B = XQ.size(0);
  auto N_H = XQ.size(1);
  // auto N_KVH = XK.size(1);
  auto N_KVH = 0;
  at::Tensor XK, XV;
  if (!update_kv) {
    assert(XK_.has_value() == false);
    XK = at::empty_like(XQ);
    // at::zeros({0, 0, 0}, at::BFloat16); // at::zeros(0);
    XV = at::empty_like(XQ);
    // at::zeros({0, 0, 0}, at::BFloat16);
  } else {
    XK = XK_.value();
    XV = XV_.value();
    N_KVH = XK.size(1);
  }

  TORCH_CHECK(XQ.size(2) % 4 == 0);
  int32_t num_warps = B * (2 * N_KVH + N_H);
  TORCH_CHECK(num_warps > 0);

  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);
  dim3 blocks(cuda_calc_xblock_count(num_warps, kWarpsPerBlock));
  auto XQ_O = at::empty_like(XQ);

  int32_t* block_tables_ptr = nullptr;
  int32_t block_tables_b_stride = 0;
  if (block_tables.has_value()) {
    block_tables_ptr = static_cast<int32_t*>(block_tables.value().data_ptr());
    block_tables_b_stride = block_tables.value().stride(0);
  }
  int64_t* actual_batch_size_ptr = nullptr;
  if (actual_batch_size.has_value()) {
    actual_batch_size_ptr =
        static_cast<int64_t*>(actual_batch_size.value().data_ptr());
  }
  auto cache_seqpos_ = cache_seqpos.value_or(seqpos);
  CacheLogicalDtype cache_logical_dtype =
      static_cast<CacheLogicalDtype>(cache_logical_dtype_int);
  if (cache_K.dtype() == at::kBFloat16) {
    nope_qkv_varseq_prefill_kernel<<<
        blocks,
        threads,
        0,
        at::cuda::getCurrentCUDAStream()>>>(
        XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
        XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
        XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
        cache_K.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
        cache_V.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
        XQ_O.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
        batch.has_value() ? batch.value().data_ptr<int32_t>() : nullptr,
        seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
        block_tables_ptr,
        page_size,
        block_tables_b_stride,
        cache_seqpos_.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
        actual_batch_size_ptr,
        update_kv);

    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    auto num_groups_ = num_groups ? num_groups.value() : 1;
    int32_t* qparam_k_ptr = nullptr;
    int32_t* qparam_v_ptr = nullptr;
    if (qparam_k.has_value()) {
      qparam_k_ptr = static_cast<int32_t*>(qparam_k.value().data_ptr());
      qparam_v_ptr = static_cast<int32_t*>(qparam_v.value().data_ptr());
    }
    auto batch_ =
        batch.has_value() ? batch.value().data_ptr<int32_t>() : nullptr;
    auto seqpos_ =
        seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>();
    if (cache_logical_dtype == CacheLogicalDtype::FP8) {
#if (defined(USE_ROCM) && ROCM_VERSION >= 60200) || \
    (defined(CUDA_VERSION) && CUDA_VERSION >= 12000)
      CUDA_KERNEL_ASSERT(num_groups_ == 1);
      if (cache_K.dtype() == at::kFloat8_e4m3fn) {
        float* amax_ptr = nullptr;
        if (amax_qkv.has_value()) {
          amax_ptr = static_cast<float*>(amax_qkv.value().data_ptr());
        }
        rope_xpos_qkv_varseq_prefill_kernel_fp8<
            PositionEmbeddingMode::NOPE,
            CacheLogicalDtype::FP8,
            1><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            cache_K.packed_accessor64<
                at::Float8_e4m3fn,
                4,
                at::RestrictPtrTraits>(),
            cache_V.packed_accessor64<
                at::Float8_e4m3fn,
                4,
                at::RestrictPtrTraits>(),
            qparam_k_ptr,
            qparam_v_ptr,
            batch_,
            seqpos_,
            0,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            cache_seqpos_
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            actual_batch_size_ptr,
            false,
            0,
            0,
            0,
            0,
            true, // write_k_back and q too if we are doing norm.
            k_norm,
            amax_ptr,
            nullptr);

        C10_CUDA_KERNEL_LAUNCH_CHECK();

      } else {
        CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL(
            1,
            CacheLogicalDtype::FP8,
            PositionEmbeddingMode::NOPE,
            batch_,
            seqpos_,
            0,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            (cache_seqpos_
                 .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
            actual_batch_size_ptr,
            false,
            0,
            0,
            0,
            0,
            false,
            k_norm);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
#else
      throw std::runtime_error("CUDA version is older than 12.0");
#endif
    } else {
      CALL_INT4_KERNEL_WITH_KV_GROUPWISE_QUANT_CHECK(
          CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL,
          num_groups_,
          CacheLogicalDtype::INT4,
          PositionEmbeddingMode::NOPE,
          batch_,
          seqpos_,
          0,
          0,
          0,
          0,
          block_tables_ptr,
          page_size,
          block_tables_b_stride,
          (cache_seqpos_
               .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
          actual_batch_size_ptr,
          false,
          0,
          0,
          0,
          0,
          false,
          k_norm);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
  return XQ_O;
}

at::Tensor rope_qkv_varseq_prefill(
    at::Tensor XQ,
    std::optional<at::Tensor> XK_,
    std::optional<at::Tensor> XV_,
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor varseq_batch,
    at::Tensor varseq_seqpos,
    double theta,
    std::optional<int64_t> num_groups,
    std::optional<at::Tensor> block_tables,
    int64_t page_size,
    std::optional<at::Tensor> varseq_cache_seqpos,
    int64_t cache_logical_dtype_int,
    bool rope_scaling = false,
    int64_t old_context_len = 8192,
    double scaling_factor = 16,
    double lo_freq_factor = 1,
    double hi_freq_factor = 32,
    std::optional<at::Tensor> qparam_k = std::nullopt,
    std::optional<at::Tensor> qparam_v = std::nullopt,
    bool write_k_back = false,
    bool k_norm = false,
    bool update_kv = true,
    std::optional<at::Tensor> amax_qkv = std::nullopt,
    std::optional<at::Tensor> kv_quant_scale_precomputed = std::nullopt) {
  auto B_T = XQ.size(0);
  auto N_H = XQ.size(1);
  auto N_KVH = 0;

  at::Tensor XK, XV;
  if (!update_kv) {
    assert(XK_.has_value() == false);
    XK = at::empty_like(XQ);
    // at::zeros({0, 0, 0}, at::BFloat16); // at::zeros(0);
    XV = at::empty_like(XQ);
    // at::zeros({0, 0, 0}, at::BFloat16);
  } else {
    XK = XK_.value();
    XV = XV_.value();
    N_KVH = XK.size(1);
  }

  TORCH_CHECK(XQ.size(2) % 4 == 0);
  TORCH_CHECK(XQ.size(2) <= 512);

  int32_t num_warps = B_T * (2 * N_KVH + N_H);
  TORCH_CHECK(num_warps > 0);

  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);
  dim3 blocks(cuda_calc_xblock_count(num_warps, kWarpsPerBlock));

  TORCH_CHECK(varseq_batch.is_contiguous());
  TORCH_CHECK(varseq_batch.numel() == B_T);
  auto XQ_O = at::empty_like(XQ);

  auto varseq_cache_seqpos_ = varseq_cache_seqpos.value_or(varseq_seqpos);

  CacheLogicalDtype cache_logical_dtype =
      static_cast<CacheLogicalDtype>(cache_logical_dtype_int);

  int32_t* block_tables_ptr = nullptr;
  int32_t block_tables_b_stride = 0;
  if (block_tables.has_value()) {
    block_tables_ptr = static_cast<int32_t*>(block_tables.value().data_ptr());
    block_tables_b_stride = block_tables.value().stride(0);
  }
  if (cache_K.dtype() == at::kBFloat16) {
    rope_xpos_qkv_varseq_prefill_kernel<PositionEmbeddingMode::ROPE>
        <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            cache_K.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
            cache_V.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
            XQ_O.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            varseq_batch.data_ptr<int32_t>(),
            varseq_seqpos
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            theta,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            varseq_cache_seqpos_
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            nullptr,
            rope_scaling,
            old_context_len,
            scaling_factor,
            lo_freq_factor,
            hi_freq_factor,
            write_k_back,
            update_kv);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    auto num_groups_ = num_groups ? num_groups.value() : 1;
    auto varseq_batch_ = varseq_batch.data_ptr<int32_t>();
    auto varseq_seqpos_ =
        varseq_seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>();
    int32_t* qparam_k_ptr = nullptr;
    int32_t* qparam_v_ptr = nullptr;
    if (qparam_k.has_value()) {
      qparam_k_ptr = static_cast<int32_t*>(qparam_k.value().data_ptr());
      qparam_v_ptr = static_cast<int32_t*>(qparam_v.value().data_ptr());
    }
    if (cache_logical_dtype == CacheLogicalDtype::FP8) {
#if (defined(USE_ROCM) && ROCM_VERSION >= 60200) || \
    (defined(CUDA_VERSION) && CUDA_VERSION >= 12000)
      CUDA_KERNEL_ASSERT(num_groups_ == 1);
      if (cache_K.dtype() == at::kFloat8_e4m3fn) {
        float* amax_ptr = nullptr;
        bool* is_precalculated_qparam = nullptr;
        if (amax_qkv.has_value()) {
          amax_ptr = static_cast<float*>(amax_qkv.value().data_ptr());
        }
        if (kv_quant_scale_precomputed.has_value()) {
          is_precalculated_qparam =
              static_cast<bool*>(kv_quant_scale_precomputed.value().data_ptr());
        }
        rope_xpos_qkv_varseq_prefill_kernel_fp8<
            PositionEmbeddingMode::ROPE,
            CacheLogicalDtype::FP8,
            1><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            cache_K.packed_accessor64<
                at::Float8_e4m3fn,
                4,
                at::RestrictPtrTraits>(),
            cache_V.packed_accessor64<
                at::Float8_e4m3fn,
                4,
                at::RestrictPtrTraits>(),
            qparam_k_ptr,
            qparam_v_ptr,
            varseq_batch_,
            varseq_seqpos_,
            theta,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            varseq_cache_seqpos_
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            nullptr,
            rope_scaling,
            old_context_len,
            scaling_factor,
            lo_freq_factor,
            hi_freq_factor,
            true,
            k_norm,
            amax_ptr,
            is_precalculated_qparam);
        C10_CUDA_KERNEL_LAUNCH_CHECK();

      } else {
        CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL(
            1,
            CacheLogicalDtype::FP8,
            PositionEmbeddingMode::ROPE,
            varseq_batch_,
            varseq_seqpos_,
            theta,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            (varseq_cache_seqpos_
                 .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
            nullptr,
            rope_scaling,
            old_context_len,
            scaling_factor,
            lo_freq_factor,
            hi_freq_factor,
            write_k_back,
            k_norm);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
#else
      throw std::runtime_error("CUDA version is older than 12.0");
#endif
    } else {
      CALL_INT4_KERNEL_WITH_KV_GROUPWISE_QUANT_CHECK(
          CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL,
          num_groups_,
          CacheLogicalDtype::INT4,
          PositionEmbeddingMode::ROPE,
          varseq_batch_,
          varseq_seqpos_,
          theta,
          0,
          0,
          0,
          block_tables_ptr,
          page_size,
          block_tables_b_stride,
          (varseq_cache_seqpos_
               .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
          nullptr,
          rope_scaling,
          old_context_len,
          scaling_factor,
          lo_freq_factor,
          hi_freq_factor,
          write_k_back,
          k_norm);

      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
  return XQ_O;
}

at::Tensor xpos_qkv_varseq_prefill(
    at::Tensor XQ,
    at::Tensor XK,
    at::Tensor XV,
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor varseq_batch,
    at::Tensor varseq_seqpos,
    double theta,
    double gamma,
    double scale_base,
    double exponent_offset,
    std::optional<int64_t> num_groups,
    std::optional<at::Tensor> block_tables,
    int64_t page_size,
    std::optional<at::Tensor> varseq_cache_seqpos,
    int64_t cache_logical_dtype_int,
    bool rope_scaling = false,
    int64_t old_context_len = 8192,
    double scaling_factor = 16,
    double lo_freq_factor = 1,
    double hi_freq_factor = 32,
    std::optional<at::Tensor> qparam_k = std::nullopt,
    std::optional<at::Tensor> qparam_v = std::nullopt) {
  auto B_T = XQ.size(0);
  auto N_H = XQ.size(1);
  auto N_KVH = XK.size(1);

  TORCH_CHECK(XQ.size(2) % 4 == 0);
  TORCH_CHECK(XQ.size(2) <= 512);

  int32_t num_warps = B_T * (2 * N_KVH + N_H);
  TORCH_CHECK(num_warps > 0);

  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);
  dim3 blocks(cuda_calc_xblock_count(num_warps, kWarpsPerBlock));

  auto XQ_O = at::empty_like(XQ);
  TORCH_CHECK(varseq_batch.is_contiguous());
  TORCH_CHECK(varseq_batch.numel() == B_T);
  auto varseq_cache_seqpos_ = varseq_cache_seqpos.value_or(varseq_seqpos);
  CacheLogicalDtype cache_logical_dtype =
      static_cast<CacheLogicalDtype>(cache_logical_dtype_int);

  int32_t* block_tables_ptr = nullptr;
  int32_t block_tables_b_stride = 0;
  if (block_tables.has_value()) {
    block_tables_ptr = static_cast<int32_t*>(block_tables.value().data_ptr());
    block_tables_b_stride = block_tables.value().stride(0);
  }

  if (cache_K.dtype() == at::kBFloat16) {
    rope_xpos_qkv_varseq_prefill_kernel<PositionEmbeddingMode::XPOS>
        <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            cache_K.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
            cache_V.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
            XQ_O.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            varseq_batch.data_ptr<int32_t>(),
            varseq_seqpos
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            theta,
            gamma,
            scale_base,
            exponent_offset,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            varseq_cache_seqpos_
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            nullptr,
            rope_scaling,
            old_context_len,
            scaling_factor,
            lo_freq_factor,
            hi_freq_factor);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    auto num_groups_ = num_groups ? num_groups.value() : 1;
    auto varseq_batch_ = varseq_batch.data_ptr<int32_t>();
    auto varseq_seqpos_ =
        varseq_seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>();
    int32_t* qparam_k_ptr = nullptr;
    int32_t* qparam_v_ptr = nullptr;
    if (qparam_k.has_value()) {
      qparam_k_ptr = static_cast<int32_t*>(qparam_k.value().data_ptr());
      qparam_v_ptr = static_cast<int32_t*>(qparam_v.value().data_ptr());
    }
    if (cache_logical_dtype == CacheLogicalDtype::FP8) {
#if (defined(USE_ROCM) && ROCM_VERSION >= 60200) || \
    (defined(CUDA_VERSION) && CUDA_VERSION >= 12000)
      CUDA_KERNEL_ASSERT(num_groups_ == 1);
      CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL(
          1,
          CacheLogicalDtype::FP8,
          PositionEmbeddingMode::XPOS,
          varseq_batch_,
          varseq_seqpos_,
          theta,
          gamma,
          scale_base,
          exponent_offset,
          block_tables_ptr,
          page_size,
          block_tables_b_stride,
          (varseq_cache_seqpos_
               .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
          nullptr,
          rope_scaling,
          old_context_len,
          scaling_factor,
          lo_freq_factor,
          hi_freq_factor,
          false,
          false);

      C10_CUDA_KERNEL_LAUNCH_CHECK();
#else
      throw std::runtime_error("CUDA version is older than 12.0");
#endif
    } else {
      CALL_INT4_KERNEL_WITH_KV_GROUPWISE_QUANT_CHECK(
          CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL,
          num_groups_,
          CacheLogicalDtype::INT4,
          PositionEmbeddingMode::XPOS,
          varseq_batch_,
          varseq_seqpos_,
          theta,
          gamma,
          scale_base,
          exponent_offset,
          block_tables_ptr,
          page_size,
          block_tables_b_stride,
          (varseq_cache_seqpos_
               .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
          nullptr,
          rope_scaling,
          old_context_len,
          scaling_factor,
          lo_freq_factor,
          hi_freq_factor,
          false,
          false);

      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
  return XQ_O;
}

at::Tensor rope_qkv_decoding(
    at::Tensor XQ,
    std::optional<at::Tensor> XK_,
    std::optional<at::Tensor> XV_,
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor seqpos,
    double theta,
    std::optional<int64_t> num_groups,
    std::optional<at::Tensor> block_tables,
    int64_t page_size,
    std::optional<at::Tensor> actual_batch_size,
    std::optional<at::Tensor> batch,
    std::optional<at::Tensor> cache_seqpos,
    int64_t cache_logical_dtype_int,
    bool rope_scaling = false,
    int64_t old_context_len = 8192,
    double scaling_factor = 16,
    double lo_freq_factor = 1,
    double hi_freq_factor = 32,
    std::optional<at::Tensor> qparam_k = std::nullopt,
    std::optional<at::Tensor> qparam_v = std::nullopt,
    bool k_norm = false,
    bool update_kv = true,
    std::optional<at::Tensor> amax_qkv = std::nullopt) {
  auto B = XQ.size(0);
  auto N_H = XQ.size(1);
  auto N_KVH = 0;
  at::Tensor XK, XV;
  if (!update_kv) {
    assert(XK_.has_value() == false);
    XK = at::empty_like(XQ);
    // at::zeros({0, 0, 0}, at::BFloat16); // at::zeros(0);
    XV = at::empty_like(XQ);
    // at::zeros({0, 0, 0}, at::BFloat16);
  } else {
    XK = XK_.value();
    XV = XV_.value();
    N_KVH = XK.size(1);
  }

  TORCH_CHECK(XQ.size(2) % 4 == 0);
  int32_t num_warps = B * (2 * N_KVH + N_H);
  TORCH_CHECK(num_warps > 0);

  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);
  dim3 blocks(cuda_calc_xblock_count(num_warps, kWarpsPerBlock));
  // TODO: not needed for fp8 attn case
  auto XQ_O = at::empty_like(XQ);

  CacheLogicalDtype cache_logical_dtype =
      static_cast<CacheLogicalDtype>(cache_logical_dtype_int);

  int32_t* block_tables_ptr = nullptr;
  int32_t block_tables_b_stride = 0;
  if (block_tables.has_value()) {
    block_tables_ptr = static_cast<int32_t*>(block_tables.value().data_ptr());
    block_tables_b_stride = block_tables.value().stride(0);
  }
  int64_t* actual_batch_size_ptr = nullptr;
  if (actual_batch_size.has_value()) {
    actual_batch_size_ptr =
        static_cast<int64_t*>(actual_batch_size.value().data_ptr());
  }
  auto cache_seqpos_ = cache_seqpos.value_or(seqpos);
  if (cache_K.dtype() == at::kBFloat16) {
    rope_xpos_qkv_varseq_prefill_kernel<PositionEmbeddingMode::ROPE>
        <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            cache_K.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
            cache_V.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
            XQ_O.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            batch.has_value() ? batch.value().data_ptr<int32_t>() : nullptr,
            seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            theta,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            cache_seqpos_
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            actual_batch_size_ptr,
            rope_scaling,
            old_context_len,
            scaling_factor,
            lo_freq_factor,
            hi_freq_factor,
            false,
            update_kv);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    auto seqpos_ =
        seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>();
    auto num_groups_ = num_groups ? num_groups.value() : 1;
    int32_t* qparam_k_ptr = nullptr;
    int32_t* qparam_v_ptr = nullptr;
    if (qparam_k.has_value()) {
      qparam_k_ptr = static_cast<int32_t*>(qparam_k.value().data_ptr());
      qparam_v_ptr = static_cast<int32_t*>(qparam_v.value().data_ptr());
    }
    if (cache_logical_dtype == CacheLogicalDtype::FP8) {
#if (defined(USE_ROCM) && ROCM_VERSION >= 60200) || \
    (defined(CUDA_VERSION) && CUDA_VERSION >= 12000)
      CUDA_KERNEL_ASSERT(num_groups_ == 1);
      if (cache_K.dtype() == at::kFloat8_e4m3fn) {
        float* amax_ptr = nullptr;
        if (amax_qkv.has_value()) {
          amax_ptr = static_cast<float*>(amax_qkv.value().data_ptr());
        }
        rope_xpos_qkv_varseq_prefill_kernel_fp8<
            PositionEmbeddingMode::ROPE,
            CacheLogicalDtype::FP8,
            1><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            cache_K.packed_accessor64<
                at::Float8_e4m3fn,
                4,
                at::RestrictPtrTraits>(),
            cache_V.packed_accessor64<
                at::Float8_e4m3fn,
                4,
                at::RestrictPtrTraits>(),
            qparam_k_ptr,
            qparam_v_ptr,
            batch.has_value() ? batch.value().data_ptr<int32_t>() : nullptr,
            seqpos_,
            theta,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            cache_seqpos_
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            actual_batch_size_ptr,
            rope_scaling,
            old_context_len,
            scaling_factor,
            lo_freq_factor,
            hi_freq_factor,
            true,
            k_norm,
            amax_ptr,
            nullptr);

        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL(
            1,
            CacheLogicalDtype::FP8,
            PositionEmbeddingMode::ROPE,
            nullptr,
            seqpos_,
            theta,
            0,
            0,
            0,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            (cache_seqpos_
                 .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
            actual_batch_size_ptr,
            rope_scaling,
            old_context_len,
            scaling_factor,
            lo_freq_factor,
            hi_freq_factor,
            false,
            k_norm);

        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
#else
      throw std::runtime_error("CUDA version is older than 12.0");
#endif
    } else {
      CALL_INT4_KERNEL_WITH_KV_GROUPWISE_QUANT_CHECK(
          CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL,
          num_groups_,
          CacheLogicalDtype::INT4,
          PositionEmbeddingMode::ROPE,
          nullptr,
          seqpos_,
          theta,
          0,
          0,
          0,
          block_tables_ptr,
          page_size,
          block_tables_b_stride,
          (cache_seqpos_
               .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
          actual_batch_size_ptr,
          rope_scaling,
          old_context_len,
          scaling_factor,
          lo_freq_factor,
          hi_freq_factor,
          false,
          k_norm);

      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }

  return XQ_O;
}

at::Tensor xpos_qkv_decoding(
    at::Tensor XQ,
    at::Tensor XK,
    at::Tensor XV,
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor seqpos,
    double theta,
    double gamma,
    double scale_base,
    double exponent_offset,
    std::optional<int64_t> num_groups,
    std::optional<at::Tensor> block_tables,
    int64_t page_size,
    std::optional<at::Tensor> actual_batch_size,
    std::optional<at::Tensor> batch,
    std::optional<at::Tensor> cache_seqpos,
    int64_t cache_logical_dtype_int,
    bool rope_scaling = false,
    int64_t old_context_len = 8192,
    double scaling_factor = 16,
    double lo_freq_factor = 1,
    double hi_freq_factor = 32,
    std::optional<at::Tensor> qparam_k = std::nullopt,
    std::optional<at::Tensor> qparam_v = std::nullopt) {
  auto B = XQ.size(0);
  auto N_H = XQ.size(1);
  auto N_KVH = XK.size(1);

  TORCH_CHECK(XQ.size(2) % 4 == 0);
  int32_t num_warps = B * (2 * N_KVH + N_H);
  TORCH_CHECK(num_warps > 0);

  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);
  dim3 blocks(cuda_calc_xblock_count(num_warps, kWarpsPerBlock));
  auto XQ_O = at::empty_like(XQ);
  CacheLogicalDtype cache_logical_dtype =
      static_cast<CacheLogicalDtype>(cache_logical_dtype_int);

  int32_t* block_tables_ptr = nullptr;
  int32_t block_tables_b_stride = 0;
  if (block_tables.has_value()) {
    block_tables_ptr = static_cast<int32_t*>(block_tables.value().data_ptr());
    block_tables_b_stride = block_tables.value().stride(0);
  }

  int64_t* actual_batch_size_ptr = nullptr;
  if (actual_batch_size.has_value()) {
    actual_batch_size_ptr =
        static_cast<int64_t*>(actual_batch_size.value().data_ptr());
  }
  auto cache_seqpos_ = cache_seqpos.value_or(seqpos);
  if (cache_K.dtype() == at::kBFloat16) {
    rope_xpos_qkv_varseq_prefill_kernel<PositionEmbeddingMode::XPOS>
        <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            XQ.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XK.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            XV.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            cache_K.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
            cache_V.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
            XQ_O.packed_accessor32<at::BFloat16, 3, at::RestrictPtrTraits>(),
            batch.has_value() ? batch.value().data_ptr<int32_t>() : nullptr,
            seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            theta,
            gamma,
            scale_base,
            exponent_offset,
            block_tables_ptr,
            page_size,
            block_tables_b_stride,
            cache_seqpos_
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            actual_batch_size_ptr,
            rope_scaling,
            old_context_len,
            scaling_factor,
            lo_freq_factor,
            hi_freq_factor);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    auto num_groups_ = num_groups ? num_groups.value() : 1;
    auto seqpos_ =
        seqpos.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>();
    int32_t* qparam_k_ptr = nullptr;
    int32_t* qparam_v_ptr = nullptr;
    if (qparam_k.has_value()) {
      qparam_k_ptr = static_cast<int32_t*>(qparam_k.value().data_ptr());
      qparam_v_ptr = static_cast<int32_t*>(qparam_v.value().data_ptr());
    }
    if (cache_logical_dtype == CacheLogicalDtype::FP8) {
#if (defined(USE_ROCM) && ROCM_VERSION >= 60200) || \
    (defined(CUDA_VERSION) && CUDA_VERSION >= 12000)
      CUDA_KERNEL_ASSERT(num_groups_ == 1);
      CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL(
          1,
          CacheLogicalDtype::FP8,
          PositionEmbeddingMode::XPOS,
          nullptr,
          seqpos_,
          theta,
          gamma,
          scale_base,
          exponent_offset,
          block_tables_ptr,
          page_size,
          block_tables_b_stride,
          (cache_seqpos_
               .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
          actual_batch_size_ptr,
          rope_scaling,
          old_context_len,
          scaling_factor,
          lo_freq_factor,
          hi_freq_factor,
          false,
          false);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
#else
      throw std::runtime_error("CUDA version is older than 12.0");
#endif
    } else {
      CALL_INT4_KERNEL_WITH_KV_GROUPWISE_QUANT_CHECK(
          CALL_ROPE_XPOS_QKV_VARSEQ_PREFILL_GROUPWISE_KERNEL,
          num_groups_,
          CacheLogicalDtype::INT4,
          PositionEmbeddingMode::XPOS,
          nullptr,
          seqpos_,
          theta,
          gamma,
          scale_base,
          exponent_offset,
          block_tables_ptr,
          page_size,
          block_tables_b_stride,
          (cache_seqpos_
               .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>()),
          actual_batch_size_ptr,
          rope_scaling,
          old_context_len,
          scaling_factor,
          lo_freq_factor,
          hi_freq_factor,
          false,
          false);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
  return XQ_O;
}

#if (defined(USE_ROCM))
/**
 * Converts the contents of a FP8 KV cache from e4m3fn (NV) to e4m3fnuz (AMD).
 * These formats differ in their support for negative zero, and in their
 * exponent bias. Negative zeros are replaced with positive zero, and the scale
 * qparam is multiplied by 2.0, because we know that the scale will be applied
 * to the k/v value and is equivalent to recomputing the exponent bias.
 *
 * This in an inplace operation.
 *
 * It is assumed that inputs will have been generated with scale_ub = max(fp16)
 * / 2 to avoid overflow. Some debug mode assertions are in place, but there are
 * no runtime guarantees.
 *
 * As written, this kernel is only valid on AMD, because it relies on threads
 * 32-63 to convert the V tensors. NV only has threads 0-31 per warp.
 */
__global__ void convert_e4m3fn_kv_cache_to_e4m3fnuz_inplace_kernel(
    at::PackedTensorAccessor64<uint8_t, 5, at::RestrictPtrTraits>
        cache_K, // [N_H_L][B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<uint8_t, 5, at::RestrictPtrTraits>
        cache_V, // [N_H_L][B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<int32_t, 5, at::RestrictPtrTraits> qparam_K,
    at::PackedTensorAccessor64<int32_t, 5, at::RestrictPtrTraits> qparam_V,
    int* errorFlag,
    half* beforeVal,
    half* afterVal) {
  auto N_KVH = cache_K.size(3);
  auto MAX_T = cache_K.size(2);
  auto D_H = cache_K.size(4);
  CUDA_KERNEL_ASSERT(D_H == 128);

  auto l = blockIdx.x;
  auto b = blockIdx.y;
  int h = 0, t = 0;
  uint8_t* head;
  __half2* shift_scale;

  for (auto t_h = threadIdx.y + blockIdx.z * blockDim.y; t_h < MAX_T * N_KVH;
       t_h += blockDim.y * gridDim.z) {
    h = t_h % N_KVH;
    t = t_h / N_KVH;

    auto tidx = threadIdx.x;
    if (threadIdx.x < 32) {
      head = &cache_K[l][b][t][h][0];
      shift_scale = reinterpret_cast<__half2*>(&qparam_K[l][b][t][h][0]);
    } else {
      head = &cache_V[l][b][t][h][0];
      shift_scale = reinterpret_cast<__half2*>(&qparam_V[l][b][t][h][0]);
      tidx -= 32;
    }
    auto D_H_idx = tidx * 4; // Reading 4 bytes at once.
    auto negative_zero = 0x80;

    // Our only goal here is to detect negative zeros that are valid
    // in e4m3fn, but not valid in e4m3fnuz, and overwrite them with positive
    // zeros.
    uint32_t packed_fp8x4_vals = *reinterpret_cast<uint32_t*>(&head[D_H_idx]);
    if (((packed_fp8x4_vals >> 24) & 0xff) == negative_zero) {
      packed_fp8x4_vals &= 0x00ffffff;
    }
    if (((packed_fp8x4_vals >> 16) & 0xff) == negative_zero) {
      packed_fp8x4_vals &= 0xff00ffff;
    }
    if (((packed_fp8x4_vals >> 8) & 0xff) == negative_zero) {
      packed_fp8x4_vals &= 0xffff00ff;
    }
    if ((packed_fp8x4_vals & 0xff) == negative_zero) {
      packed_fp8x4_vals &= 0xffffff00;
    }
    *reinterpret_cast<uint32_t*>(&head[D_H_idx]) = packed_fp8x4_vals;

    // Multiply qparam scale (member x) by 2 to compensate for the exponent
    // bias difference (1) between e4m3fn and e4m3fnuz. We only need to do
    // this once per row. In debug mode, assert that 2.0*scale as a float would
    // not exceed the max value of __half.
    if (tidx == 0) {
      __half shift = __high2half(*shift_scale);
      __half scale = __low2half(*shift_scale);

      CUDA_KERNEL_ASSERT(__half2float(scale) * 2.0f <= 65504.0f);

      __half new_scale = __hmul(scale, __float2half(2.0f));
      *shift_scale = __half2(new_scale, shift);

      if (__half2float(scale) * 2.0f > 65504.0f) {
        // Record error if scale overflows.
        *beforeVal = scale;
        *afterVal = new_scale;
        *errorFlag = 1;
      }
    }
  }
}

void convert_e4m3fn_kv_cache_to_e4m3fnuz_inplace(
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor qparam_K,
    at::Tensor qparam_V) {
  TORCH_CHECK(cache_K.is_cuda());
  TORCH_CHECK(cache_V.is_cuda());
  TORCH_CHECK(qparam_K.is_cuda());
  TORCH_CHECK(qparam_V.is_cuda());

  auto N_H_L = cache_K.size(0);
  auto B = cache_K.size(1);

  constexpr int32_t kMaxBlocks = 512;
  // Blocks: (N_H_L, B, residual from max blocks)
  dim3 blocks(N_H_L, B, std::max<int32_t>(1, kMaxBlocks / (B * N_H_L)));
  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);

  int* d_errorFlag;
  int h_errorFlag = 0;
  hipMalloc(&d_errorFlag, sizeof(int));
  hipMemcpy(d_errorFlag, &h_errorFlag, sizeof(int), hipMemcpyHostToDevice);

  half* d_beforeVal;
  half h_beforeVal = __float2half(0.0f);
  hipMalloc(&d_beforeVal, sizeof(__half));
  hipMemcpy(d_beforeVal, &h_beforeVal, sizeof(__half), hipMemcpyHostToDevice);

  half* d_afterVal;
  half h_afterVal = __float2half(0.0f);
  hipMalloc(&d_afterVal, sizeof(__half));
  hipMemcpy(d_afterVal, &h_afterVal, sizeof(__half), hipMemcpyHostToDevice);

  convert_e4m3fn_kv_cache_to_e4m3fnuz_inplace_kernel<<<
      blocks,
      threads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      cache_K.packed_accessor64<uint8_t, 5, at::RestrictPtrTraits>(),
      cache_V.packed_accessor64<uint8_t, 5, at::RestrictPtrTraits>(),
      qparam_K.packed_accessor64<int32_t, 5, at::RestrictPtrTraits>(),
      qparam_V.packed_accessor64<int32_t, 5, at::RestrictPtrTraits>(),
      d_errorFlag,
      d_beforeVal,
      d_afterVal);

  C10_CUDA_KERNEL_LAUNCH_CHECK();

  hipMemcpy(&h_errorFlag, d_errorFlag, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&h_beforeVal, d_beforeVal, sizeof(__half), hipMemcpyDeviceToHost);
  hipMemcpy(&h_afterVal, d_afterVal, sizeof(__half), hipMemcpyDeviceToHost);
  // Check the error flag
  if (h_errorFlag != 0) {
    std::stringstream ss;
    ss << "Error detected in convert_e4m3fn_kv_cache_to_e4m3fnuz_inplace execution! ["
       << h_errorFlag << "] scale before: ";
    uint16_t h_beforeVal_u16 = std::bit_cast<uint16_t>(h_beforeVal);
    for (int i = 15; i >= 0; --i) {
      ss << ((h_beforeVal_u16 >> i) & 1);
    }
    ss << " scale after: ";
    uint16_t h_afterVal_u16 = std::bit_cast<uint16_t>(h_afterVal);
    for (int i = 15; i >= 0; --i) {
      ss << ((h_afterVal_u16 >> i) & 1);
    }
    LOG(ERROR) << ss.str() << std::endl;
  }
}
#else
void convert_e4m3fn_kv_cache_to_e4m3fnuz_inplace(
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor qparam_K,
    at::Tensor qparam_V) {
  throw std::runtime_error(
      "convert_e4m3fn_kv_cache_to_e4m3fnuz_inplace is only supported on AMD");
}
#endif

#if (defined(USE_ROCM) && ROCM_VERSION >= 60200) || \
    (defined(CUDA_VERSION) && CUDA_VERSION >= 12000)

#if (defined(USE_ROCM) && ROCM_VERSION >= 60200)
template <bool ExternalQParam>
__global__ void dequantize_fp8_cache_kernel(
    // This code currently represents FP8 version not int4
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_K, // [B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_V, // [B][MAX_T][N_KVH][D_H // G]
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> kv_seqlen,
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_K_dq, // [B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_V_dq, // [B][MAX_T][N_KVH][D_H]
    int32_t* qparam_k_ptr,
    int32_t* qparam_v_ptr) {
  auto N_KVH = cache_K.size(2);
  auto MAX_T = cache_K.size(1);
  auto D_H = cache_K_dq.size(3);
  auto D_H_q = cache_K.size(3);
  // TODO: support D_H < 128 for small model used in testing.
  CUDA_KERNEL_ASSERT(D_H == 128);
  const uint8_t offset_bytes = (ExternalQParam) ? 0 : 4;
  CUDA_KERNEL_ASSERT(D_H_q - D_H == offset_bytes);

  auto b = blockIdx.x;
  // only need to dequantize this far.
  auto max_t = kv_seqlen[b];

  // one warp per T/H
  int h = 0, t = 0;
  uint8_t* row;
  c10::BFloat16* row_dq{};
  bfx4 kv_dq;
  long t_h{};
  __half2* qparam_src;
  // On AMD, we have 64 threads per warp.
  // We use the first 32 threads to process K
  // and the second 32 threads to process V
  for (t_h = threadIdx.y + blockIdx.y * blockDim.y; t_h < max_t * N_KVH;
       t_h += blockDim.y * gridDim.y) {
    h = t_h % N_KVH;
    t = t_h / N_KVH;
    size_t idx = b * (MAX_T * N_KVH) + t * N_KVH + h;
    auto tidx = threadIdx.x;
    if (threadIdx.x < 32) {
      row = &cache_K[b][t][h][0];
      row_dq = &cache_K_dq[b][t][h][0];
      if constexpr (ExternalQParam) {
        qparam_src = reinterpret_cast<__half2*>(&qparam_k_ptr[idx]);
      } else {
        qparam_src = reinterpret_cast<__half2*>(&row[0]);
      }
    } else {
      row = &cache_V[b][t][h][0];
      row_dq = &cache_V_dq[b][t][h][0];
      if constexpr (ExternalQParam) {
        qparam_src = reinterpret_cast<__half2*>(&qparam_v_ptr[idx]);
      } else {
        qparam_src = reinterpret_cast<__half2*>(&row[0]);
      }
      tidx -= 32;
    }
    uint32_t q = *reinterpret_cast<uint32_t*>(&row[tidx * 4 + offset_bytes]);
    kv_dq = dequantize_packed_fp8(q, *qparam_src);
    // now, write our outputs
    // each thread writes 4 elements of type bf16
    *reinterpret_cast<uint2*>(&row_dq[4 * tidx]) =
        *reinterpret_cast<uint2*>(&kv_dq.vals[0]);
  }

  max_t = (max_t + 127) / 128 * 128;
  max_t = max_t > MAX_T ? MAX_T : max_t;
  for (; t_h < max_t * N_KVH; t_h += blockDim.y * gridDim.y) {
    h = t_h % N_KVH;
    t = t_h / N_KVH;
    auto tidx = threadIdx.x;
    if (threadIdx.x < 32) {
      row_dq = &cache_K_dq[b][t][h][0];
    } else {
      row_dq = &cache_V_dq[b][t][h][0];
      tidx -= 32;
    }
    memset(&row_dq[4 * tidx], 0, sizeof(uint2));
  }
}

__global__ void dequantize_fp8_cache_kernel_paged(
    // This code currently represents FP8 version not int4
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_K, // [1][MAX_PAGE * PAGE_SIZE][N_KVH][D_H]
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_V, // [1][MAX_PAGE * PAGE_SIZE][N_KVH][D_H // G]
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> kv_seqlen,
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_K_dq, // [1][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_V_dq, // [1][MAX_T][N_KVH][D_H]
    int32_t* qparam_k_ptr,
    int32_t* qparam_v_ptr,
    int32_t* block_tables,
    int32_t block_tables_b_stride,
    int32_t page_size) {
  CUDA_KERNEL_ASSERT(0 && "unimplemented");
}

#else
template <bool ExternalQParam>
__global__ void dequantize_fp8_cache_kernel(
    // This code currently represents FP8 version not int4
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_K, // [B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_V, // [B][MAX_T][N_KVH][D_H // G]
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> kv_seqlen,
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_K_dq, // [B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_V_dq, // [B][MAX_T][N_KVH][D_H]
    int32_t* qparam_k_ptr,
    int32_t* qparam_v_ptr) {
  auto N_KVH = cache_K.size(2);
  auto MAX_T = cache_K.size(1);
  auto D_H = cache_K_dq.size(3);
  auto D_H_q = cache_K.size(3);
  // TODO: support D_H < 128 for small model used in testing.
  CUDA_KERNEL_ASSERT(D_H == 128);
  const uint8_t offset_bytes = (ExternalQParam) ? 0 : 4;
  CUDA_KERNEL_ASSERT(D_H_q - D_H == offset_bytes);

  auto b = blockIdx.x;
  // only need to dequantize this far.
  auto max_t = kv_seqlen[b];

  // one warp per T/H
  int h = 0, t = 0;
  uint8_t *row_k{}, *row_v{};
  c10::BFloat16 *row_k_dq{}, *row_v_dq{};
  uint64_t packed{};
  bfx8 kv_dq;
  long t_h{};
  for (t_h = threadIdx.y + blockIdx.y * blockDim.y; t_h < max_t * N_KVH;
       t_h += blockDim.y * gridDim.y) {
    h = t_h % N_KVH;
    t = t_h / N_KVH;

    row_k = &cache_K[b][t][h][0];
    row_v = &cache_V[b][t][h][0];
    row_k_dq = &cache_K_dq[b][t][h][0];
    row_v_dq = &cache_V_dq[b][t][h][0];
    // Calculate kv_dq for this row
    {
      __half2* qparam_k_src;
      __half2* qparam_v_src;
      if (ExternalQParam) {
        size_t idx = b * (MAX_T * N_KVH) + t * N_KVH + h;
        qparam_k_src = reinterpret_cast<__half2*>(&qparam_k_ptr[idx]);
        qparam_v_src = reinterpret_cast<__half2*>(&qparam_v_ptr[idx]);
      } else {
        qparam_k_src = reinterpret_cast<__half2*>(&row_k[0]);
        qparam_v_src = reinterpret_cast<__half2*>(&row_v[0]);
      }
      uint64_t kq =
          *reinterpret_cast<uint32_t*>(&row_k[threadIdx.x * 4 + offset_bytes]);
      uint64_t vq =
          *reinterpret_cast<uint32_t*>(&row_v[threadIdx.x * 4 + offset_bytes]);

      packed = kq | (vq << 32);

      kv_dq = dequantize_packed_fp8(packed, *qparam_k_src, *qparam_v_src);
    }
    // now, write our outputs
    // each thread writes 4 elements of type bf16
    *reinterpret_cast<uint2*>(&row_k_dq[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&kv_dq.vals[0]);
    *reinterpret_cast<uint2*>(&row_v_dq[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&kv_dq.vals[2]);
  }

  max_t = (max_t + 127) / 128 * 128;
  max_t = max_t > MAX_T ? MAX_T : max_t;
  for (; t_h < max_t * N_KVH; t_h += blockDim.y * gridDim.y) {
    h = t_h % N_KVH;
    t = t_h / N_KVH;
    row_k_dq = &cache_K_dq[b][t][h][0];
    row_v_dq = &cache_V_dq[b][t][h][0];

    memset(&row_k_dq[4 * threadIdx.x], 0, sizeof(uint2));
    memset(&row_v_dq[4 * threadIdx.x], 0, sizeof(uint2));
  }
}

// Cloned from dequantize_fp8_cache_kernel because
// branching inside the original kernel runs into
// "too many resources requested for launch" which
// necessitates decreasing the number of warps per block,
// which might have performance implications. Also we
// might have more diverging behaviors for paged kernel
// as noted in the comment below so we will keep a separate
// kernel for now.
__global__ void dequantize_fp8_cache_kernel_paged(
    // This code currently represents FP8 version not int4
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_K, // [1][MAX_PAGE * PAGE_SIZE][N_KVH][D_H]
    at::PackedTensorAccessor64<uint8_t, 4, at::RestrictPtrTraits>
        cache_V, // [1][MAX_PAGE * PAGE_SIZE][N_KVH][D_H // G]
    at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> kv_seqlen,
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_K_dq, // [1][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::BFloat16, 4, at::RestrictPtrTraits>
        cache_V_dq, // [1][MAX_T][N_KVH][D_H]
    int32_t* qparam_k_ptr,
    int32_t* qparam_v_ptr,
    int32_t* block_tables,
    int32_t block_tables_b_stride,
    int32_t page_size) {
  auto N_KVH = cache_K.size(2);
  auto D_H = cache_K_dq.size(3);
  auto D_H_q = cache_K.size(3);
  CUDA_KERNEL_ASSERT(D_H == 128);

  auto b = blockIdx.x;
  // only need to dequantize this far.
  auto max_t = kv_seqlen[b];

  // one warp per T/H
  for (auto t_h = threadIdx.y + blockIdx.y * blockDim.y; t_h < max_t * N_KVH;
       t_h += blockDim.y * gridDim.y) {
    auto h = t_h % N_KVH;
    auto t = t_h / N_KVH;

    int page_logical_idx = t / page_size;
    int page_offset = t % page_size;
    int page_physical_idx =
        block_tables[b * block_tables_b_stride + page_logical_idx];
    int physical_t = page_physical_idx * page_size + page_offset;

    uint8_t* row_k = &cache_K[0][physical_t][h][0];
    uint8_t* row_v = &cache_V[0][physical_t][h][0];

    bfx8 kv_dq;
    uint8_t qparam_offset_bytes;
    __half2* qparam_k_src;
    __half2* qparam_v_src;
    if (qparam_k_ptr) {
      // read from standalone qparam tensor
      qparam_offset_bytes = 0;
      auto idx = physical_t * N_KVH + h;
      qparam_k_src = reinterpret_cast<__half2*>(&qparam_k_ptr[idx]);
      qparam_v_src = reinterpret_cast<__half2*>(&qparam_v_ptr[idx]);
    } else {
      // read from first row
      qparam_offset_bytes = 4;
      qparam_k_src = reinterpret_cast<__half2*>(&row_k[0]);
      qparam_v_src = reinterpret_cast<__half2*>(&row_v[0]);
    }
    // Assert the quantized row dim is as expected
    CUDA_KERNEL_ASSERT(D_H_q - D_H == qparam_offset_bytes);
    if (4 * threadIdx.x >= D_H) {
      continue;
    }
    // each thread reads 4 x 8 bits

    uint64_t kq = *reinterpret_cast<uint32_t*>(
        &row_k[threadIdx.x * 4 + qparam_offset_bytes]);
    uint64_t vq = *reinterpret_cast<uint32_t*>(
        &row_v[threadIdx.x * 4 + qparam_offset_bytes]);

    uint64_t packed = kq | (vq << 32);

    kv_dq = dequantize_packed_fp8(packed, *qparam_k_src, *qparam_v_src);

    // now, write our outputs
    auto* row_k_dq = &cache_K_dq[0][physical_t][h][0];
    auto* row_v_dq = &cache_V_dq[0][physical_t][h][0];
    // each thread writes 4 elements of type bf16
    *reinterpret_cast<uint2*>(&row_k_dq[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&kv_dq.vals[0]);
    *reinterpret_cast<uint2*>(&row_v_dq[4 * threadIdx.x]) =
        *reinterpret_cast<uint2*>(&kv_dq.vals[2]);
  }
}
#endif

std::tuple<at::Tensor, at::Tensor> dequantize_fp8_cache(
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor kv_seqlen,
    std::optional<at::Tensor> qparam_k,
    std::optional<at::Tensor> qparam_v,
    std::optional<at::Tensor> block_tables,
    int64_t page_size) {
  TORCH_CHECK(cache_K.is_cuda());
  TORCH_CHECK(cache_V.is_cuda());
  TORCH_CHECK(kv_seqlen.is_cuda());
  auto B = kv_seqlen.size(0);
  // vanilla: B_KV = B, paged: B_KV = 1
  auto B_KV = cache_K.size(0);
  // vanilla: MAX_T = MAX_T, paged: MAX_T = MAX_PAGE * PAGE_SIZE
  auto MAX_T = cache_K.size(1);
  auto N_KVH = cache_K.size(2);
  auto D_HQ = cache_K.size(3);
  auto fp8_qparam_offset = 4;
  int32_t* qparam_k_ptr = nullptr;
  int32_t* qparam_v_ptr = nullptr;
  if (qparam_k.has_value()) {
    qparam_k_ptr = static_cast<int32_t*>(qparam_k.value().data_ptr());
    qparam_v_ptr = static_cast<int32_t*>(qparam_v.value().data_ptr());
    fp8_qparam_offset = 0;
  }
  auto D_H = (D_HQ - fp8_qparam_offset);

  // TODO:
  // The below allocates Tensors that have the same shape as cache_K and
  // cache_V to store their dequantize results. For paged KV cache, this can
  // be a bit inefficient because it has the shape of [1 x (MAX_PAGES *
  // PAGE_SIZE) x N_KVH x D_H] to accommodate pages globally across batch
  // instances, and if we have very large MAX_PAGES then we are essentially
  // allocating a very huge Tensor here. The benefit is that the following
  // users of this dequantized results can reuse the existing block_tables to
  // access their elements. If we want to be more efficient, there are two
  // possible approaches: (1) Allocate a shorter Tensor here and store the
  // dequantize results in a more compact manner, but that requires creating a
  // new block_tables here and making sure the following users all use the
  // correct block_tables. (2) From outside, keep a persistent buffer that has
  // a matching shape with the original paged KV and feed the same buffer into
  // this function at every layer to reuse it and prevent allocation.

  auto cache_K_dq = at::empty(
      {B_KV, MAX_T, N_KVH, D_H}, cache_K.options().dtype(at::kBFloat16));
  auto cache_V_dq = at::empty(
      {B_KV, MAX_T, N_KVH, D_H}, cache_K.options().dtype(at::kBFloat16));

  if (B == 0) {
    return {cache_K_dq, cache_V_dq};
  }

  int32_t* block_tables_ptr = nullptr;
  int32_t block_tables_b_stride = 0;
  if (block_tables.has_value()) {
    block_tables_ptr = static_cast<int32_t*>(block_tables.value().data_ptr());
    block_tables_b_stride = block_tables.value().stride(0);
  }

  constexpr int32_t kMaxBlocks = 512;
  dim3 blocks(B, std::max<int32_t>(1, kMaxBlocks / B));
  dim3 threads(kThreadsPerWarp, kWarpsPerBlock);
#define CALL_DEQUANTIZE_FP8_CACHE(EXTERNAL_Q_PARAM)                           \
  const auto deq_fn = dequantize_fp8_cache_kernel<EXTERNAL_Q_PARAM>;          \
  deq_fn<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(           \
      cache_K.packed_accessor64<uint8_t, 4, at::RestrictPtrTraits>(),         \
      cache_V.packed_accessor64<uint8_t, 4, at::RestrictPtrTraits>(),         \
      kv_seqlen.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),       \
      cache_K_dq.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(), \
      cache_V_dq.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(), \
      qparam_k_ptr,                                                           \
      qparam_v_ptr);                                                          \
  C10_CUDA_KERNEL_LAUNCH_CHECK()
  if (block_tables_ptr == nullptr) {
    if (qparam_k_ptr) {
      CALL_DEQUANTIZE_FP8_CACHE(true);
    } else {
      CALL_DEQUANTIZE_FP8_CACHE(false);
    }
#undef CALL_DEQUANTIZE_FP8_CACHE
  } else {
    dequantize_fp8_cache_kernel_paged<<<
        blocks,
        threads,
        0,
        at::cuda::getCurrentCUDAStream()>>>(
        cache_K.packed_accessor64<uint8_t, 4, at::RestrictPtrTraits>(),
        cache_V.packed_accessor64<uint8_t, 4, at::RestrictPtrTraits>(),
        kv_seqlen.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
        cache_K_dq.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
        cache_V_dq.packed_accessor64<at::BFloat16, 4, at::RestrictPtrTraits>(),
        qparam_k_ptr,
        qparam_v_ptr,
        block_tables_ptr,
        block_tables_b_stride,
        page_size);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  return {cache_K_dq, cache_V_dq};
}

// Function to convert and pack a single component
DEVICE_INLINE uint32_t
convertAndPack(float component, float inv_scale, float shift = 0.0) {
  // auto val = (component - shift) * inv_scale;
  auto val = fmaf(component, inv_scale, -shift * inv_scale);
  val = fmaxf(val, -FP8_E4M3_MAX::value);
  val = fminf(val, FP8_E4M3_MAX::value);
  auto x = __hip_fp8_e4m3_fnuz(val);
  return *reinterpret_cast<uint32_t*>(&x);
}
// Function to pack four components into a single uint32_t
DEVICE_INLINE uint32_t packComponents(uint32_t x_bits[4]) {
  uint32_t packed = 0;
  packed |= (x_bits[0] << 0);
  packed |= (x_bits[1] << 8);
  packed |= (x_bits[2] << 16);
  packed |= (x_bits[3] << 24);
  return packed;
}

__global__ void quantizeQKVPerHead(
    const float* xqkv_amax_head, // [B, HH]
    at::BFloat16* xqkv, // [B_T, HH, D_H]
    const int32_t* varseq_seqpos, // [B_T]
    const int32_t* varseq_batch, // [B_T]
    const bool* is_precalculated_qparam, // [B_T]
    at::PackedTensorAccessor64<at::Float8_e4m3fn, 3, at::RestrictPtrTraits>
        XQ_O, // [B_T][N_H][D]
    at::PackedTensorAccessor64<at::Float8_e4m3fn, 4, at::RestrictPtrTraits>
        cache_K, // [B][MAX_T][N_KVH][D_H]
    at::PackedTensorAccessor64<at::Float8_e4m3fn, 4, at::RestrictPtrTraits>
        cache_V, // [B][MAX_T][N_KVH][D_H]
    float* const scale_q,
    float* const scale_k,
    float* const scale_v,
    float kv_multiplier = 64.f) {
  // Launch one warp per token. Each thread handles 4 elements.
  // warps = B_T
  auto N_KVH = cache_K.size(2);
  auto N_H = XQ_O.size(1);
  auto B_T = XQ_O.size(0);
  // TODO: Support N_KVH > 1
  // CUDA_KERNEL_ASSERT(N_KVH == 1);

  auto HH = N_H + N_KVH * 2;
  auto maxHH = scale_k ? HH : N_H;

  uint2 buffer;

  // warps_per_block = blockDim.y
  // warp_id = threadIdx.y
  // block_id = blockIdx.x

  // Calculate scaling factor
  constexpr float min_scaling_factor = 1.0f / (FP8_E4M3_MAX::value * 512.f);
  int b = 0;
  int last_b = -1;
  int h = 0;
  float* qparam = nullptr;
  at::Float8_e4m3fn* dst_row_q = nullptr;
  float val = 0;
  float inv_scale = 0;

  uint d = 4 * threadIdx.x;

  auto b_t_start = blockIdx.x * blockDim.y + threadIdx.y;
  for (int b_t = b_t_start; b_t < B_T; b_t += blockDim.y * gridDim.x) {
    b = varseq_batch ? varseq_batch[b_t] : b_t;
    if (b_t > 0) {
      last_b = varseq_batch ? varseq_batch[b_t - 1] : b_t - 1;
    } else {
      last_b = -1;
    }
    {
      // Skip quantization of KV if scale is pre-calculated for K/V
      // as in decode and partial prefill cases
      bool is_precalculated_qparam_b_t =
          is_precalculated_qparam ? is_precalculated_qparam[b_t] : true;
      if (is_precalculated_qparam_b_t)
        maxHH = N_H;
    }
    val = 0;
    for (auto hh = 0; hh < N_H; hh++) {
      val = fmaxf(val, xqkv_amax_head[b * HH + hh]);
    }

    for (auto hh = 0; hh < maxHH; hh++) {
      {
        at::BFloat16* src_row = &xqkv[(b_t * HH + hh + 0) * D_H];
        buffer = *reinterpret_cast<uint2*>(&src_row[d]);
        val = (hh < N_H) ? val : xqkv_amax_head[b * HH + hh];
      }

      {
        int seqpos_t = varseq_seqpos[b_t];
        if (hh < N_H) {
          h = hh;
          qparam = scale_q + b * N_KVH + hh / (N_H / N_KVH);
          dst_row_q = &XQ_O[b_t][h][0];
          val = val * 8;
        } else if (hh < N_H + N_KVH) {
          h = hh - N_H;

          qparam = scale_k + b * N_KVH + h;
          dst_row_q = &cache_K[b][seqpos_t][h][0];
          val = kv_multiplier * val;
        } else {
          h = hh - N_H - N_KVH;

          qparam = scale_v + b * N_KVH + h;
          dst_row_q = &cache_V[b][seqpos_t][h][0];
          val = kv_multiplier * val;
        }
      }
      {
        float scale = 0;
        val = fminf(val, 12000);
        scale = fmaxf(val / FP8_E4M3_MAX::value, min_scaling_factor);
        bool is_first_token = b != last_b;
        if (threadIdx.x == 0 && h == 0 && is_first_token) {
          *qparam = scale;
        }
        inv_scale = 1 / scale;
      }

      {
        bfx4 src;
        fx4 dst;
        uint32_t x_bits[4];
        // Convert and pack data
        // 8 bytes are 4 elements of type bf16
        *reinterpret_cast<uint2*>(&src) = buffer;
        dst = bfx4_to_fx4(src);
        x_bits[0] = convertAndPack(dst.x, inv_scale);
        x_bits[1] = convertAndPack(dst.y, inv_scale);
        x_bits[2] = convertAndPack(dst.z, inv_scale);
        x_bits[3] = convertAndPack(dst.w, inv_scale);
        uint32_t packed = packComponents(x_bits);
        // CUDA_KERNEL_ASSERT(uintptr_t(&dst_row_q[d]) % 4 == 0);
        *reinterpret_cast<uint32_t*>(&dst_row_q[d]) = packed;
      }
    }
  }
}

at::Tensor quantize_qkv_per_head(
    at::Tensor xqkv_amax_row, // [B, HH]
    at::Tensor xqkv, // [B_T, HH, D_H]
    at::Tensor varseq_seqpos, // [B_T]
    std::optional<at::Tensor> varseq_batch, // [B_T]
    std::optional<at::Tensor> is_precalculated_qparam, // [B_T]
    at::Tensor cache_K, // [B][MAX_T][N_KVH][D_H]
    at::Tensor cache_V, // [B][MAX_T][N_KVH][D_H]
    at::Tensor XQ_O, // [B_T][N_H][D]
    int64_t B, // Batch size
    std::optional<at::Tensor> qparam_k = std::nullopt,
    std::optional<at::Tensor> qparam_v = std::nullopt) {
  auto N_KVH_L = cache_K.size(2);

  float* qparam_k_ptr = nullptr;
  float* qparam_v_ptr = nullptr;
  if (qparam_k.has_value()) {
    // prefill case
    qparam_k_ptr = qparam_k.value().data_ptr<float>();
    qparam_v_ptr = qparam_v.value().data_ptr<float>();
  }

  constexpr int32_t kMaxBlocks = 512;
  dim3 block_size(kThreadsPerWarp, kWarpsPerBlock);
  dim3 grid_size(kMaxBlocks);
  auto scale_q = at::zeros({B, N_KVH_L}, XQ_O.options().dtype(at::kFloat));
  float* const scale_q_ptr = scale_q.data_ptr<float>();
  // Launch the kernel
  quantizeQKVPerHead<<<
      grid_size,
      block_size,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      xqkv_amax_row.data_ptr<float>(),
      xqkv.data_ptr<at::BFloat16>(),
      varseq_seqpos.data_ptr<int32_t>(),
      varseq_batch.has_value() ? varseq_batch.value().data_ptr<int32_t>()
                               : nullptr, // not needed for decode
      is_precalculated_qparam.has_value()
          ? is_precalculated_qparam.value().data_ptr<bool>()
          : nullptr,
      XQ_O.packed_accessor64<at::Float8_e4m3fn, 3, at::RestrictPtrTraits>(),
      cache_K.packed_accessor64<at::Float8_e4m3fn, 4, at::RestrictPtrTraits>(),
      cache_V.packed_accessor64<at::Float8_e4m3fn, 4, at::RestrictPtrTraits>(),
      scale_q_ptr,
      qparam_k_ptr,
      qparam_v_ptr);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return scale_q;
}

template <typename T, KVQuantRecipe recipe>
DEVICE_INLINE void
quantize_fp8_kv(fx4 dst, T* dst_row_q, __half2* qparam, bool do_norm) {
  if (do_norm) {
    per_row_norm(dst);
  }
  float shift, scale;
  int32_t fp8_qparam_offset = 0;
  if (qparam == nullptr) {
    fp8_qparam_offset = 4;
  }
  if (recipe == KVQuantRecipe::perTokenScaling) {
    auto thread_min = fminf(fminf(fminf(dst.x, dst.y), dst.z), dst.w);
    auto thread_max = fmaxf(fmaxf(fmaxf(dst.x, dst.y), dst.z), dst.w);

    float warp_min, warp_max;
    unsigned mask = ballot_sync(4 * threadIdx.x < D_H, 0xFFFFFFFF);
    warp_min = -warpReduceMax(-thread_min, mask);
    warp_max = warpReduceMax(thread_max, mask);

    auto bounded_max = (warp_max - warp_min) / 2;
    // max FP16 value is 65504.0f.
    // Divide by 2 to avoid overflow during
    // e4m3fn (NV) to e4m3fnuz (AMD) conversion
    const float scale_ub = 65500.0f / 2;
    constexpr float min_scaling_factor = 1.0f / (FP8_E4M3_MAX::value * 512.f);
    bounded_max = std::min(bounded_max, scale_ub);
    scale = static_cast<float>(
        std::max(bounded_max / FP8_E4M3_MAX::value, min_scaling_factor));
    shift = warp_min + FP8_E4M3_MAX::value * scale;
  } else {
    // Support of per-head scaling is limited to reading a
    // pre-calculated scale from qparam tensor and using it for scaling the
    // row.
    scale = *reinterpret_cast<float*>(qparam);
    shift = 0.0f;
  }
  float inv_scale = 1 / scale;
  uint32_t x_bits[4];
  x_bits[0] = convertAndPack(dst.x, inv_scale, shift);
  x_bits[1] = convertAndPack(dst.y, inv_scale, shift);
  x_bits[2] = convertAndPack(dst.z, inv_scale, shift);
  x_bits[3] = convertAndPack(dst.w, inv_scale, shift);
  uint32_t packed = packComponents(x_bits);
  CUDA_KERNEL_ASSERT(
      uintptr_t(&dst_row_q[4 * threadIdx.x + fp8_qparam_offset]) % 4 == 0);

  *reinterpret_cast<uint32_t*>(
      &dst_row_q[4 * threadIdx.x + fp8_qparam_offset]) = packed;
  // write qparams
  if (threadIdx.x == 0 && recipe == KVQuantRecipe::perTokenScaling) {
    __half2* param_store = qparam;
    if (param_store == nullptr) {
      // If no external qparam, store the params at beginning of the quantized
      // cache.
      param_store = reinterpret_cast<__half2*>(&dst_row_q[0]);
    }
    CUDA_KERNEL_ASSERT(uintptr_t(param_store) % 4 == 0);
    *param_store = __floats2half2_rn(scale, shift);
  }
}
#else
template <typename T, KVQuantRecipe recipe>
DEVICE_INLINE void
quantize_fp8_kv(fx4 dst, T* dst_row_, __half2* qparam, bool do_norm) {}
std::vector<at::Tensor> quantize_fp8_per_tensor(
    at::Tensor input,
    std::optional<at::Tensor> bs, // batch size
    std::optional<at::Tensor> scale_ub) { // scale upperbound
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

std::tuple<at::Tensor, at::Tensor> dequantize_fp8_cache(
    at::Tensor cache_K,
    at::Tensor cache_V,
    at::Tensor kv_seqlen,
    std::optional<at::Tensor> qparam_k,
    std::optional<at::Tensor> qparam_v,
    std::optional<at::Tensor> block_tables,
    int64_t page_size) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}

at::Tensor quantize_qkv_per_head(
    at::Tensor xqkv_amax_row, // [B_T, HH]
    at::Tensor xqkv, // [B_T, HH, D_H]
    at::Tensor varseq_seqpos, // [B_T]
    std::optional<at::Tensor> varseq_batch, // [B_T]
    at::Tensor q_seqstarts, // [B+1]
    at::Tensor cache_K, // [B][MAX_T][N_KVH][D_H]
    at::Tensor cache_V, // [B][MAX_T][N_KVH][D_H]
    at::Tensor XQ_O, // [B_T][N_H][D]
    int64_t max_seq_length, // Length of the sequence
    std::optional<at::Tensor> qparam_k,
    std::optional<at::Tensor> qparam_v) {
  throw std::runtime_error(
      "CUDA version is older than 12.0"); // requires CUDA>=12
}
#endif

} // namespace fbgemm_gpu
