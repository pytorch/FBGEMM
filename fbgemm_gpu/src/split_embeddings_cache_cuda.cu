#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/TensorUtils.h>
#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <THC/THCAtomics.cuh>
#include <limits>
#include <mutex>
#include "cub/device/device_radix_sort.cuh"
#include "cub/device/device_run_length_encode.cuh"
#include "cub/device/device_select.cuh"
#include "fbgemm_gpu/dispatch_macros.h"
#include "fbgemm_gpu/fbgemm_cuda_utils.cuh"

#include "split_embeddings_utils.cuh"

constexpr size_t kCacheMaxThreads = 512;

using namespace at;
using namespace fbgemm_gpu;

// TODO: do we care about 64-bit indices? Currently we just ignore.
__host__ DEVICE_INLINE uint32_t cache_slot(int32_t h_in, int32_t C) {
  // MurmorHash3 32-bit mixing function.
  uint32_t h = (uint32_t)h_in;
  h ^= h >> 16;
  h *= 0x85ebca6b;
  h ^= h >> 13;
  h *= 0xc2b2ae35;
  h ^= h >> 16;
  // https://lemire.me/blog/2016/06/27/a-fast-alternative-to-the-modulo-reduction/
  return ((uint64_t)h * (uint64_t)C) >> 32;
}

__host__ DEVICE_INLINE uint32_t cache_slot(int64_t h_in, int32_t C) {
  // MurmurHash3 64-bit mixing function.
  uint64_t h = (uint64_t)h_in;
  h ^= h >> 33;
  h *= 0xff51afd7ed558ccd;
  h ^= h >> 33;
  h *= 0xc4ceb9fe1a85ec53;
  h ^= h >> 33;

  return h % (uint32_t)C;
}

int64_t host_lxu_cache_slot(int64_t h_in, int64_t C) {
  return static_cast<int64_t>(cache_slot(h_in, static_cast<int32_t>(C)));
}

constexpr int32_t kCacheLocationMissing = -1;
constexpr int64_t kCacheStateInvalid = -1;

template <typename emb_t, typename cache_t>
__global__ __launch_bounds__(kMaxThreads) void lxu_cache_flush_kernel(
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> weights,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        cache_hash_size_cumsum,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        cache_index_table_map,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> weights_offsets,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> D_offsets,
    PackedTensorAccessor32<int64_t, 2, RestrictPtrTraits> lxu_cache_state,
    PackedTensorAccessor64<cache_t, 2, RestrictPtrTraits> lxu_cache_weights,
    bool stochastic_rounding,
    PhiloxCudaState stochastic_rounding_philox_args) {
  int32_t B = lxu_cache_weights.size(0);
  int32_t b = blockIdx.x * blockDim.y + threadIdx.y;
  if (b >= B) {
    return;
  }
  int32_t slot = b % kWarpSize;
  int32_t cache_set = b / kWarpSize;
  int64_t current_idx = lxu_cache_state[cache_set][slot];
  if (current_idx != static_cast<int64_t>(kCacheStateInvalid)) {
    // evict from slot to backing storage
    int32_t t_current = cache_index_table_map[current_idx];
    int64_t idx_current = current_idx - cache_hash_size_cumsum[t_current];
    int64_t weights_offset_current = weights_offsets[t_current];
    int32_t D_start_current = D_offsets[t_current];
    int32_t D_end_current = D_offsets[t_current + 1];
    int32_t D_current = D_end_current - D_start_current;

    int32_t D_emb = D_current;
    if (std::is_same<emb_t, uint8_t>::value) {
      D_emb += kINT8QparamsBytes;
    }
    auto weight_row = WeightRow<emb_t, cache_t, acc_type<cache_t, true>>(
        &weights[weights_offset_current + idx_current * D_emb + 0],
        &lxu_cache_weights[b][0],
        D_current,
        nullptr);
    if (!std::is_same<emb_t, float>::value && stochastic_rounding) {
      StochasticRoundingRNGState state;
      // different for every *run* and every *thread*.
      auto stochastic_rounding_seeds =
          at::cuda::philox::unpack(stochastic_rounding_philox_args);
      stochastic_rounding_init(
          std::get<0>(stochastic_rounding_seeds) ^
              std::get<1>(stochastic_rounding_seeds),
          blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
              threadIdx.x,
          &state);
      weight_row.set_stoc_state(&state);
    }

    float2 qparams;
    if (std::is_same<emb_t, uint8_t>::value) {
      qparams =
          thrust_find_qparams<cache_t>(&lxu_cache_weights[b][0], D_current);
      if (threadIdx.x == 0) {
        weight_row.store_qparams(qparams);
      }
    }
    for (int32_t d = threadIdx.x; d * 4 < D_current; d += blockDim.x) {
      Vec4T<acc_type<cache_t, true>> cache_weights_vec =
          weight_row.load(d * 4, qparams);
      weight_row.evict(cache_weights_vec, d * 4, qparams);
    }
  }
}

void lxu_cache_flush_cuda(
    Tensor uvm_weights,
    Tensor cache_hash_size_cumsum,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    int64_t total_D,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    bool stochastic_rounding) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(lxu_cache_weights.get_device());

  int32_t T = D_offsets.numel() - 1;
  int32_t S = lxu_cache_weights.size(0);
  int32_t tx = std::min<int32_t>(total_D / 4 / T, kMaxThreads);
  dim3 threads(tx, kMaxThreads / tx);
  dim3 blocks(div_round_up(S, kMaxThreads / tx));

  DISPATCH_EMB_CACHE_TYPES(
      uvm_weights.type(),
      lxu_cache_weights.type(),
      "lxu_cache_flush_kernel_2",
      ([&] {
        PhiloxCudaState rng_engine_inputs;
        if (stochastic_rounding && std::is_same<emb_t, Half>::value) {
          auto gen = at::cuda::detail::getDefaultCUDAGenerator();
          std::lock_guard<std::mutex> lock(gen.mutex());
          rng_engine_inputs = at::check_generator<at::CUDAGeneratorImpl>(gen)
                                  ->philox_cuda_state(4);
        }
        lxu_cache_flush_kernel<emb_t, cache_t>
            <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                uvm_weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                cache_hash_size_cumsum
                    .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                cache_index_table_map
                    .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                weights_offsets
                    .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                D_offsets.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                lxu_cache_state
                    .packed_accessor32<int64_t, 2, RestrictPtrTraits>(),
                lxu_cache_weights
                    .packed_accessor64<cache_t, 2, RestrictPtrTraits>(),
                stochastic_rounding,
                rng_engine_inputs);
      }));
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return;
}

__global__ __launch_bounds__(kMaxThreads) void linearize_cache_indices_kernel(
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        cache_hash_size_cumsum,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> indices,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> offsets,
    PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        linear_cache_indices) {
  int32_t T = cache_hash_size_cumsum.size(0) - 1;
  int64_t total_cache_hash_size = cache_hash_size_cumsum[T];
  int32_t B = (offsets.size(0) - 1) / T;
  int32_t b_t = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t b = b_t % B;
  int32_t t = b_t / B;
  bool valid = t < T;

  int64_t hash_offset = valid ? cache_hash_size_cumsum[t] : -1;
  int64_t indices_start = valid ? offsets[t * B + b] : -1;
  int32_t L = valid ? offsets[t * B + b + 1] - indices_start : 0;
  int32_t lane_id = threadIdx.x % kWarpSize;

  // hash_offset < 0 for non-caching tables
  for (int32_t j = 0; j < kWarpSize; ++j) {
    int64_t indices_start_warp = __shfl_sync(0xFFFFFFFF, indices_start, j);
    int32_t L_warp = __shfl_sync(0xFFFFFFFF, L, j);
    int64_t hash_offset_warp = __shfl_sync(0xFFFFFFFF, hash_offset, j);
    if (hash_offset_warp >= 0) {
      for (int32_t i = lane_id; i < L_warp; i += kWarpSize) {
        auto idx = __ldg(&indices[indices_start_warp + i]);
        linear_cache_indices[indices_start_warp + i] = hash_offset_warp + idx;
      }
    } else {
      for (int32_t i = lane_id; i < L_warp; i += kWarpSize) {
        linear_cache_indices[indices_start_warp + i] = total_cache_hash_size;
      }
    }
  }
}

Tensor linearize_cache_indices_cuda(
    Tensor cache_hash_size_cumsum,
    Tensor indices,
    Tensor offsets) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(cache_hash_size_cumsum.get_device());

  auto T = cache_hash_size_cumsum.size(0) - 1;
  TORCH_CHECK(T > 0);
  // offsets = [B x T  + 1]
  auto B = (offsets.size(0) - 1) / T;
  TORCH_CHECK(B >= 0);

  auto linear_cache_indices = at::empty_like(indices);
  if (B == 0) {
    return linear_cache_indices;
  }
  linearize_cache_indices_kernel<<<
      div_round_up(B * T, kMaxThreads),
      kMaxThreads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      cache_hash_size_cumsum.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
      indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
      offsets.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
      linear_cache_indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return linear_cache_indices;
}

std::tuple<Tensor, Tensor, c10::optional<Tensor>> get_unique_indices_cuda(
    Tensor linear_indices,
    int64_t max_indices,
    bool compute_count) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(linear_indices.get_device());

  TORCH_CHECK(linear_indices.numel() < std::numeric_limits<int32_t>::max());
  int32_t N = linear_indices.numel();
  auto sorted_indices = at::empty_like(linear_indices);
  auto unique_indices = at::empty_like(linear_indices);
  auto unique_indices_length =
      at::empty({1}, linear_indices.options().dtype(kInt));
  c10::optional<Tensor> unique_indices_count = c10::nullopt;
  if (compute_count) {
    unique_indices_count = at::empty(
        {linear_indices.numel()}, linear_indices.options().dtype(kInt));
  }

  // sort indices
  size_t temp_storage_bytes_0 = 0;
  AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortKeys(
      nullptr,
      temp_storage_bytes_0,
      linear_indices.data_ptr<int64_t>(),
      sorted_indices.data_ptr<int64_t>(),
      N,
      0,
      int(log2(float(max_indices + 1)) + 1),
      at::cuda::getCurrentCUDAStream(),
      false));
  auto temp_storage_0 = at::empty(
      {static_cast<int64_t>(temp_storage_bytes_0)},
      linear_indices.options().dtype(kByte));
  AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortKeys(
      temp_storage_0.data_ptr(),
      temp_storage_bytes_0,
      linear_indices.data_ptr<int64_t>(),
      sorted_indices.data_ptr<int64_t>(),
      N,
      0,
      int(log2(float(max_indices + 1)) + 1),
      at::cuda::getCurrentCUDAStream(),
      false));
  // get unique indices
  if (compute_count) {
    size_t temp_storage_bytes_1 = 0;
    AT_CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
        nullptr,
        temp_storage_bytes_1,
        sorted_indices.data_ptr<int64_t>(),
        unique_indices.data_ptr<int64_t>(),
        unique_indices_count->data_ptr<int32_t>(),
        unique_indices_length.data_ptr<int32_t>(),
        N,
        at::cuda::getCurrentCUDAStream(),
        false));
    auto temp_storage_1 = at::empty(
        {static_cast<int64_t>(temp_storage_bytes_1)},
        linear_indices.options().dtype(kByte));
    AT_CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
        temp_storage_1.data_ptr(),
        temp_storage_bytes_1,
        sorted_indices.data_ptr<int64_t>(),
        unique_indices.data_ptr<int64_t>(),
        unique_indices_count->data_ptr<int32_t>(),
        unique_indices_length.data_ptr<int32_t>(),
        N,
        at::cuda::getCurrentCUDAStream(),
        false));
  } else {
    size_t temp_storage_bytes_1 = 0;
    AT_CUDA_CHECK(hipcub::DeviceSelect::Unique(
        nullptr,
        temp_storage_bytes_1,
        sorted_indices.data_ptr<int64_t>(),
        unique_indices.data_ptr<int64_t>(),
        unique_indices_length.data_ptr<int32_t>(),
        N,
        at::cuda::getCurrentCUDAStream(),
        false));
    auto temp_storage_1 = at::empty(
        {static_cast<int64_t>(temp_storage_bytes_1)},
        linear_indices.options().dtype(kByte));
    AT_CUDA_CHECK(hipcub::DeviceSelect::Unique(
        temp_storage_1.data_ptr(),
        temp_storage_bytes_1,
        sorted_indices.data_ptr<int64_t>(),
        unique_indices.data_ptr<int64_t>(),
        unique_indices_length.data_ptr<int32_t>(),
        N,
        at::cuda::getCurrentCUDAStream(),
        false));
  }
  return std::make_tuple(
      unique_indices, unique_indices_length, unique_indices_count);
}

__global__ __launch_bounds__(kMaxThreads) void lru_cache_find_uncached_kernel(
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> unique_indices,
    const int32_t* __restrict__ N_unique,
    int64_t max_indices,
    const PackedTensorAccessor32<int64_t, 2, RestrictPtrTraits> lxu_cache_state,
    PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> cache_sets,
    int64_t time_stamp,
    PackedTensorAccessor32<int64_t, 2, RestrictPtrTraits> lru_state) {
  int32_t N = unique_indices.size(0);
  int32_t C = lxu_cache_state.size(0);

  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= N) {
    return;
  }
  if (n >= *N_unique) {
    if (threadIdx.x == 0) {
      cache_sets[n] = C; // invalid index, used as sentinel
    }
    return;
  }
  int64_t idx = unique_indices[n];
  if (idx == max_indices) {
    if (threadIdx.x == 0) {
      cache_sets[n] = C; // invalid index, used as sentinel
    }
    return;
  }
  int32_t cache_set = cache_slot(idx, C);

  auto slot = threadIdx.x;
  bool found = __ldg((&lxu_cache_state[cache_set][0]) + slot) == idx;
  if (found) {
    // mark it as existing.
    cache_sets[n] = C; // invalid index, used as sentinel
    // mark it as recently accessed so we don't evict.
    lru_state[cache_set][slot] = time_stamp;
  }

  if (!__any_sync(0xFFFFFFFF, found)) {
    if (threadIdx.x == 0) {
      cache_sets[n] = cache_set;
    }
  }
}

std::pair<Tensor, Tensor> lru_cache_find_uncached_cuda(
    Tensor unique_indices,
    Tensor unique_indices_length,
    int64_t max_indices,
    Tensor lxu_cache_state,
    int64_t time_stamp,
    Tensor lru_state) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(unique_indices.get_device());

  auto cache_sets =
      empty_like(unique_indices, unique_indices.options().dtype(kInt));
  int32_t N = unique_indices.numel();
  auto sorted_cache_sets = empty_like(cache_sets);
  auto cache_set_sorted_unique_indices = empty_like(unique_indices);

  // Find uncached indices
  lru_cache_find_uncached_kernel<<<
      div_round_up(N, kMaxThreads / kWarpSize),
      dim3(kWarpSize, kMaxThreads / kWarpSize),
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      unique_indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
      unique_indices_length.data_ptr<int32_t>(),
      max_indices,
      lxu_cache_state.packed_accessor32<int64_t, 2, RestrictPtrTraits>(),
      cache_sets.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
      time_stamp,
      lru_state.packed_accessor32<int64_t, 2, RestrictPtrTraits>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  // Sort the cache sets and ids
  size_t temp_storage_bytes = 0;
  AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
      nullptr,
      temp_storage_bytes,
      cache_sets.data_ptr<int32_t>(),
      sorted_cache_sets.data_ptr<int32_t>(),
      unique_indices.data_ptr<int64_t>(),
      cache_set_sorted_unique_indices.data_ptr<int64_t>(),
      N,
      0,
      int(log2(float(lxu_cache_state.size(0) + 1)) + 1),
      at::cuda::getCurrentCUDAStream(),
      false));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      unique_indices.options().dtype(kByte));
  AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
      temp_storage.data_ptr(),
      temp_storage_bytes,
      cache_sets.data_ptr<int32_t>(),
      sorted_cache_sets.data_ptr<int32_t>(),
      unique_indices.data_ptr<int64_t>(),
      cache_set_sorted_unique_indices.data_ptr<int64_t>(),
      N,
      0,
      int(log2(float(lxu_cache_state.size(0) + 1)) + 1),
      at::cuda::getCurrentCUDAStream(),
      false));
  return {sorted_cache_sets, cache_set_sorted_unique_indices};
}

template <typename emb_t, typename cache_t>
__global__ __launch_bounds__(kMaxThreads) void lru_cache_insert_kernel(
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> weights,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        cache_hash_size_cumsum,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        cache_index_table_map,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> weights_offsets,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> D_offsets,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        sorted_cache_sets,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        cache_set_sorted_indices,
    const int32_t* __restrict__ N_unique,
    PackedTensorAccessor32<int64_t, 2, RestrictPtrTraits> lxu_cache_state,
    PackedTensorAccessor64<cache_t, 2, RestrictPtrTraits> lxu_cache_weights,
    int64_t time_stamp,
    PackedTensorAccessor32<int64_t, 2, RestrictPtrTraits> lru_state,
    bool stochastic_rounding,
    PhiloxCudaState stochastic_rounding_philox_args) {
  int32_t C = lxu_cache_state.size(0);
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= *N_unique) {
    return;
  }
  // check if this warp is responsible for this whole segment.
  bool segment_start =
      (n == 0 || sorted_cache_sets[n - 1] != sorted_cache_sets[n]);

  if (!segment_start) {
    // don't have *warp* divergence since we launch full warps in blockDim.x,
    // so we can just exit this warp entirely.
    return;
  }
  int32_t cache_set = sorted_cache_sets[n];
  if (cache_set == C) {
    // ignore the already-existing elements
    return;
  }

  int32_t SL = 1;
  while (n + SL < *N_unique && sorted_cache_sets[n + SL] == cache_set) {
    SL += 1;
  }

  // now, we need to insert the (unique!) values in indices[n:n + SL] into
  // our slots.
  int32_t slot = threadIdx.x;
  int64_t slot_time = lru_state[cache_set][slot];
  int64_t costs[1] = {slot_time};
  int32_t slots[1] = {slot};

  BitonicSort<int64_t, int32_t, 1, Comparator<int64_t>>::sort(costs, slots);
  int32_t sorted_slot = slots[0];
  int64_t sorted_lru_cost = costs[0];

  for (int32_t l = 0; l < min(SL, kWarpSize); ++l) {
    int32_t insert_slot = __shfl_sync(0xFFFFFFFF, sorted_slot, l);
    int64_t insert_current_lru_cost =
        __shfl_sync(0xFFFFFFFF, sorted_lru_cost, l);
    if (insert_current_lru_cost == time_stamp) {
      return;
    }
    int64_t insert_idx = cache_set_sorted_indices[n + l];
    int32_t t_insert = cache_index_table_map[insert_idx];
    int64_t idx_insert = insert_idx - cache_hash_size_cumsum[t_insert];
    int64_t weights_offset_insert = weights_offsets[t_insert];
    int32_t D_start_insert = D_offsets[t_insert];
    int32_t D_end_insert = D_offsets[t_insert + 1];
    int32_t D_insert = D_end_insert - D_start_insert;

    // ensure that threadIdx.x is the only thread reading/writing to
    // lxu_cache_state
    int64_t current_idx =
        threadIdx.x == 0 ? lxu_cache_state[cache_set][insert_slot] : 0;
    current_idx = __shfl_sync(0xFFFFFFFF, current_idx, 0);

    // not empty
    if (current_idx != static_cast<int64_t>(kCacheStateInvalid)) {
      // evict from slot to backing storage
      int32_t t_current = cache_index_table_map[current_idx];
      int64_t idx_current = current_idx - cache_hash_size_cumsum[t_current];
      int64_t weights_offset_current = weights_offsets[t_current];
      int32_t D_start_current = D_offsets[t_current];
      int32_t D_end_current = D_offsets[t_current + 1];
      int32_t D_current = D_end_current - D_start_current;
      int32_t D_emb = D_current;
      if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
      }
      auto weight_row = WeightRow<emb_t, cache_t, cache_t>(
          &weights[weights_offset_current + idx_current * D_emb + 0],
          &lxu_cache_weights[cache_set * kWarpSize + insert_slot][0],
          D_current,
          nullptr);
      if (!std::is_same<emb_t, float>::value && stochastic_rounding) {
        StochasticRoundingRNGState state;
        // different for every *run* and every *thread*.
        auto stochastic_rounding_seeds =
            at::cuda::philox::unpack(stochastic_rounding_philox_args);
        stochastic_rounding_init(
            std::get<0>(stochastic_rounding_seeds) ^
                std::get<1>(stochastic_rounding_seeds),
            (blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
             threadIdx.x) *
                    kWarpSize +
                l,
            &state);
        weight_row.set_stoc_state(&state);
      }
      float2 qparams;
      acc_type<cache_t, true> local_min =
          std::numeric_limits<acc_type<cache_t, true>>::max();
      acc_type<cache_t, true> local_max =
          std::numeric_limits<acc_type<cache_t, true>>::lowest();
      if (std::is_same<emb_t, uint8_t>::value) {
        for (int32_t d = threadIdx.x; d * 4 < D_current; d += blockDim.x) {
          Vec4T<cache_t> cache_weights_vec =
              weight_row.load(d * 4, qparams); // qparams not used
          local_max = max(local_max, vec4_max(cache_weights_vec));
          local_min = min(local_min, vec4_min(cache_weights_vec));
        }
        qparams = warp_find_qparams(local_min, local_max);
        if (threadIdx.x == 0) {
          weight_row.store_qparams(qparams);
        }
      }
      for (int32_t d = threadIdx.x; d * 4 < D_current; d += blockDim.x) {
        Vec4T<cache_t> cache_weights_vec = weight_row.load(d * 4, qparams);
        weight_row.evict(
            cache_weights_vec, d * 4, qparams); // FP32 -> FP16/FP32
      }
    }
    int32_t D_emb = D_insert;
    if (std::is_same<emb_t, uint8_t>::value) {
      D_emb += kINT8QparamsBytes;
    }
    // insert into cache
    auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
        &weights[weights_offset_insert + idx_insert * D_emb + 0],
        &lxu_cache_weights[cache_set * kWarpSize + insert_slot][0],
        D_insert,
        nullptr);

    auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
        &weights[weights_offset_insert + idx_insert * D_emb + 0],
        nullptr,
        D_insert,
        nullptr);

    float2 qparams;
    if (std::is_same<emb_t, uint8_t>::value) {
      qparams = weight_row_emb.load_qparams();
    }
    for (int32_t d = threadIdx.x; d * 4 < D_insert; d += blockDim.x) {
      auto row = weight_row_emb.load(d * 4, qparams);
      weight_row_cache.store(row, d * 4, qparams);
    }
    if (threadIdx.x == 0) {
      lxu_cache_state[cache_set][insert_slot] = insert_idx;
      lru_state[cache_set][insert_slot] = time_stamp;
    }
  }
}

void lru_cache_insert_cuda(
    Tensor weights,
    Tensor cache_hash_size_cumsum,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    Tensor sorted_cache_sets,
    Tensor cache_set_sorted_unique_indices,
    Tensor unique_indices_length,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    int64_t time_stamp,
    Tensor lru_state,
    bool stochastic_rounding) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(weights.get_device());

  int32_t N = cache_set_sorted_unique_indices.numel();

  DISPATCH_EMB_CACHE_TYPES(
      weights.type(),
      lxu_cache_weights.type(),
      "lru_cache_insert_kernel_2",
      ([&] {
        PhiloxCudaState rng_engine_inputs;
        if (stochastic_rounding && !std::is_same<emb_t, float>::value) {
          auto gen = at::cuda::detail::getDefaultCUDAGenerator();
          std::lock_guard<std::mutex> lock(gen.mutex());
          rng_engine_inputs = at::check_generator<at::CUDAGeneratorImpl>(gen)
                                  ->philox_cuda_state(4);
        }

        lru_cache_insert_kernel<emb_t, cache_t>
            <<<div_round_up(N, kMaxThreads / kWarpSize),
               dim3(kWarpSize, kMaxThreads / kWarpSize),
               0,
               at::cuda::getCurrentCUDAStream()>>>(
                weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                cache_hash_size_cumsum
                    .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                cache_index_table_map
                    .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                weights_offsets
                    .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                D_offsets.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                sorted_cache_sets
                    .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                cache_set_sorted_unique_indices
                    .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                unique_indices_length.data_ptr<int32_t>(),
                lxu_cache_state
                    .packed_accessor32<int64_t, 2, RestrictPtrTraits>(),
                lxu_cache_weights
                    .packed_accessor64<cache_t, 2, RestrictPtrTraits>(),
                time_stamp,
                lru_state.packed_accessor32<int64_t, 2, RestrictPtrTraits>(),
                stochastic_rounding,
                rng_engine_inputs);
      }));
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void lru_cache_populate_cuda(
    Tensor weights,
    Tensor cache_hash_size_cumsum,
    int64_t total_cache_hash_size,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    Tensor linear_cache_indices,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    int64_t time_stamp,
    Tensor lru_state,
    bool stochastic_rounding) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(weights.get_device());

  TORCH_CHECK(
      linear_cache_indices.numel() < std::numeric_limits<int32_t>::max());
  if (linear_cache_indices.numel() == 0) {
    // nothing to do
    return;
  }

  // Get unqiue indices
  Tensor unique_indices;
  Tensor unique_indices_length;
  c10::optional<Tensor> unique_indices_count;
  std::tie(unique_indices, unique_indices_length, unique_indices_count) =
      get_unique_indices_cuda(
          linear_cache_indices, total_cache_hash_size, false);

  // Find uncached indices
  auto cache_sets_and_unique_indices = lru_cache_find_uncached_cuda(
      unique_indices,
      unique_indices_length,
      total_cache_hash_size,
      lxu_cache_state,
      time_stamp,
      lru_state);
  auto sorted_cache_sets = cache_sets_and_unique_indices.first;
  auto cache_set_sorted_unique_indices = cache_sets_and_unique_indices.second;

  // insert caching weights
  lru_cache_insert_cuda(
      weights,
      cache_hash_size_cumsum,
      cache_index_table_map,
      weights_offsets,
      D_offsets,
      sorted_cache_sets,
      cache_set_sorted_unique_indices,
      unique_indices_length,
      lxu_cache_state,
      lxu_cache_weights,
      time_stamp,
      lru_state,
      stochastic_rounding);
}

__global__ __launch_bounds__(kMaxThreads) void lfu_update_counts_kernel(
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> unique_indices,
    const int32_t* __restrict__ N_unique,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        unique_indices_count,
    PackedTensorAccessor64<int64_t, 1, RestrictPtrTraits> lfu_state) {
  int32_t n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n >= *N_unique) {
    return;
  }
  int64_t idx = unique_indices[n];
  lfu_state[idx] += unique_indices_count[n];
}

void lfu_update_counts_cuda(
    Tensor unique_indices,
    Tensor unique_indices_length,
    Tensor unique_indices_count,
    Tensor lfu_state) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(unique_indices.get_device());

  int32_t N = unique_indices.size(0);
  lfu_update_counts_kernel<<<
      div_round_up(N, kMaxThreads),
      kMaxThreads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      unique_indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
      unique_indices_length.data_ptr<int32_t>(),
      unique_indices_count.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
      lfu_state.packed_accessor64<int64_t, 1, RestrictPtrTraits>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

constexpr int32_t kCacheSetBits = 24;
constexpr int32_t kLFUCounterBits = 40;
static_assert(kCacheSetBits + kLFUCounterBits == 8 * sizeof(int64_t), "");

__global__ __launch_bounds__(kMaxThreads) void lfu_cache_find_uncached_kernel(
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> unique_indices,
    const int32_t* __restrict__ N_unique,
    int64_t max_indices,
    const PackedTensorAccessor32<int64_t, 2, RestrictPtrTraits> lxu_cache_state,
    uint64_t* __restrict__ cache_sets,
    const PackedTensorAccessor64<int64_t, 1, RestrictPtrTraits> lfu_state) {
  int32_t N = unique_indices.size(0);
  int32_t C = lxu_cache_state.size(0);
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= N) {
    return;
  }
  if (n >= *N_unique) {
    if (threadIdx.x == 0) {
      cache_sets[n] =
          (static_cast<uint64_t>(C)
           << kLFUCounterBits); // invalid index, used as sentinel
    }
    return;
  }
  int64_t idx = unique_indices[n];
  if (idx == max_indices) {
    if (threadIdx.x == 0) {
      cache_sets[n] =
          (static_cast<uint64_t>(C)
           << kLFUCounterBits); // invalid index, used as sentinel
    }
    return;
  }
  uint32_t cache_set = cache_slot(idx, C);

  auto slot = threadIdx.x;
  bool found = __ldg((&lxu_cache_state[cache_set][0]) + slot) == idx;
  if (found) {
    // mark it as existing.
    cache_sets[n] =
        (static_cast<uint64_t>(C)
         << kLFUCounterBits); // invalid index, used as sentinel
  }

  if (!__any_sync(0xFFFFFFFF, found)) {
    if (threadIdx.x == 0) {
      // sort so the highest LFUs come first in the segment.
      // assume lfu_state[idx] <= 2^40 - 1 and cache_set < 2^24 -1
      cache_sets[n] = ((static_cast<uint64_t>(cache_set) << kLFUCounterBits)) |
          ((static_cast<uint64_t>(1) << kLFUCounterBits) - 1 - lfu_state[idx]);
    }
  }
}

std::pair<Tensor, Tensor> lfu_cache_find_uncached_cuda(
    Tensor unique_indices,
    Tensor unique_indices_length,
    int64_t max_indices,
    Tensor lxu_cache_state,
    Tensor lfu_state) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(unique_indices.get_device());

  auto cache_sets =
      empty_like(unique_indices, unique_indices.options().dtype(kLong));
  int32_t N = unique_indices.numel();
  auto sorted_cache_sets = empty_like(cache_sets);
  auto cache_set_sorted_unique_indices = empty_like(unique_indices);

  // Find uncached indices
  lfu_cache_find_uncached_kernel<<<
      div_round_up(N, kMaxThreads / kWarpSize),
      dim3(kWarpSize, kMaxThreads / kWarpSize),
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      unique_indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
      unique_indices_length.data_ptr<int32_t>(),
      max_indices,
      lxu_cache_state.packed_accessor32<int64_t, 2, RestrictPtrTraits>(),
      (uint64_t*)cache_sets.data_ptr<int64_t>(),
      lfu_state.packed_accessor64<int64_t, 1, RestrictPtrTraits>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  // Sort the cache sets and ids
  size_t temp_storage_bytes = 0;
  AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
      nullptr,
      temp_storage_bytes,
      (uint64_t*)cache_sets.data_ptr<int64_t>(),
      (uint64_t*)sorted_cache_sets.data_ptr<int64_t>(),
      unique_indices.data_ptr<int64_t>(),
      cache_set_sorted_unique_indices.data_ptr<int64_t>(),
      N,
      0,
      int(log2(float(lxu_cache_state.size(0) + 1)) + 1) + kLFUCounterBits,
      at::cuda::getCurrentCUDAStream(),
      false));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      unique_indices.options().dtype(kByte));
  AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
      temp_storage.data_ptr(),
      temp_storage_bytes,
      (uint64_t*)cache_sets.data_ptr<int64_t>(),
      (uint64_t*)sorted_cache_sets.data_ptr<int64_t>(),
      unique_indices.data_ptr<int64_t>(),
      cache_set_sorted_unique_indices.data_ptr<int64_t>(),
      N,
      0,
      int(log2(float(lxu_cache_state.size(0) + 1)) + 1) + kLFUCounterBits,
      at::cuda::getCurrentCUDAStream(),
      false));
  return {sorted_cache_sets, cache_set_sorted_unique_indices};
}

template <typename emb_t, typename cache_t>
__global__ __launch_bounds__(kCacheMaxThreads) void lfu_cache_insert_kernel(
    PackedTensorAccessor64<emb_t, 1, RestrictPtrTraits> weights,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        cache_hash_size_cumsum,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits>
        cache_index_table_map,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits> weights_offsets,
    const PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> D_offsets,
    const uint64_t* __restrict__ sorted_cache_sets,
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        cache_set_sorted_indices,
    const int32_t* __restrict__ N_unique,
    PackedTensorAccessor32<int64_t, 2, RestrictPtrTraits> lxu_cache_state,
    PackedTensorAccessor64<cache_t, 2, RestrictPtrTraits> lxu_cache_weights,
    const PackedTensorAccessor64<int64_t, 1, RestrictPtrTraits> lfu_state,
    bool stochastic_rounding,
    PhiloxCudaState stochastic_rounding_philox_args) {
  int32_t C = lxu_cache_state.size(0);
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= *N_unique) {
    return;
  }
  // check if this warp is responsible for this whole segment.
  bool segment_start =
      (n == 0 ||
       (sorted_cache_sets[n - 1] >> kLFUCounterBits) !=
           (sorted_cache_sets[n] >> kLFUCounterBits));

  if (!segment_start) {
    // don't have *warp* divergence since we launch full warps in blockDim.x,
    // so we can just exit this warp entirely.
    return;
  }
  uint32_t cache_set = (sorted_cache_sets[n] >> kLFUCounterBits);
  if (cache_set == C) {
    // ignore the already-existing elements
    return;
  }

  int32_t SL = 1;
  while (n + SL < *N_unique &&
         (sorted_cache_sets[n + SL] >> kLFUCounterBits) == cache_set) {
    SL += 1;
  }

  // now, we need to insert the (unique!) values in indices[n:n + SL] into
  // our slots.
  int32_t slot = threadIdx.x;
  int64_t current_idx = lxu_cache_state[cache_set][slot];
  int64_t current_lfu_cost =
      (current_idx != static_cast<int64_t>(kCacheStateInvalid))
      ? lfu_state[current_idx]
      : -1;
  int64_t costs[1] = {current_lfu_cost};
  int32_t slots[1] = {slot};

  BitonicSort<int64_t, int32_t, 1, Comparator<int64_t>>::sort(costs, slots);
  int32_t sorted_slot = slots[0];
  int64_t sorted_lfu_cost = costs[0];

  for (int32_t l = 0; l < min(SL, kWarpSize); ++l) {
    int32_t insert_slot = __shfl_sync(0xFFFFFFFF, sorted_slot, l);
    int64_t insert_current_lfu_cost =
        __shfl_sync(0xFFFFFFFF, sorted_lfu_cost, l);
    int64_t insert_idx = cache_set_sorted_indices[n + l];
    int64_t insert_lfu_cost = lfu_state[insert_idx];

    if (insert_current_lfu_cost > insert_lfu_cost) {
      // don't insert.
      // all subsequent `current_lfu_cost` values are greater, and all
      // subsequent `insert_lfu_cost` values are smaller, so we can exit
      // early here.
      return;
    }
    int32_t t_insert = cache_index_table_map[insert_idx];
    int64_t idx_insert = insert_idx - cache_hash_size_cumsum[t_insert];
    int64_t weights_offset_insert = weights_offsets[t_insert];
    int32_t D_start_insert = D_offsets[t_insert];
    int32_t D_end_insert = D_offsets[t_insert + 1];
    int32_t D_insert = D_end_insert - D_start_insert;

    // not empty
    if (insert_current_lfu_cost != -1) {
      // ensure that threadIdx.x is the only thread reading/writing to
      // lxu_cache_state
      int64_t current_idx =
          threadIdx.x == 0 ? lxu_cache_state[cache_set][insert_slot] : 0;
      current_idx = __shfl_sync(0xFFFFFFFF, current_idx, 0);
      int32_t t_current = cache_index_table_map[current_idx];
      int64_t idx_current = current_idx - cache_hash_size_cumsum[t_current];
      int64_t weights_offset_current = weights_offsets[t_current];
      int32_t D_start_current = D_offsets[t_current];
      int32_t D_end_current = D_offsets[t_current + 1];
      int32_t D_current = D_end_current - D_start_current;

      int32_t D_emb = D_current;
      if (std::is_same<emb_t, uint8_t>::value) {
        D_emb += kINT8QparamsBytes;
      }
      auto weight_row = WeightRow<emb_t, cache_t, cache_t>(
          &weights[weights_offset_current + idx_current * D_emb + 0],
          &lxu_cache_weights[cache_set * kWarpSize + insert_slot][0],
          D_current,
          nullptr);
      if (!std::is_same<emb_t, float>::value && stochastic_rounding) {
        StochasticRoundingRNGState state;
        // different for every *run* and every *thread*.
        auto stochastic_rounding_seeds =
            at::cuda::philox::unpack(stochastic_rounding_philox_args);
        stochastic_rounding_init(
            std::get<0>(stochastic_rounding_seeds) ^
                std::get<1>(stochastic_rounding_seeds),
            (blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
             threadIdx.x) *
                    kWarpSize +
                l,
            &state);
        weight_row.set_stoc_state(&state);
      }

      float2 qparams;
      acc_type<cache_t, true> local_min =
          std::numeric_limits<acc_type<cache_t, true>>::max();
      acc_type<cache_t, true> local_max =
          std::numeric_limits<acc_type<cache_t, true>>::lowest();
      if (std::is_same<emb_t, uint8_t>::value) {
        for (int32_t d = threadIdx.x; d * 4 < D_current; d += blockDim.x) {
          Vec4T<cache_t> cache_weights_vec =
              weight_row.load(d * 4, qparams); // qparams not used
          local_max = max(local_max, vec4_max(cache_weights_vec));
          local_min = min(local_min, vec4_min(cache_weights_vec));
        }
        qparams = warp_find_qparams(local_min, local_max);
        if (threadIdx.x == 0) {
          weight_row.store_qparams(qparams);
        }
      }
      for (int32_t d = threadIdx.x; d * 4 < D_current; d += blockDim.x) {
        Vec4T<cache_t> cache_weights_vec = weight_row.load(d * 4, qparams);
        weight_row.evict(cache_weights_vec, d * 4, qparams);
      }
    }
    // insert into cache
    int32_t D_emb = D_insert;
    if (std::is_same<emb_t, uint8_t>::value) {
      D_emb += kINT8QparamsBytes;
    }
    auto weight_row_cache = WeightRow<emb_t, cache_t, cache_t>(
        &weights[weights_offset_insert + idx_insert * D_emb + 0],
        &lxu_cache_weights[cache_set * kWarpSize + insert_slot][0],
        D_insert,
        nullptr);

    auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
        &weights[weights_offset_insert + idx_insert * D_emb + 0],
        nullptr,
        D_insert,
        nullptr);

    float2 qparams;
    if (std::is_same<emb_t, uint8_t>::value) {
      qparams = weight_row_emb.load_qparams();
    }
    for (int32_t d = threadIdx.x; d * 4 < D_insert; d += blockDim.x) {
      auto row = weight_row_emb.load(d * 4, qparams);
      weight_row_cache.store(row, d * 4, qparams);
    }
    if (threadIdx.x == 0) {
      lxu_cache_state[cache_set][insert_slot] = insert_idx;
    }
  }
}

void lfu_cache_insert_cuda(
    Tensor weights,
    Tensor cache_hash_size_cumsum,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    Tensor sorted_cache_sets,
    Tensor cache_set_sorted_unique_indices,
    Tensor unique_indices_length,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    Tensor lfu_state,
    bool stochastic_rounding) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(weights.get_device());

  int32_t N = cache_set_sorted_unique_indices.numel();

  DISPATCH_EMB_CACHE_TYPES(
      weights.type(),
      lxu_cache_weights.type(),
      "lfu_cache_insert_kernel_2",
      ([&] {
        PhiloxCudaState rng_engine_inputs;
        if (stochastic_rounding && !std::is_same<emb_t, float>::value) {
          auto gen = at::cuda::detail::getDefaultCUDAGenerator();
          std::lock_guard<std::mutex> lock(gen.mutex());
          rng_engine_inputs = at::check_generator<at::CUDAGeneratorImpl>(gen)
                                  ->philox_cuda_state(4);
        }

        lfu_cache_insert_kernel<emb_t, cache_t>
            <<<div_round_up(N, kCacheMaxThreads / kWarpSize),
               dim3(kWarpSize, kCacheMaxThreads / kWarpSize),
               0,
               at::cuda::getCurrentCUDAStream()>>>(
                weights.packed_accessor64<emb_t, 1, RestrictPtrTraits>(),
                cache_hash_size_cumsum
                    .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                cache_index_table_map
                    .packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                weights_offsets
                    .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                D_offsets.packed_accessor32<int32_t, 1, RestrictPtrTraits>(),
                (uint64_t*)sorted_cache_sets.data_ptr<int64_t>(),
                cache_set_sorted_unique_indices
                    .packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                unique_indices_length.data_ptr<int32_t>(),
                lxu_cache_state
                    .packed_accessor32<int64_t, 2, RestrictPtrTraits>(),
                lxu_cache_weights
                    .packed_accessor64<cache_t, 2, RestrictPtrTraits>(),
                lfu_state.packed_accessor64<int64_t, 1, RestrictPtrTraits>(),
                stochastic_rounding,
                rng_engine_inputs);
      }));
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void lfu_cache_populate_cuda(
    Tensor weights,
    Tensor cache_hash_size_cumsum,
    int64_t total_cache_hash_size,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    Tensor linear_cache_indices,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    Tensor lfu_state,
    bool stochastic_rounding) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(weights.get_device());

  TORCH_CHECK(
      linear_cache_indices.numel() < std::numeric_limits<int32_t>::max());
  if (linear_cache_indices.numel() == 0) {
    // nothing to do
    return;
  }

  // get unqiue indices
  Tensor unique_indices;
  Tensor unique_indices_length;
  c10::optional<Tensor> unique_indices_count;
  std::tie(unique_indices, unique_indices_length, unique_indices_count) =
      get_unique_indices_cuda(
          linear_cache_indices, total_cache_hash_size, true);

  // update lfu counts
  lfu_update_counts_cuda(
      unique_indices, unique_indices_length, *unique_indices_count, lfu_state);

  // find uncached indices
  auto cache_sets_and_unique_indices = lfu_cache_find_uncached_cuda(
      unique_indices,
      unique_indices_length,
      total_cache_hash_size,
      lxu_cache_state,
      lfu_state);
  auto sorted_cache_sets = cache_sets_and_unique_indices.first;
  auto cache_set_sorted_unique_indices = cache_sets_and_unique_indices.second;

  // insert caching weights
  lfu_cache_insert_cuda(
      weights,
      cache_hash_size_cumsum,
      cache_index_table_map,
      weights_offsets,
      D_offsets,
      sorted_cache_sets,
      cache_set_sorted_unique_indices,
      unique_indices_length,
      lxu_cache_state,
      lxu_cache_weights,
      lfu_state,
      stochastic_rounding);
}

__global__ __launch_bounds__(kMaxThreads) void lxu_cache_lookup_kernel(
    const PackedTensorAccessor32<int64_t, 1, RestrictPtrTraits>
        linear_cache_indices,
    const PackedTensorAccessor32<int64_t, 2, RestrictPtrTraits> lxu_cache_state,
    PackedTensorAccessor32<int32_t, 1, RestrictPtrTraits> lxu_cache_locations) {
  const int32_t C = lxu_cache_state.size(0);
  const int32_t N = linear_cache_indices.size(0);
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= N) {
    return;
  }
  int64_t idx = linear_cache_indices[n];
  int32_t cache_set = cache_slot(idx, C);
  auto slot = threadIdx.x;
  bool found = (__ldg((&lxu_cache_state[cache_set][0]) + slot) == idx);
  if (found) {
    lxu_cache_locations[n] = cache_set * kWarpSize + slot;
  }
  if (!__any_sync(0xFFFFFFFF, found)) {
    if (threadIdx.x == 0) {
      lxu_cache_locations[n] = kCacheLocationMissing;
    }
  }
}

Tensor lxu_cache_lookup_cuda(
    Tensor linear_cache_indices,
    Tensor lxu_cache_state) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(linear_cache_indices.get_device());

  const auto N = linear_cache_indices.numel();
  auto lxu_cache_locations = empty_like(
      linear_cache_indices, linear_cache_indices.options().dtype(kInt));
  if (linear_cache_indices.numel() == 0) {
    // nothing to do
    return lxu_cache_locations;
  }

  const dim3 threads(kWarpSize, kMaxThreads / kWarpSize);
  const dim3 blocks(div_round_up(N, kMaxThreads / kWarpSize));

  lxu_cache_lookup_kernel<<<
      blocks,
      threads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      linear_cache_indices.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
      lxu_cache_state.packed_accessor32<int64_t, 2, RestrictPtrTraits>(),
      lxu_cache_locations.packed_accessor32<int32_t, 1, RestrictPtrTraits>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return lxu_cache_locations;
}
