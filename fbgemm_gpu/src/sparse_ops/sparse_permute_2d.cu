#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

// Kernel for permuting the indices and weights. Used for permutation of sparse
// data
template <
    bool has_weight,
    typename offsets_t,
    typename indices_t,
    typename weights_t>
__global__ __launch_bounds__(kMaxThreads) void permute_2D_data_kernel(
    int32_t len,
    int32_t T,
    int32_t B,
    const indices_t* __restrict__ indices,
    const weights_t* __restrict__ weights,
    const int32_t* __restrict__ permute,
    const offsets_t* __restrict__ input_offsets,
    const offsets_t* __restrict__ output_offsets,
    indices_t* __restrict__ permuted_indices,
    weights_t* __restrict__ permuted_weights) {
  int32_t b_t_start = blockIdx.x * blockDim.y + threadIdx.y;
  const int stride = gridDim.x * blockDim.y;
  for (int b_t = b_t_start; b_t < B * T; b_t += stride) {
    int32_t b = b_t % B;
    int32_t t = b_t / B;
    offsets_t output_start = output_offsets[b_t];
    offsets_t segment_length;
    if (b_t == B * T - 1) {
      segment_length = len - output_offsets[b_t];
    } else {
      segment_length = output_offsets[b_t + 1] - output_offsets[b_t];
    }
    offsets_t input_start = input_offsets[permute[t] * B + b];
    for (int32_t i = threadIdx.x; i < segment_length; i += blockDim.x) {
      permuted_indices[output_start + i] = indices[input_start + i];
      if (has_weight) {
        permuted_weights[output_start + i] = weights[input_start + i];
      }
    }
  }
}

// Kernel for permuting the lengths. Used for permutation of sparse features.
template <typename index_t>
__global__ __launch_bounds__(kMaxThreads) void permute_2D_lengths_kernel(
    int32_t T,
    int32_t B,
    const index_t* __restrict__ lengths,
    const int32_t* __restrict__ permute,
    index_t* __restrict__ permuted_lengths) {
  CUDA_KERNEL_LOOP(b_t, B * T) {
    int32_t b = b_t % B;
    int32_t t = b_t / B;
    permuted_lengths[b_t] = lengths[permute[t] * B + b];
  }
}

DLL_PUBLIC std::tuple<Tensor, Tensor, std::optional<Tensor>>
permute_2D_sparse_data_cuda(
    const Tensor& permute,
    const Tensor& lengths,
    const Tensor& indices,
    const std::optional<Tensor>& weights,
    const std::optional<int64_t>& permuted_lengths_sum) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(permute, lengths, indices, weights);
  TORCH_CHECK(lengths.dim() == 2);

  CUDA_DEVICE_GUARD(indices);

  const auto permute_contig = permute.contiguous();
  const auto lengths_contig = lengths.contiguous();
  const auto indices_contig = indices.contiguous();
  // the data to permute over can be less or more with or without
  // repetitions
  const auto T = permute.numel();
  const auto B = lengths.size(1);

  if (T == 0 || B == 0) {
    // When T = 0 or B = 0, permutation will not be performed.  Return the
    // input tensors.
    return {
        lengths.clone(),
        indices.clone(),
        weights.has_value() ? c10::make_optional(weights->clone())
                            : c10::nullopt,
    };
  }

  Tensor permuted_lengths;
  Tensor permuted_indices;
  Tensor permuted_weights;

  permuted_lengths = at::empty({T, B}, lengths.options());

  constexpr int32_t threads_1 = 256;
  const auto blocks_1 = cuda_calc_xblock_count(B * T, threads_1);
  AT_DISPATCH_INDEX_TYPES(
      lengths.scalar_type(), "permute_2D_lengths_kernel", [&] {
        permute_2D_lengths_kernel<index_t>
            <<<blocks_1, threads_1, 0, at::cuda::getCurrentCUDAStream()>>>(
                T,
                B,
                lengths_contig.data_ptr<index_t>(),
                permute.data_ptr<int32_t>(),
                permuted_lengths.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

  // convert lengths to offsets
  const auto input_offsets = asynchronous_exclusive_cumsum_gpu(lengths_contig);
  const auto output_offsets =
      asynchronous_complete_cumsum_gpu(permuted_lengths.flatten());
  int64_t permuted_indices_size = 0;
  if (permuted_lengths_sum.has_value()) {
    permuted_indices_size = permuted_lengths_sum.value();
  } else {
    permuted_indices_size = output_offsets[-1].item<int64_t>();
  }

  constexpr int32_t BT_blocks = 32;
  dim3 threads_2(32, BT_blocks);
  const auto blocks_2 = cuda_calc_xblock_count(B * T, BT_blocks);
  permuted_indices = at::empty(permuted_indices_size, indices.options());

  AT_DISPATCH_INDEX_TYPES(
      input_offsets.scalar_type(), "permute_2D_data_kernel_1", [&] {
        using offsets_t = index_t;
        FBGEMM_DISPATCH_ALL_TYPES(
            indices.scalar_type(), "permute_2D_data_kernel_2", [&] {
              using indices_t = scalar_t;
              if (weights.has_value()) {
                const Tensor weights_value = weights.value();
                const auto weights_value_contig = weights_value.contiguous();
                permuted_weights =
                    at::empty(permuted_indices_size, weights_value.options());
                FBGEMM_DISPATCH_ALL_TYPES(
                    weights_value.scalar_type(),
                    "permute_2D_data_kernel_3",
                    [&] {
                      using weights_t = scalar_t;
                      permute_2D_data_kernel<
                          true,
                          offsets_t,
                          indices_t,
                          weights_t>
                          <<<blocks_2,
                             threads_2,
                             0,
                             at::cuda::getCurrentCUDAStream()>>>(
                              permuted_indices_size,
                              T,
                              B,
                              indices_contig.data_ptr<indices_t>(),
                              weights_value_contig.data_ptr<weights_t>(),
                              permute_contig.data_ptr<int32_t>(),
                              input_offsets.data_ptr<offsets_t>(),
                              output_offsets.data_ptr<offsets_t>(),
                              permuted_indices.data_ptr<indices_t>(),
                              permuted_weights.data_ptr<weights_t>());
                      C10_CUDA_KERNEL_LAUNCH_CHECK();
                    }); // for each weights_t
              } else {
                permute_2D_data_kernel<
                    false,
                    offsets_t,
                    indices_t,
                    std::nullptr_t>
                    <<<blocks_2,
                       threads_2,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        permuted_indices_size,
                        T,
                        B,
                        indices_contig.data_ptr<indices_t>(),
                        nullptr,
                        permute_contig.data_ptr<int32_t>(),
                        input_offsets.data_ptr<offsets_t>(),
                        output_offsets.data_ptr<offsets_t>(),
                        permuted_indices.data_ptr<indices_t>(),
                        nullptr);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
            }); // for each indices_t
      }); // for each offsets_t
  return {permuted_lengths, permuted_indices, permuted_weights};
}

// Kernel for permuting the indices and weights. Used for permutation of
// sparse features
template <bool has_weight, typename index_t, typename scalar_t>
__global__ __launch_bounds__(kMaxThreads) void permute_indices_weights_kernel(
    int32_t T,
    int32_t B,
    const index_t* __restrict__ indices,
    const scalar_t* __restrict__ weights,
    const int32_t* __restrict__ permute,
    const index_t* __restrict__ input_offsets,
    const index_t* __restrict__ output_offsets,
    index_t* __restrict__ permuted_indices,
    scalar_t* __restrict__ permuted_weights) {
  int32_t b_t_start = blockIdx.x * blockDim.y + threadIdx.y;
  const int stride = gridDim.x * blockDim.y;
  for (int b_t = b_t_start; b_t < B * T; b_t += stride) {
    int32_t b = b_t % B;
    int32_t t = b_t / B;
    index_t output_start = output_offsets[b_t];
    index_t segment_length = output_offsets[b_t + 1] - output_offsets[b_t];
    index_t input_start = input_offsets[permute[t] * B + b];
    for (int32_t i = threadIdx.x; i < segment_length; i += blockDim.x) {
      permuted_indices[output_start + i] = indices[input_start + i];
      if (has_weight) {
        permuted_weights[output_start + i] = weights[input_start + i];
      }
    }
  }
}

DLL_PUBLIC std::tuple<Tensor, Tensor, std::optional<Tensor>>
permute_sparse_features_cuda(
    const Tensor& permute,
    const Tensor& lengths,
    const Tensor& indices,
    const std::optional<Tensor>& weights) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(permute, lengths, indices, weights);

  CUDA_DEVICE_GUARD(indices);

  // the following implementation requires lengths and indices has the same
  // dtype if usecase comes up that requires different dtype (e.g. int32 for
  // lengths and int64 for indices, this will give a better error msg for
  // debugging
  TENSORS_HAVE_SAME_TYPE(lengths, indices);

  TORCH_CHECK(
      lengths.dim() == 2,
      "The dimension of lengths tensor should be equal to 2 to correctly infer number of features and batch size.")

  const auto permute_contig = permute.contiguous();
  const auto lengths_contig = lengths.contiguous();
  const auto indices_contig = indices.contiguous();
  // the features to permute over can be less or more with or without
  // repetitions
  const auto num_output_features = permute.numel();
  const auto num_features = lengths.size(0);
  const auto B = lengths.size(1);

  Tensor permuted_lengths;
  Tensor permuted_indices;
  Tensor permuted_weights;

  permuted_lengths = at::empty({num_output_features, B}, lengths.options());

  constexpr int32_t threads_1 = 256;
  const auto blocks_1 =
      cuda_calc_xblock_count(B * num_output_features, threads_1);
  AT_DISPATCH_INDEX_TYPES(
      lengths.scalar_type(), "permute_2D_lengths_kernel", [&] {
        fbgemm_gpu::permute_2D_lengths_kernel<index_t>
            <<<blocks_1, threads_1, 0, at::cuda::getCurrentCUDAStream()>>>(
                num_output_features,
                B,
                lengths_contig.data_ptr<index_t>(),
                permute.data_ptr<int32_t>(),
                permuted_lengths.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

  // convert lengths to offsets
  const auto input_offsets =
      fbgemm_gpu::asynchronous_exclusive_cumsum_gpu(lengths_contig);
  const auto output_offsets =
      fbgemm_gpu::asynchronous_complete_cumsum_gpu(permuted_lengths.flatten());
  int64_t permuted_lengths_sum = indices.numel();

  /* TODO: Remove the condition protecting the slow path because even when the
   * condition below is true permuted_lengths.sum() could still be needed. For
   * instance if there are three features with indices `[0, 1, 2]`, `permute`
   * can be `[0, 1, 1]` for which permuted lengths sum would be needed to
   * create permuted_{indices, weights} and `permuted_lengths_sum =
   * indices.numel() or weights.numdel() would be incorrect.
   */
  if (num_features != num_output_features) {
    permuted_lengths_sum = output_offsets[-1].item<int64_t>();
  }

  constexpr int32_t BT_blocks = 32;
  dim3 threads_2(32, BT_blocks);
  const auto blocks_2 =
      cuda_calc_xblock_count(B * num_output_features, BT_blocks);
  permuted_indices = at::empty(permuted_lengths_sum, indices.options());
  if (weights.has_value()) {
    const Tensor weights_value = weights.value();
    const auto weights_value_contig = weights_value.contiguous();
    permuted_weights = at::empty(permuted_lengths_sum, weights_value.options());
    AT_DISPATCH_INDEX_TYPES(
        input_offsets.scalar_type(), "permute_indices_weights_kernel_1", [&] {
          FBGEMM_DISPATCH_FLOATING_TYPES_AND(
              at::ScalarType::Int,
              weights_value.scalar_type(),
              "permute_indices_weights_kernel_2",
              [&] {
                permute_indices_weights_kernel<true, index_t, scalar_t>
                    <<<blocks_2,
                       threads_2,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        num_output_features,
                        B,
                        indices_contig.data_ptr<index_t>(),
                        weights_value_contig.data_ptr<scalar_t>(),
                        permute_contig.data_ptr<int32_t>(),
                        input_offsets.data_ptr<index_t>(),
                        output_offsets.data_ptr<index_t>(),
                        permuted_indices.data_ptr<index_t>(),
                        permuted_weights.data_ptr<scalar_t>());
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              });
        });
  } else {
    AT_DISPATCH_INDEX_TYPES(
        indices.scalar_type(), "permute_indices_kernel", [&] {
          permute_indices_weights_kernel<false, index_t, std::nullptr_t>
              <<<blocks_2, threads_2, 0, at::cuda::getCurrentCUDAStream()>>>(
                  num_output_features,
                  B,
                  indices_contig.data_ptr<index_t>(),
                  nullptr,
                  permute_contig.data_ptr<int32_t>(),
                  input_offsets.data_ptr<index_t>(),
                  output_offsets.data_ptr<index_t>(),
                  permuted_indices.data_ptr<index_t>(),
                  nullptr);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
  }
  return {permuted_lengths, permuted_indices, permuted_weights};
}

} // namespace fbgemm_gpu

FBGEMM_OP_DISPATCH(
    CUDA,
    "permute_sparse_data",
    fbgemm_gpu::permute_2D_sparse_data_cuda);
FBGEMM_OP_DISPATCH(
    CUDA,
    "permute_2D_sparse_data",
    fbgemm_gpu::permute_2D_sparse_data_cuda);
FBGEMM_OP_DISPATCH(
    CUDA,
    "permute_sparse_features",
    fbgemm_gpu::permute_sparse_features_cuda);
