#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

// Kernel for permuting the indices and weights. Used for permutation of
// table-wise partitioned sequence embeddings

template <typename index_t, typename scalar_t>
__global__ void permute_embeddings_kernel(
    int32_t len,
    int32_t T,
    int32_t B,
    const scalar_t* __restrict__ embeddings,
    // bag level permute
    const int32_t* __restrict__ permute,
    const index_t* __restrict__ input_offsets,
    const index_t* __restrict__ output_offsets,
    scalar_t* __restrict__ permuted_embeddings) {
  int32_t b_t_start = blockIdx.x * blockDim.y + threadIdx.y;
  const int stride = gridDim.x * blockDim.y;
  for (int b_t = b_t_start; b_t < B * T; b_t += stride) {
    int32_t b = b_t % B;
    int32_t t = b_t / B;
    index_t output_start = output_offsets[b_t];
    index_t segment_length;
    if (b_t == B * T - 1) {
      segment_length = len - output_offsets[b_t];
    } else {
      segment_length = output_offsets[b_t + 1] - output_offsets[b_t];
    }
    index_t input_start = input_offsets[permute[t] * B + b];
    for (int32_t i = threadIdx.x; i < segment_length; i += blockDim.x) {
      permuted_embeddings[output_start + i] = embeddings[input_start + i];
    }
  }
}

DLL_PUBLIC std::tuple<Tensor, Tensor> permute_sequence_embeddings_cuda(
    const Tensor& permute,
    const Tensor& lengths,
    const Tensor& embeddings) {
  // wrapper for permute_2D_sparse_data_cuda, kept for BC
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(permute, lengths, embeddings);

  CUDA_DEVICE_GUARD(embeddings);

  TORCH_CHECK(
      lengths.dim() == 2,
      "The dimension of lengths tensor should be equal to 2"
      "to correctly infer number of features and batch size.")

  Tensor permuted_lengths;
  Tensor permuted_embeddings;
  std::optional<Tensor> weights_dummy;
  std::optional<int64_t> permuted_lengths_sum_dummy;

  const auto T = permute.numel();
  const auto B = lengths.size(1);
  if (T == 0 || B == 0) {
    return {lengths.clone(), embeddings.clone()};
  }
  permuted_lengths = at::empty({T, B}, lengths.options());

  // ignore the third element in the tuple
  std::tie(permuted_lengths, permuted_embeddings, std::ignore) =
      fbgemm_gpu::permute_2D_sparse_data_cuda(
          permute,
          lengths,
          embeddings,
          weights_dummy,
          permuted_lengths_sum_dummy);

  return {permuted_lengths, permuted_embeddings};
}

} // namespace fbgemm_gpu

FBGEMM_OP_DISPATCH(
    CUDA,
    "permute_sequence_embeddings",
    fbgemm_gpu::permute_sequence_embeddings_cuda);
