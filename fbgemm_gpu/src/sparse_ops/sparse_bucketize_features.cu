#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

// Kernel for bucketize lengths, with the Cyclic distribution (vs. block,
// block-cyclic distribution). Used for bucketize sparse feature with row-wise
// partition (sparse_feature is partitioned cyclically along the sparse
// dimension into my_size blocks)
template <typename scalar_t>
__global__
__launch_bounds__(kMaxThreads) void _bucketize_sparse_features_cuda_kernel1(
    int lengths_size,
    int my_size,
    const scalar_t* __restrict__ offsets_data,
    const scalar_t* __restrict__ indices_data,
    scalar_t* __restrict__ new_lengths_data) {
  using uscalar_t = std::make_unsigned_t<scalar_t>;
  CUDA_KERNEL_LOOP(r, lengths_size) {
    scalar_t rowstart = (r == 0 ? 0 : offsets_data[r - 1]);
    scalar_t rowend = offsets_data[r];
    for (scalar_t i = rowstart; i < rowend; ++i) {
      // Need to handle negative indices if we use raw indices instead of hashed
      // indices, convert to unsigned
      uscalar_t idx = static_cast<uscalar_t>(indices_data[i]);
      uscalar_t p = idx % my_size;
      new_lengths_data[p * lengths_size + r]++;
    }
  }
}

// Kernel for bucketize offsets, indices, and positional weights, with the
// Cyclic distribution (vs. block, block-cyclic distribution). Used for
// bucketize sparse feature with row-wise partition (sparse_feature is
// partitioned cyclically along the sparse dimension into my_size blocks)
template <
    bool has_weight,
    bool bucketize_pos,
    typename index_t,
    typename scalar_t>
__global__
__launch_bounds__(kMaxThreads) void _bucketize_sparse_features_cuda_kernel2(
    int lengths_size,
    int my_size,
    const index_t* __restrict__ offsets_data,
    const index_t* __restrict__ indices_data,
    const scalar_t* __restrict__ weights_data,
    index_t* __restrict__ new_offsets_data,
    index_t* __restrict__ new_indices_data,
    scalar_t* __restrict__ new_weights_data,
    index_t* __restrict__ new_pos_data) {
  using uindex_t = std::make_unsigned_t<index_t>;
  CUDA_KERNEL_LOOP(r, lengths_size) {
    index_t rowstart = r == 0 ? 0 : offsets_data[r - 1];
    index_t rowend = offsets_data[r];
    for (index_t i = rowstart; i < rowend; ++i) {
      // Need to handle negative indices if we use raw indices instead of hashed
      // indices, convert to unsigned
      uindex_t idx = static_cast<uindex_t>(indices_data[i]);
      uindex_t p = idx % my_size;
      uindex_t new_idx = idx / my_size;
      uindex_t pos = new_offsets_data[p * lengths_size + r];
      new_indices_data[pos] = new_idx;
      new_offsets_data[p * lengths_size + r]++;
      if (has_weight) {
        new_weights_data[pos] = weights_data[i];
      }
      if (bucketize_pos) {
        new_pos_data[pos] = i - rowstart;
      }
    }
  }
}

// This function partitions sparse features
// cyclically along the sparse dimension into my_size blocks
DLL_PUBLIC
std::tuple<Tensor, Tensor, c10::optional<Tensor>, c10::optional<Tensor>>
bucketize_sparse_features_cuda(
    const Tensor& lengths,
    const Tensor& indices,
    const bool bucketize_pos,
    const int64_t my_size,
    const c10::optional<Tensor>& weights) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(lengths, indices);

  CUDA_DEVICE_GUARD(lengths);

  // allocate tensors and buffers
  const int lengths_size = lengths.numel();
  const int new_lengths_size = lengths_size * my_size;
  auto offsets = at::empty({lengths_size}, lengths.options());
  auto new_lengths = at::zeros({new_lengths_size}, lengths.options());
  auto new_offsets = at::empty({new_lengths_size}, lengths.options());
  auto new_indices = at::empty_like(indices);
  auto lengths_contig = lengths.contiguous();
  auto indices_contig = indices.contiguous();
  auto offsets_contig = offsets.contiguous();
  Tensor new_weights;
  Tensor new_pos;
  // count nonzeros
  offsets_contig = fbgemm_gpu::asynchronous_inclusive_cumsum_gpu(lengths);
  int threads_per_block = 256;
  const auto num_blocks =
      cuda_calc_xblock_count(lengths_size, threads_per_block);
  AT_DISPATCH_INDEX_TYPES(
      indices_contig.scalar_type(),
      "_bucketize_sparse_features_cuda_kernel1",
      ([&] {
        _bucketize_sparse_features_cuda_kernel1<<<
            num_blocks,
            threads_per_block,
            0,
            at::cuda::getCurrentCUDAStream()>>>(
            lengths_size,
            my_size,
            offsets_contig.data_ptr<index_t>(),
            indices_contig.data_ptr<index_t>(),
            new_lengths.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
  // bucketize nonzeros
  new_offsets = fbgemm_gpu::asynchronous_exclusive_cumsum_gpu(new_lengths);
  if (weights.has_value() & bucketize_pos) {
    Tensor weights_value = weights.value();
    auto weights_value_contig = weights_value.contiguous();
    new_weights = at::empty_like(weights_value);
    new_pos = at::empty_like(indices);
    AT_DISPATCH_INDEX_TYPES(
        indices_contig.scalar_type(),
        "_bucketize_sparse_features_weight_cuda_kernel2_1",
        ([&] {
          FBGEMM_DISPATCH_FLOAT_ONLY(
              weights_value.scalar_type(),
              "_bucketize_sparse_features_cuda_weight_kernel2_2",
              ([&] {
                _bucketize_sparse_features_cuda_kernel2<
                    true,
                    true,
                    index_t,
                    scalar_t>
                    <<<num_blocks,
                       threads_per_block,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        lengths_size,
                        my_size,
                        offsets_contig.data_ptr<index_t>(),
                        indices_contig.data_ptr<index_t>(),
                        weights_value_contig.data_ptr<scalar_t>(),
                        new_offsets.data_ptr<index_t>(),
                        new_indices.data_ptr<index_t>(),
                        new_weights.data_ptr<scalar_t>(),
                        new_pos.data_ptr<index_t>());
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }));
        }));
  } else if (weights.has_value()) {
    Tensor weights_value = weights.value();
    auto weights_value_contig = weights_value.contiguous();
    new_weights = at::empty_like(weights_value);
    AT_DISPATCH_INDEX_TYPES(
        indices_contig.scalar_type(),
        "_bucketize_sparse_features_weight_cuda_kernel2_1",
        ([&] {
          FBGEMM_DISPATCH_FLOAT_ONLY(
              weights_value.scalar_type(),
              "_bucketize_sparse_features_cuda_weight_kernel2_2",
              ([&] {
                _bucketize_sparse_features_cuda_kernel2<
                    true,
                    false,
                    index_t,
                    scalar_t>
                    <<<num_blocks,
                       threads_per_block,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        lengths_size,
                        my_size,
                        offsets_contig.data_ptr<index_t>(),
                        indices_contig.data_ptr<index_t>(),
                        weights_value_contig.data_ptr<scalar_t>(),
                        new_offsets.data_ptr<index_t>(),
                        new_indices.data_ptr<index_t>(),
                        new_weights.data_ptr<scalar_t>(),
                        nullptr);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }));
        }));
  } else if (bucketize_pos) {
    new_pos = at::empty_like(indices);
    AT_DISPATCH_INDEX_TYPES(
        indices_contig.scalar_type(),
        "_bucketize_sparse_features_cuda_kernel2",
        ([&] {
          _bucketize_sparse_features_cuda_kernel2<
              false,
              true,
              index_t,
              std::nullptr_t>
              <<<num_blocks,
                 threads_per_block,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  lengths_size,
                  my_size,
                  offsets_contig.data_ptr<index_t>(),
                  indices_contig.data_ptr<index_t>(),
                  nullptr,
                  new_offsets.data_ptr<index_t>(),
                  new_indices.data_ptr<index_t>(),
                  nullptr,
                  new_pos.data_ptr<index_t>());
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        }));
  } else {
    AT_DISPATCH_INDEX_TYPES(
        indices_contig.scalar_type(),
        "_bucketize_sparse_features_cuda_kernel2",
        ([&] {
          _bucketize_sparse_features_cuda_kernel2<
              false,
              false,
              index_t,
              std::nullptr_t>
              <<<num_blocks,
                 threads_per_block,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  lengths_size,
                  my_size,
                  offsets_contig.data_ptr<index_t>(),
                  indices_contig.data_ptr<index_t>(),
                  nullptr,
                  new_offsets.data_ptr<index_t>(),
                  new_indices.data_ptr<index_t>(),
                  nullptr,
                  nullptr);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        }));
  }

  return {new_lengths, new_indices, new_weights, new_pos};
}

} // namespace fbgemm_gpu

FBGEMM_OP_DISPATCH(
    CUDA,
    "bucketize_sparse_features",
    fbgemm_gpu::bucketize_sparse_features_cuda);
