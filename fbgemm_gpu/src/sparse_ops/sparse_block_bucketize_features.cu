#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

// Kernel for calulating lengthh idx to feature id mapping. Used for block
// bucketize sparse features with variable batch size for row-wise partition
template <typename offset_t>
__global__
__launch_bounds__(kMaxThreads) void _populate_length_to_feature_id_inplace_kernel(
    const uint64_t max_B,
    const int T,
    const offset_t* const __restrict__ batch_size_per_feature,
    const offset_t* const __restrict__ batch_size_offsets,
    offset_t* const __restrict__ length_to_feature_idx) {
  const auto b_t = blockIdx.x * blockDim.x + threadIdx.x;

  const auto t = b_t / max_B;
  const auto b = b_t % max_B;

  if (t >= T || b >= batch_size_per_feature[t]) {
    return;
  }

  length_to_feature_idx[batch_size_offsets[t] + b] = t;
}

// Kernel for bucketize lengths, with the Block distribution (vs. cyclic,
// block-cyclic distribution). Used for bucketize sparse feature, especially for
// checkpointing with row-wise partition (sparse_feature is partitioned
// continuously along the sparse dimension into my_size blocks)
template <typename offset_t, typename index_t>
__global__
__launch_bounds__(kMaxThreads) void _block_bucketize_sparse_features_cuda_kernel1(
    const int32_t lengths_size,
    const int32_t B,
    const index_t* const __restrict__ block_sizes_data,
    const int my_size,
    const offset_t* const __restrict__ offsets_data,
    const index_t* const __restrict__ indices_data,
    offset_t* const __restrict__ new_lengths_data,
    offset_t* __restrict__ length_to_feature_idx,
    const offset_t* const __restrict__ block_bucketize_pos_concat,
    const offset_t* const __restrict__ block_bucketize_pos_offsets,
    offset_t* __restrict__ indices_to_lb) {
  using uindex_t = std::make_unsigned_t<index_t>;
  const auto bt_start = blockIdx.x * blockDim.y + threadIdx.y;
  const auto stride = gridDim.x * blockDim.y;
  for (auto b_t = bt_start; b_t < lengths_size; b_t += stride) {
    const auto t = length_to_feature_idx ? length_to_feature_idx[b_t] : b_t / B;
    index_t blk_size = block_sizes_data[t];
    offset_t rowstart = (b_t == 0 ? 0 : offsets_data[b_t - 1]);
    offset_t rowend = offsets_data[b_t];
    const auto use_block_bucketize_pos =
        (block_bucketize_pos_concat != nullptr);
    // We have use cases using none-hashed raw indices that can be either
    // negative or larger than embedding table hash_size (blk_size *
    // my_size). In cases of none-hashed indices we need to ensure
    // bucketization can distribute them into different ranks and within
    // range of blk_size, we expect the later embedding module to take care
    // of hashing indices calculation.
    if (!use_block_bucketize_pos) {
      for (auto i = rowstart + threadIdx.x; i < rowend; i += blockDim.x) {
        uindex_t idx = static_cast<uindex_t>(indices_data[i]);
        uindex_t p = idx < blk_size * my_size ? idx / blk_size : idx % my_size;
        atomicAdd(&new_lengths_data[p * lengths_size + b_t], 1);
      }
      return;
    }

    for (auto i = rowstart + threadIdx.x; i < rowend; i += blockDim.x) {
      uindex_t idx = static_cast<uindex_t>(indices_data[i]);
      uindex_t p = 0;
      index_t first = block_bucketize_pos_offsets[t];
      index_t last = block_bucketize_pos_offsets[t + 1];

      while (first < last) {
        index_t middle = first + ((last - first) / 2);
        if (static_cast<uindex_t>(block_bucketize_pos_concat[middle]) <= idx) {
          first = ++middle;
        } else {
          last = middle;
        }
      }
      uindex_t lb =
          static_cast<uindex_t>(first - block_bucketize_pos_offsets[t] - 1);
      indices_to_lb[i] = lb;
      p = lb < my_size ? lb : idx % my_size;
      atomicAdd(&new_lengths_data[p * lengths_size + b_t], 1);
    }
  }
}

// Kernel for bucketize offsets, indices, and positional weights, with the Block
// distribution (vs. cyclic, block-cyclic distribution). Used for bucketize
// sparse feature, especially for checkpointing with row-wise partition
// (sparse_feature is partitioned continuously along the sparse dimension into
// my_size blocks)
template <
    bool sequence,
    bool has_weight,
    bool bucketize_pos,
    typename offset_t,
    typename index_t,
    typename scalar_t>
__global__
__launch_bounds__(kMaxThreads) void _block_bucketize_sparse_features_cuda_kernel2(
    int lengths_size,
    int32_t B,
    const index_t* __restrict__ block_sizes_data,
    int my_size,
    const offset_t* __restrict__ offsets_data,
    const index_t* __restrict__ indices_data,
    const scalar_t* __restrict__ weights_data,
    offset_t* __restrict__ new_offsets_data,
    index_t* __restrict__ new_indices_data,
    scalar_t* __restrict__ new_weights_data,
    index_t* __restrict__ new_pos_data,
    index_t* const __restrict__ unbucketize_permute_data,
    const offset_t* const __restrict__ length_to_feature_idx,
    const offset_t* const __restrict__ block_bucketize_pos_concat,
    const offset_t* const __restrict__ block_bucketize_pos_offsets,
    const offset_t* const __restrict__ indices_to_lb) {
  using uindex_t = std::make_unsigned_t<index_t>;
  using uoffset_t = std::make_unsigned_t<offset_t>;
  CUDA_KERNEL_LOOP(b_t, lengths_size) {
    const auto t = length_to_feature_idx ? length_to_feature_idx[b_t] : b_t / B;
    index_t blk_size = block_sizes_data[t];
    offset_t rowstart = (b_t == 0 ? 0 : offsets_data[b_t - 1]);
    offset_t rowend = offsets_data[b_t];
    const auto use_block_bucketize_pos =
        (block_bucketize_pos_concat != nullptr);
    for (index_t i = rowstart; i < rowend; ++i) {
      // We have use cases using none-hashed raw indices that can be either
      // negative or larger than embedding table hash_size (blk_size *
      // my_size). In cases of none-hashed indices we need to ensure
      // bucketization can distribute them into different ranks and within
      // range of blk_size, we expect the later embedding module to take care
      // of hashing indices calculation.
      uindex_t idx = static_cast<uindex_t>(indices_data[i]);
      uindex_t p = 0;
      uindex_t new_idx = 0;
      if (!use_block_bucketize_pos) {
        p = idx < blk_size * my_size ? idx / blk_size : idx % my_size;
        new_idx = idx < blk_size * my_size ? idx % blk_size : idx / my_size;
      } else {
        uindex_t lb = indices_to_lb[i];
        p = lb < my_size ? lb : idx % my_size;
        new_idx = lb < my_size ? idx -
                block_bucketize_pos_concat[lb + block_bucketize_pos_offsets[t]]
                               : idx / my_size;
      }
      uoffset_t pos = new_offsets_data[p * lengths_size + b_t];
      new_indices_data[pos] = new_idx;
      new_offsets_data[p * lengths_size + b_t]++;
      if (sequence) {
        unbucketize_permute_data[i] = pos;
      }
      if (has_weight) {
        new_weights_data[pos] = weights_data[i];
      }
      if (bucketize_pos) {
        new_pos_data[pos] = i - rowstart;
      }
    }
  }
}

// This function partitions sparse features
// continuously along the sparse dimension into my_size blocks
DLL_PUBLIC std::tuple<
    Tensor,
    Tensor,
    c10::optional<Tensor>,
    c10::optional<Tensor>,
    c10::optional<Tensor>>
block_bucketize_sparse_features_cuda(
    const Tensor& lengths,
    const Tensor& indices,
    const bool bucketize_pos,
    const bool sequence,
    const Tensor& block_sizes,
    const int64_t my_size,
    const c10::optional<Tensor>& weights,
    const c10::optional<Tensor>& batch_size_per_feature,
    const int64_t max_B,
    const c10::optional<std::vector<at::Tensor>>& block_bucketize_pos) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(lengths, indices);

  CUDA_DEVICE_GUARD(lengths);

  // allocate tensors and buffers
  const auto lengths_size = lengths.numel();
  const auto T = block_sizes.numel();
  const auto B = lengths_size / T;
  const auto new_lengths_size = lengths_size * my_size;
  auto offsets = at::empty({lengths_size}, lengths.options());
  auto new_lengths = at::zeros({new_lengths_size}, lengths.options());
  auto new_offsets = at::empty({new_lengths_size}, lengths.options());
  auto new_indices = at::empty_like(indices);
  auto lengths_contig = lengths.contiguous();
  auto indices_contig = indices.contiguous();
  auto offsets_contig = offsets.contiguous();
  auto batch_sizes_contig =
      batch_size_per_feature.value_or(at::empty({T}, lengths.options()))
          .contiguous();
  auto batch_sizes_offsets_contig =
      at::empty({T}, batch_sizes_contig.options());
  Tensor new_weights;
  Tensor new_pos;
  Tensor unbucketize_permute;
  // count nonzeros
  offsets_contig = asynchronous_inclusive_cumsum_gpu(lengths);
  if (batch_size_per_feature.has_value()) {
    TORCH_CHECK(max_B > 0);
    batch_sizes_offsets_contig =
        asynchronous_exclusive_cumsum_gpu(batch_size_per_feature.value());
  }
  auto length_to_feature_idx =
      at::empty({lengths_size}, lengths_contig.options());
  auto indices_to_lb = at::empty_like(indices);
  if (batch_size_per_feature.has_value()) {
    constexpr auto threads_per_block = 256;
    const auto num_blocks =
        cuda_calc_xblock_count(max_B * T, threads_per_block);
    AT_DISPATCH_INDEX_TYPES(
        offsets_contig.scalar_type(),
        "_populate_length_to_feature_id_inplace_kernel",
        [&] {
          using offset_t = index_t;
          _populate_length_to_feature_id_inplace_kernel<<<
              num_blocks,
              threads_per_block,
              0,
              at::cuda::getCurrentCUDAStream()>>>(
              max_B,
              T,
              batch_sizes_contig.data_ptr<offset_t>(),
              batch_sizes_offsets_contig.data_ptr<offset_t>(),
              length_to_feature_idx.data_ptr<offset_t>());
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
  }

  at::Tensor block_bucketize_pos_concat =
      at::empty({1}, lengths_contig.options());
  at::Tensor block_bucketize_pos_offsets =
      at::empty({1}, lengths_contig.options());

  if (block_bucketize_pos.has_value()) {
    block_bucketize_pos_concat = at::cat(block_bucketize_pos.value(), 0);
    std::vector<int64_t> sizes_;
    sizes_.reserve(block_bucketize_pos.value().size() + 1);
    for (auto const& t : block_bucketize_pos.value()) {
      sizes_.push_back(t.numel());
    }
    sizes_.push_back(0);
    at::Tensor sizes_vec =
        at::tensor(sizes_, at::TensorOptions().dtype(lengths_contig.dtype()));
    block_bucketize_pos_offsets = asynchronous_exclusive_cumsum_cpu(
        sizes_vec); // expect sizes_vec to be a small tensor, using cpu instead
                    // of gpu for cumsum
    block_bucketize_pos_offsets = block_bucketize_pos_offsets.to(
        block_bucketize_pos_concat.device(), true);
  }
  static_assert(kMaxThreads % kWarpSize == 0);
  const dim3 block_dims(kWarpSize, kMaxThreads / kWarpSize);
  const dim3 grid_dims(cuda_calc_xblock_count(lengths_size, block_dims.y));
  AT_DISPATCH_INDEX_TYPES(
      offsets_contig.scalar_type(),
      "_block_bucketize_sparse_features_cuda_kernel1",
      [&] {
        using offset_t = index_t;
        AT_DISPATCH_INDEX_TYPES(
            indices_contig.scalar_type(),
            "_block_bucketize_sparse_features_cuda_kernel2",
            [&] {
              _block_bucketize_sparse_features_cuda_kernel1<<<
                  grid_dims,
                  block_dims,
                  0,
                  at::cuda::getCurrentCUDAStream()>>>(
                  lengths_size,
                  B,
                  block_sizes.data_ptr<index_t>(),
                  my_size,
                  offsets_contig.data_ptr<offset_t>(),
                  indices_contig.data_ptr<index_t>(),
                  new_lengths.data_ptr<offset_t>(),
                  batch_size_per_feature.has_value()
                      ? length_to_feature_idx.data_ptr<offset_t>()
                      : static_cast<offset_t*>(nullptr),
                  block_bucketize_pos.has_value()
                      ? block_bucketize_pos_concat.data_ptr<offset_t>()
                      : static_cast<offset_t*>(nullptr),
                  block_bucketize_pos.has_value()
                      ? block_bucketize_pos_offsets.data_ptr<offset_t>()
                      : static_cast<offset_t*>(nullptr),
                  block_bucketize_pos.has_value()
                      ? indices_to_lb.data_ptr<offset_t>()
                      : static_cast<offset_t*>(nullptr));
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            });
      });
  constexpr auto threads_per_block = 256;
  const auto num_blocks =
      cuda_calc_xblock_count(lengths_size, threads_per_block);
  // bucketize nonzeros
  new_offsets = asynchronous_exclusive_cumsum_gpu(new_lengths);
  if (sequence) {
    const auto lengths_sum = indices.numel();
    unbucketize_permute = at::empty({lengths_sum}, indices.options());
    if (weights.has_value() & bucketize_pos) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          [&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                [&] {
                  FBGEMM_DISPATCH_FLOAT_ONLY(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      [&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            true,
                            true,
                            true,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                new_pos.data_ptr<index_t>(),
                                unbucketize_permute.data_ptr<index_t>(),
                                batch_size_per_feature.has_value()
                                    ? length_to_feature_idx.data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? block_bucketize_pos_concat
                                          .data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? block_bucketize_pos_offsets
                                          .data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? indices_to_lb.data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr));
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      });
                });
          });
    } else if (weights.has_value()) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          [&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                [&] {
                  FBGEMM_DISPATCH_FLOAT_ONLY(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      [&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            true,
                            true,
                            false,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                nullptr,
                                unbucketize_permute.data_ptr<index_t>(),
                                batch_size_per_feature.has_value()
                                    ? length_to_feature_idx.data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? block_bucketize_pos_concat
                                          .data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? block_bucketize_pos_offsets
                                          .data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? indices_to_lb.data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr));
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      });
                });
          });
    } else if (bucketize_pos) {
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          [&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                [&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      true,
                      false,
                      true,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          new_pos.data_ptr<index_t>(),
                          unbucketize_permute.data_ptr<index_t>(),
                          batch_size_per_feature.has_value()
                              ? length_to_feature_idx.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? block_bucketize_pos_concat.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? block_bucketize_pos_offsets.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? indices_to_lb.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr));
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                });
          });
    } else {
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          [&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                [&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      true,
                      false,
                      false,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          nullptr,
                          unbucketize_permute.data_ptr<index_t>(),
                          batch_size_per_feature.has_value()
                              ? length_to_feature_idx.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? block_bucketize_pos_concat.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? block_bucketize_pos_offsets.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? indices_to_lb.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr));
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                });
          });
    }
  } else {
    if (weights.has_value() & bucketize_pos) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          [&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                [&] {
                  FBGEMM_DISPATCH_FLOAT_ONLY(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      [&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            false,
                            true,
                            true,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                new_pos.data_ptr<index_t>(),
                                nullptr,
                                batch_size_per_feature.has_value()
                                    ? length_to_feature_idx.data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? block_bucketize_pos_concat
                                          .data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? block_bucketize_pos_offsets
                                          .data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? indices_to_lb.data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr));
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      });
                });
          });
    } else if (weights.has_value()) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          [&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                [&] {
                  FBGEMM_DISPATCH_FLOAT_ONLY(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      [&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            false,
                            true,
                            false,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                nullptr,
                                nullptr,
                                batch_size_per_feature.has_value()
                                    ? length_to_feature_idx.data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? block_bucketize_pos_concat
                                          .data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? block_bucketize_pos_offsets
                                          .data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr),
                                block_bucketize_pos.has_value()
                                    ? indices_to_lb.data_ptr<offset_t>()
                                    : static_cast<offset_t*>(nullptr));
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      });
                });
          });
    } else if (bucketize_pos) {
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          [&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                [&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      false,
                      false,
                      true,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          new_pos.data_ptr<index_t>(),
                          nullptr,
                          batch_size_per_feature.has_value()
                              ? length_to_feature_idx.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? block_bucketize_pos_concat.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? block_bucketize_pos_offsets.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? indices_to_lb.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr));
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                });
          });
    } else {
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          [&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                [&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      false,
                      false,
                      false,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          nullptr,
                          nullptr,
                          batch_size_per_feature.has_value()
                              ? length_to_feature_idx.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? block_bucketize_pos_concat.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? block_bucketize_pos_offsets.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr),
                          block_bucketize_pos.has_value()
                              ? indices_to_lb.data_ptr<offset_t>()
                              : static_cast<offset_t*>(nullptr));
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                });
          });
    }
  }

  return {new_lengths, new_indices, new_weights, new_pos, unbucketize_permute};
}

} // namespace fbgemm_gpu

FBGEMM_OP_DISPATCH(
    CUDA,
    "block_bucketize_sparse_features",
    fbgemm_gpu::block_bucketize_sparse_features_cuda);
