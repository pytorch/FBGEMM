#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

template <
    typename scalar_t,
    typename index_t,
    typename acc_t,
    int NUM_THREADS_PER_BLOCK,
    int MAX_ENTRIES_PER_BLOCK>
__global__ void index_select_scalar_cumsum_kernel(
    scalar_t* output,
    acc_t* output_cumsum,
    const scalar_t* __restrict__ input,
    const index_t* __restrict__ indices,
    const int num_batches,
    const int input_batch_size,
    const int output_batch_size,
    const int last_block_num_entries,
    int* block_flags,
    acc_t* block_sums) {
  typedef hipcub::BlockScan<acc_t, NUM_THREADS_PER_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage bs_temp_storage;
  __shared__ acc_t smem[MAX_ENTRIES_PER_BLOCK];
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int bid = tid / output_batch_size;
  const int num_entries_per_block = blockIdx.x == gridDim.x - 1
      ? last_block_num_entries
      : MAX_ENTRIES_PER_BLOCK;

  // Load data
  acc_t local_data[1];
  if (tid < num_batches * output_batch_size) {
    *local_data =
        input[bid * input_batch_size + indices[tid % output_batch_size]];
    output[tid] = *local_data;
  } else {
    *local_data = 0;
  }

  // Cumsum
  inclusive_sum_scan_kernel<acc_t, 1, NUM_THREADS_PER_BLOCK>(
      local_data,
      bs_temp_storage,
      block_flags,
      block_sums,
      &smem[0],
      num_entries_per_block,
      blockIdx.x,
      gridDim.x > 1,
      1);

  // Store data
  if (tid < num_batches * output_batch_size) {
    output_cumsum[tid] = *local_data;
  }
}

template <
    typename scalar_t,
    typename index_t,
    typename offset_t,
    typename weight_t,
    bool has_weights>
__global__ void keyed_jagged_index_select_dim1_kernel(
    scalar_t* output,
    weight_t* output_weights,
    const scalar_t* input,
    const weight_t* weights,
    const offset_t* input_offsets,
    const index_t* indices,
    const offset_t* output_offsets,
    const int num_batches,
    const int input_batch_size,
    const int output_batch_size,
    const int64_t num_outputs) {
  const int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < num_outputs) {
    // Each thread searches index position
    int index_pos;
    binary_search_range(
        &index_pos,
        output_offsets,
        (offset_t)tid,
        num_batches * output_batch_size);

    const offset_t rel_index =
        tid - (index_pos == 0 ? 0 : output_offsets[index_pos - 1]);

    // indices are the same for all batches
    const index_t index = indices[index_pos % output_batch_size];
    const int bid = index_pos / output_batch_size;
    const offset_t input_offset =
        (index == 0 && bid == 0
             ? 0
             : input_offsets[bid * input_batch_size + index - 1]) +
        rel_index;

    // Store data
    output[tid] = input[input_offset];
    if (has_weights) {
      output_weights[tid] = weights[input_offset];
    }
  }
}

template <typename scalar_t, typename index_t, typename offset_t>
__global__ void keyed_jagged_index_add_dim1_kernel(
    scalar_t* output,
    const scalar_t* input,
    const offset_t* input_offsets,
    const index_t* indices,
    const offset_t* output_offsets,
    const int num_batches,
    const int input_batch_size,
    const int output_batch_size,
    const int64_t num_inputs) {
  const int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < num_inputs) {
    // Each thread searches index position
    int index_pos;
    binary_search_range(
        &index_pos,
        input_offsets,
        (offset_t)tid,
        num_batches * input_batch_size);

    const offset_t rel_index =
        tid - (index_pos == 0 ? 0 : input_offsets[index_pos - 1]);

    // indices are the same for all batches
    const index_t index = indices[index_pos % input_batch_size];
    const int bid = index_pos / input_batch_size;
    const offset_t output_offset =
        (index == 0 && bid == 0
             ? 0
             : output_offsets[bid * output_batch_size + index - 1]) +
        rel_index;

    // Store data
    gpuAtomicAdd(&output[output_offset], input[tid]);
  }
}

namespace {

class KeyedJaggedIndexSelectDim1GPUOp
    : public torch::autograd::Function<KeyedJaggedIndexSelectDim1GPUOp> {
 public:
  static torch::autograd::variable_list forward(
      torch::autograd::AutogradContext* ctx,
      const Tensor& values,
      const Tensor& lengths,
      const Tensor& offsets,
      const Tensor& indices, // select same indices for all batches
      const int batch_size,
      const c10::optional<Tensor>& weights) {
    // TODO: Add weights support
    TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(lengths, offsets, values, indices);
    TORCH_CHECK(values.dim() == 1, "values must be a 1D tensor");
    TORCH_CHECK(lengths.dim() == 1, "lengths must be a 1D tensor");
    TORCH_CHECK(offsets.dim() == 1, "offsets must be a 1D tensor");
    TORCH_CHECK(indices.dim() == 1, "indices must be a 1D tensor");
    TORCH_CHECK(
        lengths.numel() + 1 == offsets.numel(),
        "offsets size must be lengths size + 1");
    TORCH_CHECK(lengths.numel() % batch_size == 0, "lengths");

    if (weights.has_value()) {
      const Tensor& pos_weights = weights.value();
      TENSOR_ON_CUDA_GPU(pos_weights);
      TENSORS_ON_SAME_DEVICE(pos_weights, indices);
      TORCH_CHECK(pos_weights.dim() == 1, "weights must be a 1D tensor");
      TORCH_CHECK(
          pos_weights.numel() == values.numel(),
          "weights size and values size must be the same");
    }

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(values.get_device());

    const int num_batches = lengths.numel() / batch_size;
    const int num_output_lengths = num_batches * indices.numel();
    const int MAX_CUMSUM_ENTRIES_PER_BLOCK = 256;
    auto grid_size = cuda_calc_xblock_count(
        num_output_lengths, MAX_CUMSUM_ENTRIES_PER_BLOCK);

    Tensor output_offsets =
        at::empty({num_batches * indices.numel()}, offsets.options());
    Tensor output_lengths =
        at::empty({num_batches * indices.numel()}, lengths.options());

    Tensor block_flags, block_sums;
    if (grid_size > 1) {
      block_flags = at::zeros({grid_size}, lengths.options().dtype(at::kInt));
      block_sums = at::empty({grid_size}, output_offsets.options());
    }
    // Do index select and cumsum
    AT_DISPATCH_INDEX_TYPES(
        lengths.scalar_type(), "index_select_scalar_cumsum_wrapper_1", [&] {
          using length_t = index_t;
          AT_DISPATCH_INDEX_TYPES(
              offsets.scalar_type(),
              "index_select_scalar_cumsum_wrapper_2",
              [&] {
                using offset_t = index_t;
                AT_DISPATCH_INDEX_TYPES(
                    indices.scalar_type(),
                    "index_select_scalar_cumsum_wrapper_3",
                    [&] {
                      index_select_scalar_cumsum_kernel<
                          length_t,
                          index_t,
                          offset_t,
                          MAX_CUMSUM_ENTRIES_PER_BLOCK,
                          MAX_CUMSUM_ENTRIES_PER_BLOCK>
                          <<<grid_size,
                             MAX_CUMSUM_ENTRIES_PER_BLOCK,
                             0,
                             at::cuda::getCurrentCUDAStream()>>>(
                              output_lengths.data_ptr<length_t>(),
                              output_offsets.data_ptr<offset_t>(),
                              lengths.data_ptr<length_t>(),
                              indices.data_ptr<index_t>(),
                              num_batches,
                              batch_size,
                              indices.numel(),
                              num_output_lengths -
                                  MAX_CUMSUM_ENTRIES_PER_BLOCK *
                                      (grid_size - 1),
                              grid_size > 1 ? block_flags.data_ptr<int>()
                                            : nullptr,
                              grid_size > 1 ? block_sums.data_ptr<offset_t>()
                                            : nullptr);
                      C10_CUDA_KERNEL_LAUNCH_CHECK();
                    });
              });
        });

    // TODO: Try to not do D->H transfer
    const int64_t num_outputs =
        output_offsets[output_offsets.numel() - 1].item<int64_t>();
    Tensor output = at::empty({num_outputs}, values.options());
    Tensor output_weights;
    if (weights.has_value()) {
      output_weights = at::empty({num_outputs}, weights.value().options());
    }
    grid_size = cuda_calc_xblock_count(num_outputs, kMaxThreads);

    if (grid_size != 0) {
#define LAUNCH_KERNEL(WEIGHTED, WEIGHT_TYPE, OUTPUT_WEIGHTS, WEIGHTS)      \
  {                                                                        \
    keyed_jagged_index_select_dim1_kernel<                                 \
        value_t,                                                           \
        index_t,                                                           \
        offset_t,                                                          \
        WEIGHT_TYPE,                                                       \
        WEIGHTED>                                                          \
        <<<grid_size, kMaxThreads, 0, at::cuda::getCurrentCUDAStream()>>>( \
            output.data_ptr<value_t>(),                                    \
            OUTPUT_WEIGHTS,                                                \
            values.data_ptr<value_t>(),                                    \
            WEIGHTS,                                                       \
            offsets.data_ptr<offset_t>() + 1,                              \
            indices.data_ptr<index_t>(),                                   \
            output_offsets.data_ptr<offset_t>(),                           \
            num_batches,                                                   \
            batch_size,                                                    \
            indices.numel(),                                               \
            num_outputs);                                                  \
  }
      AT_DISPATCH_ALL_TYPES_AND2(
          at::ScalarType::Half,
          at::ScalarType::BFloat16,
          values.scalar_type(),
          "keyed_jagged_index_select_dim1_warpper_1",
          [&] {
            using value_t = scalar_t;
            AT_DISPATCH_INDEX_TYPES(
                offsets.scalar_type(),
                "keyed_jagged_index_select_dim1_warpper_2",
                [&] {
                  using offset_t = index_t;
                  AT_DISPATCH_INDEX_TYPES(
                      indices.scalar_type(),
                      "keyed_jagged_index_select_dim1_warpper_3",
                      [&] {
                        if (weights.has_value()) {
                          AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                              weights.value().scalar_type(),
                              "keyed_jagged_index_select_dim1_warpper_4",
                              [&] {
                                using weight_t = scalar_t;
                                LAUNCH_KERNEL(
                                    true,
                                    weight_t,
                                    output_weights.data_ptr<weight_t>(),
                                    weights.value().data_ptr<weight_t>())
                              });
                        } else {
                          LAUNCH_KERNEL(false, scalar_t, nullptr, nullptr)
                        }
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      });
                });
          });
    }

#undef LAUNCH_KERNEL

    ctx->save_for_backward({indices, output_offsets, offsets});
    ctx->saved_data["num_outputs"] = num_outputs;
    ctx->saved_data["num_inputs"] = values.numel();
    ctx->saved_data["batch_size"] = batch_size;
    ctx->saved_data["num_batches"] = num_batches;
    ctx->saved_data["has_weights"] = weights.has_value();

    if (weights.has_value()) {
      return {output, output_lengths, output_weights};
    }
    return {output, output_lengths};
  }

  static torch::autograd::variable_list backward(
      torch::autograd::AutogradContext* ctx,
      torch::autograd::variable_list grad_outputs) {
    bool has_weights = ctx->saved_data["has_weights"].toBool();
    TORCH_CHECK(
        (has_weights && grad_outputs.size() == 3) || grad_outputs.size() == 2);

    const Tensor& grad = grad_outputs[0];
    TENSOR_ON_CUDA_GPU(grad_outputs[0]);

    const auto saved = ctx->get_saved_variables();
    auto savedItr = std::begin(saved);
    const Tensor& indices = *savedItr++;
    const Tensor& grad_offsets = *savedItr++;
    const Tensor& output_offsets = *savedItr++;

    TENSORS_ON_SAME_DEVICE(grad, indices);

    int64_t num_outputs = ctx->saved_data["num_inputs"].toInt();
    int64_t output_batch_size = ctx->saved_data["batch_size"].toInt();
    int64_t num_batches = ctx->saved_data["num_batches"].toInt();

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(grad.get_device());

    Tensor grad_input = at::zeros({num_outputs}, grad.options());
    auto grid_size = cuda_calc_xblock_count(grad.numel(), kMaxThreads);

    if (grid_size != 0) {
      AT_DISPATCH_ALL_TYPES_AND2(
          at::ScalarType::Half,
          at::ScalarType::BFloat16,
          grad.scalar_type(),
          "keyed_jagged_index_add_dim1_wrapper_1",
          [&] {
            AT_DISPATCH_INDEX_TYPES(
                grad_offsets.scalar_type(),
                "keyed_jagged_index_add_dim1_wrapper_2",
                [&] {
                  using offset_t = index_t;
                  AT_DISPATCH_INDEX_TYPES(
                      indices.scalar_type(),
                      "keyed_jagged_index_add_dim1_wrapper_3",
                      [&] {
                        keyed_jagged_index_add_dim1_kernel<<<
                            grid_size,
                            kMaxThreads,
                            0,
                            at::cuda::getCurrentCUDAStream()>>>(
                            grad_input.data_ptr<scalar_t>(),
                            grad.data_ptr<scalar_t>(),
                            grad_offsets.data_ptr<offset_t>(),
                            indices.data_ptr<index_t>(),
                            output_offsets.data_ptr<offset_t>() +
                                1, // shift it to make it inclusive cumsum
                            num_batches,
                            indices.numel(),
                            output_batch_size,
                            grad.numel());
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      });
                });
          });
    }

    return {
        grad_input,
        torch::autograd::Variable(), // lengths
        torch::autograd::Variable(), // offsets
        torch::autograd::Variable(), // indices
        torch::autograd::Variable(), // batch_size
        torch::autograd::Variable() // weights
    };
  }
};
} // namespace

std::vector<Tensor> keyed_jagged_index_select_dim_1_gpu(
    const Tensor& values,
    const Tensor& lengths,
    const Tensor& offsets,
    const Tensor& indices,
    const int64_t batch_size,
    const c10::optional<Tensor>& weights) {
  return KeyedJaggedIndexSelectDim1GPUOp::apply(
      values, lengths, offsets, indices, batch_size, weights);
}

} // namespace fbgemm_gpu

JAGGED_TENSOR_OPS_CUDA_DISPATCH(
    "keyed_jagged_index_select_dim1",
    fbgemm_gpu::keyed_jagged_index_select_dim_1_gpu);
