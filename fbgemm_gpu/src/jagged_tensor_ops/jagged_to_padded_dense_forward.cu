/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

#include "fbgemm_gpu/cub_namespace_postfix.cuh"
#include "fbgemm_gpu/cub_namespace_prefix.cuh"

#include <cub/device/device_scan.cuh>

using Tensor = at::Tensor;

namespace fbgemm_gpu {

///@ingroup jagged-tensor-ops-cuda
at::Tensor jagged_to_padded_dense_forward(
    const Tensor& values,
    const std::vector<Tensor>& offsets,
    c10::SymIntArrayRef max_lengths,
    const double padding_value) {
  const size_t num_jagged_dim = offsets.size();
  TORCH_CHECK(
      max_lengths.size() == num_jagged_dim,
      "max_lengths.size(), ",
      max_lengths.size(),
      " != num_jagged_dim, ",
      num_jagged_dim);
  CUDA_DEVICE_GUARD(values);

  const Tensor values_canonicalized = values.view(
      {values.size(0),
       std::accumulate(
           values.sizes().begin() + 1,
           values.sizes().end(),
           1,
           std::multiplies<size_t>())});
  at::SymDimVector padded_values_shape({at::SymInt(offsets[0].size(0) - 1)});
  padded_values_shape.insert(
      padded_values_shape.end(), max_lengths.begin(), max_lengths.end());

  // Canonicalize padded_values by unsqueeze the last dim if the inner dense
  // dimension is 1 and folded.
  const bool D_folded = values.dim() == 1;
  if (!D_folded) {
    padded_values_shape.push_back(values.size(-1));
  }
  Tensor padded_values =
      at::empty_symint(padded_values_shape, values.options());
  Tensor padded_values_view =
      D_folded ? padded_values.unsqueeze(-1) : padded_values;

  FBGEMM_DISPATCH_ALL_TYPES(
      values.scalar_type(), "jagged_to_padded_dense", [&] {
        jagged_dense_elementwise_dense_output_<scalar_t>(
            values_canonicalized,
            offsets,
            padded_values_view, // dummy not used in the lambda function
            padded_values_view,
            [] __device__(scalar_t x, scalar_t /*unused*/) -> scalar_t {
              return x;
            },
            static_cast<scalar_t>(padding_value));
      });

  return padded_values;
}

std::vector<Tensor> stacked_jagged_1d_to_dense_gpu(
    Tensor values,
    Tensor lengths,
    const std::vector<int64_t>& offset_per_key,
    const std::vector<int64_t>& max_lengths_per_key,
    int64_t padding_value) {
  TORCH_CHECK(values.dim() == 1);
  TORCH_CHECK(lengths.dim() == 2);
  CUDA_DEVICE_GUARD(values);

  const auto lengths_contig = lengths.contiguous();
  int32_t B = lengths.size(1);
  int32_t T = lengths.size(0);
  auto offsets = at::empty({B + 1}, lengths.options());
  offsets[0].zero_();
  std::vector<Tensor> padded_values_per_key;
  for (int32_t t = 0; t < T; t++) {
    int64_t max_L = max_lengths_per_key[t];
    size_t temp_storage_bytes = 0;
    AT_DISPATCH_INDEX_TYPES(
        lengths_contig.scalar_type(), "cub_inclusive_sum_wrapper1", [&] {
          AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
              nullptr,
              temp_storage_bytes,
              &(lengths_contig.data_ptr<index_t>()[t * B]),
              offsets.data_ptr<index_t>() + 1,
              B,
              at::cuda::getCurrentCUDAStream()));
        });
    auto temp_storage = at::empty(
        {static_cast<int64_t>(temp_storage_bytes)},
        lengths.options().dtype(at::kByte));
    AT_DISPATCH_INDEX_TYPES(
        lengths_contig.scalar_type(), "cub_inclusive_sum_wrapper2", [&] {
          AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
              temp_storage.data_ptr(),
              temp_storage_bytes,
              &(lengths_contig.data_ptr<index_t>()[t * B]),
              offsets.data_ptr<index_t>() + 1,
              B,
              at::cuda::getCurrentCUDAStream()));
        });

    padded_values_per_key.push_back(jagged_to_padded_dense_forward(
        values.slice(0, offset_per_key[t], offset_per_key[t + 1]),
        {offsets},
        at::ArrayRef<at::SymInt>({max_L}),
        padding_value));
  }
  return padded_values_per_key;
}

// stacked ops
std::tuple<std::vector<Tensor>, std::vector<Tensor>>
stacked_jagged_2d_to_dense_forward_cuda(
    Tensor values,
    Tensor lengths,
    const std::vector<int64_t>& offset_per_key,
    const std::vector<int64_t>& max_lengths_per_key,
    int64_t padding_value) {
  TORCH_CHECK(values.dim() == 2);
  TORCH_CHECK(lengths.dim() == 2);
  CUDA_DEVICE_GUARD(values);

  const auto lengths_contig = lengths.contiguous();
  int32_t D = values.size(1);
  int32_t B = lengths.size(1);
  int32_t T = lengths.size(0);
  std::vector<Tensor> padded_values_per_key;
  std::vector<Tensor> offsets_tensor_per_key;
  for (int32_t t = 0; t < T; t++) {
    int64_t max_L = max_lengths_per_key[t];
    size_t temp_storage_bytes = 0;
    auto offsets = at::empty({B + 1}, lengths.options());
    offsets[0].zero_();
    AT_DISPATCH_INDEX_TYPES(
        lengths_contig.scalar_type(), "cub_inclusive_sum_wrapper1", [&] {
          AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
              nullptr,
              temp_storage_bytes,
              &(lengths_contig.data_ptr<index_t>()[t * B]),
              offsets.data_ptr<index_t>() + 1,
              B,
              at::cuda::getCurrentCUDAStream()));
        });
    auto temp_storage = at::empty(
        {static_cast<int64_t>(temp_storage_bytes)},
        lengths.options().dtype(at::kByte));
    AT_DISPATCH_INDEX_TYPES(
        lengths_contig.scalar_type(), "cub_inclusive_sum_wrapper2", [&] {
          AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
              temp_storage.data_ptr(),
              temp_storage_bytes,
              &(lengths_contig.data_ptr<index_t>()[t * B]),
              offsets.data_ptr<index_t>() + 1,
              B,
              at::cuda::getCurrentCUDAStream()));
        });
    offsets_tensor_per_key.push_back(offsets);

    padded_values_per_key.push_back(jagged_to_padded_dense_forward(
        values.slice(0, offset_per_key[t], offset_per_key[t + 1]),
        {offsets},
        at::ArrayRef<at::SymInt>({max_L}),
        padding_value));
  }

  return std::make_tuple(padded_values_per_key, offsets_tensor_per_key);
}

Tensor stacked_jagged_2d_to_dense_backward_cuda(
    int64_t B,
    int64_t D,
    int64_t total_L,
    const std::vector<Tensor>& grad_padded_values_per_key,
    const std::vector<Tensor>& offsets_tensor_per_key,
    const std::vector<int64_t>& offset_per_key) {
  CUDA_DEVICE_GUARD(grad_padded_values_per_key[0]);

  auto grad_values =
      at::zeros({total_L, D}, grad_padded_values_per_key[0].options());
  int32_t T = grad_padded_values_per_key.size();
  for (int32_t t = 0; t < T; t++) {
    TORCH_CHECK(grad_padded_values_per_key[t].dim() == 3);
    TORCH_CHECK(grad_padded_values_per_key[t].size(0) == B);
    TORCH_CHECK(grad_padded_values_per_key[t].size(2) == D);

    Tensor grad_values_slice =
        grad_values.slice(0, offset_per_key[t], offset_per_key[t + 1]);

    FBGEMM_DISPATCH_FLOATING_TYPES(
        grad_values.scalar_type(), "jagged_2d_to_dense_backward_kernel", [&] {
          jagged_dense_elementwise_jagged_output_<scalar_t>(
              grad_values_slice, // dummy not used in the lambda function
              {offsets_tensor_per_key[t]},
              grad_padded_values_per_key[t],
              grad_values_slice,
              [] __device__(scalar_t /*unused*/, scalar_t y) -> scalar_t {
                return y;
              });
        });
  }

  return grad_values;
}

namespace {

class StackedJagged2DToDenseGPUOp
    : public torch::autograd::Function<StackedJagged2DToDenseGPUOp> {
 public:
  static torch::autograd::variable_list forward(
      torch::autograd::AutogradContext* ctx,
      Tensor values,
      Tensor lengths,
      const std::vector<int64_t>& offset_per_key,
      const std::vector<int64_t>& max_lengths_per_key,
      int64_t padding_value) {
    int64_t total_L = values.size(0);
    ctx->saved_data["B"] = lengths.size(1);
    ctx->saved_data["D"] = values.size(1);
    ctx->saved_data["total_L"] = total_L;
    ctx->saved_data["offset_per_key"] = offset_per_key;

    auto [padded_values_per_key, offsets_tensor_per_key] =
        stacked_jagged_2d_to_dense_forward_cuda(
            values,
            lengths,
            offset_per_key,
            max_lengths_per_key,
            padding_value);
    ctx->saved_data["offsets_tensor_per_key"] = offsets_tensor_per_key;

    return padded_values_per_key;
  }

  static torch::autograd::variable_list backward(
      torch::autograd::AutogradContext* ctx,
      torch::autograd::variable_list grad_outputs) {
    auto B = ctx->saved_data["B"].toInt();
    auto D = ctx->saved_data["D"].toInt();
    auto total_L = ctx->saved_data["total_L"].toInt();
    auto offset_per_key = ctx->saved_data["offset_per_key"].toIntVector();
    auto offsets_tensor_per_key =
        ctx->saved_data["offsets_tensor_per_key"].toTensorVector();

    using torch::autograd::Variable;
    auto grad_values = stacked_jagged_2d_to_dense_backward_cuda(
        B, D, total_L, grad_outputs, offsets_tensor_per_key, offset_per_key);
    return {
        grad_values,
        Variable(), // lengths
        Variable(), // offset_per_key
        Variable(), // max_lengths_per_key
        Variable(), // padding_value
    };
  }
};
} // namespace

std::vector<Tensor> stacked_jagged_2d_to_dense_gpu(
    Tensor values,
    Tensor lengths,
    const std::vector<int64_t>& offset_per_key,
    const std::vector<int64_t>& max_lengths_per_key,
    int64_t padding_value) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(values, lengths);
  TORCH_CHECK(values.dim() == 2);
  TORCH_CHECK(lengths.dim() == 2);
  return StackedJagged2DToDenseGPUOp::apply(
      values, lengths, offset_per_key, max_lengths_per_key, padding_value);
}

Tensor jagged_2d_to_dense_gpu_forward(
    Tensor values,
    Tensor offsets,
    int64_t max_sequence_length) {
  return jagged_to_padded_dense_forward(
      values,
      {offsets},
      c10::ArrayRef<c10::SymInt>({max_sequence_length}),
      /*padding_value=*/0);
}

namespace {

class JaggedDenseAddJaggedOutputGPUOp
    : public torch::autograd::Function<JaggedDenseAddJaggedOutputGPUOp> {
 public:
  static torch::autograd::variable_list forward(
      torch::autograd::AutogradContext* ctx,
      const Tensor& x_values,
      const std::vector<Tensor>& offsets,
      const Tensor& dense) {
    ctx->save_for_backward(offsets);
    ctx->saved_data["dense_shape"] = dense.sizes();

    auto output = at::empty_like(x_values);

    CUDA_DEVICE_GUARD(dense);

    AT_DISPATCH_SWITCH(
        x_values.scalar_type(),
        "jagged_dense_elementwise_jagged_output_forward",
        AT_DISPATCH_CASE(
            at::ScalarType::Half,
            [&] {
              jagged_dense_elementwise_jagged_output_opt_<scalar_t>(
                  x_values,
                  offsets,
                  dense,
                  output,
                  [] __device__(scalar_t x, scalar_t y) -> scalar_t {
                    return x + y;
                  }); // device lambda
            } // lambda
            ) // CASE
        FBGEMM_DISPATCH_FLOAT_AND_BFLOAT16_CASE([&] {
          jagged_dense_elementwise_jagged_output_<scalar_t>(
              x_values,
              offsets,
              dense,
              output,
              [] __device__(scalar_t x, scalar_t y) -> scalar_t {
                return x + y;
              }); // device lambda
        } // lambda
                                                ) // CASE_FLOATING_TYPES_AND
    ); // SWITCH

    return {output};
  }

  static torch::autograd::variable_list backward(
      torch::autograd::AutogradContext* ctx,
      torch::autograd::variable_list grad_outputs) {
    auto offsets = ctx->get_saved_variables();
    auto dense_shape = ctx->saved_data["dense_shape"].toIntVector();
    TORCH_CHECK(grad_outputs.size() == 1);
    CUDA_DEVICE_GUARD(grad_outputs[0]);

    Tensor dense_values_grad = jagged_to_padded_dense_forward(
        grad_outputs[0],
        offsets,
        c10::fromIntArrayRefKnownNonNegative(std::vector<int64_t>(
            dense_shape.begin() + 1, dense_shape.end() - 1)),
        /*padding_value=*/0);
    TORCH_CHECK(dense_values_grad.sizes() == dense_shape);

    return {
        grad_outputs[0],
        torch::autograd::Variable(), // offsets
        dense_values_grad};
  }
};
} // namespace

///@ingroup jagged-tensor-ops-cuda
/// output = x + y where x is jagged, y is dense, and output is jagged
std::tuple<Tensor, std::vector<Tensor>>
jagged_dense_elementwise_add_jagged_output_cuda(
    const Tensor& x_values,
    const std::vector<Tensor>& x_offsets,
    const Tensor& y) {
  auto sum_values =
      JaggedDenseAddJaggedOutputGPUOp::apply(x_values, x_offsets, y)[0];

  return {sum_values, x_offsets};
}

} // namespace fbgemm_gpu

FBGEMM_OP_DISPATCH(
    CUDA,
    "jagged_to_padded_dense_forward",
    fbgemm_gpu::jagged_to_padded_dense_forward);
FBGEMM_OP_DISPATCH(
    CUDA,
    "stacked_jagged_1d_to_dense",
    fbgemm_gpu::stacked_jagged_1d_to_dense_gpu);
FBGEMM_OP_DISPATCH(
    CUDA,
    "stacked_jagged_2d_to_dense",
    fbgemm_gpu::stacked_jagged_2d_to_dense_gpu);
FBGEMM_OP_DISPATCH(
    CUDA,
    "stacked_jagged_2d_to_dense_forward",
    fbgemm_gpu::stacked_jagged_2d_to_dense_forward_cuda);
FBGEMM_OP_DISPATCH(
    CUDA,
    "stacked_jagged_2d_to_dense_backward",
    fbgemm_gpu::stacked_jagged_2d_to_dense_backward_cuda);
FBGEMM_OP_DISPATCH(
    CUDA,
    "jagged_dense_elementwise_add_jagged_output",
    fbgemm_gpu::jagged_dense_elementwise_add_jagged_output_cuda);
