#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

template <typename index_t, typename scalar_t>
__global__ __launch_bounds__(kMaxThreads) void outer_prod_jagged_2d_output(
    const at::PackedTensorAccessor32<scalar_t, 2> x,
    const at::PackedTensorAccessor32<scalar_t, 2> y,
    const at::PackedTensorAccessor32<index_t, 1> offsets,
    at::PackedTensorAccessor32<scalar_t, 2> output_values) {
  const int B = offsets.size(0) - 1;
  const int H = x.size(0) / B;
  const int max_L = x.size(1);
  const int D = y.size(1);

  const int b_h_l_begin = blockIdx.x * blockDim.y + threadIdx.y;
  const int b_h_l_step = gridDim.x * blockDim.y;
  for (int b_h_l = b_h_l_begin; b_h_l < B * H * max_L; b_h_l += b_h_l_step) {
    const int b_h = b_h_l / max_L;
    const int b = b_h / H;
    const int h = b_h % H;
    const int l = b_h_l % max_L;

    const int row_start = offsets[b];
    const int row_end = offsets[b + 1];
    const int length = row_end - row_start;
    if (l < length) {
      for (int d = threadIdx.x; d < D; d += blockDim.x) {
        output_values[row_start + l][h * D + d] = x[b_h][l] * y[b_h][d];
      }
    }
  }
}

template <typename index_t, typename scalar_t>
__global__
__launch_bounds__(kMaxThreads) void dense_vec_jagged_2d_transposed_bmm(
    const at::PackedTensorAccessor32<scalar_t, 2> v,
    const at::PackedTensorAccessor32<scalar_t, 2> a_values,
    const at::PackedTensorAccessor32<index_t, 1> a_offsets,
    at::PackedTensorAccessor32<scalar_t, 2> output) {
  const int B = a_offsets.size(0) - 1;
  const int H = v.size(0) / B;
  const int max_L = output.size(1);
  const int D = v.size(1);

  const int b_h_begin = blockIdx.x * blockDim.y + threadIdx.y;
  const int b_h_step = gridDim.x * blockDim.y;
  for (int b_h = b_h_begin; b_h < B * H; b_h += b_h_step) {
    const int b = b_h / H;
    const int h = b_h % H;

    const int row_start = a_offsets[b];
    const int row_end = a_offsets[b + 1];
    const int length = std::min(row_end - row_start, max_L);
    if (D == 0) {
      for (int l = threadIdx.x; l < max_L; ++l) {
        output[b_h][l] = 0;
      }
    } else {
      int l;
      for (l = threadIdx.x; l < length; l += blockDim.x) {
        at::acc_type<scalar_t, true> acc =
            v[b_h][0] * a_values[row_start + l][h * D];
        for (int d = 1; d < D; ++d) {
          acc += v[b_h][d] * a_values[row_start + l][h * D + d];
        }
        output[b_h][l] = acc;
      }
      for (; l < max_L; l += blockDim.x) {
        output[b_h][l] = 0;
      }
    }
  }
}

std::tuple<Tensor, Tensor> batched_dense_vec_jagged_2d_mul_backward(
    const Tensor& grad_output,
    const Tensor& v,
    const Tensor& a_values,
    const Tensor& a_offsets) {
  TENSOR_ON_CUDA_GPU(grad_output);
  TENSOR_ON_CUDA_GPU(a_values);
  TENSOR_ON_CUDA_GPU(a_offsets);
  TENSOR_ON_CUDA_GPU(v);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(grad_output.get_device());

  const int B = a_offsets.numel() - 1;
  const int D = grad_output.size(-1);

  Tensor a_values_grad = at::zeros_like(a_values);
  Tensor v_grad = at::empty_like(v);

  if (B > 0 && D > 0) {
    TORCH_CHECK(
        v.size(0) % B == 0, "B, ", B, " doesn't divide v.size(0), ", v.size(0));
    const int H = v.size(0) / B;
    const int max_L = v.size(-1);

    AT_DISPATCH_INDEX_TYPES(
        a_offsets.scalar_type(),
        "dense_vec_jagged_2d_bmm_backward_kernel_1",
        [&] {
          AT_DISPATCH_FLOATING_TYPES_AND2(
              at::ScalarType::Half,
              at::ScalarType::BFloat16,
              grad_output.scalar_type(),
              "dense_vec_jagged_2d_bmm_backward_kernel_2",
              [&] {
                int block_dim_x = std::min(
                    div_round_up(max_L, kWarpSize) * kWarpSize, kMaxThreads);
                int block_dim_y = kMaxThreads / block_dim_x;

                dense_vec_jagged_2d_transposed_bmm<index_t, scalar_t>
                    <<<div_round_up(B * H, block_dim_y),
                       dim3(block_dim_x, block_dim_y),
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        grad_output.packed_accessor32<scalar_t, 2>(),
                        a_values.packed_accessor32<scalar_t, 2>(),
                        a_offsets.packed_accessor32<index_t, 1>(),
                        v_grad.packed_accessor32<scalar_t, 2>());
                C10_CUDA_KERNEL_LAUNCH_CHECK();

                block_dim_x = std::min(
                    div_round_up(D, kWarpSize) * kWarpSize, kMaxThreads);
                block_dim_y = kMaxThreads / block_dim_x;

                outer_prod_jagged_2d_output<index_t, scalar_t>
                    <<<div_round_up(B * H * max_L, block_dim_y),
                       dim3(block_dim_x, block_dim_y),
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        v.packed_accessor32<scalar_t, 2>(),
                        grad_output.packed_accessor32<scalar_t, 2>(),
                        a_offsets.packed_accessor32<index_t, 1>(),
                        a_values_grad.packed_accessor32<scalar_t, 2>());
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              });
        });
  } else {
    v_grad.zero_();
  }

  return {v_grad, a_values_grad};
}

} // namespace fbgemm_gpu

JAGGED_TENSOR_OPS_CUDA_DISPATCH(
    "batched_dense_vec_jagged_2d_mul_backward",
    fbgemm_gpu::batched_dense_vec_jagged_2d_mul_backward);
