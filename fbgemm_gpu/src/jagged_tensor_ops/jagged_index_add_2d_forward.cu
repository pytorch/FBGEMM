#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

template <typename index_t, typename offset_t, typename scalar_t>
__global__ __launch_bounds__(kMaxThreads) void jagged_index_add_2d_kernel(
    at::PackedTensorAccessor64<scalar_t, 2, at::RestrictPtrTraits> output,
    const at::PackedTensorAccessor64<scalar_t, 2, at::RestrictPtrTraits> values,
    const at::PackedTensorAccessor32<offset_t, 1, at::RestrictPtrTraits>
        input_offsets,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<offset_t, 1, at::RestrictPtrTraits>
        output_offsets,
    const int64_t num_dense_input_rows) {
  __shared__ int smem[1];
  for (offset_t dense_input_offset = blockIdx.x;
       dense_input_offset < num_dense_input_rows;
       dense_input_offset += gridDim.x) {
    // Binary search
    // TODO: use multiple threads to do bin search to reduce number of steps
    if (threadIdx.x == 0) {
      const auto num_input_rows = indices.size(0);
      binary_search_range(
          smem, &input_offsets[0], dense_input_offset, num_input_rows);
    }
    __syncthreads();

    // All threads load index_pos from shared memory and return if the index_pos
    // is invalid
    int index_pos = smem[0];

    // TODO: Can also be obtained during the binary search
    // Relative index position
    const offset_t rel_index = dense_input_offset -
        (index_pos == 0 ? 0 : input_offsets[index_pos - 1]);
    const index_t index = indices[index_pos];
    const offset_t output_offset =
        (index == 0 ? 0 : output_offsets[index - 1]) + rel_index;

    // TODO: Avoid using atoimcAdd (because it could lead to the numerical
    // indeterminism issue)
    const auto num_cols = output.size(1);
    for (int i = threadIdx.x; i < num_cols; i += blockDim.x) {
      gpuAtomicAdd(&output[output_offset][i], values[dense_input_offset][i]);
    }
  }
}

/// Add sequences from input jagged tensor to output jagged tensor based on
/// indices specified in the indices tensor (host function for dispatching
/// jagged_index_add_2d_kernel to GPU)
/// @param values               2D dense value tensor of input jagged tensor
/// @param indices              1D tensor that contains indices to be added in
///                             output jagged tensor
/// @param input_offsets        1D tensor that contains offsets of input
///                             jagged tensor
/// @param output_offsets       1D tensor that contains offsets of output
///                             jagged tensor
/// @param num_dense_input_rows The total number of rows in the 2D dense value
///                             tensor of input jagged tensor
/// @param num_output_rows      The number of sequences in jagged output tensor
Tensor jagged_index_add_2d_forward_cuda(
    const Tensor& values,
    const Tensor& indices,
    const Tensor& input_offsets,
    const Tensor& output_offsets,
    const int64_t num_dense_input_rows,
    const int64_t num_output_rows) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(
      values, indices, input_offsets, output_offsets);
  CUDA_DEVICE_GUARD(values);

  auto num_cols = values.size(1);

  const int64_t max_num_blocks = 1024; // Arbitrarily set to this number of now
  const int64_t max_num_threads = kMaxThreads;
  const int64_t num_blocks = std::min(max_num_blocks, num_dense_input_rows);
  const int64_t num_threads = std::min(max_num_threads, num_cols);
  Tensor output = at::zeros({num_output_rows, num_cols}, values.options());

  if (num_blocks > 0) {
    // input_offsets has to be contiguous since it is passed to
    // binary_search_range which accepts raw pointers
    const auto input_offsets_contig = input_offsets.expect_contiguous();
    FBGEMM_DISPATCH_ALL_TYPES(
        values.scalar_type(), "jagged_index_add_2d_kernel_wrapper_1", [&] {
          AT_DISPATCH_INDEX_TYPES(
              indices.scalar_type(),
              "jagged_index_add_2d_kernel_wrapper_2",
              [&] {
                jagged_index_add_2d_kernel<<<
                    dim3(num_blocks),
                    dim3(num_cols),
                    0,
                    at::cuda::getCurrentCUDAStream()>>>(
                    output.packed_accessor64<
                        scalar_t,
                        2,
                        at::RestrictPtrTraits>(),
                    values.packed_accessor64<
                        scalar_t,
                        2,
                        at::RestrictPtrTraits>(),
                    input_offsets_contig->packed_accessor32<
                        int64_t,
                        1,
                        at::RestrictPtrTraits>(),
                    indices
                        .packed_accessor32<index_t, 1, at::RestrictPtrTraits>(),
                    output_offsets
                        .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
                    num_dense_input_rows);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              });
        });
  }

  return output;
}
} // namespace fbgemm_gpu

FBGEMM_OP_DISPATCH(
    CUDA,
    "jagged_index_add_2d_forward",
    fbgemm_gpu::jagged_index_add_2d_forward_cuda);
