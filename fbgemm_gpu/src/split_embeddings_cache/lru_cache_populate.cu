#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

namespace {

template <typename emb_t, typename cache_t>
__global__ __launch_bounds__(kMaxThreads) void lru_cache_insert_kernel(
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> weights,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        cache_hash_size_cumsum,
    const pta::PackedTensorAccessor64<int32_t, 1, at::RestrictPtrTraits>
        cache_index_table_map,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        weights_offsets,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        D_offsets,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_cache_sets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        cache_set_sorted_indices,
    const int32_t* __restrict__ N_unique,
    pta::PackedTensorAccessor32<int64_t, 2, at::RestrictPtrTraits>
        lxu_cache_state,
    pta::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const int64_t time_stamp,
    pta::PackedTensorAccessor32<int64_t, 2, at::RestrictPtrTraits> lru_state,
    const bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args,
    const bool gather_cache_stats,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        uvm_cache_stats,
    const bool lock_cache_line,
    pta::PackedTensorAccessor32<int32_t, 2, at::RestrictPtrTraits>
        lxu_cache_locking_counter) {
  const int32_t C = lxu_cache_state.size(0);
  int32_t n_conflict_misses = 0;
  for (int32_t n = blockIdx.x * blockDim.y + threadIdx.y; n < *N_unique;
       n += gridDim.x * blockDim.y) {
    // check if this warp is responsible for this whole segment.
    const bool segment_start =
        (n == 0 || sorted_cache_sets[n - 1] != sorted_cache_sets[n]);

    if (!segment_start) {
      // don't have *warp* divergence since we launch full warps in blockDim.x,
      // so we can just exit this warp entirely.
      continue;
    }
    const int32_t cache_set = sorted_cache_sets[n];
    if (cache_set == C) {
      // ignore the already-existing elements
      continue;
    }

    int32_t SL = 1;
    while (n + SL < *N_unique && sorted_cache_sets[n + SL] == cache_set) {
      SL += 1;
    }
    int32_t n_inserted = 0; // also used as index to insert

    // now, we need to insert the (unique!) values in indices[n:n + SL] into
    // our slots.
    const int32_t slot = threadIdx.x;
    const int64_t slot_time = lru_state[cache_set][slot];
    int64_t costs[1] = {slot_time};
    int32_t slots[1] = {slot};

    BitonicSort<int64_t, int32_t, 1, Comparator<int64_t>>::sort(costs, slots);
    const int32_t sorted_slot = slots[0];
    const int64_t sorted_lru_cost = costs[0];
    const auto stoc_rounding_salt = kWarpSize *
        (blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
         threadIdx.x);

    for (int32_t l = 0; l < min(SL, kWarpSize); ++l) {
      const int32_t insert_slot = shfl_sync(sorted_slot, l);
      if (lock_cache_line) {
        auto count = lxu_cache_locking_counter[cache_set][insert_slot];
        if (count > 0) {
          continue; // cache slot is in use
        }
      }
      const int64_t insert_current_lru_cost = shfl_sync(sorted_lru_cost, l);
      if (insert_current_lru_cost == time_stamp) {
        break;
      }
      const int64_t insert_idx = cache_set_sorted_indices[n + n_inserted];
      const int32_t t_insert = cache_index_table_map[insert_idx];
      const int64_t idx_insert = insert_idx - cache_hash_size_cumsum[t_insert];
      const int64_t weights_offset_insert = weights_offsets[t_insert];
      const int32_t D_start_insert = D_offsets[t_insert];
      const int32_t D_end_insert = D_offsets[t_insert + 1];
      const int32_t D_insert = D_end_insert - D_start_insert;

      // ensure that threadIdx.x is the only thread reading/writing to
      // lxu_cache_state
      int64_t current_idx =
          threadIdx.x == 0 ? lxu_cache_state[cache_set][insert_slot] : 0;
      current_idx = shfl_sync(current_idx, 0);

      // not empty
      if (current_idx != static_cast<int64_t>(kCacheStateInvalid)) {
        // evict from slot to backing storage
        const int32_t t_current = cache_index_table_map[current_idx];
        const int64_t idx_current =
            current_idx - cache_hash_size_cumsum[t_current];
        const int64_t weights_offset_current = weights_offsets[t_current];
        const int32_t D_start_current = D_offsets[t_current];
        const int32_t D_end_current = D_offsets[t_current + 1];
        const int32_t D_current = D_end_current - D_start_current;
        int32_t D_emb = D_current;
        if constexpr (std::is_same_v<emb_t, uint8_t>) {
          D_emb += kINT8QparamsBytes;
        }

        StochasticRoundingRNGState state;
        auto weight_row = WeightRow<emb_t, cache_t, cache_t>(
            &weights[weights_offset_current + idx_current * D_emb + 0],
            &lxu_cache_weights[cache_set * kWarpSize + insert_slot][0],
            D_current,
            stochastic_rounding ? &state : nullptr,
            &stochastic_rounding_philox_args,
            stoc_rounding_salt + l);

        weight_row.warp_evict_cache(D_current, blockDim.x, threadIdx.x);
      }

      int32_t D_emb = D_insert;
      if constexpr (std::is_same_v<emb_t, uint8_t>) {
        D_emb += kINT8QparamsBytes;
      }

      auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
          &weights[weights_offset_insert + idx_insert * D_emb + 0],
          nullptr,
          D_insert);

      weight_row_emb.warp_copy_to_cache(
          &lxu_cache_weights[cache_set * kWarpSize + insert_slot][0],
          D_insert,
          blockDim.x,
          threadIdx.x);

      if (threadIdx.x == 0) {
        lxu_cache_state[cache_set][insert_slot] = insert_idx;
        lru_state[cache_set][insert_slot] = time_stamp;
        if (lock_cache_line) {
          lxu_cache_locking_counter[cache_set][insert_slot] += 1;
        }
      }

      n_inserted++;
    }
    n_conflict_misses += (SL - n_inserted);
  }
  if (gather_cache_stats && n_conflict_misses > 0 && threadIdx.x == 0) {
    atomicAdd(
        &uvm_cache_stats[uvm_cache_stats_index::num_conflict_unique_misses],
        n_conflict_misses);
  }
}

void lru_cache_insert_cuda(
    Tensor weights,
    Tensor cache_hash_size_cumsum,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    Tensor sorted_cache_sets,
    Tensor cache_set_sorted_unique_indices,
    Tensor unique_indices_length,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    const int64_t time_stamp,
    Tensor lru_state,
    const bool stochastic_rounding,
    bool gather_cache_stats,
    Tensor uvm_cache_stats,
    bool lock_cache_line,
    Tensor lxu_cache_locking_counter) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(
      weights,
      cache_hash_size_cumsum,
      cache_index_table_map,
      weights_offsets,
      D_offsets,
      sorted_cache_sets,
      cache_set_sorted_unique_indices,
      unique_indices_length,
      lxu_cache_state,
      lxu_cache_weights,
      lru_state,
      uvm_cache_stats,
      lxu_cache_locking_counter);

  CUDA_DEVICE_GUARD(weights);

  const int32_t N = cache_set_sorted_unique_indices.numel();
  DISPATCH_EMB_CACHE_TYPES(
      weights.scalar_type(),
      lxu_cache_weights.scalar_type(),
      "lru_cache_insert_kernel_2",
      ([&] {
        // Stochastic rounding is required only when emb_t and cache_t are
        // not the same type and emb_t is not float
        const bool stochastic_rounding_ = stochastic_rounding &&
            !std::is_same<emb_t, float>::value &&
            !std::is_same<emb_t, cache_t>::value;

        at::PhiloxCudaState rng_engine_inputs;
        if (stochastic_rounding_) {
          auto gen = at::cuda::detail::getDefaultCUDAGenerator();
          std::lock_guard<std::mutex> lock(gen.mutex());
          rng_engine_inputs = at::check_generator<at::CUDAGeneratorImpl>(gen)
                                  ->philox_cuda_state(4);
        }

        // During concurrent prefetch, cache lines are locked and we use less
        // SMs for some of the prefetch kernels (e.g. insert)
        // since it is not SM bound. It leaves SMs for main stream to overlap
        constexpr int ALL_TO_PREFETCH_SM_RATIO = 8;

        auto grid_size = lock_cache_line
            ? div_round_up(get_device_sm_cnt_(), ALL_TO_PREFETCH_SM_RATIO)
            : div_round_up(N, kMaxThreads / kWarpSize);

#ifdef FBGEMM_GPU_MEMCHECK
        const char* func_name = "lru_cache_insert_kernel";
#endif
        lru_cache_insert_kernel<emb_t, cache_t>
            <<<grid_size,
               dim3(kWarpSize, kMaxThreads / kWarpSize),
               0,
               at::cuda::getCurrentCUDAStream()>>>(
                MAKE_PTA_WITH_NAME(func_name, weights, emb_t, 1, 64),
                MAKE_PTA_WITH_NAME(
                    func_name, cache_hash_size_cumsum, int64_t, 1, 32),
                MAKE_PTA_WITH_NAME(
                    func_name, cache_index_table_map, int32_t, 1, 64),
                MAKE_PTA_WITH_NAME(func_name, weights_offsets, int64_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name, D_offsets, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(
                    func_name, sorted_cache_sets, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(
                    func_name, cache_set_sorted_unique_indices, int64_t, 1, 32),
                unique_indices_length.data_ptr<int32_t>(),
                MAKE_PTA_WITH_NAME(func_name, lxu_cache_state, int64_t, 2, 32),
                MAKE_PTA_WITH_NAME(
                    func_name, lxu_cache_weights, cache_t, 2, 64),
                time_stamp,
                MAKE_PTA_WITH_NAME(func_name, lru_state, int64_t, 2, 32),
                stochastic_rounding_,
                rng_engine_inputs,
                gather_cache_stats,
                MAKE_PTA_WITH_NAME(func_name, uvm_cache_stats, int32_t, 1, 32),
                lock_cache_line,
                MAKE_PTA_WITH_NAME(
                    func_name, lxu_cache_locking_counter, int32_t, 2, 32));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
}

} // namespace

DLL_PUBLIC void lru_cache_populate_cuda(
    Tensor weights,
    Tensor cache_hash_size_cumsum,
    const int64_t total_cache_hash_size,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    Tensor linear_cache_indices,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    const int64_t time_stamp,
    Tensor lru_state,
    const bool stochastic_rounding,
    bool gather_cache_stats,
    c10::optional<Tensor> uvm_cache_stats,
    bool lock_cache_line,
    c10::optional<Tensor> lxu_cache_locking_counter) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(
      weights,
      cache_hash_size_cumsum,
      cache_index_table_map,
      weights_offsets,
      D_offsets,
      linear_cache_indices,
      lxu_cache_state,
      lxu_cache_weights,
      lru_state);

  Tensor uvm_cache_stats_ = at::empty({0}, weights.options().dtype(at::kInt));
  if (gather_cache_stats) {
    TORCH_CHECK(uvm_cache_stats.has_value());
    uvm_cache_stats_ = uvm_cache_stats.value();
    TENSOR_ON_CUDA_GPU(uvm_cache_stats_);
  }

  Tensor lxu_cache_locking_counter_ =
      at::empty({0, 0}, lxu_cache_state.options().dtype(at::kInt));
  if (lock_cache_line) {
    TORCH_CHECK(lxu_cache_locking_counter.has_value());
    lxu_cache_locking_counter_ = lxu_cache_locking_counter.value();
    TENSOR_ON_CUDA_GPU(lxu_cache_locking_counter_);
  }

  CUDA_DEVICE_GUARD(weights);

  TORCH_CHECK(
      linear_cache_indices.numel() < std::numeric_limits<int32_t>::max());
  if (linear_cache_indices.numel() == 0) {
    // nothing to do
    return;
  }

  // Get unqiue indices
  auto [unique_indices, unique_indices_length, unique_indices_count] =
      get_unique_indices_cuda(
          linear_cache_indices,
          total_cache_hash_size,
          /*compute_count=*/false);

  auto
      [sorted_cache_sets,
       cache_set_sorted_unique_indices,
       cache_set_inverse_indices] =
          lru_cache_find_uncached_cuda(
              unique_indices,
              unique_indices_length,
              total_cache_hash_size,
              lxu_cache_state,
              time_stamp,
              lru_state,
              gather_cache_stats,
              uvm_cache_stats_,
              lock_cache_line,
              lxu_cache_locking_counter_,
              /*compute_inverse_indices=*/false);

  // insert caching weights
  lru_cache_insert_cuda(
      weights,
      cache_hash_size_cumsum,
      cache_index_table_map,
      weights_offsets,
      D_offsets,
      sorted_cache_sets,
      cache_set_sorted_unique_indices,
      unique_indices_length,
      lxu_cache_state,
      lxu_cache_weights,
      time_stamp,
      lru_state,
      stochastic_rounding,
      gather_cache_stats,
      uvm_cache_stats_,
      lock_cache_line,
      lxu_cache_locking_counter_);
}
