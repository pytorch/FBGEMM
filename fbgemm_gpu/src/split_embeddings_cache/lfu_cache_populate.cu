#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

namespace {

template <typename emb_t, typename cache_t>
__global__ __launch_bounds__(kCacheMaxThreads) void lfu_cache_insert_kernel(
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> weights,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        cache_hash_size_cumsum,
    const pta::PackedTensorAccessor64<int32_t, 1, at::RestrictPtrTraits>
        cache_index_table_map,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        weights_offsets,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        D_offsets,
    const uint64_t* __restrict__ sorted_cache_sets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        cache_set_sorted_indices,
    const int32_t* __restrict__ N_unique,
    pta::PackedTensorAccessor32<int64_t, 2, at::RestrictPtrTraits>
        lxu_cache_state,
    pta::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const pta::PackedTensorAccessor64<int64_t, 1, at::RestrictPtrTraits>
        lfu_state,
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args) {
  const int32_t C = lxu_cache_state.size(0);
  for (int32_t n = blockIdx.x * blockDim.y + threadIdx.y; n < *N_unique;
       n += gridDim.x * blockDim.y) {
    // check if this warp is responsible for this whole segment.
    const bool segment_start =
        (n == 0 ||
         (sorted_cache_sets[n - 1] >> kLFUCounterBits) !=
             (sorted_cache_sets[n] >> kLFUCounterBits));

    if (!segment_start) {
      // don't have *warp* divergence since we launch full warps in blockDim.x,
      // so we can just exit this warp entirely.
      continue;
    }
    const uint32_t cache_set = (sorted_cache_sets[n] >> kLFUCounterBits);
    if (cache_set == C) {
      // ignore the already-existing elements
      continue;
    }

    int32_t SL = 1;
    while (n + SL < *N_unique &&
           (sorted_cache_sets[n + SL] >> kLFUCounterBits) == cache_set) {
      SL += 1;
    }

    // now, we need to insert the (unique!) values in indices[n:n + SL] into
    // our slots.
    const int32_t slot = threadIdx.x;
    const int64_t current_idx = lxu_cache_state[cache_set][slot];
    const int64_t current_lfu_cost =
        (current_idx != static_cast<int64_t>(kCacheStateInvalid))
        ? lfu_state[current_idx]
        : -1;
    int64_t costs[1] = {current_lfu_cost};
    int32_t slots[1] = {slot};

    BitonicSort<int64_t, int32_t, 1, Comparator<int64_t>>::sort(costs, slots);
    const int32_t sorted_slot = slots[0];
    const int64_t sorted_lfu_cost = costs[0];

    for (int32_t l = 0; l < min(SL, kWarpSize); ++l) {
      const int32_t insert_slot = shfl_sync(sorted_slot, l);
      const int64_t insert_current_lfu_cost = shfl_sync(sorted_lfu_cost, l);
      const int64_t insert_idx = cache_set_sorted_indices[n + l];
      const int64_t insert_lfu_cost = lfu_state[insert_idx];

      if (insert_current_lfu_cost > insert_lfu_cost) {
        // don't insert.
        // all subsequent `current_lfu_cost` values are greater, and all
        // subsequent `insert_lfu_cost` values are smaller, so we can exit
        // early here.
        break;
      }
      const int32_t t_insert = cache_index_table_map[insert_idx];
      const int64_t idx_insert = insert_idx - cache_hash_size_cumsum[t_insert];
      const int64_t weights_offset_insert = weights_offsets[t_insert];
      const int32_t D_start_insert = D_offsets[t_insert];
      const int32_t D_end_insert = D_offsets[t_insert + 1];
      const int32_t D_insert = D_end_insert - D_start_insert;

      // not empty
      if (insert_current_lfu_cost != -1) {
        // ensure that threadIdx.x is the only thread reading/writing to
        // lxu_cache_state
        int64_t current_idx =
            threadIdx.x == 0 ? lxu_cache_state[cache_set][insert_slot] : 0;
        current_idx = shfl_sync(current_idx, 0);
        const int32_t t_current = cache_index_table_map[current_idx];
        const int64_t idx_current =
            current_idx - cache_hash_size_cumsum[t_current];
        const int64_t weights_offset_current = weights_offsets[t_current];
        const int32_t D_start_current = D_offsets[t_current];
        const int32_t D_end_current = D_offsets[t_current + 1];
        const int32_t D_current = D_end_current - D_start_current;

        int32_t D_emb = D_current;
        if constexpr (std::is_same_v<emb_t, uint8_t>) {
          D_emb += kINT8QparamsBytes;
        }
        StochasticRoundingRNGState state;
        auto weight_row = WeightRow<emb_t, cache_t, cache_t>(
            &weights[weights_offset_current + idx_current * D_emb + 0],
            &lxu_cache_weights[cache_set * kWarpSize + insert_slot][0],
            D_current,
            stochastic_rounding ? &state : nullptr,
            &stochastic_rounding_philox_args,
            (blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
             threadIdx.x) *
                    kWarpSize +
                l);

        weight_row.warp_evict_cache(D_current, blockDim.x, threadIdx.x);
      }

      // insert into cache
      int32_t D_emb = D_insert;
      if constexpr (std::is_same_v<emb_t, uint8_t>) {
        D_emb += kINT8QparamsBytes;
      }

      auto weight_row_emb = WeightRow<emb_t, cache_t, cache_t>(
          &weights[weights_offset_insert + idx_insert * D_emb + 0],
          nullptr,
          D_insert);

      weight_row_emb.warp_copy_to_cache(
          &lxu_cache_weights[cache_set * kWarpSize + insert_slot][0],
          D_insert,
          blockDim.x,
          threadIdx.x);

      if (threadIdx.x == 0) {
        lxu_cache_state[cache_set][insert_slot] = insert_idx;
      }
    }
  }
}

void lfu_cache_insert_cuda(
    Tensor weights,
    Tensor cache_hash_size_cumsum,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    Tensor sorted_cache_sets,
    Tensor cache_set_sorted_unique_indices,
    Tensor unique_indices_length,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    Tensor lfu_state,
    bool stochastic_rounding) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(
      weights,
      cache_hash_size_cumsum,
      cache_index_table_map,
      weights_offsets,
      D_offsets,
      sorted_cache_sets,
      cache_set_sorted_unique_indices,
      unique_indices_length,
      lxu_cache_state,
      lxu_cache_weights,
      lfu_state);

  CUDA_DEVICE_GUARD(weights);

  const int32_t N = cache_set_sorted_unique_indices.numel();

  DISPATCH_EMB_CACHE_TYPES(
      weights.scalar_type(),
      lxu_cache_weights.scalar_type(),
      "lfu_cache_insert_kernel_2",
      ([&] {
        // Stochastic rounding is required only when emb_t and cache_t are
        // not the same type and emb_t is not float
        const bool stochastic_rounding_ = stochastic_rounding &&
            !std::is_same<emb_t, float>::value &&
            !std::is_same<emb_t, cache_t>::value;

        at::PhiloxCudaState rng_engine_inputs;
        if (stochastic_rounding_) {
          auto gen = at::cuda::detail::getDefaultCUDAGenerator();
          std::lock_guard<std::mutex> lock(gen.mutex());
          rng_engine_inputs = at::check_generator<at::CUDAGeneratorImpl>(gen)
                                  ->philox_cuda_state(4);
        }

#ifdef FBGEMM_GPU_MEMCHECK
        const char* func_name = "lfu_cache_insert_kernel";
#endif

        lfu_cache_insert_kernel<emb_t, cache_t>
            <<<std::min(
                   div_round_up(N, kCacheMaxThreads / kWarpSize),
                   get_max_thread_blocks_for_cache_kernels_()),
               dim3(kWarpSize, kCacheMaxThreads / kWarpSize),
               0,
               at::cuda::getCurrentCUDAStream()>>>(
                MAKE_PTA_WITH_NAME(func_name, weights, emb_t, 1, 64),
                MAKE_PTA_WITH_NAME(
                    func_name, cache_hash_size_cumsum, int64_t, 1, 32),
                MAKE_PTA_WITH_NAME(
                    func_name, cache_index_table_map, int32_t, 1, 64),
                MAKE_PTA_WITH_NAME(func_name, weights_offsets, int64_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name, D_offsets, int32_t, 1, 32),
                (uint64_t*)sorted_cache_sets.data_ptr<int64_t>(),
                MAKE_PTA_WITH_NAME(
                    func_name, cache_set_sorted_unique_indices, int64_t, 1, 32),
                unique_indices_length.data_ptr<int32_t>(),
                MAKE_PTA_WITH_NAME(func_name, lxu_cache_state, int64_t, 2, 32),
                MAKE_PTA_WITH_NAME(
                    func_name, lxu_cache_weights, cache_t, 2, 64),
                MAKE_PTA_WITH_NAME(func_name, lfu_state, int64_t, 1, 64),
                stochastic_rounding_,
                rng_engine_inputs);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
}

} // namespace

DLL_PUBLIC void lfu_cache_populate_cuda(
    Tensor weights,
    Tensor cache_hash_size_cumsum,
    int64_t total_cache_hash_size,
    Tensor cache_index_table_map,
    Tensor weights_offsets,
    Tensor D_offsets,
    Tensor linear_cache_indices,
    Tensor lxu_cache_state,
    Tensor lxu_cache_weights,
    Tensor lfu_state,
    bool stochastic_rounding) {
  TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(
      weights,
      cache_hash_size_cumsum,
      cache_index_table_map,
      weights_offsets,
      D_offsets,
      linear_cache_indices,
      lxu_cache_state,
      lxu_cache_weights,
      lfu_state);

  CUDA_DEVICE_GUARD(weights);

  TORCH_CHECK(
      linear_cache_indices.numel() < std::numeric_limits<int32_t>::max());
  if (linear_cache_indices.numel() == 0) {
    // nothing to do
    return;
  }

  // get unqiue indices
  auto [unique_indices, unique_indices_length, unique_indices_count] =
      get_unique_indices_cuda(
          linear_cache_indices,
          total_cache_hash_size,
          /*compute_count=*/true);

  // update lfu counts
  lfu_update_counts_cuda(
      unique_indices, unique_indices_length, *unique_indices_count, lfu_state);

  // find uncached indices
  auto cache_sets_and_unique_indices = lfu_cache_find_uncached_cuda(
      unique_indices,
      unique_indices_length,
      total_cache_hash_size,
      lxu_cache_state,
      lfu_state);
  const auto sorted_cache_sets = cache_sets_and_unique_indices.first;
  const auto cache_set_sorted_unique_indices =
      cache_sets_and_unique_indices.second;

  // insert caching weights
  lfu_cache_insert_cuda(
      weights,
      cache_hash_size_cumsum,
      cache_index_table_map,
      weights_offsets,
      D_offsets,
      sorted_cache_sets,
      cache_set_sorted_unique_indices,
      unique_indices_length,
      lxu_cache_state,
      lxu_cache_weights,
      lfu_state,
      stochastic_rounding);
}
