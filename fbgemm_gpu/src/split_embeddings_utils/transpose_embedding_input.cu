#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "fbgemm_gpu/embedding_backward_template_helpers.cuh" // @manual
#include "fbgemm_gpu/ops_utils.h" // @manual
#include "fbgemm_gpu/split_embeddings_utils.cuh" // @manual

// clang-format off
#include "fbgemm_gpu/cub_namespace_prefix.cuh" // @manual
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_scan.cuh>
#include "fbgemm_gpu/cub_namespace_postfix.cuh" // @manual
// clang-format on

using Tensor = at::Tensor;
using namespace fbgemm_gpu;

inline at::Tensor asynchronous_complete_cumsum(at::Tensor t_in) {
  CUDA_DEVICE_GUARD(t_in);

  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == at::kInt || t_in.dtype() == at::kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK_LT(t_in.numel(), std::numeric_limits<int32_t>::max());
  TORCH_CHECK_EQ(t_in.dim(), 1);
  auto t_out = at::empty({t_in.numel() + 1}, t_in.options());
  t_out[0].zero_();
  AT_DISPATCH_INDEX_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper1", [&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<index_t>(),
            t_out.data_ptr<index_t>() + 1,
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      });
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      t_in.options().dtype(at::kByte));
  AT_DISPATCH_INDEX_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper2", [&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<index_t>(),
            t_out.data_ptr<index_t>() + 1,
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      });
  return t_out;
}

template <typename index_t, typename info_acc_t, bool nobag, bool vbe>
__global__ __launch_bounds__(kMaxThreads) void linearize_index_kernel(
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        hash_size_cumsum,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    at::PackedTensorAccessor32<info_acc_t, 1, at::RestrictPtrTraits> infos,
    at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        linear_indices,
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    const uint32_t max_T,
    const uint32_t max_B,
    // Use a raw pointer to avoid creating dummy PackedTensorAccessor
    const uint32_t* const __restrict__ vbe_b_t_map,
    FixedDivisor fd) {
  const int32_t T = hash_size_cumsum.size(0) - 1;
  auto b_t = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t b;
  int32_t t;
  const auto total_B = offsets.size(0) - 1;
  bool valid = b_t < total_B;
  // info must be uint32_t (using auto will assign int32_t to info)
  uint32_t info = 0;

  if (vbe && valid) {
    info = vbe_b_t_map[b_t];
    reinterpret_cast<uint32_t*>(&t)[0] = info >> info_B_num_bits;
    reinterpret_cast<uint32_t*>(&b)[0] = info & info_B_mask;
  } else {
    fd.DivMod(b_t, &t, &b);
  }

  const index_t hash_offset = valid ? hash_size_cumsum[t] : -1;
  const index_t indices_start = valid ? offsets[b_t] : -1;
  const int32_t L = valid ? offsets[b_t + 1] - indices_start : 0;
  const int32_t lane_id = threadIdx.x % fbgemm_gpu::kWarpSize;

  // Compile-time conditional
  if (nobag) {
    for (int32_t j = 0; j < fbgemm_gpu::kWarpSize; ++j) {
      const index_t indices_start_warp =
          fbgemm_gpu::shfl_sync(indices_start, j);
      const int32_t t_warp = fbgemm_gpu::shfl_sync(t, j);
      const int32_t L_warp = fbgemm_gpu::shfl_sync(L, j);
      const index_t hash_offset_warp = fbgemm_gpu::shfl_sync(hash_offset, j);
      for (int32_t i = lane_id; i < L_warp; i += fbgemm_gpu::kWarpSize) {
        const index_t idx = __ldg(&indices[indices_start_warp + i]);
        const int64_t l_t = (indices_start_warp + i) * T + t_warp;
        infos[indices_start_warp + i] = l_t;
        linear_indices[indices_start_warp + i] = hash_offset_warp + idx;
      }
    }
  } else {
    // Store t in upper (32 - DEFAULT_INFO_B_NUM_BITS).
    // Store b in lower (DEFAULT_INFO_B_NUM_BITS).
    if (!vbe && valid) {
      info = (reinterpret_cast<uint32_t*>(&t)[0] << info_B_num_bits) |
          reinterpret_cast<uint32_t*>(&b)[0];
    }
    for (int32_t j = 0; j < fbgemm_gpu::kWarpSize; ++j) {
      const index_t indices_start_warp =
          fbgemm_gpu::shfl_sync(indices_start, j);
      const uint32_t info_warp = fbgemm_gpu::shfl_sync(info, j);
      const int32_t L_warp = fbgemm_gpu::shfl_sync(L, j);
      const index_t hash_offset_warp = fbgemm_gpu::shfl_sync(hash_offset, j);
      for (int32_t i = lane_id; i < L_warp; i += fbgemm_gpu::kWarpSize) {
        const index_t idx = __ldg(&indices[indices_start_warp + i]);
        reinterpret_cast<uint32_t*>(&infos[0])[indices_start_warp + i] =
            info_warp;
        linear_indices[indices_start_warp + i] = hash_offset_warp + idx;
      }
    }
  }
}

template <typename index_t, typename info_acc_t>
__global__
__launch_bounds__(kMaxThreads) void linearize_index_index_select_kernel(
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        hash_size_cumsum,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        total_L_offsets,
    at::PackedTensorAccessor32<info_acc_t, 1, at::RestrictPtrTraits> infos,
    at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        linear_indices,
    FixedDivisor fd,
    int32_t fixed_L_per_warp) {
  const int32_t T = hash_size_cumsum.size(0) - 1;
  auto b_t = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t b;
  int32_t t;

  fd.DivMod(b_t, &t, &b);

  const int32_t lane_id = threadIdx.x % fbgemm_gpu::kWarpSize;

  index_t hash_offset = -1;
  index_t indices_start = -1;
  int32_t L = 0;
  int32_t L_start = 0;
  if (t < T) {
    const auto total_L_start = total_L_offsets[t];
    const auto total_L = total_L_offsets[t + 1] - total_L_start;
    L_start = b * fixed_L_per_warp;
    if (L_start < total_L) {
      hash_offset = hash_size_cumsum[t];
      indices_start = total_L_start + L_start;
      L = (total_L - L_start >= fixed_L_per_warp) ? fixed_L_per_warp
                                                  : (total_L - L_start);
    }
  }

  // Compile-time conditional
  for (int32_t j = 0; j < fbgemm_gpu::kWarpSize; ++j) {
    const index_t indices_start_warp = fbgemm_gpu::shfl_sync(indices_start, j);
    const auto t_warp = fbgemm_gpu::shfl_sync(t, j);
    const auto L_warp = fbgemm_gpu::shfl_sync(L, j);
    const auto L_start_warp = fbgemm_gpu::shfl_sync(L_start, j);
    const index_t hash_offset_warp = fbgemm_gpu::shfl_sync(hash_offset, j);
    for (int32_t i = lane_id; i < L_warp; i += fbgemm_gpu::kWarpSize) {
      const index_t idx = __ldg(&indices[indices_start_warp + i]);
      // l is the relative l in the feature (i.e., the first l in the feature
      // is 0)
      const int64_t l_t = (L_start_warp + i) * T + t_warp;
      infos[indices_start_warp + i] = l_t;
      linear_indices[indices_start_warp + i] = hash_offset_warp + idx;
    }
  }
}

DLL_PUBLIC std::tuple<
    Tensor /*linear_indices*/,
    Tensor /*linear_indices_sorted*/,
    Tensor /*infos_sorted*/,
    Tensor /*sorted_linear_indices_run*/,
    Tensor /*sorted_linear_indices_run_lengths*/,
    Tensor /*sorted_linear_indices_num_runs*/,
    Tensor /*sorted_linear_indices_cumulative_run_lengths*/>
transpose_embedding_input(
    Tensor hash_size_cumsum,
    int64_t total_hash_size_bits,
    Tensor indices,
    Tensor offsets,
    bool nobag,
    const c10::optional<Tensor>& vbe_b_t_map,
    const int64_t info_B_num_bits,
    const int64_t info_B_mask,
    const int64_t total_unique_indices,
    const bool is_index_select,
    const c10::optional<Tensor>& total_L_offsets,
    const int64_t fixed_L_per_warp,
    const int64_t num_warps_per_feature) {
  const bool vbe = vbe_b_t_map.has_value();
  TORCH_CHECK(nobag || !vbe || info_B_num_bits > 0);
  TORCH_CHECK(!vbe || info_B_mask > 0);
  TORCH_CHECK(
      !is_index_select || (fixed_L_per_warp > 0 && num_warps_per_feature > 0));

  const auto T = hash_size_cumsum.size(0) - 1;
  const auto total_B =
      !is_index_select ? (offsets.size(0) - 1) : (num_warps_per_feature * T);

  TORCH_CHECK(
      !is_index_select ||
      (total_L_offsets.has_value() &&
       total_L_offsets.value().numel() == T + 1));

  auto infos = at::empty_like(
      indices,
      indices.options().dtype(
          (nobag || is_index_select) ? at::kLong : at::kInt));
  auto infos_sorted = at::empty_like(infos);
  auto linear_indices = at::empty_like(indices);
  auto linear_indices_sorted = at::empty_like(indices);

  Tensor sorted_linear_indices_run;
  Tensor sorted_linear_indices_run_lengths;
  Tensor sorted_linear_indices_num_runs;

  using at::RestrictPtrTraits;

#define INVOKE_LINEARIZE_INDEX_KERNEL(INFO_ACC_T, NOBAG)                   \
  const auto linearize_index_kernel_ =                                     \
      (vbe ? linearize_index_kernel<index_t, INFO_ACC_T, NOBAG, true>      \
           : linearize_index_kernel<index_t, INFO_ACC_T, NOBAG, false>);   \
  linearize_index_kernel_<<<                                               \
      div_round_up(total_B, kMaxThreads),                                  \
      kMaxThreads,                                                         \
      0,                                                                   \
      at::cuda::getCurrentCUDAStream()>>>(                                 \
      hash_size_cumsum.packed_accessor32<index_t, 1, RestrictPtrTraits>(), \
      indices.packed_accessor32<index_t, 1, RestrictPtrTraits>(),          \
      offsets.packed_accessor32<index_t, 1, RestrictPtrTraits>(),          \
      infos.packed_accessor32<INFO_ACC_T, 1, RestrictPtrTraits>(),         \
      linear_indices.packed_accessor32<index_t, 1, RestrictPtrTraits>(),   \
      info_B_num_bits,                                                     \
      info_B_mask,                                                         \
      (1u << (DEFAULT_INFO_NUM_BITS - info_B_num_bits)) - 1,               \
      (1u << info_B_num_bits) - 1,                                         \
      vbe ? reinterpret_cast<uint32_t*>(vbe_b_t_map.value().data_ptr())    \
          : nullptr,                                                       \
      FixedDivisor(total_B / T));                                          \
  C10_CUDA_KERNEL_LAUNCH_CHECK()

  AT_DISPATCH_INDEX_TYPES(
      infos.scalar_type(), "transpose_embedding_input1", [&] {
        using info_t = index_t;
        AT_DISPATCH_INDEX_TYPES(
            indices.scalar_type(), "transpose_embedding_input2", [&] {
              if (!is_index_select) {
                if (!nobag) {
                  INVOKE_LINEARIZE_INDEX_KERNEL(int32_t, false);
                } else {
                  INVOKE_LINEARIZE_INDEX_KERNEL(int64_t, true);
                }
              } else {
                // index_select is a special case of TBE (dense, nobag, with
                // fixed_L_per_warp)
                linearize_index_index_select_kernel<<<
                    div_round_up(total_B, kMaxThreads),
                    kMaxThreads,
                    0,
                    at::cuda::getCurrentCUDAStream()>>>(
                    hash_size_cumsum
                        .packed_accessor32<index_t, 1, RestrictPtrTraits>(),
                    indices.packed_accessor32<index_t, 1, RestrictPtrTraits>(),
                    total_L_offsets.value()
                        .packed_accessor32<index_t, 1, RestrictPtrTraits>(),
                    infos.packed_accessor32<int64_t, 1, RestrictPtrTraits>(),
                    linear_indices
                        .packed_accessor32<index_t, 1, RestrictPtrTraits>(),
                    FixedDivisor(total_B / T),
                    fixed_L_per_warp);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
              {
                size_t temp_storage_bytes = 0;
                AT_CUDA_CHECK(
                    FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceRadixSort::SortPairs(
                        nullptr,
                        temp_storage_bytes,
                        linear_indices.data_ptr<index_t>(),
                        linear_indices_sorted.data_ptr<index_t>(),
                        infos.data_ptr<info_t>(),
                        infos_sorted.data_ptr<info_t>(),
                        linear_indices.numel(),
                        0,
                        total_hash_size_bits,
                        at::cuda::getCurrentCUDAStream(),
                        false));
                auto temp_storage = at::empty(
                    {static_cast<int64_t>(temp_storage_bytes)},
                    indices.options().dtype(at::kByte));
                AT_CUDA_CHECK(
                    FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceRadixSort::SortPairs(
                        temp_storage.data_ptr(),
                        temp_storage_bytes,
                        linear_indices.data_ptr<index_t>(),
                        linear_indices_sorted.data_ptr<index_t>(),
                        infos.data_ptr<info_t>(),
                        infos_sorted.data_ptr<info_t>(),
                        linear_indices.numel(),
                        0,
                        total_hash_size_bits,
                        at::cuda::getCurrentCUDAStream(),
                        false));
              }
              if (total_unique_indices != -1) {
                TORCH_CHECK(total_unique_indices >= 0);
                sorted_linear_indices_run =
                    at::empty({total_unique_indices}, indices.options());
                sorted_linear_indices_run_lengths = at::zeros(
                    {total_unique_indices}, indices.options().dtype(at::kInt));
              } else {
                sorted_linear_indices_run = at::empty_like(indices);
                sorted_linear_indices_run_lengths =
                    at::zeros_like(indices, indices.options().dtype(at::kInt));
              }
              sorted_linear_indices_num_runs =
                  at::zeros({1}, indices.options().dtype(at::kInt));

              {
                size_t temp_storage_bytes = 0;
                AT_CUDA_CHECK(
                    FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceRunLengthEncode::Encode(
                        nullptr,
                        temp_storage_bytes,
                        linear_indices_sorted.data_ptr<index_t>(),
                        sorted_linear_indices_run.data_ptr<index_t>(),
                        sorted_linear_indices_run_lengths.data_ptr<int32_t>(),
                        sorted_linear_indices_num_runs.data_ptr<int32_t>(),
                        linear_indices_sorted.numel(),
                        at::cuda::getCurrentCUDAStream()));
                // Allocate temporary storage
                auto temp_storage = at::empty(
                    {static_cast<int64_t>(temp_storage_bytes)},
                    indices.options().dtype(at::kByte));
                // Run encoding
                AT_CUDA_CHECK(
                    FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceRunLengthEncode::Encode(
                        temp_storage.data_ptr(),
                        temp_storage_bytes,
                        linear_indices_sorted.data_ptr<index_t>(),
                        sorted_linear_indices_run.data_ptr<index_t>(),
                        sorted_linear_indices_run_lengths.data_ptr<int32_t>(),
                        sorted_linear_indices_num_runs.data_ptr<int32_t>(),
                        linear_indices_sorted.numel(),
                        at::cuda::getCurrentCUDAStream()));
              }
            });
      });

  auto sorted_linear_indices_cumulative_run_lengths =
      asynchronous_complete_cumsum(sorted_linear_indices_run_lengths);

#undef INVOKE_LINEARIZE_INDEX_KERNEL

  return {
      linear_indices,
      linear_indices_sorted,
      infos_sorted,
      sorted_linear_indices_run,
      sorted_linear_indices_run_lengths,
      sorted_linear_indices_num_runs,
      sorted_linear_indices_cumulative_run_lengths};
}
