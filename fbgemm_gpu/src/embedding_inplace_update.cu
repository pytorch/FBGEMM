#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <c10/cuda/CUDAGuard.h>

#include "embedding_inplace_update.h"
#include "fbgemm_gpu/fbgemm_cuda_utils.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

constexpr int32_t kCacheLocationMissing = -1;

__launch_bounds__(kMaxThreads) __global__ void embedding_inplace_update_kernel(
    at::PackedTensorAccessor64<uint8_t, 1, at::RestrictPtrTraits> dev_weights,
    at::PackedTensorAccessor64<uint8_t, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        weights_offsets,
    const at::PackedTensorAccessor32<uint8_t, 1, at::RestrictPtrTraits>
        weights_tys,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        D_offsets,
    const at::PackedTensorAccessor64<uint8_t, 1, at::RestrictPtrTraits>
        update_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        update_table_idx,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        update_row_idx,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        update_offsets,
    const int64_t row_alignment,
    at::PackedTensorAccessor64<uint8_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        lxu_cache_locations) {
  // each row is updated by one warp of threads
  // blockIdx.x: block idx, threadIdx.x: thread idx in the warp,
  // threadIdx.y: warp idx in the block.
  // blockDim.x = warpSize, blockDim.y = warpsPerBlock.
  const int32_t i = blockIdx.x * blockDim.y + threadIdx.y;
  if (i >= update_row_idx.size(0)) {
    return;
  }
  const int32_t table_idx = update_table_idx[i];
  const int32_t row_idx = update_row_idx[i];

  const int32_t D_start = D_offsets[table_idx];
  const int32_t D_end = D_offsets[table_idx + 1];
  const int32_t D = D_end - D_start;
  SparseType weight_ty = static_cast<SparseType>(weights_tys[table_idx]);
  const int32_t D_bytes =
      nbit::padded_row_size_in_bytes(D, weight_ty, row_alignment);

  const int64_t weight_offset = weights_offsets[table_idx];
  uint8_t* __restrict__ weight_row;
  const auto placement =
      static_cast<PlacementType>(weights_placements[table_idx]);
  if (placement == PlacementType::DEVICE) {
    weight_row = &dev_weights[weight_offset + D_bytes * row_idx];
  } else {
    weight_row = &uvm_weights[weight_offset + D_bytes * row_idx];
  }

  const int64_t update_weight_offset = update_offsets[i];
  const uint8_t* __restrict__ update_weight_row =
      &update_weights[update_weight_offset];
  // TODO: do wider loads/stores so that entire 128B rows can be updated
  // in a single memory transaction
  for (int32_t d = threadIdx.x; d < D_bytes; d += blockDim.x) {
    weight_row[d] = update_weight_row[d];
  }

  uint8_t* __restrict__ cache_row;
  bool cache_valid = (placement == PlacementType::MANAGED_CACHING);
  int32_t cache_idx =
      cache_valid ? lxu_cache_locations[i] : kCacheLocationMissing;
  if (cache_valid && cache_idx != kCacheLocationMissing) {
    cache_row = &lxu_cache_weights[static_cast<int64_t>(cache_idx)][0];
    for (int32_t d = threadIdx.x; d < D_bytes; d += blockDim.x) {
      cache_row[d] = update_weight_row[d];
    }
  }
}

int32_t get_D_bytes(
    Tensor D_offsets,
    Tensor weights_tys,
    const int32_t table_idx,
    const int64_t row_alignment) {
  const int32_t D_start = D_offsets[table_idx].item<int32_t>();
  const int32_t D_end = D_offsets[table_idx + 1].item<int32_t>();
  const int32_t D = D_end - D_start;
  SparseType weight_ty =
      static_cast<SparseType>(weights_tys[table_idx].item<uint8_t>());
  return nbit::padded_row_size_in_bytes(D, weight_ty, row_alignment);
}

void embedding_inplace_update_cuda(
    Tensor dev_weights,
    Tensor uvm_weights,
    Tensor weights_placements,
    Tensor weights_offsets,
    Tensor weights_tys,
    Tensor D_offsets,
    Tensor update_weights,
    Tensor update_table_idx,
    Tensor update_row_idx,
    Tensor update_offsets,
    const int64_t row_alignment,
    c10::optional<Tensor> lxu_cache_weights,
    c10::optional<Tensor> lxu_cache_locations) {
  TENSOR_ON_CUDA_GPU(dev_weights);
  TENSOR_ON_CUDA_GPU(uvm_weights);
  TENSOR_ON_CUDA_GPU(weights_placements);
  TENSOR_ON_CUDA_GPU(weights_offsets);
  TENSOR_ON_CUDA_GPU(weights_tys);
  TENSOR_ON_CUDA_GPU(D_offsets);

  TENSOR_ON_CUDA_GPU(update_weights);
  TENSOR_ON_CUDA_GPU(update_offsets);
  TENSOR_ON_CUDA_GPU(update_table_idx);
  TENSOR_ON_CUDA_GPU(update_row_idx);

  if (lxu_cache_weights.has_value()) {
    TENSOR_ON_CUDA_GPU(lxu_cache_weights);
  }
  if (lxu_cache_locations.has_value()) {
    TENSOR_ON_CUDA_GPU(lxu_cache_locations);
  }

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(dev_weights.get_device());

  const int64_t N = update_row_idx.numel();
  if (N == 0) {
    return;
  }
  TORCH_CHECK(N == update_table_idx.numel());

  const int32_t warpsPerBlock = kMaxThreads / kWarpSize;

  auto lxu_cache_weights_value = lxu_cache_weights.value_or(
      at::empty({0, 0}, dev_weights.options().dtype(at::kByte)));

  auto lxu_cache_locations_value = lxu_cache_locations.value_or(
      at::empty({0}, dev_weights.options().dtype(at::kInt)));

  embedding_inplace_update_kernel<<<
      nbit::div_round_up(N, warpsPerBlock), // number of blocks needed
      dim3(kWarpSize, warpsPerBlock), // shape of each block
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      dev_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(),
      uvm_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(),
      weights_placements.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
      weights_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
      weights_tys.packed_accessor32<uint8_t, 1, at::RestrictPtrTraits>(),
      D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
      update_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(),
      update_table_idx.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
      update_row_idx.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
      update_offsets.packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
      row_alignment,
      lxu_cache_weights_value
          .packed_accessor64<uint8_t, 2, at::RestrictPtrTraits>(),
      lxu_cache_locations_value
          .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void embedding_inplace_update_host_weight_cuda(
    Tensor dev_weights,
    Tensor uvm_weights,
    const Tensor weights_placements,
    const Tensor weights_offsets,
    const Tensor weights_tys,
    const Tensor D_offsets,
    const Tensor update_weights,
    const std::vector<int32_t>& update_table_idx,
    const std::vector<int32_t>& update_row_idx,
    const int64_t row_alignment,
    c10::optional<Tensor> lxu_cache_weights,
    c10::optional<Tensor> lxu_cache_locations) {
  TENSOR_ON_CUDA_GPU(dev_weights);
  TENSOR_ON_CUDA_GPU(uvm_weights);
  TENSOR_ON_CUDA_GPU(weights_placements);
  TENSOR_ON_CUDA_GPU(weights_offsets);
  TENSOR_ON_CUDA_GPU(weights_tys);
  TENSOR_ON_CUDA_GPU(D_offsets);
  TENSOR_ON_CUDA_GPU(update_weights);

  if (lxu_cache_weights.has_value()) {
    TENSOR_ON_CUDA_GPU(lxu_cache_weights);
  }
  if (lxu_cache_locations.has_value()) {
    TENSOR_ON_CUDA_GPU(lxu_cache_locations);
  }

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(dev_weights.get_device());

  std::unordered_map<int32_t, int32_t> table_dbytes_map;
  std::vector<int64_t> update_offsets;
  int64_t update_offset = 0;
  update_offsets.push_back(0);
  for (int i = 0; i < update_table_idx.size(); ++i) {
    int32_t idx = update_table_idx[i];
    if (table_dbytes_map.find(idx) == table_dbytes_map.end()) {
      table_dbytes_map[idx] =
          get_D_bytes(D_offsets, weights_tys, idx, row_alignment);
    }
    update_offset += table_dbytes_map[idx];
    update_offsets.push_back(update_offset);
  }
  auto device = at::Device(at::kCUDA, at::cuda::current_device());
  auto update_offsets_tensor =
      at::tensor(update_offsets, at::device(device).dtype(at::kLong));
  auto table_idx_tensor =
      at::tensor(update_table_idx, at::device(device).dtype(at::kInt));
  auto row_idx_tensor =
      at::tensor(update_row_idx, at::device(device).dtype(at::kInt));

  embedding_inplace_update_cuda(
      dev_weights,
      uvm_weights,
      weights_placements,
      weights_offsets,
      weights_tys,
      D_offsets,
      update_weights,
      table_idx_tensor,
      row_idx_tensor,
      update_offsets_tensor,
      row_alignment,
      lxu_cache_weights,
      lxu_cache_locations);
}

} // namespace fbgemm_gpu
