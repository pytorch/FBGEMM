#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <c10/cuda/CUDAGuard.h>

#include "fbgemm_gpu/embedding_inplace_update.h"
#include "fbgemm_gpu/fbgemm_cuda_utils.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

constexpr int32_t kCacheLocationMissing = -1;

template <typename index_t>
__launch_bounds__(kMaxThreads) __global__ void embedding_inplace_update_kernel(
    at::PackedTensorAccessor64<uint8_t, 1, at::RestrictPtrTraits> dev_weights,
    at::PackedTensorAccessor64<uint8_t, 1, at::RestrictPtrTraits> uvm_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        weights_placements,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        weights_offsets,
    const at::PackedTensorAccessor32<uint8_t, 1, at::RestrictPtrTraits>
        weights_tys,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        D_offsets,
    const at::PackedTensorAccessor64<uint8_t, 1, at::RestrictPtrTraits>
        update_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        update_table_idx,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        update_row_idx,
    const at::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>
        update_offsets,
    const int64_t row_alignment,
    at::PackedTensorAccessor64<uint8_t, 2, at::RestrictPtrTraits>
        lxu_cache_weights,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        lxu_cache_locations) {
  // each row is updated by one warp of threads
  // blockIdx.x: block idx, threadIdx.x: thread idx in the warp,
  // threadIdx.y: warp idx in the block.
  // blockDim.x = warpSize, blockDim.y = warpsPerBlock.
  const int64_t i = blockIdx.x * blockDim.y + threadIdx.y;
  if (i >= update_row_idx.size(0)) {
    return;
  }
  const int32_t table_idx = update_table_idx[i];
  const auto row_idx = update_row_idx[i];

  const int32_t D_start = D_offsets[table_idx];
  const int32_t D_end = D_offsets[table_idx + 1];
  const int32_t D = D_end - D_start;
  SparseType weight_ty = static_cast<SparseType>(weights_tys[table_idx]);
  const int32_t D_bytes =
      nbit::padded_row_size_in_bytes(D, weight_ty, row_alignment);

  const int64_t weight_offset = weights_offsets[table_idx];
  uint8_t* __restrict__ weight_row;
  const auto placement =
      static_cast<PlacementType>(weights_placements[table_idx]);
  if (placement == PlacementType::DEVICE) {
    weight_row =
        &dev_weights
            [weight_offset +
             static_cast<int64_t>(D_bytes) * static_cast<int64_t>(row_idx)];
  } else {
    weight_row =
        &uvm_weights
            [weight_offset +
             static_cast<int64_t>(D_bytes) * static_cast<int64_t>(row_idx)];
  }

  // padded_row_size_in_bytes pad each row with row_alignment (16 bytes on GPUs)
  // So each row will be multiple of 16 bytes (uint4 = 32bit x 4 = 16 bytes)
  auto vec_weight_row = reinterpret_cast<uint4*>(weight_row);
  const int64_t update_weight_offset = update_offsets[i];
  auto update_weight_row =
      reinterpret_cast<const uint4*>(&update_weights[update_weight_offset]);
  // Do wider loads/stores so that each 16 Byte segment in the row can be
  // updated in a single memory transaction
  for (int32_t d = threadIdx.x; d * sizeof(uint4) < D_bytes; d += blockDim.x) {
    vec_weight_row[d] = update_weight_row[d];
  }

  bool cache_valid = (placement == PlacementType::MANAGED_CACHING);
  int32_t cache_idx =
      cache_valid ? lxu_cache_locations[i] : kCacheLocationMissing;
  if (cache_valid && cache_idx != kCacheLocationMissing) {
    auto vec_cache_row = reinterpret_cast<uint4*>(
        &lxu_cache_weights[static_cast<int64_t>(cache_idx)][0]);

    for (int32_t d = threadIdx.x; d * sizeof(uint4) < D_bytes;
         d += blockDim.x) {
      vec_cache_row[d] = update_weight_row[d];
    }
  }
}

void embedding_inplace_update_cuda(
    Tensor dev_weights,
    Tensor uvm_weights,
    Tensor weights_placements,
    Tensor weights_offsets,
    Tensor weights_tys,
    Tensor D_offsets,
    Tensor update_weights,
    Tensor update_table_idx,
    Tensor update_row_idx,
    Tensor update_offsets,
    const int64_t row_alignment,
    c10::optional<Tensor> lxu_cache_weights,
    c10::optional<Tensor> lxu_cache_locations) {
  TENSOR_ON_CUDA_GPU(dev_weights);
  TENSOR_ON_CUDA_GPU(uvm_weights);
  TENSOR_ON_CUDA_GPU(weights_placements);
  TENSOR_ON_CUDA_GPU(weights_offsets);
  TENSOR_ON_CUDA_GPU(weights_tys);
  TENSOR_ON_CUDA_GPU(D_offsets);

  TENSOR_ON_CUDA_GPU(update_weights);
  TENSOR_ON_CUDA_GPU(update_offsets);
  TENSOR_ON_CUDA_GPU(update_table_idx);
  TENSOR_ON_CUDA_GPU(update_row_idx);

  if (lxu_cache_weights.has_value()) {
    TENSOR_ON_CUDA_GPU(lxu_cache_weights);
  }
  if (lxu_cache_locations.has_value()) {
    TENSOR_ON_CUDA_GPU(lxu_cache_locations);
  }

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(dev_weights.get_device());

  const int64_t N = update_row_idx.numel();
  if (N == 0) {
    return;
  }
  TORCH_CHECK(N == update_table_idx.numel());

  const int32_t warpsPerBlock = kMaxThreads / kWarpSize;

  auto lxu_cache_weights_value = lxu_cache_weights.value_or(
      at::empty({0, 0}, dev_weights.options().dtype(at::kByte)));

  auto lxu_cache_locations_value = lxu_cache_locations.value_or(
      at::empty({0}, dev_weights.options().dtype(at::kInt)));

  AT_DISPATCH_INDEX_TYPES(
      update_row_idx.scalar_type(), "embedding_inplace_update_kernel", [&] {
        embedding_inplace_update_kernel<<<
            nbit::div_round_up(N, warpsPerBlock), // number of blocks needed
            dim3(kWarpSize, warpsPerBlock), // shape of each block
            0,
            at::cuda::getCurrentCUDAStream()>>>(
            dev_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(),
            uvm_weights.packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(),
            weights_placements
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            weights_offsets
                .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            weights_tys.packed_accessor32<uint8_t, 1, at::RestrictPtrTraits>(),
            D_offsets.packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            update_weights
                .packed_accessor64<uint8_t, 1, at::RestrictPtrTraits>(),
            update_table_idx
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
            update_row_idx
                .packed_accessor32<index_t, 1, at::RestrictPtrTraits>(),
            update_offsets
                .packed_accessor32<int64_t, 1, at::RestrictPtrTraits>(),
            row_alignment,
            lxu_cache_weights_value
                .packed_accessor64<uint8_t, 2, at::RestrictPtrTraits>(),
            lxu_cache_locations_value
                .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace fbgemm_gpu
