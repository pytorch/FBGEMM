#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include "fbgemm_gpu/batched_unary_embedding_ops.cuh"
#include "fbgemm_gpu/sparse_ops.cuh"
#include "fbgemm_gpu/sparse_ops.h"
#include "fbgemm_gpu/sparse_ops_utils.h"

#include <ATen/ATen.h>
#include <ATen/core/op_registration/op_registration.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/library.h>

// clang-format off
#include "fbgemm_gpu/cub_namespace_prefix.cuh"
#include "cub/device/device_scan.cuh"
#include "fbgemm_gpu/cub_namespace_postfix.cuh"
// clang-format on

#include "fbgemm_gpu/fbgemm_cuda_utils.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

std::tuple<uint32_t, uint32_t, uint32_t> calc_offsets_range_thread_block(
    const int64_t output_size,
    const int64_t num_seq) {
  uint32_t threads_per_block;
  uint32_t vector_size;
  if (output_size / num_seq < 2) {
    threads_per_block = 512;
    vector_size = 2;
  } else if (output_size / num_seq < 4) {
    threads_per_block = 512;
    vector_size = 4;
  } else if (output_size / num_seq < 64) {
    threads_per_block = 512;
    vector_size = 8;
  } else if (output_size / num_seq < 128) {
    threads_per_block = 512;
    vector_size = 16;
  } else {
    threads_per_block = 512;
    vector_size = 32;
  }
  uint32_t rows_per_block = threads_per_block / vector_size;
  const auto num_blocks = cuda_calc_xblock_count(num_seq, rows_per_block);

  return std::make_tuple(num_blocks, rows_per_block, vector_size);
}

Tensor offsets_range_cuda(const Tensor& offsets, int64_t range_size) {
  TENSOR_ON_CUDA_GPU(offsets);
  TENSOR_NDIM_EQUALS(offsets, 1);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(offsets.get_device());

  auto offsets_arg = at::TensorArg(offsets, "offsets", 1);
  checkScalarTypes("_offsets_range_cuda", offsets_arg, {at::kLong, at::kInt});
  auto range = at::empty(range_size, offsets.options());
  if (range_size == 0) {
    return range;
  }
  auto offsets_contig = offsets.contiguous();
  int64_t N = offsets_contig.numel();

  uint32_t vector_size;
  uint32_t rows_per_block;
  uint32_t num_blocks;
  std::tie(num_blocks, rows_per_block, vector_size) =
      calc_offsets_range_thread_block(range_size, N);
  dim3 threads(vector_size, rows_per_block);
  AT_DISPATCH_INDEX_TYPES(
      offsets_contig.scalar_type(), "offsets_range_kernel", [&]() {
        _offsets_range_cuda_kernel<index_t>
            <<<num_blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                N,
                range_size,
                offsets_contig.data_ptr<index_t>(),
                range.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

  return range;
}

Tensor segment_sum_csr_cuda(
    const int64_t batch_size,
    const Tensor& csr_seg,
    const Tensor& values) {
  TENSOR_ON_CUDA_GPU(csr_seg);
  TENSOR_ON_CUDA_GPU(values);

  TENSORS_ON_SAME_DEVICE(csr_seg, values);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(values.get_device());

  auto output = at::empty(csr_seg.numel() - 1, values.options());
  constexpr uint32_t threads_per_block = 256;
  const uint32_t num_blocks = csr_seg.numel() - 1;
  AT_DISPATCH_ALL_TYPES(values.type(), "_segment_sum_csr_cuda", [&]() {
    _segment_sum_csr_cuda_kernel<scalar_t>
        <<<num_blocks,
           threads_per_block,
           0,
           at::cuda::getCurrentCUDAStream()>>>(
            csr_seg.numel() - 1,
            batch_size,
            csr_seg.data_ptr<int>(),
            values.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
  return output;
}

Tensor asynchronous_inclusive_cumsum_gpu(const Tensor& t_in) {
  TENSOR_ON_CUDA_GPU(t_in);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == at::kInt || t_in.dtype() == at::kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  auto t_out = at::empty_like(t_in);
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      t_in.options().dtype(at::kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

Tensor asynchronous_exclusive_cumsum_gpu(const Tensor& t_in) {
  TENSOR_ON_CUDA_GPU(t_in);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == at::kInt || t_in.dtype() == at::kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  auto t_out = at::empty_like(t_in);
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_exclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::ExclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      t_in.options().dtype(at::kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_exclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::ExclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

Tensor asynchronous_complete_cumsum_gpu(const Tensor& t_in) {
  TENSOR_ON_CUDA_GPU(t_in);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == at::kInt || t_in.dtype() == at::kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  TORCH_CHECK(t_in.dim() == 1);
  auto t_out = at::empty({t_in.numel() + 1}, t_in.options());
  t_out[0].zero_();
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>() + 1,
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      t_in.options().dtype(at::kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>() + 1,
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

std::tuple<Tensor, Tensor, c10::optional<Tensor>> permute_sparse_data_cuda(
    const Tensor& permute,
    const Tensor& lengths,
    const Tensor& indices,
    const c10::optional<Tensor>& weights,
    const c10::optional<int64_t>& permuted_lengths_sum) {
  TENSOR_ON_CUDA_GPU(permute);
  TENSOR_ON_CUDA_GPU(lengths);
  TENSOR_ON_CUDA_GPU(indices);
  TENSOR_ON_CUDA_GPU(weights);

  TENSORS_ON_SAME_DEVICE(permute, lengths);
  TENSORS_ON_SAME_DEVICE(permute, indices);
  TENSORS_ON_SAME_DEVICE(permute, weights);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(indices.get_device());

  const auto permute_contig = permute.contiguous();
  const auto lengths_contig = lengths.contiguous();
  const auto indices_contig = indices.contiguous();
  // the data to permute over can be less or more with or without
  // repetitions
  const auto T = permute.numel();
  const auto T_ = lengths.size(0);
  const auto B = lengths.view({lengths.sizes()[0], -1}).sizes()[1];

  Tensor permuted_lengths;
  Tensor permuted_indices;
  Tensor permuted_weights;

  permuted_lengths = at::empty({T, B}, lengths.options());

  constexpr int32_t threads_1 = 256;
  const auto blocks_1 = cuda_calc_xblock_count(B * T, threads_1);
  AT_DISPATCH_INDEX_TYPES(
      lengths.scalar_type(), "permute_lengths_kernel", ([&] {
        permute_lengths_kernel<index_t>
            <<<blocks_1, threads_1, 0, at::cuda::getCurrentCUDAStream()>>>(
                T,
                B,
                lengths_contig.data_ptr<index_t>(),
                permute.data_ptr<int32_t>(),
                permuted_lengths.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));

  // convert lengths to offsets
  const auto input_offsets = asynchronous_exclusive_cumsum_gpu(lengths_contig);
  const auto output_offsets =
      asynchronous_exclusive_cumsum_gpu(permuted_lengths);
  int64_t permuted_indices_size = 0;
  if (permuted_lengths_sum.has_value()) {
    permuted_indices_size = permuted_lengths_sum.value();
  } else {
    permuted_indices_size = permuted_lengths.sum().item<int64_t>();
  }

  constexpr int32_t BT_blocks = 32;
  dim3 threads_2(32, BT_blocks);
  const auto blocks_2 = cuda_calc_xblock_count(B * T, BT_blocks);
  permuted_indices = at::empty(permuted_indices_size, indices.options());

  AT_DISPATCH_INDEX_TYPES(
      input_offsets.scalar_type(), "permute_data_kernel_1", ([&] {
        using offsets_t = index_t;
        AT_DISPATCH_ALL_TYPES_AND(
            at::ScalarType::Half,
            indices.scalar_type(),
            "permute_data_kernel_2",
            ([&] {
              using indices_t = scalar_t;
              if (weights.has_value()) {
                const Tensor weights_value = weights.value();
                const auto weights_value_contig = weights_value.contiguous();
                permuted_weights =
                    at::empty(permuted_indices_size, weights_value.options());
                AT_DISPATCH_ALL_TYPES_AND(
                    at::ScalarType::Half,
                    weights_value.scalar_type(),
                    "permute_data_kernel_3",
                    ([&] {
                      using weights_t = scalar_t;
                      permute_data_kernel<true, offsets_t, indices_t, weights_t>
                          <<<blocks_2,
                             threads_2,
                             0,
                             at::cuda::getCurrentCUDAStream()>>>(
                              permuted_indices_size,
                              T,
                              B,
                              indices_contig.data_ptr<indices_t>(),
                              weights_value_contig.data_ptr<weights_t>(),
                              permute_contig.data_ptr<int32_t>(),
                              input_offsets.data_ptr<offsets_t>(),
                              output_offsets.data_ptr<offsets_t>(),
                              permuted_indices.data_ptr<indices_t>(),
                              permuted_weights.data_ptr<weights_t>());
                      C10_CUDA_KERNEL_LAUNCH_CHECK();
                    })); // for each weights_t
              } else {
                permute_data_kernel<false, offsets_t, indices_t, std::nullptr_t>
                    <<<blocks_2,
                       threads_2,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        permuted_indices_size,
                        T,
                        B,
                        indices_contig.data_ptr<indices_t>(),
                        nullptr,
                        permute_contig.data_ptr<int32_t>(),
                        input_offsets.data_ptr<offsets_t>(),
                        output_offsets.data_ptr<offsets_t>(),
                        permuted_indices.data_ptr<indices_t>(),
                        nullptr);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
            })); // for each indices_t
      })); // for each offsets_t
  return {permuted_lengths, permuted_indices, permuted_weights};
}

// This function partitions sparse features
// continuously along the sparse dimension into my_size blocks
std::tuple<
    Tensor,
    Tensor,
    c10::optional<Tensor>,
    c10::optional<Tensor>,
    c10::optional<Tensor>>
block_bucketize_sparse_features_cuda(
    Tensor lengths,
    Tensor indices,
    bool bucketize_pos,
    bool sequence,
    Tensor block_sizes,
    int64_t my_size,
    c10::optional<Tensor> weights) {
  TENSOR_ON_CUDA_GPU(lengths);
  TENSOR_ON_CUDA_GPU(indices);
  TENSORS_ON_SAME_DEVICE(lengths, indices);
  TENSOR_ON_CUDA_GPU(weights);
  TENSORS_ON_SAME_DEVICE(lengths, weights);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(lengths.get_device());
  // allocate tensors and buffers
  const int lengths_size = lengths.numel();
  const int T = block_sizes.numel();
  const int B = lengths_size / T;
  const int new_lengths_size = lengths_size * my_size;
  auto offsets = at::empty({lengths_size}, lengths.options());
  auto new_lengths = at::zeros({new_lengths_size}, lengths.options());
  auto new_offsets = at::empty({new_lengths_size}, lengths.options());
  auto new_indices = at::empty_like(indices);
  auto lengths_contig = lengths.contiguous();
  auto indices_contig = indices.contiguous();
  auto offsets_contig = offsets.contiguous();
  Tensor new_weights;
  Tensor new_pos;
  Tensor unbucketize_permute;
  // count nonzeros
  offsets_contig = asynchronous_inclusive_cumsum_gpu(lengths);
  int threads_per_block = 256;
  int num_blocks = (lengths_size + threads_per_block - 1) / threads_per_block;
  AT_DISPATCH_INDEX_TYPES(
      offsets_contig.scalar_type(),
      "_block_bucketize_sparse_features_cuda_kernel1",
      ([&] {
        using offset_t = index_t;
        AT_DISPATCH_INDEX_TYPES(
            indices_contig.scalar_type(),
            "_block_bucketize_sparse_features_cuda_kernel2",
            ([&] {
              _block_bucketize_sparse_features_cuda_kernel1<<<
                  num_blocks,
                  threads_per_block,
                  0,
                  at::cuda::getCurrentCUDAStream()>>>(
                  lengths_size,
                  B,
                  block_sizes.data_ptr<index_t>(),
                  my_size,
                  offsets_contig.data_ptr<offset_t>(),
                  indices_contig.data_ptr<index_t>(),
                  new_lengths.data_ptr<offset_t>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));

  // bucketize nonzeros
  new_offsets = asynchronous_exclusive_cumsum_gpu(new_lengths);
  if (sequence) {
    const auto lengths_sum = indices.numel();
    unbucketize_permute = at::empty({lengths_sum}, indices.options());
    if (weights.has_value() & bucketize_pos) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            true,
                            true,
                            true,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                new_pos.data_ptr<index_t>(),
                                unbucketize_permute.data_ptr<index_t>());
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));
    } else if (weights.has_value()) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            true,
                            true,
                            false,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                nullptr,
                                unbucketize_permute.data_ptr<index_t>());
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (bucketize_pos) {
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      true,
                      false,
                      true,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          new_pos.data_ptr<index_t>(),
                          unbucketize_permute.data_ptr<index_t>());
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));

    } else {
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      true,
                      false,
                      false,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          nullptr,
                          unbucketize_permute.data_ptr<index_t>());
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));
    }
  } else {
    if (weights.has_value() & bucketize_pos) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            false,
                            true,
                            true,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                new_pos.data_ptr<index_t>(),
                                nullptr);
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (weights.has_value()) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            false,
                            true,
                            false,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                nullptr,
                                nullptr);
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (bucketize_pos) {
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      false,
                      false,
                      true,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          new_pos.data_ptr<index_t>(),
                          nullptr);
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));

    } else {
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      false,
                      false,
                      false,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          nullptr,
                          nullptr);
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));
    }
  }

  return {new_lengths, new_indices, new_weights, new_pos, unbucketize_permute};
}

template <typename Dtype>
__global__ void reorder_batched_ad_lengths_kernel(
    // reorder lengths from (ragged) [B  x T x #num_ads_b)] to
    // [T][B][#num_ads_b], i.e. [T][sum(#num_ads_b)].
    const at::PackedTensorAccessor32<Dtype, 1, at::RestrictPtrTraits>
        cat_ad_lengths,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        batch_offsets,
    at::PackedTensorAccessor32<Dtype, 1, at::RestrictPtrTraits>
        reordered_cat_ad_lengths,
    int32_t T) {
  const int32_t B = batch_offsets.size(0) - 1;

  const int32_t num_ads_in_batch = batch_offsets[B];
  // warp-per-segment.
  const int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
  const int32_t b = b_t % B;
  const int32_t t = b_t / B;
  if (t >= T) {
    return;
  }

  const int32_t num_ads_b = batch_offsets[b + 1] - batch_offsets[b];
  const int32_t input_segment_start = T * batch_offsets[b] + t * num_ads_b;
  const int32_t output_segment_start = t * num_ads_in_batch + batch_offsets[b];

  for (int32_t i = threadIdx.x; i < num_ads_b; i += blockDim.x) {
    reordered_cat_ad_lengths[output_segment_start + i] =
        cat_ad_lengths[input_segment_start + i];
  }
}

Tensor reorder_batched_ad_lengths_gpu(
    const Tensor& cat_ad_lengths,
    const Tensor& batch_offsets,
    const int64_t num_ads_in_batch) {
  TENSOR_ON_CUDA_GPU(cat_ad_lengths);
  TENSOR_ON_CUDA_GPU(batch_offsets);
  TENSORS_ON_SAME_DEVICE(cat_ad_lengths, batch_offsets);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(cat_ad_lengths.get_device());

  const int64_t B = batch_offsets.numel() - 1;
  const int64_t T = cat_ad_lengths.numel() / num_ads_in_batch;

  Tensor reordered_cat_ad_lengths = at::empty_like(cat_ad_lengths);

  const dim3 threads(32, 32);
  const dim3 blocks((B * T + 32 - 1) / 32);

  AT_DISPATCH_ALL_TYPES(
      cat_ad_lengths.type(), "reorder_batched_ad_lengths_gpu_kernel", ([&] {
        reorder_batched_ad_lengths_kernel<scalar_t>
            <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                cat_ad_lengths
                    .packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
                batch_offsets
                    .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                reordered_cat_ad_lengths
                    .packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
                T);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
  return reordered_cat_ad_lengths;
}

template <typename Dtype>
__global__ void reorder_batched_ad_indices_kernel(
    // reorder indices from (ragged) [B  x T x #num_ads_b x length_{b, t, a})]
    // to [T][B][#num_ads_b][length_{b, t, a}], i.e. [sum(length_{b, t, a})],
    // laid out as [T][B][A][L] (if all lengths were equal).
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        cat_ad_offsets,
    const at::PackedTensorAccessor32<Dtype, 1, at::RestrictPtrTraits>
        cat_ad_indices,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        reordered_cat_ad_offsets,
    at::PackedTensorAccessor32<Dtype, 1, at::RestrictPtrTraits>
        reordered_cat_ad_indices,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        batch_offsets,
    int32_t T) {
  const int32_t B = batch_offsets.size(0) - 1;
  const int32_t num_ads_in_batch = batch_offsets[B];
  // warp-per-segment.
  const int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
  const int32_t b = b_t % B;
  const int32_t t = b_t / B;
  if (t >= T) {
    return;
  }
  // for each ad,
  const int32_t num_ads_b = batch_offsets[b + 1] - batch_offsets[b];
  const int32_t b_t_start = T * batch_offsets[b] + t * num_ads_b;
  const int32_t input_segment_offset_start =
      T * batch_offsets[b] + t * num_ads_b;
  const int32_t input_segment_offset_end =
      T * batch_offsets[b] + t * num_ads_b + num_ads_b;

  // Idea: we want to copy the entire segment of size sum_a(length_{b, t, a})
  // from starting point (given by cat_ad_offsets[b, t])
  // to end point (given by reordered_cat_ad_indices[t][b])
  const int32_t input_segment_start =
      cat_ad_offsets[input_segment_offset_start];
  const int32_t input_segment_end = cat_ad_offsets[input_segment_offset_end];

  const int32_t output_segment_offset_start =
      t * num_ads_in_batch + batch_offsets[b];
  const int32_t output_segment_start =
      reordered_cat_ad_offsets[output_segment_offset_start];

  for (auto i = threadIdx.x; i < input_segment_end - input_segment_start;
       i += blockDim.x) {
    reordered_cat_ad_indices[output_segment_start + i] =
        cat_ad_indices[input_segment_start + i];
  }
}

Tensor reorder_batched_ad_indices_gpu(
    const Tensor& cat_ad_offsets,
    const Tensor& cat_ad_indices,
    const Tensor& reordered_cat_ad_offsets,
    const Tensor& batch_offsets,
    const int64_t num_ads_in_batch) {
  TENSOR_ON_CUDA_GPU(cat_ad_offsets);
  TENSOR_ON_CUDA_GPU(cat_ad_indices);
  TENSOR_ON_CUDA_GPU(reordered_cat_ad_offsets);
  TENSOR_ON_CUDA_GPU(batch_offsets);
  TENSORS_ON_SAME_DEVICE(cat_ad_offsets, cat_ad_indices);
  TENSORS_ON_SAME_DEVICE(cat_ad_offsets, reordered_cat_ad_offsets);
  TENSORS_ON_SAME_DEVICE(cat_ad_offsets, batch_offsets);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(cat_ad_offsets.get_device());

  const int64_t B = batch_offsets.numel() - 1;
  const int64_t T = (cat_ad_offsets.numel() - 1) / num_ads_in_batch;
  Tensor reordered_cat_ad_indices = at::empty_like(cat_ad_indices);

  const dim3 threads(32, 32);
  const dim3 blocks((B * T + 32 - 1) / 32);

  AT_DISPATCH_ALL_TYPES(
      cat_ad_indices.type(), "reorder_batched_ad_indices_gpu_kernel", ([&] {
        reorder_batched_ad_indices_kernel<scalar_t>
            <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                cat_ad_offsets
                    .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                cat_ad_indices
                    .packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
                reordered_cat_ad_offsets
                    .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                reordered_cat_ad_indices
                    .packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
                batch_offsets
                    .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                T);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
  return reordered_cat_ad_indices;
}

Tensor batched_unary_embeddings_forward_cuda(
    const Tensor& weight,
    const Tensor& table_offsets,
    const Tensor& offsets,
    const Tensor& indices) {
  TENSOR_CONTIGUOUS_AND_ON_CUDA_GPU(table_offsets);
  TENSOR_CONTIGUOUS_AND_ON_CUDA_GPU(weight);
  TENSOR_CONTIGUOUS_AND_ON_CUDA_GPU(offsets);
  TENSOR_CONTIGUOUS_AND_ON_CUDA_GPU(indices);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(weight.get_device());
  // N: number of tasks, T: number of tables, B: batch size
  const int32_t N = weight.size(0);
  const int32_t T = table_offsets.numel() - 1;
  const int32_t B = (offsets.numel() - 1) / T;
  TORCH_CHECK(N > 0);
  TORCH_CHECK(B > 0);
  TORCH_CHECK(T > 0);
  TORCH_CHECK(T <= 65535);
  TORCH_CHECK(N <= 65535);
  int32_t threads = std::min<int32_t>(B, 512);
  dim3 blocks(cuda_calc_xblock_count(B, threads), T, N);
  auto output = at::empty({N, B, T}, weight.options());
  AT_DISPATCH_INDEX_TYPES(
      indices.type(), "batched_unary_embeddings_forward_kernel", ([&] {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
            weight.type(), "batched_unary_embeddings_forward_kernel", ([&] {
              batched_unary_embeddings_forward_kernel<scalar_t>
                  <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                      N,
                      B,
                      T,
                      weight.data_ptr<scalar_t>(),
                      table_offsets.data_ptr<index_t>(),
                      offsets.data_ptr<index_t>(),
                      indices.data_ptr<index_t>(),
                      output.data_ptr<scalar_t>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));
  return output;
}

Tensor batched_unary_embeddings_backward_cuda(
    const Tensor& grad_output,
    const Tensor& weight,
    const Tensor& table_offsets,
    const Tensor& offsets,
    const Tensor& indices) {
  TENSOR_ON_CUDA_GPU(grad_output);
  TENSOR_ON_CUDA_GPU(weight);
  TENSOR_ON_CUDA_GPU(table_offsets);
  TENSOR_ON_CUDA_GPU(offsets);
  TENSOR_ON_CUDA_GPU(indices);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(grad_output.get_device());

  // N: number of tasks, T: number of tables, B: batch size
  const int32_t N = grad_output.size(0);
  const int32_t B = grad_output.size(1);
  const int32_t T = grad_output.size(2);
  TORCH_CHECK(N > 0);
  TORCH_CHECK(B > 0);
  TORCH_CHECK(T > 0);
  int threads = std::min<int32_t>(N * T, 512);
  dim3 blocks(cuda_calc_xblock_count(N * T, threads));
  auto grad_weight = at::zeros_like(weight);
  AT_DISPATCH_INDEX_TYPES(
      indices.type(), "batched_unary_embeddings_backward_kernel", ([&] {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
            grad_output.type(),
            "batched_unary_embeddings_backward_kernel",
            ([&] {
              batched_unary_embeddings_backward_kernel<scalar_t>
                  <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                      N,
                      B,
                      T,
                      grad_output.data_ptr<scalar_t>(),
                      table_offsets.data_ptr<index_t>(),
                      offsets.data_ptr<index_t>(),
                      indices.data_ptr<index_t>(),
                      grad_weight.data_ptr<scalar_t>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));
  return grad_weight;
}

} // namespace fbgemm_gpu
