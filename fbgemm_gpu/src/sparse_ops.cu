#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include "fbgemm_gpu/quantize_ops.cuh"
#include "fbgemm_gpu/sparse_ops.cuh"
#include "fbgemm_gpu/sparse_ops.h"
#include "fbgemm_gpu/sparse_ops_utils.h"

#include <ATen/ATen.h>
#include <ATen/core/op_registration/op_registration.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/library.h>

#include "ATen/Parallel.h"
#include "cub/device/device_scan.cuh"

namespace at {
Tensor asynchronous_inclusive_cumsum(const Tensor& t_in) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == kInt || t_in.dtype() == kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  auto t_out = at::empty_like(t_in);
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(hipcub::DeviceScan::InclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)}, t_in.options().dtype(kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(hipcub::DeviceScan::InclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

Tensor asynchronous_exclusive_cumsum(const Tensor& t_in) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == kInt || t_in.dtype() == kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  auto t_out = at::empty_like(t_in);
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_exclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)}, t_in.options().dtype(kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_exclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

std::tuple<Tensor, Tensor, c10::optional<Tensor>> permute_sparse_data_cuda(
    const Tensor& permute,
    const Tensor& lengths,
    const Tensor& indices,
    const c10::optional<Tensor>& weights,
    const c10::optional<int64_t>& permuted_lengths_sum) {
  TENSOR_ON_CUDA_GPU(permute);
  TENSOR_ON_CUDA_GPU(lengths);
  TENSOR_ON_CUDA_GPU(indices);
  TENSOR_ON_CUDA_GPU(weights);

  TENSORS_ON_SAME_DEVICE(permute, lengths);
  TENSORS_ON_SAME_DEVICE(permute, indices);
  TENSORS_ON_SAME_DEVICE(permute, weights);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(indices.get_device());

  const auto permute_contig = permute.contiguous();
  const auto lengths_contig = lengths.contiguous();
  const auto indices_contig = indices.contiguous();
  // the data to permute over can be less or more with or without
  // repetitions
  const auto T = permute.numel();
  const auto T_ = lengths.size(0);
  const auto B = lengths.view({lengths.sizes()[0], -1}).sizes()[1];

  Tensor permuted_lengths;
  Tensor permuted_indices;
  Tensor permuted_weights;

  permuted_lengths = at::empty({T, B}, lengths.options());

  constexpr int32_t threads_1 = 256;
  const auto blocks_1 = cuda_calc_xblock_count(B * T, threads_1);
  AT_DISPATCH_INDEX_TYPES(
      lengths.scalar_type(), "permute_lengths_kernel", ([&] {
        permute_lengths_kernel<index_t>
            <<<blocks_1, threads_1, 0, at::cuda::getCurrentCUDAStream()>>>(
                T,
                B,
                lengths_contig.data_ptr<index_t>(),
                permute.data_ptr<int32_t>(),
                permuted_lengths.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));

  // convert lengths to offsets
  const auto input_offsets = asynchronous_exclusive_cumsum(lengths_contig);
  const auto output_offsets = asynchronous_exclusive_cumsum(permuted_lengths);
  int64_t permuted_indices_size = 0;
  if (permuted_lengths_sum.has_value()) {
    permuted_indices_size = permuted_lengths_sum.value();
  } else {
    permuted_indices_size = permuted_lengths.sum().item<int64_t>();
  }

  constexpr int32_t BT_blocks = 32;
  dim3 threads_2(32, BT_blocks);
  const auto blocks_2 = cuda_calc_xblock_count(B * T, BT_blocks);
  permuted_indices = at::empty(permuted_indices_size, indices.options());

  AT_DISPATCH_INDEX_TYPES(
      input_offsets.scalar_type(), "permute_data_kernel_1", ([&] {
        using offsets_t = index_t;
        AT_DISPATCH_ALL_TYPES(
            indices.scalar_type(), "permute_data_kernel_2", ([&] {
              using indices_t = scalar_t;
              if (weights.has_value()) {
                const Tensor weights_value = weights.value();
                const auto weights_value_contig = weights_value.contiguous();
                permuted_weights =
                    at::empty(permuted_indices_size, weights_value.options());
                AT_DISPATCH_FLOATING_TYPES(
                    weights_value.scalar_type(), "permute_data_kernel_3", ([&] {
                      using weights_t = scalar_t;
                      permute_data_kernel<true, offsets_t, indices_t, weights_t>
                          <<<blocks_2,
                             threads_2,
                             0,
                             at::cuda::getCurrentCUDAStream()>>>(
                              permuted_indices_size,
                              T,
                              B,
                              indices_contig.data_ptr<indices_t>(),
                              weights_value_contig.data_ptr<weights_t>(),
                              permute_contig.data_ptr<int32_t>(),
                              input_offsets.data_ptr<offsets_t>(),
                              output_offsets.data_ptr<offsets_t>(),
                              permuted_indices.data_ptr<indices_t>(),
                              permuted_weights.data_ptr<weights_t>());
                      C10_CUDA_KERNEL_LAUNCH_CHECK();
                    })); // for each weights_t
              } else {
                permute_data_kernel<false, offsets_t, indices_t, std::nullptr_t>
                    <<<blocks_2,
                       threads_2,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        permuted_indices_size,
                        T,
                        B,
                        indices_contig.data_ptr<indices_t>(),
                        nullptr,
                        permute_contig.data_ptr<int32_t>(),
                        input_offsets.data_ptr<offsets_t>(),
                        output_offsets.data_ptr<offsets_t>(),
                        permuted_indices.data_ptr<indices_t>(),
                        nullptr);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
            })); // for each indices_t
      })); // for each offsets_t
  return {permuted_lengths, permuted_indices, permuted_weights};
}

// This function partitions sparse features
// continuously along the sparse dimension into my_size blocks
std::tuple<
    Tensor,
    Tensor,
    c10::optional<Tensor>,
    c10::optional<Tensor>,
    c10::optional<Tensor>>
block_bucketize_sparse_features_cuda(
    Tensor lengths,
    Tensor indices,
    bool bucketize_pos,
    bool sequence,
    Tensor block_sizes,
    int64_t my_size,
    c10::optional<Tensor> weights) {
  TENSOR_ON_CUDA_GPU(lengths);
  TENSOR_ON_CUDA_GPU(indices);
  TENSORS_ON_SAME_DEVICE(lengths, indices);
  TENSOR_ON_CUDA_GPU(weights);
  TENSORS_ON_SAME_DEVICE(lengths, weights);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(lengths.get_device());
  // allocate tensors and buffers
  const int lengths_size = lengths.numel();
  const int T = block_sizes.numel();
  const int B = lengths_size / T;
  const int new_lengths_size = lengths_size * my_size;
  auto offsets = at::empty({lengths_size}, lengths.options());
  auto new_lengths = at::zeros({new_lengths_size}, lengths.options());
  auto new_offsets = at::empty({new_lengths_size}, lengths.options());
  auto new_indices = at::empty_like(indices);
  auto lengths_contig = lengths.contiguous();
  auto indices_contig = indices.contiguous();
  auto offsets_contig = offsets.contiguous();
  Tensor new_weights;
  Tensor new_pos;
  Tensor unbucketize_permute;
  // count nonzeros
  offsets_contig = asynchronous_inclusive_cumsum(lengths);
  int threads_per_block = 256;
  int num_blocks = (lengths_size + threads_per_block - 1) / threads_per_block;
  AT_DISPATCH_INDEX_TYPES(
      offsets_contig.scalar_type(),
      "_block_bucketize_sparse_features_cuda_kernel1",
      ([&] {
        using offset_t = index_t;
        AT_DISPATCH_INDEX_TYPES(
            indices_contig.scalar_type(),
            "_block_bucketize_sparse_features_cuda_kernel2",
            ([&] {
              _block_bucketize_sparse_features_cuda_kernel1<<<
                  num_blocks,
                  threads_per_block,
                  0,
                  at::cuda::getCurrentCUDAStream()>>>(
                  lengths_size,
                  B,
                  block_sizes.data_ptr<index_t>(),
                  my_size,
                  offsets_contig.data_ptr<offset_t>(),
                  indices_contig.data_ptr<index_t>(),
                  new_lengths.data_ptr<offset_t>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));

  // bucketize nonzeros
  new_offsets = asynchronous_exclusive_cumsum(new_lengths);
  if (sequence) {
    const auto lengths_sum = indices.numel();
    unbucketize_permute = at::empty({lengths_sum}, indices.options());
    if (weights.has_value() & bucketize_pos) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            true,
                            true,
                            true,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                new_pos.data_ptr<index_t>(),
                                unbucketize_permute.data_ptr<index_t>());
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));
    } else if (weights.has_value()) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            true,
                            true,
                            false,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                nullptr,
                                unbucketize_permute.data_ptr<index_t>());
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (bucketize_pos) {
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      true,
                      false,
                      true,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          new_pos.data_ptr<index_t>(),
                          unbucketize_permute.data_ptr<index_t>());
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));

    } else {
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      true,
                      false,
                      false,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          nullptr,
                          unbucketize_permute.data_ptr<index_t>());
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));
    }
  } else {
    if (weights.has_value() & bucketize_pos) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            false,
                            true,
                            true,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                new_pos.data_ptr<index_t>(),
                                nullptr);
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (weights.has_value()) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            false,
                            true,
                            false,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                nullptr,
                                nullptr);
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (bucketize_pos) {
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      false,
                      false,
                      true,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          new_pos.data_ptr<index_t>(),
                          nullptr);
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));

    } else {
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      false,
                      false,
                      false,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          nullptr,
                          nullptr);
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));
    }
  }

  return {new_lengths, new_indices, new_weights, new_pos, unbucketize_permute};
}

at::Tensor _float_to_fused8bitrowwise_gpu(const at::Tensor& input) {
  TENSOR_ON_CUDA_GPU(input);
  TORCH_CHECK(input.is_contiguous(), "input must be contiguous");

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(input.get_device());

  const auto input_sizes = input.sizes();
  const auto last_dim = input_sizes.size() - 1;
  const int nrows = c10::size_to_dim_(last_dim, input_sizes);
  const int ncols = input_sizes[last_dim];
  const int ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int output_columns = ncols_aligned + 2 * sizeof(float);

  // Global memory instructions support reading or writing words of size equal
  // to 1, 2, 4, 8, or 16 bytes. Any access (via a variable or a pointer) to
  // data residing in global memory compiles to a single global memory
  // instruction if and only if the size of the data type is 1, 2, 4, 8, or 16
  // bytes and the data is naturally aligned (i.e., its address is a multiple of
  // that size).
  auto output_dims = input_sizes.vec();
  output_dims[last_dim] = output_columns;
  auto output = at::empty(
      output_dims, // 4 = sizeof(float)
      input.options().dtype(at::kByte));

  if (nrows == 0 || ncols == 0) {
    return output;
  }

  constexpr int threads_per_block = 256;
  const auto num_blocks = cuda_calc_xblock_count(nrows, threads_per_block);
  // think unsigned as we use 0, 255

  if (nrows <= 20) {
    _float_to_fused8bitrowwise_cuda_kernel<<<
        num_blocks,
        threads_per_block,
        0,
        at::cuda::getCurrentCUDAStream()>>>(
        input.data_ptr<float>(), nrows, ncols, output.data_ptr<std::uint8_t>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    // range_tensor is used to store the range for each embedding row.
    // We save range/255.0f as row scale, and use 255.0f / (range + kEpsilon) to
    // quantize. This will guarantee the numerical match but bring some perf
    // regression.
    auto range_tensor = at::empty({nrows}, input.options().dtype(at::kFloat));

    {
      // we need a blockDim.x that is a power of 2 no larger than the warp size
      // of 32

      int blockDim_x = 1;
      if (ncols > 16) {
        // max warp size
        blockDim_x = 32;
      } else {
        while (blockDim_x < ncols) {
          blockDim_x <<= 1;
        }
      }

      const int rows_per_block = threads_per_block / blockDim_x;
      const auto num_blocks_warp =
          cuda_calc_xblock_count(nrows, rows_per_block);

      _get_8bit_qparam_cuda_kernel<<<
          num_blocks_warp,
          dim3(blockDim_x, rows_per_block),
          0,
          at::cuda::getCurrentCUDAStream()>>>(
          input.data_ptr<float>(),
          nrows,
          ncols,
          output.data_ptr<std::uint8_t>(),
          range_tensor.data_ptr<float>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }

    {
      const int blockDim_x = std::min(ncols, threads_per_block);
      dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);
      const auto gridDim_x = cuda_calc_xblock_count(ncols, blockDim.x);
      const auto gridDim_y = cuda_calc_block_count(nrows, blockDim.y);
      dim3 gridDim(gridDim_x, gridDim_y);

      _compute_8bit_quantize_cuda_kernel<<<
          gridDim,
          blockDim,
          0,
          at::cuda::getCurrentCUDAStream()>>>(
          input.data_ptr<float>(),
          range_tensor.data_ptr<float>(),
          nrows,
          ncols,
          output.data_ptr<std::uint8_t>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }

  return output;
}

at::Tensor _fused8bitrowwise_to_float_gpu(const at::Tensor& input) {
  TENSOR_ON_CUDA_GPU(input);
  TORCH_CHECK(input.is_contiguous(), "input must be contiguous");

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(input.get_device());

  const auto input_sizes = input.sizes();
  const auto last_dim = input_sizes.size() - 1;
  const int nrows = c10::size_to_dim_(last_dim, input_sizes);
  const int ncols = input_sizes[last_dim];
  const int ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int output_columns = ncols_aligned - 2 * sizeof(float);

  // Global memory instructions support reading or writing words of size equal
  // to 1, 2, 4, 8, or 16 bytes. Any access (via a variable or a pointer) to
  // data residing in global memory compiles to a single global memory
  // instruction if and only if the size of the data type is 1, 2, 4, 8, or 16
  // bytes and the data is naturally aligned (i.e., its address is a multiple of
  // that size).
  auto output_dims = input_sizes.vec();
  output_dims[last_dim] = output_columns;
  auto output = at::empty(
      output_dims, // 4 = sizeof(float)
      input.options().dtype(at::kFloat));

  if (nrows == 0 || output_columns == 0) {
    return output;
  }

  constexpr int threads_per_block = 256;

  const int blockDim_x = std::min(threads_per_block, output_columns);
  dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);

  const auto gridDim_x = cuda_calc_xblock_count(output_columns, blockDim.x);
  const auto gridDim_y = cuda_calc_block_count(nrows, blockDim.y);
  dim3 gridDim(gridDim_x, gridDim_y);

  _fused8bitrowwise_to_float_cuda_kernel<<<
      gridDim,
      blockDim,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      input.data_ptr<std::uint8_t>(), nrows, ncols, output.data_ptr<float>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return output;
}

at::Tensor _float_to_fusednbitrowwise_gpu(
    const at::Tensor& input,
    const int64_t bit_rate) {
  TENSOR_ON_CUDA_GPU(input);
  TENSOR_NDIM_EQUALS(input, 2);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(input.get_device());

  const int nrows = input.size(0);
  const int ncols = input.size(1);
  const int num_elem_per_byte = 8 / bit_rate;
  TORCH_CHECK(
      ncols % (2 * num_elem_per_byte) == 0,
      "ncols needs to be multiple of 2 Bytes (half type size) to make the address aligned");
  const int output_columns =
      (ncols + num_elem_per_byte - 1) / num_elem_per_byte +
      2 * sizeof(at::Half);

  // Global memory instructions support reading or writing words of size equal
  // to 1, 2, 4, 8, or 16 bytes. Any access (via a variable or a pointer) to
  // data residing in global memory compiles to a single global memory
  // instruction if and only if the size of the data type is 1, 2, 4, 8, or 16
  // bytes and the data is naturally aligned (i.e., its address is a multiple of
  // that size).
  auto output = at::empty(
      {nrows, output_columns},
      input.options().dtype(at::kByte)); // at::kBytes for uint8_t

  if (nrows == 0 || ncols == 0) {
    return output;
  }

  constexpr auto threads_per_block = 256;
  const auto num_blocks = cuda_calc_xblock_count(nrows, threads_per_block);
  // think unsigned as we use 0, 255

  _float_to_fusednbitrowwise_cuda_kernel<<<
      num_blocks,
      threads_per_block,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      bit_rate,
      input.data_ptr<float>(),
      nrows,
      ncols,
      output.data_ptr<std::uint8_t>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return output;
}

at::Tensor _fusednbitrowwise_to_float_gpu(
    const at::Tensor& input,
    const int64_t bit_rate) {
  TENSOR_ON_CUDA_GPU(input);
  TENSOR_NDIM_EQUALS(input, 2);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(input.get_device());

  const int nrows = input.size(0);
  const int ncols = input.size(1);
  const int num_elem_per_byte = 8 / bit_rate;
  const int output_columns = (ncols - 2 * sizeof(at::Half)) * num_elem_per_byte;

  // Global memory instructions support reading or writing words of size equal
  // to 1, 2, 4, 8, or 16 bytes. Any access (via a variable or a pointer) to
  // data residing in global memory compiles to a single global memory
  // instruction if and only if the size of the data type is 1, 2, 4, 8, or 16
  // bytes and the data is naturally aligned (i.e., its address is a multiple of
  // that size).
  auto output = at::empty(
      {nrows, output_columns}, // 4 = sizeof(float)
      input.options().dtype(at::kFloat)); // at::kBytes for uint8_t

  if (nrows == 0 || output_columns == 0) {
    return output;
  }

  constexpr int threads_per_block = 256;

  const int blockDim_x = std::min(output_columns, threads_per_block);
  dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);
  const auto gridDim_x = cuda_calc_xblock_count(output_columns, blockDim.x);
  const auto gridDim_y = cuda_calc_block_count(nrows, blockDim.y);
  dim3 gridDim(gridDim_x, gridDim_y);

  _fusednbitrowwise_to_float_cuda_kernel<<<
      gridDim,
      blockDim,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      bit_rate,
      input.data_ptr<uint8_t>(),
      nrows,
      ncols,
      output.data_ptr<float>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return output;
}

} // namespace at
