#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include "fbgemm_gpu/sparse_ops.cuh"
#include "fbgemm_gpu/sparse_ops.h"
#include "fbgemm_gpu/sparse_ops_utils.h"

#include <ATen/ATen.h>
#include <ATen/core/op_registration/op_registration.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/library.h>

// clang-format off
#include "fbgemm_gpu/cub_namespace_prefix.cuh"
#include "cub/device/device_scan.cuh"
#include "fbgemm_gpu/cub_namespace_postfix.cuh"
// clang-format on

#include "fbgemm_gpu/embedding_backward_template_helpers.cuh"
#include "fbgemm_gpu/fbgemm_cuda_utils.cuh"
#include "fbgemm_gpu/split_embeddings_utils.cuh"

using Tensor = at::Tensor;

namespace fbgemm_gpu {

std::tuple<uint32_t, uint32_t, uint32_t> calc_offsets_range_thread_block(
    const int64_t output_size,
    const int64_t num_seq) {
  uint32_t threads_per_block;
  uint32_t vector_size;
  if (output_size / num_seq < 2) {
    threads_per_block = 512;
    vector_size = 2;
  } else if (output_size / num_seq < 4) {
    threads_per_block = 512;
    vector_size = 4;
  } else if (output_size / num_seq < 64) {
    threads_per_block = 512;
    vector_size = 8;
  } else if (output_size / num_seq < 128) {
    threads_per_block = 512;
    vector_size = 16;
  } else {
    threads_per_block = 512;
    vector_size = 32;
  }
  uint32_t rows_per_block = threads_per_block / vector_size;
  const auto num_blocks = cuda_calc_xblock_count(num_seq, rows_per_block);

  return std::make_tuple(num_blocks, rows_per_block, vector_size);
}

Tensor offsets_range_cuda(const Tensor& offsets, int64_t range_size) {
  TENSOR_ON_CUDA_GPU(offsets);
  TENSOR_NDIM_EQUALS(offsets, 1);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(offsets.get_device());

  auto offsets_arg = at::TensorArg(offsets, "offsets", 1);
  checkScalarTypes("_offsets_range_cuda", offsets_arg, {at::kLong, at::kInt});
  auto range = at::empty(range_size, offsets.options());
  if (range_size == 0) {
    return range;
  }
  auto offsets_contig = offsets.contiguous();
  int64_t N = offsets_contig.numel();

  uint32_t vector_size;
  uint32_t rows_per_block;
  uint32_t num_blocks;
  std::tie(num_blocks, rows_per_block, vector_size) =
      calc_offsets_range_thread_block(range_size, N);
  dim3 threads(vector_size, rows_per_block);
  AT_DISPATCH_INDEX_TYPES(
      offsets_contig.scalar_type(), "offsets_range_kernel", [&]() {
        _offsets_range_cuda_kernel<index_t>
            <<<num_blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                N,
                range_size,
                offsets_contig.data_ptr<index_t>(),
                range.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

  return range;
}

Tensor segment_sum_csr_cuda(
    const int64_t batch_size,
    const Tensor& csr_seg,
    const Tensor& values) {
  TENSOR_ON_CUDA_GPU(csr_seg);
  TENSOR_ON_CUDA_GPU(values);

  TENSORS_ON_SAME_DEVICE(csr_seg, values);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(values.get_device());

  auto output = at::empty(csr_seg.numel() - 1, values.options());
  constexpr uint32_t threads_per_block = 256;
  const uint32_t num_blocks = csr_seg.numel() - 1;
  AT_DISPATCH_ALL_TYPES(values.type(), "_segment_sum_csr_cuda", [&]() {
    _segment_sum_csr_cuda_kernel<scalar_t>
        <<<num_blocks,
           threads_per_block,
           0,
           at::cuda::getCurrentCUDAStream()>>>(
            csr_seg.numel() - 1,
            batch_size,
            csr_seg.data_ptr<int>(),
            values.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
  return output;
}

Tensor asynchronous_inclusive_cumsum_gpu(const Tensor& t_in) {
  TENSOR_ON_CUDA_GPU(t_in);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == at::kInt || t_in.dtype() == at::kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  auto t_out = at::empty_like(t_in);
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      t_in.options().dtype(at::kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

Tensor asynchronous_exclusive_cumsum_gpu(const Tensor& t_in) {
  TENSOR_ON_CUDA_GPU(t_in);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == at::kInt || t_in.dtype() == at::kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  auto t_out = at::empty_like(t_in);
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_exclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::ExclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      t_in.options().dtype(at::kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_exclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::ExclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

Tensor asynchronous_complete_cumsum_gpu(const Tensor& t_in) {
  TENSOR_ON_CUDA_GPU(t_in);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == at::kInt || t_in.dtype() == at::kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  TORCH_CHECK(t_in.dim() == 1);
  auto t_out = at::empty({t_in.numel() + 1}, t_in.options());
  t_out[0].zero_();
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>() + 1,
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      t_in.options().dtype(at::kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>() + 1,
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

std::tuple<Tensor, Tensor, c10::optional<Tensor>> permute_sparse_data_cuda(
    const Tensor& permute,
    const Tensor& lengths,
    const Tensor& indices,
    const c10::optional<Tensor>& weights,
    const c10::optional<int64_t>& permuted_lengths_sum) {
  TENSOR_ON_CUDA_GPU(permute);
  TENSOR_ON_CUDA_GPU(lengths);
  TENSOR_ON_CUDA_GPU(indices);
  TENSOR_ON_CUDA_GPU(weights);

  TENSORS_ON_SAME_DEVICE(permute, lengths);
  TENSORS_ON_SAME_DEVICE(permute, indices);
  TENSORS_ON_SAME_DEVICE(permute, weights);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(indices.get_device());

  const auto permute_contig = permute.contiguous();
  const auto lengths_contig = lengths.contiguous();
  const auto indices_contig = indices.contiguous();
  // the data to permute over can be less or more with or without
  // repetitions
  const auto T = permute.numel();
  const auto T_ = lengths.size(0);
  const auto B = lengths.view({lengths.sizes()[0], -1}).sizes()[1];

  Tensor permuted_lengths;
  Tensor permuted_indices;
  Tensor permuted_weights;

  permuted_lengths = at::empty({T, B}, lengths.options());

  constexpr int32_t threads_1 = 256;
  const auto blocks_1 = cuda_calc_xblock_count(B * T, threads_1);
  AT_DISPATCH_INDEX_TYPES(
      lengths.scalar_type(), "permute_lengths_kernel", ([&] {
        permute_lengths_kernel<index_t>
            <<<blocks_1, threads_1, 0, at::cuda::getCurrentCUDAStream()>>>(
                T,
                B,
                lengths_contig.data_ptr<index_t>(),
                permute.data_ptr<int32_t>(),
                permuted_lengths.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));

  // convert lengths to offsets
  const auto input_offsets = asynchronous_exclusive_cumsum_gpu(lengths_contig);
  const auto output_offsets =
      asynchronous_exclusive_cumsum_gpu(permuted_lengths);
  int64_t permuted_indices_size = 0;
  if (permuted_lengths_sum.has_value()) {
    permuted_indices_size = permuted_lengths_sum.value();
  } else {
    permuted_indices_size = permuted_lengths.sum().item<int64_t>();
  }

  constexpr int32_t BT_blocks = 32;
  dim3 threads_2(32, BT_blocks);
  const auto blocks_2 = cuda_calc_xblock_count(B * T, BT_blocks);
  permuted_indices = at::empty(permuted_indices_size, indices.options());

  AT_DISPATCH_INDEX_TYPES(
      input_offsets.scalar_type(), "permute_data_kernel_1", ([&] {
        using offsets_t = index_t;
        AT_DISPATCH_ALL_TYPES_AND(
            at::ScalarType::Half,
            indices.scalar_type(),
            "permute_data_kernel_2",
            ([&] {
              using indices_t = scalar_t;
              if (weights.has_value()) {
                const Tensor weights_value = weights.value();
                const auto weights_value_contig = weights_value.contiguous();
                permuted_weights =
                    at::empty(permuted_indices_size, weights_value.options());
                AT_DISPATCH_ALL_TYPES_AND(
                    at::ScalarType::Half,
                    weights_value.scalar_type(),
                    "permute_data_kernel_3",
                    ([&] {
                      using weights_t = scalar_t;
                      permute_data_kernel<true, offsets_t, indices_t, weights_t>
                          <<<blocks_2,
                             threads_2,
                             0,
                             at::cuda::getCurrentCUDAStream()>>>(
                              permuted_indices_size,
                              T,
                              B,
                              indices_contig.data_ptr<indices_t>(),
                              weights_value_contig.data_ptr<weights_t>(),
                              permute_contig.data_ptr<int32_t>(),
                              input_offsets.data_ptr<offsets_t>(),
                              output_offsets.data_ptr<offsets_t>(),
                              permuted_indices.data_ptr<indices_t>(),
                              permuted_weights.data_ptr<weights_t>());
                      C10_CUDA_KERNEL_LAUNCH_CHECK();
                    })); // for each weights_t
              } else {
                permute_data_kernel<false, offsets_t, indices_t, std::nullptr_t>
                    <<<blocks_2,
                       threads_2,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        permuted_indices_size,
                        T,
                        B,
                        indices_contig.data_ptr<indices_t>(),
                        nullptr,
                        permute_contig.data_ptr<int32_t>(),
                        input_offsets.data_ptr<offsets_t>(),
                        output_offsets.data_ptr<offsets_t>(),
                        permuted_indices.data_ptr<indices_t>(),
                        nullptr);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
            })); // for each indices_t
      })); // for each offsets_t
  return {permuted_lengths, permuted_indices, permuted_weights};
}

// This function partitions sparse features
// continuously along the sparse dimension into my_size blocks
std::tuple<
    Tensor,
    Tensor,
    c10::optional<Tensor>,
    c10::optional<Tensor>,
    c10::optional<Tensor>>
block_bucketize_sparse_features_cuda(
    Tensor lengths,
    Tensor indices,
    bool bucketize_pos,
    bool sequence,
    Tensor block_sizes,
    int64_t my_size,
    c10::optional<Tensor> weights) {
  TENSOR_ON_CUDA_GPU(lengths);
  TENSOR_ON_CUDA_GPU(indices);
  TENSORS_ON_SAME_DEVICE(lengths, indices);
  TENSOR_ON_CUDA_GPU(weights);
  TENSORS_ON_SAME_DEVICE(lengths, weights);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(lengths.get_device());
  // allocate tensors and buffers
  const int lengths_size = lengths.numel();
  const int T = block_sizes.numel();
  const int B = lengths_size / T;
  const int new_lengths_size = lengths_size * my_size;
  auto offsets = at::empty({lengths_size}, lengths.options());
  auto new_lengths = at::zeros({new_lengths_size}, lengths.options());
  auto new_offsets = at::empty({new_lengths_size}, lengths.options());
  auto new_indices = at::empty_like(indices);
  auto lengths_contig = lengths.contiguous();
  auto indices_contig = indices.contiguous();
  auto offsets_contig = offsets.contiguous();
  Tensor new_weights;
  Tensor new_pos;
  Tensor unbucketize_permute;
  // count nonzeros
  offsets_contig = asynchronous_inclusive_cumsum_gpu(lengths);
  int threads_per_block = 256;
  int num_blocks = (lengths_size + threads_per_block - 1) / threads_per_block;
  AT_DISPATCH_INDEX_TYPES(
      offsets_contig.scalar_type(),
      "_block_bucketize_sparse_features_cuda_kernel1",
      ([&] {
        using offset_t = index_t;
        AT_DISPATCH_INDEX_TYPES(
            indices_contig.scalar_type(),
            "_block_bucketize_sparse_features_cuda_kernel2",
            ([&] {
              _block_bucketize_sparse_features_cuda_kernel1<<<
                  num_blocks,
                  threads_per_block,
                  0,
                  at::cuda::getCurrentCUDAStream()>>>(
                  lengths_size,
                  B,
                  block_sizes.data_ptr<index_t>(),
                  my_size,
                  offsets_contig.data_ptr<offset_t>(),
                  indices_contig.data_ptr<index_t>(),
                  new_lengths.data_ptr<offset_t>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));

  // bucketize nonzeros
  new_offsets = asynchronous_exclusive_cumsum_gpu(new_lengths);
  if (sequence) {
    const auto lengths_sum = indices.numel();
    unbucketize_permute = at::empty({lengths_sum}, indices.options());
    if (weights.has_value() & bucketize_pos) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            true,
                            true,
                            true,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                new_pos.data_ptr<index_t>(),
                                unbucketize_permute.data_ptr<index_t>());
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));
    } else if (weights.has_value()) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            true,
                            true,
                            false,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                nullptr,
                                unbucketize_permute.data_ptr<index_t>());
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (bucketize_pos) {
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      true,
                      false,
                      true,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          new_pos.data_ptr<index_t>(),
                          unbucketize_permute.data_ptr<index_t>());
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));

    } else {
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      true,
                      false,
                      false,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          nullptr,
                          unbucketize_permute.data_ptr<index_t>());
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));
    }
  } else {
    if (weights.has_value() & bucketize_pos) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            false,
                            true,
                            true,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                new_pos.data_ptr<index_t>(),
                                nullptr);
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (weights.has_value()) {
      Tensor weights_value = weights.value();
      auto weights_value_contig = weights_value.contiguous();
      new_weights = at::empty_like(weights_value);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_bucketize_sparse_features_weight_cuda_kernel2_2",
                ([&] {
                  AT_DISPATCH_FLOATING_TYPES(
                      weights_value.scalar_type(),
                      "_block_bucketize_sparse_features_cuda_weight_kernel2_3",
                      ([&] {
                        _block_bucketize_sparse_features_cuda_kernel2<
                            false,
                            true,
                            false,
                            offset_t,
                            index_t,
                            scalar_t>
                            <<<num_blocks,
                               threads_per_block,
                               0,
                               at::cuda::getCurrentCUDAStream()>>>(
                                lengths_size,
                                B,
                                block_sizes.data_ptr<index_t>(),
                                my_size,
                                offsets_contig.data_ptr<offset_t>(),
                                indices_contig.data_ptr<index_t>(),
                                weights_value_contig.data_ptr<scalar_t>(),
                                new_offsets.data_ptr<offset_t>(),
                                new_indices.data_ptr<index_t>(),
                                new_weights.data_ptr<scalar_t>(),
                                nullptr,
                                nullptr);
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                      }));
                }));
          }));

    } else if (bucketize_pos) {
      new_pos = at::empty_like(indices);
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      false,
                      false,
                      true,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          new_pos.data_ptr<index_t>(),
                          nullptr);
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));

    } else {
      AT_DISPATCH_INDEX_TYPES(
          offsets_contig.scalar_type(),
          "_bucketize_sparse_features_weight_cuda_kernel2_1",
          ([&] {
            using offset_t = index_t;
            AT_DISPATCH_INDEX_TYPES(
                indices_contig.scalar_type(),
                "_block_bucketize_sparse_features_cuda_kernel2_2",
                ([&] {
                  _block_bucketize_sparse_features_cuda_kernel2<
                      false,
                      false,
                      false,
                      offset_t,
                      index_t,
                      std::nullptr_t>
                      <<<num_blocks,
                         threads_per_block,
                         0,
                         at::cuda::getCurrentCUDAStream()>>>(
                          lengths_size,
                          B,
                          block_sizes.data_ptr<index_t>(),
                          my_size,
                          offsets_contig.data_ptr<offset_t>(),
                          indices_contig.data_ptr<index_t>(),
                          nullptr,
                          new_offsets.data_ptr<offset_t>(),
                          new_indices.data_ptr<index_t>(),
                          nullptr,
                          nullptr,
                          nullptr);
                  C10_CUDA_KERNEL_LAUNCH_CHECK();
                }));
          }));
    }
  }

  return {new_lengths, new_indices, new_weights, new_pos, unbucketize_permute};
}

template <typename Dtype>
__global__ void reorder_batched_ad_lengths_kernel(
    // reorder lengths from (ragged) [B  x T x #num_ads_b)] to
    // [T][B][#num_ads_b], i.e. [T][sum(#num_ads_b)].
    const at::PackedTensorAccessor32<Dtype, 1, at::RestrictPtrTraits>
        cat_ad_lengths,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        batch_offsets,
    at::PackedTensorAccessor32<Dtype, 1, at::RestrictPtrTraits>
        reordered_cat_ad_lengths,
    int32_t T) {
  const int32_t B = batch_offsets.size(0) - 1;

  const int32_t num_ads_in_batch = batch_offsets[B];
  // warp-per-segment.
  const int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
  const int32_t b = b_t % B;
  const int32_t t = b_t / B;
  if (t >= T) {
    return;
  }

  const int32_t num_ads_b = batch_offsets[b + 1] - batch_offsets[b];
  const int32_t input_segment_start = T * batch_offsets[b] + t * num_ads_b;
  const int32_t output_segment_start = t * num_ads_in_batch + batch_offsets[b];

  for (int32_t i = threadIdx.x; i < num_ads_b; i += blockDim.x) {
    reordered_cat_ad_lengths[output_segment_start + i] =
        cat_ad_lengths[input_segment_start + i];
  }
}

Tensor reorder_batched_ad_lengths_gpu(
    const Tensor& cat_ad_lengths,
    const Tensor& batch_offsets,
    const int64_t num_ads_in_batch) {
  TENSOR_ON_CUDA_GPU(cat_ad_lengths);
  TENSOR_ON_CUDA_GPU(batch_offsets);
  TENSORS_ON_SAME_DEVICE(cat_ad_lengths, batch_offsets);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(cat_ad_lengths.get_device());

  const int64_t B = batch_offsets.numel() - 1;
  const int64_t T = cat_ad_lengths.numel() / num_ads_in_batch;

  Tensor reordered_cat_ad_lengths = at::empty_like(cat_ad_lengths);

  const dim3 threads(32, 32);
  const dim3 blocks((B * T + 32 - 1) / 32);

  AT_DISPATCH_ALL_TYPES(
      cat_ad_lengths.type(), "reorder_batched_ad_lengths_gpu_kernel", ([&] {
        reorder_batched_ad_lengths_kernel<scalar_t>
            <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                cat_ad_lengths
                    .packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
                batch_offsets
                    .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                reordered_cat_ad_lengths
                    .packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
                T);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
  return reordered_cat_ad_lengths;
}

template <typename Dtype>
__global__ void reorder_batched_ad_indices_kernel(
    // reorder indices from (ragged) [B  x T x #num_ads_b x length_{b, t, a})]
    // to [T][B][#num_ads_b][length_{b, t, a}], i.e. [sum(length_{b, t, a})],
    // laid out as [T][B][A][L] (if all lengths were equal).
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        cat_ad_offsets,
    const at::PackedTensorAccessor32<Dtype, 1, at::RestrictPtrTraits>
        cat_ad_indices,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        reordered_cat_ad_offsets,
    at::PackedTensorAccessor32<Dtype, 1, at::RestrictPtrTraits>
        reordered_cat_ad_indices,
    const at::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        batch_offsets,
    int32_t T) {
  const int32_t B = batch_offsets.size(0) - 1;
  const int32_t num_ads_in_batch = batch_offsets[B];
  // warp-per-segment.
  const int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
  const int32_t b = b_t % B;
  const int32_t t = b_t / B;
  if (t >= T) {
    return;
  }
  // for each ad,
  const int32_t num_ads_b = batch_offsets[b + 1] - batch_offsets[b];
  const int32_t b_t_start = T * batch_offsets[b] + t * num_ads_b;
  const int32_t input_segment_offset_start =
      T * batch_offsets[b] + t * num_ads_b;
  const int32_t input_segment_offset_end =
      T * batch_offsets[b] + t * num_ads_b + num_ads_b;

  // Idea: we want to copy the entire segment of size sum_a(length_{b, t, a})
  // from starting point (given by cat_ad_offsets[b, t])
  // to end point (given by reordered_cat_ad_indices[t][b])
  const int32_t input_segment_start =
      cat_ad_offsets[input_segment_offset_start];
  const int32_t input_segment_end = cat_ad_offsets[input_segment_offset_end];

  const int32_t output_segment_offset_start =
      t * num_ads_in_batch + batch_offsets[b];
  const int32_t output_segment_start =
      reordered_cat_ad_offsets[output_segment_offset_start];

  for (int32_t i = threadIdx.x; i < input_segment_end - input_segment_start;
       i += blockDim.x) {
    reordered_cat_ad_indices[output_segment_start + i] =
        cat_ad_indices[input_segment_start + i];
  }
}

Tensor reorder_batched_ad_indices_gpu(
    const Tensor& cat_ad_offsets,
    const Tensor& cat_ad_indices,
    const Tensor& reordered_cat_ad_offsets,
    const Tensor& batch_offsets,
    const int64_t num_ads_in_batch) {
  TENSOR_ON_CUDA_GPU(cat_ad_offsets);
  TENSOR_ON_CUDA_GPU(cat_ad_indices);
  TENSOR_ON_CUDA_GPU(reordered_cat_ad_offsets);
  TENSOR_ON_CUDA_GPU(batch_offsets);
  TENSORS_ON_SAME_DEVICE(cat_ad_offsets, cat_ad_indices);
  TENSORS_ON_SAME_DEVICE(cat_ad_offsets, reordered_cat_ad_offsets);
  TENSORS_ON_SAME_DEVICE(cat_ad_offsets, batch_offsets);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(cat_ad_offsets.get_device());

  const int64_t B = batch_offsets.numel() - 1;
  const int64_t T = (cat_ad_offsets.numel() - 1) / num_ads_in_batch;
  Tensor reordered_cat_ad_indices = at::empty_like(cat_ad_indices);

  const dim3 threads(32, 32);
  const dim3 blocks((B * T + 32 - 1) / 32);

  AT_DISPATCH_ALL_TYPES(
      cat_ad_indices.type(), "reorder_batched_ad_indices_gpu_kernel", ([&] {
        reorder_batched_ad_indices_kernel<scalar_t>
            <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                cat_ad_offsets
                    .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                cat_ad_indices
                    .packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
                reordered_cat_ad_offsets
                    .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                reordered_cat_ad_indices
                    .packed_accessor32<scalar_t, 1, at::RestrictPtrTraits>(),
                batch_offsets
                    .packed_accessor32<int32_t, 1, at::RestrictPtrTraits>(),
                T);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
  return reordered_cat_ad_indices;
}

// Forward kernel for batched unary embedding op
template <typename scalar_t, typename index_t>
__global__ void batched_unary_embeddings_forward_kernel(
    const int32_t N,
    const int32_t B,
    const int32_t T,
    const scalar_t* __restrict__ weight, // N * sum(E) * 1 (embedding dimension
                                         // is 1)
    const index_t* __restrict__ table_offsets,
    const index_t* __restrict__ offsets,
    const index_t* __restrict__ indices,
    scalar_t* __restrict__ output // N * B * T
) {
  index_t sum_E = table_offsets[T];
  int32_t b = blockIdx.x * blockDim.x + threadIdx.x;
  if (b >= B) {
    return;
  }
  int32_t t = blockIdx.y;
  int32_t n = blockIdx.z;
  index_t table_offset = table_offsets[t];
  index_t indices_start = offsets[t * B + b];
  index_t indices_end = offsets[t * B + b + 1];
  int32_t L = indices_end - indices_start;
  at::acc_type<scalar_t, true> sum = 0.0;
  for (int32_t l = 0; l < L; ++l) {
    auto idx = __ldg(&indices[indices_start + l]);
    sum += weight[n * sum_E + table_offset + idx + 0];
  }
  output[(n * B + b) * T + t] = sum;
}

Tensor batched_unary_embeddings_forward_cuda(
    const Tensor& weight,
    const Tensor& table_offsets,
    const Tensor& offsets,
    const Tensor& indices) {
  TENSOR_CONTIGUOUS_AND_ON_CUDA_GPU(table_offsets);
  TENSOR_CONTIGUOUS_AND_ON_CUDA_GPU(weight);
  TENSOR_CONTIGUOUS_AND_ON_CUDA_GPU(offsets);
  TENSOR_CONTIGUOUS_AND_ON_CUDA_GPU(indices);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(weight.get_device());
  // N: number of tasks, T: number of tables, B: batch size
  const int32_t N = weight.size(0);
  const int32_t T = table_offsets.numel() - 1;
  const int32_t B = (offsets.numel() - 1) / T;
  TORCH_CHECK(N > 0);
  TORCH_CHECK(B > 0);
  TORCH_CHECK(T > 0);
  TORCH_CHECK(T <= 65535);
  TORCH_CHECK(N <= 65535);
  int32_t threads = std::min<int32_t>(B, 512);
  dim3 blocks(cuda_calc_xblock_count(B, threads), T, N);
  auto output = at::empty({N, B, T}, weight.options());
  AT_DISPATCH_INDEX_TYPES(
      indices.type(), "batched_unary_embeddings_forward_kernel", ([&] {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
            weight.type(), "batched_unary_embeddings_forward_kernel", ([&] {
              batched_unary_embeddings_forward_kernel<scalar_t>
                  <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                      N,
                      B,
                      T,
                      weight.data_ptr<scalar_t>(),
                      table_offsets.data_ptr<index_t>(),
                      offsets.data_ptr<index_t>(),
                      indices.data_ptr<index_t>(),
                      output.data_ptr<scalar_t>());
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));
  return output;
}

// Backward kernel for batched unary embedding op
// We sort input indices so we don't have race conditions, an approach similar
// to the usual split table batched embedding backward.
// We can think of the following alternatives but each with challenges:
// 1) Assign output elements to different threads. Each thread scan all indices
//    corresponding to the table it owns but only accumulate gradients when an
//    index value matches with the output element it owns.
//    A challenge is each thread need to binary search to map from [0 .. sum_E]
//    to table id.
// 2) Densify indices and offsets to create [B, sum_E] matrix. Then, do batched
//    GEMM where ith GEMM multiplies [N, B] submatrix of grad_output with
//    [B, E_i] submatrix where E_i is the num of embeddings of ith table.
//    Concatenating the GEMM outputs will result in [N, B, T]
//    A challenge is there's no available batched GEMM routine with varying K
//    dimension.
template <typename scalar_t, typename index_t>
__global__ void batched_unary_embeddings_backward_kernel(
    const int32_t N,
    const int32_t B,
    const int32_t T,
    const scalar_t* __restrict__ grad_output, // [N * B * T]
    const index_t* __restrict__ table_offsets,
    scalar_t* __restrict__ grad_weight, // [N * sum_E * 1] (embedding
                                        // dimension is 1)
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_run,
    const int32_t* __restrict__ sorted_linear_indices_cumulative_run_lengths,
    const int32_t* __restrict__ sorted_infos,
    const int32_t* __restrict__ sorted_linear_indices_num_runs,
    FixedDivisor fd) {
  int32_t run_id = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t n = blockIdx.y;
  if (n >= N) {
    return;
  }
  if (run_id >= sorted_linear_indices_run.size(0)) {
    return;
  }
  if (run_id >= sorted_linear_indices_num_runs[0]) {
    return;
  }
  int64_t linear_index = sorted_linear_indices_run[run_id];
  int32_t segment_start = sorted_linear_indices_cumulative_run_lengths[run_id];
  int32_t segment_end =
      sorted_linear_indices_cumulative_run_lengths[run_id + 1];
  int32_t SL = segment_end - segment_start;

  if (SL == 0) {
    return;
  }

  // now, each segment corresponds to exactly one table `t` and row in
  // that table (`idx`). Thus, we can hoist out some of the book-keeping.
  auto info = sorted_infos[segment_start];
  int t = fd.Div(info);

  at::acc_type<scalar_t, true> grad_sum = 0.0;
  for (int32_t sl = 0; sl < SL; ++sl) {
    int32_t b = fd.Mod(sorted_infos[segment_start + sl]);
    grad_sum += grad_output[(n * B + b) * T + t];
  }

  index_t table_offset = table_offsets[t];
  index_t sum_E = table_offsets[T];
  int64_t idx = linear_index - table_offset;
  grad_weight[n * sum_E + table_offset + idx] = grad_sum;
}

Tensor batched_unary_embeddings_backward_cuda(
    const Tensor& grad_output,
    const Tensor& weight,
    const Tensor& table_offsets,
    const Tensor& offsets,
    const Tensor& indices) {
  TENSOR_ON_CUDA_GPU(grad_output);
  TENSOR_ON_CUDA_GPU(weight);
  TENSOR_ON_CUDA_GPU(table_offsets);
  TENSOR_ON_CUDA_GPU(offsets);
  TENSOR_ON_CUDA_GPU(indices);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(grad_output.get_device());

  // N: number of tasks, T: number of tables, B: batch size
  const int32_t N = grad_output.size(0);
  const int32_t B = grad_output.size(1);
  const int32_t T = grad_output.size(2);
  TORCH_CHECK(N > 0);
  TORCH_CHECK(B > 0);
  TORCH_CHECK(T > 0);

  // weight: [N, sum_E]
  // total_hash_size_bits = log2(sum_E)
  int64_t total_hash_size_bits = log2(weight.numel() / N) + 1;

  Tensor linear_indices, linear_indices_sorted;
  Tensor infos_sorted;
  Tensor sorted_linear_indices_run, sorted_linear_indices_run_lengths,
      sorted_linear_indices_num_runs,
      sorted_linear_indices_cumulative_run_lengths;
  std::tie(
      linear_indices,
      linear_indices_sorted,
      infos_sorted,
      sorted_linear_indices_run,
      sorted_linear_indices_run_lengths,
      sorted_linear_indices_num_runs,
      sorted_linear_indices_cumulative_run_lengths) =
      transpose_embedding_input(
          table_offsets, total_hash_size_bits, indices, offsets);

  int threads = std::min<int32_t>(sorted_linear_indices_run.numel(), 512);
  dim3 blocks(
      cuda_calc_xblock_count(sorted_linear_indices_run.numel(), threads), N);
  auto grad_weight = at::zeros_like(weight);
  AT_DISPATCH_INDEX_TYPES(
      indices.type(), "batched_unary_embeddings_backward_kernel", ([&] {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
            grad_output.type(),
            "batched_unary_embeddings_backward_kernel",
            ([&] {
              batched_unary_embeddings_backward_kernel<scalar_t>
                  <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                      N,
                      B,
                      T,
                      grad_output.data_ptr<scalar_t>(),
                      table_offsets.data_ptr<index_t>(),
                      grad_weight.data_ptr<scalar_t>(),
                      sorted_linear_indices_run.packed_accessor32<
                          index_t,
                          1,
                          at::RestrictPtrTraits>(),
                      sorted_linear_indices_cumulative_run_lengths
                          .data_ptr<int32_t>(),
                      infos_sorted.data_ptr<int32_t>(),
                      sorted_linear_indices_num_runs.data_ptr<int32_t>(),
                      FixedDivisor(B));
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));
  return grad_weight;
}

} // namespace fbgemm_gpu
