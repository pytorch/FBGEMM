#include <iostream>
#include <ctime>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "util.cuh"
#include <chrono>
#include <thread>


int main(int argc, char* argv[]) {

    // 1171 MBs
    int32_t rows = 1200000;
    int32_t dim = 256;
    int32_t n_loop = 65536;
    float milliseconds;

    if(argc == 3){
        rows = atoi(argv[1]);
        dim = atoi(argv[2]);
    }

    at::Device device(at::kCUDA);
    auto float_tensor_options = at::TensorOptions().device(device).dtype(at::kFloat);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ============================= empty =============================
    hipEventRecord(start, 0);
    for(int i=0;i<n_loop;i++){
        auto grad_dev_weights = at::empty({rows * dim}, float_tensor_options);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("at::empty -- Total (ms): %.3f, avg_kernel latency:%.3f ms/iter\n", milliseconds, milliseconds/float(n_loop));


    // ============================= zeros =============================
    hipEventRecord(start, 0);
    for(int i=0;i<n_loop;i++){
        auto grad_dev_weights = at::zeros({rows * dim}, float_tensor_options);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("at::zeros -- Total (ms): %.3f, avg_kernel latency:%.3f ms/iter\n", milliseconds, milliseconds/float(n_loop));

    // ============================= empty + zero =============================
    hipEventRecord(start, 0);
    for(int i=0;i<n_loop;i++){
        auto grad_dev_weights = at::empty({rows * dim}, float_tensor_options);
        grad_dev_weights.zero_();
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("empty + zero -- Total (ms): %.3f, avg_kernel latency:%.3f ms/iter\n", milliseconds, milliseconds/float(n_loop));

    // ============================= empty + hipMemset =============================
    hipEventRecord(start, 0);
    for(int i=0;i<n_loop;i++){
        auto grad_dev_weights = at::empty({rows * dim}, float_tensor_options);
        hipMemset(grad_dev_weights.data_ptr<float>(), 0, rows * dim * sizeof(float));
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("empty + hipMemset -- Total (ms): %.3f, avg_kernel latency:%.3f ms/iter\n", milliseconds, milliseconds/float(n_loop));

    return 0;
}
