#include "hip/hip_runtime.h"
#include "fbgemm_put_tbe_backward.cuh"
#include "fbgemm_gpu/split_embeddings_utils.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper Functions
////////////////////////////////////////////////////////////////////////////////

std::tuple<int32_t, uint32_t> adjust_info_B_num_bits(
    int32_t B,
    int32_t T) {
  int32_t info_B_num_bits = DEFAULT_INFO_B_NUM_BITS;
  uint32_t info_B_mask = DEFAULT_INFO_B_MASK;
  uint32_t max_T = MAX_T;
  uint32_t max_B = MAX_B;
  bool invalid_T = T > max_T;
  bool invalid_B = B > max_B;

  TORCH_CHECK(
      !(invalid_T && invalid_B),
      "Not enough infos bits to accommodate T and B. Default num bits = ",
      DEFAULT_INFO_NUM_BITS);

  if (invalid_T) {
    // Reduce info_B_num_bits
    while (invalid_T && !invalid_B && info_B_num_bits > 0) {
      info_B_num_bits--;
      max_T = ((max_T + 1) << 1) - 1;
      max_B = ((max_B + 1) >> 1) - 1;
      invalid_T = T > max_T;
      invalid_B = B > max_B;
    }
  } else if (invalid_B) {
    // Increase info_B_num_bits
    while (!invalid_T && invalid_B && info_B_num_bits < DEFAULT_INFO_NUM_BITS) {
      info_B_num_bits++;
      max_T = ((max_T + 1) >> 1) - 1;
      max_B = ((max_B + 1) << 1) - 1;
      invalid_T = T > max_T;
      invalid_B = B > max_B;
    }
  }

  TORCH_CHECK(
      !invalid_T && !invalid_B,
      "Not enough infos bits to accommodate T and B. Default num bits = ",
      DEFAULT_INFO_NUM_BITS);

  // Recompute info_B_mask using new info_B_num_bits
  info_B_mask = (1u << info_B_num_bits) - 1;

  return {info_B_num_bits, info_B_mask};
}

inline at::Tensor asynchronous_complete_cumsum(at::Tensor t_in) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == at::kInt || t_in.dtype() == at::kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK_LT(t_in.numel(), std::numeric_limits<int32_t>::max());
  TORCH_CHECK_EQ(t_in.dim(), 1);
  auto t_out = at::empty({t_in.numel() + 1}, t_in.options());
  t_out[0].zero_();
  AT_DISPATCH_INDEX_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper1", [&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<index_t>(),
            t_out.data_ptr<index_t>() + 1,
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      });
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)},
      t_in.options().dtype(at::kByte));
  AT_DISPATCH_INDEX_TYPES(
      t_in.scalar_type(), "cub_inclusive_sum_wrapper2", [&] {
        AT_CUDA_CHECK(FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceScan::InclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<index_t>(),
            t_out.data_ptr<index_t>() + 1,
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      });
  return t_out;
}

template <
    typename emb_t,
    typename cache_t,
    size_t kMaxVecsPerThread,
    int32_t kThreadGroupSize = kWarpSize,
    int32_t VEC_WIDTH
>
DEVICE_INLINE void split_sgd_table_update_kernel(
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits>& dev_weights,
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits>& uvm_weights,
    pta::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits>& lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>& weights_placements,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits>& weights_offsets,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>& sorted_lxu_cache_locations,
    Vec4T<at::acc_type<cache_t, true>>* grad_sum,
    const bool stochastic_rounding,
    const at::PhiloxCudaState& stochastic_rounding_philox_args,
    const uint32_t run_id,
    const int32_t D,
    const int32_t t,
    const int64_t idx,
    const int32_t segment_start,
    const uint32_t shfl_sync_mask,
    const int32_t shared_weight_offset,
    float learning_rate = 0
) {
    constexpr auto is_int8 = std::is_same<emb_t, uint8_t>::value;
    const int64_t weights_offset = weights_offsets[t];
    emb_t* __restrict__ weights {nullptr};
    cache_t* __restrict__ cache_weights {nullptr};
    int32_t D_emb = D;
    if (is_int8) {
        D_emb += kINT8QparamsBytes;
    }
    const auto weights_placement = static_cast<PlacementType>(weights_placements[t]);
    if (weights_placement == PlacementType::DEVICE) {
        weights = &dev_weights[weights_offset + idx * D_emb];
    } else {
        weights = &uvm_weights[weights_offset + idx * D_emb];
    }

    struct SharedMemory<Vec4T<at::acc_type<cache_t, true>>> weight_update_buffer;
    Vec4T<at::acc_type<cache_t, true>>* shared_weight_update_row =
        is_int8 ? weight_update_buffer.getPointer() : nullptr;
    auto weight_row_template =
        WeightRow<emb_t, cache_t, at::acc_type<cache_t, true>>(
            weights, cache_weights, D, nullptr);

    weight_row_template.set_stochastic_rounding(
      stochastic_rounding,
      stochastic_rounding_philox_args,
      threadIdx.x + run_id * blockDim.x
    );

    float2 qparams_template;
    if (is_int8 && !cache_weights) {
        qparams_template = weight_row_template.load_qparams();
    }



    float2 qparams_new;
#pragma unroll kMaxVecsPerThread
    for (int32_t i = 0;
        i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
        ++i) {
        int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
        Vec4T<at::acc_type<cache_t, true>> weight_new = weight_row_template.load(d, qparams_template);
        auto& grad = grad_sum[i];

      weight_new.fma_(grad, -learning_rate);

        if (is_int8 && !cache_weights) {
            shared_weight_update_row[
                threadIdx.x + (i * kThreadGroupSize) + shared_weight_offset] = weight_new;
        } else {
            // qparams_new not used if type is not int8
            weight_row_template.store(weight_new, d, qparams_new);
        }
    }

    if (is_int8 && !cache_weights) {
        // Calculate new qparams after row update
        qparams_new = thrust_find_qparams<at::acc_type<cache_t, true>>(
            &shared_weight_update_row[shared_weight_offset], D);
        weight_row_template.store_qparams(qparams_new);

        // Fetch cached updated row from shared mem and quantize on-the-fly
        // when saving to lowp embedding
#pragma unroll kMaxVecsPerThread
        for (int32_t i = 0;
            i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
            ++i) {
            const int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
            weight_row_template.store(
                shared_weight_update_row[threadIdx.x + (i * kThreadGroupSize) + shared_weight_offset],
                d,
                qparams_new);
        }
    }


}

using namespace fbgemm_gpu;

template <typename index_t, typename info_acc_t, bool nobag, bool vbe>
__global__ __launch_bounds__(kMaxThreads) void linearize_index_kernel(
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        hash_size_cumsum,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> indices,
    const at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits> offsets,
    at::PackedTensorAccessor32<info_acc_t, 1, at::RestrictPtrTraits> infos,
    at::PackedTensorAccessor32<index_t, 1, at::RestrictPtrTraits>
        linear_indices,
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    const uint32_t max_T,
    const uint32_t max_B,
    // Use a raw pointer to avoid creating dummy PackedTensorAccessor
    const uint32_t* const __restrict__ vbe_b_t_map,
    FixedDivisor fd) {
  const int32_t T = hash_size_cumsum.size(0) - 1;
  auto b_t = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t b;
  int32_t t;
  const auto total_B = offsets.size(0) - 1;
  bool valid = b_t < total_B;
  // info must be uint32_t (using auto will assign int32_t to info)
  uint32_t info = 0;

  if (vbe && valid) {
    info = vbe_b_t_map[b_t];
    reinterpret_cast<uint32_t*>(&t)[0] = info >> info_B_num_bits;
    reinterpret_cast<uint32_t*>(&b)[0] = info & info_B_mask;
  } else {
    fd.DivMod(b_t, &t, &b);
  }

  const index_t hash_offset = valid ? hash_size_cumsum[t] : -1;
  const index_t indices_start = valid ? offsets[b_t] : -1;
  const int32_t L = valid ? offsets[b_t + 1] - indices_start : 0;
  const int32_t lane_id = threadIdx.x % kWarpSize;

  // Compile-time conditional
  if (nobag) {
    for (int32_t j = 0; j < kWarpSize; ++j) {
      const index_t indices_start_warp =
          fbgemm_gpu::shfl_sync(indices_start, j);
      const int32_t t_warp = fbgemm_gpu::shfl_sync(t, j);
      const int32_t L_warp = fbgemm_gpu::shfl_sync(L, j);
      const index_t hash_offset_warp = fbgemm_gpu::shfl_sync(hash_offset, j);
      for (int32_t i = lane_id; i < L_warp; i += kWarpSize) {
        const index_t idx = __ldg(&indices[indices_start_warp + i]);
        const int64_t l_t = (indices_start_warp + i) * T + t_warp;
        infos[indices_start_warp + i] = l_t;
        linear_indices[indices_start_warp + i] = hash_offset_warp + idx;
      }
    }
  } else {
    // Store t in upper (32 - DEFAULT_INFO_B_NUM_BITS).
    // Store b in lower (DEFAULT_INFO_B_NUM_BITS).
    if (!vbe && valid) {
      info = (reinterpret_cast<uint32_t*>(&t)[0] << info_B_num_bits) |
          reinterpret_cast<uint32_t*>(&b)[0];
    }
    for (int32_t j = 0; j < kWarpSize; ++j) {
      const index_t indices_start_warp =
          fbgemm_gpu::shfl_sync(indices_start, j);
      const uint32_t info_warp = fbgemm_gpu::shfl_sync(info, j);
      const int32_t L_warp = fbgemm_gpu::shfl_sync(L, j);
      const index_t hash_offset_warp = fbgemm_gpu::shfl_sync(hash_offset, j);
      for (int32_t i = lane_id; i < L_warp; i += kWarpSize) {
        const index_t idx = __ldg(&indices[indices_start_warp + i]);
        reinterpret_cast<uint32_t*>(&infos[0])[indices_start_warp + i] =
            info_warp;
        linear_indices[indices_start_warp + i] = hash_offset_warp + idx;
      }
    }
  }
}


std::tuple<
    Tensor /*linear_indices*/,
    Tensor /*linear_indices_sorted*/,
    Tensor /*infos_sorted*/,
    Tensor /*sorted_linear_indices_run*/,
    Tensor /*sorted_linear_indices_run_lengths*/,
    Tensor /*sorted_linear_indices_num_runs*/,
    Tensor /*sorted_linear_indices_cumulative_run_lengths*/>
transpose_embedding_input_local(
    Tensor hash_size_cumsum,
    int64_t total_hash_size_bits,
    Tensor indices,
    Tensor offsets,
    bool nobag,
    const c10::optional<Tensor>& vbe_b_t_map,
    const int64_t info_B_num_bits,
    const int64_t info_B_mask,
    const int64_t total_unique_indices) {

  const bool vbe = vbe_b_t_map.has_value();
  TORCH_CHECK(nobag || !vbe || info_B_num_bits > 0);
  TORCH_CHECK(!vbe || info_B_mask > 0);

  const auto total_B = offsets.size(0) - 1;
  const auto T = hash_size_cumsum.size(0) - 1;

  auto infos = at::empty_like(
      indices, indices.options().dtype(nobag ? at::kLong : at::kInt));
  auto infos_sorted = at::empty_like(infos);
  auto linear_indices = at::empty_like(indices);
  auto linear_indices_sorted = at::empty_like(indices);

  Tensor sorted_linear_indices_run;
  Tensor sorted_linear_indices_run_lengths;
  Tensor sorted_linear_indices_num_runs;

  using at::RestrictPtrTraits;

#define INVOKE_LINEARIZE_INDEX_KERNEL(INFO_ACC_T, NOBAG)                   \
  const auto linearize_index_kernel_ =                                     \
      (vbe ? linearize_index_kernel<index_t, INFO_ACC_T, NOBAG, true>      \
           : linearize_index_kernel<index_t, INFO_ACC_T, NOBAG, false>);   \
  linearize_index_kernel_<<<                                               \
      div_round_up(total_B, kMaxThreads),                                  \
      kMaxThreads,                                                         \
      0,                                                                   \
      at::cuda::getCurrentCUDAStream()>>>(                                 \
      hash_size_cumsum.packed_accessor32<index_t, 1, RestrictPtrTraits>(), \
      indices.packed_accessor32<index_t, 1, RestrictPtrTraits>(),          \
      offsets.packed_accessor32<index_t, 1, RestrictPtrTraits>(),          \
      infos.packed_accessor32<INFO_ACC_T, 1, RestrictPtrTraits>(),         \
      linear_indices.packed_accessor32<index_t, 1, RestrictPtrTraits>(),   \
      info_B_num_bits,                                                     \
      info_B_mask,                                                         \
      (1u << (DEFAULT_INFO_NUM_BITS - info_B_num_bits)) - 1,               \
      (1u << info_B_num_bits) - 1,                                         \
      vbe ? reinterpret_cast<uint32_t*>(vbe_b_t_map.value().data_ptr())    \
          : nullptr,                                                       \
      FixedDivisor(total_B / T));                                          \
  C10_CUDA_KERNEL_LAUNCH_CHECK()

  AT_DISPATCH_INDEX_TYPES(
      infos.scalar_type(), "transpose_embedding_input1", [&] {
        using info_t = index_t;
        AT_DISPATCH_INDEX_TYPES(
            indices.scalar_type(), "transpose_embedding_input2", [&] {
              if (!nobag) {
                INVOKE_LINEARIZE_INDEX_KERNEL(int32_t, false);
              } else {
                INVOKE_LINEARIZE_INDEX_KERNEL(int64_t, true);
              }
              {
                size_t temp_storage_bytes = 0;
                AT_CUDA_CHECK(
                    FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceRadixSort::SortPairs(
                        nullptr,
                        temp_storage_bytes,
                        linear_indices.data_ptr<index_t>(),
                        linear_indices_sorted.data_ptr<index_t>(),
                        infos.data_ptr<info_t>(),
                        infos_sorted.data_ptr<info_t>(),
                        linear_indices.numel(),
                        0,
                        total_hash_size_bits,
                        at::cuda::getCurrentCUDAStream(),
                        false));
                auto temp_storage = at::empty(
                    {static_cast<int64_t>(temp_storage_bytes)},
                    indices.options().dtype(at::kByte));
                AT_CUDA_CHECK(
                    FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceRadixSort::SortPairs(
                        temp_storage.data_ptr(),
                        temp_storage_bytes,
                        linear_indices.data_ptr<index_t>(),
                        linear_indices_sorted.data_ptr<index_t>(),
                        infos.data_ptr<info_t>(),
                        infos_sorted.data_ptr<info_t>(),
                        linear_indices.numel(),
                        0,
                        total_hash_size_bits,
                        at::cuda::getCurrentCUDAStream(),
                        false));
              }
            //   std::cout<< "linear_indices:" << linear_indices << std::endl;
            //   std::cout<< "linear_indices_sorted:" << linear_indices_sorted << std::endl;
            //   std::cout<< "infos:" << infos << std::endl;
            //   std::cout<< "infos_sorted:" << infos_sorted << std::endl;

              if (total_unique_indices != -1) {
                TORCH_CHECK(total_unique_indices >= 0);
                sorted_linear_indices_run =
                    at::empty({total_unique_indices}, indices.options());
                sorted_linear_indices_run_lengths = at::zeros(
                    {total_unique_indices}, indices.options().dtype(at::kInt));
              } else {
                sorted_linear_indices_run = at::empty_like(indices);
                sorted_linear_indices_run_lengths =
                    at::zeros_like(indices, indices.options().dtype(at::kInt));
              }
              sorted_linear_indices_num_runs =
                  at::zeros({1}, indices.options().dtype(at::kInt));

              {
                size_t temp_storage_bytes = 0;
                AT_CUDA_CHECK(
                    FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceRunLengthEncode::Encode(
                        nullptr,
                        temp_storage_bytes,
                        linear_indices_sorted.data_ptr<index_t>(),
                        sorted_linear_indices_run.data_ptr<index_t>(),
                        sorted_linear_indices_run_lengths.data_ptr<int32_t>(),
                        sorted_linear_indices_num_runs.data_ptr<int32_t>(),
                        linear_indices_sorted.numel(),
                        at::cuda::getCurrentCUDAStream()));
                // Allocate temporary storage
                auto temp_storage = at::empty(
                    {static_cast<int64_t>(temp_storage_bytes)},
                    indices.options().dtype(at::kByte));
                // Run encoding
                AT_CUDA_CHECK(
                    FBGEMM_GPU_CUB_NS_PREFIX hipcub::DeviceRunLengthEncode::Encode(
                        temp_storage.data_ptr(),
                        temp_storage_bytes,
                        linear_indices_sorted.data_ptr<index_t>(), // d_input: 1,1,1,2,2,3,3,3,3,4,5,5,5,6
                        sorted_linear_indices_run.data_ptr<index_t>(), // d_unique_out: 1,2,3,4,5,6
                        sorted_linear_indices_run_lengths.data_ptr<int32_t>(), // d_counts_out: 3,2,4,1,3,1
                        sorted_linear_indices_num_runs.data_ptr<int32_t>(), // d_num_runs_out: 6
                        linear_indices_sorted.numel(),
                        at::cuda::getCurrentCUDAStream()));
              }
            });
      });

  auto sorted_linear_indices_cumulative_run_lengths =
      asynchronous_complete_cumsum(sorted_linear_indices_run_lengths);

#undef INVOKE_LINEARIZE_INDEX_KERNEL

  return {
      linear_indices,
      linear_indices_sorted,
      infos_sorted,
      sorted_linear_indices_run,
      sorted_linear_indices_run_lengths,
      sorted_linear_indices_num_runs,
      sorted_linear_indices_cumulative_run_lengths};
}

////////////////////////////////////////////////////////////////////////////////
// FBGEMM Kernels
////////////////////////////////////////////////////////////////////////////////

template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    size_t kMaxVecsPerThread,
    int32_t kThreadGroupSize >
__global__ __launch_bounds__(kMaxThreads) void
split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1(
    const pta::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits> grad_output,
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    pta::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements, // if optimizer != "none"
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> hash_size_cumsum,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_linear_indices_run,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_cumulative_run_lengths,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_ids,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> num_long_run_ids,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_lxu_cache_locations,
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args, // if not dense and optimizer != "none"
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> long_run_id_to_really_long_run_ids,
    pta::PackedTensorAccessor32<at::acc_type<cache_t, true>, 2, at::RestrictPtrTraits> temp_grad_accum,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> grad_accum_counter,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms,
    float learning_rate) {
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
  const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
  const unsigned int shfl_sync_mask = 0xffffffffu;
#endif
  constexpr int VEC_WIDTH = 4;
  int32_t T = weights_offsets.size(0);
  const int32_t num_long_runs = num_long_run_ids[0];
  for (int32_t long_run_id = blockIdx.x; long_run_id < num_long_runs; long_run_id += gridDim.x) {
        // The first thread block in the really long run has run_id in long_run_ids
        // and the rest have the negative of its offset (see find_long_segments kernel).
        int32_t cta_rank_on_current_run = 0;
        int32_t current_run_id = long_run_ids[long_run_id];
        if (current_run_id < 0) {
            cta_rank_on_current_run = -long_run_ids[long_run_id];
            current_run_id = long_run_ids[long_run_id - cta_rank_on_current_run];
        }
        const int32_t run_length =
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1] -
            sorted_linear_indices_cumulative_run_lengths[current_run_id];
        // This computation must agree with how we compute num_ctas_for_run in
        // find_long_segments kernel!
        const int32_t num_ctas_on_current_run =
            use_deterministic_algorithms ? 1 : div_round_up(run_length, max_segment_length_per_cta);


        const int64_t linear_index = sorted_linear_indices_run[current_run_id];
        const int32_t segment_start =
            sorted_linear_indices_cumulative_run_lengths[current_run_id] +
            cta_rank_on_current_run * max_segment_length_per_cta;
        // const int32_t segment_end = std::min(
        const int32_t segment_end = min(
            use_deterministic_algorithms ? INT_MAX : segment_start + max_segment_length_per_cta,
            sorted_linear_indices_cumulative_run_lengths[current_run_id + 1]);
        const int32_t SL = segment_end - segment_start;
        const int32_t warp_id = threadIdx.y;
        const int32_t lane_id = threadIdx.x;

        // Note that with shared embedding tables we can have multiple tables
        // (i.e. different values of `t` sharing the same segment).
        //
        const auto info_0 = reinterpret_cast<const uint32_t*>(&sorted_infos[0])[segment_start];
        const auto t_0 = info_0 >> info_B_num_bits;

        int64_t hash_size = hash_size_cumsum[t_0];
        int32_t D = D_offsets[t_0 + 1] - D_offsets[t_0];
        int64_t idx = linear_index - hash_size;

        const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
        const int32_t sl_start = SL_per_warp * warp_id;
        const int32_t sl_end = min(SL_per_warp * (warp_id + 1), SL);
        Vec4T<at::acc_type<cache_t, true>> grad_sum[kMaxVecsPerThread];
        for (int32_t sl = sl_start; sl < sl_end; sl += kThreadGroupSize) {
            int32_t sl_j = sl + threadIdx.x;
            const auto b_t = sl_j < sl_end ? reinterpret_cast<const uint32_t*>(&sorted_infos[0])[segment_start + sl_j] : 0;
            const auto b = b_t & info_B_mask;
            const auto t = b_t >> info_B_num_bits; // if vbe
            int32_t D_start = sl_j < sl_end ? D_offsets[t] : 0; // if vbe // if not nobag
            for (int32_t j = 0; j < kThreadGroupSize && sl + j < sl_end; ++j) {
                int32_t b_j = SHFL_SYNC(b, j);
                int32_t D_start_j = SHFL_SYNC(D_start, j);

                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                    i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    Vec4T<at::acc_type<grad_t, true>> grad_out_vec(
                        &grad_output[b_j][0] + D_start_j + d
                    );
                    grad_sum[i].add_(grad_out_vec);
                }
            }
        }
        // do shared memory reduction only if we used multiple warps.
        if (SL > SL_per_warp) {
            struct SharedMemory<Vec4T<at::acc_type<cache_t, true>>> smem;
            Vec4T<at::acc_type<cache_t, true>>* shared_grad_sums = smem.getPointer();

            #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
            shared_grad_sums
                [lane_id + i * kThreadGroupSize +
                warp_id * kMaxVecsPerThread * kThreadGroupSize] = grad_sum[i];
            }
            __syncthreads();
            if (blockDim.y >= 32) {
            if (warp_id < 16) {
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 16) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 16) {
            if (warp_id < 8) {
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 8) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 8) {
            if (warp_id < 4) {
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 4) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (blockDim.y >= 4) {
            if (warp_id < 2) {
                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0; i < kMaxVecsPerThread &&
                    (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                    ++i) {
                shared_grad_sums
                    [lane_id + i * kThreadGroupSize +
                    warp_id * kMaxVecsPerThread * kThreadGroupSize] =
                        vec4_acc(
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                warp_id * kMaxVecsPerThread * kThreadGroupSize],
                            shared_grad_sums
                                [lane_id + i * kThreadGroupSize +
                                (warp_id + 2) * kMaxVecsPerThread * kThreadGroupSize]);
                }
            }
            __syncthreads();
            }
            if (warp_id == 0) {
            #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                grad_sum[i] = vec4_acc(
                    shared_grad_sums
                        [lane_id + i * kThreadGroupSize +
                        warp_id * kMaxVecsPerThread * kThreadGroupSize],
                    shared_grad_sums
                        [lane_id + i * kThreadGroupSize +
                        (warp_id + 1) * kMaxVecsPerThread * kThreadGroupSize]);
            }
            }
        }

        if (warp_id != 0) {
            continue;
        }

        if (num_ctas_on_current_run > 1) {
            int really_long_run_id = long_run_id_to_really_long_run_ids[long_run_id];
            Vec4T<at::acc_type<cache_t, true>> *temp_grad_accum_ptr =
                reinterpret_cast<Vec4T<at::acc_type<cache_t, true>>*>(&temp_grad_accum[really_long_run_id][0]);
            #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.x, grad_sum[i].acc.x);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.y, grad_sum[i].acc.y);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.z, grad_sum[i].acc.z);
                gpuAtomicAdd(&temp_grad_accum_ptr[lane_id + i * kThreadGroupSize].acc.w, grad_sum[i].acc.w);
            }
            int counter;
            if (threadIdx.x == 0) {
                __threadfence();
                counter = gpuAtomicAdd(&grad_accum_counter[really_long_run_id], -1);
            }
            counter = SHFL_SYNC(counter, 0);
            // Only the thread block accumulated the gradient last does the weight update.
            if (counter > 1) {
                continue;
            }
            CUDA_KERNEL_ASSERT(counter == 1 && "Invalid grad_accum_counter. Race condition?");
            #pragma unroll kMaxVecsPerThread
            for (int32_t i = 0;
                i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                ++i) {
                grad_sum[i] = temp_grad_accum_ptr[lane_id + i * kThreadGroupSize];
            }
        }

        // ========= Write grad test =========
        // const int64_t weights_offset = long_run_id * D;
        // idx = 0;
    	// #pragma unroll kMaxVecsPerThread
        // for (int32_t i = 0;
        //     i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
        //     ++i) {
        //     int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
        //     auto& grad = grad_sum[i];
        //     grad.store(&dev_weights[weights_offset + idx * D + d]);
        // } // if not dense and optimizer != "none"
        // ========= Write grad test =========

        split_sgd_table_update_kernel
          <emb_t, cache_t, kMaxVecsPerThread, kThreadGroupSize, VEC_WIDTH>(
              dev_weights,
              uvm_weights,
              lxu_cache_weights,
              weights_placements,
              weights_offsets,
              sorted_lxu_cache_locations,
              grad_sum,
              stochastic_rounding,
              stochastic_rounding_philox_args,
              current_run_id,
              D,
              t_0,
              idx,
              segment_start,
              shfl_sync_mask,
              0, // shared_weight_offset
              learning_rate);
    } // for each run
}


template <
    typename emb_t,
    typename grad_t,
    typename cache_t,
    size_t kMaxVecsPerThread,
    int32_t kThreadGroupSize >
__global__ __launch_bounds__(kBackwardMaxThreads) void
split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1(
    const pta::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits> grad_output,
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> dev_weights,
    pta::PackedTensorAccessor64<emb_t, 1, at::RestrictPtrTraits> uvm_weights,
    pta::PackedTensorAccessor64<cache_t, 2, at::RestrictPtrTraits> lxu_cache_weights,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> weights_placements,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> weights_offsets,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> hash_size_cumsum,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> sorted_linear_indices_run,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_cumulative_run_lengths,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_infos,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_lxu_cache_locations,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> sorted_linear_indices_num_runs,
    int32_t max_segment_length_per_warp,
    bool stochastic_rounding,
    at::PhiloxCudaState stochastic_rounding_philox_args, // if not dense and optimizer != "none"
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    float learning_rate) {
    int32_t T = D_offsets.size(0) - 1;
    const int32_t start_run_id = blockIdx.x * blockDim.y + threadIdx.y;

#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
    const unsigned int shfl_sync_mask =
        ((1L << kThreadGroupSize) - 1) <<
        (threadIdx.y % (kWarpSize / kThreadGroupSize) * kThreadGroupSize);
#else
    const unsigned int shfl_sync_mask = 0xffffffffu;
#endif
    constexpr int VEC_WIDTH = 4;

    for (uint32_t run_id = start_run_id;
         run_id < sorted_linear_indices_run.size(0) && run_id < sorted_linear_indices_num_runs[0];
             run_id += gridDim.x * blockDim.y) {

        const int64_t linear_index = sorted_linear_indices_run[run_id];
        const int32_t segment_start =
            sorted_linear_indices_cumulative_run_lengths[run_id];
        const int32_t segment_end =
            sorted_linear_indices_cumulative_run_lengths[run_id + 1];
        const int32_t SL = segment_end - segment_start;

        // if( start_run_id == 0 && threadIdx.x == 0){
        //     printf("First %d,%d,%d,%d\n", sorted_linear_indices_cumulative_run_lengths[0], sorted_linear_indices_cumulative_run_lengths[1],
        //     sorted_linear_indices_cumulative_run_lengths[2],sorted_linear_indices_cumulative_run_lengths[3]);
        //     printf("First %d,%d, SL:%d, kThreadGroupSize:%d\n", sorted_linear_indices_cumulative_run_lengths[run_id], sorted_linear_indices_cumulative_run_lengths[run_id+1], SL, kThreadGroupSize);
        // }

        if (SL >= max_segment_length_per_warp) {
            continue;
        }

        // now, each segment corresponds to exactly one table `t` and row in
        // that table (`idx`). Thus, we can hoist out some of the book-keeping.
        const auto info_0 = reinterpret_cast<const uint32_t*>(&sorted_infos[0])[segment_start];
        const auto t_0 = info_0 >> info_B_num_bits;

        int64_t hash_size = hash_size_cumsum[t_0];
        int32_t D = D_offsets[t_0 + 1] - D_offsets[t_0];
        int64_t idx = linear_index - hash_size;

        const int32_t SL_per_warp = div_round_up(SL, blockDim.y);
        const int32_t sl_start = 0;
        const int32_t sl_end = SL;
        Vec4T<at::acc_type<cache_t, true>> grad_sum[kMaxVecsPerThread];
        for (int32_t sl = sl_start; sl < sl_end; sl += kThreadGroupSize) {
            int32_t sl_j = sl + threadIdx.x;
            const auto b_t = sl_j < sl_end ? reinterpret_cast<const uint32_t*>(&sorted_infos[0])[segment_start + sl_j] : 0;
            const auto b = b_t & info_B_mask;
            const auto t = b_t >> info_B_num_bits; // if vbe

            // if( start_run_id == 0 && threadIdx.x == 0){
            //     printf("info_B_mask:%d, info_B_num_bits:%d, b_t:%d\n", info_B_mask, info_B_num_bits, b_t);
            // }
            int32_t D_start = sl_j < sl_end ? D_offsets[t] : 0; // if vbe // if not nobag

            for (int32_t j = 0; j < kThreadGroupSize && sl + j < sl_end; ++j) {
                int32_t b_j = SHFL_SYNC(b, j);
                int32_t D_start_j = SHFL_SYNC(D_start, j);

                #pragma unroll kMaxVecsPerThread
                for (int32_t i = 0;
                        i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
                        ++i) {
                    int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
                    Vec4T<at::acc_type<grad_t, true>> grad_out_vec(
                        &grad_output[b_j][0] + D_start_j + d
                    );
                    grad_sum[i].add_(grad_out_vec);

                }
            }
        }

        // ========= Write grad test =========
        // const int64_t weights_offset = run_id * D;
        // idx = 0;
    	// #pragma unroll kMaxVecsPerThread
        // for (int32_t i = 0;
        //     i < kMaxVecsPerThread && (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH < D;
        //     ++i) {
        //     int32_t d = (i * kThreadGroupSize + threadIdx.x) * VEC_WIDTH;
        //     auto& grad = grad_sum[i];
        //     grad.store(&dev_weights[weights_offset + idx * D + d]);
        // } // if not dense and optimizer != "none"
        // ========= Write grad test =========


        split_sgd_table_update_kernel
          <emb_t, cache_t, kMaxVecsPerThread, kThreadGroupSize, VEC_WIDTH>(
              dev_weights,
              uvm_weights,
              lxu_cache_weights,
              weights_placements,
              weights_offsets,
              sorted_lxu_cache_locations,
              grad_sum,
              stochastic_rounding,
              stochastic_rounding_philox_args,
              run_id,
              D,
              t_0,
              idx,
              segment_start,
              shfl_sync_mask,
              threadIdx.y * kMaxVecsPerThread * kThreadGroupSize, // shared_weight_offset
              learning_rate); // if not dense and optimizer != "none"

    }
}


__global__ __launch_bounds__(kMaxThreads) void
split_embedding_backward_codegen_find_long_segments(
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_num_runs,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        sorted_linear_indices_run_lengths,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        long_run_ids,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        num_long_run_ids,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        long_run_id_to_really_long_run_ids,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        num_really_long_run_ids,
    pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits>
        grad_accum_counter,
    const int32_t max_segment_length_per_warp,
    const int32_t max_segment_length_per_cta,
    const bool use_deterministic_algorithms) {
  const int32_t num_runs = sorted_linear_indices_num_runs[0];
  for (auto run_id = blockIdx.x * blockDim.x + threadIdx.x; run_id < num_runs; run_id += blockDim.x * gridDim.x) {
    if (sorted_linear_indices_run_lengths[run_id] >= max_segment_length_per_warp) {
        // A segment with length > max_segment_length_per_cta is handled by more than 1 thread block.
        const int num_ctas_for_run =
            use_deterministic_algorithms ? 1 : div_round_up(sorted_linear_indices_run_lengths[run_id], max_segment_length_per_cta);
        const auto long_run_idx = gpuAtomicAdd(&num_long_run_ids[0], num_ctas_for_run);
        // The first thread block in the really long run gets run_id in long_run_ids
        // and the rest get the negative of its offset.
        long_run_ids[long_run_idx] = run_id;
        for (int i = 1; i < num_ctas_for_run; ++i) {
            long_run_ids[long_run_idx + i] = -i;
        }
        if (num_ctas_for_run > 1) {
            const auto really_long_run_idx = gpuAtomicAdd(&num_really_long_run_ids[0], 1);
            grad_accum_counter[really_long_run_idx] = num_ctas_for_run;
            for (int i = 0; i < num_ctas_for_run; ++i) {
                long_run_id_to_really_long_run_ids[long_run_idx + i] = really_long_run_idx;
            }
        }
    }
  }
}


template <typename grad_t>
__global__ __launch_bounds__(kMaxThreads) void grad_mean_kernel(
    pta::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits>
        grad_output_mean,
    const pta::PackedTensorAccessor64<grad_t, 2, at::RestrictPtrTraits>
        grad_output,
    const pta::PackedTensorAccessor32<int32_t, 1, at::RestrictPtrTraits> D_offsets,
    const pta::PackedTensorAccessor32<int64_t, 1, at::RestrictPtrTraits> offsets,

    FixedDivisor fd_B

) {
  int32_t T = D_offsets.size(0) - 1;
  int32_t b_t = blockIdx.x * blockDim.y + threadIdx.y;
  int32_t b;
  int32_t t;
  const auto total_B = offsets.size(0) - 1;

  if (b_t >= total_B) {
    return;
  }

  fd_B.DivMod(b_t, &t, &b);


  int32_t D_start = D_offsets[t];
  int32_t D_end = D_offsets[t + 1];
  int32_t D = D_end - D_start;
  int64_t indices_start = offsets[b_t];
  int64_t indices_end = offsets[b_t + 1];
  int32_t L = indices_end - indices_start;


  const auto grad_offset = D_start;
  const auto grad_outer_offset = b;


  const grad_t* shifted_grad_output = &grad_output[grad_outer_offset][grad_offset];
  grad_t* shifted_grad_output_mean = &grad_output_mean[grad_outer_offset][grad_offset];

  if (L != 0) {
    for (int32_t d = threadIdx.x; d * 4 < D; d += blockDim.x) {
      Vec4T<grad_t> grad_out_vec(&shifted_grad_output[d * 4]);
      grad_out_vec.mul_(1.0 / L);
      grad_out_vec.store(&shifted_grad_output_mean[d * 4]);
    }
  } else {
    for (int32_t d = threadIdx.x; d * 4 < D; d += blockDim.x) {
      Vec4T<grad_t> grad_out_vec(&shifted_grad_output[d * 4]);
      grad_out_vec.store(&shifted_grad_output_mean[d * 4]);
    }
  }
}



////////////////////////////////////////////////////////////////////////////////
// Operator Code
////////////////////////////////////////////////////////////////////////////////

Tensor split_embedding_backward_codegen_sgd_unweighted_exact_cuda(
    Tensor grad_output,
    Tensor dev_weights,
    Tensor uvm_weights,
    Tensor lxu_cache_weights,
    Tensor weights_placements,
    Tensor weights_offsets,
    Tensor D_offsets,
    int64_t max_D,
    Tensor hash_size_cumsum,
    int64_t total_hash_size_bits,
    Tensor indices,
    Tensor offsets,
    int64_t pooling_mode,
    Tensor lxu_cache_locations,
    int64_t unused_,
    int64_t max_segment_length_per_warp,
    bool stochastic_rounding,
    const int32_t info_B_num_bits,
    const uint32_t info_B_mask,
    double learning_rate
    ) {

   TENSORS_ON_SAME_CUDA_GPU_IF_NOT_OPTIONAL(
        dev_weights,
        uvm_weights,
        lxu_cache_weights,
        weights_placements,
        weights_offsets,
        D_offsets,
        hash_size_cumsum,
        indices,
        offsets,
        lxu_cache_locations,
        grad_output);

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(dev_weights.get_device());
    TORCH_CHECK(max_D <= 1024);
    // Set total_unique_indices to total num indices by default
    const auto total_unique_indices = indices.numel();

    // short-circuit if there are zero indices.
    if (indices.numel() == 0) {
        return Tensor();
    }
    int32_t T = D_offsets.numel() - 1;

    TORCH_CHECK(T > 0);
    // offsets = [B x T  + 1]
    const auto total_B = offsets.size(0) - 1;
    TORCH_CHECK(total_B > 0);
    auto BT_block_size = kMaxThreads / kWarpSize;
    TORCH_CHECK(BT_block_size * kWarpSize <= kMaxThreads);

    // V100: 96 KB; A100: 160 KB; H100: 228 KB.
    int max_shared_bytes = 0;
#ifndef __HIP_PLATFORM_HCC__
    hipDeviceGetAttribute(&max_shared_bytes, hipDeviceAttributeSharedMemPerBlockOptin, dev_weights.get_device());
#else
    // MI100 has 64 KB local memory (shared memory) per workgroup
    max_shared_bytes = 64 << 10;
#endif
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    int shared_kb = max_shared_bytes >> 10;
    // V100: 64 KB; A100: 96 KB; H100: 144 KB
#ifndef __HIP_PLATFORM_HCC__
    // Use 2/3 of the available GPU shared mem; leave rooms for L1$.
    int used_shared_kb = round_down(shared_kb * 2 / 3, 16);
    TORCH_CHECK(used_shared_kb > 0);
#else
    // MI100 has independent shared mem and L1
    int used_shared_kb = shared_kb;
#endif
    int used_shared_bytes = used_shared_kb << 10;

    Tensor linear_indices, linear_indices_sorted;
    Tensor infos_sorted;
    Tensor sorted_linear_indices_run, sorted_linear_indices_run_lengths,
        sorted_linear_indices_num_runs,
        sorted_linear_indices_cumulative_run_lengths;
    std::tie(
        linear_indices,
        linear_indices_sorted,
        infos_sorted,
        sorted_linear_indices_run,
        sorted_linear_indices_run_lengths,
        sorted_linear_indices_num_runs,
        sorted_linear_indices_cumulative_run_lengths) =
        transpose_embedding_input_local(
        // transpose_embedding_input(
            hash_size_cumsum,
            total_hash_size_bits,
            indices,
            offsets,
            false,
            c10::optional<Tensor>(),
            info_B_num_bits,
            info_B_mask,
            total_unique_indices);
    auto lxu_cache_locations_sorted = at::empty_like(lxu_cache_locations);
    if (lxu_cache_locations.size(0) > 0) {
        size_t temp_storage_bytes = 0;
        AT_CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(
            // radix_sort_pairs(
            nullptr,
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            lxu_cache_locations.data_ptr<int32_t>(),
            lxu_cache_locations_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
        auto temp_storage = at::empty(
            {static_cast<int64_t>(temp_storage_bytes)},
            indices.options().dtype(at::kByte));
        AT_CUDA_CHECK( hipcub::DeviceRadixSort::SortPairs(
            // radix_sort_pairs(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            linear_indices.data_ptr<int64_t>(),
            linear_indices_sorted.data_ptr<int64_t>(),
            lxu_cache_locations.data_ptr<int32_t>(),
            lxu_cache_locations_sorted.data_ptr<int32_t>(),
            linear_indices.numel(),
            0,
            total_hash_size_bits,
            at::cuda::getCurrentCUDAStream(),
            false));
    }

    DISPATCH_EMB_GRAD_CACHE_TYPES(
        dev_weights.scalar_type(),
        grad_output.scalar_type(),
        lxu_cache_weights.scalar_type(),
            "split_embedding_backward_sgd_exact_kernel",
        [&] {

            // early memory release
            linear_indices.reset();
            linear_indices_sorted.reset();

            auto grad_output_accessor = MAKE_PTA_WITH_NAME("split_embedding_backward_codegen_sgd_unweighted_exact_cuda.1", grad_output, grad_t, 2, 64);
            Tensor grad_output_mean;
            if (static_cast<PoolingMode>(pooling_mode) == PoolingMode::MEAN) {
              grad_output_mean = at::empty_like(grad_output);

#ifdef FBGEMM_GPU_MEMCHECK
              const auto func_name1 = "grad_mean_kernel";
#endif

              grad_mean_kernel<<<
                    div_round_up(total_B, kMaxThreads / kWarpSize),
                    dim3(kWarpSize, kMaxThreads / kWarpSize),
                    0,
                    at::cuda::getCurrentCUDAStream()>>>
                    (
                        MAKE_PTA_WITH_NAME(func_name1, grad_output_mean, grad_t, 2, 64),
                        MAKE_PTA_WITH_NAME(func_name1, grad_output, grad_t, 2, 64),
                        MAKE_PTA_WITH_NAME(func_name1, D_offsets, int32_t, 1, 32),
                        MAKE_PTA_WITH_NAME(func_name1, offsets, int64_t, 1, 32),
                        FixedDivisor(total_B / T)
                    );
              C10_CUDA_KERNEL_LAUNCH_CHECK(); // if not dense or not vbe


              grad_output_accessor = MAKE_PTA_WITH_NAME("split_embedding_backward_codegen_sgd_unweighted_exact_cuda.2", grad_output_mean, grad_t, 2, 64);
            }
            at::PhiloxCudaState rng_engine_inputs;
            if (stochastic_rounding && !std::is_same<emb_t, float>::value) {
                auto gen = at::cuda::detail::getDefaultCUDAGenerator();
                std::lock_guard<std::mutex> lock(gen.mutex());
                rng_engine_inputs =
                    at::check_generator<at::CUDAGeneratorImpl>(gen)
                        ->philox_cuda_state(4);
            }
            // kMaxElemPerThread is # of elements handled by thread if we use a full warp for a row
            // We consider kMaxElemPerThread 1 and 2, and then a multiple of 4.
            if (max_D <= 32) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 1 / 4 >= 1 ? 1 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 1, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 64) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 2 / 4 >= 1 ? 2 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 2, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 128) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 4 / 4 >= 1 ? 4 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 4, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 256) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 8 / 4 >= 1 ? 8 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 8, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 384) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 12 / 4 >= 1 ? 12 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 12, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 512) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 16 / 4 >= 1 ? 16 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 16, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 640) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 20 / 4 >= 1 ? 20 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 20, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 768) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 24 / 4 >= 1 ? 24 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 24, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 896) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 28 / 4 >= 1 ? 28 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 28, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
            if (max_D <= 1024) {
            // hipcc can't use max in constexpr
            constexpr int kMaxVecsPerThread = 32 / 4 >= 1 ? 32 / 4 : 1;
            // If max_D is small, use fewer number of threads than kWarpSize.
#ifdef FBGEMM_USE_SUBWARP_SHUFFLE
            constexpr int kThreadGroupSize = kWarpSize / std::max(4 / 32, 1);
#else
            constexpr int kThreadGroupSize = kWarpSize;
#endif
            // Stay under used_shared_kb of shared memory (V100: 64 KB; A100: 96 KB; H100: 144 KB), BT_block_size must be a power of two.
            while (BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread >= used_shared_bytes) {
                BT_block_size /= 2;
            }
            TORCH_CHECK(BT_block_size >= 1);
            if (std::is_same<emb_t, double>::value) {
                // Otherwise we see CUDA kernel launch failures despite the above checks.
                BT_block_size = 1;
            }

            auto long_run_ids = at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            auto num_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));

            const bool use_deterministic_algorithms = at::globalContext().deterministicAlgorithms();
            const int max_segment_length_per_cta = use_deterministic_algorithms ? INT_MAX : 1024;
            Tensor long_run_id_to_really_long_run_ids;
            if (use_deterministic_algorithms) {
                long_run_id_to_really_long_run_ids =
                    at::empty(0, sorted_linear_indices_run_lengths.options());
            } else {
                long_run_id_to_really_long_run_ids =
                    at::empty({indices.numel()}, sorted_linear_indices_run_lengths.options());
            }
            auto num_really_long_run_ids = at::zeros({1}, indices.options().dtype(at::kInt));
            auto grad_accum_counter = at::empty(
                use_deterministic_algorithms ? 0 : (indices.numel() / max_segment_length_per_cta),
                indices.options().dtype(at::kInt));

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name2 = "split_embedding_backward_codegen_find_long_segments";
#endif

            split_embedding_backward_codegen_find_long_segments<<<
                div_round_up(total_unique_indices, kMaxThreads),
                kMaxThreads,
                0,
                at::cuda::getCurrentCUDAStream()
            >>>(
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_num_runs, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, sorted_linear_indices_run_lengths, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, num_really_long_run_ids, int32_t, 1, 32),
                MAKE_PTA_WITH_NAME(func_name2, grad_accum_counter, int32_t, 1, 32),
                max_segment_length_per_warp,
                max_segment_length_per_cta,
                use_deterministic_algorithms);
            C10_CUDA_KERNEL_LAUNCH_CHECK();

            // A temp buffer to accumulate gradients with atomics.
            auto temp_grad_accum = at::zeros(
                {use_deterministic_algorithms ? 0 : grad_accum_counter.numel(), max_D},
                grad_output.options().dtype(std::is_same<cache_t, double>::value ? at::kDouble : at::kFloat));

            int32_t grid_size = std::min(
                div_round_up(total_unique_indices, kMaxThreads),
                get_max_thread_blocks_());

            // Check https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory-7-x
            // "Compute capability 7.x devices allow a single thread block to
            // address the full capacity of shared memory: 96 KB on Volta,
            // 64 KB on Turing. Kernels relying on shared memory allocations
            // over 48 KB per block are architecture-specific, as such they
            // must use dynamic shared memory (rather than statically sized
            // arrays) and require an explicit opt-in using hipFuncSetAttribute()".

#ifndef __HIP_PLATFORM_HCC__
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t),
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            C10_CUDA_KERNEL_LAUNCH_CHECK();

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name3 = "split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1";
#endif

            // dividing by kMaxThreads is a heuristic to avoid num of blocks far exceeding num_long_run_ids[0]
            split_embedding_backward_codegen_sgd_unweighted_kernel_cta_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, BT_block_size),
                    BT_block_size * sizeof(at::acc_type<cache_t, true>) * 4 * kWarpSize *
                        kMaxVecsPerThread,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name3, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name3, weights_placements, int32_t, 1, 32), // if optimizer != "none"
                    MAKE_PTA_WITH_NAME(func_name3, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, num_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name3, lxu_cache_locations_sorted, int32_t, 1, 32),
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    MAKE_PTA_WITH_NAME(func_name3, long_run_id_to_really_long_run_ids, int32_t, 1, 32),
                    MAKE_PTA_ACC_WITH_NAME(func_name3, temp_grad_accum, cache_t, 2, 32),
                    MAKE_PTA_WITH_NAME(func_name3, grad_accum_counter, int32_t, 1, 32),
                    max_segment_length_per_cta,
                    use_deterministic_algorithms,
                    learning_rate);

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            grid_size = std::min(
                div_round_up(total_unique_indices, kBackwardMaxThreads / kThreadGroupSize),
                get_max_thread_blocks_());

            // Shared memory is not needed for non uint8_t weights
            size_t shmem_bytes = 0;
            if (std::is_same<emb_t, uint8_t>::value) {
                shmem_bytes = BT_block_size * sizeof(
                    at::acc_type<cache_t, true>) * 4 * kWarpSize * kMaxVecsPerThread;
#ifndef __HIP_PLATFORM_HCC__
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                    emb_t),
                    grad_t,
                    cache_t,
                    kMaxVecsPerThread,
                    kThreadGroupSize>,
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    used_shared_bytes); // V100: 64 KB; A100: 96 KB; H100: 144 KB
#endif
            }

#ifdef FBGEMM_GPU_MEMCHECK
            const auto func_name4 = "split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1";
#endif

            C10_CUDA_KERNEL_LAUNCH_CHECK();
            split_embedding_backward_codegen_sgd_unweighted_kernel_warp_per_row_1<
                emb_t,
                grad_t,
                cache_t,
                kMaxVecsPerThread,
                kThreadGroupSize>
                <<<grid_size,
                    dim3(kThreadGroupSize, kBackwardMaxThreads / kThreadGroupSize),
                    shmem_bytes,
                    at::cuda::getCurrentCUDAStream()>>>(
                    grad_output_accessor,
                    MAKE_PTA_WITH_NAME(func_name4, dev_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, uvm_weights, emb_t, 1, 64),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_weights, cache_t, 2, 64),
                    MAKE_PTA_WITH_NAME(func_name4, weights_placements, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, weights_offsets, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, D_offsets, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, hash_size_cumsum, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_run, int64_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_cumulative_run_lengths, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, infos_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, lxu_cache_locations_sorted, int32_t, 1, 32),
                    MAKE_PTA_WITH_NAME(func_name4, sorted_linear_indices_num_runs, int32_t, 1, 32),
                    max_segment_length_per_warp,
                    stochastic_rounding,
                    rng_engine_inputs, // if not dense and optimizer != "none"
                    info_B_num_bits,
                    info_B_mask,
                    learning_rate);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
            return;
        }
        });
    return Tensor();
}
