#include <iostream>
#include <ctime>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "tbe_data_loader.cuh"
#include "fbgemm_put_tbe.cuh"

#include <mpi.h>
#include <torch/torch.h>

#include <chrono>
#include <thread>

#include "nccl.h"
#include "util.cuh"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h> // for at::cuda::current_device
#include <c10/cuda/CUDAStream.h> // for at::cuda::CUDAStream::getCurrentCUDAStream



int main(int argc, char* argv[]) {
    int32_t if_save_result = atoi(argv[1]); // if exp_type == 0: fbgemm kernel; exp_type == 1: nvshmem kernel, exp_type == 2: profiling
    int32_t if_all_to_all = atoi(argv[2]); // if exp_type == 0: fbgemm kernel; exp_type == 1: nvshmem kernel, exp_type == 2: profiling
    int32_t n_loop = 2048;
    if(argc > 3){
        n_loop = atoi(argv[3]);
    }

    // MPI and NVSHMEM and NCCL
    // init.===============================================================================================================================
    // MPI
    int rank, nranks;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    MPI_Comm mpi_comm = MPI_COMM_WORLD;

    int deviceId, dev_count;
    CUDA_CHECK(hipGetDevice(&deviceId));
    CUDA_CHECK(hipGetDeviceCount(&dev_count));
    CUDA_CHECK(hipSetDevice(rank % dev_count));

    // NCCL
    ncclUniqueId id;
    ncclComm_t comm;
    if (rank == 0) ncclGetUniqueId(&id);
    MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
    ncclCommInitRank(&comm, nranks, id, rank);
    hipStream_t stream;
    hipStreamCreate(&stream);

    int gpu_per_node = nranks;
    int local_rank = rank % gpu_per_node;
    printf(
        "rank:%d, nranks:%d, cuda_device_id:%d, device_cnt:%d, local_rank:%d\n",
        rank,
        nranks,
        deviceId,
        dev_count,
        local_rank);

    // Loading sharding parameters ================================================================================================
    std::string home_directory = std::getenv("HOME");
    std::string sharding_param_dir = home_directory + "/tmp/test_1";
    std::string data_dir = home_directory + "/tmp/test_1/data";
    std::string result_dir = home_directory + "/tmp/test_1/result";
    TBE_Dataloader* dataloader = new TBE_Dataloader(sharding_param_dir, data_dir, rank);

    auto sharding_param = dataloader->get_sharing_param();
    std::cout << sharding_param <<"\n";

    at::Device device(at::kCUDA);
    auto float_tensor_options = at::TensorOptions().device(device).dtype(at::kFloat);
    auto int32_tensor_options = at::TensorOptions().device(device).dtype(at::kInt);
    auto int64_tensor_options = at::TensorOptions().device(device).dtype(at::kLong);

    torch::manual_seed(rank);

    // Load and init sharding params and embedding table parameters.
    std::string file_name = sharding_param_dir + "/weight_" + std::to_string(rank) + ".bin";
    at::Tensor dev_weights = load_float_tensor(file_name, sharding_param["dev_weights"][0]).to(device);
    // std::cout << "dev_weights:" << dev_weights.sizes() << std::endl;
    // at::Tensor dev_weight_cpu = load_float_tensor(file_name, sharding_param["dev_weights"][0]);
    // at::Tensor dev_weights = at::randn(dev_weight_cpu.sizes(), float_tensor_options);

    at::Tensor uvm_weights = at::empty({sharding_param["uvm_weights"][0]}, float_tensor_options);
    // std::cout << "uvm_weights:" << uvm_weights.sizes() << std::endl;

    at::Tensor lxu_cache_weights = at::empty({sharding_param["lxu_cache_weights"][0], sharding_param["lxu_cache_weights"][1]}, float_tensor_options);
    // std::cout << "lxu_cache_weights:" << lxu_cache_weights.sizes() << std::endl;

    std::vector<int> weights_placements_data = sharding_param["weights_placements"];
    at::Tensor weights_placements = at::tensor(weights_placements_data, int32_tensor_options);
    // std::cout << "weights_placements:" << weights_placements.sizes() << std::endl;

    std::vector<int64_t> weights_offsets_data = sharding_param["weights_offsets"];
    at::Tensor weights_offsets = at::tensor(weights_offsets_data, int64_tensor_options);
    // std::cout << "weights_offsets:" << weights_offsets.sizes() << std::endl;

    std::vector<int> D_offsets_data = sharding_param["D_offsets"];
    at::Tensor D_offsets = at::tensor(D_offsets_data, int32_tensor_options);
    // std::cout << "D_offsets:" << D_offsets.sizes() << std::endl;

    at::Tensor lxu_cache_locations = at::empty({0}, int32_tensor_options);
    // std::cout << "lxu_cache_locations:" << lxu_cache_locations.sizes() << std::endl;

    std::vector<int64_t> hash_size_cumsum_data = sharding_param["hash_size_cumsum"];
    at::Tensor hash_size_cumsum = at::tensor(hash_size_cumsum_data, int64_tensor_options);
    // std::cout << "hash_size_cumsum:" << hash_size_cumsum.sizes() << std::endl;

    // All-to-All information:
    std::vector<int> dim_sum_per_rank_data = sharding_param["dim_sum_per_rank"];
    std::vector<int> dim_offset_per_rank_data = sharding_param["dim_offset_per_rank"];
    at::Tensor dim_sum_per_rank = at::tensor(dim_sum_per_rank_data, int32_tensor_options);
    at::Tensor dim_offset_per_rank = at::tensor(dim_offset_per_rank_data, int32_tensor_options);
    int32_t total_dim_output = sharding_param["total_dim_output"];

    int64_t total_D = sharding_param["total_D"];
    int64_t max_D = sharding_param["max_D"];
    int64_t pooling_mode = sharding_param["pooling_mode"];
    int64_t output_dtype = sharding_param["output_dtype"];
    bool is_experimental = sharding_param["is_experimental"];
    int32_t n_local_Table = D_offsets.numel() - 1; // n_local_Table

    // backward params
    int64_t unused_ = 0;
    int32_t max_segment_length_per_warp = 32;
    int64_t total_hash_size_bits = sharding_param["total_hash_size_bits"];
    bool stochastic_rounding = sharding_param["stochastic_rounding"];
    double learning_rate = sharding_param["learning_rate"];
    int32_t T = D_offsets.numel() - 1; // the number of local table;
    int32_t total_B;
    int32_t info_B_num_bits;
    uint32_t info_B_mask;

    std::cout<< "total_D:" << total_D << ", max_D:" << max_D << ", pooling_mode:" <<
    pooling_mode << ", output_dtype:" << output_dtype << ", is_experimental:" << is_experimental << "\n";

    // Init nvshmem buffer =====================================================================================================
    at::Tensor tmp_idx, tmp_offset;
    std::tie(tmp_idx, tmp_offset) = dataloader->next_input_tensor();
    dataloader->reset_iter();
    total_B = tmp_offset.size(0) - 1;
    int32_t local_batch_size = total_B / n_local_Table / nranks;
    float *output_buffer, *h_output_buffer;
    h_output_buffer = (float *) malloc (local_batch_size * total_dim_output * sizeof(float)); // malloc host EMT

    std::vector<at::Tensor> all_to_all_buffer = init_NCCL_AlltoAll_buffer(nranks, rank, dim_sum_per_rank_data, local_batch_size);

    MPI_Barrier(MPI_COMM_WORLD);

    std::cout << "rank:" << rank << ", finished loading sharding params and init grad buffer\n";

    // Compute and save result ================================================================================================
    // at::Tensor indices, offsets;
    // std::tie(indices, offsets) = dataloader->next_input_tensor();

    // at::Tensor embedding = split_embedding_codegen_forward_unweighted_cuda_local(
    //     dev_weights,
    //     uvm_weights,
    //     lxu_cache_weights,
    //     weights_placements,
    //     weights_offsets,
    //     D_offsets,
    //     total_D,
    //     max_D,
    //     indices,
    //     offsets,
    //     pooling_mode,
    //     lxu_cache_locations,
    //     output_dtype,
    //     is_experimental
    // );

    // hipDeviceSynchronize();
    // at::Tensor fwd_a2a_result = NCCL_AlltoAll_forward(embedding, all_to_all_buffer, dim_sum_per_rank_data, comm, local_batch_size, nranks, rank);

    // hipMemcpy(h_output_buffer, fwd_a2a_result.data_ptr<float>(), local_batch_size * total_dim_output * sizeof(float), hipMemcpyDeviceToHost);
    // std::string all_to_all_file = result_dir + "/fbgemm_all_to_all_result_" + std::to_string(rank) + "_" + std::to_string(0) + ".bin";
    // save_float(h_output_buffer, all_to_all_file, local_batch_size * total_dim_output);

    at::Tensor indices, offsets;
    if(if_save_result){
        std::tie(indices, offsets) = dataloader->next_input_tensor();

        at::Tensor embedding = split_embedding_codegen_forward_unweighted_cuda_local(
            dev_weights,
            uvm_weights,
            lxu_cache_weights,
            weights_placements,
            weights_offsets,
            D_offsets,
            total_D,
            max_D,
            indices,
            offsets,
            pooling_mode,
            lxu_cache_locations,
            output_dtype,
            is_experimental
        );

        hipDeviceSynchronize();
        at::Tensor fwd_a2a_result = NCCL_AlltoAll_forward(embedding, all_to_all_buffer, dim_sum_per_rank_data, comm, local_batch_size, nranks, rank);

        // save result
        hipMemcpy(h_output_buffer, fwd_a2a_result.data_ptr<float>(), local_batch_size * total_dim_output * sizeof(float), hipMemcpyDeviceToHost);
        std::string all_to_all_file = result_dir + "/fbgemm_all_to_all_result_" + std::to_string(rank) + "_" + std::to_string(0) + ".bin";
        save_float(h_output_buffer, all_to_all_file, local_batch_size * total_dim_output);

        std::cout << "Saved FBGEMM Result\n";
    }

    // warm up ================================================================================================
    for(int i=0; i<256; i++){
        std::tie(indices, offsets) = dataloader->next_input_tensor();

        at::Tensor embedding = split_embedding_codegen_forward_unweighted_cuda_local(
            dev_weights,
            uvm_weights,
            lxu_cache_weights,
            weights_placements,
            weights_offsets,
            D_offsets,
            total_D,
            max_D,
            indices,
            offsets,
            pooling_mode,
            lxu_cache_locations,
            output_dtype,
            is_experimental
        );

        if(if_all_to_all){
            hipDeviceSynchronize();
            at::Tensor fwd_a2a_result = NCCL_AlltoAll_forward(embedding, all_to_all_buffer, dim_sum_per_rank_data, comm, local_batch_size, nranks, rank);
        }
        if(rank==0 && (i+1)%16==0)
            printf("finished iter warmup:%d\n", i);
    }
    hipDeviceSynchronize();

    // Profiling the throughput ================================================================================================
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    for(int i=0;i<n_loop;i++){
        std::tie(indices, offsets) = dataloader->next_input_tensor();

        at::Tensor embedding = split_embedding_codegen_forward_unweighted_cuda_local(
            dev_weights,
            uvm_weights,
            lxu_cache_weights,
            weights_placements,
            weights_offsets,
            D_offsets,
            total_D,
            max_D,
            indices,
            offsets,
            pooling_mode,
            lxu_cache_locations,
            output_dtype,
            is_experimental
        );

        if(if_all_to_all){
            hipDeviceSynchronize();
            at::Tensor fwd_a2a_result = NCCL_AlltoAll_forward(embedding, all_to_all_buffer, dim_sum_per_rank_data, comm, local_batch_size, nranks, rank);
        }
        if(rank==0 && (i+1)%1024==0)
            printf("finished iter:%d\n", i);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Rank-%d, Total (ms): %.3f, avg_kernel latency:%.3f ms/iter\n", rank, milliseconds, milliseconds/float(n_loop));

    MPI_Barrier(MPI_COMM_WORLD);

    // Release Memeory ========================================================================================================
    std::cout << "rank:" << rank << ", finished save result\n";
    free(h_output_buffer);
    MPI_Finalize();
    return 0;
}
