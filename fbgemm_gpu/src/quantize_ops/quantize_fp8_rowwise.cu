#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "common.cuh"

using Tensor = at::Tensor;

/// @defgroup quantize-data-cuda Quantization Data CUDA Operators
/// The following are CUDA Operators

namespace fbgemm_gpu {

namespace {

// FP32/FP16 -> FP8 rowwise kernel
template <typename input_t>
__global__ inline void _float_to_FP8rowwise_cuda_kernel(
    const input_t* __restrict__ input,
    const int64_t nrows,
    const int64_t ncols,
    std::uint8_t* __restrict__ output,
    const bool forward) {
  constexpr float kEpsilon = 1e-20f;
  const int ebit = forward ? 4 : 5;
  const int bias = forward ? 15 : 31;
  const float max_pos = forward ? 0.9375 : 0.875;

  const int64_t ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int64_t output_columns = ncols_aligned + 2 * sizeof(float);

  const int64_t row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < nrows) {
    const input_t* input_row = input + row * ncols;
    std::uint8_t* output_row = output + row * output_columns;
    float* output_row_scale_bias =
        reinterpret_cast<float*>(output_row + ncols_aligned);

    const float minimum_element = fbgemm_gpu::min(input_row, input_row + ncols);
    const float maximum_element = fbgemm_gpu::max(input_row, input_row + ncols);

    const auto scale =
        max_pos / (kEpsilon + fmaxf(maximum_element, -minimum_element));
    output_row_scale_bias[0] = scale;
    for (int64_t col = 0; col < ncols; ++col) {
      if constexpr (std::is_same<input_t, at::BFloat16>::value) {
        output_row[col] = float_to_hfp8(
            __bfloat162float(input_row[col]) * scale, ebit, bias, max_pos);
      } else if constexpr (std::is_same<input_t, at::Half>::value) {
        output_row[col] = float_to_hfp8(
            __half2float(input_row[col]) * scale, ebit, bias, max_pos);
      } else {
        output_row[col] =
            float_to_hfp8(input_row[col] * scale, ebit, bias, max_pos);
      }
    }
  }
}

template <typename input_t>
__global__ inline void _get_FP8_qparam_cuda_kernel(
    const input_t* __restrict__ input,
    const int64_t nrows,
    const int64_t ncols,
    uint8_t* __restrict__ output,
    float* __restrict__ range_list,
    const bool forward) {
  const int64_t row = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int64_t output_columns = ncols_aligned + 2 * sizeof(float);
  float max_pos;
  if (forward) {
    max_pos = 0.9375;
  } else {
    max_pos = 0.875;
  }
  // starting values for future reductions
  constexpr float kEpsilon = 1e-20f;
  float maximum_element = kEpsilon;
  // always a power of 2 up to size 32. Multiple rows can share the same warp
  // when smaller than 32.
  const int lane_width = blockDim.x;

  // March warp-wise through the row, doing thread local min and max reductions.
  // This loop will only execute once when ncol <= 32
  if (row < nrows) {
    const input_t* const input_row = input + row * ncols;

    for (int64_t col = threadIdx.x; col < ncols; col += lane_width) {
      // Get thread-local minmax. These are the smallest min and max ever seen
      // by this thread.
      if constexpr (std::is_same<input_t, at::BFloat16>::value) {
        maximum_element =
            fmaxf(maximum_element, fabs(__bfloat162float(input_row[col])));
      } else if constexpr (std::is_same<input_t, at::Half>::value) {
        maximum_element =
            fmaxf(maximum_element, fabs(__half2float(input_row[col])));
      } else {
        maximum_element = fmaxf(maximum_element, fabs(input_row[col]));
      }
    }
  }

  // Perform warp-wide min and max reductions. All threads in the warp
  // participate, even if they aren't assigned to a row, since we can't assume
  // the existence of the `*_sync` warp primitives with support for masking.
  for (int offset = lane_width >> 1; offset > 0; offset >>= 1) {
    maximum_element =
        fmaxf(maximum_element, shfl_xor(maximum_element, offset, lane_width));
  }

  // only the leading thread in the warp is needed to return the final result in
  // output. Additionally, threads mapped to non-existent rows do not write to
  // the output array.
  if (threadIdx.x != 0 || row >= nrows) {
    return;
  }
  float* const output_row_qparams =
      reinterpret_cast<float*>(output + row * output_columns + ncols_aligned);

  output_row_qparams[0] = max_pos / (kEpsilon + maximum_element);
}

template <typename input_t>
__global__ inline void _compute_FP8_quantize_cuda_kernel(
    const input_t* const __restrict__ input,
    const float* const __restrict__ range_list,
    const int64_t nrows,
    const int64_t ncols,
    std::uint8_t* const __restrict__ output,
    const bool forward) {
  int ebit;
  int bias;
  float max_pos;
  if (forward) {
    ebit = 4;
    bias = 15;
    max_pos = 0.9375;
  } else {
    ebit = 5;
    bias = 31;
    max_pos = 0.875;
  }

  const int64_t ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int64_t output_columns = ncols_aligned + 2 * sizeof(float);

  int64_t row = blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t col = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t row_incre = blockDim.y * gridDim.y;
  for (/*row*/; row < nrows; row += row_incre) {
    if (col < ncols) {
      float* row_qparams = reinterpret_cast<float*>(
          output + row * output_columns + ncols_aligned);
      const float scale = row_qparams[0];
      const auto input_idx = row * ncols + col;
      uint8_t* output_addr = output + row * output_columns + col;
      // TODO: lift range_list into shared memory. However, when nrows is large,
      // it might exceed the size of shared memory.
      // output_addr[0] = lrintf((input[input_idx] - bias) * inverse_scale);
      if constexpr (std::is_same<input_t, at::BFloat16>::value) {
        output_addr[0] = float_to_hfp8(
            __bfloat162float(input[input_idx]) * scale, ebit, bias, max_pos);
      } else if constexpr (std::is_same<input_t, at::Half>::value) {
        output_addr[0] = float_to_hfp8(
            __half2float(input[input_idx]) * scale, ebit, bias, max_pos);
      } else {
        output_addr[0] =
            float_to_hfp8(input[input_idx] * scale, ebit, bias, max_pos);
      }
    }
  }
}

template <typename output_t>
__global__ inline void _FP8rowwise_to_float_cuda_kernel(
    const std::uint8_t* const __restrict__ input,
    const int nrows,
    const int ncols,
    output_t* const __restrict__ output,
    const bool forward) {
  const int output_columns = ncols - 2 * sizeof(float);
  const int ebit = forward ? 4 : 5;
  const int bias = forward ? 15 : 31;

  int row = (int)blockIdx.y * blockDim.y + threadIdx.y;
  const int col = (int)blockIdx.x * blockDim.x + threadIdx.x;
  const int row_incre = blockDim.y * gridDim.y;
  for (/*row*/; row < nrows; row += row_incre) {
    if (col < output_columns) {
      const std::uint8_t* input_row = input + row * ncols;
      const float* input_row_scale_bias =
          reinterpret_cast<const float*>(input_row + output_columns);
      output_t* output_row = output + row * output_columns;

      const float output_ =
          hfp8_to_float(input_row[col], ebit, bias) / input_row_scale_bias[0];

      if constexpr (std::is_same<output_t, at::BFloat16>::value) {
        *reinterpret_cast<__hip_bfloat16*>(&output_row[col]) =
            __float2bfloat16(output_);
      } else if constexpr (std::is_same<output_t, at::Half>::value) {
        output_row[col] = __half2float(output_);
      } else {
        output_row[col] = output_;
      }
    }
  }
}

} // namespace

// revising INT8 rowwise template for FP8 rowwise quantization
template <typename input_t>
Tensor _float_to_FP8rowwise_gpu_t(const Tensor& input, const bool forward) {
  TENSOR_ON_CUDA_GPU(input);
  TORCH_CHECK(input.is_contiguous(), "input must be contiguous");

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(input.get_device());

  const auto input_sizes = input.sizes();
  const auto last_dim = input_sizes.size() - 1;
  const int nrows = c10::size_to_dim_(last_dim, input_sizes);
  const int ncols = input_sizes[last_dim];
  const int ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int output_columns = ncols_aligned + 2 * sizeof(float);

  // Global memory instructions support reading or writing words of size equal
  // to 1, 2, 4, 8, or 16 bytes. Any access (via a variable or a pointer) to
  // data residing in global memory compiles to a single global memory
  // instruction if and only if the size of the data type is 1, 2, 4, 8, or 16
  // bytes and the data is naturally aligned (i.e., its address is a multiple of
  // that size).
  auto output_dims = input_sizes.vec();
  output_dims[last_dim] = output_columns;
  auto output = at::empty(
      output_dims, // 4 = sizeof(float)
      input.options().dtype(at::kByte));

  if (nrows == 0 || ncols == 0) {
    return output;
  }

  constexpr int threads_per_block = 256;
  const auto num_blocks = cuda_calc_xblock_count(nrows, threads_per_block);
  // think unsigned as we use 0, 255

  if (nrows <= 20) {
    FBGEMM_DISPATCH_FLOAT_HALF_AND_BFLOAT16(
        input.scalar_type(), "_float_to_FP8rowwise_cuda_kernel", [&] {
          _float_to_FP8rowwise_cuda_kernel<scalar_t>
              <<<num_blocks,
                 threads_per_block,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  input.data_ptr<scalar_t>(),
                  nrows,
                  ncols,
                  output.data_ptr<std::uint8_t>(),
                  forward);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
  } else {
    // range_tensor is used to store the range for each embedding row.
    // We save max_pos/max_val(rowwise) as row scale to quantize
    // unlike INT8, FP8 does not have zero shift
    // This will guarantee the numerical match but bring some perf
    // regression.
    auto range_tensor = at::empty({nrows}, input.options().dtype(at::kFloat));

    {
      // we need a blockDim.x that is a power of 2 no larger than the warp size
      // of 32

      int blockDim_x = 1;
      if (ncols > 16) {
        // max warp size
        blockDim_x = 32;
      } else {
        while (blockDim_x < ncols) {
          blockDim_x <<= 1;
        }
      }

      const int rows_per_block = threads_per_block / blockDim_x;
      const auto num_blocks_warp =
          cuda_calc_xblock_count(nrows, rows_per_block);

      FBGEMM_DISPATCH_FLOAT_HALF_AND_BFLOAT16(
          input.scalar_type(), "_get_FP8_qparam_cuda_kernel", [&] {
            _get_FP8_qparam_cuda_kernel<scalar_t>
                <<<num_blocks_warp,
                   dim3(blockDim_x, rows_per_block),
                   0,
                   at::cuda::getCurrentCUDAStream()>>>(
                    input.data_ptr<scalar_t>(),
                    nrows,
                    ncols,
                    output.data_ptr<std::uint8_t>(),
                    range_tensor.data_ptr<float>(),
                    forward);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
    }

    {
      const int blockDim_x = std::min(ncols, threads_per_block);
      dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);
      const auto gridDim_x = cuda_calc_xblock_count(ncols, blockDim.x);
      const auto gridDim_y = cuda_calc_block_count(nrows, blockDim.y);
      dim3 gridDim(gridDim_x, gridDim_y);

      FBGEMM_DISPATCH_FLOAT_HALF_AND_BFLOAT16(
          input.scalar_type(), "_compute_FP8_quantize_cuda_kernel", [&] {
            _compute_FP8_quantize_cuda_kernel<scalar_t>
                <<<gridDim, blockDim, 0, at::cuda::getCurrentCUDAStream()>>>(
                    input.data_ptr<scalar_t>(),
                    range_tensor.data_ptr<float>(),
                    nrows,
                    ncols,
                    output.data_ptr<std::uint8_t>(),
                    forward);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
    }
  }

  return output;
}

///@ingroup quantize-data-cuda
DLL_PUBLIC Tensor
_float_to_FP8rowwise_gpu(const Tensor& input, const bool forward) {
  auto input_type = input.dtype();
  if (input_type == at::kHalf) {
    return _float_to_FP8rowwise_gpu_t<half>(input, forward);
  } else if (input_type == at::kBFloat16) {
    return _float_to_FP8rowwise_gpu_t<__hip_bfloat16>(input, forward);
  } else {
    return _float_to_FP8rowwise_gpu_t<float>(input, forward);
  }
}

template <typename output_t>
Tensor _FP8rowwise_to_float_gpu_t(const Tensor& input, bool forward) {
  TENSOR_ON_CUDA_GPU(input);
  TORCH_CHECK(input.is_contiguous(), "input must be contiguous");

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(input.get_device());

  const auto input_sizes = input.sizes();
  const auto last_dim = input_sizes.size() - 1;
  const int nrows = c10::size_to_dim_(last_dim, input_sizes);
  const int ncols = input_sizes[last_dim];
  const int ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int output_columns = ncols_aligned - 2 * sizeof(float);

  // Global memory instructions support reading or writing words of size equal
  // to 1, 2, 4, 8, or 16 bytes. Any access (via a variable or a pointer) to
  // data residing in global memory compiles to a single global memory
  // instruction if and only if the size of the data type is 1, 2, 4, 8, or 16
  // bytes and the data is naturally aligned (i.e., its address is a multiple of
  // that size).
  auto output_dims = input_sizes.vec();
  output_dims[last_dim] = output_columns;
  Tensor output;
  if constexpr (std::is_same_v<output_t, float>) {
    output = at::empty(
        output_dims, // 4 = sizeof(float)
        input.options().dtype(at::kFloat));
  } else if constexpr (std::is_same_v<output_t, half>) { // T = at::Half
    output = at::empty(
        output_dims, // 4 = sizeof(float)
        input.options().dtype(at::kHalf));
  } else if constexpr (std::is_same_v<
                           output_t,
                           __hip_bfloat16>) { // T = at::BFloat16
    output = at::empty(
        output_dims, // 4 = sizeof(float)
        input.options().dtype(at::kBFloat16));
  } else {
    TORCH_CHECK(false);
  }

  if (nrows == 0 || output_columns == 0) {
    return output;
  }

  constexpr int threads_per_block = 256;

  const int blockDim_x = std::min(threads_per_block, output_columns);
  const dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);

  const auto gridDim_x = cuda_calc_xblock_count(output_columns, blockDim.x);
  const auto gridDim_y = cuda_calc_block_count(nrows, blockDim.y);
  const dim3 gridDim(gridDim_x, gridDim_y);

  FBGEMM_DISPATCH_FLOAT_HALF_AND_BFLOAT16(
      output.scalar_type(), "FP8rowwise_to_float_cuda_kernel", [&] {
        _FP8rowwise_to_float_cuda_kernel<scalar_t>
            <<<gridDim, blockDim, 0, at::cuda::getCurrentCUDAStream()>>>(
                input.data_ptr<std::uint8_t>(),
                nrows,
                ncols,
                output.data_ptr<scalar_t>(),
                forward);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

  return output;
}

DLL_PUBLIC at::Tensor _FP8rowwise_to_float_gpu(
    const at::Tensor& input,
    bool forward,
    const int64_t output_dtype) {
  SparseType output_sparse_dtype = static_cast<SparseType>(output_dtype);
  Tensor output;
  switch (output_sparse_dtype) {
    case SparseType::FP32:
      output = _FP8rowwise_to_float_gpu_t<float>(input, forward);
      break;
    case SparseType::FP16:
      output = _FP8rowwise_to_float_gpu_t<half>(input, forward);
      break;
    case SparseType::BF16:
      output = _FP8rowwise_to_float_gpu_t<__hip_bfloat16>(input, forward);
      break;
    default:
      TORCH_CHECK(false);
  }
  return output;
}

} // namespace fbgemm_gpu
