/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

// clang-format off
#include "fbgemm_gpu/cub_namespace_prefix.cuh"
#include "cub/device/device_scan.cuh"
#include "fbgemm_gpu/cub_namespace_postfix.cuh"
// clang-format on

#include "fbgemm_gpu/layout_transform_ops.cuh"
#include "fbgemm_gpu/sparse_ops.h"
#include "fbgemm_gpu/sparse_ops_utils.h"

#include <ATen/ATen.h>
#include <ATen/core/op_registration/op_registration.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/library.h>

#include "ATen/Parallel.h"

using Tensor = at::Tensor;

///@defgroup layout-transform-cuda Layout Transformation CUDA Operators
///

namespace fbgemm_gpu {

///@ingroup layout-transform-cuda
Tensor recat_embedding_grad_output_cuda(
    Tensor grad_output, // [B_local][T_global][D]
    const std::vector<int64_t>& num_features_per_rank) {
  TENSOR_ON_CUDA_GPU(grad_output);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(grad_output.get_device());

  TORCH_CHECK(grad_output.is_contiguous());
  const auto B_local = grad_output.size(0);
  const auto T_global = grad_output.size(1);
  const auto D = grad_output.size(2);

  Tensor sharded_grad_output =
      at::empty({grad_output.numel()}, grad_output.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "recat_embedding_gradients", [&] {
        const auto go = grad_output.accessor<scalar_t, 3>();
        auto sgo = sharded_grad_output.accessor<scalar_t, 1>();
        int64_t feature_offset = 0;
        int64_t sgo_offset = 0;
        for (auto num_features : num_features_per_rank) {
          if (num_features == 0) {
            continue;
          }
          AT_CUDA_CHECK(hipMemcpy2DAsync(
              &sgo[sgo_offset],
              num_features * D * sizeof(scalar_t),
              &go[0][feature_offset][0],
              T_global * D * sizeof(scalar_t),
              num_features * D * sizeof(scalar_t),
              B_local,
              hipMemcpyDeviceToDevice,
              at::cuda::getCurrentCUDAStream()));
          feature_offset += num_features;
          sgo_offset += B_local * num_features * D;
        }
        TORCH_CHECK(sgo_offset == grad_output.numel());
        TORCH_CHECK(feature_offset == T_global);
      });
  return sharded_grad_output;
}

///@ingroup layout-transform-cuda
Tensor recat_embedding_grad_output_mixed_D_cuda(
    const Tensor& grad_output, // [B_local][Sum_T_global(D)]
    const std::vector<int64_t>& dim_sum_per_rank) {
  TENSOR_ON_CUDA_GPU(grad_output);
  TORCH_CHECK(grad_output.is_contiguous());

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(grad_output.get_device());

  const auto B_local = grad_output.size(0);
  const auto global_dim_sum = at::sum_integers(dim_sum_per_rank);

  Tensor sharded_grad_output =
      at::empty({grad_output.numel()}, grad_output.options());

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "recat_embedding_gradients", [&] {
        const auto go = grad_output.accessor<scalar_t, 2>();
        auto sgo = sharded_grad_output.accessor<scalar_t, 1>();
        int64_t sgo_offset = 0;
        int64_t accum_dim_sum = 0;
        for (auto dim_sum : dim_sum_per_rank) {
          if (dim_sum == 0) {
            continue;
          }
          AT_CUDA_CHECK(hipMemcpy2DAsync(
              &sgo[sgo_offset],
              dim_sum * sizeof(scalar_t),
              &go[0][accum_dim_sum],
              global_dim_sum * sizeof(scalar_t),
              dim_sum * sizeof(scalar_t),
              B_local,
              hipMemcpyDeviceToDevice,
              at::cuda::getCurrentCUDAStream()));
          sgo_offset += B_local * dim_sum;
          accum_dim_sum += dim_sum;
        }
        TORCH_CHECK(sgo_offset == grad_output.numel());
        TORCH_CHECK(accum_dim_sum == global_dim_sum);
      });

  return sharded_grad_output;
}

///@ingroup layout-transform-cuda
Tensor recat_embedding_grad_output_mixed_D_batch_cuda(
    const Tensor& grad_output, // [B_local][Sum_T_global(D)]
    const Tensor& dim_sum_per_rank,
    const Tensor& cumsum_dim_sum_per_rank) {
  TENSOR_ON_CUDA_GPU(grad_output);
  TENSOR_ON_CUDA_GPU(dim_sum_per_rank);
  TENSOR_ON_CUDA_GPU(cumsum_dim_sum_per_rank);
  TORCH_CHECK(grad_output.is_contiguous());

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(grad_output.get_device());

  const auto B_local = grad_output.size(0);
  Tensor sharded_grad_output =
      at::empty({grad_output.numel()}, grad_output.options());
  const auto dim_num = dim_sum_per_rank.size(0);
  const auto dim_sum = grad_output.size(1);

  const dim3 threads(
      fbgemm_gpu::kWarpSize, fbgemm_gpu::kMaxThreads / fbgemm_gpu::kWarpSize);
  const dim3 blocks(fbgemm_gpu::div_round_up(
      (B_local * dim_num), fbgemm_gpu::kMaxThreads / fbgemm_gpu::kWarpSize));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "recat_embedding_gradients", [&] {
        recat_copy_async_kernel<scalar_t>
            <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
                dim_sum_per_rank.data_ptr<int64_t>(),
                cumsum_dim_sum_per_rank.data_ptr<int64_t>(),
                grad_output.data_ptr<scalar_t>(),
                sharded_grad_output.data_ptr<scalar_t>(),
                dim_num,
                B_local,
                dim_sum);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

  return sharded_grad_output;
}

} // namespace fbgemm_gpu
